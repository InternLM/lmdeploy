#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2020-2023, NVIDIA CORPORATION.  All rights reserved.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "src/fastertransformer/kernels/decoder_masked_multihead_attention.h"
#include "src/fastertransformer/kernels/decoder_masked_multihead_attention_utils.h"
#include "src/fastertransformer/utils/cuda_bf16_wrapper.h"
#include "src/fastertransformer/utils/cuda_utils.h"
#include <assert.h>
#include <float.h>
#include <type_traits>

#include "decoder_masked_multihead_attention_template.cuh"

////////////////////////////////////////////////////////////////////////////////////////////////////

#define MMHA_LAUNCH_KERNEL(T, Dh, Dh_MAX, THDS_PER_KEY, THDS_PER_VALUE, THDS_PER_BLOCK, HAS_BEAMS, stream)             \
    size_t smem_sz = mmha::smem_size_in_bytes<T>(params, THDS_PER_VALUE, THDS_PER_BLOCK);                              \
    dim3   grid(params.num_heads, params.batch_size);                                                                  \
    mmha::masked_multihead_attention_kernel<T, Dh, Dh_MAX, THDS_PER_KEY, THDS_PER_VALUE, THDS_PER_BLOCK, HAS_BEAMS>    \
        <<<grid, THDS_PER_BLOCK, smem_sz, stream>>>(params)

////////////////////////////////////////////////////////////////////////////////////////////////////

// !!! Specialize the launcher for Cross attention
template<typename T, int Dh, int Dh_MAX, typename KERNEL_PARAMS_TYPE>
void mmha_launch_kernel(const KERNEL_PARAMS_TYPE& params, const hipStream_t& stream)
{
    constexpr int THREADS_PER_VALUE = threads_per_value_t<T, Dh_MAX>::value;
    // constexpr bool DO_CROSS_ATTENTION = std::is_same<KERNEL_PARAMS_TYPE, Cross_multihead_attention_params<T>>::value;
    int tlength = params.timestep;

    FT_CHECK(params.cache_indir == nullptr);

    if (tlength < 32) {
        MMHA_LAUNCH_KERNEL(T, Dh, Dh_MAX, 4, THREADS_PER_VALUE, 64, false, stream);
    }
    else if (tlength < 2048) {
        MMHA_LAUNCH_KERNEL(T, Dh, Dh_MAX, 2, THREADS_PER_VALUE, 128, false, stream);
    }
    else {
        MMHA_LAUNCH_KERNEL(T, Dh, Dh_MAX, 1, THREADS_PER_VALUE, 256, false, stream);
    }
}

////////////////////////////////////////////////////////////////////////////////////////////////////

template void mmha_launch_kernel<float, 128, 128, Masked_multihead_attention_params<float>>(
    const Masked_multihead_attention_params<float>& params, const hipStream_t& stream);
template void mmha_launch_kernel<uint16_t, 128, 128, Masked_multihead_attention_params<uint16_t>>(
    const Masked_multihead_attention_params<uint16_t>& params, const hipStream_t& stream);
#ifdef ENABLE_BF16
template void mmha_launch_kernel<__hip_bfloat16, 128, 128, Masked_multihead_attention_params<__hip_bfloat16>>(
    const Masked_multihead_attention_params<__hip_bfloat16>& params, const hipStream_t& stream);
#endif
#ifdef ENABLE_FP8
template void mmha_launch_kernel<__hip_fp8_e4m3_fnuz, 128, 128, Masked_multihead_attention_params<__hip_fp8_e4m3_fnuz>>(
    const Masked_multihead_attention_params<__hip_fp8_e4m3_fnuz>& params, const hipStream_t& stream);
#endif

#undef MMHA_LAUNCH_KERNEL
