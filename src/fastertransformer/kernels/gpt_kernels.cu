#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2020-2023, NVIDIA CORPORATION.  All rights reserved.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "src/fastertransformer/utils/cuda_fp8_utils.h"
#ifndef CUDART_VERSION
#error CUDART_VERSION Undefined!
#elif (CUDART_VERSION >= 11050)
#include <hipcub/hipcub.hpp>
#else
#include "3rdparty/hipcub/hipcub.hpp"
#endif
#include "src/fastertransformer/kernels/gpt_kernels.h"
#include "src/fastertransformer/utils/memory_utils.h"

namespace fastertransformer {

// PROMPT_SRC: 0 --> no prompts, 1 --> from loaded prompts, 2 --> from request prompts
template<typename T, bool OUTPUT_ID, int PROMPT_SRC>
__global__ void start_id_embedding_position_lookups_kernel(T*                    from_tensor,
                                                           int*                  output_ids,
                                                           const T*              embedding_table,
                                                           const T*              pos_table,
                                                           pPromptTuningParam<T> prompt_param,
                                                           const int*            input_ids,
                                                           const int             start_step,
                                                           const int             length,
                                                           const int             max_length,
                                                           const int             batch_size,
                                                           const int64_t         hidden_units)
{
    for (int index = blockIdx.x * blockDim.x + threadIdx.x; index < batch_size * length * hidden_units;
         index += blockDim.x * gridDim.x) {
        // transpose the input_ids [batch, length] (part of [batch, max_length]) to output_ids [length, batch]
        if (OUTPUT_ID && index < batch_size * max_length) {
            // for p/prompt_tuning (have prompt templates like [input1, prompt1, input2, prompt2])
            // we have to process it to like [input1, input2, prompt1, prompt2], and then remove the prompts during post
            // processing
            if (PROMPT_SRC > 0) {
                if (index < batch_size) {
                    int no_prompt_output_seq_id = 0;
#pragma unroll 1
                    for (int seq_id = 0; seq_id < max_length; seq_id++) {
                        int current_input_id = input_ids[index * max_length + seq_id];
                        if (current_input_id < prompt_param.p_prompt_tuning_id_start) {
                            output_ids[no_prompt_output_seq_id * batch_size + index] = current_input_id;
                            no_prompt_output_seq_id++;
                        }
                    }
                }
            }
            else {
                const int seq_id   = index % max_length;
                const int batch_id = index / max_length;
                if (seq_id < length) {
                    output_ids[seq_id * batch_size + batch_id] = input_ids[index];
                }
            }
        }

        // embedding lookup from word ids [batch, length] (part of [batch, max_length]) and [vocab, hidden] to generate
        // embedding [batch, length, hidden]
        const int word_index      = index / hidden_units;
        const int word_index_row  = word_index / length;  // batch_id
        const int word_index_col  = word_index % length;
        const int real_word_index = word_index_row * max_length + word_index_col;
        const int step            = start_step + word_index % length;
        const int col_index       = index % hidden_units;
        const int input_id        = input_ids == nullptr ? real_word_index : input_ids[real_word_index];
        const int prompt_id       = input_id - prompt_param.p_prompt_tuning_id_start;
        T         embedding       = (T)0.0f;
        if (PROMPT_SRC > 0 && prompt_id >= 0) {
            if (PROMPT_SRC == 1) {
                // from loaded prompt embedding tables
                embedding =
                    prompt_param.p_prompt_tuning_batch_weights[word_index_row][prompt_id * hidden_units + col_index];
            }
            else {
                // from request prompt embedding
                embedding =
                    prompt_param
                        .request_prompt_embedding[word_index_row * prompt_param.request_prompt_max_length * hidden_units
                                                  + prompt_id * hidden_units + col_index];
            }
        }
        else {
            embedding = embedding_table[input_id * hidden_units + col_index];
        }
        T pos_embed        = pos_table == nullptr ? (T)0.f : pos_table[(step - 1) * hidden_units + col_index];
        from_tensor[index] = embedding + pos_embed;
    }
}

#define WORD_POS_EMBEDDING_LOOPUP_KERNEL(OUTPUT_ID, PROMPT_SRC)                                                        \
    start_id_embedding_position_lookups_kernel<T, OUTPUT_ID, PROMPT_SRC><<<grid, block, 0, stream>>>(from_tensor,      \
                                                                                                     output_ids,       \
                                                                                                     embedding_table,  \
                                                                                                     pos_table,        \
                                                                                                     prompt_param,     \
                                                                                                     input_ids,        \
                                                                                                     start_step,       \
                                                                                                     length,           \
                                                                                                     max_length,       \
                                                                                                     batch_size,       \
                                                                                                     hidden_units);

template<typename T>
void invokeInputIdsEmbeddingLookupPosEncoding(T*                    from_tensor,
                                              int*                  output_ids,
                                              const T*              embedding_table,  // can also be inputs_embeds
                                              const T*              pos_table,
                                              pPromptTuningParam<T> prompt_param,
                                              const int*            input_ids,
                                              const int             start_step,
                                              const int             length,
                                              const int             max_length,
                                              const int             batch_size,
                                              const int             hidden_units,
                                              hipStream_t          stream)
{
    dim3       grid(min(batch_size * length, 65536));
    dim3       block(min(hidden_units, 512));
    const bool has_output_ids = output_ids != nullptr;
    FT_CHECK(!(has_output_ids && input_ids == nullptr));

    if (has_output_ids) {
        if (prompt_param.use_request_p_prompt_embedding) {
            WORD_POS_EMBEDDING_LOOPUP_KERNEL(true, 2);
        }
        else if (prompt_param.p_prompt_tuning_batch_weights != nullptr) {
            WORD_POS_EMBEDDING_LOOPUP_KERNEL(true, 1);
        }
        else {
            WORD_POS_EMBEDDING_LOOPUP_KERNEL(true, 0);
        }
    }
    else {
        if (prompt_param.use_request_p_prompt_embedding) {
            WORD_POS_EMBEDDING_LOOPUP_KERNEL(false, 2);
        }
        else if (prompt_param.p_prompt_tuning_batch_weights != nullptr) {
            WORD_POS_EMBEDDING_LOOPUP_KERNEL(false, 1);
        }
        else {
            WORD_POS_EMBEDDING_LOOPUP_KERNEL(false, 0);
        }
    }
}

template void invokeInputIdsEmbeddingLookupPosEncoding(float*                    from_tensor,
                                                       int*                      output_ids,
                                                       const float*              embedding_table,
                                                       const float*              pos_table,
                                                       pPromptTuningParam<float> prompt_param,
                                                       const int*                input_ids,
                                                       const int                 start_step,
                                                       const int                 length,
                                                       const int                 max_length,
                                                       const int                 batch_size,
                                                       const int                 hidden_units,
                                                       hipStream_t              stream);

template void invokeInputIdsEmbeddingLookupPosEncoding(half*                    from_tensor,
                                                       int*                     output_ids,
                                                       const half*              embedding_table,
                                                       const half*              pos_table,
                                                       pPromptTuningParam<half> prompt_param,
                                                       const int*               input_ids,
                                                       const int                start_step,
                                                       const int                length,
                                                       const int                max_length,
                                                       const int                batch_size,
                                                       const int                hidden_units,
                                                       hipStream_t             stream);

#ifdef ENABLE_BF16
template void invokeInputIdsEmbeddingLookupPosEncoding(__hip_bfloat16*                    from_tensor,
                                                       int*                              output_ids,
                                                       const __hip_bfloat16*              embedding_table,
                                                       const __hip_bfloat16*              pos_table,
                                                       pPromptTuningParam<__hip_bfloat16> prompt_param,
                                                       const int*                        input_ids,
                                                       const int                         start_step,
                                                       const int                         length,
                                                       const int                         max_length,
                                                       const int                         batch_size,
                                                       const int                         hidden_units,
                                                       hipStream_t                      stream);
#endif

template<typename T>
__global__ void inputIdsEmbeddingLookupPosEncodingSoftPrompt(inputIdsEmbeddingLookupPosEncodingSoftPromptParam<T> param)
{
    // 1. Copy the input ids to output ids and transpose output ids to [seq_len, batch_size, beam_width].
    // 2. Embedding lookup by input ids and concat with soft prompt. The axis of concatenation is on axis of seq_len.

    // Assume batch size is 2 and prompts are [[t1, t2], [t3], [t4, t5]], input_ids are [[s1, s2], [s3], [s4]]
    // then the order of output_ids is
    // [ [?, ?, s1, s2]
    //   [?, s3, padding, padding]
    //   [?, ?, s4, padding] ]
    // and the order of embedding is
    // [ [t1, t2, s1, s2]
    //   [t3, s3, padding, padding]
    //   [t4, t5, s4, padding] ]
    // where "?" means undefined values and we should attach it.

    for (int index = blockIdx.x * blockDim.x + threadIdx.x;
         index < param.batch_size * param.beam_width * (param.max_prefix_soft_prompt_length + param.max_input_length)
                     * param.hidden_units;
         index += blockDim.x * gridDim.x) {
        // transpose the input_ids [batch, length] (part of [batch, beam, max_input_length]) to
        // output_ids [length, batch, beam].
        // ouptut_ids need to add padding in the beginning for soft prompting.

        if (index < param.batch_size * param.beam_width * param.max_input_length) {
            int       tmp_index = index;
            const int seq_id    = tmp_index % param.max_input_length;
            tmp_index           = (tmp_index - seq_id) / param.max_input_length;
            const int beam_id   = tmp_index % param.beam_width;
            tmp_index           = (tmp_index - beam_id) / param.beam_width;
            const int batch_id  = tmp_index % param.batch_size;
            if (seq_id < param.max_input_length) {
                param.output_ids[(param.prefix_soft_prompt_lengths[batch_id] + seq_id) * param.batch_size
                                     * param.beam_width
                                 + batch_id * param.beam_width + beam_id] = param.input_ids[index];
            }
        }

        // embedding lookup from word ids [batch, beam, length] (part of [batch, beam, max_input_length]), [vocab,
        // hidden] and [batch, max_prefix_soft_prompt_length, hidden] to generate embedding [batch, beam, length +
        // max_prefix_soft_prompt_length, hidden]
        int       tmp_index = index;
        const int hidden_id = tmp_index % param.hidden_units;
        tmp_index           = (tmp_index - hidden_id) / param.hidden_units;
        const int seq_id    = tmp_index % (param.max_prefix_soft_prompt_length + param.max_input_length);
        tmp_index           = (tmp_index - seq_id) / (param.max_prefix_soft_prompt_length + param.max_input_length);
        const int beam_id   = tmp_index % param.beam_width;
        tmp_index           = (tmp_index - beam_id) / param.beam_width;
        const int batch_id  = tmp_index % param.batch_size;
        const int64_t hidden_units = param.hidden_units;
        T         embedding =
            (seq_id < param.prefix_soft_prompt_lengths[batch_id]) ?
                        (T)param.prefix_soft_prompt_embedding[batch_id * param.max_prefix_soft_prompt_length * hidden_units
                                                      + seq_id * hidden_units + hidden_id] :
                            param.embedding_table[param.input_ids[batch_id * param.beam_width * param.max_input_length
                                                      + beam_id * param.max_input_length
                                                      + (seq_id - param.prefix_soft_prompt_lengths[batch_id])]
                                          * hidden_units
                                      + hidden_id];

        T pos_embed              = param.pos_table == nullptr ?
                                       (T)0.0f :
                                       param.pos_table[(param.start_step + seq_id - 1) * hidden_units + hidden_id];
        param.from_tensor[index] = embedding + pos_embed;

        if (seq_id == 0 && hidden_id == 0) {
            param.input_lengths[batch_id * param.beam_width + beam_id] += param.prefix_soft_prompt_lengths[batch_id];
        }
    }
}

template<typename T>
void invokeInputIdsEmbeddingLookupPosEncodingSoftPrompt(inputIdsEmbeddingLookupPosEncodingSoftPromptParam<T> param)
{
    dim3 grid(min(param.batch_size * param.beam_width * (param.max_input_length + param.max_prefix_soft_prompt_length),
                  65536));
    dim3 block(min(param.hidden_units, 512));
    inputIdsEmbeddingLookupPosEncodingSoftPrompt<T><<<grid, block, 0, param.stream>>>(param);
}

template void
invokeInputIdsEmbeddingLookupPosEncodingSoftPrompt(inputIdsEmbeddingLookupPosEncodingSoftPromptParam<float> param);

template void
invokeInputIdsEmbeddingLookupPosEncodingSoftPrompt(inputIdsEmbeddingLookupPosEncodingSoftPromptParam<half> param);

#ifdef ENABLE_BF16
template void invokeInputIdsEmbeddingLookupPosEncodingSoftPrompt(
    inputIdsEmbeddingLookupPosEncodingSoftPromptParam<__hip_bfloat16> param);
#endif

// TODO Add half2 implementation
template<typename T>
__global__ void transposeAxis01(T* out, T* in, const int dim0, const int dim1, const int dim2)
{
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    if (index < dim0 * dim1 * dim2) {
        const int input_dim2_index = index % dim2;
        index                      = (index - input_dim2_index) / dim2;
        const int input_dim1_index = index % dim1;
        index                      = (index - input_dim1_index) / dim1;
        const int input_dim0_index = index % dim0;

        out[input_dim1_index * dim0 * dim2 + input_dim0_index * dim2 + input_dim2_index] =
            in[input_dim0_index * dim1 * dim2 + input_dim1_index * dim2 + input_dim2_index];
    }
}

template<typename T>
void invokeTransposeAxis01(T* out, T* in, const int dim0, const int dim1, const int dim2, hipStream_t stream)
{
    dim3 block(512);
    dim3 grid((int)(ceil(dim0 * dim1 * dim2 / 512.)));
    transposeAxis01<<<grid, block, 0, stream>>>(out, in, dim0, dim1, dim2);
}

template void
invokeTransposeAxis01(float* out, float* in, const int dim0, const int dim1, const int dim2, hipStream_t stream);

template void
invokeTransposeAxis01(half* out, half* in, const int dim0, const int dim1, const int dim2, hipStream_t stream);

template void
invokeTransposeAxis01(int* out, int* in, const int dim0, const int dim1, const int dim2, hipStream_t stream);

template<typename T>
__global__ void transposeAxis01(T* out, T* in, const int* in_skipping_dim1, const int dim0, const int dim1)
{
    // out: [dim1, dim0]
    // in: [dim0, dim1]
    // in_skipping_dim1: [dim1]

    int index = threadIdx.x + blockIdx.x * blockDim.x;
    if (index < dim0 * dim1) {
        const int input_dim1_index = index % dim1;
        index                      = (index - input_dim1_index) / dim1;
        const int input_dim0_index = index % dim0;
        const int in_offset        = in_skipping_dim1 == nullptr ? 0 : in_skipping_dim1[input_dim1_index] * dim1;

        out[input_dim1_index * dim0 + input_dim0_index] = in[in_offset + input_dim0_index * dim1 + input_dim1_index];
    }
}

template<typename T>
void invokeTransposeAxis01(
    T* out, T* in, const int* in_skipping_dim1, const int dim0, const int dim1, hipStream_t stream)
{
    dim3 block(512);
    dim3 grid((int)(ceil(dim0 * dim1 / 512.)));
    transposeAxis01<<<grid, block, 0, stream>>>(out, in, in_skipping_dim1, dim0, dim1);
}

template void invokeTransposeAxis01(
    int* out, int* in, const int* in_skipping_dim1, const int dim0, const int dim1, hipStream_t stream);

template<typename T, bool PREFIX_PROMPT>
__global__ void buildDecoderAttentionMaskKernel(T*         attention_mask,
                                                const int* sequence_lengths,
                                                const int* prefix_prompt_lengths,
                                                const int  max_seq_len,
                                                const int  max_prompt_length)
{
    // sequence_lengths: [batch_size]
    // attention_mask: [batch_size, 1, max_seq_len, max_seq_len + max_prompt_length]
    const int max_prompt_seq_length = max_seq_len + max_prompt_length;
    const int mask_size_per_seq     = max_seq_len * max_prompt_seq_length;
    attention_mask += blockIdx.x * mask_size_per_seq;
    const int seq_length    = sequence_lengths[blockIdx.x];
    const int prompt_length = PREFIX_PROMPT ? prefix_prompt_lengths[blockIdx.x] : 0;
    for (int i = threadIdx.x; i < mask_size_per_seq; i += blockDim.x) {
        int row_id = i / max_prompt_seq_length;
        int col_id = i % max_prompt_seq_length;
        if (row_id < seq_length && col_id <= (row_id + prompt_length)) {
            attention_mask[i] = (T)(1.0f);
        }
        else {
            attention_mask[i] = (T)(0.0f);
        }
    }
}

template<typename T>
void invokeBuildDecoderAttentionMask(T*           attention_mask,
                                     const int*   sequence_lengths,
                                     const int*   prefix_prompt_lengths,
                                     const int    batch_size,
                                     const int    max_seq_len,
                                     const int    max_prompt_length,
                                     hipStream_t stream)
{
    if (max_prompt_length == 0) {
        buildDecoderAttentionMaskKernel<T, false><<<batch_size, 256, 0, stream>>>(
            attention_mask, sequence_lengths, prefix_prompt_lengths, max_seq_len, max_prompt_length);
    }
    else {
        buildDecoderAttentionMaskKernel<T, true><<<batch_size, 256, 0, stream>>>(
            attention_mask, sequence_lengths, prefix_prompt_lengths, max_seq_len, max_prompt_length);
    }
}

template void invokeBuildDecoderAttentionMask(float*       attention_mask,
                                              const int*   sequence_lengths,
                                              const int*   prefix_prompt_lengths,
                                              const int    batch_size,
                                              const int    max_seq_len,
                                              const int    max_prompt_length,
                                              hipStream_t stream);
template void invokeBuildDecoderAttentionMask(half*        attention_mask,
                                              const int*   sequence_lengths,
                                              const int*   prefix_prompt_lengths,
                                              const int    batch_size,
                                              const int    max_seq_len,
                                              const int    max_prompt_length,
                                              hipStream_t stream);
#ifdef ENABLE_BF16
template void invokeBuildDecoderAttentionMask(__hip_bfloat16* attention_mask,
                                              const int*     sequence_lengths,
                                              const int*     prefix_prompt_lengths,
                                              const int      batch_size,
                                              const int      max_seq_len,
                                              const int      max_prompt_length,
                                              hipStream_t   stream);
#endif
#ifdef ENABLE_FP8
template void invokeBuildDecoderAttentionMask(__hip_fp8_e4m3_fnuz* attention_mask,
                                              const int*     sequence_lengths,
                                              const int*     prefix_prompt_lengths,
                                              const int      batch_size,
                                              const int      max_seq_len,
                                              const int      max_prompt_length,
                                              hipStream_t   stream);
#endif

template<typename T>
__launch_bounds__(1024, 1) __global__ void lookupHiddenStateOfLastToken(T*         from_tensor,
                                                                        const T*   hidden_state,
                                                                        const int* input_lengths,
                                                                        const int  max_input_length,
                                                                        const int  batch_size,
                                                                        const int  hidden_units)
{
    for (int index = blockIdx.x * blockDim.x + threadIdx.x; index < batch_size * hidden_units;
         index += blockDim.x * gridDim.x) {
        const int col_index = index % hidden_units;
        const int batch_id  = index / hidden_units;
        from_tensor[index]  = hidden_state[batch_id * max_input_length * hidden_units
                                          + (input_lengths[batch_id] - 1) * hidden_units + col_index];
    }
}

template<typename T>
void invokeLookupHiddenStateOfLastToken(T*           from_tensor,
                                        const T*     hidden_state,
                                        const int*   input_lengths,
                                        const int    max_input_length,
                                        const int    batch_size,
                                        const int    hidden_units,
                                        hipStream_t stream)
{
    const int grid_size = (int)(ceil(batch_size * hidden_units / 1024.));
    dim3      grid(min(grid_size, 65536));
    dim3      block(min(hidden_units, 1024));
    lookupHiddenStateOfLastToken<T><<<grid, block, 0, stream>>>(
        from_tensor, hidden_state, input_lengths, max_input_length, batch_size, hidden_units);
}

template void invokeLookupHiddenStateOfLastToken(float*       from_tensor,
                                                 const float* hidden_state,
                                                 const int*   input_lengths,
                                                 const int    max_input_length,
                                                 const int    batch_size,
                                                 const int    hidden_units,
                                                 hipStream_t stream);

template void invokeLookupHiddenStateOfLastToken(half*        from_tensor,
                                                 const half*  hidden_state,
                                                 const int*   input_lengths,
                                                 const int    max_input_length,
                                                 const int    batch_size,
                                                 const int    hidden_units,
                                                 hipStream_t stream);

#ifdef ENABLE_BF16
template void invokeLookupHiddenStateOfLastToken(__hip_bfloat16*       from_tensor,
                                                 const __hip_bfloat16* hidden_state,
                                                 const int*           input_lengths,
                                                 const int            max_input_length,
                                                 const int            batch_size,
                                                 const int            hidden_units,
                                                 hipStream_t         stream);
#endif

template<bool PREFIX_PROMPT>
__global__ void tileGptPromptInputs(int*       tiled_input_ids,
                                    int*       tiled_input_lengths,
                                    int*       tiled_prompt_lengths,
                                    const int* input_ids,
                                    const int* input_lengths,
                                    const int* prefix_prompt_lengths,
                                    const int  max_input_length)
{
    if (threadIdx.x == 0) {
        tiled_input_lengths[blockIdx.x * gridDim.y + blockIdx.y] = input_lengths[blockIdx.x];
        if (PREFIX_PROMPT) {
            tiled_prompt_lengths[blockIdx.x * gridDim.y + blockIdx.y] = prefix_prompt_lengths[blockIdx.x];
        }
    }
    for (int index = threadIdx.x; index < max_input_length; index += blockDim.x) {
        tiled_input_ids[(blockIdx.x * gridDim.y + blockIdx.y) * max_input_length + index] =
            input_ids[blockIdx.x * max_input_length + index];
    }
}

void invokeTileGptPromptInputs(int*         tiled_input_ids,
                               int*         tiled_input_lengths,
                               int*         tiled_prompt_lengths,
                               const int*   input_ids,
                               const int*   input_lengths,
                               const int*   prefix_prompt_lengths,
                               const int    batch_size,
                               const int    beam_width,
                               const int    max_input_length,
                               hipStream_t stream)
{
    dim3 grid(batch_size, beam_width);
    dim3 block(min(1024, max_input_length));
    if (prefix_prompt_lengths != nullptr) {
        tileGptPromptInputs<true><<<grid, block, 0, stream>>>(tiled_input_ids,
                                                              tiled_input_lengths,
                                                              tiled_prompt_lengths,
                                                              input_ids,
                                                              input_lengths,
                                                              prefix_prompt_lengths,
                                                              max_input_length);
    }
    else {
        tileGptPromptInputs<false><<<grid, block, 0, stream>>>(tiled_input_ids,
                                                               tiled_input_lengths,
                                                               tiled_prompt_lengths,
                                                               input_ids,
                                                               input_lengths,
                                                               prefix_prompt_lengths,
                                                               max_input_length);
    }
}

void invokeTileGptInputs(int*         tiled_input_ids,
                         int*         tiled_input_lengths,
                         const int*   input_ids,
                         const int*   input_lengths,
                         const int    batch_size,
                         const int    beam_width,
                         const int    max_input_length,
                         hipStream_t stream)
{
    invokeTileGptPromptInputs(tiled_input_ids,
                              tiled_input_lengths,
                              nullptr,
                              input_ids,
                              input_lengths,
                              nullptr,
                              batch_size,
                              beam_width,
                              max_input_length,
                              stream);
}

void setSeqLimitLen(uint32_t* seq_len_d, Tensor seq_len, int limit_len_offset, int batch_size)
{
    std::vector<uint32_t> seq_len_h(batch_size);
    for (int i = 0; i < batch_size; i++) {
        seq_len_h[i] = seq_len.getPtr<uint32_t>()[i] + limit_len_offset;
    }
    cudaH2Dcpy(seq_len_d, seq_len_h.data(), batch_size);
}

template<int TB_SIZE>
__global__ void
find_context_dups(int* shared_contexts, const int* input_ids, const size_t batch_size, const size_t input_seq_len)
{
    /* We compare all context pairs (i, j), with i (tgt) < j (src) , to detect duplicate
     * inputs. If there's a match between i and j, we store i at the
     * j-th position of shared_context. So that we know that j can be
     * represented by i. shared_contexts is initialized like shared_contexts[i] = i
     * and when there's a match, we actually use shared_contexts[j] = min(shared_contexts[j], i)
     * so that in the end, shared_contexts effectively contains an index
     * to the match with the lowest index context.
     * Note that shared_contexts[i] <= i, a property that will be used when uncompacting
     * inputs.
     */
    typedef hipcub::BlockReduce<int, TB_SIZE>       BlockReduce;
    __shared__ typename BlockReduce::TempStorage temp_storage;
    __shared__ bool                              match;

    /* Each block is responsible for a (i, j) pair. To map the block space to
     * the i < j space, we need to convert a linear addressing to a triangle, of
     * size (batch_size * (batch_size - 1)) / 2
     * For more information, check https://en.wikipedia.org/wiki/Triangular_number
     */

    // blockIdx = [0, 1, 2, ... n(n-1)/2] -> base_index = [0, 1, 1, 2, 2, 2, 3, 3, 3, 3, ..., n - 2]
    const int base_index = floorf(0.5f * (sqrtf(1 + 8 * blockIdx.x) - 1));
    const int src_idx    = base_index + 1;  // base_index \in [1, batch_size)

    const int rev_base_index = base_index * (base_index + 1) / 2;
    const int tgt_idx        = blockIdx.x - rev_base_index;  // tgt_idx \in [0, src_idx)

    const int padded_length = TB_SIZE * ((input_seq_len + TB_SIZE - 1) / TB_SIZE);

    int sum = 0;
    for (int i = threadIdx.x; i < padded_length; i += TB_SIZE) {
        int compare =
            (i >= input_seq_len) ? 1 : input_ids[tgt_idx * input_seq_len + i] == input_ids[src_idx * input_seq_len + i];

        sum = BlockReduce(temp_storage).Sum(compare);

        if (threadIdx.x == 0) {
            match = (sum == TB_SIZE);
        }

        __syncthreads();

        if (!match) {
            break;
        }
    }

    if (threadIdx.x == 0 && match) {
        atomicMin(&shared_contexts[src_idx], tgt_idx);
    }
}

constexpr int DUPS_INDICES_BLOCK_SIZE = 128;

__global__ void generate_dups_indices(int*         batch_to_compact,
                                      int*         compact_to_batch,
                                      int*         compact_size,
                                      const int*   shared_contexts,
                                      const size_t batch_size,
                                      const size_t input_seq_len)
{
    const int padded_batchsize = blockDim.x * ((batch_size + blockDim.x - 1) / blockDim.x);

    typedef hipcub::BlockScan<int, DUPS_INDICES_BLOCK_SIZE, hipcub::BLOCK_SCAN_WARP_SCANS> BlockScan;
    __shared__ typename BlockScan::TempStorage                                       temp_storage;
    __shared__ int                                                                   scan_offset;

    int scan = 0;
    for (int batch = threadIdx.x; batch < padded_batchsize; batch += blockDim.x) {
        bool masked     = (batch >= batch_size);
        bool first_iter = batch < blockDim.x;

        int is_first_occur = masked ? 0 : shared_contexts[batch] == batch;
        BlockScan(temp_storage).ExclusiveSum(is_first_occur, scan);

        if (!masked && is_first_occur) {
            int compact_idx = scan + (first_iter ? 0 : scan_offset);
            // Context rep. writes initial index
            batch_to_compact[batch]       = compact_idx;
            compact_to_batch[compact_idx] = batch;
        }

        if (threadIdx.x == blockDim.x - 1) {
            scan_offset = scan + is_first_occur + (first_iter ? 0 : scan_offset);
        }

        __syncthreads();

        if (!masked && !is_first_occur) {
            // Fill the rest of batch_to_compact based on what rep. wrote
            const int src_idx       = batch_to_compact[shared_contexts[batch]];
            batch_to_compact[batch] = src_idx;
        }
    }

    if (threadIdx.x == 0) {
        *compact_size = scan_offset;
    }
}

__global__ void init_shared_contexts(int* shared_contexts, const size_t batch_size)
{
    const int global_idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (global_idx >= batch_size) {
        return;
    }
    shared_contexts[global_idx] = global_idx;
}

void invokeFindContextDups(int*         shared_contexts,
                           int*         batch_to_compact,
                           int*         compact_to_batch,
                           int*         compact_size,
                           const int*   input_ids,
                           const size_t batch_size,
                           const size_t input_seq_len,
                           hipStream_t stream)
{
    dim3 block{512};
    dim3 grid{((int)batch_size + block.x - 1) / block.x};
    init_shared_contexts<<<grid, block, 0, stream>>>(shared_contexts, batch_size);

    grid = dim3{(unsigned int)(batch_size * (batch_size - 1)) / 2};
    if (input_seq_len <= 128) {
        block = 128;
        find_context_dups<128><<<grid, block, 0, stream>>>(shared_contexts, input_ids, batch_size, input_seq_len);
    }
    else {
        block = 256;
        find_context_dups<256><<<grid, block, 0, stream>>>(shared_contexts, input_ids, batch_size, input_seq_len);
    }

    generate_dups_indices<<<1, DUPS_INDICES_BLOCK_SIZE, 0, stream>>>(
        batch_to_compact, compact_to_batch, compact_size, shared_contexts, batch_size, input_seq_len);
}

template<typename T>
__global__ void compact_inputs(T*         compact_input,
                               T*         compact_attention_mask,
                               int*       compact_input_lengths,
                               const T*   decoder_input,
                               const T*   decoder_mask,
                               const int* input_lengths,
                               const int* compact_idx,
                               size_t     compact_size,
                               size_t     seq_len,
                               size_t     hidden_dimension)
{
    const int global_idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (global_idx < compact_size * seq_len * hidden_dimension) {
        const int h_id     = global_idx % hidden_dimension;
        const int seq_id   = (global_idx / hidden_dimension) % seq_len;
        const int batch_id = global_idx / (hidden_dimension * seq_len);

        compact_input[global_idx] = decoder_input[(compact_idx[batch_id] * seq_len + seq_id) * hidden_dimension + h_id];
    }

    if (global_idx < compact_size * seq_len * seq_len) {
        const int seq1_id  = global_idx % seq_len;
        const int seq2_id  = (global_idx / seq_len) % seq_len;
        const int batch_id = global_idx / (seq_len * seq_len);

        compact_attention_mask[global_idx] =
            decoder_mask[(compact_idx[batch_id] * seq_len + seq2_id) * seq_len + seq1_id];
    }

    if (global_idx < compact_size) {
        compact_input_lengths[global_idx] = input_lengths[compact_idx[global_idx]];
    }
}

template<typename T>
void invokeCompactInputs(T*           compact_input,
                         T*           compact_attention_mask,
                         int*         compact_input_lengths,
                         const T*     decoder_input,
                         const T*     decoder_mask,
                         const int*   input_lengths,
                         const int*   compact_idx,
                         size_t       compact_size,
                         size_t       seq_len,
                         size_t       hidden_dimension,
                         hipStream_t stream)
{
    /* Compact relevant decoder_layer inputs based on the identical contexts.
     * For example, decoder_input is [batch_size, seq_len, H]. It's compacted
     * into compact_input [compact_size, seq_len, H] such that
     * compact_input[i, ...] = decoder_input[compact_idx[i], ...] */
    const size_t elems_n = compact_size * seq_len * max(hidden_dimension, seq_len);
    const dim3   blockDim(512);
    const dim3   gridDim((elems_n + 512 - 1) / 512);

    compact_inputs<T><<<gridDim, blockDim, 0, stream>>>(compact_input,
                                                        compact_attention_mask,
                                                        compact_input_lengths,
                                                        decoder_input,
                                                        decoder_mask,
                                                        input_lengths,
                                                        compact_idx,
                                                        compact_size,
                                                        seq_len,
                                                        hidden_dimension);
}

#define INSTANTIATE_INVOKE_COMPACT_INPUTS(T)                                                                           \
    template void invokeCompactInputs<T>(T * compact_input,                                                            \
                                         T * compact_attention_mask,                                                   \
                                         int*         compact_input_lengths,                                           \
                                         const T*     decoder_input,                                                   \
                                         const T*     decoder_mask,                                                    \
                                         const int*   input_lengths,                                                   \
                                         const int*   compact_idx,                                                     \
                                         size_t       compact_size,                                                    \
                                         size_t       seq_len,                                                         \
                                         size_t       hidden_dimension,                                                \
                                         hipStream_t stream)
INSTANTIATE_INVOKE_COMPACT_INPUTS(half);
INSTANTIATE_INVOKE_COMPACT_INPUTS(float);
#ifdef ENABLE_BF16
INSTANTIATE_INVOKE_COMPACT_INPUTS(__hip_bfloat16);
#endif
#undef INSTANTIATE_INVOKE_COMPACT_INPUTS

template<typename T>
__global__ void uncompact_outputs(T*         uncompact_buffer,
                                  const T*   compact_buffer,
                                  const int* batch_to_compact_idx,
                                  size_t     batch_size,
                                  size_t     buffer_stride)
{
    /* Uncompact a buffer IN of size [Compact, Stride] into OUT of size [Batch, Stride]
     * so that \forall i, OUT[i, :] = IN[batch_to_compact_idx[i], :]
     */
    const int global_idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (global_idx >= batch_size * buffer_stride) {
        return;
    }

    const int stride_idx = global_idx % buffer_stride;
    const int batch_idx  = global_idx / buffer_stride;

    const int src                = batch_to_compact_idx[batch_idx];
    uncompact_buffer[global_idx] = compact_buffer[src * buffer_stride + stride_idx];
}

template<typename T>
void invokeUnCompactOutputs(T*           uncompact_buffer,
                            const T*     compact_buffer,
                            const int*   batch_to_compact_idx,
                            size_t       batch_size,
                            size_t       buffer_stride,
                            hipStream_t stream)
{
    const size_t num_elems = batch_size * buffer_stride;
    const dim3   blockDim(1024);
    const dim3   gridDim((num_elems + blockDim.x - 1) / blockDim.x);

    uncompact_outputs<T><<<gridDim, blockDim, 0, stream>>>(
        uncompact_buffer, compact_buffer, batch_to_compact_idx, batch_size, buffer_stride);
}

#define INSTANTIATE_INVOKE_UNCOMPACT_OUTPUTS(T)                                                                        \
    template void invokeUnCompactOutputs(T*           uncompact_buffer,                                                \
                                         const T*     compact_buffer,                                                  \
                                         const int*   batch_to_compact_idx,                                            \
                                         size_t       batch_size,                                                      \
                                         size_t       buffer_stride,                                                   \
                                         hipStream_t stream)
INSTANTIATE_INVOKE_UNCOMPACT_OUTPUTS(half);
INSTANTIATE_INVOKE_UNCOMPACT_OUTPUTS(float);
#ifdef ENABLE_BF16
INSTANTIATE_INVOKE_UNCOMPACT_OUTPUTS(__hip_bfloat16);
#endif
#undef INSTANTIATE_INVOKE_UNCOMPACT_OUTPUTS

template<typename T>
__global__ void uncompact_caches(T*         uncompact_k_cache,
                                 T*         uncompact_v_cache,
                                 const T*   compact_k_cache,
                                 const T*   compact_v_cache,
                                 const int* batch_to_compact_idx,
                                 size_t     batch_size,
                                 size_t     num_heads,
                                 size_t     max_seq_len,
                                 size_t     seq_len,
                                 size_t     size_per_head,
                                 size_t     local_batch_size,
                                 size_t     ite)
{
    const int hidden_dimension    = num_heads * size_per_head;
    const int num_elems_per_batch = seq_len * hidden_dimension;
    const int num_elems_cache     = batch_size * num_elems_per_batch;
    const int x_size              = 16 / sizeof(T);

    for (int global_idx = blockIdx.x * blockDim.x + threadIdx.x; global_idx < 2 * num_elems_cache;
         global_idx += blockDim.x * gridDim.x) {

        const bool     handle_k  = global_idx < num_elems_cache;
        const T* const cache_src = handle_k ? compact_k_cache : compact_v_cache;
        T* const       cache_dst = handle_k ? uncompact_k_cache : uncompact_v_cache;
        const int      idx       = handle_k ? global_idx : global_idx - num_elems_cache;

        const int src_offset = idx % num_elems_per_batch;
        const int batch_idx  = idx / num_elems_per_batch;
        const int batch_src  = batch_to_compact_idx[batch_idx] - ite * local_batch_size;

        if (batch_src < 0 || batch_src >= local_batch_size) {
            continue;
        }

        int dst_offset;
        if (handle_k) {
            const int i0 = idx % (x_size * seq_len);
            const int i1 = (idx / (x_size * seq_len)) % (num_heads * size_per_head / x_size);
            dst_offset   = i1 * max_seq_len * x_size + i0;
        }
        else {
            const int i0 = idx % (size_per_head * seq_len);
            const int i1 = (idx / (size_per_head * seq_len)) % (num_heads);
            dst_offset   = i1 * max_seq_len * size_per_head + i0;
        }

        cache_dst[batch_idx * max_seq_len * hidden_dimension + dst_offset] =
            cache_src[batch_src * num_elems_per_batch + src_offset];
    }
}

template<typename T>
void invokeUnCompactCaches(T*           uncompact_k_cache,
                           T*           uncompact_v_cache,
                           const T*     compact_k_cache,
                           const T*     compact_v_cache,
                           const int*   batch_to_compact_idx,
                           size_t       batch_size,
                           size_t       num_heads,
                           size_t       max_seq_len,
                           size_t       seq_len,
                           size_t       size_per_head,
                           size_t       local_batch_size,
                           size_t       ite,
                           hipStream_t stream)
{
    const dim3 blockDim(512);
    const dim3 gridDim(1024);
    uncompact_caches<T><<<gridDim, blockDim, 0, stream>>>(uncompact_k_cache,
                                                          uncompact_v_cache,
                                                          compact_k_cache,
                                                          compact_v_cache,
                                                          batch_to_compact_idx,
                                                          batch_size,
                                                          num_heads,
                                                          max_seq_len,
                                                          seq_len,
                                                          size_per_head,
                                                          local_batch_size,
                                                          ite);
}

#define INSTANTIATE_INVOKE_UNCOMPACT_CACHES(T)                                                                         \
    template void invokeUnCompactCaches(T*           uncompact_k_cache,                                                \
                                        T*           uncompact_v_cache,                                                \
                                        const T*     compact_k_cache,                                                  \
                                        const T*     compact_v_cache,                                                  \
                                        const int*   batch_to_compact_idx,                                             \
                                        size_t       batch_size,                                                       \
                                        size_t       num_heads,                                                        \
                                        size_t       max_seq_len,                                                      \
                                        size_t       seq_len,                                                          \
                                        size_t       size_per_head,                                                    \
                                        size_t       local_batch_size,                                                 \
                                        size_t       ite,                                                              \
                                        hipStream_t stream)
INSTANTIATE_INVOKE_UNCOMPACT_CACHES(half);
INSTANTIATE_INVOKE_UNCOMPACT_CACHES(float);
#ifdef ENABLE_BF16
INSTANTIATE_INVOKE_UNCOMPACT_CACHES(__hip_bfloat16);
#endif
#undef INSTANTIATE_INVOKE_UNCOMPACT_CACHES

template<bool PREFIX_PROMPT>
__global__ void update_padding_count(int*       total_padding_count,
                                     const int* input_lengths,
                                     const int* tiled_prompt_lengths,
                                     size_t     max_input_length,
                                     size_t     max_prompt_length,
                                     size_t     batch_size,
                                     size_t     beam_width)
{
    const int gidx = blockIdx.x * blockDim.x + threadIdx.x;

    if (gidx >= batch_size * beam_width) {
        return;
    }

    const int batch_idx = gidx / beam_width;

    total_padding_count[gidx] +=
        PREFIX_PROMPT ? (max_input_length + max_prompt_length - input_lengths[batch_idx] - tiled_prompt_lengths[gidx]) :
                        (max_input_length - input_lengths[batch_idx]);
}

void invokeUpdatePaddingCount(int*         total_padding_count,
                              const int*   input_lengths,
                              const int*   tiled_prompt_lengths,
                              size_t       max_input_length,
                              size_t       max_prompt_length,
                              size_t       batch_size,
                              size_t       beam_width,
                              hipStream_t stream)
{
    dim3 blockSize(256);
    dim3 gridSize((batch_size * beam_width + blockSize.x - 1) / blockSize.x);

    if (tiled_prompt_lengths != nullptr) {
        update_padding_count<true><<<gridSize, blockSize, 0, stream>>>(total_padding_count,
                                                                       input_lengths,
                                                                       tiled_prompt_lengths,
                                                                       max_input_length,
                                                                       max_prompt_length,
                                                                       batch_size,
                                                                       beam_width);
    }
    else {
        update_padding_count<false><<<gridSize, blockSize, 0, stream>>>(total_padding_count,
                                                                        input_lengths,
                                                                        tiled_prompt_lengths,
                                                                        max_input_length,
                                                                        max_prompt_length,
                                                                        batch_size,
                                                                        beam_width);
    }
}

template<bool PREFIX_PROMPT>
__global__ void mask_padding_tokens(bool*        masked_tokens,
                                    const int*   input_lengths,
                                    const int*   tiled_prefix_prompt_lengths,
                                    const size_t memory_len,
                                    const size_t max_input_length,
                                    const size_t initial_step,
                                    size_t       beam_width)
{
    const int seq_len = PREFIX_PROMPT ?
                            (input_lengths[blockIdx.x / beam_width] + tiled_prefix_prompt_lengths[blockIdx.x]) :
                            input_lengths[blockIdx.x / beam_width];
    for (int step = initial_step + seq_len + threadIdx.x; step < initial_step + max_input_length; step += blockDim.x) {
        masked_tokens[blockIdx.x * memory_len + step % memory_len] = true;
    }
}

void invokeMaskPaddingTokens(bool*        masked_tokens,
                             const int*   input_lengths,
                             const int*   tiled_prefix_prompt_lengths,
                             const size_t memory_len,
                             const size_t max_input_length,
                             const size_t initial_step,
                             size_t       batch_size,
                             size_t       beam_width,
                             hipStream_t stream)
{
    dim3 blockSize(128);
    dim3 gridSize(batch_size * beam_width);
    if (tiled_prefix_prompt_lengths != nullptr) {
        mask_padding_tokens<true><<<gridSize, blockSize, 0, stream>>>(masked_tokens,
                                                                      input_lengths,
                                                                      tiled_prefix_prompt_lengths,
                                                                      memory_len,
                                                                      max_input_length,
                                                                      initial_step,
                                                                      beam_width);
    }
    else {
        mask_padding_tokens<false><<<gridSize, blockSize, 0, stream>>>(masked_tokens,
                                                                       input_lengths,
                                                                       tiled_prefix_prompt_lengths,
                                                                       memory_len,
                                                                       max_input_length,
                                                                       initial_step,
                                                                       beam_width);
    }
}

template<typename T>
__global__ void sum_length_dimension(
    float* out_buf, const T* in_buf, const size_t batch_size, const size_t input_length, const size_t hidden_dim)
{
    const int bidx = blockIdx.x;

    for (int hidx = threadIdx.x; hidx < hidden_dim; hidx += blockDim.x) {
        float accum = 0.0f;
        for (int step = 0; step < input_length; step++) {
            accum += static_cast<float>(in_buf[(bidx * input_length + step) * hidden_dim + hidx]);
        }
        out_buf[bidx * hidden_dim + hidx] = accum;
    }
}

template<typename T>
void invokeSumLengthDimension(float*       out_buf,
                              const T*     in_buf,
                              const size_t batch_size,
                              const size_t input_length,
                              const size_t hidden_dim,
                              hipStream_t stream)
{
    dim3 gridSize(batch_size);
    dim3 blockSize(256);

    sum_length_dimension<<<gridSize, blockSize, 0, stream>>>(out_buf, in_buf, batch_size, input_length, hidden_dim);
}

#define INSTANTIATE_INVOKE_SUM_LENGTH_DIMENSION(T)                                                                     \
    template void invokeSumLengthDimension(float*       out_buf,                                                       \
                                           const T*     in_buf,                                                        \
                                           const size_t batch_size,                                                    \
                                           const size_t input_length,                                                  \
                                           const size_t hidden_dim,                                                    \
                                           hipStream_t stream)
INSTANTIATE_INVOKE_SUM_LENGTH_DIMENSION(half);
INSTANTIATE_INVOKE_SUM_LENGTH_DIMENSION(float);
#ifdef ENABLE_BF16
INSTANTIATE_INVOKE_SUM_LENGTH_DIMENSION(__hip_bfloat16);
#endif
#undef INSTANTIATE_INVOKE_SUM_LENGTH_DIMENSION

}  // namespace fastertransformer
