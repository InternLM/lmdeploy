#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2022-2023, NVIDIA CORPORATION.  All rights reserved.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <assert.h>
#include <hip/hip_fp16.h>
#include <hip/hip_runtime.h>

#ifndef CUDART_VERSION
#error CUDART_VERSION Undefined!
#elif (CUDART_VERSION >= 11050)
#include <hipcub/hipcub.hpp>
#else
#include "3rdparty/hipcub/hipcub.hpp"
#endif

#include "src/fastertransformer/kernels/logprob_kernels.h"
#include "src/fastertransformer/kernels/reduce_kernel_utils.cuh"
#include "src/fastertransformer/utils/logger.h"

namespace fastertransformer {

template<typename T>
__global__ void log_probs_kernel(float*       log_probs,
                                 const T*     logits,
                                 const int*   ids,
                                 const int*   lengths,
                                 const size_t max_input_length,
                                 const size_t batch_size,
                                 const size_t vocab_size,
                                 const size_t vocab_size_padded,
                                 bool         batch_first)
{
    // Calculate the log probability from logits.
    //   log_probs[t, :] = log(softmax(logits))[ids[t + 1, :]]
    //
    // log_probs: [max_length - 1, batch_size] or [batch_size, max_length -1],
    //     log probabilities of each token.
    // logits: [max_length, batch_size, vocab_size_padded] or [batch_size, max_length, vocab_size_padded]
    // lengths: [batch_size], sequence lengths
    // ids: [max_length, batch_size], token ids.
    // batch_size: [1], batch_size. in case of beam > 1, batch x beam.
    // vocab_size: [1], vocab_size,
    // vocab_size: [1], vocab_size_padded, padded vocab size.

    const bool IS_FP16   = std::is_same<T, half>::value;
    const T    MAX_T_VAL = (IS_FP16) ? HALF_FLT_MAX : FLT_MAX;

    int tidx = threadIdx.x;                            // vocab dim
    int bidx = batch_first ? blockIdx.x : blockIdx.y;  // batch dim
    int step = batch_first ? blockIdx.y : blockIdx.x;  // step dim

    __shared__ float s_max_logit;

    if (bidx < batch_size && step < lengths[bidx] - 1) {
        // reposition logits to data for the current batch.
        int step_offset  = batch_first ? step * vocab_size_padded : step * batch_size * vocab_size_padded;
        int batch_offset = batch_first ? bidx * max_input_length * vocab_size_padded : bidx * vocab_size_padded;
        logits += step_offset + batch_offset;

        // Find max(logits).
        float local_max = -MAX_T_VAL;
        float val       = -MAX_T_VAL;
        for (int i = tidx; i < vocab_size; i += blockDim.x) {
            val       = static_cast<float>(logits[i]);
            local_max = fmax(local_max, val);
        }

        float max_val = blockDim.x <= 32 ? warpReduceMax(local_max) : blockReduceMax<float>(local_max);
        if (tidx == 0) {
            s_max_logit = max_val;
        }
        __syncthreads();

        // Calculate the denominator: sum_i exp(logits[i])
        float local_sum_exp = 0.0f;
        for (int i = tidx; i < vocab_size; i += blockDim.x) {
            val = __expf(static_cast<float>(logits[i]) - s_max_logit);
            local_sum_exp += val;
        }

        float sum_exp = blockDim.x <= 32 ? warpReduceSum(local_sum_exp) : blockReduceSum<float>(local_sum_exp);
        if (tidx == 0) {
            int idx = batch_first ? step + bidx * (max_input_length - 1) : step * batch_size + bidx;
            // log_probs[step, ...] is the log probability of a token at step t + 1.
            int token_idx  = batch_first ? step + 1 + bidx * max_input_length : (step + 1) * batch_size + bidx;
            log_probs[idx] = static_cast<float>(logits[ids[token_idx]]) - s_max_logit - __logf(sum_exp + 1e-9f);
        }
    }
}

__global__ void accumulate_log_probs(float*       cum_log_probs,
                                     const float* log_probs,
                                     const int*   lengths,
                                     const size_t max_input_length,
                                     const size_t batch_size,
                                     const bool   batch_first)
{
    // Accumulate the log probability along with the sequence dimension.
    //   cum_log_probs[j] = sum_i log(softmax(logits))[ids[i,j]]
    //
    // cum_log_probs: [batch_size], cumulative log probability
    // log_probs: [max_length - 1, batch_size] or [batch_size, max_length - 1],
    //   log probability of each token
    // lengths: [batch_size], sequence lengths
    // batch_size: [1], batch_size. in case of beam > 1, batch x beam.

    int bidx = blockIdx.x;   // batch dim
    int tidx = threadIdx.x;  // step dim

    if (bidx < batch_size) {
        int length = lengths[bidx];
        // reposition logits to data for the current batch.
        log_probs += batch_first ? bidx * (max_input_length - 1) : bidx;
        int   stride      = batch_first ? 1 : batch_size;  // stride along with seq dim.
        float local_accum = 0.0f;
        for (int step = tidx; step < length - 1; step += blockDim.x) {
            local_accum += static_cast<float>(log_probs[step * stride]);
        }
        float accum = blockDim.x <= 32 ? warpReduceSum(local_accum) : blockReduceSum<float>(local_accum);
        if (tidx == 0) {
            cum_log_probs[bidx] = accum;
        }
    }
}

template<typename T>
void invokeLogProbFromLogits(float*       cum_log_probs,
                             const T*     logits,
                             const int*   input_ids,
                             const int*   input_lengths,
                             const size_t max_input_length,
                             const size_t batch_size,
                             const size_t vocab_size,
                             const size_t vocab_size_padded,
                             void*        workspace,
                             const size_t workspace_size,
                             hipStream_t stream,
                             const bool   batch_first)
{
    // A batched version of log prob computation.
    //
    // cum_log_probs: [batch_size]
    // logits: [max_input_length, batch_size, vocab_size] or [batch_size, max_input_length, vocab_size]
    // input_ids: [max_input_length, batch_size] or [max_input_length, batch_size]
    // input_lengths: [batch_size]
    // workspace: workspace buffer of size at least sizeof(float) * max_input_length * batch_size.

    FT_LOG_DEBUG(__PRETTY_FUNCTION__);
    // block_size should be multiple of 32 to use warpReduceMax.
    const int block_size = vocab_size < 1024 ? (vocab_size + 31) / 32 * 32 : 1024;
    assert(block_size % 32 == 0);
    assert(workspace != nullptr && workspace_size >= sizeof(float) * max_input_length * batch_size);
    assert(vocab_size <= vocab_size_padded);

    float* log_probs = reinterpret_cast<float*>(workspace);
    int    gx        = batch_first ? batch_size : max_input_length - 1;
    int    gy        = batch_first ? max_input_length - 1 : batch_size;
    dim3   grid(gx, gy);
    log_probs_kernel<T><<<grid, block_size, 0, stream>>>(log_probs,
                                                         logits,
                                                         input_ids,
                                                         input_lengths,
                                                         max_input_length,
                                                         batch_size,
                                                         vocab_size,
                                                         vocab_size_padded,
                                                         batch_first);
    accumulate_log_probs<<<batch_size, block_size, 0, stream>>>(
        cum_log_probs, log_probs, input_lengths, max_input_length, batch_size, batch_first);
}

template void invokeLogProbFromLogits(float*       cum_log_probs,
                                      const float* logits,
                                      const int*   input_ids,
                                      const int*   input_lengths,
                                      const size_t max_input_length,
                                      const size_t batch_size,
                                      const size_t vocab_size,
                                      const size_t vocab_size_padded,
                                      void*        workspace,
                                      const size_t workspace_size,
                                      hipStream_t stream,
                                      const bool   batch_first);

template void invokeLogProbFromLogits(float*       cum_log_probs,
                                      const half*  logits,
                                      const int*   input_ids,
                                      const int*   input_lengths,
                                      const size_t max_input_length,
                                      const size_t batch_size,
                                      const size_t vocab_size,
                                      const size_t vocab_size_padded,
                                      void*        workspace,
                                      const size_t workspace_size,
                                      hipStream_t stream,
                                      const bool   batch_first);
}  // end of namespace fastertransformer
