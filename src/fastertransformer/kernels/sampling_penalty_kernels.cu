#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2020-2023, NVIDIA CORPORATION.  All rights reserved.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <assert.h>
#include <float.h>

#include "src/fastertransformer/kernels/sampling_penalty_kernels.h"

namespace fastertransformer {

// TODO Add half2 implementation
template<typename T>
__global__ void applyTemperaturePenalty(T*          logits,
                                        const T*    bias,
                                        const float temperature_inverse,
                                        const int   m,
                                        const int   vocab_size,
                                        const int   vocab_size_padd)
{
    const bool IS_FP16   = std::is_same<T, half>::value;
    const T    MAX_T_VAL = (IS_FP16) ? 65504.F : FLT_MAX;
    for (int index = blockIdx.x * blockDim.x + threadIdx.x; index < m * vocab_size_padd;
         index += blockDim.x * gridDim.x) {
        T bias_val = bias == nullptr ? (T)(0.0f) : bias[index % vocab_size_padd];
        if (index % vocab_size_padd < vocab_size) {
            logits[index] = (logits[index] + bias_val) * (T)temperature_inverse;
        }
        else {
            logits[index] = -MAX_T_VAL;
        }
    }
}

template<>
__global__ void applyTemperaturePenalty(half2*       logits,
                                        const half2* bias,
                                        const float  temperature_inverse,
                                        const int    batch_size,
                                        const int    vocab_size,
                                        const int    vocab_size_padded)
{
    assert(vocab_size % 2 == 0);
    assert(vocab_size_padded % 2 == 0);
    const half2 mask_val = __float2half2_rn(-65504.0f);
    const half2 temp_inv = __float2half2_rn(temperature_inverse);

    const int half_vocab_size        = vocab_size / 2;
    const int half_vocab_size_padded = vocab_size_padded / 2;
    for (int index = blockIdx.x * blockDim.x + threadIdx.x; index < batch_size * half_vocab_size_padded;
         index += blockDim.x * gridDim.x) {
        int   vocab_idx = index % half_vocab_size_padded;
        half2 logit     = vocab_idx < half_vocab_size ? __ldg(&logits[index]) : mask_val;
        if (vocab_idx < half_vocab_size) {
            if (bias != nullptr) {
                logit = __hadd2(logit, bias[vocab_idx]);
            }
            logits[index] = __hmul2(logit, temp_inv);
        }
    }
}

template<typename T>
void invokeApplyTemperaturePenalty(T*           logits,
                                   const T*     bias,
                                   const float  temperature,
                                   const int    batch_size,
                                   const int    vocab_size,
                                   const int    vocab_size_padd,
                                   hipStream_t stream)
{
    dim3    block(min(vocab_size_padd, 1024));
    dim3    grid(min(batch_size * vocab_size_padd / block.x, 65536));
    const T temperature_inverse = (T)(1.f / (temperature + 1e-6f));
    if (std::is_same<T, half>::value && vocab_size % 2 == 0 && vocab_size_padd % 2 == 0) {
        applyTemperaturePenalty<<<grid, block, 0, stream>>>(reinterpret_cast<half2*>(logits),
                                                            reinterpret_cast<const half2*>(bias),
                                                            temperature_inverse,
                                                            batch_size,
                                                            vocab_size,
                                                            vocab_size_padd);
    }
    else {
        applyTemperaturePenalty<T>
            <<<grid, block, 0, stream>>>(logits, bias, temperature_inverse, batch_size, vocab_size, vocab_size_padd);
    }
}

template void invokeApplyTemperaturePenalty(float*       logits,
                                            const float* bias,
                                            const float  temperature,
                                            const int    batch_size,
                                            const int    vocab_size,
                                            const int    vocab_size_padd,
                                            hipStream_t stream);

template void invokeApplyTemperaturePenalty(half*        logits,
                                            const half*  bias,
                                            const float  temperature,
                                            const int    batch_size,
                                            const int    vocab_size,
                                            const int    vocab_size_padd,
                                            hipStream_t stream);

template<typename T>
__global__ void batchApplyTemperaturePenalty(T*           logits,
                                             const T*     bias,
                                             const float* temperatures,
                                             const int    batch_size,
                                             const int    vocab_size,
                                             const int    vocab_size_padd)
{
    // TODO: Add macro or device function to get MAX_T_VAL.
    const bool              IS_FP16   = std::is_same<T, half>::value;
    const T                 MAX_T_VAL = (IS_FP16) ? 65504.F : FLT_MAX;
    extern __shared__ float inv_temperatures[];
    if (threadIdx.x < batch_size) {
        inv_temperatures[threadIdx.x] = 1.0f / (temperatures[threadIdx.x] + 1e-6f);
    }
    __syncthreads();

    for (int index = blockIdx.x * blockDim.x + threadIdx.x; index < batch_size * vocab_size_padd;
         index += blockDim.x * gridDim.x) {
        int batch_idx = index / vocab_size_padd;
        int vocab_idx = index % vocab_size_padd;
        T   logit     = (vocab_idx < vocab_size) ? logits[index] : -MAX_T_VAL;
        if (vocab_idx < vocab_size) {
            if (bias != nullptr) {
                logit += bias[vocab_idx];
            }
            logit *= inv_temperatures[batch_idx];
        }
        logits[index] = logit;
    }
}

__global__ void batchApplyTemperaturePenalty_h2(half2*       logits,
                                                const half2* bias,
                                                const float* temperatures,
                                                const int    batch_size,
                                                const int    vocab_size,
                                                const int    vocab_size_padded)
{
    assert(vocab_size % 2 == 0);
    assert(vocab_size_padded % 2 == 0);
    extern __shared__ half2 h2_inv_temperatures[];
    if (threadIdx.x < batch_size) {
        h2_inv_temperatures[threadIdx.x] = __float2half2_rn(1.f / (temperatures[threadIdx.x] + 1e-6f));
    }
    __syncthreads();

    const half2 mask_val               = __float2half2_rn(-65504.0f);
    const int   half_vocab_size        = vocab_size / 2;
    const int   half_vocab_size_padded = vocab_size_padded / 2;
    for (int index = blockIdx.x * blockDim.x + threadIdx.x; index < batch_size * half_vocab_size_padded;
         index += blockDim.x * gridDim.x) {
        int   batch_idx = index / half_vocab_size_padded;
        int   vocab_idx = index % half_vocab_size_padded;
        half2 logit     = vocab_idx < half_vocab_size ? __ldg(&logits[index]) : mask_val;
        if (vocab_idx < half_vocab_size) {
            if (bias != nullptr) {
                logit = __hadd2(logit, bias[vocab_idx]);
            }
            logits[index] = __hmul2(logit, h2_inv_temperatures[batch_idx]);
        }
    }
}

template<typename T>
void invokeBatchApplyTemperaturePenalty(T*           logits,
                                        const T*     bias,
                                        const float* temperatures,
                                        const int    batch_size,
                                        const int    vocab_size,
                                        const int    vocab_size_padd,
                                        hipStream_t stream)
{
    dim3 block(min(vocab_size_padd, 1024));
    dim3 grid(min(batch_size * vocab_size_padd / block.x, 65536));
    if (std::is_same<T, half>::value && vocab_size % 2 == 0 && vocab_size_padd % 2 == 0) {
        size_t smem_size = sizeof(half2) * batch_size;
        batchApplyTemperaturePenalty_h2<<<grid, block, smem_size, stream>>>(reinterpret_cast<half2*>(logits),
                                                                            reinterpret_cast<const half2*>(bias),
                                                                            temperatures,
                                                                            batch_size,
                                                                            vocab_size,
                                                                            vocab_size_padd);
    }
    else {
        size_t smem_size = sizeof(float) * batch_size;
        batchApplyTemperaturePenalty<T>
            <<<grid, block, smem_size, stream>>>(logits, bias, temperatures, batch_size, vocab_size, vocab_size_padd);
    }
}

template void invokeBatchApplyTemperaturePenalty(float*       logits,
                                                 const float* bias,
                                                 const float* temperatures,
                                                 const int    batch_size,
                                                 const int    vocab_size,
                                                 const int    vocab_size_padd,
                                                 hipStream_t stream);

template void invokeBatchApplyTemperaturePenalty(half*        logits,
                                                 const half*  bias,
                                                 const float* temperatures,
                                                 const int    batch_size,
                                                 const int    vocab_size,
                                                 const int    vocab_size_padd,
                                                 hipStream_t stream);

template<typename T, RepetitionPenaltyType penalty_type>
__global__ void applyRepetitionPenalty(T*          logits,
                                       const float penalty,
                                       const int*  start_ids,
                                       int*        output_ids,
                                       const int   batch_size,
                                       const int   local_batch_size,
                                       const int   vocab_size,
                                       const int   vocab_size_padd,
                                       const int*  input_lengths,
                                       const int   max_input_len,
                                       const int   step)
{
    extern __shared__ float penalty_logits[];
    int*                    penalty_indices = (int*)(penalty_logits + step);

    logits                 = logits + blockIdx.x * vocab_size_padd;
    const int input_length = input_lengths != nullptr ? input_lengths[blockIdx.x] : max_input_len;
    for (int index = threadIdx.x; index < step; index += blockDim.x) {

        if (index >= input_length && index < max_input_len) {
            continue;
        }

        // output_ids shape: (input_len + output_len, batch_size)
        int penalty_index = output_ids[index * batch_size + blockIdx.x];
        if (penalty_index >= vocab_size) {
            continue;
        }
        penalty_indices[index] = penalty_index;
        float logit            = (float)logits[penalty_index];
        if (penalty_type == RepetitionPenaltyType::Additive) {
            penalty_logits[index] = logit - penalty;
        }
        else if (penalty_type == RepetitionPenaltyType::Multiplicative) {
            penalty_logits[index] = logit < 0.0f ? logit * penalty : logit / penalty;
        }
        else if (penalty_type == RepetitionPenaltyType::None) {
            penalty_logits[index] = logit;
        }
        else {
            // Unsupported type
            assert(false);
        }
    }

    if (blockDim.x > 32) {
        __syncthreads();
    }

    for (int index = threadIdx.x; index < step; index += blockDim.x) {

        if (index >= input_length && index < max_input_len) {
            continue;
        }

        // output_ids shape: (input_len + output_len, batch_size)
        if (penalty_indices[index] >= vocab_size) {
            continue;
        }
        logits[penalty_indices[index]] = penalty_logits[index];
    }
}

template<typename T>
void invokeApplyRepetitionPenalty(T*                          logits,
                                  const float                 penalty,
                                  const int*                  start_ids,
                                  int*                        output_ids,
                                  const int                   batch_size,
                                  const int                   local_batch_size,
                                  const int                   vocab_size,
                                  const int                   vocab_size_padd,
                                  const int*                  input_lengths,
                                  const int                   max_input_len,
                                  const int                   step,
                                  const RepetitionPenaltyType penalty_type,
                                  hipStream_t                stream)
{
    dim3   block(min(step, 1024));
    dim3   grid(local_batch_size);
    size_t smem_size = step * (sizeof(float) + sizeof(int));

    if (penalty_type == RepetitionPenaltyType::Additive) {
        applyRepetitionPenalty<T, RepetitionPenaltyType::Additive><<<grid, block, smem_size, stream>>>(logits,
                                                                                                       penalty,
                                                                                                       start_ids,
                                                                                                       output_ids,
                                                                                                       batch_size,
                                                                                                       local_batch_size,
                                                                                                       vocab_size,
                                                                                                       vocab_size_padd,
                                                                                                       input_lengths,
                                                                                                       max_input_len,
                                                                                                       step);
    }
    else if (penalty_type == RepetitionPenaltyType::Multiplicative) {
        applyRepetitionPenalty<T, RepetitionPenaltyType::Multiplicative>
            <<<grid, block, smem_size, stream>>>(logits,
                                                 penalty,
                                                 start_ids,
                                                 output_ids,
                                                 batch_size,
                                                 local_batch_size,
                                                 vocab_size,
                                                 vocab_size_padd,
                                                 input_lengths,
                                                 max_input_len,
                                                 step);
    }
    else if (penalty_type == RepetitionPenaltyType::None) {
        // do nothing
    }
}

template void invokeApplyRepetitionPenalty(float*                      logits,
                                           const float                 penalty,
                                           const int*                  start_ids,
                                           int*                        output_ids,
                                           const int                   batch_size,
                                           const int                   local_batch_size,
                                           const int                   vocab_size,
                                           const int                   vocab_size_padd,
                                           const int*                  input_lengths,
                                           const int                   max_input_len,
                                           const int                   step,
                                           const RepetitionPenaltyType penalty_type,
                                           hipStream_t                stream);

template void invokeApplyRepetitionPenalty(half*                       logits,
                                           const float                 penalty,
                                           const int*                  start_ids,
                                           int*                        output_ids,
                                           const int                   batch_size,
                                           const int                   local_batch_size,
                                           const int                   vocab_size,
                                           const int                   vocab_size_padd,
                                           const int*                  input_lengths,
                                           const int                   max_input_len,
                                           const int                   step,
                                           const RepetitionPenaltyType penalty_type,
                                           hipStream_t                stream);

template<typename T, RepetitionPenaltyType penalty_type>
__global__ void batchApplyRepetitionPenalty(T*           logits,
                                            const float* penalties,
                                            const int*   output_ids,
                                            const int    batch_size,
                                            const int    vocab_size,
                                            const int*   input_lengths,
                                            const int    max_input_length,
                                            const int    step)
{
    extern __shared__ float penalty_logits[];
    int*                    penalty_indices = (int*)(penalty_logits + step);
    const int               batch_idx       = blockIdx.x;
    const float             penalty         = penalties[batch_idx];
    const int               input_length    = input_lengths != nullptr ? input_lengths[batch_idx] : max_input_length;

    logits += batch_idx * vocab_size;

    // Phase 1. Find indices to penalize and keep the penalized values.
    // A vocab id can appear multiple times but should be penalized once.
    for (int index = threadIdx.x; index < step; index += blockDim.x) {
        // Skip the padding tokens in input sequences.
        if (index >= input_length && index < max_input_length) {
            continue;
        }
        // output_ids shape: (input_len + output_len, batch_size)
        int penalty_index = output_ids[index * batch_size + batch_idx];
        assert(penalty_index < vocab_size);
        penalty_indices[index] = penalty_index;
        float logit            = (float)logits[penalty_index];
        if (penalty_type == RepetitionPenaltyType::Additive) {
            penalty_logits[index] = logit - penalty;
        }
        else if (penalty_type == RepetitionPenaltyType::Multiplicative) {
            penalty_logits[index] = logit < 0.0f ? logit * penalty : logit / penalty;
        }
        else if (penalty_type == RepetitionPenaltyType::None) {
            penalty_logits[index] = logit;
        }
        else {
            // Unsupported type
            assert(false);
        }
    }

    if (blockDim.x > 32) {
        __syncthreads();
    }

    // Phase 2. Replace a logit value by the penalized one.
    for (int index = threadIdx.x; index < step; index += blockDim.x) {
        // Skip the padding tokens in input sequences.
        if (index >= input_length && index < max_input_length) {
            continue;
        }
        logits[penalty_indices[index]] = penalty_logits[index];
    }
}

template<typename T>
void invokeBatchApplyRepetitionPenalty(T*                    logits,
                                       const float*          penalties,
                                       const int*            output_ids,
                                       const int             batch_size,
                                       const int             local_batch_size,
                                       const int             vocab_size,
                                       const int*            input_lengths,
                                       const int             max_input_length,
                                       const int             step,
                                       RepetitionPenaltyType penalty_type,
                                       hipStream_t          stream)
{
    // Inputs
    //   logits [local_batch_size, vocab_size] : logit values.
    //   penalties [local_batch_size] : repetition penalty factors.
    //   output_ids [step, batch_size] : output token ids (with offset ite * local_batch_size).
    //   input_lengths [local_batch_size], input lengths (optional).
    //      Padding tokens at [input_length, max_input_length) of input will not be penalized.
    dim3   block(min(step, 1024));
    dim3   grid(local_batch_size);
    size_t smem_size = step * (sizeof(float) + sizeof(int));
    if (penalty_type == RepetitionPenaltyType::Additive) {
        batchApplyRepetitionPenalty<T, RepetitionPenaltyType::Additive><<<grid, block, smem_size, stream>>>(
            logits, penalties, output_ids, batch_size, vocab_size, input_lengths, max_input_length, step);
    }
    else if (penalty_type == RepetitionPenaltyType::Multiplicative) {
        batchApplyRepetitionPenalty<T, RepetitionPenaltyType::Multiplicative><<<grid, block, smem_size, stream>>>(
            logits, penalties, output_ids, batch_size, vocab_size, input_lengths, max_input_length, step);
    }
    else if (penalty_type == RepetitionPenaltyType::None) {
        // do nothing
    }
}

template void invokeBatchApplyRepetitionPenalty(float*                logits,
                                                const float*          penalties,
                                                const int*            output_ids,
                                                const int             batch_size,
                                                const int             local_batch_size,
                                                const int             vocab_size,
                                                const int*            input_lengths,
                                                const int             max_input_length,
                                                const int             step,
                                                RepetitionPenaltyType penalty_type,
                                                hipStream_t          stream);

template void invokeBatchApplyRepetitionPenalty(half*                 logits,
                                                const float*          penalties,
                                                const int*            output_ids,
                                                const int             batch_size,
                                                const int             local_batch_size,
                                                const int             vocab_size,
                                                const int*            input_lengths,
                                                const int             max_input_length,
                                                const int             step,
                                                RepetitionPenaltyType penalty_type,
                                                hipStream_t          stream);

template<typename T>
__global__ void batchApplyMinLengthPenalty(T*         logits,
                                           const int* min_lengths,
                                           const int* end_ids,
                                           const int* sequence_lengths,
                                           const int  max_input_length,
                                           const int  vocab_size_padded)
{
    int bid = threadIdx.x + blockIdx.x * blockDim.x;  // batch index
    // We need +1 because sequence_lengths = max_input_length + num_gen_tokens - 1,
    // which is equal to the length of k/v caches.
    if (sequence_lengths[bid] + 1 - max_input_length < min_lengths[bid]) {
        T mask_val                                     = (std::is_same<T, half>::value) ? -65504.0f : -FLT_MAX;
        logits[bid * vocab_size_padded + end_ids[bid]] = mask_val;
    }
}

template<typename T>
void invokeMinLengthPenalty(T*           logits,
                            const int*   min_lengths,
                            const int*   end_ids,
                            const int*   sequnece_lengths,
                            const int    max_input_length,
                            const int    batch_size,
                            const int    vocab_size_padded,
                            hipStream_t stream)

{
    const int block_size = min(batch_size, 1024);
    const int grid_size  = (batch_size + block_size - 1) / block_size;
    batchApplyMinLengthPenalty<<<grid_size, block_size, 0, stream>>>(
        logits, min_lengths, end_ids, sequnece_lengths, max_input_length, vocab_size_padded);
}

template void invokeMinLengthPenalty(float*       logits,
                                     const int*   min_lengths,
                                     const int*   end_ids,
                                     const int*   sequnece_lengths,
                                     const int    max_input_length,
                                     const int    batch_size,
                                     const int    vocab_size_padded,
                                     hipStream_t stream);

template void invokeMinLengthPenalty(half*        logits,
                                     const int*   min_lengths,
                                     const int*   end_ids,
                                     const int*   sequnece_lengths,
                                     const int    max_input_length,
                                     const int    batch_size,
                                     const int    vocab_size_padded,
                                     hipStream_t stream);

}  // namespace fastertransformer
