#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2019-2023, NVIDIA CORPORATION.  All rights reserved.
 * Copyright (c) 2021, NAVER Corp.  Authored by CLOVA.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <stdexcept>
#ifndef CUDART_VERSION
#error CUDART_VERSION Undefined!
#elif (CUDART_VERSION >= 11050)
#include <hipcub/hipcub.hpp>
#else
#include "3rdparty/hipcub/hipcub.hpp"
#endif

#include "src/fastertransformer/kernels/reduce_kernel_utils.cuh"
#include "src/fastertransformer/kernels/sampling_topk_kernels.h"

namespace fastertransformer {

__global__ void curandInitialize(hiprandState_t* state, const int size, const unsigned long long random_seed)
{
    if (threadIdx.x + blockIdx.x * blockDim.x < size) {
        hiprand_init(random_seed, 0, 0, &state[blockIdx.x * blockDim.x + threadIdx.x]);
    }
}

void invokeCurandInitialize(hiprandState_t*           state,
                            const size_t             batch_size,
                            const unsigned long long random_seed,
                            hipStream_t             stream)
{
    dim3 block(256);
    dim3 grid((int)(ceil(batch_size * 1.0 / 256)));
    curandInitialize<<<grid, block, 0, stream>>>(state, batch_size, random_seed);
}

__global__ void curandBatchInitialize(hiprandState_t* states, const int size, const unsigned long long* random_seeds)
{
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    if (idx < size) {
        hiprand_init(random_seeds[idx], 0, 0, &states[idx]);
    }
}

void invokeCurandBatchInitialize(hiprandState_t*            states,
                                 const size_t              batch_size,
                                 const unsigned long long* random_seeds,
                                 hipStream_t              stream)
{
    dim3 block(256);
    dim3 grid((int)(ceil(batch_size * 1.0 / 256)));
    curandBatchInitialize<<<grid, block, 0, stream>>>(states, batch_size, random_seeds);
}

template<typename T>
__global__ void addBiasEndMask(T*          logits,
                               const T*    bias,
                               const int*  end_ids,
                               const bool* finished,
                               const int   vocab_size,
                               const int   vocab_size_padded)
{
    int  bid    = blockIdx.x;
    bool finish = finished != nullptr ? finished[bid] : false;
    int  offset = bid * vocab_size_padded;

    const bool IS_FP16   = std::is_same<T, half>::value;
    const T    MAX_T_VAL = (IS_FP16) ? HALF_FLT_MAX : FLT_MAX;
    for (int tid = threadIdx.x; tid < vocab_size_padded; tid += blockDim.x) {
        if (tid >= vocab_size) {
            logits[offset + tid] = -MAX_T_VAL;
        }
        else if (finish) {
            logits[offset + tid] = (tid == end_ids[bid]) ? MAX_T_VAL : -MAX_T_VAL;
        }
        else {
            if (bias != nullptr) {
                logits[offset + tid] += bias[tid];
            }
        }
    }
}

template<typename T>
void invokeAddBiasEndMask(T*           logits,
                          const T*     bias,
                          const int*   end_ids,
                          const bool*  finished,
                          const int    batch_size,
                          const int    vocab_size,
                          const int    vocab_size_padded,
                          hipStream_t stream)
{
    dim3 grid(batch_size);
    dim3 block(min(vocab_size_padded, 1024));
    /*n is the vocab_size, e.g., 30000, 7000.... vocab_size is usually very big. */
    addBiasEndMask<<<grid, block, 0, stream>>>(logits, bias, end_ids, finished, vocab_size, vocab_size_padded);
}

template void invokeAddBiasEndMask(float*       logits,
                                   const float* bias,
                                   const int*   end_ids,
                                   const bool*  finished,
                                   const int    batch_size,
                                   const int    vocab_size,
                                   const int    vocab_size_padded,
                                   hipStream_t stream);

template void invokeAddBiasEndMask(half*        logits,
                                   const half*  bias,
                                   const int*   end_ids,
                                   const bool*  finished,
                                   const int    batch_size,
                                   const int    vocab_size,
                                   const int    vocab_size_padded,
                                   hipStream_t stream);

template<typename T, int BLOCK_SIZE_, int BLOCKS_PER_BEAM_>
__global__ void topk_stage1(const T* __restrict log_probs,
                            T*          tmp_log_probs,
                            int*        topk_tmp_id_buf,
                            T*          topk_tmp_val_buf,
                            const bool* finished,
                            const int   max_top_k,
                            const int*  top_ks,
                            const int   vocab_size,
                            const int*  end_ids,
                            const bool* skip_decode)
{
    typedef hipcub::BlockReduce<TopK_2<T>, BLOCK_SIZE_> BlockReduce;
    __shared__ typename BlockReduce::TempStorage     temp_storage;

    const int tid = threadIdx.x;
    const int bid = blockIdx.x;

    const int batch_id = bid / BLOCKS_PER_BEAM_;  // row id for log_probs
    if (skip_decode != nullptr && skip_decode[batch_id]) {
        return;
    }
    const int block_lane = bid % BLOCKS_PER_BEAM_;                              // block id for a beam
    const int k          = (top_ks != nullptr) ? top_ks[batch_id] : max_top_k;  // batch_id = batch index

    const int tmp_log_buf_index  = batch_id * vocab_size;
    const int tmp_topk_buf_index = batch_id * BLOCKS_PER_BEAM_ * max_top_k + block_lane * k;

    TopK_2<T>  partial;
    const bool IS_FP16   = std::is_same<T, half>::value;
    const T    MAX_T_VAL = (IS_FP16) ? HALF_FLT_MAX : FLT_MAX;

    if (finished != nullptr && finished[batch_id] == true) {
        if (tid < k) {
            const int index = tmp_topk_buf_index + tid;
            if (block_lane == 0 && tid == 0) {
                const int end_id        = end_ids[batch_id];
                topk_tmp_id_buf[index]  = tmp_log_buf_index + end_id;
                topk_tmp_val_buf[index] = log_probs[tmp_log_buf_index + end_id];
            }
            else {
                topk_tmp_id_buf[index]  = -1;
                topk_tmp_val_buf[index] = -MAX_T_VAL;
            }
        }
        return;
    }

    for (int elem_id = tid + block_lane * BLOCK_SIZE_; elem_id < vocab_size;
         elem_id += BLOCK_SIZE_ * BLOCKS_PER_BEAM_) {
        int index            = elem_id + tmp_log_buf_index;
        tmp_log_probs[index] = log_probs[index];
    }

    for (int ite = 0; ite < k; ite++) {
        partial.init();
#pragma unroll
        for (int elem_id = tid + block_lane * BLOCK_SIZE_; elem_id < vocab_size;
             elem_id += BLOCK_SIZE_ * BLOCKS_PER_BEAM_) {
            int index = elem_id + tmp_log_buf_index;
            partial.insert(tmp_log_probs[index], index);
        }

        TopK_2<T> total = BlockReduce(temp_storage).Reduce(partial, reduce_topk_op_2<T>);

        if (tid == 0) {
            const int index         = tmp_topk_buf_index + ite;
            topk_tmp_id_buf[index]  = total.p;
            topk_tmp_val_buf[index] = total.u;
            tmp_log_probs[total.p]  = -MAX_T_VAL;
        }
        __syncthreads();
    }
}

template<typename T, int BLOCK_SIZE_, int BLOCKS_PER_BEAM_>
__global__ void topk_stage2_sampling(const int* __restrict topk_tmp_id_buf,
                                     T*             topk_tmp_val_buf,
                                     int*           ids,
                                     int*           sequence_length,
                                     bool*          finished,
                                     float*         cum_log_probs,
                                     float*         output_log_probs,
                                     const int      max_top_k,
                                     const int*     top_ks,
                                     const float    top_p,
                                     const float*   top_ps,
                                     hiprandState_t* curandstate,
                                     const int*     end_ids,
                                     const int      vocab_size,
                                     const bool*    skip_decode)
{
    const bool IS_FP16   = std::is_same<T, half>::value;
    const T    MAX_T_VAL = (IS_FP16) ? HALF_FLT_MAX : FLT_MAX;

    const int tid      = threadIdx.x;
    const int batch_id = blockIdx.x;
    if (skip_decode != nullptr && skip_decode[batch_id]) {
        return;
    }

    const int   k              = (top_ks != nullptr) ? top_ks[batch_id] : max_top_k;
    const float prob_threshold = (top_ps != nullptr) ? top_ps[batch_id] : top_p;
    const int   size           = k * BLOCKS_PER_BEAM_;
    const int   stride         = max_top_k * BLOCKS_PER_BEAM_;

    typedef hipcub::BlockReduce<TopK_2<float>, BLOCK_SIZE_> BlockReduce;
    __shared__ typename BlockReduce::TempStorage         temp_storage;
    extern __shared__ char                               array[];
    __shared__ float                                     rand_num;
    __shared__ float                                     s_sum;
    __shared__ float                                     s_max;
    T*                                                   s_val = topk_tmp_val_buf + batch_id * stride;
    int*                                                 s_id  = reinterpret_cast<int*>(array);
    if (tid == 0) {
        s_sum = 0.0f;
    }
    TopK_2<float> partial;

    if (finished != nullptr && finished[batch_id] == true) {
        ids[batch_id] = end_ids[batch_id];
        return;
    }

    float* s_val2 = reinterpret_cast<float*>(s_id + k);
    for (int ite = 0; ite < k; ite++) {
        partial.init();
#pragma unroll
        for (int i = tid; i < size; i += BLOCK_SIZE_) {
            partial.insert((float)s_val[i], i);
        }

        TopK_2<float> total = BlockReduce(temp_storage).Reduce(partial, reduce_topk_op_2<float>);

        if (tid == 0) {
            if (ite == 0) {
                s_max = total.u;
            }
            s_id[ite]      = total.p;
            s_val[total.p] = -MAX_T_VAL;

            // when cum_log_probs are computed, topk_tmp_val_buf (logits_buf_) are already pre-processed by
            // softmax_kernel
            if (cum_log_probs == nullptr && output_log_probs == nullptr) {
                total.u = __expf(total.u - s_max);
            }
            s_val2[ite] = total.u;
            s_sum += total.u;
        }
        __syncthreads();
    }

    if (tid == 0) {
        rand_num = (float)hiprand_uniform(curandstate + blockIdx.x) * prob_threshold * s_sum;
        for (int i = 0; i < k; i++) {
            float exp_logit = s_val2[i];
            rand_num        = rand_num - exp_logit;
            if (rand_num <= 0.0f || i == k - 1) {
                ids[batch_id] = topk_tmp_id_buf[batch_id * stride + s_id[i]] % vocab_size;
                if (cum_log_probs != nullptr || output_log_probs != nullptr) {
                    float log_prob = logf(exp_logit);
                    if (cum_log_probs != nullptr) {
                        cum_log_probs[batch_id] += log_prob;
                    }
                    if (output_log_probs != nullptr) {
                        // 'output_log_probs' is the probability induced by the top-k sampling.
                        // We normalize the probability 'exp_logit' of the selected token by
                        // the probability 's_sum' of a set of top-k tokens, meaning the log_prob
                        // is the probability of the selected token, conditioned on the event that
                        // it is selected, i.e.,
                        //   log_prob = log P(i | i is in top-k) = log(exp_logit / s_sum).
                        output_log_probs[batch_id] = log_prob - logf(s_sum);
                    }
                }
                break;
            }
        }
        if (sequence_length != nullptr && finished != nullptr) {
            sequence_length[batch_id] = finished[batch_id] ? sequence_length[batch_id] : sequence_length[batch_id] + 1;
            finished[batch_id]        = ids[batch_id] == end_ids[batch_id] ? true : false;
        }
    }
}

#define CASE_K(K_MIN, K_MAX, BLOCK_SIZE_1_, BLOCK_SIZE_2_, BLOCKS_PER_BEAM_)                                           \
    case K_MIN ... K_MAX:                                                                                              \
        topk_stage1<T, BLOCK_SIZE_1_, BLOCKS_PER_BEAM_>                                                                \
            <<<batch_size * BLOCKS_PER_BEAM_, BLOCK_SIZE_1_, 0, stream>>>(log_probs,                                   \
                                                                          temp_log_probs,                              \
                                                                          topk_tmp_id_buf,                             \
                                                                          topk_tmp_val_buf,                            \
                                                                          finished,                                    \
                                                                          max_top_k,                                   \
                                                                          top_ks,                                      \
                                                                          vocab_size,                                  \
                                                                          end_ids,                                     \
                                                                          skip_decode);                                \
        topk_stage2_sampling<T, BLOCK_SIZE_2_, BLOCKS_PER_BEAM_>                                                       \
            <<<batch_size, BLOCK_SIZE_2_, K_MAX * sizeof(int) + K_MAX * sizeof(float), stream>>>(topk_tmp_id_buf,      \
                                                                                                 topk_tmp_val_buf,     \
                                                                                                 ids,                  \
                                                                                                 sequence_length,      \
                                                                                                 finished,             \
                                                                                                 cum_log_probs,        \
                                                                                                 output_log_probs,     \
                                                                                                 max_top_k,            \
                                                                                                 top_ks,               \
                                                                                                 top_p,                \
                                                                                                 top_ps,               \
                                                                                                 curandstate,          \
                                                                                                 end_ids,              \
                                                                                                 vocab_size,           \
                                                                                                 skip_decode);         \
        break;

template<typename T>
void invokeBatchTopKSampling(void*          workspace,
                             size_t&        workspace_size,
                             const T*       log_probs,
                             int*           ids,
                             int*           sequence_length,
                             bool*          finished,
                             float*         cum_log_probs,
                             float*         output_log_probs,
                             hiprandState_t* curandstate,
                             const int      max_top_k,
                             const int*     top_ks,
                             const float    top_p,
                             const float*   top_ps,
                             const int      vocab_size_padded,
                             const int*     end_ids,
                             hipStream_t   stream,
                             const int      batch_size,
                             const bool*    skip_decode)
{
    // Not allow an ambiguous inputs top_p and top_ps.
    assert(top_p == 1.0f || top_ps == nullptr);
    const int vocab_size              = vocab_size_padded;
    const int max_block_per_beam      = 8;
    int       temp_log_probs_buf_size = batch_size * vocab_size;                      // type float
    int       topk_tmp_ids_buf_size   = batch_size * max_top_k * max_block_per_beam;  // type int
    int       topk_tmp_val_buf_size   = batch_size * max_top_k * max_block_per_beam;  // type float

    // prevent memory misaligned address
    temp_log_probs_buf_size = (int)(ceil(temp_log_probs_buf_size / 4.)) * 4;
    topk_tmp_ids_buf_size   = (int)(ceil(topk_tmp_ids_buf_size / 4.)) * 4;
    topk_tmp_val_buf_size   = (int)(ceil(topk_tmp_val_buf_size / 4.)) * 4;

    if (workspace == nullptr) {
        workspace_size = sizeof(T) * temp_log_probs_buf_size + sizeof(int) * topk_tmp_ids_buf_size
                         + sizeof(T) * topk_tmp_val_buf_size;
        return;
    }

    T*   temp_log_probs   = (T*)workspace;
    int* topk_tmp_id_buf  = (int*)(temp_log_probs + temp_log_probs_buf_size);
    T*   topk_tmp_val_buf = (T*)(topk_tmp_id_buf + topk_tmp_ids_buf_size);

    switch (max_top_k) {
        CASE_K(1, 16, 128, 128, 8);
        CASE_K(17, 32, 256, 128, 8);
        CASE_K(33, 64, 256, 256, 8);
        CASE_K(65, 1024, 256, 256, 8);
        default:
            throw std::domain_error(fmtstr("top-k kernel supports 1<=k<=1024 but got k=%d", max_top_k));
    }
}

#undef CASE_K

template void invokeBatchTopKSampling(void*          workspace,
                                      size_t&        workspace_size,
                                      const float*   log_probs,
                                      int*           ids,
                                      int*           sequence_length,
                                      bool*          finished_buf,
                                      float*         cum_log_probs,
                                      float*         output_log_probs,
                                      hiprandState_t* curandstate,
                                      const int      max_top_k,
                                      const int*     top_ks,
                                      const float    top_p,
                                      const float*   top_ps,
                                      const int      vocab_size_padded,
                                      const int*     end_ids,
                                      hipStream_t   stream,
                                      const int      batch_size,
                                      const bool*    skip_decode);

template void invokeBatchTopKSampling(void*          workspace,
                                      size_t&        workspace_size,
                                      const half*    log_probs,
                                      int*           ids,
                                      int*           sequence_length,
                                      bool*          finished_buf,
                                      float*         cum_log_probs,
                                      float*         output_log_probs,
                                      hiprandState_t* curandstate,
                                      const int      max_top_k,
                                      const int*     top_ks,
                                      const float    top_p,
                                      const float*   top_ps,
                                      const int      vocab_size_padded,
                                      const int*     end_ids,
                                      hipStream_t   stream,
                                      const int      batch_size,
                                      const bool*    skip_decode);

template<typename T>
void invokeTopKSampling(void*          workspace,
                        size_t&        workspace_size,
                        const T*       log_probs,
                        int*           ids,
                        int*           sequence_length,
                        bool*          finished_buf,
                        float*         cum_log_probs,
                        float*         output_log_probs,
                        hiprandState_t* curandstate,
                        const int      top_k,
                        const float    top_p,
                        const int      vocab_size_padded,
                        const int*     end_ids,
                        hipStream_t   stream,
                        const int      batch_size,
                        const bool*    skip_decode)
{
    invokeBatchTopKSampling(workspace,
                            workspace_size,
                            log_probs,
                            ids,
                            sequence_length,
                            finished_buf,
                            cum_log_probs,
                            output_log_probs,
                            curandstate,
                            top_k,
                            nullptr,
                            top_p,
                            nullptr,
                            vocab_size_padded,
                            end_ids,
                            stream,
                            batch_size,
                            skip_decode);
}

template void invokeTopKSampling(void*          workspace,
                                 size_t&        workspace_size,
                                 const float*   log_probs,
                                 int*           ids,
                                 int*           sequence_length,
                                 bool*          finished_buf,
                                 float*         cum_log_probs,
                                 float*         output_log_probs,
                                 hiprandState_t* curandstate,
                                 const int      top_k,
                                 const float    top_p,
                                 const int      vocab_size_padded,
                                 const int*     end_ids,
                                 hipStream_t   stream,
                                 const int      batch_size,
                                 const bool*    skip_decode);

template void invokeTopKSampling(void*          workspace,
                                 size_t&        workspace_size,
                                 const half*    log_probs,
                                 int*           ids,
                                 int*           sequence_length,
                                 bool*          finished_buf,
                                 float*         cum_log_probs,
                                 float*         output_log_probs,
                                 hiprandState_t* curandstate,
                                 const int      top_k,
                                 const float    top_p,
                                 const int      vocab_size_padded,
                                 const int*     end_ids,
                                 hipStream_t   stream,
                                 const int      batch_size,
                                 const bool*    skip_decode);

template<typename T>
void invokeTopKTopPSampling(void*          workspace,
                            size_t&        workspace_size,
                            int*           output_ids,
                            const T*       logits,
                            int*           sequence_length,
                            bool*          finished_buf,
                            float*         cum_log_probs,
                            float*         output_log_probs,
                            hiprandState_t* curandstate,
                            const int      batch_size,
                            const int      top_k,
                            const float    top_p,
                            const int      vocab_size_padded,
                            const int*     end_ids,
                            hipStream_t   stream)
{
    // invokeTopKTopPSampling will be deprecated. Please use invokeTopKSampling instead.
    invokeTopKSampling(workspace,
                       workspace_size,
                       logits,
                       output_ids,
                       sequence_length,
                       finished_buf,
                       cum_log_probs,
                       output_log_probs,
                       curandstate,
                       top_k,
                       top_p,
                       vocab_size_padded,
                       end_ids,
                       stream,
                       batch_size,
                       nullptr);
}

template void invokeTopKTopPSampling(void*          workspace,
                                     size_t&        workspace_size,
                                     int*           output_ids,
                                     const float*   logits,
                                     int*           sequence_length,
                                     bool*          finished_buf,
                                     float*         cum_log_probs,
                                     float*         output_log_probs,
                                     hiprandState_t* curandstate,
                                     const int      batch_size,
                                     const int      top_k,
                                     const float    top_p,
                                     const int      vocab_size_padded,
                                     const int*     end_ids,
                                     hipStream_t   stream);

template void invokeTopKTopPSampling(void*          workspace,
                                     size_t&        workspace_size,
                                     int*           output_ids,
                                     const half*    logits,
                                     int*           sequence_length,
                                     bool*          finished_buf,
                                     float*         cum_log_probs,
                                     float*         output_log_probs,
                                     hiprandState_t* curandstate,
                                     const int      batch_size,
                                     const int      top_k,
                                     const float    top_p,
                                     const int      vocab_size_padded,
                                     const int*     end_ids,
                                     hipStream_t   stream);

}  // namespace fastertransformer
