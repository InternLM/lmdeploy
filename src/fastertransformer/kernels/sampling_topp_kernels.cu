#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2019-2023, NVIDIA CORPORATION.  All rights reserved.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#ifndef CUDART_VERSION
#error CUDART_VERSION Undefined!
#elif (CUDART_VERSION >= 11050)
#include <hipcub/hipcub.hpp>
#else
#include "3rdparty/hipcub/hipcub.hpp"
#endif

#include "src/fastertransformer/kernels/reduce_kernel_utils.cuh"
#include "src/fastertransformer/kernels/sampling_topp_kernels.h"
#include "src/fastertransformer/utils/cuda_utils.h"

constexpr int   ENABLE_SINGLE_PASS_TOP_P = 0;
constexpr float SINGLE_PASS_THRESHOLD    = 0.9;

namespace fastertransformer {

namespace segmented_topp_impl {

template<int HALF_ELEMENTS_PER_WARP_LOAD>
using Copy_half_t = typename std::conditional<
    HALF_ELEMENTS_PER_WARP_LOAD == 32,
    half,
    typename std::conditional<HALF_ELEMENTS_PER_WARP_LOAD == 64,
                              int,
                              typename std::conditional<HALF_ELEMENTS_PER_WARP_LOAD == 128, int2, int4>::type>::type>::
    type;

template<typename T, int ELEMENTS_PER_WARP_LOAD>
using Copy_t = Copy_half_t<sizeof(T) / sizeof(half) * ELEMENTS_PER_WARP_LOAD>;

template<typename T>
struct Float_as_int_ {
};
template<>
struct Float_as_int_<float> {
    using Type = uint32_t;
};
template<>
struct Float_as_int_<__half> {
    using Type = uint16_t;
};

using kernel_params_float   = Segmented_topk_kernel_params<float, int32_t, 256, 2>;
using kernel_params_float_1 = Segmented_topk_kernel_params<float, int32_t, 256, 1>;
using kernel_params_half    = Segmented_topk_kernel_params<__half, int32_t, 256, 4>;
using kernel_params_half_1  = Segmented_topk_kernel_params<__half, int32_t, 256, 1>;

///////////////////////////////////////////////////////////////////////////////////////////////////

static inline __device__ float to_float(uint32_t src)
{
    return __int_as_float(src);
}

///////////////////////////////////////////////////////////////////////////////////////////////////

static inline __device__ float to_float(uint16_t src)
{
    __half dst = __ushort_as_half(src);
    return __half2float(dst);
}

///////////////////////////////////////////////////////////////////////////////////////////////////

// sort one segment per cta
template<typename T_SCORE, int BLOCK_THREADS, int ELEMENTS_PER_THREAD>
__global__ void blockSortKernel(const T_SCORE* d_keys_in,
                                T_SCORE*       d_keys_out,
                                const int32_t* d_values_in,
                                int32_t*       d_values_out,
                                const int32_t* active_counts,
                                int            num_items_,
                                int            stride_items,
                                int            num_segments)
{
    // Specialize BlockRadixSort for a 1D block
    typedef hipcub::BlockRadixSort<T_SCORE, BLOCK_THREADS, ELEMENTS_PER_THREAD, int32_t> BlockRadixSort;

    // Allocate shared memory for BlockRadixSort
    __shared__ typename BlockRadixSort::TempStorage temp_storage;

    if (blockIdx.x >= num_segments) {
        return;
    }

    int num_items = active_counts[blockIdx.x];  // > num_items_ ? num_items_ : active_counts[blockIdx.x];

    if (num_items == 0) {
        return;
    }

    // Obtain a segment of consecutive items that are blocked across threads
    T_SCORE thread_keys[ELEMENTS_PER_THREAD];
    int32_t thread_values[ELEMENTS_PER_THREAD];

    int32_t block_offset = blockIdx.x * stride_items;
    hipcub::LoadDirectStriped<BLOCK_THREADS>(threadIdx.x, d_keys_out + block_offset, thread_keys, num_items, 0);
    hipcub::LoadDirectStriped<BLOCK_THREADS>(threadIdx.x, d_values_out + block_offset, thread_values, num_items, -1);
    __syncthreads();

    // Collectively sort the keys and values among block threads
    BlockRadixSort(temp_storage).SortDescendingBlockedToStriped(thread_keys, thread_values);

    // Store output in striped fashion
    hipcub::StoreDirectStriped<BLOCK_THREADS>(threadIdx.x, d_keys_out + block_offset, thread_keys, num_items);
    hipcub::StoreDirectStriped<BLOCK_THREADS>(threadIdx.x, d_values_out + block_offset, thread_values, num_items);
}

///////////////////////////////////////////////////////////////////////////////////////////////////

/// block sort kernel
template<typename T_SCORE>
void blockSort(const T_SCORE* d_keys_in,
               T_SCORE*       d_keys_out,
               const int32_t* d_values_in,
               int32_t*       d_values_out,
               const int32_t* active_counts,
               int            num_items,
               int            stride_items,
               int            num_segments,
               hipStream_t   stream)
{
    if (num_items == 0) {
        return;
    }

    int kernel_index  = div_up(num_items, 128) - 1;
    int warps_per_cta = (kernel_index + 1) * 128 / 32;
    if (kernel_index > 7) {
        kernel_index  = 7 + div_up(num_items, 1024) - 1;
        warps_per_cta = 1024 / 32;
    }
    assert(warps_per_cta <= 32);

    dim3 block(warps_per_cta * 32);
    dim3 grid(num_segments);

    using kernel_func = void (*)(const T_SCORE* d_keys_in,
                                 T_SCORE*       d_keys_out,
                                 const int32_t* d_values_in,
                                 int32_t*       d_values_out,
                                 const int32_t* active_counts,
                                 int            num_items,
                                 int            stride_items,
                                 int            num_segments);

    static const kernel_func kernel_funcs[] = {
        &blockSortKernel<T_SCORE, 128, 1>,
        &blockSortKernel<T_SCORE, 256, 1>,
        &blockSortKernel<T_SCORE, 384, 1>,
        &blockSortKernel<T_SCORE, 512, 1>,
        &blockSortKernel<T_SCORE, 640, 1>,
        &blockSortKernel<T_SCORE, 768, 1>,
        &blockSortKernel<T_SCORE, 896, 1>,
        &blockSortKernel<T_SCORE, 1024, 1>,
        &blockSortKernel<T_SCORE, 1024, 2>,
        &blockSortKernel<T_SCORE, 1024, 4>,
        //&blockSortKernel<T_SCORE, 1024, 6>,
    };
    kernel_funcs[kernel_index]<<<grid, block, 0, stream>>>(
        d_keys_in, d_keys_out, d_values_in, d_values_out, active_counts, num_items, stride_items, num_segments);
}

///////////////////////////////////////////////////////////////////////////////////////////////////

struct BlockPrefixCallbackOp {
    // Running prefix
    int running_total;
    // Constructor
    __device__ BlockPrefixCallbackOp(uint32_t running_total): running_total(running_total) {}
    // Callback operator to be entered by the first warp of threads in the block.
    // Thread-0 is responsible for returning a value for seeding the block-wide scan.
    __device__ int operator()(uint32_t block_aggregate)
    {
        uint32_t old_prefix = running_total;
        running_total += block_aggregate;
        return old_prefix;
    }
};

///////////////////////////////////////////////////////////////////////////////////////////////////

#define DO_DEBUG_PRINT 0

// governs the split between regs and smem
constexpr float SMEM_FRACTION = 0.5F;
constexpr float P_EPSILON     = 0.01F;

constexpr int MAX_TOP_K = 3072;
constexpr int WARP_SZ   = 32;

template<typename Kernel_params, int ITEMS_PER_THREAD>
__global__ __launch_bounds__(Kernel_params::BLOCK_THREADS,
                             1) void segmented_top_p_single_pass(TopKPerSegmentParams params)
{
#if DO_DEBUG_PRINT
    constexpr int debug_block_id = 26;
#endif

    using Key_Data_Type     = typename Kernel_params::Key_Data_Type;
    using Int_Key_Data_Type = typename Float_as_int_<Key_Data_Type>::Type;

    // 4 fp16 keys or 2 fp32 keys
    constexpr int                                         KEYS_PER_LDG = Kernel_params::KEYS_PER_LDG;
    typedef Copy_t<Key_Data_Type, WARP_SZ * KEYS_PER_LDG> copy_t;
    union access_t {
        copy_t            v;
        Int_Key_Data_Type x[KEYS_PER_LDG];  // supported size 1,2,4
    };

    constexpr int BLOCK_THREADS = Kernel_params::BLOCK_THREADS;

    constexpr int ITEMS_PER_THREAD_IN_REGS = ITEMS_PER_THREAD * (1.0F - SMEM_FRACTION);
    constexpr int ITEMS_PER_THREAD_IN_SMEM = ITEMS_PER_THREAD - ITEMS_PER_THREAD_IN_REGS;

#if DO_DEBUG_PRINT == 1
    if (blockIdx.x == 0 && threadIdx.x == 0) {
        printf("ITEMS_PER_THREAD, ITEMS_PER_THREAD_IN_REGS, ITEMS_PER_THREAD_IN_SMEM = %d, %d, %d\n",
               ITEMS_PER_THREAD,
               ITEMS_PER_THREAD_IN_REGS,
               ITEMS_PER_THREAD_IN_SMEM);
    }
#endif

    constexpr int          MIN_KEY            = 0;
    constexpr int          ENABLED_PER_THREAD = (ITEMS_PER_THREAD + 32 - 1) / 32;
    extern __shared__ int2 dynamic_smem[];
    int2*                  smem_selected_elements = dynamic_smem;
    Int_Key_Data_Type*     smem_thread_items = reinterpret_cast<Int_Key_Data_Type*>(smem_selected_elements + MAX_TOP_K);

    __shared__ unsigned int smem_selected_count;

    // Specialize BlockScan type for our thread block
    typedef hipcub::BlockScan<uint32_t, BLOCK_THREADS> BlockScan;

    // Specialize BlockScan type for our thread block
    typedef hipcub::BlockReduce<float, BLOCK_THREADS> BlockReduce;
    __shared__ float                               smem_p_sum_total;

    __shared__ union {
        typename BlockScan::TempStorage scan;

        typename BlockReduce::TempStorage reduce;
    } temp_storage;
    // Initialize running total
    BlockPrefixCallbackOp prefix_op(0);

    unsigned int old_selected_count;

    uint32_t segment = blockIdx.y * gridDim.x + blockIdx.x;

    // Preceding TopK has shortcutted this segment
    if (params.gmem_begin_offsets[segment] == params.gmem_end_offsets[segment]) {
        if (threadIdx.x == 0) {
            params.gmem_active_count_per_segment[segment] = 1;
            atomicMax(params.gmem_active_count_total, 1);
        }
        return;
    }

    Int_Key_Data_Type* gmem_src_keys = reinterpret_cast<Int_Key_Data_Type*>(params.gmem_src_keys);
    Int_Key_Data_Type* gmem_dst_keys = reinterpret_cast<Int_Key_Data_Type*>(params.gmem_dst_keys);
    int32_t*           gmem_dst_vals = reinterpret_cast<int32_t*>(params.gmem_dst_vals);

    constexpr int BITS_IN_KEY = sizeof(Key_Data_Type) * 8;

    int items       = params.num_items / params.num_segments;
    int first_index = segment * items;
    gmem_src_keys += first_index;
    gmem_dst_keys += first_index;
    gmem_dst_vals += first_index;

    int               index_limit                            = items;
    Int_Key_Data_Type thread_items[ITEMS_PER_THREAD_IN_REGS] = {0};

    // Load all keys into registers and smem
    const int     lane_id   = threadIdx.x % WARP_SZ;
    const int     warp_id   = threadIdx.x / WARP_SZ;
    constexpr int NUM_WARPS = BLOCK_THREADS / WARP_SZ;

    access_t ZERO;
    for (int i = 0; i < KEYS_PER_LDG; i++) {
        ZERO.x[i] = MIN_KEY;
    }

    // registers
    for (int iter = 0; iter < ITEMS_PER_THREAD_IN_REGS; iter++) {
        int offset         = (iter + threadIdx.x * ITEMS_PER_THREAD);
        thread_items[iter] = (offset < index_limit) ? gmem_src_keys[offset] : MIN_KEY;
    }

    // shared memory
    for (int c = warp_id; c < BLOCK_THREADS; c += NUM_WARPS) {
        for (int iter = lane_id * KEYS_PER_LDG; iter < ITEMS_PER_THREAD_IN_SMEM; iter += WARP_SZ * KEYS_PER_LDG) {
            int      offset = iter + c * ITEMS_PER_THREAD + ITEMS_PER_THREAD_IN_REGS;
            access_t val;
            val.v = (offset < index_limit) ? *reinterpret_cast<copy_t*>(&gmem_src_keys[offset]) : ZERO.v;
            for (int i = 0; i < KEYS_PER_LDG; i++) {
                smem_thread_items[c + (iter + i) * BLOCK_THREADS] = val.x[i];
            }
            // smem_thread_items[c  + iter * BLOCK_THREADS] = (offset < index_limit)? gmem_src_keys[offset] : MIN_KEY;
        }
    }

    Int_Key_Data_Type select_mask = 0;
    Int_Key_Data_Type save_mask   = 0;

    // Int_Key_Data_Type save_bit = 0;
    // set to true when we finish with too few keys, so we go back to last_save_mask one more time
    bool is_last_iter = false;

    if (threadIdx.x == 0) {
        smem_selected_count = 0;
        old_selected_count  = 0;
    }

    // iterate over bits.
    // skip the first two bits,
    // * bit 31 is the sign bit. all values are positive
    // * bit 30 is only set for values >= 2, but the input consists only of values in
    // the range of [0,1]
    constexpr int               START_BIT = BITS_IN_KEY - 1;
    constexpr int               SKIP_BITS = 2;
    constexpr Int_Key_Data_Type ONE       = (Int_Key_Data_Type)1;
    uint32_t                    selected;
    uint32_t                    sc;
    float                       p_sum_total     = 0.0F;
    float                       old_p_sum_total = 0.0F;
    uint32_t                    offset          = 0;
    for (Int_Key_Data_Type bit = START_BIT - SKIP_BITS; true; --bit) {
        __syncthreads();
        Int_Key_Data_Type bit_mask = select_mask | (ONE << bit);

        uint32_t enabled[ENABLED_PER_THREAD] = {0};
        float    thread_sum                  = 0.0F;

        for (int item = 0; item < ITEMS_PER_THREAD_IN_REGS; ++item) {
            // check if all the bits from bit mask are contained in the thread_item. If yes, set respective
            // bit of enabled
            auto     val        = thread_items[item];
            uint32_t is_enabled = uint32_t(((val ^ bit_mask) & bit_mask) == 0);
            // thread_sum += (is_enabled)? to_float(val) : 0.0F;
            thread_sum += is_enabled * to_float(val);
            enabled[item / 32] |= is_enabled << (item % 32);
        }

        for (int item = 0; item < ITEMS_PER_THREAD_IN_SMEM; ++item) {
            int idx = threadIdx.x + item * BLOCK_THREADS;
            // int idx = item + ITEMS_PER_THREAD_IN_SMEM * threadIdx.x;
            auto     val        = smem_thread_items[idx];
            uint32_t is_enabled = uint32_t(((val ^ bit_mask) & bit_mask) == 0);
            // thread_sum += (is_enabled)? to_float(val) : 0.0F;
            thread_sum += is_enabled * to_float(val);
            enabled[(ITEMS_PER_THREAD_IN_REGS + item) / 32] |= is_enabled << ((ITEMS_PER_THREAD_IN_REGS + item) % 32);
        }

        selected = 0;
#pragma unroll
        for (int i = 0; i < ENABLED_PER_THREAD; i++) {
            selected += __popc(enabled[i]);
        }

        float p_sum = BlockReduce(temp_storage.reduce).Sum(thread_sum);

        if (threadIdx.x == 0) {
            p_sum_total += p_sum;
            smem_p_sum_total = p_sum_total;
        }

        __syncthreads();
        p_sum_total = smem_p_sum_total;
        __syncthreads();

        BlockScan(temp_storage.scan).ExclusiveSum(selected, offset, prefix_op);

        if (threadIdx.x == 0) {
            smem_selected_count = prefix_op.running_total;
        }

        __syncthreads();
        sc = smem_selected_count;
        __syncthreads();

        // float p_diff = params.top_p - p_sum_total;
        float p_diff = p_sum_total - params.top_p;

        if ((p_sum_total <= params.top_p + P_EPSILON && p_sum_total > 0)
            || (p_sum_total > params.top_p && sc <= MAX_TOP_K) || (bit == 0 && p_sum_total > 0) || is_last_iter) {

#if DO_DEBUG_PRINT == 1
            __syncthreads();
            if (threadIdx.x == 0 && blockIdx.x == debug_block_id) {
                sc = smem_selected_count;
                printf("bit %d bit_mask %d offset %d (%d, %d), sc = %d, p_sum = %f, p_sum_total = %f\n",
                       bit,
                       bit_mask,
                       offset,
                       blockIdx.x,
                       threadIdx.x,
                       sc,
                       p_sum,
                       p_sum_total);
            }
            __syncthreads();
#endif

            for (int item = 0; item < ITEMS_PER_THREAD_IN_REGS; ++item) {
                // last condition should not trigger with well trained weights, but we will get
                // illegal mewmory access if we do not have one in those rare cases
                if (enabled[item / 32] & (ONE << (item % 32)) && offset < MAX_TOP_K) {
                    smem_selected_elements[offset] =
                        make_int2(thread_items[item], item + threadIdx.x * ITEMS_PER_THREAD);
                    ++offset;
                    thread_items[item] = MIN_KEY;
                }
            }

            for (int item = 0; item < ITEMS_PER_THREAD_IN_SMEM; ++item) {
                if (enabled[(item + ITEMS_PER_THREAD_IN_REGS) / 32] & (ONE << ((item + ITEMS_PER_THREAD_IN_REGS) % 32))
                    && offset < MAX_TOP_K) {
                    int idx = threadIdx.x + item * BLOCK_THREADS;
                    // int idx = item + ITEMS_PER_THREAD_IN_SMEM * threadIdx.x;
                    // if (idx <  params.num_items_per_segment_in_smem)
                    {
                        smem_selected_elements[offset] = make_int2(
                            smem_thread_items[idx], item + threadIdx.x * ITEMS_PER_THREAD + ITEMS_PER_THREAD_IN_REGS);
                        ++offset;
                        smem_thread_items[idx] = MIN_KEY;
                    }
                }
            }
        }

#if DO_DEBUG_PRINT == 1
        if (threadIdx.x == 0 && blockIdx.x == debug_block_id) {
            printf("!!!! bit %d bit_mask %d offset %d (%d, %d), sc = %d, p_sum = %f, p_sum_total = %f\n",
                   bit,
                   bit_mask,
                   offset,
                   blockIdx.x,
                   threadIdx.x,
                   sc,
                   p_sum,
                   p_sum_total);
        }
#endif

        if (p_diff <= P_EPSILON && p_diff >= 0 || (p_sum_total > params.top_p && sc <= MAX_TOP_K) || bit == 0) {

            break;
        }
        // p > top_p
        else if (p_diff > P_EPSILON) {
            // There are too many bits in the current selection
            // Save the current state and go to the next bit
            // If there are not enough items left using the next bit
            // it's necessary to restart here with the current bit not set
            save_mask = bit_mask;
            select_mask |= bit_mask;

            if (threadIdx.x == 0) {
                smem_selected_count = old_selected_count;
                p_sum_total         = old_p_sum_total;

                prefix_op.running_total = old_selected_count;
            }
        }
        else {
            // sc < num_top_k branch
            if (save_mask) {
                select_mask = save_mask;

                save_mask = 0;
            }
            if (threadIdx.x == 0) {
                old_selected_count = smem_selected_count;
                old_p_sum_total    = p_sum_total;
            }
        }
    }

    __syncthreads();

    // store data to global memory
    sc = (p_sum_total < params.top_p) ? params.num_items / params.num_segments : smem_selected_count;
    if (threadIdx.x == 0) {
        params.gmem_active_count_per_segment[segment] = sc;
        atomicMax(params.gmem_active_count_total, sc);
    }
    if (sc >= MAX_TOP_K) {
        return;
    }
    for (int i = threadIdx.x; i < sc; i += blockDim.x) {
        int2 selected_element = smem_selected_elements[i];
        gmem_dst_keys[i]      = selected_element.x;
        gmem_dst_vals[i]      = selected_element.y;
    }
}

///////////////////////////////////////////////////////////////////////////////////////////////////

template<typename Kernel_params>
int getSmemSizeAndCheck(const TopKPerSegmentContext& context, const TopKPerSegmentParams& params)
{
    constexpr int BLOCK_THREADS         = Kernel_params::BLOCK_THREADS;
    using Key_Data_Type                 = typename Kernel_params::Key_Data_Type;
    int           num_items_per_segment = params.num_items / params.num_segments;
    constexpr int ITEMS_INCREMENT       = Kernel_params::ITEMS_INCREMENT;
    int           kernel_index          = div_up(num_items_per_segment, BLOCK_THREADS * ITEMS_INCREMENT) - 1;

    int       smem_size                = MAX_TOP_K * sizeof(int2);
    const int items_per_thread         = (kernel_index + 1) * ITEMS_INCREMENT;
    const int items_per_thread_in_regs = items_per_thread * (1.0F - SMEM_FRACTION);
    const int items_per_thread_in_smem = items_per_thread - items_per_thread_in_regs;

    smem_size += items_per_thread_in_smem * BLOCK_THREADS * sizeof(typename Float_as_int_<Key_Data_Type>::Type);

    int keys_per_ldg = 2 * sizeof(Key_Data_Type) / 2;
    if (smem_size + BLOCK_THREADS * sizeof(float) > (size_t)context.sm_shared_size ||  // dynamic + static memory
        items_per_thread_in_regs + items_per_thread_in_smem != items_per_thread || params.top_p + P_EPSILON > 1.0F
        || items_per_thread_in_regs % keys_per_ldg != 0 || items_per_thread_in_smem % keys_per_ldg != 0
        || num_items_per_segment % keys_per_ldg != 0) {
        return -1;
    }

    return smem_size;
}

///////////////////////////////////////////////////////////////////////////////////////////////////

int getSmemSizeAndCheck(const TopKPerSegmentContext& context,
                        const TopKPerSegmentParams&  params,
                        const DType_t                DT_SCORE)
{
    int num_items_per_segment = params.num_items / params.num_segments;
    if (DT_SCORE == kFLOAT) {
        if (num_items_per_segment % 2 == 0) {
            return getSmemSizeAndCheck<kernel_params_float>(context, params);
        }
        else {
            return getSmemSizeAndCheck<kernel_params_float_1>(context, params);
        }
    }
    else {
        if (num_items_per_segment % 4 == 0) {
            return getSmemSizeAndCheck<kernel_params_half>(context, params);
        }
        else {
            return getSmemSizeAndCheck<kernel_params_half_1>(context, params);
        }
    }
}

///////////////////////////////////////////////////////////////////////////////////////////////////

template<typename Kernel_params>
void segmentedTopPSinglePass_dispatch(const TopKPerSegmentParams&  params,
                                      const TopKPerSegmentContext& context,
                                      hipStream_t                 stream)
{

    constexpr int BLOCK_THREADS = Kernel_params::BLOCK_THREADS;
    using Key_Data_Type         = typename Kernel_params::Key_Data_Type;
    using Value_Data_Type       = typename Kernel_params::Value_Data_Type;

    int num_items_per_segment = params.num_items / params.num_segments;

    constexpr int ITEMS_INCREMENT = Kernel_params::ITEMS_INCREMENT;
    int           kernel_index    = div_up(num_items_per_segment, BLOCK_THREADS * ITEMS_INCREMENT) - 1;

#define KERNEL_RUN(INDEX)                                                                                              \
    {                                                                                                                  \
        if (smem_size > 0)                                                                                             \
            check_cuda_error(                                                                                          \
                hipFuncSetAttribute(reinterpret_cast<const void*>(segmented_top_p_single_pass<Kernel_params), ITEMS_INCREMENT*(INDEX + 1)>,          \
                                     hipFuncAttributeMaxDynamicSharedMemorySize,                                      \
                                     smem_size));                                                                      \
        segmented_top_p_single_pass<Kernel_params, ITEMS_INCREMENT*(INDEX + 1)>                                        \
            <<<grid_dim, Kernel_params::BLOCK_THREADS, smem_size, stream>>>(params);                                   \
    }

    int smem_size = getSmemSizeAndCheck<Kernel_params>(context, params);

    dim3 grid_dim(params.num_segments, 1);

    switch (kernel_index) {
        case 0:
            KERNEL_RUN(0) break;
        case 1:
            KERNEL_RUN(1) break;
        case 2:
            KERNEL_RUN(2) break;
        case 3:
            KERNEL_RUN(3) break;
        case 4:
            KERNEL_RUN(4) break;
        case 5:
            KERNEL_RUN(5) break;
        case 6:
            KERNEL_RUN(6) break;
        case 7:
            KERNEL_RUN(7) break;
        default:
            exit(1);
    }
}

///////////////////////////////////////////////////////////////////////////////////////////////////

template<typename Kernel_params>
void topPPerSegment_dispatch(const TopKPerSegmentContext& context,
                             TopKPerSegmentParams&        params,
                             void*                        temp_storage,
                             size_t&                      temp_storage_bytes,
                             hipStream_t                 stream)
{

    using Key_Data_Type   = typename Kernel_params::Key_Data_Type;
    using Value_Data_Type = typename Kernel_params::Value_Data_Type;

    if (temp_storage == nullptr) {
        if (params.num_segments > 1) {
            hipcub::DeviceSegmentedRadixSort::SortPairsDescending(temp_storage,
                                                               temp_storage_bytes,
                                                               reinterpret_cast<Key_Data_Type*>(params.gmem_src_keys),
                                                               reinterpret_cast<Key_Data_Type*>(params.gmem_dst_keys),
                                                               reinterpret_cast<Value_Data_Type*>(params.gmem_src_vals),
                                                               reinterpret_cast<Value_Data_Type*>(params.gmem_dst_vals),
                                                               params.num_items,
                                                               params.num_segments,
                                                               params.gmem_begin_offsets,
                                                               params.gmem_end_offsets,
                                                               0,
                                                               sizeof(Key_Data_Type) * 8,
                                                               stream);
        }
        else {
            hipcub::DeviceRadixSort::SortPairsDescending(temp_storage,
                                                      temp_storage_bytes,
                                                      reinterpret_cast<Key_Data_Type*>(params.gmem_src_keys),
                                                      reinterpret_cast<Key_Data_Type*>(params.gmem_dst_keys),
                                                      reinterpret_cast<Value_Data_Type*>(params.gmem_src_vals),
                                                      reinterpret_cast<Value_Data_Type*>(params.gmem_dst_vals),
                                                      params.num_items,
                                                      0,
                                                      sizeof(Key_Data_Type) * 8,
                                                      stream);
        }
        temp_storage_bytes = div_up(temp_storage_bytes, 256) * 256;
        // total active counts
        temp_storage_bytes += div_up(sizeof(int), 256) * 256;
        // storage for gmem_end_offsets
        temp_storage_bytes += div_up(sizeof(int) * params.num_segments, 256) * 256;
        return;
    }

    size_t cub_temp_storage_bytes =
        temp_storage_bytes - div_up(sizeof(int), 256) * 256 - div_up(sizeof(int) * params.num_segments, 256) * 256;
    void* cub_temp_storage         = temp_storage;
    params.gmem_active_count_total = reinterpret_cast<int*>((char*)temp_storage + cub_temp_storage_bytes);
    params.gmem_active_count_per_segment =
        reinterpret_cast<int*>((char*)params.gmem_active_count_total + div_up(sizeof(int), 256) * 256);

    int num_items_per_segment = params.num_items / params.num_segments;

    hipMemsetAsync(params.gmem_active_count_total, 0, sizeof(int), stream);
    hipMemsetAsync(params.gmem_dst_keys, 0, params.num_items * sizeof(Key_Data_Type), stream);
    segmentedTopPSinglePass_dispatch<Kernel_params>(params, context, stream);

    int max_num_items = 0;
    hipMemcpyAsync(&max_num_items, params.gmem_active_count_total, sizeof(int), hipMemcpyDeviceToHost, stream);

    hipStreamSynchronize(stream);

    if (max_num_items >= MAX_TOP_K || max_num_items == 0) {
        if (params.num_segments > 1) {
            hipcub::DeviceSegmentedRadixSort::SortPairsDescending(cub_temp_storage,
                                                               cub_temp_storage_bytes,
                                                               reinterpret_cast<Key_Data_Type*>(params.gmem_src_keys),
                                                               reinterpret_cast<Key_Data_Type*>(params.gmem_dst_keys),
                                                               reinterpret_cast<Value_Data_Type*>(params.gmem_src_vals),
                                                               reinterpret_cast<Value_Data_Type*>(params.gmem_dst_vals),
                                                               params.num_items,
                                                               params.num_segments,
                                                               params.gmem_begin_offsets,
                                                               params.gmem_end_offsets,
                                                               0,
                                                               sizeof(Key_Data_Type) * 8,
                                                               stream);
        }
        else {
            hipcub::DeviceRadixSort::SortPairsDescending(cub_temp_storage,
                                                      cub_temp_storage_bytes,
                                                      reinterpret_cast<Key_Data_Type*>(params.gmem_src_keys),
                                                      reinterpret_cast<Key_Data_Type*>(params.gmem_dst_keys),
                                                      reinterpret_cast<Value_Data_Type*>(params.gmem_src_vals),
                                                      reinterpret_cast<Value_Data_Type*>(params.gmem_dst_vals),
                                                      params.num_items,
                                                      0,
                                                      sizeof(Key_Data_Type) * 8,
                                                      stream);
        }
    }
    else {
        // run at max supported value
        blockSort<Key_Data_Type>((const Key_Data_Type*)(params.gmem_dst_keys),
                                 (Key_Data_Type*)(params.gmem_dst_keys),
                                 (const Value_Data_Type*)(params.gmem_dst_vals),
                                 (Value_Data_Type*)(params.gmem_dst_vals),
                                 params.gmem_active_count_per_segment,
                                 max_num_items,
                                 num_items_per_segment,
                                 params.num_segments,
                                 stream);
    }
}

///////////////////////////////////////////////////////////////////////////////////////////////////

int topPPerSegment(const TopKPerSegmentContext& context,
                   TopKPerSegmentParams&        params,
                   const DType_t                DT_SCORE,
                   void*                        temp_storage,
                   size_t&                      temp_storage_bytes,
                   hipStream_t                 stream)
{
    int num_items_per_segment = params.num_items / params.num_segments;
    if (DT_SCORE == kFLOAT) {
        if (num_items_per_segment % 2 == 0) {
            topPPerSegment_dispatch<kernel_params_float>(context, params, temp_storage, temp_storage_bytes, stream);
        }
        else {
            topPPerSegment_dispatch<kernel_params_float_1>(context, params, temp_storage, temp_storage_bytes, stream);
        }
    }
    else {
        if (num_items_per_segment % 4 == 0) {
            topPPerSegment_dispatch<kernel_params_half>(context, params, temp_storage, temp_storage_bytes, stream);
        }
        else {
            topPPerSegment_dispatch<kernel_params_half_1>(context, params, temp_storage, temp_storage_bytes, stream);
        }
    }

    return 0;
}

///////////////////////////////////////////////////////////////////////////////////////////////////

}  // namespace segmented_topp_impl

__global__ void topPInitialize(
    int* topp_id_val_buf, int* topp_offset_buf, int* begin_topp_offset_buf_, const int batch_size, const int n)
{
    int tid = threadIdx.x;
    int bid = blockIdx.x;

    if (bid == 0) {
        for (int i = tid; i < batch_size + 1; i += blockDim.x) {
            topp_offset_buf[i]        = i * n;
            begin_topp_offset_buf_[i] = topp_offset_buf[i];
        }
    }

    int index = tid + bid * blockDim.x;

    while (index < batch_size * n) {
        topp_id_val_buf[index] = index % n;
        index += blockDim.x * gridDim.x;
    }
}

void invokeTopPInitialize(int*         topp_id_val_buf,
                          int*         topp_offset_buf,
                          int*         begin_topp_offset_buf_,
                          const size_t batch_size,
                          const int    n,
                          hipStream_t stream)
{
    // n: the column number of logits_buffer for top_p sampling
    topPInitialize<<<32, 512, 0, stream>>>(topp_id_val_buf, topp_offset_buf, begin_topp_offset_buf_, batch_size, n);
}

template<typename T, int MAX_K, int THREADBLOCK_SIZE>
__launch_bounds__(THREADBLOCK_SIZE) __global__ void topp_beam_topk_kernel(const T*     log_probs,  // prob.
                                                                          int*         topk_tmp_id_buf,
                                                                          T*           topk_tmp_val_buf,
                                                                          const int    vocab_size,
                                                                          int*         offset_buf,
                                                                          int*         begin_offset_buf,
                                                                          const float  top_p,
                                                                          const float* top_ps,
                                                                          const bool*  skip_decode)
{
    int thread_id = threadIdx.x;
    int batch_id  = blockIdx.x;
    if (skip_decode != nullptr && skip_decode[batch_id]) {
        return;
    }
    float p_threshold = (top_ps != nullptr) ? top_ps[batch_id] : top_p;

    typedef hipcub::BlockReduce<TopK<T, MAX_K>, THREADBLOCK_SIZE> BlockReduce;
    __shared__ typename BlockReduce::TempStorage               temp_storage;
    TopK<T, MAX_K>                                             partial;

    const bool IS_FP16   = std::is_same<T, half>::value;
    const T    MAX_T_VAL = (IS_FP16) ? HALF_FLT_MAX : FLT_MAX;

#pragma unroll
    for (int i = 0; i < MAX_K; ++i) {
        partial.p[i] = -1;
        partial.u[i] = -MAX_T_VAL;
    }

#pragma unroll
    for (int elem_id = thread_id; elem_id < vocab_size; elem_id += THREADBLOCK_SIZE) {
        int index = elem_id + batch_id * vocab_size;
        partial.insert(log_probs[index], index);
    }

    TopK<T, MAX_K> total = BlockReduce(temp_storage).Reduce(partial, reduce_topk_op<T, MAX_K>);

    if (thread_id == 0) {
        begin_offset_buf[batch_id] = offset_buf[batch_id];
        T sum_prob                 = (T)(0.0f);

#pragma unroll
        for (int i = 0; i < MAX_K; i++) {
            sum_prob += total.u[i];
        }

        if ((float)sum_prob >= p_threshold) {
            begin_offset_buf[batch_id] += vocab_size;
            int index = batch_id * vocab_size;

#pragma unroll
            for (int i = 0; i < MAX_K; ++i) {
                topk_tmp_id_buf[index + i]  = total.p[i] % vocab_size;
                topk_tmp_val_buf[index + i] = total.u[i];
            }
        }
    }
}

struct BlockPrefixCallbackOp {
    // Running prefix
    float running_total;
    // Constructor
    __device__ BlockPrefixCallbackOp(float running_total): running_total(running_total) {}
    // Callback operator to be entered by the first warp of threads in the block.
    // Thread-0 is responsible for returning a value for seeding the block-wide scan.
    __device__ float operator()(float block_aggregate)
    {
        float old_prefix = running_total;
        running_total += block_aggregate;
        return old_prefix;
    }
};

template<typename T, int BLOCK_SIZE>
__global__ void topp_sampling(T*             sorted_log_probs,
                              int*           sorted_id_vals,
                              int*           ids,
                              int*           sequence_length,
                              bool*          finished_buf,
                              float*         cum_log_probs,
                              float*         output_log_probs,
                              const int*     begin_offset_buf,
                              const int*     offset_buf,
                              const int      vocab_size,
                              hiprandState_t* curandstate,
                              const float    top_p,
                              const float*   top_ps,
                              const int*     end_ids,
                              const int      batch_size,
                              const bool*    skip_decode)
{
    __shared__ int   stop_shared;
    __shared__ float rand_num_s;

    const int tid      = threadIdx.x;
    const int batch_id = blockIdx.x;
    if (skip_decode != nullptr && skip_decode[batch_id]) {
        return;
    }

    constexpr int WARP_SIZE      = 32;
    constexpr int NUM_WARPS      = BLOCK_SIZE / WARP_SIZE;
    const int     lane_id        = threadIdx.x % WARP_SIZE;
    const int     warp_id        = threadIdx.x / WARP_SIZE;
    const float   prob_threshold = (top_ps != nullptr) ? top_ps[batch_id] : top_p;

    if (threadIdx.x == 0) {
        stop_shared = 0;
        rand_num_s  = hiprand_uniform(curandstate + blockIdx.x) * prob_threshold;
    }

    // if begin_offset_buf and offset_buf of sorting have same value,
    // this means that we have find best one in beam_topK_kernel_for_topP
    // and skip the sorting. So, we can skip then during sampling.
    if (begin_offset_buf[batch_id] == offset_buf[batch_id]) {
        if (tid == 0) {
            int offset    = batch_id * vocab_size;
            ids[batch_id] = sorted_id_vals[offset];

            if (cum_log_probs != nullptr || output_log_probs != nullptr) {
                float lprob = logf(sorted_log_probs[offset]);
                if (cum_log_probs != nullptr) {
                    cum_log_probs[batch_id] += lprob;
                }
                if (output_log_probs != nullptr) {
                    output_log_probs[batch_id] = lprob;
                }
            }
            if (sequence_length != nullptr && finished_buf != nullptr) {
                sequence_length[batch_id] =
                    finished_buf[batch_id] ? sequence_length[batch_id] : sequence_length[batch_id] + 1;
                finished_buf[batch_id] = ids[batch_id] == end_ids[batch_id] ? 1 : 0;
            }
        }
        return;
    }

    typedef hipcub::BlockScan<float, BLOCK_SIZE>  BlockScan;
    __shared__ typename BlockScan::TempStorage temp_storage;
    __shared__ uint32_t                        selected_shared[NUM_WARPS];
    // Initialize running total
    BlockPrefixCallbackOp prefix_op(0);

    if (lane_id == 0) {
        selected_shared[warp_id] = 0;
    }

    __syncthreads();

    int offset          = batch_id * vocab_size;
    ids[batch_id]       = sorted_id_vals[offset];
    int   end           = ((vocab_size + BLOCK_SIZE - 1) / BLOCK_SIZE) * BLOCK_SIZE;
    int   i_active      = 0;
    float thread_offset = 0;
    for (int i = tid; i < end; i += BLOCK_SIZE) {
        float thread_count = (i < vocab_size) ? (float)sorted_log_probs[offset + i] : 0.f;
        BlockScan(temp_storage).InclusiveSum(thread_count, thread_offset, prefix_op);

        uint32_t active_mask = __ballot_sync(0xFFFFFFFF, rand_num_s <= thread_offset);

        i_active = i;
        if (active_mask != 0) {
            if (lane_id == 0) {
                atomicAdd(&stop_shared, 1);
                selected_shared[warp_id] = active_mask;
            }
        }
        __syncthreads();
        if (stop_shared > 0) {
            break;
        }
    };

    // select first active warp
    bool skip = (selected_shared[warp_id] > 0) ? false : true;
    for (int i = 0; i < warp_id; i++) {
        if (selected_shared[i] != 0) {
            skip = true;
        }
    }
    if (!skip) {
        int active_lane_id = WARP_SIZE - __popc(selected_shared[warp_id]);
        if (lane_id == active_lane_id) {
            ids[batch_id] = sorted_id_vals[offset + i_active];
            if (cum_log_probs != nullptr || output_log_probs != nullptr) {
                float lprob = logf(sorted_log_probs[offset + i_active]);
                if (cum_log_probs != nullptr) {
                    cum_log_probs[batch_id] += lprob;
                }
                if (output_log_probs != nullptr) {
                    output_log_probs[batch_id] = lprob;
                }
            }
            if (sequence_length != nullptr && finished_buf != nullptr) {
                sequence_length[batch_id] =
                    finished_buf[batch_id] ? sequence_length[batch_id] : sequence_length[batch_id] + 1;
                finished_buf[batch_id] = ids[batch_id] == end_ids[batch_id] ? 1 : 0;
            }
        }
    }
}

template<typename T>
void invokeBatchTopPSampling(void*           workspace,
                             size_t&         workspace_size,
                             size_t&         cub_temp_storage_size,
                             int*            output_ids,
                             int*            sequence_length,
                             bool*           finished_buf,
                             float*          cum_log_probs,
                             float*          output_log_probs,
                             const T*        log_probs,
                             const int*      id_vals,
                             int*            offset_buf,
                             int*            begin_offset_buf,
                             hiprandState_t*  curandstate,
                             const int       batch_size,
                             const size_t    vocab_size_padded,
                             const int*      end_ids,
                             const float     max_top_p,
                             const float*    top_ps,
                             hipStream_t    stream,
                             hipDeviceProp_t* cuda_device_prop,
                             const bool*     skip_decode)
{
    // Here, we put batch size as an argument because the batch size of initialization
    // and inference may be different due to pipeline parallelism.
    const int vocab_size = vocab_size_padded;
    const int block_size = 256;

    size_t sorted_log_prob_buf_size = batch_size * vocab_size * sizeof(T);    // type T
    size_t sorted_id_vals_buf_size  = batch_size * vocab_size * sizeof(int);  // type int
    sorted_log_prob_buf_size        = div_up(sorted_log_prob_buf_size, 256) * 256;
    sorted_id_vals_buf_size         = div_up(sorted_id_vals_buf_size, 256) * 256;

    void* cub_temp_storage = workspace;
    T*    sorted_log_probs = (T*)((char*)cub_temp_storage + cub_temp_storage_size);
    int*  sorted_id_vals   = (int*)((char*)sorted_log_probs + sorted_log_prob_buf_size);

    bool do_radix_sort = (ENABLE_SINGLE_PASS_TOP_P == 0 || max_top_p >= SINGLE_PASS_THRESHOLD);
    int  smem_size     = -1;

    segmented_topp_impl::TopKPerSegmentContext context;
    segmented_topp_impl::TopKPerSegmentParams  params;
    segmented_topp_impl::DType_t               dataTypeKind =
        (std::is_same<T, float>::value) ? segmented_topp_impl::kFLOAT : segmented_topp_impl::kHALF;

    if (!do_radix_sort) {
        FT_CHECK(cuda_device_prop != nullptr);
        memset(&context, 0, sizeof(context));
        context.sm_count       = cuda_device_prop->multiProcessorCount;
        context.sm_shared_size = cuda_device_prop->sharedMemPerMultiprocessor;
        context.sm_version     = cuda_device_prop->major * 100 + cuda_device_prop->minor * 10;

        memset(&params, 0, sizeof(params));
        params.gmem_src_keys        = reinterpret_cast<void*>(const_cast<T*>(log_probs));
        params.gmem_dst_keys        = sorted_log_probs;
        params.gmem_src_vals        = reinterpret_cast<void*>(const_cast<int*>(id_vals));
        params.gmem_dst_vals        = reinterpret_cast<void*>(sorted_id_vals);
        params.gmem_begin_offsets   = begin_offset_buf;
        params.gmem_end_offsets     = offset_buf + 1;
        params.workspace            = nullptr;
        params.num_items            = vocab_size * batch_size;
        params.num_segments         = batch_size;
        params.top_p                = max_top_p;
        params.confidence_threshold = 0.0F;

        smem_size     = getSmemSizeAndCheck(context, params, dataTypeKind);
        do_radix_sort = smem_size < 0;
    }

    if (do_radix_sort) {
        if (workspace == nullptr) {
            check_cuda_error(
                hipcub::DeviceSegmentedRadixSort::SortPairsDescending(nullptr,
                                                                   cub_temp_storage_size,
                                                                   log_probs,
                                                                   (T*)nullptr,
                                                                   id_vals,
                                                                   (int*)nullptr,
                                                                   vocab_size * batch_size,
                                                                   batch_size,
                                                                   begin_offset_buf,
                                                                   offset_buf + 1,
                                                                   0,              // begin_bit
                                                                   sizeof(T) * 8,  // end_bit = sizeof(KeyT) * 8
                                                                   stream));       // hipStream_t
            cub_temp_storage_size = div_up(cub_temp_storage_size, 256) * 256;
            workspace_size        = sorted_log_prob_buf_size + sorted_id_vals_buf_size + cub_temp_storage_size;
            return;
        }

        topp_beam_topk_kernel<T, 1, block_size><<<batch_size, block_size, 0, stream>>>(log_probs,
                                                                                       sorted_id_vals,
                                                                                       sorted_log_probs,
                                                                                       vocab_size,
                                                                                       offset_buf,
                                                                                       begin_offset_buf,
                                                                                       max_top_p,
                                                                                       top_ps,
                                                                                       skip_decode);

        check_cuda_error(
            hipcub::DeviceSegmentedRadixSort::SortPairsDescending(cub_temp_storage,
                                                               cub_temp_storage_size,
                                                               log_probs,
                                                               sorted_log_probs,
                                                               id_vals,
                                                               sorted_id_vals,
                                                               vocab_size * batch_size,
                                                               batch_size,
                                                               begin_offset_buf,
                                                               offset_buf + 1,
                                                               0,              // begin_bit
                                                               sizeof(T) * 8,  // end_bit = sizeof(KeyT) * 8
                                                               stream));       // hipStream_t
    }
    else {
        if (workspace == nullptr) {
            segmented_topp_impl::topPPerSegment(
                context, params, dataTypeKind, cub_temp_storage, cub_temp_storage_size, stream);
            workspace_size = sorted_log_prob_buf_size + sorted_id_vals_buf_size + cub_temp_storage_size;
            return;
        }
        else {
            topp_beam_topk_kernel<T, 1, block_size><<<batch_size, block_size, 0, stream>>>(log_probs,
                                                                                           sorted_id_vals,
                                                                                           sorted_log_probs,
                                                                                           vocab_size,
                                                                                           offset_buf,
                                                                                           begin_offset_buf,
                                                                                           max_top_p,
                                                                                           top_ps,
                                                                                           skip_decode);
            segmented_topp_impl::topPPerSegment(
                context, params, dataTypeKind, cub_temp_storage, cub_temp_storage_size, stream);
        }
    }

    constexpr int SAMPLING_BLOCK_SIZE = 256;
    dim3          grid(batch_size);
    topp_sampling<T, SAMPLING_BLOCK_SIZE><<<grid, SAMPLING_BLOCK_SIZE, 0, stream>>>(sorted_log_probs,
                                                                                    sorted_id_vals,
                                                                                    output_ids,
                                                                                    sequence_length,
                                                                                    finished_buf,
                                                                                    cum_log_probs,
                                                                                    output_log_probs,
                                                                                    begin_offset_buf,
                                                                                    offset_buf + 1,
                                                                                    vocab_size,
                                                                                    curandstate,
                                                                                    max_top_p,
                                                                                    top_ps,
                                                                                    end_ids,
                                                                                    batch_size,
                                                                                    skip_decode);
}

template void invokeBatchTopPSampling(void*           workspace,
                                      size_t&         workspace_size,
                                      size_t&         cub_temp_storage_size,
                                      int*            output_ids,
                                      int*            sequence_length,
                                      bool*           finished_buf,
                                      float*          cum_log_probs,
                                      float*          output_log_probs,
                                      const float*    log_probs,
                                      const int*      id_vals,
                                      int*            offset_buf,
                                      int*            begin_offset_buf,
                                      hiprandState_t*  curandstate,
                                      const int       batch_size,
                                      const size_t    vocab_size_padded,
                                      const int*      end_ids,
                                      const float     max_top_p,
                                      const float*    top_ps,
                                      hipStream_t    stream,
                                      hipDeviceProp_t* cuda_device_prop,
                                      const bool*     skip_decode);

template void invokeBatchTopPSampling(void*           workspace,
                                      size_t&         workspace_size,
                                      size_t&         cub_temp_storage_size,
                                      int*            output_ids,
                                      int*            sequence_length,
                                      bool*           finished_buf,
                                      float*          cum_log_probs,
                                      float*          output_log_probs,
                                      const half*     log_probs,
                                      const int*      id_vals,
                                      int*            offset_buf,
                                      int*            begin_offset_buf,
                                      hiprandState_t*  curandstate,
                                      const int       batch_size,
                                      const size_t    vocab_size_padded,
                                      const int*      end_ids,
                                      const float     max_top_p,
                                      const float*    top_ps,
                                      hipStream_t    stream,
                                      hipDeviceProp_t* cuda_device_prop,
                                      const bool*     skip_decode);

template<typename T>
void invokeTopPSampling(void*           workspace,
                        size_t&         workspace_size,
                        size_t&         cub_temp_storage_size,
                        int*            output_ids,
                        int*            sequence_length,
                        bool*           finished_buf,
                        float*          cum_log_probs,
                        float*          output_log_probs,
                        const T*        log_probs,
                        const int*      id_vals,
                        int*            offset_buf,
                        int*            begin_offset_buf,
                        hiprandState_t*  curandstate,
                        const int       batch_size,
                        const size_t    vocab_size_padded,
                        const int*      end_ids,
                        const float     top_p,
                        hipStream_t    stream,
                        hipDeviceProp_t* cuda_device_prop,
                        const bool*     skip_decode)
{
    invokeBatchTopPSampling(workspace,
                            workspace_size,
                            cub_temp_storage_size,
                            output_ids,
                            sequence_length,
                            finished_buf,
                            cum_log_probs,
                            output_log_probs,
                            log_probs,
                            id_vals,
                            offset_buf,
                            begin_offset_buf,
                            curandstate,
                            batch_size,
                            vocab_size_padded,
                            end_ids,
                            top_p,
                            nullptr,
                            stream,
                            cuda_device_prop,
                            skip_decode);
}

template void invokeTopPSampling(void*           workspace,
                                 size_t&         workspace_size,
                                 size_t&         cub_temp_storage_size,
                                 int*            output_ids,
                                 int*            sequence_length,
                                 bool*           finished_buf,
                                 float*          cum_log_probs,
                                 float*          output_log_probs,
                                 const float*    log_probs,
                                 const int*      id_vals,
                                 int*            offset_buf,
                                 int*            begin_offset_buf,
                                 hiprandState_t*  curandstate,
                                 const int       batch_size,
                                 const size_t    vocab_size_padded,
                                 const int*      end_ids,
                                 const float     top_p,
                                 hipStream_t    stream,
                                 hipDeviceProp_t* cuda_device_prop,
                                 const bool*     skip_decode);

template void invokeTopPSampling(void*           workspace,
                                 size_t&         workspace_size,
                                 size_t&         cub_temp_storage_size,
                                 int*            output_ids,
                                 int*            sequence_length,
                                 bool*           finished_buf,
                                 float*          cum_log_probs,
                                 float*          output_log_probs,
                                 const half*     log_probs,
                                 const int*      id_vals,
                                 int*            offset_buf,
                                 int*            begin_offset_buf,
                                 hiprandState_t*  curandstate,
                                 const int       batch_size,
                                 const size_t    vocab_size_padded,
                                 const int*      end_ids,
                                 const float     top_p,
                                 hipStream_t    stream,
                                 hipDeviceProp_t* cuda_device_prop,
                                 const bool*     skip_decode);

template<typename T>
__global__ void
addBiasSoftMax(T* logits, const T* bias, const int* end_ids, const bool* finished, const int n_padded, const int n)
{
    int  bid    = blockIdx.x;
    bool finish = (finished != nullptr) ? finished[bid] : false;
    int  offset = bid * n_padded;

    float            max_val   = -1 * FLT_MAX;
    const bool       IS_FP16   = std::is_same<T, half>::value;
    const T          MAX_T_VAL = (IS_FP16) ? HALF_FLT_MAX : FLT_MAX;
    __shared__ float s_max_val;
    __shared__ float s_sum_val;

    for (int tid = threadIdx.x; tid < n_padded; tid += blockDim.x) {
        if (tid < n) {
            if (finish) {
                logits[offset + tid] = (tid == end_ids[bid]) ? MAX_T_VAL : -MAX_T_VAL;
            }
            else {
                T bias_val = (bias != nullptr) ? bias[tid] : (T)0.0f;
                logits[offset + tid] += bias_val;
            }
        }
        else {
            logits[offset + tid] = -MAX_T_VAL;
        }
        max_val = max(max_val, (float)logits[offset + tid]);
    }

    max_val = blockReduceMax<float>((float)max_val);
    if (threadIdx.x == 0) {
        s_max_val = max_val;
    }
    __syncthreads();

    float sum_val = 0.0f;
    for (int tid = threadIdx.x; tid < n_padded; tid += blockDim.x) {
        logits[offset + tid] = __expf((float)logits[offset + tid] - s_max_val);
        sum_val += (float)logits[offset + tid];
    }

    sum_val = blockReduceSum<float>(sum_val);
    if (threadIdx.x == 0) {
        s_sum_val = sum_val;
    }
    __syncthreads();

    for (int tid = threadIdx.x; tid < n_padded; tid += blockDim.x) {
        logits[offset + tid] = ((float)logits[offset + tid] / (s_sum_val + 1e-6f));
    }
}

template<typename T>
void invokeAddBiasSoftMax(T*           logits,
                          const T*     bias,
                          const int*   end_ids,
                          const bool*  finished,
                          const int    m,
                          const int    n_padded,
                          const int    n,
                          hipStream_t stream)
{
    dim3 grid(m);
    dim3 block(min(n, 1024));
    /*n is the vocab_size, e.g., 30000, 7000.... vocab_size is usually very big. */
    addBiasSoftMax<<<grid, block, 0, stream>>>(logits, bias, end_ids, finished, n_padded, n);
}

template void invokeAddBiasSoftMax(float*       logits,
                                   const float* bias,
                                   const int*   end_ids,
                                   const bool*  finished,
                                   const int    m,
                                   const int    n_padded,
                                   const int    n,
                                   hipStream_t stream);

template void invokeAddBiasSoftMax(half*        logits,
                                   const half*  bias,
                                   const int*   end_ids,
                                   const bool*  finished,
                                   const int    m,
                                   const int    n_padded,
                                   const int    n,
                                   hipStream_t stream);

__global__ void computeToppDecay(float*         runtime_top_p,
                                 const float*   runtime_initial_top_p,
                                 const int*     output_ids,
                                 const float*   top_p_decay,
                                 const float*   top_p_min,
                                 const int32_t* top_p_reset_ids,
                                 const int      local_batch_size)
{
    /**
     * @brief Compute the topp decay by https://arxiv.org/pdf/2206.04624.pdf
     *        In short, the formula is
     *          runtime_top_p = max(runtime_top_p * top_p_decay, top_p_min)
     *        If generating the top_p_reset_ids, then reset the runtime_top_p.
     *
     * \param runtime_top_p          [local_batch_size]
     * \param runtime_initial_top_p  [local_batch_size]
     * \param output_ids             [local_batch_size]
     * \param top_p_decay            [local_batch_size]
     * \param top_p_min              [local_batch_size]
     * \param top_p_reset_ids         [local_batch_size]
     * \param local_batch_size
     *
     */

    int idx = blockDim.x * blockIdx.x + threadIdx.x;
    if (output_ids[idx] == top_p_reset_ids[idx]) {
        runtime_top_p[idx] = runtime_initial_top_p[idx];
    }
    else {
        runtime_top_p[idx] = max(runtime_top_p[idx] * top_p_decay[idx], top_p_min[idx]);
    }
}

void invokeComputeToppDecay(float*         runtime_top_p,
                            const float*   runtime_initial_top_p,
                            const int*     output_ids,
                            const float*   top_p_decay,
                            const float*   top_p_min,
                            const int32_t* top_p_reset_ids,
                            const int      local_batch_size,
                            hipStream_t   stream)
{
    dim3 block(min(local_batch_size, 512));
    dim3 grid((local_batch_size + block.x - 1) / block.x);
    computeToppDecay<<<grid, block, 0, stream>>>(
        runtime_top_p, runtime_initial_top_p, output_ids, top_p_decay, top_p_min, top_p_reset_ids, local_batch_size);
}

}  // namespace fastertransformer
