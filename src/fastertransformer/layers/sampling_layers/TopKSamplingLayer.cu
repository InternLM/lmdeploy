#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2019-2023, NVIDIA CORPORATION.  All rights reserved.
 * Copyright (c) 2021, NAVER Corp.  Authored by CLOVA.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <float.h>

#include "src/fastertransformer/kernels/sampling_topk_kernels.h"
#include "src/fastertransformer/kernels/sampling_topp_kernels.h"
#include "src/fastertransformer/layers/sampling_layers/TopKSamplingLayer.h"
#include "src/fastertransformer/utils/logger.h"
#include "src/fastertransformer/utils/memory_utils.h"

namespace fastertransformer {

template<uint TOP_K_MAX>
__global__ void setup_topk_runtime_args(int    batch_size,
                                        uint   top_k,
                                        uint*  top_ks,
                                        int    top_ks_size,
                                        float  top_p,
                                        float* top_ps,
                                        int    top_ps_size,
                                        bool*  skip_decode)
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    for (int i = index; i < batch_size; i += gridDim.x * blockDim.x) {
        uint  k = top_ks_size > 1 ? top_ks[i] : top_k;
        float p = top_ps_size > 1 ? top_ps[i] : top_p;
        if (k == 0 && p == 0.0f) {
            // FT's topp implementation does not support topp = 0.0f, but it equivalent to greedy search.
            // So, we set the topk = 1 as an alternative solution.
            k = 1;
        }
        if (k > 0 && p == 0.0f) {
            // for compatibility <= FT5.0.
            // This case corresponds to the old topk sampling, which is equivalent to
            // the old topk_topp sampling with topp=1.0f. TopKSamplingLayer and
            // TopKTopPSamplingLayer are now merged by TopKSamplingLayer. Thus, we
            // replace the case topk>0 and topp=0.0f by topk>0 and topp=1.0f for the
            // compatibility.
            p = 1.0f;
        }
        // Clip k value. A topk sampling kernel supports up to TOP_K_MAX=64.
        top_ks[i] = k > TOP_K_MAX ? TOP_K_MAX : k;
        if (k > TOP_K_MAX) {
            printf("[WARNING] topk (%d) is larger than max supported number (%d) for token %d"
                   " clip to max supported number %d. \n",
                   k,
                   TOP_K_MAX,
                   i,
                   top_ks[i]);
        }
        // Clip p value if it is out of range. range = [0.0, 1.0].
        top_ps[i] = p < 0.0f ? 0.0f : (p > 1.0f ? 1.0f : p);
        if (p < 0.0f || p > 1.0f) {
            printf("[WARNING] topp (%f) is out of range ([0.0, 1.0f]) for token %d"
                   " clip to closest number %f.\n",
                   p,
                   i,
                   top_ps[i]);
        }
        skip_decode[i] = k == 0;
    }
}

template<typename T>
void TopKSamplingLayer<T>::allocateBuffer()
{
    FT_CHECK(false);
}

template<typename T>
void TopKSamplingLayer<T>::allocateBuffer(size_t batch_size, Tensor top_k, Tensor top_p)
{
    FT_LOG_DEBUG(__PRETTY_FUNCTION__);
    BaseSamplingLayer<T>::allocateBuffer(batch_size, top_k, top_p);
    uint max_top_k = top_k.size() > 0 ? top_k.max<uint>() : 1;
    if (max_top_k == 0) {
        // for safety. TopKSamplingLayer handles a case of top_k=0 and top_p=0 as
        // a greedy decode, i.e. top_k=1, although such case has max_top_k=0.
        max_top_k = 1;
    }
    invokeTopKSampling<T>(nullptr,
                          sampling_workspace_size_,
                          nullptr,
                          nullptr,
                          nullptr,
                          nullptr,
                          nullptr,
                          nullptr,
                          nullptr,
                          max_top_k,
                          1.0f,
                          vocab_size_padded_,
                          nullptr,
                          stream_,
                          batch_size,
                          skip_decode_buf_);
    sampling_workspace_ = allocator_->reMalloc(sampling_workspace_, sampling_workspace_size_, false);
    runtime_top_k_buf_ =
        reinterpret_cast<uint*>(allocator_->reMalloc(runtime_top_k_buf_, sizeof(uint) * batch_size, false));
    runtime_top_p_buf_ =
        reinterpret_cast<float*>(allocator_->reMalloc(runtime_top_p_buf_, sizeof(float) * batch_size, false));
    is_allocate_buffer_ = true;
}

template<typename T>
void TopKSamplingLayer<T>::freeBuffer()
{
    FT_LOG_DEBUG(__PRETTY_FUNCTION__);
    if (is_allocate_buffer_) {
        allocator_->free((void**)(&sampling_workspace_));
        allocator_->free((void**)(&runtime_top_k_buf_));
        allocator_->free((void**)(&runtime_top_p_buf_));
    }
    BaseSamplingLayer<T>::freeBuffer();
    is_allocate_buffer_ = false;
}

template<typename T>
void TopKSamplingLayer<T>::setup(const size_t batch_size, const size_t beam_width, TensorMap* runtime_args)
{
    // Setup runtime topk and topp arguments.
    //
    // runtime_args:
    //     runtime_top_k [1] or [batch_size] on cpu, optional, uint.
    //     runtime_top_p [1] or [batch_size] on cpu, optional, float.
    //     temperature [1] or [batch_size] on cpu, optional
    //     repetition_penalty [1] or [batch_size] on cpu, optional
    FT_LOG_DEBUG(__PRETTY_FUNCTION__);
    BaseSamplingLayer<T>::setup(batch_size, beam_width, runtime_args);

    uint         tmp_top_k     = 0;
    const Tensor runtime_top_k = runtime_args->isExist("runtime_top_k") ?
                                     runtime_args->at("runtime_top_k") :
                                     Tensor(MEMORY_CPU, TYPE_UINT32, {1}, &tmp_top_k);
    const Tensor runtime_top_p = runtime_args->isExist("runtime_top_p") ? runtime_args->at("runtime_top_p") : Tensor();
    const size_t runtime_top_k_size = runtime_top_k.size();
    const size_t runtime_top_p_size = runtime_top_p.size();

    uint  top_k = runtime_top_k.max<uint>();
    float top_p = runtime_top_p_size == 0 ? 0.0f : runtime_top_p.getVal<float>();

    if (runtime_top_k_size > 1) {
        FT_CHECK_WITH_INFO(
            runtime_top_k.size() == batch_size,
            fmtstr("runtime_top_k.size() (%d) == batch_size (%d) is not satisfied!", runtime_top_k.size(), batch_size));
        cudaAutoCpy(runtime_top_k_buf_, runtime_top_k.getPtr<uint>(), batch_size, stream_);
    }
    if (runtime_top_p_size > 1) {
        FT_CHECK_WITH_INFO(
            runtime_top_p.size() == batch_size,
            fmtstr("runtime_top_p.size() (%d) == batch_size (%d) is not satisfied!", runtime_top_p.size(), batch_size));
        cudaAutoCpy(runtime_top_p_buf_, runtime_top_p.getPtr<float>(), batch_size, stream_);
    }

    dim3 block(std::min((int)batch_size, 256));
    dim3 grid(div_up((int)batch_size, (int)block.x));
    // support top_k up to 1024.
    setup_topk_runtime_args<1024><<<grid, block, 0, stream_>>>(batch_size,
                                                               top_k,
                                                               runtime_top_k_buf_,
                                                               runtime_top_k_size,
                                                               top_p,
                                                               runtime_top_p_buf_,
                                                               runtime_top_p_size,
                                                               skip_decode_buf_);
    cudaAutoCpy(skip_decode_, skip_decode_buf_, batch_size, stream_);
    uint* runtime_top_ks = new uint[batch_size];
    cudaAutoCpy(runtime_top_ks, runtime_top_k_buf_, batch_size, stream_);
    runtime_max_top_k_ = static_cast<int>(*std::max_element(runtime_top_ks, runtime_top_ks + batch_size));
    delete[] runtime_top_ks;
}

template<typename T>
void TopKSamplingLayer<T>::runSampling(TensorMap* output_tensors, TensorMap* input_tensors)
{
    // input_tensors:
    //      logits [local_batch_size, vocab_size_padded]
    //      embedding_bias [vocab_size_padded], optional
    //      step [1] on cpu
    //      max_input_length [1] on cpu
    //      input_lengths [local_batch_size], optional
    //      ite [1] on cpu

    // output_tensors:
    //      output_ids [max_seq_len, batch_size]
    //      finished [local_batch_size], optional
    //      sequence_length [local_batch_size], optional
    //      cum_log_probs [batch_size], must be float*, optional
    //          The cumultative log probability of generated tokens.
    //      output_log_probs [local_batch_size], must be float*, optional
    //          The log probs at the current step.

    FT_LOG_DEBUG(__PRETTY_FUNCTION__);
    FT_CHECK(input_tensors->size() >= 4);
    FT_CHECK(output_tensors->size() >= 1);

    const int batch_size       = output_tensors->at("output_ids").shape[1];
    const int local_batch_size = input_tensors->at("logits").shape[0];
    const int ite              = input_tensors->at("ite").getVal<int>();
    const int step             = input_tensors->at("step").getVal<int>();

    // in case of skip any, the logit value is already copied and processed.
    T* logits = !skip_any_ ? input_tensors->at("logits").getPtr<T>() : runtime_logits_buf_;

    invokeAddBiasEndMask(logits,
                         (T*)(nullptr),
                         input_tensors->at("end_id").getPtr<const int>(),
                         output_tensors->at("finished", Tensor{MEMORY_GPU, TYPE_INVALID, {}, nullptr}).getPtr<bool>(),
                         local_batch_size,
                         vocab_size_,
                         vocab_size_padded_,
                         stream_);
    sync_check_cuda_error();

    float* cum_log_probs =
        output_tensors->isExist("cum_log_probs") ? output_tensors->at("cum_log_probs").getPtr<float>() : nullptr;
    float* output_log_probs =
        output_tensors->isExist("output_log_probs") ? output_tensors->at("output_log_probs").getPtr<float>() : nullptr;

    if (cum_log_probs != nullptr || output_log_probs != nullptr) {
        invokeAddBiasSoftMax(
            logits,
            (T*)(nullptr),
            input_tensors->at("end_id").getPtr<int>(),
            output_tensors->at("finished", Tensor{MEMORY_GPU, TYPE_INVALID, {}, nullptr}).getPtr<bool>(),
            local_batch_size,
            vocab_size_padded_,
            vocab_size_,
            stream_);
        sync_check_cuda_error();
    }

    invokeBatchTopKSampling(
        sampling_workspace_,
        sampling_workspace_size_,
        logits,
        output_tensors->at("output_ids").getPtrWithOffset<int>(step * batch_size + ite * local_batch_size),
        output_tensors->at("sequence_length", Tensor{MEMORY_GPU, TYPE_INVALID, {}, nullptr}).getPtr<int>(),
        output_tensors->at("finished", Tensor{MEMORY_GPU, TYPE_INVALID, {}, nullptr}).getPtr<bool>(),
        cum_log_probs,
        output_log_probs,
        curandstate_buf_ + ite * local_batch_size,
        (int)runtime_max_top_k_,  // useless because runtime_top_k_buf_ is never nullptr. Keep for legacy.
        (int*)(runtime_top_k_buf_ + ite * local_batch_size),
        1.0f,  // useless because runtime_top_p_buf_ is never nullptr. Keep for legacy.
        runtime_top_p_buf_ + ite * local_batch_size,
        vocab_size_padded_,
        input_tensors->at("end_id").getPtr<int>(),
        stream_,
        local_batch_size,
        skip_decode_buf_ + ite * local_batch_size);
    sync_check_cuda_error();
}

template<typename T>
TopKSamplingLayer<T>::TopKSamplingLayer(size_t             max_batch_size,
                                        size_t             vocab_size,
                                        size_t             vocab_size_padded,
                                        int                end_id,
                                        size_t             top_k,
                                        unsigned long long random_seed,
                                        float              temperature,
                                        float              len_penalty,
                                        float              repetition_penalty,
                                        hipStream_t       stream,
                                        cublasMMWrapper*   cublas_wrapper,
                                        IAllocator*        allocator,
                                        bool               is_free_buffer_after_forward):
    BaseSamplingLayer<T>(max_batch_size,
                         vocab_size,
                         vocab_size_padded,
                         end_id,
                         top_k,
                         0.0f,
                         random_seed,
                         temperature,
                         len_penalty,
                         repetition_penalty,
                         stream,
                         cublas_wrapper,
                         allocator,
                         is_free_buffer_after_forward,
                         nullptr)
{
}

template<typename T>
TopKSamplingLayer<T>::TopKSamplingLayer(TopKSamplingLayer<T> const& top_k_sampling_layer):
    BaseSamplingLayer<T>(top_k_sampling_layer)
{
}

template<typename T>
TopKSamplingLayer<T>::~TopKSamplingLayer()
{
    FT_LOG_DEBUG(__PRETTY_FUNCTION__);
    freeBuffer();
}

template class TopKSamplingLayer<float>;
template class TopKSamplingLayer<half>;

}  // namespace fastertransformer
