#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2022-2023, NVIDIA CORPORATION.  All rights reserved.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "cublasFP8MMWrapper.h"
#include "cuda_utils.h"

namespace fastertransformer {

#define CUBLAS_WORKSPACE_1MB 1048576
cublasFP8MMWrapper::cublasFP8MMWrapper(hipblasLtHandle_t cublaslt_handle,
                                       hipStream_t     stream,
                                       cublasAlgoMap*   cublas_algo_map,
                                       std::mutex*      mu,
                                       IAllocator*      allocator):
    cublasMMWrapper(nullptr, cublaslt_handle, stream, cublas_algo_map, mu, allocator)
{
    FT_LOG_DEBUG(__PRETTY_FUNCTION__);
    FT_CHECK_WITH_INFO(allocator != nullptr, "must pass allocator to cublasFP8MMWrapper");
    cublasVersionCheck();

    if (allocator_ != nullptr) {
        cublas_workspace_qgemm_ = allocator_->reMalloc(cublas_workspace_qgemm_, CUBLAS_WORKSPACE_1MB, true);
    }
}

cublasFP8MMWrapper::cublasFP8MMWrapper(hipblasHandle_t   cublas_handle,
                                       hipblasLtHandle_t cublaslt_handle,
                                       hipStream_t     stream,
                                       cublasAlgoMap*   cublas_algo_map,
                                       std::mutex*      mu,
                                       IAllocator*      allocator):
    cublasMMWrapper(cublas_handle, cublaslt_handle, stream, cublas_algo_map, mu, allocator)
{
    FT_LOG_DEBUG(__PRETTY_FUNCTION__);
    FT_CHECK_WITH_INFO(allocator != nullptr, "must pass allocator to cublasFP8MMWrapper");
    cublasVersionCheck();
    if (allocator_ != nullptr) {
        cublas_workspace_qgemm_ = allocator_->reMalloc(cublas_workspace_qgemm_, CUBLAS_WORKSPACE_1MB, true);
    }
}

cublasFP8MMWrapper::~cublasFP8MMWrapper()
{
    FT_LOG_DEBUG(__PRETTY_FUNCTION__);
    mu_ = nullptr;
    if (allocator_ != nullptr) {
        allocator_->free((void**)(&cublas_workspace_qgemm_));
    }
}

cublasFP8MMWrapper::cublasFP8MMWrapper(const cublasFP8MMWrapper& wrapper):
    cublasMMWrapper(wrapper.cublas_handle_,
                    wrapper.cublaslt_handle_,
                    wrapper.stream_,
                    wrapper.cublas_algo_map_,
                    wrapper.mu_,
                    wrapper.allocator_)
{
    FT_LOG_DEBUG(__PRETTY_FUNCTION__);
    cublasVersionCheck();
}

void cublasFP8MMWrapper::cublasVersionCheck()
{
    cublasGetProperty(MAJOR_VERSION, &version_major_);
    cublasGetProperty(MINOR_VERSION, &version_minor_);
    cublasGetProperty(PATCH_LEVEL, &version_patch_);
    size_t cublasVersion = (version_major_ * 10000 + version_minor_ * 100 + version_patch_);
#if defined(FP8_MHA) || !defined(FP8_GEMM_OUTPUT_QUANT_DISABLE)
    FT_CHECK_WITH_INFO((version_major_ > 11) || (version_major_ == 11 && version_minor_ == 11 && version_patch_ >= 4),
                       "FP8 MHA needs d-scale, which is only supported after cublas 11.11.4 !");

#endif
}

void cublasFP8MMWrapper::Gemm(__hip_bfloat16*       res,
                              int                  batchCount,
                              int                  m,
                              int                  n,
                              int                  k,
                              int64_t              strideA,
                              int64_t              strideB,
                              int64_t              strideD,
                              const float*         alpha,
                              const float*         beta,
                              const __hip_fp8_e4m3_fnuz* input,
                              const __hip_fp8_e4m3_fnuz* kernel,
                              const float*         input_scale,
                              const float*         kernel_scale)
{
    Gemm(res,
         batchCount,
         m,
         n,
         k,
         strideA,
         strideB,
         strideD,
         alpha,
         beta,
         input,
         kernel,
         input_scale,
         kernel_scale,
         (hipStream_t)0);
}

void cublasFP8MMWrapper::Gemm(__hip_bfloat16*       res,
                              int                  batchCount,
                              int                  m,
                              int                  n,
                              int                  k,
                              int64_t              strideA,
                              int64_t              strideB,
                              int64_t              strideD,
                              const float*         alpha,
                              const float*         beta,
                              const __hip_fp8_e4m3_fnuz* input,
                              const __hip_fp8_e4m3_fnuz* kernel,
                              const float*         input_scale,
                              const float*         kernel_scale,
                              hipStream_t         stream,
                              bool                 fastAccum)
{
    FT_LOG_DEBUG(__PRETTY_FUNCTION__);
    mu_->lock();

    const void*  devAscalePtr = (const void*)kernel_scale;
    const void*  devBscalePtr = (const void*)input_scale;
    const size_t wsSizeBytes  = CUBLAS_WORKSPACE_SIZE;

    const auto aType       = HIP_R_8F_E4M3;
    const auto bType       = HIP_R_8F_E4M3;
    const auto dType       = HIP_R_16BF;
    const auto computeType = HIPBLAS_COMPUTE_32F;
    const auto scaleType   = HIP_R_32F;
    // const auto epilogueAuxType = HIP_R_16BF;

    const hipblasOperation_t tA = HIPBLAS_OP_T;
    const hipblasOperation_t tB = HIPBLAS_OP_N;

    //------- init, desc & tensors
    hipblasLtMatmulDesc_t   matmulDesc;
    hipblasLtMatrixLayout_t Adesc;
    hipblasLtMatrixLayout_t Bdesc;
    hipblasLtMatrixLayout_t Ddesc;

    {
        check_cuda_error(hipblasLtMatmulDescCreate(&matmulDesc, computeType, scaleType));
        check_cuda_error(hipblasLtMatmulDescSetAttribute(matmulDesc, HIPBLASLT_MATMUL_DESC_TRANSA, &tA, sizeof(tA)));
        check_cuda_error(hipblasLtMatmulDescSetAttribute(matmulDesc, HIPBLASLT_MATMUL_DESC_TRANSB, &tB, sizeof(tB)));

        if (version_major_ >= 11 && version_minor_ >= 11 && version_patch_ > 0 && fastAccum) {
            const int8_t fastAccuMode = 1;  // enable fast imprecise accum
            check_cuda_error(hipblasLtMatmulDescSetAttribute(
                matmulDesc, CUBLASLT_MATMUL_DESC_FAST_ACCUM, &fastAccuMode, sizeof(decltype(fastAccuMode))));
        }

        // TODO: Check that do we need to set these attributes
        // TODO: comment them for compiler first
        check_cuda_error(hipblasLtMatmulDescSetAttribute(
            matmulDesc, HIPBLASLT_MATMUL_DESC_A_SCALE_POINTER, &devAscalePtr, sizeof(devAscalePtr)));
        check_cuda_error(hipblasLtMatmulDescSetAttribute(
            matmulDesc, HIPBLASLT_MATMUL_DESC_B_SCALE_POINTER, &devBscalePtr, sizeof(devBscalePtr)));
    }

    {
        const int64_t lda = k;
        const int64_t ldb = k;
        const int64_t ldd = n;

        // create matrix descriptors, we are good with the details here so no need
        // to set any extra attributes
        check_cuda_error(
            hipblasLtMatrixLayoutCreate(&Adesc, aType, tA == HIPBLAS_OP_N ? n : k, tA == HIPBLAS_OP_N ? k : n, lda));
        if (batchCount > 1) {
            check_cuda_error(hipblasLtMatrixLayoutSetAttribute(
                Adesc, HIPBLASLT_MATRIX_LAYOUT_BATCH_COUNT, &batchCount, sizeof(batchCount)));
            check_cuda_error(hipblasLtMatrixLayoutSetAttribute(
                Adesc, HIPBLASLT_MATRIX_LAYOUT_STRIDED_BATCH_OFFSET, &strideA, sizeof(strideA)));
        }

        check_cuda_error(
            hipblasLtMatrixLayoutCreate(&Bdesc, bType, tB == HIPBLAS_OP_N ? k : m, tB == HIPBLAS_OP_N ? m : k, ldb));
        if (batchCount > 1) {
            check_cuda_error(hipblasLtMatrixLayoutSetAttribute(
                Bdesc, HIPBLASLT_MATRIX_LAYOUT_BATCH_COUNT, &batchCount, sizeof(batchCount)));
            check_cuda_error(hipblasLtMatrixLayoutSetAttribute(
                Bdesc, HIPBLASLT_MATRIX_LAYOUT_STRIDED_BATCH_OFFSET, &strideB, sizeof(strideB)));
        }

        check_cuda_error(hipblasLtMatrixLayoutCreate(&Ddesc, dType, n, m, ldd));
        if (batchCount > 1) {
            check_cuda_error(hipblasLtMatrixLayoutSetAttribute(
                Ddesc, HIPBLASLT_MATRIX_LAYOUT_BATCH_COUNT, &batchCount, sizeof(batchCount)));
            check_cuda_error(hipblasLtMatrixLayoutSetAttribute(
                Ddesc, HIPBLASLT_MATRIX_LAYOUT_STRIDED_BATCH_OFFSET, &strideD, sizeof(strideD)));
        }
    }

    bool                    findAlgo = cublas_algo_map_->isExist(batchCount, n, m, k, FP8_DATATYPE);
    cublasLtMatmulAlgo_info info     = cublas_algo_map_->getAlgo(batchCount, n, m, k, FP8_DATATYPE);
    if (info.stages == -1) {
        findAlgo = false;
    }

    hipblasLtMatmulAlgo_t algo;
    int                  workspaceSize = cublas_workspace_ == NULL ? 0 : CUBLAS_WORKSPACE_SIZE;
    if (findAlgo) {
        if (info.workspaceSize > workspaceSize) {
            findAlgo = false;
        }
        else {
            cublasLtMatmulAlgoInit(
                cublaslt_handle_, computeType, scaleType, aType, bType, dType, dType, info.algoId, &algo);
            cublasLtMatmulAlgoConfigSetAttribute(
                &algo, CUBLASLT_ALGO_CONFIG_CUSTOM_OPTION, &(info.customOption), sizeof(info.customOption));
            cublasLtMatmulAlgoConfigSetAttribute(&algo, CUBLASLT_ALGO_CONFIG_TILE_ID, &(info.tile), sizeof(info.tile));
            cublasLtMatmulAlgoConfigSetAttribute(
                &algo, CUBLASLT_ALGO_CONFIG_SPLITK_NUM, &(info.splitK_val), sizeof(info.splitK_val));
            cublasLtMatmulAlgoConfigSetAttribute(
                &algo, CUBLASLT_ALGO_CONFIG_CTA_SWIZZLING, &(info.swizzle), sizeof(info.swizzle));
            cublasLtMatmulAlgoConfigSetAttribute(
                &algo, CUBLASLT_ALGO_CONFIG_REDUCTION_SCHEME, &(info.reductionScheme), sizeof(info.reductionScheme));

#if (CUDART_VERSION >= 11000)
            cublasLtMatmulAlgoConfigSetAttribute(
                &algo, CUBLASLT_ALGO_CONFIG_STAGES_ID, &(info.stages), sizeof(info.stages));
#endif

#if (CUBLAS_VER_MAJOR == 11 && CUBLAS_VER_MINOR == 11 && CUBLAS_VER_PATCH >= 3)
            cublasLtMatmulAlgoConfigSetAttribute(
                &algo, CUBLASLT_ALGO_CONFIG_INNER_SHAPE_ID, &(info.inner_shapeId), sizeof(info.inner_shapeId));
            cublasLtMatmulAlgoConfigSetAttribute(
                &algo, CUBLASLT_ALGO_CONFIG_CLUSTER_SHAPE_ID, &(info.cluster_shapeId), sizeof(info.cluster_shapeId));
#elif (CUBLAS_VER_MAJOR == 11 && CUBLAS_VER_MINOR == 11 && CUBLAS_VER_PATCH < 3)
            cublasLtMatmulAlgoConfigSetAttribute(
                &algo, CUBLASLT_ALGO_CONFIG_MMA_SHAPE_ID, &(info.mma_shapeId), sizeof(info.mma_shapeId));
            cublasLtMatmulAlgoConfigSetAttribute(
                &algo, CUBLASLT_ALGO_CONFIG_CGA_SHAPE_ID, &(info.cga_shapeId), sizeof(info.cga_shapeId));
            cublasLtMatmulAlgoConfigSetAttribute(
                &algo, CUBLASLT_ALGO_CONFIG_SCHEDULING_MODE, &(info.sche_mode), sizeof(info.sche_mode));
#endif
        }
    }

    {
        hipblasStatus_t status = hipblasLtMatmul(cublaslt_handle_,
                                               matmulDesc,
                                               alpha,
                                               kernel,
                                               Adesc,
                                               input,
                                               Bdesc,
                                               beta,
                                               nullptr,  // Cptr, not used here
                                               Ddesc,
                                               res,
                                               Ddesc,
                                               (findAlgo ? (&algo) : NULL),
                                               cublas_workspace_,
                                               wsSizeBytes,
                                               stream);
        check_cuda_error(status);
    }

    if (Ddesc) {
        check_cuda_error(hipblasLtMatrixLayoutDestroy(Ddesc));
    }
    if (Bdesc) {
        check_cuda_error(hipblasLtMatrixLayoutDestroy(Bdesc));
    }
    if (Adesc) {
        check_cuda_error(hipblasLtMatrixLayoutDestroy(Adesc));
    }
    if (matmulDesc) {
        check_cuda_error(hipblasLtMatmulDescDestroy(matmulDesc));
    }

    mu_->unlock();
}

void cublasFP8MMWrapper::Gemm(__hip_fp8_e4m3_fnuz*       res,
                              int                  batchCount,
                              int                  m,
                              int                  n,
                              int                  k,
                              int64_t              strideA,
                              int64_t              strideB,
                              int64_t              strideD,
                              const float*         alpha,
                              const float*         beta,
                              const __hip_fp8_e4m3_fnuz* input,
                              const __hip_fp8_e4m3_fnuz* kernel,
                              const float*         input_scale,
                              const float*         kernel_scale,
                              const float*         output_scale)
{
    Gemm(res,
         batchCount,
         m,
         n,
         k,
         strideA,
         strideB,
         strideD,
         alpha,
         beta,
         input,
         kernel,
         input_scale,
         kernel_scale,
         output_scale,
         0);
}

void cublasFP8MMWrapper::Gemm(__hip_fp8_e4m3_fnuz*       res,
                              int                  batchCount,
                              int                  m,
                              int                  n,
                              int                  k,
                              int64_t              strideA,
                              int64_t              strideB,
                              int64_t              strideD,
                              const float*         alpha,
                              const float*         beta,
                              const __hip_fp8_e4m3_fnuz* input,
                              const __hip_fp8_e4m3_fnuz* kernel,
                              const float*         input_scale,
                              const float*         kernel_scale,
                              const float*         output_scale,
                              hipStream_t         stream,
                              bool                 fastAccum)
{
    FT_LOG_DEBUG(__PRETTY_FUNCTION__);
    mu_->lock();

    const void* devAscalePtr = (const void*)kernel_scale;
    const void* devBscalePtr = (const void*)input_scale;
    const void* devDscalePtr = (const void*)output_scale;

    FT_CHECK(cublas_workspace_ != nullptr);
    const size_t wsSizeBytes = CUBLAS_WORKSPACE_SIZE;

    const auto aType       = HIP_R_8F_E4M3;
    const auto bType       = HIP_R_8F_E4M3;
    const auto cType       = HIP_R_16BF;
    const auto dType       = HIP_R_8F_E4M3;
    const auto computeType = HIPBLAS_COMPUTE_32F;
    const auto scaleType   = HIP_R_32F;

    const hipblasOperation_t tA = HIPBLAS_OP_T;
    const hipblasOperation_t tB = HIPBLAS_OP_N;

    //------- init, desc & tensors
    hipblasLtMatmulDesc_t   matmulDesc;
    hipblasLtMatrixLayout_t Adesc;
    hipblasLtMatrixLayout_t Bdesc;
    hipblasLtMatrixLayout_t Cdesc;
    hipblasLtMatrixLayout_t Ddesc;

    {
        check_cuda_error(hipblasLtMatmulDescCreate(&matmulDesc, computeType, scaleType));
        check_cuda_error(hipblasLtMatmulDescSetAttribute(matmulDesc, HIPBLASLT_MATMUL_DESC_TRANSA, &tA, sizeof(tA)));
        check_cuda_error(hipblasLtMatmulDescSetAttribute(matmulDesc, HIPBLASLT_MATMUL_DESC_TRANSB, &tB, sizeof(tB)));

        if (version_major_ >= 11 && version_minor_ >= 11 && version_patch_ > 0 && fastAccum) {
            const int8_t fastAccuMode = 1;  // enable fast imprecise accum
            check_cuda_error(hipblasLtMatmulDescSetAttribute(
                matmulDesc, CUBLASLT_MATMUL_DESC_FAST_ACCUM, &fastAccuMode, sizeof(decltype(fastAccuMode))));
        }

        // TODO: Check that do we need to set these attributes
        // TODO: comment them for compiler first
        check_cuda_error(hipblasLtMatmulDescSetAttribute(
            matmulDesc, HIPBLASLT_MATMUL_DESC_A_SCALE_POINTER, &devAscalePtr, sizeof(devAscalePtr)));
        check_cuda_error(hipblasLtMatmulDescSetAttribute(
            matmulDesc, HIPBLASLT_MATMUL_DESC_B_SCALE_POINTER, &devBscalePtr, sizeof(devBscalePtr)));
        // check_cuda_error(hipblasLtMatmulDescSetAttribute(
        //     matmulDesc, HIPBLASLT_MATMUL_DESC_C_SCALE_POINTER, &devDscalePtr, sizeof(devDscalePtr)));
        check_cuda_error(hipblasLtMatmulDescSetAttribute(
            matmulDesc, HIPBLASLT_MATMUL_DESC_D_SCALE_POINTER, &devDscalePtr, sizeof(devDscalePtr)));
    }

    {
        const int64_t lda = k;
        const int64_t ldb = k;
        const int64_t ldd = n;

        // create matrix descriptors, we are good with the details here so no need
        // to set any extra attributes
        check_cuda_error(
            hipblasLtMatrixLayoutCreate(&Adesc, aType, tA == HIPBLAS_OP_N ? n : k, tA == HIPBLAS_OP_N ? k : n, lda));
        if (batchCount > 1) {
            check_cuda_error(hipblasLtMatrixLayoutSetAttribute(
                Adesc, HIPBLASLT_MATRIX_LAYOUT_BATCH_COUNT, &batchCount, sizeof(batchCount)));
            check_cuda_error(hipblasLtMatrixLayoutSetAttribute(
                Adesc, HIPBLASLT_MATRIX_LAYOUT_STRIDED_BATCH_OFFSET, &strideA, sizeof(strideA)));
        }

        check_cuda_error(
            hipblasLtMatrixLayoutCreate(&Bdesc, bType, tB == HIPBLAS_OP_N ? k : m, tB == HIPBLAS_OP_N ? m : k, ldb));
        if (batchCount > 1) {
            check_cuda_error(hipblasLtMatrixLayoutSetAttribute(
                Bdesc, HIPBLASLT_MATRIX_LAYOUT_BATCH_COUNT, &batchCount, sizeof(batchCount)));
            check_cuda_error(hipblasLtMatrixLayoutSetAttribute(
                Bdesc, HIPBLASLT_MATRIX_LAYOUT_STRIDED_BATCH_OFFSET, &strideB, sizeof(strideB)));
        }

        check_cuda_error(hipblasLtMatrixLayoutCreate(&Cdesc, cType, n, m, ldd));
        if (batchCount > 1) {
            check_cuda_error(hipblasLtMatrixLayoutSetAttribute(
                Cdesc, HIPBLASLT_MATRIX_LAYOUT_BATCH_COUNT, &batchCount, sizeof(batchCount)));
            check_cuda_error(hipblasLtMatrixLayoutSetAttribute(
                Cdesc, HIPBLASLT_MATRIX_LAYOUT_STRIDED_BATCH_OFFSET, &strideD, sizeof(strideD)));
        }
        check_cuda_error(hipblasLtMatrixLayoutCreate(&Ddesc, dType, n, m, ldd));
        if (batchCount > 1) {
            check_cuda_error(hipblasLtMatrixLayoutSetAttribute(
                Ddesc, HIPBLASLT_MATRIX_LAYOUT_BATCH_COUNT, &batchCount, sizeof(batchCount)));
            check_cuda_error(hipblasLtMatrixLayoutSetAttribute(
                Ddesc, HIPBLASLT_MATRIX_LAYOUT_STRIDED_BATCH_OFFSET, &strideD, sizeof(strideD)));
        }
    }

    bool                    findAlgo = cublas_algo_map_->isExist(batchCount, n, m, k, FP8_DATATYPE);
    cublasLtMatmulAlgo_info info     = cublas_algo_map_->getAlgo(batchCount, n, m, k, FP8_DATATYPE);
    if (info.stages == -1) {
        findAlgo = false;
    }

    hipblasLtMatmulAlgo_t algo;
    int                  workspaceSize = cublas_workspace_ == NULL ? 0 : CUBLAS_WORKSPACE_SIZE;
    if (findAlgo) {
        if (info.workspaceSize > workspaceSize) {
            findAlgo = false;
        }
        else {
            cublasLtMatmulAlgoInit(
                cublaslt_handle_, computeType, scaleType, aType, bType, cType, dType, info.algoId, &algo);
            cublasLtMatmulAlgoConfigSetAttribute(
                &algo, CUBLASLT_ALGO_CONFIG_CUSTOM_OPTION, &(info.customOption), sizeof(info.customOption));
            cublasLtMatmulAlgoConfigSetAttribute(&algo, CUBLASLT_ALGO_CONFIG_TILE_ID, &(info.tile), sizeof(info.tile));
            cublasLtMatmulAlgoConfigSetAttribute(
                &algo, CUBLASLT_ALGO_CONFIG_SPLITK_NUM, &(info.splitK_val), sizeof(info.splitK_val));
            cublasLtMatmulAlgoConfigSetAttribute(
                &algo, CUBLASLT_ALGO_CONFIG_CTA_SWIZZLING, &(info.swizzle), sizeof(info.swizzle));
            cublasLtMatmulAlgoConfigSetAttribute(
                &algo, CUBLASLT_ALGO_CONFIG_REDUCTION_SCHEME, &(info.reductionScheme), sizeof(info.reductionScheme));

#if (CUDART_VERSION >= 11000)
            cublasLtMatmulAlgoConfigSetAttribute(
                &algo, CUBLASLT_ALGO_CONFIG_STAGES_ID, &(info.stages), sizeof(info.stages));
#endif

#if (CUBLAS_VER_MAJOR == 11 && CUBLAS_VER_MINOR == 11 && CUBLAS_VER_PATCH >= 3)
            cublasLtMatmulAlgoConfigSetAttribute(
                &algo, CUBLASLT_ALGO_CONFIG_INNER_SHAPE_ID, &(info.inner_shapeId), sizeof(info.inner_shapeId));
            cublasLtMatmulAlgoConfigSetAttribute(
                &algo, CUBLASLT_ALGO_CONFIG_CLUSTER_SHAPE_ID, &(info.cluster_shapeId), sizeof(info.cluster_shapeId));
#elif (CUBLAS_VER_MAJOR == 11 && CUBLAS_VER_MINOR == 11 && CUBLAS_VER_PATCH < 3)
            cublasLtMatmulAlgoConfigSetAttribute(
                &algo, CUBLASLT_ALGO_CONFIG_MMA_SHAPE_ID, &(info.mma_shapeId), sizeof(info.mma_shapeId));
            cublasLtMatmulAlgoConfigSetAttribute(
                &algo, CUBLASLT_ALGO_CONFIG_CGA_SHAPE_ID, &(info.cga_shapeId), sizeof(info.cga_shapeId));
            cublasLtMatmulAlgoConfigSetAttribute(
                &algo, CUBLASLT_ALGO_CONFIG_SCHEDULING_MODE, &(info.sche_mode), sizeof(info.sche_mode));
#endif
        }
    }

    {
        hipblasStatus_t status = hipblasLtMatmul(cublaslt_handle_,
                                               matmulDesc,
                                               alpha,
                                               kernel,
                                               Adesc,
                                               input,
                                               Bdesc,
                                               beta,
                                               nullptr,  // Cptr, not used here
                                               Cdesc,
                                               res,
                                               Ddesc,
                                               (findAlgo ? (&algo) : NULL),
                                               cublas_workspace_,
                                               wsSizeBytes,
                                               stream);
        check_cuda_error(status);
    }

    if (Ddesc) {
        check_cuda_error(hipblasLtMatrixLayoutDestroy(Ddesc));
    }
    if (Cdesc) {
        check_cuda_error(hipblasLtMatrixLayoutDestroy(Cdesc));
    }
    if (Bdesc) {
        check_cuda_error(hipblasLtMatrixLayoutDestroy(Bdesc));
    }
    if (Adesc) {
        check_cuda_error(hipblasLtMatrixLayoutDestroy(Adesc));
    }
    if (matmulDesc) {
        check_cuda_error(hipblasLtMatmulDescDestroy(matmulDesc));
    }

    mu_->unlock();
}

template<bool RELU, bool GELU>
void cublasFP8MMWrapper::Conv1x1Gemm(__hip_fp8_e4m3_fnuz*       res,
                                     int                  m,
                                     int                  n,
                                     int                  k,
                                     const __hip_fp8_e4m3_fnuz* input,
                                     const __hip_fp8_e4m3_fnuz* kernel,
                                     const __hip_bfloat16* bias,
                                     const float          input_scale,
                                     const float          kernel_scale,
                                     const float          output_scale,
                                     hipStream_t         stream)
{
    FT_LOG_DEBUG(__PRETTY_FUNCTION__);
    mu_->lock();
    size_t workspace_size = 0;
    // get workspace size
    qgmmaLauncher.getWorkSpaceSize<RELU, GELU>(n, workspace_size);

    if (workspace_size > CUBLAS_WORKSPACE_1MB) {
        throw std::runtime_error("Need to rellocate workspace for qgemm. It is not supported");
        // cublas_workspace_qgemm_ = allocator_->reMalloc(cublas_workspace_qgemm_, workspace_size);
    }

    qgmmaLauncher.invokeQgmma1x1<RELU, GELU>(
        res, m, n, k, input, kernel, bias, input_scale, kernel_scale, output_scale, cublas_workspace_qgemm_, stream);
    sync_check_cuda_error();
    mu_->unlock();
}

template void cublasFP8MMWrapper::Conv1x1Gemm<true, false>(__hip_fp8_e4m3_fnuz*       res,
                                                           int                  m,
                                                           int                  n,
                                                           int                  k,
                                                           const __hip_fp8_e4m3_fnuz* input,
                                                           const __hip_fp8_e4m3_fnuz* kernel,
                                                           const __hip_bfloat16* bias,
                                                           const float          input_scale,
                                                           const float          kernel_scale,
                                                           const float          output_scale,
                                                           hipStream_t         stream);
template void cublasFP8MMWrapper::Conv1x1Gemm<true, true>(__hip_fp8_e4m3_fnuz*       res,
                                                          int                  m,
                                                          int                  n,
                                                          int                  k,
                                                          const __hip_fp8_e4m3_fnuz* input,
                                                          const __hip_fp8_e4m3_fnuz* kernel,
                                                          const __hip_bfloat16* bias,
                                                          const float          input_scale,
                                                          const float          kernel_scale,
                                                          const float          output_scale,
                                                          hipStream_t         stream);
template void cublasFP8MMWrapper::Conv1x1Gemm<false, false>(__hip_fp8_e4m3_fnuz*       res,
                                                            int                  m,
                                                            int                  n,
                                                            int                  k,
                                                            const __hip_fp8_e4m3_fnuz* input,
                                                            const __hip_fp8_e4m3_fnuz* kernel,
                                                            const __hip_bfloat16* bias,
                                                            const float          input_scale,
                                                            const float          kernel_scale,
                                                            const float          output_scale,
                                                            hipStream_t         stream);
template void cublasFP8MMWrapper::Conv1x1Gemm<false, true>(__hip_fp8_e4m3_fnuz*       res,
                                                           int                  m,
                                                           int                  n,
                                                           int                  k,
                                                           const __hip_fp8_e4m3_fnuz* input,
                                                           const __hip_fp8_e4m3_fnuz* kernel,
                                                           const __hip_bfloat16* bias,
                                                           const float          input_scale,
                                                           const float          kernel_scale,
                                                           const float          output_scale,
                                                           hipStream_t         stream);

template<bool RELU, bool GELU>
void cublasFP8MMWrapper::Gemm_Bias_Act(__hip_bfloat16*       res,
                                       int                  batchCount,
                                       int                  m,
                                       int                  n,
                                       int                  k,
                                       int64_t              strideA,
                                       int64_t              strideB,
                                       int64_t              strideD,
                                       const float*         alpha,
                                       const float*         beta,
                                       const __hip_fp8_e4m3_fnuz* input,
                                       const __hip_fp8_e4m3_fnuz* kernel,
                                       const float*         input_scale,
                                       const float*         kernel_scale,
                                       const __hip_bfloat16* bias,
                                       const float*         output_scale,
                                       hipStream_t         stream)
{
    FT_LOG_DEBUG(__PRETTY_FUNCTION__);
    mu_->lock();

    const void*  devAscalePtr = (const void*)kernel_scale;
    const void*  devBscalePtr = (const void*)input_scale;
    const void*  devDscalePtr = (const void*)output_scale;
    const size_t wsSizeBytes  = CUBLAS_WORKSPACE_SIZE;

    const auto aType       = HIP_R_8F_E4M3;
    const auto bType       = HIP_R_8F_E4M3;
    const auto dType       = HIP_R_16BF;
    const auto computeType = HIPBLAS_COMPUTE_32F;
    const auto scaleType   = HIP_R_32F;
    // const auto epilogueAuxType = HIP_R_16BF;

    const hipblasOperation_t tA = HIPBLAS_OP_T;
    const hipblasOperation_t tB = HIPBLAS_OP_N;

    //------- init, desc & tensors
    hipblasLtMatmulDesc_t   matmulDesc;
    hipblasLtMatrixLayout_t Adesc;
    hipblasLtMatrixLayout_t Bdesc;
    hipblasLtMatrixLayout_t Ddesc;

    {
        check_cuda_error(hipblasLtMatmulDescCreate(&matmulDesc, computeType, scaleType));
        check_cuda_error(hipblasLtMatmulDescSetAttribute(matmulDesc, HIPBLASLT_MATMUL_DESC_TRANSA, &tA, sizeof(tA)));
        check_cuda_error(hipblasLtMatmulDescSetAttribute(matmulDesc, HIPBLASLT_MATMUL_DESC_TRANSB, &tB, sizeof(tB)));

        if (version_major_ >= 11 && version_minor_ >= 11 && version_patch_ > 0) {
            const int8_t fastAccuMode = 1;  // enable fast imprecise accum
            check_cuda_error(hipblasLtMatmulDescSetAttribute(
                matmulDesc, CUBLASLT_MATMUL_DESC_FAST_ACCUM, &fastAccuMode, sizeof(decltype(fastAccuMode))));
        }

        // TODO: Check that do we need to set these attributes
        // TODO: comment them for compiler first
        check_cuda_error(hipblasLtMatmulDescSetAttribute(
            matmulDesc, HIPBLASLT_MATMUL_DESC_A_SCALE_POINTER, &devAscalePtr, sizeof(devAscalePtr)));
        check_cuda_error(hipblasLtMatmulDescSetAttribute(
            matmulDesc, HIPBLASLT_MATMUL_DESC_B_SCALE_POINTER, &devBscalePtr, sizeof(devBscalePtr)));

        hipblasLtEpilogue_t epi = HIPBLASLT_EPILOGUE_BIAS;
        if (RELU == true) {
            epi = HIPBLASLT_EPILOGUE_RELU_BIAS;
        }
        else if (GELU == true) {
            epi = HIPBLASLT_EPILOGUE_GELU_BIAS;
        }
        // hipblasLtEpilogue_t epi = HIPBLASLT_EPILOGUE_BIAS;
        hipblasLtMatmulDescSetAttribute(matmulDesc, HIPBLASLT_MATMUL_DESC_EPILOGUE, &epi, sizeof(hipblasLtEpilogue_t));
        hipblasLtMatmulDescSetAttribute(matmulDesc, HIPBLASLT_MATMUL_DESC_BIAS_POINTER, &bias, sizeof(const void*));
    }

    {
        const int64_t lda = k;
        const int64_t ldb = k;
        const int64_t ldd = n;

        // create matrix descriptors, we are good with the details here so no need
        // to set any extra attributes
        check_cuda_error(
            hipblasLtMatrixLayoutCreate(&Adesc, aType, tA == HIPBLAS_OP_N ? n : k, tA == HIPBLAS_OP_N ? k : n, lda));
        if (batchCount > 1) {
            check_cuda_error(hipblasLtMatrixLayoutSetAttribute(
                Adesc, HIPBLASLT_MATRIX_LAYOUT_BATCH_COUNT, &batchCount, sizeof(batchCount)));
            check_cuda_error(hipblasLtMatrixLayoutSetAttribute(
                Adesc, HIPBLASLT_MATRIX_LAYOUT_STRIDED_BATCH_OFFSET, &strideA, sizeof(strideA)));
        }

        check_cuda_error(
            hipblasLtMatrixLayoutCreate(&Bdesc, bType, tB == HIPBLAS_OP_N ? k : m, tB == HIPBLAS_OP_N ? m : k, ldb));
        if (batchCount > 1) {
            check_cuda_error(hipblasLtMatrixLayoutSetAttribute(
                Bdesc, HIPBLASLT_MATRIX_LAYOUT_BATCH_COUNT, &batchCount, sizeof(batchCount)));
            check_cuda_error(hipblasLtMatrixLayoutSetAttribute(
                Bdesc, HIPBLASLT_MATRIX_LAYOUT_STRIDED_BATCH_OFFSET, &strideB, sizeof(strideB)));
        }

        check_cuda_error(hipblasLtMatrixLayoutCreate(&Ddesc, dType, n, m, ldd));
        if (batchCount > 1) {
            check_cuda_error(hipblasLtMatrixLayoutSetAttribute(
                Ddesc, HIPBLASLT_MATRIX_LAYOUT_BATCH_COUNT, &batchCount, sizeof(batchCount)));
            check_cuda_error(hipblasLtMatrixLayoutSetAttribute(
                Ddesc, HIPBLASLT_MATRIX_LAYOUT_STRIDED_BATCH_OFFSET, &strideD, sizeof(strideD)));
        }
    }

    const int                       requestedAlgoCount = 1;
    hipblasLtMatmulHeuristicResult_t heuristicResult;
    hipblasLtMatmulPreference_t      preference;
    int                             returnedAlgoCount = -1;
    check_cuda_error(hipblasLtMatmulPreferenceCreate(&preference));
    check_cuda_error(hipblasLtMatmulPreferenceSetAttribute(
        preference, HIPBLASLT_MATMUL_PREF_MAX_WORKSPACE_BYTES, &wsSizeBytes, sizeof(wsSizeBytes)));

    check_cuda_error(hipblasLtMatmulAlgoGetHeuristic(cublaslt_handle_,
                                                    matmulDesc,
                                                    Adesc,
                                                    Bdesc,
                                                    Ddesc,
                                                    Ddesc,
                                                    preference,
                                                    requestedAlgoCount,
                                                    &heuristicResult,
                                                    &returnedAlgoCount));

    {
        hipblasStatus_t status = hipblasLtMatmul(cublaslt_handle_,
                                               matmulDesc,
                                               alpha,
                                               kernel,
                                               Adesc,
                                               input,
                                               Bdesc,
                                               beta,
                                               res,
                                               Ddesc,
                                               res,
                                               Ddesc,
                                               &heuristicResult.algo,
                                               cublas_workspace_,
                                               wsSizeBytes,
                                               stream);
        check_cuda_error(status);
    }

    if (Ddesc) {
        check_cuda_error(hipblasLtMatrixLayoutDestroy(Ddesc));
    }
    if (Bdesc) {
        check_cuda_error(hipblasLtMatrixLayoutDestroy(Bdesc));
    }
    if (Adesc) {
        check_cuda_error(hipblasLtMatrixLayoutDestroy(Adesc));
    }
    if (matmulDesc) {
        check_cuda_error(hipblasLtMatmulDescDestroy(matmulDesc));
    }

    mu_->unlock();
}

template<bool RELU, bool GELU>
void cublasFP8MMWrapper::Gemm_Bias_Act(__hip_fp8_e4m3_fnuz*       res,
                                       int                  batchCount,
                                       int                  m,
                                       int                  n,
                                       int                  k,
                                       int64_t              strideA,
                                       int64_t              strideB,
                                       int64_t              strideD,
                                       const float*         alpha,
                                       const float*         beta,
                                       const __hip_fp8_e4m3_fnuz* input,
                                       const __hip_fp8_e4m3_fnuz* kernel,
                                       const float*         input_scale,
                                       const float*         kernel_scale,
                                       const __hip_bfloat16* bias,
                                       const float*         output_scale,
                                       hipStream_t         stream)
{
    FT_LOG_DEBUG(__PRETTY_FUNCTION__);
    mu_->lock();

    const void*  devAscalePtr = (const void*)kernel_scale;
    const void*  devBscalePtr = (const void*)input_scale;
    const void*  devDscalePtr = (const void*)output_scale;
    const size_t wsSizeBytes  = CUBLAS_WORKSPACE_SIZE;

    const auto aType       = HIP_R_8F_E4M3;
    const auto bType       = HIP_R_8F_E4M3;
    const auto cType       = HIP_R_16BF;
    const auto dType       = HIP_R_8F_E4M3;
    const auto computeType = HIPBLAS_COMPUTE_32F;
    const auto scaleType   = HIP_R_32F;
    // const auto epilogueAuxType = HIP_R_16BF;

    const hipblasOperation_t tA = HIPBLAS_OP_T;
    const hipblasOperation_t tB = HIPBLAS_OP_N;

    //------- init, desc & tensors
    hipblasLtMatmulDesc_t   matmulDesc;
    hipblasLtMatrixLayout_t Adesc;
    hipblasLtMatrixLayout_t Bdesc;
    hipblasLtMatrixLayout_t Cdesc;
    hipblasLtMatrixLayout_t Ddesc;

    {
        check_cuda_error(hipblasLtMatmulDescCreate(&matmulDesc, computeType, scaleType));
        check_cuda_error(hipblasLtMatmulDescSetAttribute(matmulDesc, HIPBLASLT_MATMUL_DESC_TRANSA, &tA, sizeof(tA)));
        check_cuda_error(hipblasLtMatmulDescSetAttribute(matmulDesc, HIPBLASLT_MATMUL_DESC_TRANSB, &tB, sizeof(tB)));

        if (version_major_ >= 11 && version_minor_ >= 11 && version_patch_ > 0) {
            const int8_t fastAccuMode = 1;  // enable fast imprecise accum
            check_cuda_error(hipblasLtMatmulDescSetAttribute(
                matmulDesc, CUBLASLT_MATMUL_DESC_FAST_ACCUM, &fastAccuMode, sizeof(decltype(fastAccuMode))));
        }

        // TODO: Check that do we need to set these attributes
        // TODO: comment them for compiler first
        check_cuda_error(hipblasLtMatmulDescSetAttribute(
            matmulDesc, HIPBLASLT_MATMUL_DESC_A_SCALE_POINTER, &devAscalePtr, sizeof(devAscalePtr)));
        check_cuda_error(hipblasLtMatmulDescSetAttribute(
            matmulDesc, HIPBLASLT_MATMUL_DESC_B_SCALE_POINTER, &devBscalePtr, sizeof(devBscalePtr)));
        check_cuda_error(hipblasLtMatmulDescSetAttribute(
            matmulDesc, HIPBLASLT_MATMUL_DESC_D_SCALE_POINTER, &devDscalePtr, sizeof(devDscalePtr)));

        hipblasLtEpilogue_t epi = HIPBLASLT_EPILOGUE_GELU_BIAS;
        // hipblasLtEpilogue_t epi = HIPBLASLT_EPILOGUE_BIAS;
        hipblasLtMatmulDescSetAttribute(matmulDesc, HIPBLASLT_MATMUL_DESC_EPILOGUE, &epi, sizeof(hipblasLtEpilogue_t));
        hipblasLtMatmulDescSetAttribute(matmulDesc, HIPBLASLT_MATMUL_DESC_BIAS_POINTER, &bias, sizeof(const void*));
    }

    {
        const int64_t lda = k;
        const int64_t ldb = k;
        const int64_t ldd = n;

        // create matrix descriptors, we are good with the details here so no need
        // to set any extra attributes
        check_cuda_error(
            hipblasLtMatrixLayoutCreate(&Adesc, aType, tA == HIPBLAS_OP_N ? n : k, tA == HIPBLAS_OP_N ? k : n, lda));
        if (batchCount > 1) {
            check_cuda_error(hipblasLtMatrixLayoutSetAttribute(
                Adesc, HIPBLASLT_MATRIX_LAYOUT_BATCH_COUNT, &batchCount, sizeof(batchCount)));
            check_cuda_error(hipblasLtMatrixLayoutSetAttribute(
                Adesc, HIPBLASLT_MATRIX_LAYOUT_STRIDED_BATCH_OFFSET, &strideA, sizeof(strideA)));
        }

        check_cuda_error(
            hipblasLtMatrixLayoutCreate(&Bdesc, bType, tB == HIPBLAS_OP_N ? k : m, tB == HIPBLAS_OP_N ? m : k, ldb));
        if (batchCount > 1) {
            check_cuda_error(hipblasLtMatrixLayoutSetAttribute(
                Bdesc, HIPBLASLT_MATRIX_LAYOUT_BATCH_COUNT, &batchCount, sizeof(batchCount)));
            check_cuda_error(hipblasLtMatrixLayoutSetAttribute(
                Bdesc, HIPBLASLT_MATRIX_LAYOUT_STRIDED_BATCH_OFFSET, &strideB, sizeof(strideB)));
        }

        check_cuda_error(hipblasLtMatrixLayoutCreate(&Cdesc, cType, n, m, ldd));
        // (TODO Hongbinl)Not sure if the implementation makes sense
        if (batchCount > 1) {
            check_cuda_error(hipblasLtMatrixLayoutSetAttribute(
                Cdesc, HIPBLASLT_MATRIX_LAYOUT_BATCH_COUNT, &batchCount, sizeof(batchCount)));
            check_cuda_error(hipblasLtMatrixLayoutSetAttribute(
                Cdesc, HIPBLASLT_MATRIX_LAYOUT_STRIDED_BATCH_OFFSET, &strideD, sizeof(strideD)));
        }

        check_cuda_error(hipblasLtMatrixLayoutCreate(&Ddesc, dType, n, m, ldd));
        if (batchCount > 1) {
            check_cuda_error(hipblasLtMatrixLayoutSetAttribute(
                Ddesc, HIPBLASLT_MATRIX_LAYOUT_BATCH_COUNT, &batchCount, sizeof(batchCount)));
            check_cuda_error(hipblasLtMatrixLayoutSetAttribute(
                Ddesc, HIPBLASLT_MATRIX_LAYOUT_STRIDED_BATCH_OFFSET, &strideD, sizeof(strideD)));
        }
    }

    const int                       requestedAlgoCount = 1;
    hipblasLtMatmulHeuristicResult_t heuristicResult;
    hipblasLtMatmulPreference_t      preference;
    int                             returnedAlgoCount = -1;
    check_cuda_error(hipblasLtMatmulPreferenceCreate(&preference));
    check_cuda_error(hipblasLtMatmulPreferenceSetAttribute(
        preference, HIPBLASLT_MATMUL_PREF_MAX_WORKSPACE_BYTES, &wsSizeBytes, sizeof(wsSizeBytes)));
#if (CUBLAS_VERSION) <= 12000
    uint32_t pointer_mode_mask = 0;
    check_cuda_error(hipblasLtMatmulPreferenceSetAttribute(
        preference, CUBLASLT_MATMUL_PREF_EPILOGUE_MASK, &pointer_mode_mask, sizeof(pointer_mode_mask)));
#endif

    check_cuda_error(hipblasLtMatmulAlgoGetHeuristic(cublaslt_handle_,
                                                    matmulDesc,
                                                    Adesc,
                                                    Bdesc,
                                                    Cdesc,
                                                    Ddesc,
                                                    preference,
                                                    requestedAlgoCount,
                                                    &heuristicResult,
                                                    &returnedAlgoCount));

    {
        hipblasStatus_t status = hipblasLtMatmul(cublaslt_handle_,
                                               matmulDesc,
                                               alpha,
                                               kernel,
                                               Adesc,
                                               input,
                                               Bdesc,
                                               beta,
                                               res,
                                               Cdesc,
                                               res,
                                               Ddesc,
                                               &heuristicResult.algo,
                                               cublas_workspace_,
                                               wsSizeBytes,
                                               stream);
        check_cuda_error(status);
    }

    if (Ddesc) {
        check_cuda_error(hipblasLtMatrixLayoutDestroy(Ddesc));
    }
    if (Bdesc) {
        check_cuda_error(hipblasLtMatrixLayoutDestroy(Bdesc));
    }
    if (Adesc) {
        check_cuda_error(hipblasLtMatrixLayoutDestroy(Adesc));
    }
    if (matmulDesc) {
        check_cuda_error(hipblasLtMatmulDescDestroy(matmulDesc));
    }

    mu_->unlock();
}

template void cublasFP8MMWrapper::Gemm_Bias_Act<false, true>(__hip_bfloat16*       res,
                                                             int                  batchCount,
                                                             int                  m,
                                                             int                  n,
                                                             int                  k,
                                                             int64_t              strideA,
                                                             int64_t              strideB,
                                                             int64_t              strideD,
                                                             const float*         alpha,
                                                             const float*         beta,
                                                             const __hip_fp8_e4m3_fnuz* input,
                                                             const __hip_fp8_e4m3_fnuz* kernel,
                                                             const float*         input_scale,
                                                             const float*         kernel_scale,
                                                             const __hip_bfloat16* bias,
                                                             const float*         output_scale,
                                                             hipStream_t         stream);
template void cublasFP8MMWrapper::Gemm_Bias_Act<false, true>(__hip_fp8_e4m3_fnuz*       res,
                                                             int                  batchCount,
                                                             int                  m,
                                                             int                  n,
                                                             int                  k,
                                                             int64_t              strideA,
                                                             int64_t              strideB,
                                                             int64_t              strideD,
                                                             const float*         alpha,
                                                             const float*         beta,
                                                             const __hip_fp8_e4m3_fnuz* input,
                                                             const __hip_fp8_e4m3_fnuz* kernel,
                                                             const float*         input_scale,
                                                             const float*         kernel_scale,
                                                             const __hip_bfloat16* bias,
                                                             const float*         output_scale,
                                                             hipStream_t         stream);
template void cublasFP8MMWrapper::Gemm_Bias_Act<true, false>(__hip_bfloat16*       res,
                                                             int                  batchCount,
                                                             int                  m,
                                                             int                  n,
                                                             int                  k,
                                                             int64_t              strideA,
                                                             int64_t              strideB,
                                                             int64_t              strideD,
                                                             const float*         alpha,
                                                             const float*         beta,
                                                             const __hip_fp8_e4m3_fnuz* input,
                                                             const __hip_fp8_e4m3_fnuz* kernel,
                                                             const float*         input_scale,
                                                             const float*         kernel_scale,
                                                             const __hip_bfloat16* bias,
                                                             const float*         output_scale,
                                                             hipStream_t         stream);
template void cublasFP8MMWrapper::Gemm_Bias_Act<true, false>(__hip_fp8_e4m3_fnuz*       res,
                                                             int                  batchCount,
                                                             int                  m,
                                                             int                  n,
                                                             int                  k,
                                                             int64_t              strideA,
                                                             int64_t              strideB,
                                                             int64_t              strideD,
                                                             const float*         alpha,
                                                             const float*         beta,
                                                             const __hip_fp8_e4m3_fnuz* input,
                                                             const __hip_fp8_e4m3_fnuz* kernel,
                                                             const float*         input_scale,
                                                             const float*         kernel_scale,
                                                             const __hip_bfloat16* bias,
                                                             const float*         output_scale,
                                                             hipStream_t         stream);
template void cublasFP8MMWrapper::Gemm_Bias_Act<false, false>(__hip_fp8_e4m3_fnuz*       res,
                                                              int                  batchCount,
                                                              int                  m,
                                                              int                  n,
                                                              int                  k,
                                                              int64_t              strideA,
                                                              int64_t              strideB,
                                                              int64_t              strideD,
                                                              const float*         alpha,
                                                              const float*         beta,
                                                              const __hip_fp8_e4m3_fnuz* input,
                                                              const __hip_fp8_e4m3_fnuz* kernel,
                                                              const float*         input_scale,
                                                              const float*         kernel_scale,
                                                              const __hip_bfloat16* bias,
                                                              const float*         output_scale,
                                                              hipStream_t         stream);

}  // namespace fastertransformer
