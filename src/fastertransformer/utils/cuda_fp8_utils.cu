#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2022-2023, NVIDIA CORPORATION.  All rights reserved.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "cuda_fp8_utils.h"

namespace fastertransformer {
#ifdef ENABLE_FP8

template<typename T_OUT, typename T_IN, QUANTIZE_MODE quantize_mode>
__global__ void quantizeMatrix(T_OUT* output, float const* input_scale, T_IN const* input, uint32_t size, uint32_t n)
{
    for (uint32_t i = threadIdx.x + blockIdx.x * blockDim.x; i < size; i += blockDim.x * gridDim.x) {
        if (quantize_mode == QUANTIZE_MODE::PER_CHANNEL) {
            output[i] = T_OUT((float)(input[i]) * __ldg(input_scale + (i % n)));
        }
        else {
            output[i] = T_OUT((float)(input[i]) * __ldg(input_scale));
        }
    }
}

template<typename T_OUT, typename T_IN, QUANTIZE_MODE quantize_mode>
void invokeQuantizeMatrix(
    T_OUT* output, float const* input_scale, T_IN const* input, uint32_t size, uint32_t n, hipStream_t stream)
{
    dim3 grid(32);
    dim3 block(256);
    quantizeMatrix<T_OUT, T_IN, quantize_mode><<<grid, block, 0, stream>>>(output, input_scale, input, size, n);
}

#define defineinvokeQuantizeMatrix(type_out, type_in, mode)                                                            \
    template void invokeQuantizeMatrix<type_out, type_in, mode>(type_out * output,                                     \
                                                                float const*   input_scale,                            \
                                                                type_in const* input,                                  \
                                                                uint32_t       size,                                   \
                                                                uint32_t       n,                                      \
                                                                hipStream_t   stream);

defineinvokeQuantizeMatrix(__hip_fp8_e4m3_fnuz, float, QUANTIZE_MODE::PER_CHANNEL);
defineinvokeQuantizeMatrix(__hip_fp8_e4m3_fnuz, float, QUANTIZE_MODE::PER_TENSOR);
defineinvokeQuantizeMatrix(__hip_fp8_e4m3_fnuz, half, QUANTIZE_MODE::PER_CHANNEL);
defineinvokeQuantizeMatrix(__hip_fp8_e4m3_fnuz, half, QUANTIZE_MODE::PER_TENSOR);
defineinvokeQuantizeMatrix(half, __hip_fp8_e4m3_fnuz, QUANTIZE_MODE::PER_CHANNEL);
defineinvokeQuantizeMatrix(half, __hip_fp8_e4m3_fnuz, QUANTIZE_MODE::PER_TENSOR);
defineinvokeQuantizeMatrix(float, __hip_fp8_e4m3_fnuz, QUANTIZE_MODE::PER_CHANNEL);
defineinvokeQuantizeMatrix(float, __hip_fp8_e4m3_fnuz, QUANTIZE_MODE::PER_TENSOR);
#ifdef ENABLE_BF16
defineinvokeQuantizeMatrix(__hip_fp8_e4m3_fnuz, __hip_bfloat16, QUANTIZE_MODE::PER_CHANNEL);
defineinvokeQuantizeMatrix(__hip_fp8_e4m3_fnuz, __hip_bfloat16, QUANTIZE_MODE::PER_TENSOR);
defineinvokeQuantizeMatrix(__hip_bfloat16, __hip_fp8_e4m3_fnuz, QUANTIZE_MODE::PER_CHANNEL);
defineinvokeQuantizeMatrix(__hip_bfloat16, __hip_fp8_e4m3_fnuz, QUANTIZE_MODE::PER_TENSOR);
#endif

template<typename T_OUT, typename T_IN, typename T_FAKE>
__global__ void fakeQuantize(T_OUT* dst, const T_IN* src, const int size)
{
    for (int tid = threadIdx.x + blockIdx.x * blockDim.x; tid < size; tid += blockDim.x * gridDim.x) {
        T_FAKE tmp = (T_FAKE)((float)src[tid]);
        dst[tid]   = (T_OUT)((float)tmp);
    }
}

template<typename T_OUT, typename T_IN, typename T_FAKE>
void invokeFakeQuantize(T_OUT* dst, const T_IN* src, const int size, hipStream_t stream)
{
    fakeQuantize<T_OUT, T_IN, T_FAKE><<<256, 256, 0, stream>>>(dst, src, size);
}

template void
invokeFakeQuantize<float, float, __hip_fp8_e4m3_fnuz>(float* dst, const float* src, const int size, hipStream_t stream);
template void
invokeFakeQuantize<half, half, __hip_fp8_e4m3_fnuz>(half* dst, const half* src, const int size, hipStream_t stream);
template void invokeFakeQuantize<__hip_bfloat16, __hip_bfloat16, __hip_fp8_e4m3_fnuz>(__hip_bfloat16*       dst,
                                                                              const __hip_bfloat16* src,
                                                                              const int            size,
                                                                              hipStream_t         stream);

template<typename T_W>
__global__ void computeFP8QuantizeScale(float* quant_ptr, const T_W* weights, const int k, const int n)
{
    float max = -10000.f;
    for (int i = 0; i < k; i++) {
        float val = fabs((float)weights[i * n + blockIdx.x * blockDim.x + threadIdx.x]);
        max       = max > val ? max : val;
        if (threadIdx.x == 0 && blockIdx.x == 0 && i % 100 == 0) {
            printf("max: %f, val: %f \n", max, val);
        }
    }
    // quant_ptr[blockIdx.x * blockDim.x + threadIdx.x] = 1.0f;
    // quant_ptr[blockIdx.x * blockDim.x + threadIdx.x] = FP8_E4M3_MAX / max;
    quant_ptr[blockIdx.x * blockDim.x + threadIdx.x] = std::max(max / FP8_E4M3_MAX, 1.0f / 32.f);
}

template<typename T_W>
void invokeComputeFP8QuantizeScale(float* quant_ptr, const T_W* weights, const int k, const int n, hipStream_t stream)
{
    dim3 block(256);
    dim3 grid;
    grid.x = (n + 255) / 256;
    computeFP8QuantizeScale<T_W><<<grid, block, 0, stream>>>(quant_ptr, weights, k, n);
}

#ifdef ENABLE_BF16
template void invokeComputeFP8QuantizeScale(
    float* quant_ptr, const __hip_bfloat16* weights, const int k, const int n, hipStream_t stream);
#endif
template void
invokeComputeFP8QuantizeScale(float* quant_ptr, const float* weights, const int k, const int n, hipStream_t stream);

#endif  // ENABLE_FP8
}  // namespace fastertransformer