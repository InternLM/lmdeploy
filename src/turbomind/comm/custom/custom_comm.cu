
#include <memory>
#include <mutex>

#include <hip/hip_runtime.h>

#include "src/turbomind/comm/custom/custom_comm.h"

#include "mscclpp/core.hpp"
#include "mscclpp/semaphore_device.hpp"

#include "src/turbomind/comm/custom/bootstrap.h"
#include "src/turbomind/utils/cuda_utils.h"

namespace turbomind::comm {

CustomComm::CustomComm(std::shared_ptr<mscclpp::Bootstrap> bootstrap):
    Comm{bootstrap->getNranks(), bootstrap->getRank()}
{
    comm_ = std::make_shared<mscclpp::Communicator>(std::move(bootstrap));
}

void CustomComm::Initialize()
{
    FT_CHECK(comm_->bootstrap()->getNranks() == comm_->bootstrap()->getNranksPerNode());
    comm_->bootstrap()->barrier();
    {
        std::vector<mscclpp::NonblockingFuture<std::shared_ptr<mscclpp::Connection>>> connections;
        for (int i = 0; i < world_size_; ++i) {
            if (i == rank_) {
                continue;
            }
            connections.push_back(comm_->connectOnSetup(i, 0, mscclpp::Transport::CudaIpc));
        }
        comm_->setup();
        for (auto& c : connections) {
            connections_.push_back(c.get());
        }
    }

    for (int c = 0; c < kChannelsPerConn; ++c) {
        for (size_t i = 0; i < connections_.size(); ++i) {
            semaphores_.push_back(std::make_shared<mscclpp::SmDevice2DeviceSemaphore>(*comm_, connections_[i]));
        }
    }

    comm_->setup();

    hipMallocAsync(
        &device_semaphores_, sizeof(mscclpp::SmDevice2DeviceSemaphoreDeviceHandle) * semaphores_.size(), {});
    std::vector<mscclpp::SmDevice2DeviceSemaphoreDeviceHandle> device_semaphores;
    for (auto& s : semaphores_) {
        device_semaphores.push_back(s->deviceHandle());
    }
    hipMemcpyAsync(device_semaphores_,
                    device_semaphores.data(),
                    sizeof(mscclpp::SmDevice2DeviceSemaphoreDeviceHandle) * semaphores_.size(),
                    hipMemcpyDefault,
                    {});

    hipMallocAsync(&device_syncer_, sizeof(mscclpp::DeviceSyncer), {});
    hipMemsetAsync(device_syncer_, 0, sizeof(mscclpp::DeviceSyncer), {});
    hipStreamSynchronize({});

    hipMalloc(&packet_buff_, kPacketBuffSize);
    hipMemset(packet_buff_, 0, kPacketBuffSize);

    hipMalloc(&scratch_buff_, kScratchBuffSize);
    hipMemset(scratch_buff_, 0, kScratchBuffSize);

    RegisterBuffer(packet_buff_, kPacketBuffSize);
    RegisterBuffer(scratch_buff_, kScratchBuffSize);
}

void CustomComm::RegisterBuffer(void* ptr, size_t size)
{
    FT_CHECK(registered_channels_.count(ptr) == 0);

    mscclpp::RegisteredMemory memory = comm_->registerMemory(ptr, size, mscclpp::Transport::CudaIpc);
    std::vector<mscclpp::NonblockingFuture<mscclpp::RegisteredMemory>> futures;

    for (int i = 0; i < world_size_; ++i) {
        if (i == rank_) {
            continue;
        }
        futures.push_back(comm_->recvMemoryOnSetup(i, 0));
        comm_->sendMemoryOnSetup(memory, i, 0);
    }

    comm_->setup();

    std::vector<mscclpp::SmChannel>        channels;
    std::vector<mscclpp::RegisteredMemory> memories;

    for (size_t i = 0; i < connections_.size(); ++i) {
        mscclpp::RegisteredMemory remote_memory = futures[i].get();
        memories.push_back(remote_memory);
        channels.emplace_back(semaphores_[i], remote_memory, ptr, nullptr);
    }

    registered_channels_.emplace(ptr, std::move(channels));
    registered_memories_.emplace(ptr, std::move(memories));
}

Array<void*, kMaxNearPeers> CustomComm::get_near_impl(void* ptr)
{
    auto& memories = registered_memories_.at(ptr);
    FT_CHECK(memories.size() <= kMaxNearPeers);
    Array<void*, kMaxNearPeers> ret{};
    for (size_t i = 0; i < memories.size(); ++i) {
        ret[i] = memories[i].data();
    }
    return ret;
}

class LocalGroupId: public GroupId {
public:
    void Initialize() override
    {
        internal_ = std::make_shared<Internal>();
    }

    void Export(std::ostream& os) override
    {
        FT_CHECK((bool)internal_);  // `Initialize` must come befor `Export`

        const void* ptr = this;
        os.write((const char*)&ptr, sizeof(ptr));
    }

    void Import(std::istream& is) override
    {
        void* ptr{};
        is.read((char*)&ptr, sizeof(ptr));
        internal_ = reinterpret_cast<LocalGroupId*>(ptr)->internal_;

        FT_CHECK((bool)internal_);
    }

    std::unique_ptr<Comm> CreateCommunicator(int rank, int world_size) override
    {
        auto init_shared_state = [&] {  //
            internal_->state = std::make_shared<LocalBootstrap::State>(world_size);
        };

        FT_CHECK((bool)internal_);

        // one of the rank initialize the shared state
        std::call_once(internal_->flag, init_shared_state);

        FT_CHECK((bool)internal_->state);

        auto bootstrap = std::make_shared<LocalBootstrap>(world_size, rank, internal_->state);

        std::vector<hipCtx_t> ctx(world_size);
        CUDRVCHECK(hipCtxGetCurrent(&ctx[rank]));

        bootstrap->allGather(ctx.data(), sizeof(hipCtx_t));

        for (int i = 0; i < world_size; ++i) {
            if (i != rank) {
                auto ec = hipCtxEnablePeerAccess(ctx[i], 0);
                FT_CHECK(ec == hipSuccess || ec == hipErrorPeerAccessAlreadyEnabled);
            }
        }

        bootstrap->barrier();

        auto comm = std::make_unique<CustomComm>(bootstrap);

        comm->Initialize();

        return comm;
    }

private:
    struct Internal {
        std::once_flag                         flag;
        std::shared_ptr<LocalBootstrap::State> state;
    };

private:
    std::shared_ptr<Internal> internal_;
};

std::unique_ptr<GroupId> CreateCustomGroupId()
{
    return std::make_unique<LocalGroupId>();
}

}  // namespace turbomind::comm