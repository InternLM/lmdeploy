#include "hip/hip_runtime.h"

#include <atomic>
#include <stdexcept>
#include <type_traits>

#include "cub/block/block_reduce.cuh"

#include "src/turbomind/comm/custom/custom_comm.h"

#include "src/turbomind/comm/custom/device_semaphore.h"

#include "src/turbomind/kernels/core/array_ops.h"
#include "src/turbomind/kernels/core/common.h"
#include "src/turbomind/kernels/core/meta.h"

#include "src/turbomind/kernels/norm/rms_norm.h"
#include "src/turbomind/utils/Tensor.h"
#include "src/turbomind/utils/cuda_utils.h"

namespace turbomind {

template<class T, int vec_size, int block_dim, bool aligned, class Relaxed>
__global__ void AllreduceResidualBiasRMSnormKernel_Simple_v2(T*                                             buf,
                                                             T*                                             res,
                                                             const T*                                       bias,
                                                             const T*                                       weights,
                                                             Array<T*, kMaxNearPeers>                       chns,
                                                             mscclpp::SmDevice2DeviceSemaphoreDeviceHandle* semaphores,
                                                             int                                            rank,
                                                             int                                            peers,
                                                             int                                            slice,
                                                             int                                            count,
                                                             int                                            vdim,
                                                             float                                          inv_dim,
                                                             float                                          eps,
                                                             constant<vec_size>,
                                                             constant<block_dim>,
                                                             constant<aligned>,
                                                             Relaxed relaxed)
{
    const int bi        = blockIdx.x;
    const int block_num = gridDim.x;

    DeviceSemaphore sem;

    if (threadIdx.x < peers) {
        sem.Load(&semaphores[blockIdx.x * peers + threadIdx.x]);
        sem.SignalAndWait(relaxed);
    }

    __syncthreads();

    using Vec = Array<T, vec_size>;

    using namespace ops;

    const int  di       = threadIdx.x;
    const bool is_valid = di < vdim;

    if (aligned || is_valid) {

        const int first = rank * slice;
        const int last  = min(count, first + slice);

        for (int p = 0; p < peers - 1; ++p) {
            const int  peer = p + rank < peers ? p + rank : p + rank - peers;
            const auto chn  = cvta_generic_to_global(chns[peer]);
            Vec        acc, tmp;
            for (int ti = first + bi; ti < last; ti += block_num) {
                const int idx = (ti * vdim + di) * vec_size;
                Load(tmp, chn + idx);
                Load(acc, buf + idx);
                acc = acc + tmp;
                Store(buf + idx, acc);
            }
        }

        {  // last peer
            const int p    = peers - 1;
            const int peer = p + rank < peers ? p + rank : p + rank - peers;
            auto      chn  = cvta_generic_to_global(chns[peer]);
            Vec       acc, tmp;
            for (int ti = first + bi; ti < last; ti += block_num) {
                const int idx = (ti * vdim + di) * vec_size;
                Load(tmp, chn + idx);
                Load(acc, buf + idx);
                acc = acc + tmp;
                Vec r_vec, x_vec;
                Load(r_vec, res + idx);
                r_vec = r_vec + acc;
                if (bias) {
                    Load(x_vec, bias + di * vec_size);
                    r_vec = r_vec + x_vec;
                }
                Store(res + idx, r_vec);
                float sum{};
                PRAGMA_UNROLL
                for (int i = 0; i < vec_size; ++i) {
                    sum += (float)r_vec[i] * (float)r_vec[i];
                }
                using BlockReduce = hipcub::BlockReduce<float, block_dim>;
                __shared__ typename BlockReduce::TempStorage temp_storage;
                __shared__ float                             shared_sum;
                sum = BlockReduce{temp_storage}.Sum(sum);
                if (di == 0) {
                    shared_sum = rsqrtf(sum * inv_dim + eps);
                }
                __syncthreads();
                sum = shared_sum;
                Load(x_vec, weights + di * vec_size);
                PRAGMA_UNROLL
                for (int i = 0; i < vec_size; ++i) {
                    r_vec[i] = static_cast<T>(((float)r_vec[i] * sum)) * x_vec[i];
                }
                Store(buf + idx, r_vec);
            }
        }
    }

    __syncthreads();

    if (threadIdx.x < peers) {
        sem.SignalAndWait(relaxed);
    }

    __syncthreads();

    if (aligned || is_valid) {
        for (int p = 0; p < peers; ++p) {
            const int peer      = p + rank < peers ? p + rank : p + rank - peers;
            const int peer_rank = peer < rank ? peer : peer + 1;
            const int first     = slice * peer_rank;
            const int last      = min(count, first + slice);
            auto      chn       = cvta_generic_to_global(chns[peer]);
            for (int ti = first + bi; ti < last; ti += block_num) {
                const int idx = (ti * vdim + di) * vec_size;
                Vec       vec;
                Load(vec, chn + idx);
                Store(buf + idx, vec);
            }
        }
    }

    __syncthreads();

    if (threadIdx.x < peers) {
        // this and the `__syncthreads` above are used to block later kernels from modifying shared `buf` before all
        // ranks done copying from it
        sem.SignalAndWait(true);
        sem.Save(&semaphores[blockIdx.x * peers + threadIdx.x]);
    }
}

template<class T, int vec_size, int block_dim, bool aligned, class Peers, class Relaxed>
__global__ void AllreduceResidualBiasRMSnormKernel_Simple_v3(T*                                             buf,
                                                             T*                                             res,
                                                             const T*                                       bias,
                                                             const T*                                       weights,
                                                             Array<T*, kMaxNearPeers>                       chns,
                                                             mscclpp::SmDevice2DeviceSemaphoreDeviceHandle* semaphores,
                                                             int                                            rank,
                                                             Peers                                          peers,
                                                             int                                            slice,
                                                             int                                            count,
                                                             int                                            vdim,
                                                             float                                          inv_dim,
                                                             float                                          eps,
                                                             constant<vec_size>,
                                                             constant<block_dim>,
                                                             constant<aligned>,
                                                             Relaxed relaxed)
{
    const int bi        = blockIdx.x;
    const int block_num = gridDim.x;

    DeviceSemaphore sem;

    if (threadIdx.x < peers) {
        sem.Load(&semaphores[blockIdx.x * peers + threadIdx.x]);
        sem.SignalAndWait(relaxed);
    }

    __syncthreads();

    using Vec = Array<T, vec_size>;

    using namespace ops;

    const int  di       = threadIdx.x;
    const bool is_valid = di < vdim;

    if (aligned || is_valid) {

        const int first = rank * slice;
        const int last  = min(count, first + slice);

        __shared__ const T* chs[8];
        for (int p = 0; p < peers; ++p) {
            const int peer = p + rank < peers ? p + rank : p + rank - peers;
            chs[p]         = chns[peer];
        }

        for (int ti = first + bi; ti < last; ti += block_num) {
            const int idx = (ti * vdim + di) * vec_size;
            Vec       acc;
            Load(acc, buf + idx);
            for (int p = 0; p < peers; ++p) {
                Vec tmp;
                Load(tmp, chs[p] + idx);
                acc = acc + tmp;
            }
            Vec r_vec, x_vec;
            Load(r_vec, res + idx);
            r_vec = r_vec + acc;
            if (bias) {
                Load(x_vec, bias + di * vec_size);
                r_vec = r_vec + x_vec;
            }
            Store(res + idx, r_vec);
            float sum{};
            PRAGMA_UNROLL
            for (int i = 0; i < vec_size; ++i) {
                sum += (float)r_vec[i] * (float)r_vec[i];
            }
            using BlockReduce = hipcub::BlockReduce<float, block_dim>;
            __shared__ typename BlockReduce::TempStorage temp_storage;
            __shared__ float                             shared_sum;
            sum = BlockReduce{temp_storage}.Sum(sum);
            if (di == 0) {
                shared_sum = rsqrtf(sum * inv_dim + eps);
            }
            __syncthreads();
            sum = shared_sum;
            Load(x_vec, weights + di * vec_size);
            PRAGMA_UNROLL
            for (int i = 0; i < vec_size; ++i) {
                r_vec[i] = static_cast<T>(((float)r_vec[i] * sum)) * x_vec[i];
            }
            Store(buf + idx, r_vec);
        }
    }

    __syncthreads();

    if (threadIdx.x < peers) {
        sem.SignalAndWait(relaxed);
    }

    __syncthreads();

    if (aligned || is_valid) {
        for (int p = 0; p < peers; ++p) {
            const int peer      = p + rank < peers ? p + rank : p + rank - peers;
            const int peer_rank = peer < rank ? peer : peer + 1;
            const int first     = slice * peer_rank;
            const int last      = min(count, first + slice);
            auto      chn       = cvta_generic_to_global(chns[peer]);
            Vec       vec;
            for (int ti = first + bi; ti < last; ti += block_num) {
                const int idx = (ti * vdim + di) * vec_size;
                Load(vec, chn + idx);
                Store(buf + idx, vec);
            }
        }
    }

    __syncthreads();

    if (threadIdx.x < peers) {
        sem.SignalAndWait(relaxed);
        sem.Save(&semaphores[blockIdx.x * peers + threadIdx.x]);
    }
}

void CustomComm::AllreduceResidualBiasRMSnorm(void*        hidden,
                                              void*        residual,
                                              const void*  bias,
                                              const void*  weights,
                                              float        eps,
                                              int          dim,
                                              int          token_num,
                                              DataType     dtype,
                                              hipStream_t stream)
{

    auto invoke = [&](auto t) {
        using T                = decltype(t);
        const auto    near     = get_near((T*)hidden);
        const int     slice    = (token_num + world_size_ - 1) / world_size_;
        const int     count    = token_num;
        constexpr int threads  = 1024;
        const int     blocks   = std::min(token_num, 48);
        constexpr int vec_size = sizeof(uint4) / sizeof(T);
        AllreduceResidualBiasRMSnormKernel_Simple_v2<<<blocks, threads, 0, stream>>>((T*)hidden,
                                                                                     (T*)residual,
                                                                                     (const T*)bias,
                                                                                     (const T*)weights,
                                                                                     near,
                                                                                     device_semaphores_,
                                                                                     rank_,
                                                                                     world_size_ - 1,
                                                                                     slice,
                                                                                     count,
                                                                                     dim / vec_size,
                                                                                     1.f / dim,
                                                                                     eps,
                                                                                     constant<vec_size>{},
                                                                                     constant<threads>{},
                                                                                     constant<false>{},
                                                                                     std::true_type{});
    };

    const size_t elemsize = get_elem_size(dtype);
    const size_t bytesize = elemsize * token_num * dim;
    const int    vec_size = sizeof(uint4) / elemsize;

    if (dim % vec_size == 0 && bytesize > (1 << 20)) {
        switch (dtype) {
            case DataType::TYPE_FP16:
                return invoke(half{});
            case DataType::TYPE_BF16:
                return invoke(hip_bfloat16{});
            default:
                FT_CHECK(0);
        }
    };

    // fallback
    AllReduceSum(hidden, hidden, token_num * dim, dtype, stream);
    invokeResidualBiasRMSNorm(hidden, residual, weights, bias, dtype, dim, token_num, eps, stream);
}

}  // namespace turbomind