// Copyright (c) OpenMMLab. All rights reserved.

#include <memory>
#include <mutex>
#include <vector>

#include <hip/hip_runtime.h>

#include "src/turbomind/comm/comm.h"
#include "src/turbomind/comm/native/bootstrap.h"
#include "src/turbomind/comm/native/native_comm.h"

#include "src/turbomind/utils/cuda_utils.h"
#include "src/turbomind/utils/logger.h"

namespace turbomind::comm {

NativeComm::NativeComm(std::shared_ptr<LocalBootstrap> bootstrap): Comm{bootstrap->getNranks(), bootstrap->getRank()}
{
    bootstrap_ = bootstrap;

    // Exchange device ordinals
    ordinals_.resize(world_size_);
    check_cuda_error(hipGetDevice(&ordinals_[rank_]));
    bootstrap_->allGather(ordinals_.data(), sizeof(int));

    // Prepare allocation properties & granularity
    alloc_prop_.type          = hipMemAllocationTypePinned;
    alloc_prop_.location.type = hipMemLocationTypeDevice;
    alloc_prop_.location.id   = ordinals_[rank_];
    CUDRVCHECK(hipMemGetAllocationGranularity(&alloc_granularity_, &alloc_prop_, hipMemAllocationGranularityRecommended));

    // Prepare access descriptors
    alloc_access_descs_.resize(world_size_);
    for (int r = 0; r < world_size_; ++r) {
        alloc_access_descs_[r].location.id   = ordinals_[r];
        alloc_access_descs_[r].location.type = hipMemLocationTypeDevice;
        alloc_access_descs_[r].flags         = hipMemAccessFlagsProtReadWrite;
    }
}

NativeComm::~NativeComm()
{
    Deregister(scratch_buff_);
    Deregister(packet_buff_);
    // device_semaphores_ is not registered

    Free(scratch_buff_);
    Free(packet_buff_);
    Free(device_semaphore_data_);

    for (const auto& [ptr, _] : registered_memories_) {
        TM_LOG_WARNING("[TM][COMM][%d] Buffer %p is not deregistered", rank_, ptr);
    }

    for (const auto& [ptr, alloc] : allocations_) {
        TM_LOG_WARNING("[TM][COMM][%d] Allocation (%p, %lu) is not freed", rank_, ptr, alloc.size);
    }

    // check_cuda_error(hipFreeAsync(device_syncer_, 0));
    check_cuda_error(hipFreeAsync(device_semaphores_, 0));
    check_cuda_error(hipStreamSynchronize(0));
}

void NativeComm::Initialize()
{
    const int flags_size = 3 * sizeof(uint64_t) * kChannelsPerConn * (world_size_ - 1);
    uint64_t* flags      = (uint64_t*)Allocate(flags_size);
    check_cuda_error(hipMemsetAsync(flags, 0, flags_size));
    device_semaphore_data_ = flags;

    std::vector<uint64_t*> all_flags(world_size_);
    all_flags[rank_] = flags;
    bootstrap_->allGather(all_flags.data(), sizeof(uint64_t*));

    const int peers = world_size_ - 1;

    std::vector<mscclpp::SmDevice2DeviceSemaphoreDeviceHandle> device_semaphores;
    for (int c = 0; c < kChannelsPerConn; ++c) {
        for (int r = 0; r < world_size_; ++r) {
            if (r != rank_) {
                const int p     = r < rank_ ? r : r - 1;
                const int inv_p = Rank{rank_, peers}.inverse_peer(p);
                //
                mscclpp::SmDevice2DeviceSemaphoreDeviceHandle handle{};
                handle.inboundSemaphoreId         = flags + c * peers + p;                                  // local
                handle.outboundSemaphoreId        = handle.inboundSemaphoreId + kChannelsPerConn * peers;   // local
                handle.expectedInboundSemaphoreId = handle.outboundSemaphoreId + kChannelsPerConn * peers;  // local
                handle.remoteInboundSemaphoreId   = all_flags[r] + c * peers + inv_p;                       // near
                device_semaphores.push_back(handle);
            }
        }
    }

    check_cuda_error(hipMallocAsync(
        &device_semaphores_, sizeof(mscclpp::SmDevice2DeviceSemaphoreDeviceHandle) * device_semaphores.size(), 0));

    check_cuda_error(hipMemcpyAsync(device_semaphores_,
                                     device_semaphores.data(),
                                     sizeof(mscclpp::SmDevice2DeviceSemaphoreDeviceHandle) * device_semaphores.size(),
                                     hipMemcpyHostToDevice));

    packet_buff_ = Allocate(kPacketBuffSize);
    check_cuda_error(hipMemsetAsync(packet_buff_, 0, kPacketBuffSize));

    scratch_buff_ = Allocate(kScratchBuffSize);
    check_cuda_error(hipMemsetAsync(scratch_buff_, 0, kScratchBuffSize));

    check_cuda_error(hipStreamSynchronize(0));

    Register(packet_buff_, kPacketBuffSize);
    Register(scratch_buff_, kScratchBuffSize);
}

void* NativeComm::Allocate(size_t size)
{
    hipMemGenericAllocationHandle_t handle{};
    size = (size + alloc_granularity_ - 1) / alloc_granularity_ * alloc_granularity_;
    CUDRVCHECK(hipMemCreate(&handle, size, &alloc_prop_, 0));
    hipDeviceptr_t dptr{};
    CUDRVCHECK(hipMemAddressReserve(&dptr, size, 0, 0, 0));
    CUDRVCHECK(hipMemMap(dptr, size, 0, handle, 0));
    CUDRVCHECK(hipMemSetAccess(dptr, size, alloc_access_descs_.data(), alloc_access_descs_.size()));
    void* ptr = reinterpret_cast<void*>(dptr);
    allocations_.emplace(ptr, Allocation{handle, size});
    return ptr;
}

void NativeComm::Free(void* ptr)
{
    if (auto it = allocations_.find(ptr); it != allocations_.end()) {
        auto allocation = it->second;
        auto dptr       = reinterpret_cast<hipDeviceptr_t>(ptr);
        CUDRVCHECK(hipMemUnmap(dptr, allocation.size));
        CUDRVCHECK(hipMemRelease(allocation.handle));
        CUDRVCHECK(hipMemAddressFree(dptr, allocation.size));
        allocations_.erase(it);
    }
    else {
        TM_LOG_WARNING("[TM][COMM][%d] Freeing %p which is not allocated by this module", rank_, ptr);
    }
}

void NativeComm::Register(void* ptr, size_t size)
{
    if (!registered_memories_.count(ptr)) {
        using Buffer = std::pair<void*, size_t>;

        std::vector<Buffer> buffers(world_size_);
        buffers[rank_] = {ptr, size};
        bootstrap_->allGather(buffers.data(), sizeof(Buffer));

        std::vector<Buffer> bufs;
        for (int i = 0; i < world_size_; ++i) {
            if (i != rank_) {
                bufs.push_back(buffers[i]);
            }
        }

        registered_memories_.emplace(ptr, std::move(bufs));
    }
    else {
        TM_LOG_WARNING("[TM][COMM][%d] Duplicated registration on (%p, %lu)", rank_, ptr, size);
    }
}

void NativeComm::Deregister(void* ptr)
{
    if (int erased = registered_memories_.erase(ptr); erased == 0) {
        TM_LOG_WARNING("[TM][COMM][%d] Deregistering non-registered address %p", rank_, ptr);
    }
}

int NativeComm::Query(QueryAttr attr) const noexcept
{
    if (attr == kHasAllGather2D) {
        return 1;
    }
    return 0;
}

Array<void*, kMaxNearPeers> NativeComm::get_near_impl(void* ptr)
{
    auto& memories = registered_memories_.at(ptr);
    FT_CHECK(memories.size() <= kMaxNearPeers);
    Array<void*, kMaxNearPeers> ret{};
    for (size_t i = 0; i < memories.size(); ++i) {
        ret[i] = memories[i].first;
    }
    return ret;
}

class LocalGroupId: public GroupId {
public:
    void Initialize() override
    {
        internal_ = std::make_shared<Internal>();
    }

    void Export(std::ostream& os) override
    {
        FT_CHECK((bool)internal_);  // `Initialize` must come befor `Export`

        const void* ptr = this;
        os.write((const char*)&ptr, sizeof(ptr));
    }

    void Import(std::istream& is) override
    {
        void* ptr{};
        is.read((char*)&ptr, sizeof(ptr));
        internal_ = reinterpret_cast<LocalGroupId*>(ptr)->internal_;

        FT_CHECK((bool)internal_);
    }

    std::unique_ptr<Comm> CreateCommunicator(int rank, int world_size) override
    {
        auto init_shared_state = [&] {  //
            internal_->state = std::make_shared<LocalBootstrap::State>(world_size);
        };

        FT_CHECK((bool)internal_);

        // One of the rank initialize the shared state
        std::call_once(internal_->flag, init_shared_state);

        FT_CHECK((bool)internal_->state);

        auto bootstrap = std::make_shared<LocalBootstrap>(world_size, rank, internal_->state);

        auto comm = std::make_unique<NativeComm>(bootstrap);

        comm->Initialize();

        return comm;
    }

private:
    struct Internal {
        std::once_flag                         flag;
        std::shared_ptr<LocalBootstrap::State> state;
    };

private:
    std::shared_ptr<Internal> internal_;
};

std::unique_ptr<GroupId> CreateNativeGroupId()
{
    return std::make_unique<LocalGroupId>();
}

}  // namespace turbomind::comm
