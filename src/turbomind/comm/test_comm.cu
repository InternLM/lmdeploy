#include "hip/hip_runtime.h"
// Copyright (c) OpenMMLab. All rights reserved.

#include <algorithm>
#include <cmath>
#include <cstdio>
#include <numeric>
#include <optional>
#include <ostream>
#include <random>
#include <sstream>
#include <thread>

#include <hip/hip_runtime_api.h>
#include <hip/hip_runtime.h>

#include "src/turbomind/comm/barrier.h"
#include "src/turbomind/comm/comm.h"

using namespace turbomind::comm;

[[maybe_unused]] static constexpr bool is_ncu = 0;

struct Context {

    hipStream_t stream;

    hipEvent_t ev_start;
    hipEvent_t ev_end;

    std::vector<void*> buffers;

    template<class F>
    float exec(F func)
    {
        hipStreamSynchronize(stream);
        hipEventRecord(ev_start, stream);

        func(stream);

        hipEventRecord(ev_end, stream);
        hipEventSynchronize(ev_end);
        float ms{};
        hipEventElapsedTime(&ms, ev_start, ev_end);
        return ms;
    }

    template<class T>
    T* malloc(size_t count)
    {
        T* data;
        hipMallocAsync(&data, sizeof(T) * count, stream);
        buffers.push_back(data);
        return data;
    }

    template<class T>
    void copy_n(const T* src, size_t count, T* dst)
    {
        hipMemcpyAsync(dst, src, sizeof(T) * count, hipMemcpyDefault, stream);
    }

    Context(int device_id)
    {
        hipSetDevice(device_id);
        hipStreamCreateWithFlags(&stream, hipStreamNonBlocking);
        hipEventCreate(&ev_start);
        hipEventCreate(&ev_end);
    }
    ~Context()
    {
        for (auto& p : buffers) {
            hipFreeAsync(p, stream);
            p = {};
        }
        hipStreamSynchronize(stream);

        hipEventDestroy(ev_end);
        hipEventDestroy(ev_start);
        hipStreamDestroy(stream);
    }
};

struct TestComm {

    std::vector<std::unique_ptr<Comm>> comm_;

    int              warmup_;
    int              iters_;
    std::vector<int> tokens_;
    size_t           max_tokens_;

    std::optional<Barrier> barrier_;

    static auto Init(int world_size, const std::string& backend) -> std::vector<std::unique_ptr<Comm>>
    {
        std::unique_ptr<GroupId> group_id = CreateGroupId(backend);
        std::string              group_id_str;
        if (1) {  // master
            group_id->Initialize();
            std::stringstream ss;
            group_id->Export(ss);
            group_id_str = ss.str();
        }

        std::vector<std::unique_ptr<Comm>> comm(world_size);

        auto init = [&](int rank) {
            hipSetDevice(rank);
            std::stringstream        ss(group_id_str);
            std::unique_ptr<GroupId> group_id = CreateGroupId(backend);
            group_id->Import(ss);
            comm[rank] = group_id->CreateCommunicator(rank, world_size);
        };

        std::vector<std::thread> threads;
        for (int i = 0; i < world_size; ++i) {
            threads.emplace_back(init, i);
        }
        for (auto& t : threads) {
            t.join();
        }

        return comm;
    }

    void Run(int hidden_dim, int vocab_size, int tp, int warmup, int iters, std::vector<int> tokens)
    {
        int device_num{};
        hipGetDeviceCount(&device_num);

        std::cout << "Device count: " << device_num << "\n";

        if (tp < 0) {
            tp = device_num;
        }

        barrier_.emplace(device_num);

        comm_ = Init(device_num, "native");

        warmup_ = warmup;
        iters_  = iters;
        tokens_ = tokens;

        max_tokens_ = *std::max_element(tokens_.begin(), tokens_.end());

        TestAllReduce<half>(hidden_dim);
        TestAllreduceResidualBiasRMSnorm<half>(hidden_dim);
        TestAllGather<half>(hidden_dim / tp);  // tp embedding
        TestAllGather<float>(vocab_size / tp);
    }

    template<class T>
    void TestAllReduce(size_t dim)
    {
        std::mt19937                       gen{};
        std::uniform_int_distribution<int> dist{0, 31};  // 5 mantissa bits
        std::vector<std::vector<T>>        data;
        std::vector<T>                     ref_data(max_tokens_ * dim);

        std::cout << "preparing data ... " << std::flush;

        for (int i = 0; i < (int)comm_.size(); ++i) {
            auto& rank_data = data.emplace_back(ref_data.size());
            for (size_t j = 0; j < rank_data.size(); ++j) {
                rank_data[j] = T(dist(gen));
                ref_data[j] += rank_data[j];
            }
        }

        std::cout << "done.\n";

        auto func = [&](Comm& comm) {
            const int    rank = comm.rank();
            Context      ctx{rank};
            const size_t max_count   = ref_data.size();
            T*           d_rank_data = ctx.malloc<T>(max_count);
            T*           d_tmp       = (T*)comm.Allocate(sizeof(T) * max_count);
            comm.Register(d_tmp, sizeof(T) * max_count);
            ctx.copy_n(data[rank].data(), max_count, d_rank_data);

            [[maybe_unused]] auto verify = [&](auto count) {
                std::vector<T> res(count);
                ctx.copy_n(d_tmp, count, res.data());
                hipStreamSynchronize(ctx.stream);
                size_t diff = 0;
                for (size_t i = 0; i < count; ++i) {
                    diff += res[i] != ref_data[i];
                    if (diff == 1) {
                        printf("%d: %f vs %f\n", (int)i, (float)res[i], (float)ref_data[i]);
                    }
                }
                if (diff) {
                    printf("[rank %d] count = %d, diff = %lu\n", rank, (int)count, diff);
                    std::this_thread::sleep_for(std::chrono::seconds(1));
                    std::abort();
                }
            };

            std::vector<float> deltas;
            for (const auto& n : tokens_) {
                const size_t count = (size_t)n * dim;
                auto&        delta = deltas.emplace_back();

                barrier_->arrive_and_wait();

                for (int i = 0; i < warmup_ + iters_; ++i) {
                    ctx.copy_n(d_rank_data, count, d_tmp);
                    auto ms = ctx.exec([&](auto stream) {  //
                        if (is_ncu && i == warmup_) {
                            barrier_->arrive_and_wait();
                            if (rank == 0) {
                                hipProfilerStart();
                            }
                            barrier_->arrive_and_wait();
                        }

                        comm.AllReduceSum(d_tmp, d_tmp, count, stream);

                        if (is_ncu && i == warmup_) {
                            barrier_->arrive_and_wait();
                            if (rank == 0) {
                                hipProfilerStop();
                            }
                            barrier_->arrive_and_wait();
                        }
                    });
                    if (i >= warmup_) {
                        delta += ms;
                    }
                    // verify(count);
                }
                verify(count);
            }

            if (rank == 0) {
                SummaryHeader("allreduce", dim, comm.world_size());
                for (size_t i = 0; i < tokens_.size(); ++i) {
                    const float  avg   = deltas[i] / iters_;
                    const size_t count = tokens_[i] * dim;
                    const float  algbw = sizeof(T) * count / 1e9f / avg * 1000.f;
                    const float  busbw = algbw * (2 * (comm.world_size() - 1)) / comm.world_size();
                    SummaryEntry(tokens_[i], count, sizeof(T), avg, algbw, busbw);
                }
            }

            comm.Deregister(d_tmp);
            comm.Free(d_tmp);
        };

        std::vector<std::thread> threads;
        for (auto& comm : comm_) {
            threads.emplace_back(func, std::ref(*comm));
        }
        for (auto& t : threads) {
            t.join();
        }
    }

    template<class T>
    void TestAllreduceResidualBiasRMSnorm(size_t dim)
    {
        std::mt19937                       gen{};
        std::uniform_int_distribution<int> dist{0, 31};  // 5 mantissa bits
        std::vector<std::vector<T>>        data;
        std::vector<T>                     ref_data(max_tokens_ * dim);
        std::vector<T>                     residual(max_tokens_ * dim);
        std::vector<T>                     ref_residual(max_tokens_ * dim);
        std::vector<T>                     weight(dim);
        std::vector<T>                     bias(dim);
        constexpr float                    eps      = 1e-5;
        constexpr bool                     has_bias = true;

        std::cout << "preparing data ... " << std::flush;

        for (size_t i = 0; i < dim; ++i) {
            weight[i] = T(dist(gen));
        }

        if (has_bias) {
            for (size_t i = 0; i < dim; ++i) {
                bias[i] = T(dist(gen));
            }
        }

        for (int i = 0; i < (int)comm_.size(); ++i) {
            auto& rank_data = data.emplace_back(ref_data.size());
            for (size_t j = 0; j < rank_data.size(); ++j) {
                rank_data[j] = T(dist(gen));
                ref_data[j] += rank_data[j];  // sum over all ranks
            }
        }

        for (size_t i = 0; i < max_tokens_; ++i) {
            float sum = 0.f;
            for (size_t d = 0; d < dim; ++d) {
                const size_t index  = i * dim + d;
                residual[index]     = T(dist(gen));
                ref_residual[index] = residual[index] + ref_data[index] + bias[d];  // r' <- r + (h + b)
                sum += (float)ref_residual[index] * (float)ref_residual[index];
            }
            sum = rsqrtf(sum / dim + eps);
            for (size_t d = 0; d < dim; ++d) {
                const size_t index = i * dim + d;
                float        tmp   = (float)ref_residual[index];
                ref_data[index]    = tmp * sum * (float)weight[d];  // h' <- norm(r) * w
            }
        }

        std::cout << "done.\n";

        auto func = [&](Comm& comm) noexcept {
            const int    rank = comm.rank();
            Context      ctx{rank};
            const size_t max_count   = ref_data.size();
            T*           d_rank_data = ctx.malloc<T>(max_count);
            T*           d_residual  = ctx.malloc<T>(max_count);
            T*           d_bias      = ctx.malloc<T>(dim);
            T*           d_weight    = ctx.malloc<T>(dim);
            T*           d_tmp_res   = ctx.malloc<T>(max_count);
            T*           d_tmp_data  = (T*)comm.Allocate(sizeof(T) * max_count);

            comm.Register(d_tmp_data, sizeof(T) * max_count);

            ctx.copy_n(data[rank].data(), max_count, d_rank_data);
            ctx.copy_n(residual.data(), max_count, d_residual);
            ctx.copy_n(bias.data(), dim, d_bias);
            ctx.copy_n(weight.data(), dim, d_weight);

            [[maybe_unused]] auto verify = [&](auto token_num) {
                const size_t   count = (size_t)token_num * dim;
                std::vector<T> h_data(count);
                std::vector<T> h_res(count);
                ctx.copy_n(d_tmp_data, count, h_data.data());
                ctx.copy_n(d_tmp_res, count, h_res.data());
                hipStreamSynchronize(ctx.stream);
                const int    world_size = comm.world_size();
                const size_t slice      = (token_num + world_size - 1) / world_size * dim;
                const size_t first      = rank * slice;
                const size_t last       = std::min(first + slice, count);
                size_t       res_diff   = 0;
                for (size_t i = first; i < last; ++i) {
                    int is_diff = !(h_res[i] == ref_residual[i]);
                    if (!res_diff && is_diff) {
                        printf("[rank %d], %d: %f vs %f\n",
                               rank,
                               (int)(i - first),
                               (float)h_res[i],
                               (float)ref_residual[i]);
                    }
                    res_diff += is_diff;
                }
                float data_diff = 0;
                for (size_t i = 0; i < count; ++i) {
                    float diff = (float)h_data[i] - (float)ref_data[i];
                    data_diff += std::abs(diff);
                }
                data_diff /= count;
                if (rank == 0) {
                    printf("[rank %d] count = %d, data_diff = %f\n", rank, (int)token_num, data_diff);
                }
                if (res_diff || data_diff > 0.1f || std::isnan(data_diff)) {
                    printf("[rank %d] count = %d, res_diff = %lu, data_diff = %f\n",
                           rank,
                           (int)token_num,
                           res_diff,
                           data_diff);
                    std::this_thread::sleep_for(std::chrono::seconds(5));
                    std::abort();
                }
            };

            std::vector<float> deltas;
            for (const auto& n : tokens_) {
                const size_t count = (size_t)n * dim;
                auto&        delta = deltas.emplace_back();
                barrier_->arrive_and_wait();
                for (int i = 0; i < warmup_ + iters_; ++i) {

                    ctx.copy_n(d_rank_data, count, d_tmp_data);
                    ctx.copy_n(d_residual, count, d_tmp_res);

                    auto ms = ctx.exec([&](auto stream) {  //
                        if (is_ncu && i == warmup_) {
                            barrier_->arrive_and_wait();
                            if (rank == 0) {
                                hipProfilerStart();
                            }
                            barrier_->arrive_and_wait();
                        }
                        comm.AllreduceResidualBiasRMSnorm(
                            d_tmp_data, d_tmp_res, has_bias ? d_bias : nullptr, d_weight, eps, dim, n, stream);

                        if (is_ncu && i == warmup_) {
                            barrier_->arrive_and_wait();
                            if (rank == 0) {
                                hipProfilerStop();
                            }
                            barrier_->arrive_and_wait();
                        }
                    });
                    if (i >= warmup_) {
                        delta += ms;
                    }
                    // verify(n);
                }
                verify(n);
            }

            comm.Deregister(d_tmp_data);
            comm.Free(d_tmp_data);

            if (rank == 0) {
                SummaryHeader("allreduce | rmsnorm", dim, comm.world_size());
                for (size_t i = 0; i < tokens_.size(); ++i) {
                    const float  avg   = deltas[i] / iters_;
                    const size_t count = tokens_[i] * dim;
                    const float  algbw = sizeof(T) * count / 1e9f / avg * 1000.f;
                    const float  busbw = algbw * (2 * (comm.world_size() - 1)) / comm.world_size();
                    SummaryEntry(tokens_[i], count, sizeof(T), avg, algbw, busbw);
                }
            }
        };

        std::vector<std::thread> threads;
        for (auto& comm : comm_) {
            threads.emplace_back(func, std::ref(*comm));
        }
        for (auto& t : threads) {
            t.join();
        }
    }

    template<class T>
    void TestAllGather(size_t dim)
    {
        std::mt19937                       gen{};
        std::uniform_int_distribution<int> dist{0, 100};
        std::vector<std::vector<T>>        data;

        std::cout << "preparing data ... " << std::flush;

        for (int i = 0; i < (int)comm_.size(); ++i) {
            auto& rank_data = data.emplace_back(max_tokens_ * dim);
            for (size_t j = 0; j < rank_data.size(); ++j) {
                rank_data[j] = T(dist(gen));
            }
        }

        std::cout << "done.\n";

        auto func = [&](Comm& comm) {
            const int    rank       = comm.rank();
            const int    world_size = comm.world_size();
            Context      ctx{rank};
            const size_t max_count   = max_tokens_ * dim;
            T*           d_rank_data = ctx.malloc<T>(max_count);
            T*           d_tmp       = (T*)comm.Allocate(sizeof(T) * max_count * world_size);
            comm.Register(d_tmp, sizeof(T) * max_count * world_size);
            ctx.copy_n(data[rank].data(), max_count, d_rank_data);
            [[maybe_unused]] auto verify = [&](int64_t count) {
                auto           total_count = count * world_size;
                std::vector<T> res(total_count);
                ctx.copy_n(d_tmp, total_count, res.data());
                hipStreamSynchronize(ctx.stream);
                size_t diff = 0;
                for (int r = 0; r < world_size; ++r) {
                    for (auto i = 0; i < count; ++i) {
                        diff += res[r * count + i] != data[r][i];
                        if (diff == 1) {
                            printf("%d: %f vs %f\n", (int)i, (float)res[r * count + i], (float)data[r][i]);
                        }
                    }
                }
                if (diff) {
                    printf("[rank %d] count = %d, diff = %lu\n", rank, (int)count, diff);
                    std::this_thread::sleep_for(std::chrono::seconds(1));
                    std::abort();
                }
            };

            std::vector<float> deltas;
            for (const auto& n : tokens_) {
                const size_t count = (size_t)n * dim;  // dim = hidden_dim / tp
                auto&        delta = deltas.emplace_back();

                barrier_->arrive_and_wait();

                for (int i = 0; i < warmup_ + iters_; ++i) {
                    hipMemsetAsync(d_tmp, 0, sizeof(T) * count * comm.world_size(), ctx.stream);
                    ctx.copy_n(d_rank_data, count, d_tmp + rank * count);
                    auto ms = ctx.exec([&](auto stream) {  //
                        if (comm.Query(kHasAllGather2D)) {
                            comm.AllGather2D(d_tmp + rank * count, d_tmp, dim, count, dim, n, {1, 1}, stream);
                        }
                        else {
                            comm.AllGather(d_tmp + rank * count, d_tmp, count, stream);
                        }
                    });
                    if (i >= warmup_) {
                        delta += ms;
                    }
                    // verify(count);
                }

                verify(count);
            }

            if (rank == 0) {
                SummaryHeader("allgather", dim, comm.world_size());
                for (size_t i = 0; i < tokens_.size(); ++i) {
                    const float  avg   = deltas[i] / iters_;
                    const size_t count = comm.world_size() * tokens_[i] * dim;
                    const float  algbw = sizeof(T) * count / 1e9f / avg * 1000.f;
                    const float  busbw = algbw * (comm.world_size() - 1) / comm.world_size();
                    SummaryEntry(tokens_[i], count, sizeof(T), avg, algbw, busbw);
                }
            }

            comm.Deregister(d_tmp);
            comm.Free(d_tmp);
        };

        std::vector<std::thread> threads;
        for (auto& comm : comm_) {
            threads.emplace_back(func, std::ref(*comm));
        }
        for (auto& t : threads) {
            t.join();
        }
    }

    void SummaryHeader(const char* name, int dim, int world_size)
    {
        printf("[%s] dim %d tp %d warmup %d iters %d\n", name, dim, world_size, warmup_, iters_);
        printf("%15s%15s%15s%15s%15s%15s\n", "num", "count", "size", "time", "algbw", "busbw");
        printf("%15s%15s%15s%15s%15s%15s\n", "(tokens)", "(elements)", "(MB)", "(us)", "(GB/s)", "(GB/s)");
    }

    void SummaryEntry(int num, size_t count, size_t elem_size, float time, float algbw, float busbw)
    {
        float mb_size = count * elem_size / (1024.f * 1024);
        printf("%15d%15ld%15.2f%15.3f%15.3f%15.3f\n", num, count, mb_size, time * 1e3f, algbw, busbw);
    }
};

int main(int argc, char* argv[])
{
    TestComm test;

    test.Run(8192,  //
             128000,
             -1,
             10,
             100,
             //  {1024});
             // {512});
             //  {1, 2, 3, 4, 5, 6, 7, 8, 12, 16, 24, 32, 48, 64, 96, 128});
             //  {128, 256, 512, 1024, 2048, 4096, 8192});
             //  {8, 16, 24, 32, 48, 64, 96, 128, 192, 256, 384, 512, 768, 1024, 1536, 2048, 4096, 6144, 8192});
             //   {8192, 16384, 32768});
             //  {1, 2, 4, 8, 16, 24, 32, 48, 64, 96, 128, 192, 256, 384, 512, 768, 1024});
             {1,   2,   4,   6,   8,   12,   16,   24,   32,   48,   64,   96,  128,
              192, 256, 384, 512, 768, 1024, 1536, 2048, 3072, 4096, 6144, 8192});

    return 0;
}
