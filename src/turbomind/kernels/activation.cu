#include "hip/hip_runtime.h"

#include "src/turbomind/core/data_type.h"

#include "src/turbomind/kernels/activation.h"
#include "src/turbomind/kernels/core/array_ops.h"
#include "src/turbomind/kernels/core/common.h"

namespace turbomind {

template<class T>
struct SiluGptOss {
    __device__ T operator()(T gate, T up) const noexcept
    {
        gate = __hmin((T)7.f, gate);
        up   = __hmax((T)-7.f, __hmin((T)7.f, up));
        return static_cast<T>(fdividef((float)gate, 1.f + expf((float)-gate * 1.702f)) * (1.f + (float)up));
    }
};

template<class T>
struct Silu {
    __device__ T operator()(T gate, T up) const noexcept
    {
        return static_cast<T>(fdividef((float)gate, 1.f + expf(-(float)gate)) * (float)up);
    }
};

template<int vec_size, class Activation, class T>
__global__ void ActivationKernel(
    T* gate_buf, const T* __restrict__ up_buf, Activation activation, int64_t stride, int token_num, int dim)
{
    if constexpr (TURBOMIND_ARCH_DTYPE_GUARD(data_type_v<T>)) {
        const int di = threadIdx.x + blockIdx.y * blockDim.x;
        const int ti = blockIdx.x;

        dim /= vec_size;

        if (di >= dim) {
            return;
        }

        using Vec = Array<T, vec_size>;

        auto p_gate = reinterpret_cast<Vec*>(gate_buf + ti * stride);
        auto p_up   = reinterpret_cast<const Vec*>(up_buf + ti * stride);

        Vec gate;
        Load(gate, (const T*)&p_gate[di]);

        Vec up;
        Ldg(up, (T*)&p_up[di]);

        PRAGMA_UNROLL
        for (int i = 0; i < vec_size; ++i) {
            gate[i] = activation(gate[i], up[i]);
        }

        Store((T*)&p_gate[di], gate);
    }
}

void Activation(Ref<Tensor> gate_, const Tensor& up, ActivationType type, hipStream_t stream)
{
    auto& gate = gate_.get();

    TM_CHECK(gate.shape() == up.shape());

    int num, dim;
    std::tie(num, dim) = gate.shapes(0, 1);

    auto invoke = [&](auto t, auto act) {
        using T = decltype(t);

        constexpr int vec_size = 4;
        constexpr int threads  = 512;

        const dim3 blocks(num, cdiv(dim, threads * vec_size));

        ActivationKernel<vec_size><<<blocks, threads, 0, stream>>>(gate.data<T>(),  //
                                                                   up.data<T>(),
                                                                   act,
                                                                   gate.stride(0),
                                                                   num,
                                                                   dim);
    };

    auto dispatch = [&](auto t) {
        using T = decltype(t);
        if (type == ActivationType::kSilu) {
            return invoke(t, Silu<T>{});
        }
        else if (type == ActivationType::kSiluGptOss) {
            return invoke(t, SiluGptOss<T>{});
        }
        else {
            TM_CHECK(0) << "unknown activation type: " << (int)type;
        }
    };

    TM_DISPATCH_PRIMARY_DTYPES(gate.dtype(), dispatch);
}

template<int vec_size, class Activation, class T>
__global__ void ActivationKernel(
    T* gate_up, const T* bias, const int* group_ids, int64_t stride, Activation activation, int token_num, int dim)
{
    if constexpr (TURBOMIND_ARCH_DTYPE_GUARD(data_type_v<T>)) {
        const int di = (threadIdx.x + blockIdx.y * blockDim.x) * vec_size;
        const int ti = blockIdx.x;
        const int gi = group_ids ? group_ids[ti] : 0;

        if (di >= dim) {
            return;
        }

        using Vec = Array<T, vec_size>;

        Vec gate_bias{}, up_bias{};
        Ldg(gate_bias, &bias[gi * stride + di]);
        Ldg(up_bias, &bias[gi * stride + dim + di]);

        Vec gate, up;
        Load(gate, &gate_up[ti * stride + di]);
        Load(up, &gate_up[ti * stride + dim + di]);

        {
            using namespace ops;
            gate = gate + gate_bias;
            up   = up + up_bias;
        }

        PRAGMA_UNROLL
        for (int i = 0; i < vec_size; ++i) {
            gate[i] = activation(gate[i], up[i]);
        }

        Store(&gate_up[ti * stride + di], gate);
    }
}

void Activation(Tensor&             gate_up,  //
                const Tensor&       bias,
                const Buffer_<int>& group_ids,
                ActivationType      type,
                hipStream_t        stream)
{
    const int num = gate_up.shape(0);
    const int dim = gate_up.shape(1) / 2;

    if (!bias) {
        Activation(gate_up.slice({0, 0}, {-1, dim}),  //
                   gate_up.slice({0, dim}, {-1, -1}),
                   type,
                   stream);
        return;
    }

    TM_CHECK_EQ(gate_up.shape(-1), bias.shape(-1));

    auto invoke = [&](auto t, auto act) {
        using T = decltype(t);

        constexpr int vec_size = 4;
        constexpr int threads  = 512;

        const dim3 blocks(num, cdiv(dim, threads * vec_size));

        ActivationKernel<vec_size><<<blocks, threads, 0, stream>>>(gate_up.data<T>(),  //
                                                                   bias.data_or((T*)nullptr),
                                                                   group_ids.data_or(nullptr),
                                                                   gate_up.stride(0),
                                                                   act,
                                                                   num,
                                                                   dim);
    };

    auto dispatch = [&](auto t) {
        using T = decltype(t);
        if (type == ActivationType::kSilu) {
            return invoke(t, Silu<T>{});
        }
        else if (type == ActivationType::kSiluGptOss) {
            return invoke(t, SiluGptOss<T>{});
        }
        else {
            TM_CHECK(0) << "unknown activation type: " << (int)type;
        }
    };

    TM_DISPATCH_PRIMARY_DTYPES(gate_up.dtype(), dispatch);
}

}  // namespace turbomind
