#include "hip/hip_runtime.h"
// Modified from xgrammar python/xgrammar/kernels/apply_token_bitmask_inplace_cuda.cu

/*
 * SPDX-FileCopyrightText: Copyright (c) 2025 NVIDIA CORPORATION & AFFILIATES. All rights reserved.
 * SPDX-License-Identifier: Apache-2.0
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 * http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

// clang-format off
#include <hip/hip_bf16.h>
#include <hip/hip_fp16.h>
#include <hip/hip_runtime.h>

#include "src/turbomind/core/context.h"
#include "src/turbomind/kernels/apply_token_bitmask_inplace_cuda.h"
// clang-format on

using namespace std;

#ifndef HIPRT_INF_FP16
#define HIPRT_INF_FP16 __ushort_as_half((unsigned short)0x7C00U)
#endif

#if __CUDA_ARCH__ >= 800
#ifndef CUDART_INF_BF16
#define CUDART_INF_BF16 __ushort_as_bfloat16((unsigned short)0x7F80U)
#endif
#endif

constexpr int32_t BITS_PER_BLOCK           = 32;
constexpr int32_t THREADS_PER_THREAD_BLOCK = 256;

template<typename T>
__device__ T NegativeInfinity()
{
    return -INFINITY;
}

template<>
__device__ __half NegativeInfinity<__half>()
{
    return -HIPRT_INF_FP16;
}

#if __CUDA_ARCH__ >= 800
template<>
__device__ __hip_bfloat16 NegativeInfinity<__hip_bfloat16>()
{
    return -CUDART_INF_BF16;
}
#endif

template<typename T, typename PackedT>
__device__ PackedT PackedNegativeInfinity()
{
    constexpr int kAlignment = sizeof(PackedT) / sizeof(T);
    T             packed[kAlignment];
#pragma unroll
    for (int i = 0; i < kAlignment; i++) {
        packed[i] = NegativeInfinity<T>();
    }
    return *reinterpret_cast<PackedT*>(packed);
}

template<typename T, typename PackedT, int32_t kBitsPerThread>
__global__ void __launch_bounds__(THREADS_PER_THREAD_BLOCK) LogitsBitmaskKernel(T* __restrict__ logits,
                                                                                const int32_t* __restrict__ bitmask,
                                                                                const int32_t* __restrict__ indices,
                                                                                int32_t vocab_size,
                                                                                int32_t logits_stride,
                                                                                int32_t bitmask_stride)
{
    constexpr int      kAlignment  = sizeof(PackedT) / sizeof(T);
    constexpr uint32_t kPackedMask = (1 << kAlignment) - 1;

    const int batch_idx = (indices == nullptr) ? blockIdx.y : indices[blockIdx.y];

    const int      block_offset      = blockIdx.x * THREADS_PER_THREAD_BLOCK * kBitsPerThread;
    T*             logits_gmem_ptr   = logits + batch_idx * logits_stride + block_offset;
    const int32_t* bitmask_gmem_ptr  = bitmask + batch_idx * bitmask_stride + block_offset / BITS_PER_BLOCK;
    const int      bitmask_inner_idx = threadIdx.x % (BITS_PER_BLOCK / kAlignment);
    T              logits_reg[kAlignment];

#pragma unroll
    for (int offset = threadIdx.x * kAlignment; offset < THREADS_PER_THREAD_BLOCK * kBitsPerThread;
         offset += THREADS_PER_THREAD_BLOCK * kAlignment) {
        if (block_offset + offset >= vocab_size) {
            break;
        }

        const uint32_t bitmask_val =
            (~bitmask_gmem_ptr[offset / BITS_PER_BLOCK] >> (bitmask_inner_idx * kAlignment)) & kPackedMask;

        if (bitmask_val == 0) {
            continue;
        }

        if (bitmask_val == kPackedMask) {
            *reinterpret_cast<PackedT*>(logits_gmem_ptr + offset) = PackedNegativeInfinity<T, PackedT>();
            continue;
        }

        *reinterpret_cast<PackedT*>(logits_reg) = *reinterpret_cast<PackedT*>(logits_gmem_ptr + offset);
#pragma unroll
        for (int i = 0; i < kAlignment; i++) {
            if (((bitmask_val >> i) & 1)) {
                logits_reg[i] = NegativeInfinity<T>();
            }
        }
        *reinterpret_cast<PackedT*>(logits_gmem_ptr + offset) = *reinterpret_cast<PackedT*>(logits_reg);
    }
}

template<typename T, typename = std::enable_if_t<std::is_integral<T>::value>>
constexpr auto CeilDiv(T numerator, T denominator)
{
    return (numerator + denominator - 1) / denominator;
}

template<typename T, typename PackedT>
void ApplyTokenBitmaskInplaceDispatchToBitsPerThread(T* __restrict__ logits,
                                                     const int32_t* __restrict__ bitmask,
                                                     const int32_t* __restrict__ indices,
                                                     int32_t vocab_size,
                                                     int32_t logits_stride,
                                                     int32_t bitmask_stride,
                                                     int32_t num_rows)
{
    constexpr int kAlignment          = sizeof(PackedT) / sizeof(T);
    const int32_t num_blocks_per_row  = CeilDiv(2048 / THREADS_PER_THREAD_BLOCK * 128, num_rows);
    const int32_t num_bits_per_thread = CeilDiv(vocab_size, THREADS_PER_THREAD_BLOCK * num_blocks_per_row);

    const dim3  block(THREADS_PER_THREAD_BLOCK);
    const auto& stream = turbomind::core::Context::stream();

    if (num_bits_per_thread <= 4 && kAlignment <= 4) {
        const dim3 grid(CeilDiv(vocab_size, THREADS_PER_THREAD_BLOCK * 4), num_rows);
        LogitsBitmaskKernel<T, PackedT, 4>
            <<<grid, block, 0, stream.handle()>>>(logits, bitmask, indices, vocab_size, logits_stride, bitmask_stride);
    }
    else if (num_bits_per_thread <= 8 && kAlignment <= 8) {
        const dim3 grid(CeilDiv(vocab_size, THREADS_PER_THREAD_BLOCK * 8), num_rows);
        LogitsBitmaskKernel<T, PackedT, 8>
            <<<grid, block, 0, stream.handle()>>>(logits, bitmask, indices, vocab_size, logits_stride, bitmask_stride);
    }
    else if (num_bits_per_thread <= 16 && kAlignment <= 16) {
        const dim3 grid(CeilDiv(vocab_size, THREADS_PER_THREAD_BLOCK * 16), num_rows);
        LogitsBitmaskKernel<T, PackedT, 16>
            <<<grid, block, 0, stream.handle()>>>(logits, bitmask, indices, vocab_size, logits_stride, bitmask_stride);
    }
    else {
        const dim3 grid(CeilDiv(vocab_size, THREADS_PER_THREAD_BLOCK * 32), num_rows);
        LogitsBitmaskKernel<T, PackedT, 32>
            <<<grid, block, 0, stream.handle()>>>(logits, bitmask, indices, vocab_size, logits_stride, bitmask_stride);
    }
}

template<typename T>
void ApplyTokenBitmaskInplaceDispatchToPackedT(T* __restrict__ logits,
                                               const int32_t* __restrict__ bitmask,
                                               const int32_t* __restrict__ indices,
                                               int32_t vocab_size,
                                               int32_t logits_stride,
                                               int32_t bitmask_stride,
                                               int32_t num_rows)
{
    if (logits_stride % (sizeof(float4) / sizeof(T)) == 0) {
        ApplyTokenBitmaskInplaceDispatchToBitsPerThread<T, float4>(
            logits, bitmask, indices, vocab_size, logits_stride, bitmask_stride, num_rows);
    }
    else {
        ApplyTokenBitmaskInplaceDispatchToBitsPerThread<T, T>(
            logits, bitmask, indices, vocab_size, logits_stride, bitmask_stride, num_rows);
    }
}

namespace turbomind {
using namespace turbomind::core;

void ApplyTokenBitmaskInplace(Tensor logits, Tensor bitmask, std::optional<Tensor> indices)
{
    std::pair<int32_t, int32_t> logits_shape =
        logits.ndim() == 2 ?
            std::make_pair(static_cast<int32_t>(logits.shape(0)), static_cast<int32_t>(logits.shape(1))) :
            std::make_pair(1, static_cast<int32_t>(logits.shape(0)));

    std::pair<int32_t, int32_t> bitmask_shape =
        bitmask.ndim() == 2 ?
            std::make_pair(static_cast<int32_t>(bitmask.shape(0)), static_cast<int32_t>(bitmask.shape(1))) :
            std::make_pair(1, static_cast<int32_t>(bitmask.shape(0)));

    int vocab_size = std::min(logits_shape.second, bitmask_shape.second * BITS_PER_BLOCK);

    int32_t  num_rows    = logits_shape.first;
    int32_t* indices_ptr = nullptr;
    if (indices) {
        num_rows    = indices->shape(0);
        indices_ptr = indices->data<int32_t>();
    }
    else {
        TM_CHECK(logits_shape.first == bitmask_shape.first) << "logits and bitmask must have the same batch size.";
    }

    // Currently we use only float logits.
    TM_CHECK(logits.dtype() == kFloat32);
    ApplyTokenBitmaskInplaceDispatchToPackedT(logits.data<float>(),
                                              bitmask.data<int32_t>(),
                                              indices_ptr,
                                              vocab_size,
                                              logits.stride(0),
                                              bitmask.stride(0),
                                              num_rows);
}
}  // namespace turbomind
