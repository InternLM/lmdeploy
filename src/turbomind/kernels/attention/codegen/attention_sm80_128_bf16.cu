#include "hip/hip_runtime.h"
// Copyright (c) OpenMMLab. All rights reserved.

#include "../attention_config.h"
#include "../attention_template.h"

namespace turbomind {

using namespace attention;

template void invokeAttention<typename AttentionConfig<arch::Sm80, hip_bfloat16, 128, CacheType::kLinear>::Kernel>(
    const AttentionParams<hip_bfloat16>& params);

template void invokeAttention<typename AttentionConfig<arch::Sm80, hip_bfloat16, 128, CacheType::kBlock>::Kernel>(
    const AttentionParams<hip_bfloat16>& params);

}  // namespace turbomind
