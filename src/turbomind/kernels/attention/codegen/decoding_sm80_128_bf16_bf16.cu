#include "hip/hip_runtime.h"
// Copyright (c) OpenMMLab. All rights reserved.

#include "../decoding_config.h"
#include "../decoding_template.h"

namespace turbomind {

using namespace attention;

template bool
invokeDecoding<Decoding<arch::Sm80, hip_bfloat16, hip_bfloat16, 1, 128>>(const AttentionParams<hip_bfloat16>& params);

template bool
invokeDecoding<Decoding<arch::Sm80, hip_bfloat16, hip_bfloat16, 4, 128>>(const AttentionParams<hip_bfloat16>& params);

template bool
invokeDecoding<Decoding<arch::Sm80, hip_bfloat16, hip_bfloat16, 6, 128>>(const AttentionParams<hip_bfloat16>& params);

template bool
invokeDecoding<Decoding<arch::Sm80, hip_bfloat16, hip_bfloat16, 8, 128>>(const AttentionParams<hip_bfloat16>& params);

}  // namespace turbomind
