#include "hip/hip_runtime.h"
// Copyright (c) OpenMMLab. All rights reserved.

#include "../decoding_config.h"
#include "../decoding_template.h"

namespace turbomind {

using namespace attention;

template bool invokeDecoding<Decoding<arch::Sm80, hip_bfloat16, uint4_t, 8, 128>>(const AttentionParams<hip_bfloat16>&);

template bool invokeDecoding<Decoding<arch::Sm80, hip_bfloat16, uint4_t, 16, 128>>(const AttentionParams<hip_bfloat16>&);

}  // namespace turbomind
