#include "hip/hip_runtime.h"
// Copyright (c) OpenMMLab. All rights reserved.

#include "../decoding_config.h"
#include "../decoding_template.h"

namespace turbomind {

using namespace attention;

template bool
invokeDecoding<Decoding<arch::Sm80, hip_bfloat16, hip_bfloat16, 1, 64>>(const AttentionParams<hip_bfloat16>& params);

template bool
invokeDecoding<Decoding<arch::Sm80, hip_bfloat16, hip_bfloat16, 2, 64>>(const AttentionParams<hip_bfloat16>& params);

template bool
invokeDecoding<Decoding<arch::Sm80, hip_bfloat16, hip_bfloat16, 8, 64>>(const AttentionParams<hip_bfloat16>& params);

template bool
invokeDecoding<Decoding<arch::Sm80, hip_bfloat16, hip_bfloat16, 16, 64>>(const AttentionParams<hip_bfloat16>& params);

}  // namespace turbomind
