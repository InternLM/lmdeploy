#include "hip/hip_runtime.h"
// Copyright (c) OpenMMLab. All rights reserved.

#include "attention.h"
#include "block.h"
#include "decoding.h"
#include "kv_cache_utils_v2.h"
#include "src/turbomind/kernels/attention/attention_params.h"
#include "src/turbomind/kernels/attention/reference.h"
#include "src/turbomind/models/llama/llama_utils.h"
#include "src/turbomind/utils/cuda_utils.h"
#include "test_utils.h"
#include <algorithm>
#include <cmath>
#include <iostream>
#include <numeric>
#include <random>
#include <thrust/universal_vector.h>
#include <utility>

using namespace turbomind;

// [b, h, s, d] : current -> stride_h=s, stride_s=1, stride_b=hs
// [cu_q, h, d] : qkvgemm -> stride_h=1, stride_s=h, stride_b=0
// [h, cu_s, d] : prefill -> stride_h=s, stride_s=1, stride_b=0

template<class T, class Tkv>
struct Config {
    int head_dim_;
    int head_num_;
    int block_len_;

    TM_HOST_DEVICE constexpr int t_bits() const
    {
        if constexpr (std::is_same_v<T, Tkv>) {
            return 0;
        }
        else {
            return bitsof<T>;
        }
    }

    TM_HOST_DEVICE constexpr int q_bits() const
    {
        return bitsof<Tkv>;
    }

    TM_HOST_DEVICE constexpr int head_dim() const
    {
        return head_dim_;
    }

    TM_HOST_DEVICE int head_num() const
    {
        return head_num_;
    }

    TM_HOST_DEVICE constexpr int block_len() const
    {
        return block_len_;
    }
};

// [S/S, H, S, D] <-> [S/b, H, b, D]
template<class Tkv, class T>
void TestBlocks(const thrust::universal_vector<T>& k_cache,        // [B, H, S, D]
                const thrust::universal_vector<T>& v_cache,        // [B, H, S, D]
                thrust::universal_vector<char>&    blocks,         // block data
                thrust::universal_vector<char*>&   k_ptrs,         // block ptrs
                thrust::universal_vector<int>&     cu_block_cnts,  // cumulative block counts
                const size_t                       head_num,
                const size_t                       head_dim,
                const size_t                       block_seq_len,
                const size_t                       batch_size,
                int                                quant_policy)
{
    const size_t seq_len  = k_cache.size() / (head_dim * head_num * batch_size);
    const size_t n_blocks = (seq_len + block_seq_len - 1) / block_seq_len;

    Config<T, Tkv> config{(int)head_dim, (int)head_num, (int)block_seq_len};
    block::Layout  layout{config};

    dump(layout);

    const size_t kHSD = head_num * seq_len * head_dim;

    std::cout << "batch_size = " << batch_size << ", seq_len = " << seq_len << ", block_size = " << block_seq_len
              << ", block_num = " << n_blocks << "\n";

    thrust::universal_vector<T> kv_cache(k_cache.size() * 2);  // [B, 2, H, S, D]

    {  // interleave K/V
        auto k_src = k_cache.begin();
        auto v_src = v_cache.begin();
        auto dst   = kv_cache.begin();
        for (size_t i = 0; i < batch_size; ++i) {
            dst = thrust::copy_n(k_src, kHSD, dst);
            dst = thrust::copy_n(v_src, kHSD, dst);
            k_src += kHSD;
            v_src += kHSD;
        }
    }

    // const int kHsD = head_num * block_seq_len * head_dim;

    // [B, S/s, 2, H, s, D]
    // blocks.resize(batch_size * n_blocks * 2 * kHsD);
    blocks.resize(batch_size * n_blocks * layout.block_size(1));
    thrust::fill(blocks.begin(), blocks.end(), NAN);
    k_ptrs.resize(batch_size * n_blocks + 1);  // +1 padding

    std::vector<size_t> idxs(batch_size * n_blocks);
    std::iota(idxs.begin(), idxs.end(), 0);

    std::random_device rd;
    std::mt19937       g(rd());
    std::shuffle(idxs.begin(), idxs.end(), g);

    for (size_t i = 0; i < idxs.size(); ++i) {
        // k_ptrs[i] = blocks.data().get() + idxs[i] * 2 * kHsD;
        k_ptrs[i] = blocks.data().get() + idxs[i] * layout.block_size(1);
    }

    thrust::universal_vector<int> seq_lens(batch_size);
    thrust::universal_vector<int> cu_seq_lens(batch_size + 1);
    thrust::fill(seq_lens.begin(), seq_lens.end(), seq_len);
    for (size_t i = 0; i <= batch_size; ++i) {
        cu_seq_lens[i] = i * seq_len;
    }

    std::vector<int> n_blocks_vec(batch_size + 1, n_blocks);
    cu_block_cnts.resize(batch_size + 1);
    std::exclusive_scan(n_blocks_vec.begin(), n_blocks_vec.end(), cu_block_cnts.begin(), 0);

    hipDeviceSynchronize();

    // [B, 2H, S, D] -> [B, S/s] x [2H, s, D]
    for (int i = 0; i < 1; ++i) {
        // (B, 2, H, S, D) -> blocks
        invokeProcessKV_v2(k_ptrs.data().get(),
                           kv_cache.data().get(),
                           kv_cache.data().get() + head_num * seq_len * head_dim,
                           (T*)nullptr,
                           (T*)nullptr,
                           cu_seq_lens.data().get(),
                           cu_seq_lens.data().get(),
                           cu_block_cnts.data().get(),
                           nullptr,
                           1.,
                           2 * head_num * seq_len,
                           0,
                           seq_len,
                           1,
                           block_seq_len,
                           0,
                           seq_len,
                           head_num,
                           head_dim,
                           batch_size,
                           quant_policy);
    }

    thrust::universal_vector<T> kv_cache_2(kv_cache.size());

    // round trip test
    for (int i = 0; i < 1; ++i) {
        // kv_cache_2 is [B, 2, H, S, D]
        invokeFlattenKV_v2(kv_cache_2.data().get(),
                           kv_cache_2.data().get() + head_num * seq_len * head_dim,
                           k_ptrs.data().get(),
                           cu_seq_lens.data().get(),
                           cu_block_cnts.data().get(),
                           nullptr,
                           1.,
                           2 * head_num * seq_len,
                           0,
                           seq_len,
                           1,
                           block_seq_len,
                           0,
                           seq_len,
                           head_num,
                           head_dim,
                           batch_size,
                           quant_policy);
    }

    hipDeviceSynchronize();

    if (0) {
        std::cout << ">>> Compare\n";
        Compare(
            kv_cache_2.data().get(), kv_cache.data().get(), head_dim, head_dim, batch_size * 2 * head_num * seq_len, 0);
        std::cout << "<<< Compare\n";
    }
}

#define KV_INT8 1

#define KV_INT4 0

#define DECODING 1

template<class T>
int test_attention()
{
    AttentionParams<T> params{};

    constexpr size_t kHeadDim = 128;

#if DECODING
    // constexpr size_t kHeadNum   = 32;
    // constexpr size_t kBatchSize = 64;
    constexpr size_t kHeadNum   = 40;
    constexpr size_t KvHeadNum  = kHeadNum / 5;
    constexpr size_t kBatchSize = 128;
    constexpr size_t kInputLen  = 1;
    // constexpr size_t kSequenceLen = 63;
    // constexpr size_t kSequenceLen = 4095;
    // constexpr size_t kSequenceLen = 511;
    // constexpr size_t kSequenceLen = 2047;
    constexpr size_t kSequenceLen = 4095;
    // constexpr size_t kSequenceLen = 8191;
    // constexpr size_t kSequenceLen = 32767;
    // constexpr size_t kSequenceLen = 65535;
    // constexpr size_t kSequenceLen = 131071;
    // constexpr size_t kSequenceLen = 262143;
    // constexpr size_t kSequenceLen = (1 << 20) - 1;  // 1M
    // constexpr size_t kSequenceLen = (1 << 22) - 1;  // 4M
    // constexpr size_t kSequenceLen = (1 << 24) - 1;  // 16M
    // constexpr int kSequenceLen = 2047;
    constexpr int kBlockSz   = 128;
    constexpr int kMaxSplitK = 1;
#else

    // append
    // constexpr size_t kHeadNum     = 32;
    // constexpr size_t KvHeadNum    = kHeadNum;
    // constexpr size_t kBatchSize   = 1;
    // constexpr size_t kInputLen    = 128;
    // constexpr size_t kSequenceLen = 65536;
    // constexpr int    kMaxSplitK   = 128;

    // constexpr size_t kHeadNum     = 1;
    // constexpr size_t KvHeadNum    = kHeadNum;
    // constexpr size_t kBatchSize   = 1;
    // constexpr size_t kInputLen    = 64;
    // constexpr size_t kSequenceLen = 65536;
    // constexpr int    kMaxSplitK   = 1;

    // prefill
    constexpr size_t kHeadNum     = 32;
    constexpr size_t KvHeadNum    = kHeadNum;
    constexpr size_t kBatchSize   = 1;
    constexpr size_t kInputLen    = 16384;
    constexpr size_t kSequenceLen = 0;
    constexpr int    kMaxSplitK   = 1;

    constexpr int kBlockSz     = 128;

#endif

#if KV_INT8
    using Tkv                  = uint8_t;
    constexpr int kQuantPolicy = QuantPolicy::kCacheKVInt8;
#elif KV_INT4
    using Tkv                  = uint4_t;
    constexpr int kQuantPolicy = QuantPolicy::kCacheKVInt4;
#else
    using Tkv                  = T;
    constexpr int kQuantPolicy = 0;
#endif

    static_assert(KvHeadNum > 0);

    constexpr size_t kContextLen = kSequenceLen + kInputLen;
    constexpr size_t kTokenNum   = kBatchSize * kInputLen;
    constexpr int    kTestIter   = 10;

    constexpr float kRoPEBase = 10000.f;
    constexpr int   kDump     = 0;

    RNG rng{};

    thrust::universal_vector<T> k_cache(kBatchSize * KvHeadNum * kContextLen * kHeadDim);
    thrust::universal_vector<T> v_cache(kBatchSize * KvHeadNum * kContextLen * kHeadDim);

    thrust::universal_vector<T> kv_cache(KvHeadNum * 2 * kBatchSize * kContextLen * kHeadDim);

    thrust::universal_vector<T> qkv(kBatchSize * kInputLen * (kHeadNum + KvHeadNum * 2) * kHeadDim);
    thrust::universal_vector<T> output(kBatchSize * kInputLen * kHeadNum * kHeadDim);

    thrust::universal_vector<bool>  finished(kBatchSize);
    thrust::universal_vector<int>   sequence_length(kBatchSize);
    thrust::universal_vector<int>   input_length(kBatchSize);
    thrust::universal_vector<int>   context_length(kBatchSize);
    thrust::universal_vector<float> rope_base(kBatchSize);
    thrust::universal_vector<int>   cu_seqlens(kBatchSize + 1);
    thrust::universal_vector<int>   cu_kv_lens(kBatchSize + 1);

    thrust::universal_vector<float> partial_M(kTokenNum * kHeadNum * kMaxSplitK);
    thrust::universal_vector<float> partial_L(kTokenNum * kHeadNum * kMaxSplitK);
    thrust::universal_vector<float> partial_O(kTokenNum * kHeadNum * kMaxSplitK * kHeadDim);
    thrust::universal_vector<int>   split_cnt(kTokenNum);
    thrust::universal_vector<int>   semaphores(kTokenNum * kHeadNum * kMaxSplitK);

    thrust::universal_vector<T> kv_cache_quant_data(kBatchSize * KvHeadNum * 2 * kContextLen * 2);
    thrust::fill(kv_cache_quant_data.begin(), kv_cache_quant_data.end(), T{0.});

    thrust::universal_vector<float> qk_buf((size_t)kDump * kBatchSize * kHeadNum * kInputLen * kContextLen);
    thrust::universal_vector<T>     pr_buf((size_t)kDump * kBatchSize * kHeadNum * kInputLen * kContextLen);

    std::fill(semaphores.begin(), semaphores.end(), 0);

    rng.GenerateNormal(qkv.data().get(), qkv.size(), 1.f, 0.f);

    rng.GenerateNormal(k_cache.data().get(), kBatchSize * KvHeadNum * kContextLen * kHeadDim);
    rng.GenerateNormal(v_cache.data().get(), kBatchSize * KvHeadNum * kContextLen * kHeadDim);

    if (0) {
        // Set input range to zero
        // (BH, SD)
        hipMemset2DAsync(k_cache.data().get() + kSequenceLen * kHeadDim,
                          sizeof(T) * kContextLen * kHeadDim,
                          0,
                          sizeof(T) * kInputLen * kHeadDim,
                          kBatchSize * KvHeadNum);
        hipMemset2DAsync(v_cache.data().get() + kSequenceLen * kHeadDim,
                          sizeof(T) * kContextLen * kHeadDim,
                          0,
                          sizeof(T) * kInputLen * kHeadDim,
                          kBatchSize * KvHeadNum);
    }

    invokeApplyRotaryEmbedding(k_cache.data().get(), kContextLen, KvHeadNum, kHeadDim, kRoPEBase, kBatchSize);

    thrust::universal_vector<T> k_cache_ref = k_cache;
    thrust::universal_vector<T> v_cache_ref = v_cache;

    thrust::universal_vector<char>  blocks;
    thrust::universal_vector<char*> k_ptrs;
    thrust::universal_vector<int>   cu_block_cnts;

    TestBlocks<Tkv>(
        k_cache, v_cache, blocks, k_ptrs, cu_block_cnts, KvHeadNum, kHeadDim, kBlockSz, kBatchSize, kQuantPolicy);

    thrust::universal_vector<T>     output_ref = output;
    thrust::universal_vector<void*> k_cache_ref_ptrs(kBatchSize);
    thrust::universal_vector<void*> v_cache_ref_ptrs(kBatchSize);

    thrust::universal_vector<T> bias_QKV(kHeadNum * kHeadDim + 2 * KvHeadNum * kHeadDim);

    rng.GenerateNormal(bias_QKV.data().get(), bias_QKV.size(), 0.1f, 0.f);

    hipDeviceSynchronize();

    for (size_t i = 0; i <= kBatchSize; ++i) {
        cu_seqlens[i] = i * kInputLen;
        cu_kv_lens[i] = i * kContextLen;
    }

    for (size_t i = 0; i < kBatchSize; ++i) {
        input_length[i]     = kInputLen;
        sequence_length[i]  = kSequenceLen;
        context_length[i]   = kContextLen;
        k_cache_ref_ptrs[i] = k_cache_ref.data().get() + i * k_cache_ref.size() / kBatchSize;
        v_cache_ref_ptrs[i] = v_cache_ref.data().get() + i * v_cache_ref.size() / kBatchSize;
        rope_base[i]        = kRoPEBase;
    }

    // getchar();

    params.out = output_ref.data().get();
    params.q   = qkv.data().get();
    params.k   = params.q + kHeadNum * kHeadDim;
    params.v   = params.k + KvHeadNum * kHeadDim;

    params.q_bias = bias_QKV.data().get();
    params.k_bias = params.q_bias + kHeadNum * kHeadDim;
    params.v_bias = params.k_bias + KvHeadNum * kHeadDim;

    params.stride = (kHeadNum + 2 * KvHeadNum) * kHeadDim;

    params.token_num  = kTokenNum;
    params.batch_size = kBatchSize;
    params.max_q_len  = kInputLen;
    params.max_k_len  = kContextLen;

    params.block_iter_params = BlockIteratorParams{k_ptrs.data().get(),  //
                                                   cu_block_cnts.data().get(),
                                                   0,
                                                   kBlockSz};

    params.linear_iter_params = LinearIteratorParams{kv_cache.data().get(),  //
                                                     int(2 * kBatchSize * kContextLen * kHeadDim),
                                                     int(kBatchSize * kContextLen * kHeadDim)};

    params.quant_policy = kQuantPolicy;

    params.finished   = finished.data().get();
    params.rope_theta = rope_base.data().get();
    params.cu_q_len   = cu_seqlens.data().get();
    params.cu_k_len   = cu_kv_lens.data().get();

    params.num_heads     = kHeadNum;
    params.num_kv_heads  = KvHeadNum;
    params.size_per_head = kHeadDim;
    params.inv_sqrt_dh   = (float)std::log2(expf(1.)) / std::sqrt((float)params.size_per_head);

    params.rotary_embedding_dim  = kHeadDim;
    params.rotary_embedding_base = kRoPEBase;
    params.rope_ti_scale         = 1.;

    params.split_cnt = split_cnt.data().get();
    params.partial_L = partial_L.data().get();
    params.partial_M = partial_M.data().get();
    params.partial_O = partial_O.data().get();
    params.locks     = semaphores.data().get();

    params.max_split_k = kMaxSplitK;
    params.arch        = getSMVersion();

    params.qk = qk_buf.data().get();
    params.pr = pr_buf.data().get();

    Reference<T> reference(kDump ? Reference<T>::kUNFUSED : Reference<T>::kFLASH_ATTENTION, {});
    // Reference<T> reference(Reference<T>::kUNFUSED, {});
    reference.Reshape(kInputLen, kContextLen, kHeadNum, kHeadDim, KvHeadNum, kBatchSize);

    for (int i = 0; i < 1; ++i) {
        reference.Execute(params.out,  //
                          k_cache_ref.data().get(),
                          v_cache_ref.data().get(),
                          qkv.data().get(),
                          bias_QKV.data().get());
    }

    hipDeviceSynchronize();

    if constexpr (kDump) {
        for (size_t b = 0; b < kBatchSize; ++b) {
            for (size_t h = 0; h < kHeadNum; ++h) {
                for (size_t q = 0; q < kInputLen; ++q) {
                    auto qk = reference.qk() + b * kHeadNum * kInputLen * kContextLen + h * kInputLen * kContextLen
                              + q * kContextLen;
                    for (size_t k = 0; k < kContextLen; ++k) {
                        std::cout << qk[k] * params.inv_sqrt_dh << " ";
                    }
                    std::cout << "\n";
                }
                std::cout << "\n";
            }
            std::cout << "\n";
        }
    }

    if (auto err = hipGetLastError(); err != hipSuccess) {
        std::cout << hipGetErrorString(err) << "\n";
        return -1;
    }
    std::cout << "---------------------------------------------------\n";

    params.out = output.data().get();

    std::vector<thrust::universal_vector<T>> outputs;

    for (int i = 0; i < std::max(kTestIter, 1); ++i) {

#if DECODING
        dispatchDecoding<T>(params);
#else
        // input -> blocked
        invokeProcessKV_v2_(params);
        // blocked -> linear
        invokeFlattenKV_v2_(params, cu_kv_lens[kBatchSize]);

        // auto tmp = std::exchange(params.linear_iter_params.kv_cache, nullptr);
        dispatchAttention(params);
        // params.linear_iter_params.kv_cache = std::exchange(tmp, nullptr);
#endif
        if (auto err = hipGetLastError(); err != hipSuccess) {
            std::cout << hipGetErrorString(err) << "\n";
            return -1;
        }
        if (1) {
            outputs.push_back(output);
        }
    }

    if (kDump) {
        hipDeviceSynchronize();
        for (size_t b = 0; b < kBatchSize; ++b) {
            for (size_t h = 0; h < kHeadNum; ++h) {
                for (size_t q = 0; q < kInputLen; ++q) {
                    auto ref = reference.qk() + b * kHeadNum * kInputLen * kContextLen + h * kInputLen * kContextLen
                               + q * kContextLen;
                    auto data = qk_buf.data().get() + b * kHeadNum * kInputLen * kContextLen
                                + h * kInputLen * kContextLen + q * kContextLen;
                    for (size_t k = 0; k < kContextLen; ++k) {
                        // std::cout << std::max(0.f, std::abs(data[k] - (float)ref[k]) - 1e-5f) << " ";
                        std::cout << data[k] * params.inv_sqrt_dh << " ";
                        // std::cout << (float)data[k] << " ";
                    }
                    std::cout << "\n";
                }
                std::cout << "\n";
            }
            std::cout << "\n";
        }
    }

    invokeFlattenKV_v2(k_cache.data().get(),  // [B, H, S, D]
                       v_cache.data().get(),
                       k_ptrs.data().get(),
                       cu_kv_lens.data().get(),
                       cu_block_cnts.data().get(),
                       nullptr,  // DECODING ? nullptr : params.rope_theta,
                       1.,
                       KvHeadNum * kContextLen,
                       0,
                       kContextLen,
                       1,
                       kBlockSz,
                       0,
                       kContextLen,
                       KvHeadNum,
                       kHeadDim,
                       kBatchSize,
                       kQuantPolicy);
    hipDeviceSynchronize();

    if (outputs.size() > 1) {
        std::cout << "Evaluating consistency..." << std::endl;
        for (size_t i = 1; i < outputs.size(); ++i) {
            Compare(outputs[i].data().get(), outputs[i - 1].data().get(), kHeadDim, kHeadDim, kHeadNum, 0, 0, 0);
        }
    }

    std::cout << "---------------------------------------------------\n";

    // [B, S, H, D]
    Compare(output.data().get(),  //
            output_ref.data().get(),
            kHeadNum * kHeadDim,
            kHeadNum * kHeadDim,
            kBatchSize * kInputLen,
            0);

    // [BH, SD]
    Compare(k_cache.data().get() + kSequenceLen * kHeadDim,
            k_cache_ref.data().get() + kSequenceLen * kHeadDim,
            kContextLen * kHeadDim,
            kInputLen * kHeadDim,
            kBatchSize * KvHeadNum,
            0);
    Compare(v_cache.data().get() + kSequenceLen * kHeadDim,
            v_cache_ref.data().get() + kSequenceLen * kHeadDim,
            kContextLen * kHeadDim,
            kInputLen * kHeadDim,
            kBatchSize * KvHeadNum);

    return 0;
}

int main(int argc, char* argv[])
{
    test_attention<half>();

    // test_attention<hip_bfloat16>();
}
