#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2022-2023, NVIDIA CORPORATION.  All rights reserved.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "src/fastertransformer/kernels/ban_bad_words.h"
#include "src/fastertransformer/utils/cuda_utils.h"

namespace fastertransformer {

template<typename T>
__global__ void ban_bad_words(T*         logits,
                              const int* output_ids_buf,
                              const int* parent_ids_buf,
                              int        batch_size,
                              int        beam_width,
                              const int* bad_words,
                              size_t     bad_words_len,
                              bool       share_words,
                              int        id_offset,
                              int        vocab_size_padded,
                              size_t     step)
{
    const int id        = blockIdx.x * blockDim.x + threadIdx.x;
    const int batch_idx = blockIdx.y / beam_width;
    const int beam_idx  = blockIdx.y % beam_width;

    const int* base_bad_words         = share_words ? bad_words : bad_words + batch_idx * 2 * bad_words_len;
    const int* base_bad_words_offsets = base_bad_words + bad_words_len;

    if (id >= bad_words_len || base_bad_words_offsets[id] < 0) {
        return;
    }

    const int item_end   = base_bad_words_offsets[id];
    const int item_start = (id > 0) ? base_bad_words_offsets[id - 1] : 0;
    const int item_size  = item_end - item_start;

    /* The single-token case unconditionally bans the token */
    bool should_ban = item_size == 1;

    /* Multi-token case and enough previously generated tokens to look for a match */
    if (item_size > 1 && step >= item_size - 1) {
        should_ban             = true;
        int        parent_id   = beam_idx;
        const bool gather_beam = beam_width > 1;

        for (int token_idx = item_size - 2; token_idx >= 0; token_idx--) {
            const int previous_token = output_ids_buf[(step - (item_size - 1) + token_idx) * batch_size * beam_width
                                                      + id_offset + batch_idx * beam_width + parent_id];

            if (previous_token != base_bad_words[item_start + token_idx]) {
                should_ban = false;
                break;
            }
            if (gather_beam) {
                parent_id = parent_ids_buf[(step - (item_size - 1) + token_idx) * beam_width * batch_size + id_offset
                                           + batch_idx * beam_width + parent_id];

                if (parent_id < 0 || parent_id >= beam_width) {
                    should_ban = false;
                    break;
                }
            }
        }
    }

    if (should_ban) {
        int banned_token = base_bad_words[item_end - 1];
        if (0 < banned_token && banned_token < vocab_size_padded) {
            logits[batch_idx * beam_width * vocab_size_padded + beam_idx * vocab_size_padded + banned_token] =
                static_cast<T>(-INFINITY);
        }
    }
}

template<typename T>
void invokeBanBadWords(T*           logits,
                       const int*   output_ids_buf,
                       const int*   parent_ids_buf,
                       int          batch_size,
                       int          local_batch_size,
                       int          beam_width,
                       const int*   bad_words,
                       bool         share_words,
                       size_t       bad_words_len,
                       int          id_offset,
                       int          vocab_size_padded,
                       size_t       step,
                       hipStream_t stream)
{
    dim3 block, grid;
    block.x = min(((bad_words_len + 32 - 1) / 32) * 32, 256UL);
    grid.x  = (bad_words_len + block.x - 1) / block.x;
    grid.y  = local_batch_size * beam_width;

    ban_bad_words<<<grid, block, 0, stream>>>(logits,
                                              output_ids_buf,
                                              parent_ids_buf,
                                              batch_size,
                                              beam_width,
                                              bad_words,
                                              bad_words_len,
                                              share_words,
                                              id_offset,
                                              vocab_size_padded,
                                              step);
    sync_check_cuda_error();
}

template void invokeBanBadWords(half*        logits,
                                const int*   output_ids_buf,
                                const int*   parent_ids_buf,
                                int          batch_size,
                                int          local_batch_size,
                                int          beam_width,
                                const int*   bad_words,
                                bool         share_words,
                                size_t       bad_words_len,
                                int          id_offset,
                                int          vocab_size_padded,
                                size_t       step,
                                hipStream_t stream);
#ifdef ENABLE_BF16
template void invokeBanBadWords(__hip_bfloat16* logits,
                                const int*     output_ids_buf,
                                const int*     parent_ids_buf,
                                int            batch_size,
                                int            local_batch_size,
                                int            beam_width,
                                const int*     bad_words,
                                bool           share_words,
                                size_t         bad_words_len,
                                int            id_offset,
                                int            vocab_size_padded,
                                size_t         step,
                                hipStream_t   stream);
#endif
template void invokeBanBadWords(float*       logits,
                                const int*   output_ids_buf,
                                const int*   parent_ids_buf,
                                int          batch_size,
                                int          local_batch_size,
                                int          beam_width,
                                const int*   bad_words,
                                bool         share_words,
                                size_t       bad_words_len,
                                int          id_offset,
                                int          vocab_size_padded,
                                size_t       step,
                                hipStream_t stream);

}  // namespace fastertransformer
