#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2020-2023, NVIDIA CORPORATION.  All rights reserved.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#ifndef CUDART_VERSION
#error CUDART_VERSION Undefined!
#elif (CUDART_VERSION >= 11050)
#include <hipcub/hipcub.hpp>
#else
#include "3rdparty/hipcub/hipcub.hpp"
#endif

#include "src/fastertransformer/kernels/beam_search_topk_kernels.h"
#include "src/fastertransformer/kernels/reduce_kernel_utils.cuh"
#include "src/fastertransformer/utils/cuda_type_utils.cuh"
#include "src/fastertransformer/utils/cuda_utils.h"
#include "src/fastertransformer/utils/logger.h"

namespace fastertransformer {

template<typename T>
__device__ __forceinline__ T apply_length_penalty(T log_prob, int length, float length_penalty)
{
    // score = log(prob) / (length)^length_penalty.
    if (length_penalty == 0.0f || length == 1) {
        return log_prob;
    }
    return log_prob / static_cast<T>(powf((float)length, length_penalty));
}

template<typename T, int MAX_K, int THREADBLOCK_SIZE>
__launch_bounds__(THREADBLOCK_SIZE) __global__ void beam_topK_kernel(const T*    log_probs,
                                                                     int*        topk_tmp_id_buf,
                                                                     T*          topk_tmp_val_buf,
                                                                     const bool* finished,
                                                                     const int*  sequence_lengths,
                                                                     const int   vocab_size,
                                                                     T           diversity_rate,
                                                                     float       length_penalty)
{
    typedef hipcub::BlockReduce<TopK<T, MAX_K>, THREADBLOCK_SIZE> BlockReduce;
    __shared__ typename BlockReduce::TempStorage               temp_storage;

    int            thread_id = threadIdx.x;
    int            block_id  = blockIdx.x;  // batch beam index.
    TopK<T, MAX_K> partial;

    const bool IS_FP16   = std::is_same<T, half>::value;
    const T    MAX_T_VAL = (IS_FP16) ? HALF_FLT_MAX : FLT_MAX;

#pragma unroll
    for (int i = 0; i < MAX_K; ++i) {
        partial.p[i] = -1;
        partial.u[i] = -MAX_T_VAL;
    }

#pragma unroll
    for (int elem_id = thread_id; elem_id < vocab_size; elem_id += THREADBLOCK_SIZE) {
        int index = elem_id + block_id * vocab_size;
        T   score = length_penalty == 0.0f ? log_probs[index] :
                                             apply_length_penalty(log_probs[index],
                                                                finished[block_id] ? sequence_lengths[block_id] :
                                                                                       sequence_lengths[block_id] + 1,
                                                                length_penalty);
        partial.insert(score, index);
    }

    TopK<T, MAX_K> total = BlockReduce(temp_storage).Reduce(partial, reduce_topk_op<T, MAX_K>);

    if (thread_id == 0) {
        int index = block_id * MAX_K;

#pragma unroll
        for (int i = 0; i < MAX_K; ++i) {
            topk_tmp_id_buf[index + i]  = total.p[i];
            topk_tmp_val_buf[index + i] = total.u[i] + diversity_rate * (T)i;
        }
    }
}

template<typename T, int MAX_K, int THREADBLOCK_SIZE>
__launch_bounds__(THREADBLOCK_SIZE) __global__
    void batch_topK_kernel(int* topk_tmp_id_buf, T* topk_tmp_val_buf, int* id_buf)
{
    int            thread_id = threadIdx.x;
    int            block_id  = blockIdx.x;
    const bool     IS_FP16   = std::is_same<T, half>::value;
    const T        MAX_T_VAL = (IS_FP16) ? HALF_FLT_MAX : FLT_MAX;
    TopK<T, MAX_K> partial;
    if (thread_id == 0) {
        for (int i = 0; i < MAX_K; ++i) {
            partial.p[i] = -1;
            partial.u[i] = -MAX_T_VAL;
        }

        int index = block_id * MAX_K * MAX_K;
        for (int i = 0; i < MAX_K * MAX_K; i++) {
            partial.insert((T)topk_tmp_val_buf[index + i], topk_tmp_id_buf[index + i]);
        }

        index = block_id * MAX_K;
        for (int i = 0; i < MAX_K; i++) {
            id_buf[index + i] = partial.p[i];
        }
    }
}

template<typename T, int MAX_K, int THREADBLOCK_SIZE>
__launch_bounds__(THREADBLOCK_SIZE) __global__
    void batch_topK_kernel_v2(int* topk_tmp_id_buf, T* topk_tmp_val_buf, int* id_buf)
{
    typedef hipcub::BlockReduce<TopK<T, MAX_K>, THREADBLOCK_SIZE> BlockReduce;
    __shared__ typename BlockReduce::TempStorage               temp_storage;

    int            tid = threadIdx.x;
    int            bid = blockIdx.x;
    TopK<T, MAX_K> partial;
    const bool     IS_FP16   = std::is_same<T, half>::value;
    const T        MAX_T_VAL = (IS_FP16) ? HALF_FLT_MAX : FLT_MAX;

#pragma unroll
    for (int i = 0; i < MAX_K; ++i) {
        partial.p[i] = -1;
        partial.u[i] = -MAX_T_VAL;
    }

    int ite = MAX_K * MAX_K / THREADBLOCK_SIZE;
#pragma unroll
    for (int i = 0; i < ite; i++) {
        int index = bid * MAX_K * MAX_K + i * THREADBLOCK_SIZE + tid;
        partial.insert((T)topk_tmp_val_buf[index], topk_tmp_id_buf[index]);
    }

    TopK<T, MAX_K> total = BlockReduce(temp_storage).Reduce(partial, reduce_topk_op<T, MAX_K>);

    if (tid == 0) {
#pragma unroll
        for (int i = 0; i < MAX_K; i++) {
            id_buf[bid * MAX_K + i] = total.p[i];
        }
    }
}

template<typename T, int BLOCK_SIZE_, int BLOCKS_PER_BEAM_>
__global__ void topk_stage_1_opt3(const T* __restrict log_probs,
                                  T*          tmp_log_probs,
                                  int*        topk_tmp_id_buf,
                                  T*          topk_tmp_val_buf,
                                  const bool* finished,
                                  const int*  sequence_lengths,
                                  const int   k,
                                  const int   vocab_size,
                                  const float length_penalty,
                                  const int*  end_ids)
{
    typedef hipcub::BlockReduce<TopK_2<T>, BLOCK_SIZE_> BlockReduce;
    __shared__ typename BlockReduce::TempStorage     temp_storage;

    const int tid = threadIdx.x;
    const int bid = blockIdx.x;

    const int  row_id             = bid / BLOCKS_PER_BEAM_;  // row id for log_probs (batchbeam index)
    const int  block_lane         = bid % BLOCKS_PER_BEAM_;  // block id for a beam
    const int  tmp_log_buf_index  = row_id * vocab_size;
    const int  tmp_topk_buf_index = row_id * BLOCKS_PER_BEAM_ * k + block_lane * k;
    TopK_2<T>  partial;
    const bool IS_FP16   = std::is_same<T, half>::value;
    const T    MAX_T_VAL = (IS_FP16) ? HALF_FLT_MAX : FLT_MAX;

    if (finished != nullptr && finished[row_id] == true) {
        if (tid < k) {
            const int index = tmp_topk_buf_index + tid;
            if (block_lane == 0 && tid == 0) {
                const int end_id        = end_ids[row_id / k];
                topk_tmp_id_buf[index]  = tmp_log_buf_index + end_id;
                topk_tmp_val_buf[index] = log_probs[tmp_log_buf_index + end_id];
            }
            else {
                topk_tmp_id_buf[index]  = -1;
                topk_tmp_val_buf[index] = -MAX_T_VAL;
            }
        }
        return;
    }

    for (int elem_id = tid + block_lane * BLOCK_SIZE_; elem_id < vocab_size;
         elem_id += BLOCK_SIZE_ * BLOCKS_PER_BEAM_) {
        int index            = elem_id + tmp_log_buf_index;
        tmp_log_probs[index] = log_probs[index];
    }

    for (int ite = 0; ite < k; ite++) {
        partial.init();
#pragma unroll
        for (int elem_id = tid + block_lane * BLOCK_SIZE_; elem_id < vocab_size;
             elem_id += BLOCK_SIZE_ * BLOCKS_PER_BEAM_) {
            int index = elem_id + tmp_log_buf_index;
            partial.insert(tmp_log_probs[index], index);
        }

        TopK_2<T> total = BlockReduce(temp_storage).Reduce(partial, reduce_topk_op_2<T>);

        if (tid == 0) {
            const int index         = tmp_topk_buf_index + ite;
            topk_tmp_id_buf[index]  = total.p;
            topk_tmp_val_buf[index] = total.u;
            tmp_log_probs[total.p]  = -MAX_T_VAL;
        }
        __syncthreads();
    }
}

template<typename T, int BLOCK_SIZE_, int BLOCKS_PER_BEAM_>
__global__ void topk_stage_2_opt3(const int* __restrict topk_tmp_id_buf,
                                  T*             topk_tmp_val_buf,
                                  int*           ids,
                                  BeamHypotheses beam_hyps,
                                  const int*     end_ids,
                                  const int      vocab_size,
                                  const int      k)
{
    const int  size      = k * k * BLOCKS_PER_BEAM_;
    const int  tid       = threadIdx.x;
    const int  batch_id  = blockIdx.x;
    const bool IS_FP16   = std::is_same<T, half>::value;
    const T    MAX_T_VAL = (IS_FP16) ? HALF_FLT_MAX : FLT_MAX;

    typedef hipcub::BlockReduce<TopK_2<T>, BLOCK_SIZE_> BlockReduce;
    __shared__ typename BlockReduce::TempStorage     temp_storage;
    extern __shared__ char                           array[];
    T*                                               s_val = topk_tmp_val_buf + batch_id * size;
    int*                                             s_id  = (int*)(array);

    __shared__ int  selected_beams;
    __shared__ bool is_stop;

    if (tid == 0) {
        selected_beams = 0;
        is_stop        = false;
    }
    __syncthreads();
    if (beam_hyps.num_beams != nullptr) {
        const int global_batch_idx = beam_hyps.ite * beam_hyps.local_batch_size + batch_id;
        if (beam_hyps.num_beams[global_batch_idx] == 0 && tid == 0) {
            // initialize the buffer
            beam_hyps.min_normed_scores[global_batch_idx] = FLT_MAX;
        }
        else if (beam_hyps.num_beams[global_batch_idx] == k) {
            return;
        }
    }

    TopK_2<T> partial;

    // In some cases, we may encounter k finished sentences, but scores are bad. So, the max iteration
    // is 2*k here
    for (int ite = 0; ite < 2 * k; ite++) {
        partial.init();
#pragma unroll
        for (int i = tid; i < size; i += BLOCK_SIZE_) {
            partial.insert(s_val[i], i);
        }

        TopK_2<T> total = BlockReduce(temp_storage).Reduce(partial, reduce_topk_op_2<T>);

        if (tid == 0) {
            if (beam_hyps.num_beams != nullptr
                && topk_tmp_id_buf[batch_id * size + total.p] % vocab_size == end_ids[batch_id]) {
                // if beam_token does not belong to top num_beams tokens, it should not be added. Refer from
                // https://github.com/huggingface/transformers/blob/v4.24.0/src/transformers/generation_beam_search.py#L257
                if (ite >= k) {
                    s_val[total.p] = -MAX_T_VAL;
                }
                else {
                    const int   global_batch_idx = beam_hyps.ite * beam_hyps.local_batch_size + batch_id;
                    const float normed_score =
                        apply_length_penalty(s_val[total.p], beam_hyps.step, beam_hyps.length_penalty);
                    const int num_beam = beam_hyps.num_beams[global_batch_idx];
                    int       beam_idx = num_beam;
                    // If there are beam_width finished sentences, check that the score of selected candidatet
                    // is higher than min_normed_score or not. If current score is better, replace worst one
                    // and update the min_normed_score.
                    if (num_beam == k) {
                        if (normed_score < beam_hyps.min_normed_scores[global_batch_idx]) {
                            // end the tracing and exist this for loop
                            selected_beams = k;
                            is_stop        = true;
                            break;
                        }
                        else {
                            // find the beam index which's score = min_normed_score, erase it.
                            for (int j = 0; j < k; j++) {
                                if (beam_hyps.normed_scores[global_batch_idx * k + j]
                                    == beam_hyps.min_normed_scores[global_batch_idx]) {
                                    beam_idx = j;
                                    beam_hyps.num_beams[global_batch_idx]--;

                                    beam_hyps.min_normed_scores[global_batch_idx]     = FLT_MAX;
                                    beam_hyps.normed_scores[global_batch_idx * k + j] = normed_score;
                                    for (int l = 0; l < k; l++) {
                                        beam_hyps.min_normed_scores[global_batch_idx] =
                                            min(beam_hyps.min_normed_scores[global_batch_idx],
                                                beam_hyps.normed_scores[global_batch_idx * k + l]);
                                    }
                                    break;
                                }
                            }
                        }
                    }
                    const int tgt_id_offset = ((batch_id + beam_hyps.ite * beam_hyps.local_batch_size) * k + beam_idx)
                                              * (beam_hyps.max_seq_len);
                    beam_hyps.output_ids_tgt[tgt_id_offset + beam_hyps.step] = end_ids[batch_id];

                    int prev_id = (topk_tmp_id_buf[batch_id * size + total.p] / vocab_size) % k;
                    for (int j = beam_hyps.step - 1; j >= 0; j--) {
                        const int src_idx = j * beam_hyps.batch_size * k
                                            + beam_hyps.ite * beam_hyps.local_batch_size * k + batch_id * k + prev_id;

                        beam_hyps.output_ids_tgt[tgt_id_offset + j] = beam_hyps.output_ids_src[src_idx];
                        prev_id                                     = beam_hyps.parent_ids_src[src_idx];
                    }
                    const int tgt_beam_idx                       = global_batch_idx * k + beam_idx;
                    beam_hyps.sequence_lengths_tgt[tgt_beam_idx] = beam_hyps.step;
                    beam_hyps.normed_scores[tgt_beam_idx]        = normed_score;
                    beam_hyps.min_normed_scores[global_batch_idx] =
                        min(beam_hyps.min_normed_scores[global_batch_idx], beam_hyps.normed_scores[tgt_beam_idx]);

                    s_val[total.p] = -MAX_T_VAL;

                    beam_hyps.num_beams[global_batch_idx]++;
                }
            }
            else {
                s_id[selected_beams] = total.p;
                s_val[total.p]       = -MAX_T_VAL;
                selected_beams++;
            }
        }
        __syncthreads();
        if (selected_beams >= k) {
            break;
        }
    }
    if (tid < k && is_stop == false) {
        ids[batch_id * k + tid] = topk_tmp_id_buf[batch_id * size + s_id[tid]];
    }
}

template<typename T, int BLOCK_SIZE, int BLOCKS_PER_BEAM>
__global__ void topk_stage_1_opt2_general(const T* __restrict log_probs,
                                          T*          tmp_log_probs,
                                          int*        topk_tmp_id_buf,
                                          T*          topk_tmp_val_buf,
                                          const bool* finished,
                                          const int*  sequence_lengths,
                                          const int   k,
                                          const int   vocab_size,
                                          const float length_penalty)
{
    const bool                                      IS_FP16   = std::is_same<T, half>::value;
    const T                                         MAX_T_VAL = (IS_FP16) ? HALF_FLT_MAX : FLT_MAX;
    typedef hipcub::BlockReduce<TopK_2<T>, BLOCK_SIZE> BlockReduce;
    __shared__ typename BlockReduce::TempStorage    temp_storage;

    const int tid                = threadIdx.x;
    const int bid                = blockIdx.x;
    const int row_id             = bid / BLOCKS_PER_BEAM;  // row id for log_probs
    const int block_lane         = bid % BLOCKS_PER_BEAM;  // block id for a beam
    const int tmp_log_buf_index  = row_id * vocab_size;
    const int tmp_topk_buf_index = row_id * BLOCKS_PER_BEAM * k + block_lane * k;
    TopK_2<T> partial;

    for (int elem_id = tid + block_lane * BLOCK_SIZE; elem_id < vocab_size; elem_id += BLOCK_SIZE * BLOCKS_PER_BEAM) {
        int index            = elem_id + tmp_log_buf_index;
        tmp_log_probs[index] = log_probs[index];
    }

    for (int ite = 0; ite < k; ite++) {
        partial.init();
#pragma unroll
        for (int elem_id = tid + block_lane * BLOCK_SIZE; elem_id < vocab_size;
             elem_id += BLOCK_SIZE * BLOCKS_PER_BEAM) {
            int index = elem_id + tmp_log_buf_index;
            partial.insert(tmp_log_probs[index], index);
        }

        TopK_2<T> total = BlockReduce(temp_storage).Reduce(partial, reduce_topk_op_2<T>);

        if (tid == 0) {
            const int index         = tmp_topk_buf_index + ite;
            topk_tmp_id_buf[index]  = total.p;
            topk_tmp_val_buf[index] = total.u;
            tmp_log_probs[total.p]  = -MAX_T_VAL;
        }
        __syncthreads();
    }
}

template<typename T, int BLOCK_SIZE, int BLOCKS_PER_BEAM>
__global__ void topk_stage_2_opt2_general(const int* __restrict topk_tmp_id_buf,
                                          T*             topk_tmp_val_buf,
                                          int*           ids,
                                          BeamHypotheses beam_hyps,
                                          const int*     end_ids,
                                          const int      k,
                                          const int      vocab_size)
{
    const int  size      = k * k * BLOCKS_PER_BEAM;
    const int  tid       = threadIdx.x;
    const int  batch_id  = blockIdx.x;
    const bool IS_FP16   = std::is_same<T, half>::value;
    const T    MAX_T_VAL = (IS_FP16) ? HALF_FLT_MAX : FLT_MAX;

    typedef hipcub::BlockReduce<TopK_2<T>, BLOCK_SIZE> BlockReduce;
    __shared__ typename BlockReduce::TempStorage    temp_storage;
    extern __shared__ char                          array[];
    T*                                              s_val = topk_tmp_val_buf + batch_id * size;
    int*                                            s_id  = (int*)(array);

    __shared__ int  selected_beams;
    __shared__ bool is_stop;

    if (tid == 0) {
        selected_beams = 0;
        is_stop        = false;
    }
    __syncthreads();
    if (beam_hyps.num_beams != nullptr) {
        const int global_batch_idx = beam_hyps.ite * beam_hyps.local_batch_size + batch_id;
        if (beam_hyps.num_beams[global_batch_idx] == 0 && tid == 0) {
            beam_hyps.min_normed_scores[global_batch_idx] = FLT_MAX;
        }
        else if (beam_hyps.num_beams[global_batch_idx] == k) {
            return;
        }
    }

    TopK_2<T> partial;

    // In some cases, we may encounter k finished sentences, but scores are bad. So, the max iteration
    // is 2*k here
    for (int ite = 0; ite < 2 * k; ite++) {
        partial.init();
#pragma unroll
        for (int i = tid; i < size; i += BLOCK_SIZE) {
            partial.insert(s_val[i], i);
        }

        TopK_2<T> total = BlockReduce(temp_storage).Reduce(partial, reduce_topk_op_2<T>);

        if (tid == 0) {
            if (beam_hyps.num_beams != nullptr
                && topk_tmp_id_buf[batch_id * size + total.p] % vocab_size == end_ids[batch_id]) {
                // if beam_token does not belong to top num_beams tokens, it should not be added. Refer from
                // https://github.com/huggingface/transformers/blob/v4.24.0/src/transformers/generation_beam_search.py#L257
                if (ite >= k) {
                    s_val[total.p] = -MAX_T_VAL;
                }
                else {
                    const int   global_batch_idx = beam_hyps.ite * beam_hyps.local_batch_size + batch_id;
                    const float normed_score =
                        apply_length_penalty(s_val[total.p], beam_hyps.step, beam_hyps.length_penalty);
                    const int num_beam = beam_hyps.num_beams[global_batch_idx];
                    int       beam_idx = num_beam;
                    // If there are beam_width finished sentences, check that the score of selected candidatet
                    // is higher than min_normed_score or not. If current score is better, replace worst one
                    // and update the min_normed_score.
                    if (num_beam == k) {
                        if (normed_score < beam_hyps.min_normed_scores[global_batch_idx]) {
                            // end the tracing and exist this for loop
                            selected_beams = k;
                            is_stop        = true;
                            break;
                        }
                        else {
                            // find the beam index which's score = min_normed_score, erase it.
                            for (int j = 0; j < k; j++) {
                                if (beam_hyps.normed_scores[global_batch_idx * k + j]
                                    == beam_hyps.min_normed_scores[global_batch_idx]) {
                                    beam_idx = j;
                                    beam_hyps.num_beams[global_batch_idx]--;

                                    beam_hyps.min_normed_scores[global_batch_idx]     = FLT_MAX;
                                    beam_hyps.normed_scores[global_batch_idx * k + j] = normed_score;
                                    for (int l = 0; l < k; l++) {
                                        beam_hyps.min_normed_scores[global_batch_idx] =
                                            min(beam_hyps.min_normed_scores[global_batch_idx],
                                                beam_hyps.normed_scores[global_batch_idx * k + l]);
                                    }
                                    break;
                                }
                            }
                        }
                    }
                    const int tgt_id_offset = ((batch_id + beam_hyps.ite * beam_hyps.local_batch_size) * k + beam_idx)
                                              * (beam_hyps.max_seq_len);
                    beam_hyps.output_ids_tgt[tgt_id_offset + beam_hyps.step] = end_ids[batch_id];

                    int prev_id = (topk_tmp_id_buf[batch_id * size + total.p] / vocab_size) % k;
                    for (int j = beam_hyps.step - 1; j >= 0; j--) {
                        const int src_idx = j * beam_hyps.batch_size * k
                                            + beam_hyps.ite * beam_hyps.local_batch_size * k + batch_id * k + prev_id;

                        beam_hyps.output_ids_tgt[tgt_id_offset + j] = beam_hyps.output_ids_src[src_idx];
                        prev_id                                     = beam_hyps.parent_ids_src[src_idx];
                    }
                    const int tgt_beam_idx                       = global_batch_idx * k + beam_idx;
                    beam_hyps.sequence_lengths_tgt[tgt_beam_idx] = beam_hyps.step;
                    beam_hyps.normed_scores[tgt_beam_idx]        = normed_score;
                    beam_hyps.min_normed_scores[global_batch_idx] =
                        min(beam_hyps.min_normed_scores[global_batch_idx], beam_hyps.normed_scores[tgt_beam_idx]);

                    s_val[total.p] = -MAX_T_VAL;

                    beam_hyps.num_beams[global_batch_idx]++;
                }
            }
            else {
                s_id[selected_beams] = total.p;
                s_val[total.p]       = -MAX_T_VAL;
                selected_beams++;
            }
        }
        __syncthreads();
        if (selected_beams >= k) {
            break;
        }
    }
    if (tid < k && is_stop == false) {
        ids[batch_id * k + tid] = topk_tmp_id_buf[batch_id * size + s_id[tid]];
    }
}

#define CASE_K_DIV(K, BLOCK_SIZE_1, BLOCK_SIZE_2)                                                                      \
    case K:                                                                                                            \
        beam_topK_kernel<T, K, BLOCK_SIZE_2><<<batch_size * beam_width, BLOCK_SIZE_2, 0, stream>>>(log_probs,          \
                                                                                                   topk_tmp_id_buf,    \
                                                                                                   topk_tmp_val_buf,   \
                                                                                                   finished,           \
                                                                                                   sequence_lengths,   \
                                                                                                   vocab_size,         \
                                                                                                   diversity_rate,     \
                                                                                                   length_penalty);    \
        if (K < 10)                                                                                                    \
            batch_topK_kernel<T, K, BLOCK_SIZE_1>                                                                      \
                <<<batch_size, BLOCK_SIZE_1, 0, stream>>>(topk_tmp_id_buf, topk_tmp_val_buf, ids);                     \
        else                                                                                                           \
            batch_topK_kernel_v2<T, K, 32><<<batch_size, 32, 0, stream>>>(topk_tmp_id_buf, topk_tmp_val_buf, ids);     \
        break;

#define CASE_K(K, BLOCK_SIZE_1_, BLOCK_SIZE_2_, BLOCKS_PER_BEAM_)                                                      \
    case K:                                                                                                            \
        topk_stage_1_opt3<float, BLOCK_SIZE_1_, BLOCKS_PER_BEAM_>                                                      \
            <<<batch_size * K * BLOCKS_PER_BEAM_, BLOCK_SIZE_1_, 0, stream>>>(log_probs,                               \
                                                                              temp_log_probs,                          \
                                                                              topk_tmp_id_buf,                         \
                                                                              topk_tmp_val_buf,                        \
                                                                              finished,                                \
                                                                              sequence_lengths,                        \
                                                                              beam_width,                              \
                                                                              vocab_size,                              \
                                                                              length_penalty,                          \
                                                                              end_ids);                                \
        topk_stage_2_opt3<float, BLOCK_SIZE_2_, BLOCKS_PER_BEAM_>                                                      \
            <<<batch_size, BLOCK_SIZE_2_, K * sizeof(int), stream>>>(                                                  \
                topk_tmp_id_buf, topk_tmp_val_buf, ids, *beam_hyps, end_ids, vocab_size, beam_width);                  \
        sync_check_cuda_error();                                                                                       \
        break;

template<typename T>
void invokeTopkBeamSearch(void*           workspace,
                          size_t&         workspace_size,
                          T*              log_probs,
                          int*            ids,
                          BeamHypotheses* beam_hyps,
                          const bool*     finished,
                          const int*      sequence_lengths,
                          const int       batch_size,
                          const int       beam_width,
                          const int       vocab_size_padded_,
                          const T         diversity_rate,
                          const float     length_penalty,
                          const int*      end_ids,
                          hipStream_t    stream)
{
    FT_LOG_DEBUG("%s", __PRETTY_FUNCTION__);
    // log_probs: (batch, beam, vocab) cumulative log_probs of beams ending with a token.
    const int vocab_size = vocab_size_padded_;
    // Beam size should be less than or equal to vocab size.
    assert(beam_width <= vocab_size);
    // Beam search needs the sequence lengths of beams to apply length penalty.
    assert(length_penalty == 0.0f || sequence_lengths != nullptr);
    const int max_block_per_beam      = 8;
    int       temp_log_probs_buf_size = batch_size * beam_width * vocab_size;                       // type float
    int       topk_tmp_ids_buf_size   = batch_size * beam_width * beam_width * max_block_per_beam;  // type int
    int       topk_tmp_val_buf_size   = batch_size * beam_width * beam_width * max_block_per_beam;  // type float

    // prevent memory misaligned address
    temp_log_probs_buf_size = (int)(ceil(temp_log_probs_buf_size / 4.)) * 4;
    topk_tmp_ids_buf_size   = (int)(ceil(topk_tmp_ids_buf_size / 4.)) * 4;
    topk_tmp_val_buf_size   = (int)(ceil(topk_tmp_val_buf_size / 4.)) * 4;

    if (workspace == nullptr) {
        workspace_size = sizeof(float) * temp_log_probs_buf_size + sizeof(int) * topk_tmp_ids_buf_size
                         + sizeof(float) * topk_tmp_val_buf_size;
        return;
    }
    else {
        T*   temp_log_probs   = (T*)workspace;
        int* topk_tmp_id_buf  = (int*)(temp_log_probs + temp_log_probs_buf_size);
        T*   topk_tmp_val_buf = (T*)(topk_tmp_id_buf + topk_tmp_ids_buf_size);
        if (diversity_rate == 0.0f) {
            switch (beam_width) {
                CASE_K(1, 128, 128, 8);
                CASE_K(4, 128, 128, 8);
                CASE_K(10, 128, 128, 8);
                CASE_K(16, 128, 128, 5);
                CASE_K(32, 256, 128, 1);
                CASE_K(64, 256, 256, 1);
                default:
                    topk_stage_1_opt2_general<T, 128, 1>
                        <<<batch_size * beam_width * 1, 128, 0, stream>>>(log_probs,
                                                                          temp_log_probs,
                                                                          topk_tmp_id_buf,
                                                                          topk_tmp_val_buf,
                                                                          finished,
                                                                          sequence_lengths,
                                                                          beam_width,
                                                                          vocab_size,
                                                                          length_penalty);
                    topk_stage_2_opt2_general<T, 128, 1>
                        <<<batch_size,
                           128,
                           beam_width * beam_width * 1 * sizeof(float) + beam_width * sizeof(int),
                           stream>>>(
                            topk_tmp_id_buf, topk_tmp_val_buf, ids, *beam_hyps, end_ids, beam_width, vocab_size);
                    break;
            }
        }
        else {
            switch (beam_width) {
                CASE_K_DIV(1, 256, 256);
                CASE_K_DIV(4, 256, 256);
                CASE_K_DIV(16, 256, 64);
                CASE_K_DIV(32, 256, 64);
                CASE_K_DIV(64, 256, 64);
                default:
                    FT_CHECK_WITH_INFO(false, fmtstr("Topk kernel does not support beamwidth = %d \n", beam_width));
                    break;
            }
        }
        return;
    }
}

#undef CASE_K
#undef CASE_K_DIV

template void invokeTopkBeamSearch(void*           workspace,
                                   size_t&         workspace_size,
                                   float*          log_probs,
                                   int*            ids,
                                   BeamHypotheses* beam_hyps,
                                   const bool*     finished,
                                   const int*      sequence_lengths,
                                   const int       batch_size,
                                   const int       beam_width,
                                   const int       vocab_size_padded_,
                                   const float     diversity_rate,
                                   const float     length_penalty,
                                   const int*      end_ids,
                                   hipStream_t    stream);

template<typename T>
__global__ void tileEncoderResults(T*         tiled_output,
                                   int*       tiled_sequence_length,
                                   const T*   output,
                                   const int* sequence_length,
                                   const uint batch_size,
                                   const uint beam_width,
                                   const uint d_model)
{
    if (blockIdx.x == 0) {
        for (uint i = threadIdx.x; i < batch_size * beam_width; i += blockDim.x) {
            tiled_sequence_length[i] = sequence_length[i / beam_width];
        }
    }

    int tgt_offset =
        blockIdx.x * gridDim.y * gridDim.z * d_model + blockIdx.y * gridDim.z * d_model + blockIdx.z * d_model;
    int src_offset = blockIdx.x * gridDim.z * d_model + blockIdx.z * d_model;
    for (uint i = threadIdx.x; i < d_model; i += blockDim.x) {
        tiled_output[i + tgt_offset] = output[i + src_offset];
    }
}

template<typename T>
void invokeTileEncoderResults(T*           tiled_output,
                              int*         tiled_sequence_length,
                              const T*     output,
                              const int*   sequence_length,
                              const size_t batch_size,
                              const size_t beam_width,
                              const size_t mem_max_seq_len,
                              const size_t d_model,
                              hipStream_t stream)
{
    // tiled_output: [batch_size, beam_width, mem_max_seq_len, d_model]
    // tiled_sequence_length: [batch_size, beam_width]

    // output: [batch_size, mem_max_seq_len, d_model]
    // sequence_length [batch_size]

    dim3 grid(batch_size, beam_width, mem_max_seq_len);
    bool is_half2 = (std::is_same<T, half>::value) && (d_model % 2 == 0);

    if (is_half2) {
        using T2 = typename TypeConverter<T>::Type;  // fp16 to half2, bf16 to bf162
        dim3 block(min(512, (int)(d_model / 2)));
        tileEncoderResults<T2><<<grid, block, 0, stream>>>((T2*)tiled_output,
                                                           tiled_sequence_length,
                                                           (const T2*)output,
                                                           sequence_length,
                                                           batch_size,
                                                           beam_width,
                                                           d_model / 2);
    }
    else {
        dim3 block(min(512, (int)d_model));
        tileEncoderResults<T><<<grid, block, 0, stream>>>(
            tiled_output, tiled_sequence_length, output, sequence_length, batch_size, beam_width, d_model);
    }
}

template void invokeTileEncoderResults(float*       tiled_output,
                                       int*         tiled_sequence_length,
                                       const float* output,
                                       const int*   sequence_length,
                                       const size_t batch_size,
                                       const size_t beam_width,
                                       const size_t mem_max_seq_len,
                                       const size_t d_model,
                                       hipStream_t stream);

template void invokeTileEncoderResults(half*        tiled_output,
                                       int*         tiled_sequence_length,
                                       const half*  output,
                                       const int*   sequence_length,
                                       const size_t batch_size,
                                       const size_t beam_width,
                                       const size_t mem_max_seq_len,
                                       const size_t d_model,
                                       hipStream_t stream);

template void invokeTileEncoderResults(half2*       tiled_output,
                                       int*         tiled_sequence_length,
                                       const half2* output,
                                       const int*   sequence_length,
                                       const size_t batch_size,
                                       const size_t beam_width,
                                       const size_t mem_max_seq_len,
                                       const size_t d_model,
                                       hipStream_t stream);
#ifdef ENABLE_BF16
template void invokeTileEncoderResults(__hip_bfloat16*       tiled_output,
                                       int*                 tiled_sequence_length,
                                       const __hip_bfloat16* output,
                                       const int*           sequence_length,
                                       const size_t         batch_size,
                                       const size_t         beam_width,
                                       const size_t         mem_max_seq_len,
                                       const size_t         d_model,
                                       hipStream_t         stream);
#endif

__global__ void insertUnfinishedPath(BeamHypotheses beam_hyps,
                                     const bool*    finished,
                                     const float*   cum_log_probs,
                                     const int      batch_size,
                                     const int      beam_width)
{
    const int bid           = blockIdx.x;
    const int tgt_start_idx = beam_hyps.num_beams[bid];
    if (beam_hyps.is_done[bid]) {
        return;
    }
    for (int i = 0; i < beam_width; i++) {
        if (threadIdx.x == 0) {
            const int src_beam_idx = bid * beam_width + i;
            const int tgt_beam_idx = bid * beam_width * 2 + i + tgt_start_idx;

            const int length = beam_hyps.sequence_lengths_src[src_beam_idx];

            beam_hyps.output_ids_tgt[(tgt_beam_idx) * (beam_hyps.max_seq_len + 1) + length] =
                beam_hyps.output_ids_src[length * batch_size * beam_width + src_beam_idx];
            if (beam_hyps.log_probs != nullptr && beam_hyps.log_probs_src != nullptr) {
                beam_hyps.log_probs[(tgt_beam_idx) * (beam_hyps.max_seq_len + 1) + length] =
                    beam_hyps.log_probs_src[length * batch_size * beam_width + src_beam_idx];
            }
            int prev_id = beam_hyps.parent_ids_src[length * batch_size * beam_width + src_beam_idx];
            for (int j = length - 1; j >= 0; j--) {
                // output_ids_tgt need to use max_seq_len + 1 because its shape is
                // [bs, beam_width, max_seq_len + 1]
                beam_hyps.output_ids_tgt[(tgt_beam_idx) * (beam_hyps.max_seq_len + 1) + j] =
                    beam_hyps.output_ids_src[j * batch_size * beam_width + bid * beam_width + prev_id];
                if (beam_hyps.log_probs != nullptr && beam_hyps.log_probs_src != nullptr) {
                    beam_hyps.log_probs[(tgt_beam_idx) * (beam_hyps.max_seq_len + 1) + j] =
                        beam_hyps.log_probs_src[j * batch_size * beam_width + bid * beam_width + prev_id];
                }
                prev_id = beam_hyps.parent_ids_src[j * batch_size * beam_width + bid * beam_width + prev_id];
            }
            beam_hyps.sequence_lengths_tgt[tgt_beam_idx] = length;

            beam_hyps.normed_scores[tgt_beam_idx] = apply_length_penalty(
                cum_log_probs[src_beam_idx], finished[src_beam_idx] ? length + 1 : length, beam_hyps.length_penalty);
            beam_hyps.cum_log_probs[tgt_beam_idx] = cum_log_probs[src_beam_idx];

            beam_hyps.num_beams[bid]++;
        }
    }
}

void invokeInsertUnfinishedPath(BeamHypotheses beam_hyps,
                                const bool*    finished,
                                const float*   cum_log_probs,
                                const int      batch_size,
                                const int      beam_width,
                                hipStream_t   stream)
{
    insertUnfinishedPath<<<batch_size, 256, 0, stream>>>(beam_hyps, finished, cum_log_probs, batch_size, beam_width);
}

}  // namespace fastertransformer
