#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2020-2023, NVIDIA CORPORATION.  All rights reserved.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "src/turbomind/kernels/decoder_masked_multihead_attention.h"
#include "src/turbomind/kernels/decoder_masked_multihead_attention/decoder_masked_multihead_attention_template.cuh"
#include "src/turbomind/kernels/decoder_masked_multihead_attention_utils.h"
#include "src/turbomind/utils/cuda_bf16_wrapper.h"
#include <assert.h>
#include <float.h>
#include <type_traits>

template<typename T, typename KERNEL_PARAMS_TYPE>
void multihead_attention_(const KERNEL_PARAMS_TYPE& params, const hipStream_t& stream)
{
    switch (params.hidden_size_per_head) {
        case 128:
            mmha_launch_kernel<T, 128, 128, KERNEL_PARAMS_TYPE>(params, stream);
            break;
        default:
            assert(false);
    }
}

////////////////////////////////////////////////////////////////////////////////////////////////////

void masked_multihead_attention(const Masked_multihead_attention_params<float>& params, const hipStream_t& stream)
{
    multihead_attention_<float, Masked_multihead_attention_params<float>>(params, stream);
}

////////////////////////////////////////////////////////////////////////////////////////////////////

void masked_multihead_attention(const Masked_multihead_attention_params<uint16_t>& params, const hipStream_t& stream)
{
    multihead_attention_<uint16_t, Masked_multihead_attention_params<uint16_t>>(params, stream);
}

////////////////////////////////////////////////////////////////////////////////////////////////////

#ifdef ENABLE_BF16
void masked_multihead_attention(const Masked_multihead_attention_params<__hip_bfloat16>& params,
                                const hipStream_t&                                     stream)
{
    multihead_attention_<__hip_bfloat16, Masked_multihead_attention_params<__hip_bfloat16>>(params, stream);
}
#endif
