#include "hip/hip_runtime.h"
// Copyright (c) OpenMMLab. All rights reserved.

#include "test_utils.h"
#include <hipblas.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <fstream>
#include <iostream>

#define _CG_ABI_EXPERIMENTAL
#include <hip/hip_cooperative_groups.h>
#include <cooperative_groups/memcpy_async.h>
#include <cooperative_groups/reduce.h>

#include "src/turbomind/kernels/decoder_masked_multihead_attention.h"

namespace turbomind {

hipblasHandle_t cublas_handle{};
hipStream_t   cublas_stream{};

template<typename T>
void Compare(const T* src, const T* ref, size_t stride, int m, int n, bool show, float rtol, float atol)
{
    float asums{};
    float rsums{};
    int   outliers{};
    for (int nn = 0; nn < n; ++nn) {
        float abs_diff_sum{};
        float rel_diff_sum{};
        for (int mm = 0; mm < m; ++mm) {
            auto x = float(src[nn * stride + mm]);
            auto y = float(ref[nn * stride + mm]);
            // if (show) {
            //     std::cout << x << "\t" << y << std::endl;
            // }
            auto abs_diff = std::abs(x - y);
            auto rel_diff = abs_diff / std::abs(y + 1e-6f);
            if (abs_diff > atol + rtol * std::abs(y)) {
                ++outliers;
                if (show) {
                    std::cout << nn << "," << mm << "\t" << x << "\t" << y << std::endl;
                }
            }
            abs_diff_sum += abs_diff;
            rel_diff_sum += rel_diff;
        }
        asums += abs_diff_sum / m;
        rsums += rel_diff_sum / m;
    }
    std::cout << "abs_diff = " << asums / n << " rel_diff = " << rsums / n << " outliers = " << outliers / (float)n
              << std::endl;
}

template void Compare(const half* src, const half* ref, size_t stride, int m, int n, bool show, float rtol, float atol);
template void
Compare(const float* src, const float* ref, size_t stride, int m, int n, bool show, float rtol, float atol);

void LoadBinary(const std::string& path, size_t size, void* dst)
{
    std::ifstream ifs(path, std::ios::binary | std::ios::in);
    if (!ifs.is_open()) {
        std::cerr << "failed to open " << path << "\n";
        std::abort();
    }
    ifs.seekg(0, ifs.end);
    auto actual_size_in_bytes = ifs.tellg();
    ifs.seekg(0, ifs.beg);
    if (size != actual_size_in_bytes) {
        std::cerr << "[warning] file " << path << " has " << actual_size_in_bytes << " bytes, while " << size
                  << " bytes is requested\n";
    }
    ifs.read((char*)dst, size);
    std::cerr << "[info] " << path << " " << size << "\n";
}

namespace cg = cooperative_groups;

__global__ void hiprand_init(hiprandState* state)
{
    auto tid = cg::this_grid().thread_rank();
    hiprand_init(0xe4c45822e90461ddULL, tid, 0, state + tid);
}

template<typename T>
__global__ void hiprand_uniform(hiprandState* state, size_t count, T* result, float scale, float shift)
{
    auto grid = cg::this_grid();
    for (auto i = grid.thread_rank(); i < count; i += grid.size()) {
        float tmp = hiprand_uniform(state + grid.thread_rank());
        result[i] = T(scale * tmp + shift);
    }
}

template<typename T>
__global__ void hiprand_normal(hiprandState* state, size_t count, T* result, float scale, float shift)
{
    auto grid = cg::this_grid();
    for (auto i = grid.thread_rank(); i < count; i += grid.size()) {
        float tmp = hiprand_normal(state + grid.thread_rank());
        result[i] = T(scale * tmp + shift);
    }
}

__global__ void curand_bytes(hiprandState* state, size_t count, uint* result)
{
    auto grid = cg::this_grid();
    for (auto i = grid.thread_rank(); i < count; i += grid.size()) {
        result[i] = hiprand(state + grid.thread_rank());
    }
}

struct RNG::Impl {

    hiprandState* states{};

    Impl()
    {
        hipMalloc(&states, sizeof(hiprandState) * 64 * 64);
        hiprand_init<<<64, 64>>>(states);
    }

    ~Impl()
    {
        hipFree(states);
    }

    void GenerateUInt(uint* out, size_t count)
    {
        curand_bytes<<<64, 64>>>(states, count, out);
    }

    template<typename T>
    void GenerateUniform(T* out, size_t count, float scale, float shift)
    {
        hiprand_uniform<<<64, 64>>>(states, count, out, scale, shift);
    }

    template<typename T>
    void GenerateNormal(T* out, size_t count, float scale, float shift)
    {
        hiprand_normal<<<64, 64>>>(states, count, out, scale, shift);
    }
};

RNG::RNG(): impl_(std::make_unique<Impl>()) {}

RNG::~RNG() = default;

void RNG::GenerateUInt(uint* out, size_t count)
{
    impl_->GenerateUInt(out, count);
}

template<typename T>
void RNG::GenerateUniform(T* out, size_t count, float scale, float shift)
{
    std::cout << count << std::endl;
    impl_->GenerateUniform(out, count, scale, shift);
}

template<typename T>
void RNG::GenerateNormal(T* out, size_t count, float scale, float shift)
{
    impl_->GenerateNormal(out, count, scale, shift);
}

template void RNG::GenerateUniform(half* out, size_t count, float scale, float shift);
template void RNG::GenerateUniform(float* out, size_t count, float scale, float shift);

template void RNG::GenerateNormal(half* out, size_t count, float scale, float shift);
template void RNG::GenerateNormal(float* out, size_t count, float scale, float shift);

template<typename T>
struct SATypeConverter {
    using Type = T;
};

template<>
struct SATypeConverter<half> {
    using Type = uint16_t;
};

template<typename T>
void mmha_ft_reference(const DecoderMultiHeadAttentionParams<T>& p, hipStream_t st)
{
    using DataType = typename SATypeConverter<T>::Type;

    // Prepare the parameters.
    Masked_multihead_attention_params<DataType> params{};
    params.q_bias = reinterpret_cast<const DataType*>(p.q_bias);
    params.k_bias = reinterpret_cast<const DataType*>(p.k_bias);
    params.v_bias = reinterpret_cast<const DataType*>(p.v_bias);

    // Set the output buffer.
    params.out = reinterpret_cast<DataType*>(p.out);

    // Set the input buffers.
    // [B, nH + kvH, D]
    params.q = reinterpret_cast<const DataType*>(p.q);
    params.k = reinterpret_cast<const DataType*>(p.k);
    params.v = reinterpret_cast<const DataType*>(p.v);

    params.stride   = p.stride;
    params.finished = (bool*)p.finished;

    params.k_cache_per_sample         = reinterpret_cast<DataType**>(p.per_sample_k_cache);
    params.v_cache_per_sample         = reinterpret_cast<DataType**>(p.per_sample_v_cache);
    params.kv_cache_per_sample_offset = p.layer_offset;
    params.batch_size                 = p.batch_size;
    params.beam_width                 = 1;
    params.memory_max_len             = p.max_seq_len;
    params.prefix_prompt_lengths      = 0;
    params.max_prefix_prompt_length   = 0;
    params.length_per_sample          = p.per_sample_length;  // max_input_length + current output length

    for (int i = 0; i < p.batch_size; ++i) {
        params.timestep = std::max(p.per_sample_length[i], params.timestep);
    }

    std::cout << "timestep = " << params.timestep << "\n";

    params.num_heads    = p.num_heads;
    params.num_kv_heads = p.num_kv_heads;

    params.hidden_size_per_head    = p.size_per_head;
    params.rotary_embedding_dim    = p.rotary_embedding_dim;
    params.max_position_embeddings = p.max_position_embeddings;
    params.use_dynamic_ntk         = false;
    params.use_logn_attn           = p.use_logn_attn;

    // Note: keep norm factor (sqrt(K_dim)) when adopting megatron T5 structure (may adjust)
    params.inv_sqrt_dh = 1.F / (sqrtf((float)params.hidden_size_per_head) * 1.f);

    params.int8_mode = 0;

    masked_multihead_attention(params, st);
}

template void mmha_ft_reference(const DecoderMultiHeadAttentionParams<half>& params, hipStream_t st);

}  // namespace turbomind
