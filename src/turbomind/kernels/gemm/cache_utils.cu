

#include "src/turbomind/kernels/gemm/cache_utils.h"
#include <iostream>

namespace turbomind::gemm {

CacheFlushing::CacheFlushing()
{
    hipDeviceProp_t props{};
    hipGetDeviceProperties(&props, 0);

    size_ = props.l2CacheSize;

    std::cout << "L2 flushing size: " << (size_ >> 20) << " MB\n";

    hipMalloc(&buffer_, size_);
}

void CacheFlushing::flush(hipStream_t stream)
{
    thread_local CacheFlushing inst{};
    inst(stream);
}

void CacheFlushing::operator()(hipStream_t stream) const
{
    hipMemsetAsync(buffer_, 0, size_, stream);
}

}  // namespace turbomind::gemm
