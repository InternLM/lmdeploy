// Copyright (c) OpenMMLab. All rights reserved.

#include "src/turbomind/kernels/attention/quantization.h"
#include "src/turbomind/kernels/core/common.h"
#include "src/turbomind/kernels/core/math.h"
#include "src/turbomind/kernels/gemm/config/sm80_hmma_16816.h"
#include "src/turbomind/kernels/gemm/convert_v2.h"
#include "src/turbomind/kernels/gemm/format.h"
#include "src/turbomind/kernels/gemm/gemm.h"
#include "src/turbomind/kernels/gemm/operand.h"
#include "src/turbomind/kernels/gemm/types.h"

namespace turbomind::gemm {

namespace {

// template<class Ti, class To>
// struct _Converter {
//     __device__ _Converter(): impl_(1, 0) {}
//     template<class T>
//     __device__ auto operator()(T&& t) const
//     {
//         return impl_((T&&)t);
//     }
//     ConvertKvCache<Ti, To> impl_;
// };

constexpr bool is_AB(Op_Tag op)
{
    if (op == OPERAND_A || op == OPERAND_B) {
        return true;
    }
    else {
        return false;
    }
}

constexpr bool is_UV(Op_Tag op)
{
    return !is_AB(op);
}

}  // namespace

// MMA     : H_16816, H_1688, H_884, H_SIMT
// Operand : A, B, U, V
// Order   : row, col
// Dtype   : u16, u8, u4 (u6, u3)
// PackNum : 1, 2, 4

template<MMA_Tag MMA, Op_Tag Op, Order Ord, class Stype_, class Dtype_, int PackNum>
struct Config {
    static constexpr int CTA_M = 32;
    static constexpr int CTA_K = 32;

    static constexpr int BLOCK_SIZE = 32;

    using Stype = Stype_;
    using Dtype = Dtype_;

    using Operand = typename GetOperand<MMA, Op, Stype, Ord, false>::Operand;

    using Kernel = ConvertOperand<CTA_M, CTA_K, PackNum, Operand, Dtype, Converter<Stype, Dtype>>;
};

template<class Config>
void Convert_v2_Impl(const void* S, const MatrixLayout& Sdesc, void* D, const MatrixLayout& Ddesc, hipStream_t stream)
{
    using Kernel = typename Config::Kernel;
    using Stype  = typename Config::Stype;
    using Dtype  = typename Config::Dtype;

    constexpr int CTA_M = 32;

    static constexpr int kSmemSize = sizeof(typename Kernel::SharedStorage);

    if (kSmemSize > (48 << 10)) {
        hipFuncSetAttribute(reinterpret_cast<const void*>(convert_kernel<Kernel>), hipFuncAttributeMaxDynamicSharedMemorySize, kSmemSize);
    }

    using PointerD = typename Kernel::PtrD;
    typename Kernel::Param param{Sdesc.rows, Sdesc.cols, (const Stype*)S, Sdesc.ld, PointerD{(Dtype*)D}, Ddesc.ld};

    constexpr int threads = Config::BLOCK_SIZE;
    const int     blocks  = ceil_div(Sdesc.rows, CTA_M);

    convert_kernel<Kernel><<<blocks, threads, kSmemSize, stream>>>(param);
}

int Convert(const void*         S,  //
            const MatrixLayout& _Sdesc,
            void*               D,
            const MatrixLayout& _Ddesc,
            hipStream_t        stream)
{
    const Op_Tag op_tag = get_operand_tag(_Ddesc.pack);
    const bool   trans  = op_tag == OPERAND_B;

    // (k, n) -> (n, k)
    MatrixLayout Sdesc = trans ? transpose(_Sdesc) : _Sdesc;
    MatrixLayout Ddesc = trans ? transpose(_Ddesc) : _Ddesc;

    auto invoke = [&](auto mma, auto operand, auto order, auto stype, auto dtype, auto pack_num) -> bool {
        using Stype = typename decltype(stype)::type;
        using Dtype = typename decltype(dtype)::type;

        if constexpr (GetOperand<mma, operand, Stype, order, false>::value) {  // is operand exist?

            // Make args constexpr explictly, some compilers failed to see const-ness of the args
            constexpr MMA_Tag mma_tag      = mma;
            constexpr Op_Tag  op_tag       = operand;
            constexpr Order   order_tag    = order;
            constexpr int     pack_num_tag = pack_num;

            using Config = Config<mma_tag, op_tag, order_tag, Stype, Dtype, pack_num_tag>;
            Convert_v2_Impl<Config>(S, Sdesc, D, Ddesc, stream);

            return true;
        }

        return false;
    };

    auto dispatch_4 = [&](auto mma, auto operand, auto order, auto stype, auto dtype) -> bool {
        switch (get_pack_num(Ddesc.pack)) {
            case 1:
                return invoke(mma, operand, order, stype, dtype, constant<1>{});
            // case 2:
            //     return invoke(mma, operand, order, dtype, constant<2>{});
            // case 4:
            //     return invoke(mma, operand, order, dtype, constant<4>{});
            default:
                return false;
        }
    };

    auto dispatch_3 = [&](auto mma, auto operand, auto order) -> bool {
        /// TODO: add U8, U4
        if constexpr (is_AB(operand)) {
            switch (Ddesc.type) {
                case DataType::F16:
                    return dispatch_4(mma, operand, order, type_c<uint16_t>, type_c<uint16_t>);
                case DataType::U8:
                    return dispatch_4(mma, operand, order, type_c<uint16_t>, type_c<uint8_t>);
                case DataType::U4:
                    return dispatch_4(mma, operand, order, type_c<uint16_t>, type_c<uint4_t>);
                default:
                    return false;
            }
        }
        else {  // UV: U16, U32
            switch (Ddesc.type) {
                case DataType::U32:
                    return dispatch_4(mma, operand, order, type_c<uint32_t>, type_c<uint32_t>);
                default:
                    return false;
            }
        }

        return false;
    };

    auto dispatch_2 = [&](auto mma, auto operand) -> bool {
        switch (Ddesc.order) {
            case Order::kRowMajor:
                return dispatch_3(mma, operand, constant<kRowMajor>{});
            case Order::kColMajor:
                return dispatch_3(mma, operand, constant<kColMajor>{});
        }
        return false;
    };

    auto dispatch_1 = [&](auto mma) -> bool {
        /// TODO: add U, V
        switch (get_operand_tag(Ddesc.pack)) {
            case OPERAND_A:
                return dispatch_2(mma, constant<OPERAND_A>{});
            case OPERAND_B:
                return dispatch_2(mma, constant<OPERAND_B>{});
            case OPERAND_U:
                return dispatch_2(mma, constant<OPERAND_U>{});
            case OPERAND_V:
                return dispatch_2(mma, constant<OPERAND_V>{});
            default:
                return false;
        }
    };

    auto dispatch = [&]() -> bool {
        /// TODO: add HMMA_1688, HMMA_884, HMMA_SIMT
        switch (get_mma_tag(Ddesc.pack)) {
            case HMMA_16816:
                return dispatch_1(constant<HMMA_16816>{});
            default:
                return false;
        }
    };

    // -1 on failure
    return dispatch() - 1;
}

}  // namespace turbomind::gemm