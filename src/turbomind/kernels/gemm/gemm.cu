#include "hip/hip_runtime.h"
// Copyright (c) OpenMMLab. All rights reserved.

#include "src/turbomind/kernels/gemm/cache_utils.h"
#include "src/turbomind/kernels/gemm/desc.h"
#include "src/turbomind/kernels/gemm/gemm.h"
#include "src/turbomind/kernels/gemm/gpu_metric.h"
#include "src/turbomind/kernels/gemm/kernel.h"
#include "src/turbomind/kernels/gemm/registry.h"
#include "src/turbomind/kernels/gemm/types.h"
#include <algorithm>
#include <limits>
#include <map>
#include <memory>
#include <numeric>
#include <vector>

namespace turbomind::gemm {

void ExportDispatchCache(std::ostream& os, const std::vector<std::pair<GemmDesc, LaunchSpec>>& entries);

void ImportDispatchCache(std::istream&                                 is,
                         std::vector<std::pair<GemmDesc, LaunchSpec>>& entries,
                         const std::vector<std::unique_ptr<Kernel>>&   kernels);

namespace {

inline decltype(auto) as_tuple(const GemmDesc& d)
{
    return std::tie(d.arch,
                    d.type_a,
                    d.type_b,
                    d.type_c,
                    d.order_a,
                    d.order_b,
                    d.order_c,
                    d.pack_a,
                    d.pack_b,
                    d.pack_u,
                    d.pack_v,
                    d.quant_a.type,
                    d.quant_a.group_size,
                    d.quant_b.type,
                    d.quant_b.group_size,
                    // d.epilogue,
                    d.n,
                    d.k,
                    d.m);
}

inline bool is_compatible(GemmDesc a, GemmDesc b)
{
    // skip batch dim & epilogue flags
    a.m = b.m  = 0;
    a.epilogue = b.epilogue = Epilogue::kNone;
    return as_tuple(a) == as_tuple(b);
}

}  // namespace

inline bool operator<(const GemmDesc& a, const GemmDesc& b)
{
    return as_tuple(a) < as_tuple(b);
}

struct Gemm::Impl {

    Impl(): props_{GetCudaDeviceProps()}, arch_{props_->major * 100 + props_->minor * 10}, registry_{props_}
    {
        l2_bytes_per_second_ = MeasureL2CacheThroughput();
        fma_per_second_      = MeasureMmaThroughput();
    }

    // find launch spec in dispatch cache, dispatch by heuristic on cache miss
    LaunchSpec Dispatch(DispatchPolicy policy, GemmDesc desc, size_t barriers_size, size_t partials_size)
    {
        if (policy & DispatchPolicy::kReuse) {
            auto it = dispatch_cache_.lower_bound(desc);
            if (it != dispatch_cache_.end() && is_compatible(it->first, desc) && it->second.kernel->is_feasible(desc)) {
                return it->second;
            }
            // if (it != dispatch_cache_.end()) {
            //     std::cout << is_compatible(it->first, desc) << " " << it->second.kernel->is_feasible(desc) << "\n";
            // }
            std::cout << "Failed to find a feasible kernel in the cache, will dispatch by heuristic.\n";
        }

        if (auto it = dispatch_cache_.find(desc); it != dispatch_cache_.end()) {
            return it->second;
        }

        auto specs = Find(desc, barriers_size, partials_size, 1);

        if (specs.empty()) {
            return {};
        }

        const auto& [spec, _] = specs.front();

        dispatch_cache_.emplace(desc, spec);

        return spec;
    }

    std::vector<std::pair<LaunchSpec, float>>
    Find(const GemmDesc& desc, size_t barrier_size, size_t partials_size, int top_k)
    {
        std::vector<Kernel*> kernels;

        for (const auto& k : registry_.kernels()) {
            if (k->is_feasible(desc)) {
                kernels.push_back(k.get());
            }
        }

        if (kernels.empty()) {
            return {};
        }

        // is a better than b
        auto compare = [&](const Kernel* a, const Kernel* b) {
            const int m_a = a->cta_tile_size().x;
            const int m_b = b->cta_tile_size().x;
            if (std::max(m_a, m_b) <= desc.m) {  // m_0 < m_1 <= M
                return m_a > m_b;
            }
            if (desc.m <= std::min(m_a, m_b)) {  // M <= m_0 < m_1
                return m_a < m_b;
            }
            // m_0 <= M <= m_1
            return m_a > m_b;
        };

        auto best_cta_m = (*std::min_element(kernels.begin(), kernels.end(), compare))->cta_tile_size().x;
        kernels.erase(
            std::remove_if(kernels.begin(), kernels.end(), [&](auto k) { return k->cta_tile_size().x != best_cta_m; }),
            kernels.end());

        //                    cost     splits
        std::vector<std::pair<float, int>> costs;

        for (const auto& k : kernels) {
            // std::cout << "\n" << k->name() << "\n";

            int max_splits = k->GetMaxSplits(desc.m, desc.n, barrier_size, partials_size);
            max_splits     = std::min(max_splits, 4);

            auto [splits, cost] = k->Estimate(desc.m,  //
                                              desc.n,
                                              desc.k,
                                              max_splits,
                                              props_->multiProcessorCount,
                                              8,
                                              1,
                                              l2_bytes_per_second_,
                                              fma_per_second_)
                                      .front();
            costs.emplace_back(cost, splits);
        }

        std::vector<int> idxs(kernels.size());
        std::iota(idxs.begin(), idxs.end(), 0);

        top_k = std::min<int>(idxs.size(), top_k);

        std::partial_sort(idxs.begin(), idxs.begin() + top_k, idxs.end(), [&](int i, int j) {
            return costs[i] < costs[j];  //
        });

        std::vector<std::pair<LaunchSpec, float>> ret;
        ret.reserve(top_k);

        for (int i = 0; i < top_k; ++i) {
            const auto& [cost, splits] = costs[idxs[i]];
            ret.emplace_back(LaunchSpec{kernels[idxs[i]], 3, splits}, static_cast<float>(cost));
        }

        return ret;
    }

    template<class LaunchFunc>
    int Measure(const GemmDesc& desc,
                size_t          barriers_size,
                size_t          partials_size,
                int             top_k,
                LaunchFunc      launch_func,
                hipStream_t    st)
    {
        if (dispatch_cache_.find(desc) != dispatch_cache_.end()) {
            return 0;
        }

        std::cout << "GEMM: " << desc.m << "x" << desc.n << "x" << desc.k << "\n";

        std::vector<Kernel*> kernels;
        for (const auto& k : registry_.kernels()) {
            if (k->is_feasible(desc)) {
                kernels.push_back(k.get());
            }
        }

        std::vector<LaunchSpec> specs;
        for (const auto& k : kernels) {
            std::cout << k->name() << "\n";
            int max_splits = k->GetMaxSplits(desc.m, desc.n, barriers_size, partials_size);
            max_splits     = std::min(max_splits, 8);
            auto splits    = k->Estimate(desc.m,  //
                                      desc.n,
                                      desc.k,
                                      max_splits,
                                      props_->multiProcessorCount,
                                      32,
                                      10,
                                      l2_bytes_per_second_,
                                      fma_per_second_);
            for (const auto& [split_k, cost] : splits) {
                for (const auto& swizzle : {0, 1, 2, 3}) {
                    if (auto s = k->GetSwizzle(desc.m, desc.n, desc.k, split_k, swizzle); s != swizzle) {
                        // Skip when swizzle is starting to get truncated
                        break;
                    }
                    specs.push_back(LaunchSpec{k, swizzle, split_k, cost});
                }
            }
        }

        /// TODO: filter kernels by heuristic
        constexpr int   kMinIteration = 5;
        constexpr int   kMaxIteration = 50;
        constexpr float kMaxDuration  = 25;  // std::milli

        constexpr float kFloatInf = std::numeric_limits<float>::infinity();

        hipEvent_t ev_beg;
        hipEvent_t ev_end;

        hipEventCreate(&ev_beg);
        hipEventCreate(&ev_end);

        // std::vector<float> measurements;

        for (auto& spec : specs) {
            int                iter = 0;
            float              accum{};
            std::vector<float> duration;
            // std::cout << "measuring " << spec.kernel->name() << " with swizzle=" << spec.swizzle
            //           << ", splits=" << spec.splits << "\n";
            while (true) {
                CacheFlushing::flush(st);
                // hipStreamSynchronize(st);

                hipEventRecord(ev_beg, st);

                launch_func(spec, st);

                hipEventRecord(ev_end, st);
                hipEventSynchronize(ev_end);

                auto err = hipGetLastError();

                float delta{};
                hipEventElapsedTime(&delta, ev_beg, ev_end);
                duration.push_back(delta);

                ++iter;
                accum += delta;

                if (err != hipSuccess) {
                    std::cout << hipGetErrorString(err) << "\n";
                    // std::abort();
                    duration.back() = kFloatInf;
                    break;
                }

                if (iter >= kMaxIteration) {
                    break;
                }
                if (iter >= kMinIteration && accum >= kMaxDuration) {
                    break;
                }
            }

            spec.measured = accum / static_cast<float>(iter);
            // measurements.push_back(accum / static_cast<float>(iter));
        }

        hipEventDestroy(ev_beg);
        hipEventDestroy(ev_end);

        std::vector<int> idxs(specs.size());
        std::iota(idxs.begin(), idxs.end(), 0);
        std::sort(idxs.begin(), idxs.end(), [&](int i, int j) {  //
            return specs[i].measured < specs[j].measured;
        });

        for (const auto& i : idxs) {
            std::cout << specs[i].kernel->name()                              //
                      << " swizzle=" << specs[i].swizzle                      //
                      << ", splits=" << specs[i].splits                       //
                      << ", estimated=" << specs[i].estimated * 1e3f << "ms"  //
                      << ", measured=" << specs[i].measured << "ms\n";
        }

        LaunchSpec spec{};
        if (!idxs.empty() && specs[idxs[0]].measured != kFloatInf) {
            spec                  = specs[idxs[0]];
            dispatch_cache_[desc] = spec;
        }
        else {
            std::cout << "No valid kernel found for problem.\n";
            return -1;
        }

        return 0;
    }

    int Export(std::ostream& os)
    {
        std::vector<std::pair<GemmDesc, LaunchSpec>> entries;
        for (const auto& entry : dispatch_cache_) {
            entries.push_back(entry);
        }
        ExportDispatchCache(os, entries);
        Summary(entries);
        return dispatch_cache_.size();
    }

    int Import(std::istream& is)
    {
        std::vector<std::pair<GemmDesc, LaunchSpec>> entries;
        ImportDispatchCache(is, entries, registry_.kernels());
        for (const auto& entry : entries) {
            dispatch_cache_.insert(entry);
        }
        return dispatch_cache_.size();
    }

    void Summary(const std::vector<std::pair<GemmDesc, LaunchSpec>>& entries)
    {
        std::vector<Kernel*> uses{nullptr};
        for (const auto& k : registry_.kernels()) {
            uses.push_back(k.get());
        }
        for (const auto& [_, s] : entries) {
            uses.push_back(s.kernel);
        }
        std::sort(uses.begin(), uses.end());
        assert(uses[0] == nullptr);
        std::vector<std::pair<int, Kernel*>> count;
        for (size_t i = 1; i < uses.size(); ++i) {
            if (uses[i] != uses[i - 1]) {
                count.emplace_back(-1, uses[i]);
            }
            ++count.back().first;
        }
        std::sort(count.begin(), count.end(), std::greater<>{});
        for (const auto& [n, k] : count) {
            std::cout << k->name() << ": " << n << "\n";
        }
    }

    /// TODO: move to cuda utils
    static std::unique_ptr<hipDeviceProp_t> GetCudaDeviceProps()
    {
        auto props     = std::make_unique<hipDeviceProp_t>();
        int  device_id = -1;
        hipGetDevice(&device_id);
        hipGetDeviceProperties(props.get(), device_id);
        return props;
    }

    std::shared_ptr<hipDeviceProp_t> props_;
    int                             arch_;
    Registry                        registry_;

    float l2_bytes_per_second_;
    float fma_per_second_;

    std::map<GemmDesc, LaunchSpec> dispatch_cache_;
};

// implementation of GEMM interfaces

Gemm::Gemm(): impl_{new Impl{}} {}

Gemm::~Gemm() = default;

int Gemm::Run(const Operation&    operation,
              float               alpha,
              const void*         A,
              const MatrixLayout& Adesc,
              const void*         U,
              const MatrixLayout& Udesc,
              const void*         B,
              const MatrixLayout& Bdesc,
              const void*         V,
              const MatrixLayout& Vdesc,
              float               beta,
              const void*         C,
              const MatrixLayout& Cdesc,
              void*               D,
              const MatrixLayout& Ddesc,
              const Workspace&    workspace,
              hipStream_t        stream)
{

    if (Adesc.rows != Ddesc.rows || Bdesc.cols != Ddesc.cols || Adesc.cols != Bdesc.rows) {
        return -1;
    }

    const int m = Ddesc.rows;
    const int n = Ddesc.cols;
    const int k = Adesc.cols;

    const GemmDesc desc{
        impl_->arch_,
        Adesc.type,
        Bdesc.type,
        Cdesc.type,
        Adesc.order,
        Bdesc.order,
        Cdesc.order,
        Adesc.pack,
        Bdesc.pack,
        Udesc.pack,
        Vdesc.pack,
        operation.quant_a,
        operation.quant_b,
        operation.epilogue,
        m,
        n,
        k,
    };

    const auto launch = [&](LaunchSpec spec, hipStream_t st) {
        auto _workspace = workspace;
        return spec.kernel->Launch(operation,
                                   alpha,
                                   A,
                                   Adesc,
                                   U,
                                   Udesc,
                                   B,
                                   Bdesc,
                                   V,
                                   Vdesc,
                                   beta,
                                   C,
                                   Cdesc,
                                   D,
                                   Ddesc,
                                   spec.swizzle,
                                   spec.splits,
                                   _workspace,
                                   st);
    };

    LaunchSpec spec{};

    if (operation.dispatch & DispatchPolicy::kMeasure) {
        impl_->Measure(desc, workspace.barriers_size, workspace.partials_size, 1, launch, stream);
    }

    spec = impl_->Dispatch(operation.dispatch, desc, workspace.barriers_size, workspace.partials_size);

    if (spec.kernel) {
        // std::cout << "[Gemm] dispatch: " << spec.kernel->name()  //
        //           << " split_k=" << spec.splits                  //
        //           << " swizzle=" << spec.swizzle << std::endl;
        return launch(spec, stream);
    }

    printf("No feasible kernel found for the problem.\n");

    return -1;
}

int Gemm::Export(std::ostream& os)
{
    return impl_->Export(os);
}

int Gemm::Import(std::istream& is)
{
    return impl_->Import(is);
}

}  // namespace turbomind::gemm
