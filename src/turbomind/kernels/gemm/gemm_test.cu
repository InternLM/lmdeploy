
#include "src/turbomind/kernels/attention/quantization.h"

#include "src/turbomind/kernels/gemm/cache_utils.h"
#include "src/turbomind/kernels/gemm/convert_v2.h"
#include "src/turbomind/kernels/gemm/gemm.h"
#include "src/turbomind/kernels/gemm/gpu_metric.h"
#include "src/turbomind/kernels/gemm/kernel.h"
#include "src/turbomind/kernels/gemm/quantization.h"
#include "src/turbomind/kernels/gemm/test_utils.h"
#include "src/turbomind/kernels/gemm/testbed.h"
#include "src/turbomind/kernels/gemm/types.h"
#include <fstream>
#include <limits>
#include <thrust/universal_vector.h>

#include <type_traits>

using namespace turbomind;
using namespace gemm;
using thrust::universal_vector;

hipblasHandle_t cublas_handle{};

void ComputeRefCpu(half* C, const half* A, const half* B, int m, int n, int k)
{
    for (int mm = 0; mm < m; ++mm) {
        for (int nn = 0; nn < n; ++nn) {
            float c = 0;
            for (int kk = 0; kk < k; ++kk) {
                c += (float)A[mm * k + kk] * (float)B[nn * k + kk];
            }
            C[mm * n + nn] = c;
        }
    }
}

template<class T>
T& gTestbed()
{
    static T inst{turbomind::gemm::DispatchPolicy::kDefault, "tmp"};
    return inst;
}

template<class T, class Tb>
void Run(int m, int n, int k, int g = 128)
{
    constexpr Pack kPackA = 0;  // HMMA_16816 | OPERAND_A | 1;
    constexpr Pack kPackU = 0;  // HMMA_16816 | OPERAND_U | 1;
    constexpr Pack kPackB = HMMA_16816 | OPERAND_B | 2;
    constexpr Pack kPackV = HMMA_16816 | OPERAND_V | 1;
    auto&          test =
        gTestbed<gemm::Testbed<half, uint4_t, half, kRowMajor, kColMajor, kRowMajor, kPackA, kPackB, kPackU, kPackV>>();

    // constexpr Pack kPackA = 0;
    // constexpr Pack kPackU = 0;
    // constexpr Pack kPackB = HMMA_SIMT | OPERAND_B | 2;
    // constexpr Pack kPackV = HMMA_SIMT | OPERAND_V | 2;
    // auto& test = gTestbed<gemm::Testbed<half, uint4_t, half, kRowMajor, kColMajor, kPackA, kPackB, kPackU,
    // kPackV>>();

    // constexpr Pack kPackA = 0;
    // constexpr Pack kPackU = 0;
    // constexpr Pack kPackB = HMMA_884 | OPERAND_B | 2;
    // constexpr Pack kPackV = HMMA_884 | OPERAND_V | 2;
    // auto& test = gTestbed<gemm::Testbed<half, uint4_t, half, kRowMajor, kColMajor, kPackA, kPackB, kPackU,
    // kPackV>>();

    test.Initialize(m, n, k, g, 0);
    for (int i = 0; i < 10; ++i) {
        test.Run();
    }

    // test.CompareB();
    test.CompareC();

    return;
}

template<class T, class Tb>
void Test(int bsz, int tp)
{
    // Run<T, Tb>(8192 - 64, 8192 , 8192);
    // Run<T, Tb>(bsz, 8192, 8192);
    // Run<T, Tb>(bsz, 4096, 4096);
    // Run<half, uint4_t>(64, 11008, 4096);
    // Run<half, uint4_t>(128, 128, 32);
    // Run<half, uint4_t>(128, 128, 128);

    // llama2-7b
    // Run<T, Tb>(bsz, 2 * 11008 / tp, 4096);  // mlp.up/gate

    // Run<T, Tb>(bsz, 4096, 11008 / tp);  // mlp.down
    // Run<T, Tb>(bsz, 12288 / tp, 4096);  // w_qkv
    // Run<T, Tb>(bsz, 4096, 4096);        // w_o

    // llama2-70b
    // Run<T, Tb>(bsz, 10240 / tp, 8192);  // attn.qkv

    // Run<T, Tb>(8, 128, 512);

    // Run<T, Tb>(16, 16, 64);

    Run<T, Tb>(16384, 16384, 16384);

    // Run<T, Tb>(8192, 8192, 8192);

    // Run<T, Tb>(4096, 4096, 4096);

    // Run<T, Tb>(1024, 1024, 16384);

    // Run<T, Tb>(128, 128 * (2 + 8) * 2, 8192);

    // Run<T, Tb>(16, 4096, 4096);

    // Run<T, Tb>(1, 22016, 4096);

    // Run<T, Tb>(256, 8192, 8192 * 3);

    // Run<T, Tb>(128, 256, 8192);

    // Run<T, Tb>(16, 32, 16384);

    // Run<T, Tb>(16, 16, 16);

    // Run<T, Tb>(16, 32, 16);
}

namespace turbomind::gemm {

Kernel& gKernel();

}

int main(int argc, char* argv[])
{
    // gemm::MeasureL2CacheThroughput();
    // gemm::MeasureMmaThroughput();
    // Test<half, uint4_t>(1, 1);
    // Test<half, uint4_t>(8, 1);
    Test<half, half>(16, 1);
    return 0;
    // Test<half, uint4_t>(32, 1);
    // Test<half, uint4_t>(64, 1);
    // Test<half, uint4_t>(128, 1);
    // Test<half, uint4_t>(256, 1);
    // Test<half, uint4_t>(512, 1);
    // Test<half, uint4_t>(1024, 1);
    // Test<half, uint4_t>(2048, 1);
    // Test<half, uint4_t>(4096, 1);
    // Test<half, uint4_t>(8192, 1);

    const int M = 16;
    const int N = 16;
    const int K = 16;

    universal_vector<half> a(M * K);
    universal_vector<half> p(M * K);

    for (int i = 0; i < M; ++i) {
        for (int j = 0; j < K; ++j) {
            a[i + j * M] = i + j * M;
        }
    }

    MatrixLayout a_desc{DataType::F16, Order::kColMajor, M, K, M};
    MatrixLayout p_desc{DataType::F16, Order::kColMajor, M, K, 0, HMMA_16816 | OPERAND_A | 1};

    Convert(a.data().get(), a_desc, p.data().get(), p_desc, 0);

    hipDeviceSynchronize();

    for (int i = 0; i < M; ++i) {
        for (int j = 0; j < K; ++j) {
            int index = (int)p[i + j * M];
            int row   = index % M;
            int col   = index / M;
            printf("(%2d,%2d) ", row, col);
        }
        printf("\n");
    }

    // universal_vector<half> b(N * K);
    // thrust::fill_n(b.begin(), b.size(), 1);

    // universal_vector<half> c(M * N);

    // Workspace workspace{};

    // const float alpha = 1.f;
    // const float beta  = 0.f;

    // const MatrixLayout c_desc{DataType::F16, Order::kRowMajor, M, N, N};

    // (void)Gemm{}.Run({},
    //                  &alpha,
    //                  p.data().get(),
    //                  p_desc,
    //                  nullptr,
    //                  MatrixLayout{},
    //                  b.data().get(),
    //                  MatrixLayout{DataType::F16, Order::kColMajor, K, N, K},
    //                  nullptr,
    //                  MatrixLayout{},
    //                  &beta,
    //                  c.data().get(),
    //                  c_desc,
    //                  c.data().get(),
    //                  c_desc,
    //                  workspace,
    //                  0);

    // hipDeviceSynchronize();

    // for (int i = 0; i < M; ++i) {
    //     for (int j = 0; j < N; ++j) {
    //         printf("%2.0f ", (float)c[i * N + j]);
    //     }
    //     printf("\n");
    // }

    return 0;
}