#include "hip/hip_runtime.h"
// Copyright (c) OpenMMLab. All rights reserved.

#include "src/turbomind/kernels/core/math.h"
#include "src/turbomind/kernels/gemm/arch.h"
#include "src/turbomind/kernels/gemm/desc.h"
#include "src/turbomind/kernels/gemm/kernel.h"
#include "src/turbomind/kernels/gemm/types.h"
#include <algorithm>
#include <iostream>
#include <numeric>
#include <sstream>

namespace turbomind::gemm {

bool Kernel::is_feasible(const GemmDesc& desc) const noexcept
{
    constexpr bool debug = 0;

    if constexpr (debug)
        printf("S\n");

    if (!is_arch_compatible(desc_.arch, desc.arch)) {
        return false;
    }

    if constexpr (debug)
        printf("S0\n");

    if (std::tie(desc.order_a, desc.order_b, desc.order_c) != std::tie(desc_.order_a, desc_.order_b, desc_.order_c)) {
        return false;
    }

    if constexpr (debug)
        printf("A\n");

    if (std::tie(desc.type_a, desc.type_b, desc.type_c) != std::tie(desc_.type_a, desc_.type_b, desc_.type_c)) {
        return false;
    }

    if constexpr (debug) {
        printf("B\n");
        printf("%X %X %X %X\n", desc.pack_a, desc_.pack_a, desc.pack_u, desc_.pack_u);
    }

    if (std::tie(desc.pack_a, desc.pack_u) != std::tie(desc_.pack_a, desc_.pack_u)) {
        return false;
    }

    if constexpr (debug) {
        printf("C\n");
        printf("%X %X %X %X\n", desc.pack_b, desc_.pack_b, desc.pack_v, desc_.pack_v);
    }

    if (std::tie(desc.pack_b, desc.pack_v) != std::tie(desc_.pack_b, desc_.pack_v)) {
        return false;
    }

    if constexpr (debug)
        printf("D\n");

    if (desc.quant_a.type != desc_.quant_a.type || desc.quant_a.group_size != desc_.quant_a.group_size) {
        return false;
    }

    if constexpr (debug)
        printf("E\n");

    if (desc.quant_b.type != desc_.quant_b.type || desc.quant_b.group_size != desc_.quant_b.group_size) {
        return false;
    }

    if constexpr (debug)
        printf("F\n");

    if (desc.m % desc_.align.x || desc.n % desc_.align.y || desc.k % desc_.align.z) {
        return false;
    }

    if constexpr (debug)
        printf("success\n");

    return true;
}

std::vector<std::pair<int, KernelMetric>>
Kernel::Estimate_v2(std::array<int, 3> size, int max_splits, int max_waves, int sm_count) const
{
    const auto [m, n, k]        = size;
    const int64_t tiled_shape_m = ceil_div(m, desc_.cta_tile.x);
    const int64_t tiled_shape_n = ceil_div(n, desc_.cta_tile.y);
    const int     chunk_cnt_k   = ceil_div(k, chunk_size_k_);

    // Despite we only have sm_count * constant tensor cores, this is the granularity for scheduling
    const int   concurrency     = sm_count * desc_.max_active_ctas;
    const float waves_per_split = float(tiled_shape_m * tiled_shape_n) / concurrency;
    const float splits_per_wave = 1.f / waves_per_split;

    // Tile quantization
    const int64_t ceil_m = tiled_shape_m * desc_.cta_tile.x;
    const int64_t ceil_n = tiled_shape_n * desc_.cta_tile.y;

    std::vector<std::pair<int, KernelMetric>> metrics;

    for (int splits = 1; splits <= max_splits; ++splits) {
        // Split quantization, penalize uneven splits
        const int64_t split_ceil_k = ceil_div(chunk_cnt_k, splits) * chunk_size_k_;
        // Footprint for single split
        const int64_t split_mma_cost = ceil_m * ceil_n * split_ceil_k;
        // Footprint for single wave
        const int64_t wave_mma_cost = split_mma_cost * splits_per_wave;

        // Wave quantization
        // const int waves = (int)std::ceil(wave_per_split * splits);

        // Bold simulation of thread block scheduling
        const int   grid_size    = tiled_shape_m * tiled_shape_n * splits;
        const int   full_waves   = grid_size / concurrency;
        const int   residue      = grid_size % concurrency;
        const float partial_wave = (float)ceil_div(residue, sm_count) / desc_.max_active_ctas;
        const float waves        = full_waves + partial_wave;

        if (splits > 1 && waves > max_waves) {
            break;
        }
        // ceil(tiled_mn / C * splits) * C / tiled_mn * ceil_m * ceil_n * split_ceil_k
        const int64_t mma_cost = wave_mma_cost * waves;

        // IO has less severe quantization effect
        const int64_t mio_cost_a = get_size(desc_.type_a, tiled_shape_n * m * split_ceil_k) * splits;
        const int64_t mio_cost_b = get_size(desc_.type_b, tiled_shape_m * n * split_ceil_k) * splits;
        /// TODO: read type from `desc_.accum` when added
        const int64_t mio_cost_c = get_size(DataType::F32, (int64_t)m * n) * (splits - 1) * 2;
        const int64_t mio_cost   = mio_cost_a + mio_cost_b + mio_cost_c;

        // std::cout << name() << " " << splits << " " << waves << " " << (float)mio_cost << " " << (float)mma_cost
        //           << "\n";

        metrics.emplace_back(splits, KernelMetric{mio_cost, mma_cost});
    }

    return metrics;
}

std::string Kernel::GetName() const
{
    std::stringstream ss;

    ss << "sm" << desc_.arch / 10;
    ss << "_" << to_string(desc_.type_a);  //
    if ((int)desc_.quant_a.type) {
        ss << "g" << desc_.quant_a.group_size;
    }
    ss << "_" << to_string(desc_.type_b);  //
    if ((int)desc_.quant_b.type) {
        ss << "g" << desc_.quant_b.group_size;
    }
    ss << "_" << to_string(desc_.type_c);
    ss << "_"                                                                            //
       << (desc_.order_a == kColMajor ? 'n' : 't')                                       //
       << (desc_.order_b == kColMajor ? 'n' : 't')                                       //
       << (desc_.order_c == kColMajor ? 'n' : 't');                                      //
    ss << "_" << desc_.cta_tile.x << "x" << desc_.cta_tile.y << "x" << desc_.cta_tile.z  //
       << "_" << desc_.stages                                                            //
       << "_" << to_string(desc_.op_class)                                               //
       << "_" << desc_.mma_tile.x << "x" << desc_.mma_tile.y << "x" << desc_.mma_tile.z  //
       << "_c" << desc_.c_tile.x << "x" << desc_.c_tile.y                                //
       << "_a" << desc_.align.x << "x" << desc_.align.y << "x" << desc_.align.z          //
       << "_" << desc_.policy_a << desc_.policy_b;

    return ss.str();
}

template<class Op>
inline static bool cmp(const int3& a, const int3& b, Op op)
{
    return op(std::tie(a.x, a.y, a.z), std::tie(b.x, b.y, b.z));
}

std::vector<std::vector<LaunchSpec>> Cluster(const std::vector<LaunchSpec>& specs, const ClusteringParam& param)
{
    std::vector<const LaunchSpec*> ptrs;  // pointer into `specs`
    for (auto& s : specs) {
        ptrs.push_back(&s);
    }

    auto less = [&](const LaunchSpec* u, const LaunchSpec* v) {
        const auto& a = u->kernel->desc();
        const auto& b = v->kernel->desc();
        if (!cmp(a.cta_tile, b.cta_tile, std::equal_to<>{})) {
            return cmp(a.cta_tile, b.cta_tile, std::less<>{});
        }
        if (!cmp(a.mma_tile, b.mma_tile, std::equal_to<>{})) {
            return cmp(a.mma_tile, b.mma_tile, std::less<>{});
        }
        if (param.cache_policy) {
            const auto pa = std::tie(a.policy_a, a.policy_b);
            const auto pb = std::tie(b.policy_a, b.policy_b);
            if (pa != pb) {
                return pa < pb;
            }
        }
        if (param.max_active_ctas) {
            if (a.max_active_ctas != b.max_active_ctas) {
                return a.max_active_ctas < b.max_active_ctas;
            }
        }
        return u->splits < v->splits;
    };

    std::stable_sort(ptrs.begin(), ptrs.end(), less);

    if (ptrs.empty()) {
        return {};
    }
    std::vector<std::vector<LaunchSpec>> clusters{{*ptrs[0]}};

    auto equal = [&](const LaunchSpec* u, const LaunchSpec* v) {  //
        return !less(u, v) && !less(v, u);
    };
    int p = 0;
    for (size_t i = 1; i < ptrs.size(); ++i) {
        if (equal(ptrs[p], ptrs[i])) {
            clusters.back().push_back(*ptrs[i]);
        }
        else {
            clusters.push_back({*ptrs[i]});
            p = i;
        }
    }

    return clusters;
}

}  // namespace turbomind::gemm
