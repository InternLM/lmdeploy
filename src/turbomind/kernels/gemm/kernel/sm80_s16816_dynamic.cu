#include "hip/hip_runtime.h"
// Copyright (c) OpenMMLab. All rights reserved.

#include "src/turbomind/core/data_type.h"
#include "src/turbomind/kernels/gemm/arch/config_sm80_s16816.h"
#include "src/turbomind/kernels/gemm/cta_map.h"
#include "src/turbomind/kernels/gemm/registry.h"
#include "src/turbomind/kernels/gemm/transform.h"
#include "src/turbomind/kernels/gemm/types.h"

namespace turbomind::gemm {

using namespace sm80_s16816;
using namespace cache_policy;
using S = cache_policy::Stream;
using D = cache_policy::Default;

namespace {

template<class T, int N>
using Config_e4m3 = Sm80_s16816<Sm80,
                                T,
                                Operand_A_Pack<fp8_e4m3_t, kColMajor, 1>,  // A
                                Transform_HMMA_16816<0, 1>,                // tarnsform A
                                Operand_UV_Pack<uint16_t, false>,          // U
                                Operand_B<T, kRowMajor, N>,                // B
                                Transform_Default,                         // transform B
                                VoidOperand,                               // V
                                kColMajor,                                 // order_C
                                T,                                         // Tc
                                Striding::kBlocked,
                                Striding::kIndexed,  // indexed input
                                Striding::kBlocked,
                                kRowMajor,
                                1>;

}  // namespace

template<class T>
void Registry::sm80_s16816_dynamic()
{
#if 0
    if constexpr (std::is_same_v<T, half>) {
        using C = Sm80_s16816<Sm80,
                              half,
                              Operand_A<half, kRowMajor>,          // A
                              Transform_Default,                   // tarnsform A
                              VoidOperand,                         // U
                              Operand_B_Pack<half, kRowMajor, 1>,  // B
                              Transform_Default,                   // transform B
                              VoidOperand,                         // V
                              kRowMajor,                           // order_C
                              half,                                // Tc
                              Striding::kIndexed,                  // indexed input
                              Striding::kBlocked,
                              Striding::kBlocked,
                              kColMajor,
                              0>;

        // clang-format off
        Add<C::Type<256, 128,  64, 4, 2, 1, D, D, 3,   0 , 1, 1>>();
        Add<C::Type<128, 256,  64, 2, 4, 1, D, D, 3,   0 , 1, 1>>(); // 10
        Add<C::Type<128, 256,  32, 2, 4, 1, D, D, 3,   0 , 1, 1>>();
        Add<C::Type<128, 128,  32, 2, 2, 1, D, D, 3, true, 1, 1>>(); // 6
        Add<C::Type<128, 128,  64, 2, 2, 1, D, D, 3, true, 1, 1>>();
        Add<C::Type<128, 128,  32, 2, 2, 1, D, D, 5, true, 1, 1>>();
        Add<C::Type< 96,  64,  64, 2, 2, 1, D, D, 3, true, 1, 1>>(); // 2
        Add<C::Type< 64, 128,  64, 1, 4, 1, D, S, 3, true, 1, 1>>();
        Add<C::Type< 64,  64,  64, 2, 2, 1, D, S, 3, true, 1, 1>>(); // *
        Add<C::Type< 64,  64,  64, 2, 2, 1, D, S, 5, true, 1, 1>>();
        Add<C::Type< 64,  64, 128, 1, 2, 2, D, S, 3, true, 1, 1>>(); // 4
        Add<C::Type< 32,  64, 128, 1, 2, 2, D, S, 3, true, 1, 1>>();
        Add<C::Type< 32, 128,  64, 1, 4, 1, D, S, 3, true, 1, 1>>();
        Add<C::Type< 16,  64, 128, 1, 2, 2, D, S, 3, true, 1, 1>>(); // 10
        Add<C::Type< 16, 128,  64, 1, 4, 1, D, S, 3, true, 1, 1>>();
        // clang-format on
    }
    else if constexpr (std::is_same_v<T, hip_bfloat16>) {
        using C = Sm80_s16816<Sm80,
                              hip_bfloat16,
                              Operand_A<hip_bfloat16, kRowMajor>,          // A
                              Transform_Default,                          // tarnsform A
                              VoidOperand,                                // U
                              Operand_B_Pack<hip_bfloat16, kRowMajor, 1>,  // B
                              Transform_Default,                          // transform B
                              VoidOperand,                                // V
                              kRowMajor,                                  // order_C
                              hip_bfloat16,                                // Tc
                              Striding::kIndexed,                         // indexed input
                              Striding::kBlocked,
                              Striding::kBlocked,
                              kColMajor,
                              0>;

        // clang-format off
        Add<C::Type<256, 128,  64, 4, 2, 1, D, D, 3,   0 , 1, 1>>();
        Add<C::Type<128, 256,  64, 2, 4, 1, D, D, 3,   0 , 1, 1>>(); // 10
        Add<C::Type<128, 256,  32, 2, 4, 1, D, D, 3,   0 , 1, 1>>();
        Add<C::Type<128, 128,  32, 2, 2, 1, D, D, 3, true, 1, 1>>(); // 6
        Add<C::Type<128, 128,  64, 2, 2, 1, D, D, 3, true, 1, 1>>();
        Add<C::Type<128, 128,  32, 2, 2, 1, D, D, 5, true, 1, 1>>();
        Add<C::Type< 96,  64,  64, 2, 2, 1, D, D, 3, true, 1, 1>>(); // 2
        Add<C::Type< 64, 128,  64, 1, 4, 1, D, S, 3, true, 1, 1>>();
        Add<C::Type< 64,  64,  64, 2, 2, 1, D, S, 3, true, 1, 1>>(); // *
        Add<C::Type< 64,  64,  64, 2, 2, 1, D, S, 5, true, 1, 1>>();
        Add<C::Type< 64,  64, 128, 1, 2, 2, D, S, 3, true, 1, 1>>(); // 4
        Add<C::Type< 32,  64, 128, 1, 2, 2, D, S, 3, true, 1, 1>>();
        Add<C::Type< 32, 128,  64, 1, 4, 1, D, S, 3, true, 1, 1>>();
        Add<C::Type< 16,  64, 128, 1, 2, 2, D, S, 3, true, 1, 1>>(); // 10
        Add<C::Type< 16, 128,  64, 1, 4, 1, D, S, 3, true, 1, 1>>();
        // clang-format on
    }

    if constexpr (std::is_same_v<T, half>) {
        using C = Sm80_s16816<Sm80,
                              half,
                              Operand_A<half, kRowMajor>,             // A
                              Transform_Default,                      // tarnsform A
                              VoidOperand,                            // U
                              Operand_B_Pack<uint4_t, kRowMajor, 2>,  // B
                              Transform_HMMA_16816<1, 0>,             // transform B,
                              Operand_UV_Pack<uint32_t, true>,        // V
                              kRowMajor,                              // order_C
                              half,                                   // Tc
                              Striding::kIndexed,                     // indexed input
                              Striding::kBlocked,
                              Striding::kBlocked,
                              kColMajor,
                              0>;

        // clang-format off
        Add<C::Type<128, 256,  32, 2, 4, 1, D, D, 3,   0 , 1, 128>>();  // 10 + 5 + 4 + 10 + 10, 37
        Add<C::Type<128, 128,  32, 1, 4, 1, D, D, 3, true, 1, 128>>();  // 1 + 6 + 4 + 4 + 2, 3
        Add<C::Type< 64, 128,  64, 1, 4, 1, D, S, 3, true, 1, 128>>();  // 7 + 4 + 6 + 2 + 4, 26
        Add<C::Type< 64, 256,  32, 1, 4, 1, D, S, 3, true, 1, 128>>();  // 18
        Add<C::Type< 32,  64, 128, 1, 2, 2, D, S, 3, true, 1, 128>>();  // 2
        Add<C::Type< 32, 128,  64, 1, 4, 1, D, S, 5, true, 1, 128>>();  // 1 + 2 + 2 + 2 + 2, 2
        Add<C::Type< 32, 256,  64, 1, 4, 1, D, S, 3, true, 1, 128>>();  // 9
        Add<C::Type< 16, 256,  64, 1, 4, 1, D, S, 3, true, 1, 128>>();  // 22
        Add<C::Type< 16, 256,  32, 1, 4, 1, D, S, 3, true, 1, 128>>();  // 8
        Add<C::Type< 16, 128,  64, 1, 4, 1, D, S, 3, true, 1, 128>>();  // 1 + 13 + 9 + 13 + 7, 7
        Add<C::Type< 16,  64, 128, 1, 2, 2, D, S, 3, true, 1, 128>>();  // 12 + 2 + 6 + 2 + 8, 42
        // clang-format on
    }
#endif

    if constexpr (std::is_same_v<T, bfloat16_t>) {
        using C = Config_e4m3<bfloat16_t, 16>;

        // clang-format off
        Add<C::Type<128, 128,  32, 4, 1, 1, D, D, 3, true, 128, 1, 128, 64>>();  
        Add<C::Type<128,  96,  32, 4, 1, 1, D, D, 3, true, 128, 1>>();
        Add<C::Type<128,  64,  32, 4, 1, 1, D, D, 3, true, 128, 1>>();
        Add<C::Type<128,  32,  32, 4, 1, 1, S, D, 3, true, 128, 1>>();
        Add<C::Type<128,  16,  64, 4, 1, 1, S, D, 3, true, 128, 1>>();
        Add<C::Type<128,  16,  32, 4, 1, 1, S, D, 5, true, 128, 1>>();
        // clang-format on

        using C8 = Config_e4m3<bfloat16_t, 8>;
        Add<C8::Type<128, 8, 128, 4, 1, 1, S, D, 3, true, 128, 1>>();
        Add<C8::Type<128, 8, 64, 4, 1, 1, S, D, 3, true, 128, 1>>();
    }
}

template void Registry::sm80_s16816_dynamic<half>();
template void Registry::sm80_s16816_dynamic<hip_bfloat16>();

}  // namespace turbomind::gemm
