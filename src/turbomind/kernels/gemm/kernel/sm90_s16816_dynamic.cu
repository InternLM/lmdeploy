#include "hip/hip_runtime.h"
// Copyright (c) OpenMMLab. All rights reserved.

#include "src/turbomind/kernels/gemm/arch/config_sm80_s16816.h"
#include "src/turbomind/kernels/gemm/cta_map.h"
#include "src/turbomind/kernels/gemm/registry.h"
#include "src/turbomind/kernels/gemm/transform.h"
#include "src/turbomind/kernels/gemm/types.h"

namespace turbomind::gemm {

using namespace sm80_s16816;
using namespace cache_policy;
using S = cache_policy::Stream;
using D = cache_policy::Default;

namespace {

template<class T, int N>
using Config_e4m3 = Sm80_s16816<Sm90,
                                T,
                                Operand_A_Pack<fp8_e4m3_t, kColMajor, 1>,  // A
                                Transform_HMMA_16816<0, 1>,                // tarnsform A
                                Operand_UV_Pack<uint16_t, false>,          // U
                                Operand_B<T, kRowMajor, N>,                // B
                                Transform_Default,                         // transform B
                                VoidOperand,                               // V
                                kColMajor,                                 // order_C
                                T,                                         // Tc
                                Striding::kBlocked,
                                Striding::kIndexed,  // indexed input
                                Striding::kBlocked,
                                kColMajor,
                                1>;

}  // namespace

template<class T>
void Registry::sm90_s16816_dynamic()
{
    if constexpr (std::is_same_v<T, half>) {
        using C = Sm80_s16816<Sm90,
                              half,
                              Operand_A<half, kRowMajor>,          // A
                              Transform_Default,                   // tarnsform A
                              VoidOperand,                         // U
                              Operand_B_Pack<half, kRowMajor, 1>,  // B
                              Transform_Default,                   // transform B
                              VoidOperand,                         // V
                              kRowMajor,                           // order_C
                              half,                                // Tc
                              Striding::kIndexed,                  // indexed input
                              Striding::kBlocked,
                              Striding::kBlocked,
                              kColMajor,
                              0>;

        // clang-format off
        Add<C::Type<256, 128,  64, 4, 2, 1, D, D, 3,   0 , 1, 1>>();
        Add<C::Type<128, 256,  64, 2, 4, 1, D, D, 3,   0 , 1, 1>>();
        Add<C::Type<128, 256,  32, 2, 4, 1, D, D, 3,   0 , 1, 1>>();
        Add<C::Type<128, 128,  32, 2, 2, 1, D, D, 3, true, 1, 1>>();
        Add<C::Type<128, 128,  64, 2, 2, 1, D, D, 3, true, 1, 1>>();
        Add<C::Type<128, 128,  32, 2, 2, 1, D, D, 5, true, 1, 1>>();
        Add<C::Type< 96,  64,  64, 2, 2, 1, D, D, 3, true, 1, 1>>();
        Add<C::Type< 64, 128,  64, 1, 4, 1, D, D, 3, true, 1, 1>>();
        Add<C::Type< 64,  64,  64, 2, 2, 1, D, D, 3, true, 1, 1>>();
        Add<C::Type< 64,  64,  64, 2, 2, 1, D, D, 5, true, 1, 1>>();
        Add<C::Type< 64,  64, 128, 1, 2, 2, D, D, 3, true, 1, 1>>();
        Add<C::Type< 32,  64, 128, 1, 2, 2, D, D, 3, true, 1, 1>>();
        Add<C::Type< 32, 128,  64, 1, 4, 1, D, D, 3, true, 1, 1>>();
        Add<C::Type< 16,  64, 128, 1, 2, 2, D, D, 3, true, 1, 1>>();
        Add<C::Type< 16, 128,  64, 1, 4, 1, D, D, 3, true, 1, 1>>();
        // clang-format on
    }
    else if constexpr (std::is_same_v<T, hip_bfloat16>) {
        using C = Sm80_s16816<Sm90,
                              hip_bfloat16,
                              Operand_A<hip_bfloat16, kRowMajor>,          // A
                              Transform_Default,                          // tarnsform A
                              VoidOperand,                                // U
                              Operand_B_Pack<hip_bfloat16, kRowMajor, 1>,  // B
                              Transform_Default,                          // transform B
                              VoidOperand,                                // V
                              kRowMajor,                                  // order_C
                              hip_bfloat16,                                // Tc
                              Striding::kIndexed,                         // indexed input
                              Striding::kBlocked,
                              Striding::kBlocked,
                              kColMajor,
                              0>;

        // clang-format off
        Add<C::Type<256, 128,  64, 4, 2, 1, D, D, 3,   0 , 1, 1>>();
        Add<C::Type<128, 256,  64, 2, 4, 1, D, D, 3,   0 , 1, 1>>();
        Add<C::Type<128, 256,  32, 2, 4, 1, D, D, 3,   0 , 1, 1>>();
        Add<C::Type<128, 128,  32, 2, 2, 1, D, D, 3, true, 1, 1>>();
        Add<C::Type<128, 128,  64, 2, 2, 1, D, D, 3, true, 1, 1>>();
        Add<C::Type<128, 128,  32, 2, 2, 1, D, D, 5, true, 1, 1>>();
        Add<C::Type< 96,  64,  64, 2, 2, 1, D, D, 3, true, 1, 1>>();
        Add<C::Type< 64, 128,  64, 1, 4, 1, D, D, 3, true, 1, 1>>();
        Add<C::Type< 64,  64,  64, 2, 2, 1, D, D, 3, true, 1, 1>>();
        Add<C::Type< 64,  64,  64, 2, 2, 1, D, D, 5, true, 1, 1>>();
        Add<C::Type< 64,  64, 128, 1, 2, 2, D, D, 3, true, 1, 1>>();
        Add<C::Type< 32,  64, 128, 1, 2, 2, D, D, 3, true, 1, 1>>();
        Add<C::Type< 32, 128,  64, 1, 4, 1, D, D, 3, true, 1, 1>>();
        Add<C::Type< 16,  64, 128, 1, 2, 2, D, D, 3, true, 1, 1>>();
        Add<C::Type< 16, 128,  64, 1, 4, 1, D, D, 3, true, 1, 1>>();
        // clang-format on
    }

    if constexpr (std::is_same_v<T, half>) {
        using C = Sm80_s16816<Sm90,
                              half,
                              Operand_A<half, kRowMajor>,             // A
                              Transform_Default,                      // tarnsform A
                              VoidOperand,                            // U
                              Operand_B_Pack<uint4_t, kRowMajor, 2>,  // B
                              Transform_HMMA_16816<1, 0>,             // transform B,
                              Operand_UV_Pack<uint32_t, true>,        // V
                              kRowMajor,                              // order_C
                              half,                                   // Tc
                              Striding::kIndexed,                     // indexed input
                              Striding::kBlocked,
                              Striding::kBlocked,
                              kColMajor,
                              0>;

        // clang-format off
        Add<C::Type<128, 256,  32, 2, 4, 1, D, D, 3,   0 , 1, 128>>();
        Add<C::Type<128, 128,  32, 1, 4, 1, D, D, 3, true, 1, 128>>();
        Add<C::Type< 64, 128,  64, 1, 4, 1, D, D, 3, true, 1, 128>>();
        Add<C::Type< 64, 256,  32, 1, 4, 1, D, D, 3, true, 1, 128>>();
        Add<C::Type< 32,  64, 128, 1, 2, 2, D, D, 3, true, 1, 128>>();
        Add<C::Type< 32, 128,  64, 1, 4, 1, D, D, 5, true, 1, 128>>();
        Add<C::Type< 32, 256,  64, 1, 4, 1, D, D, 3, true, 1, 128>>();
        Add<C::Type< 16, 256,  64, 1, 4, 1, D, D, 3, true, 1, 128>>();
        Add<C::Type< 16, 256,  32, 1, 4, 1, D, D, 3, true, 1, 128>>();
        Add<C::Type< 16, 128,  64, 1, 4, 1, D, D, 3, true, 1, 128>>();
        Add<C::Type< 16,  64, 128, 1, 2, 2, D, D, 3, true, 1, 128>>();
        // clang-format on
    }

    if constexpr (std::is_same_v<T, bfloat16_t>) {
        using C_ = Config_e4m3<bfloat16_t, 16>;
        using C8 = Config_e4m3<bfloat16_t, 8>;

        // clang-format off
        Add<C_::Type<128, 128,  32, 4, 1, 1, D, D, 3, true, 128, 1, 128, 64>>();  // sm_120
        Add<C_::Type<128,  96,  32, 4, 1, 1, D, D, 3, true, 128, 1>>();
        Add<C_::Type<128,  64,  32, 4, 1, 1, D, D, 3, true, 128, 1>>();
        Add<C_::Type<128,  32,  32, 4, 1, 1, D, D, 3, true, 128, 1>>();
        Add<C_::Type<128,  16,  64, 4, 1, 1, D, D, 3, true, 128, 1>>();
        Add<C8::Type<128,   8,  64, 4, 1, 1, D, D, 3, true, 128, 1>>();
        Add<C8::Type<128,   8, 128, 4, 1, 1, D, D, 3, true, 128, 1>>();
        // clang-format on
    }
}

template void Registry::sm90_s16816_dynamic<half>();
template void Registry::sm90_s16816_dynamic<hip_bfloat16>();

}  // namespace turbomind::gemm
