#include "hip/hip_runtime.h"
// Copyright (c) OpenMMLab. All rights reserved.

#include <algorithm>
#include <cstdio>
#include <limits>
#include <numeric>
#include <random>

#include <cub/block/block_reduce.cuh>
#include <cub/block/block_scan.cuh>
#include <cub/warp/warp_scan.cuh>

#include "src/turbomind/kernels/core/array_ops.h"
#include "src/turbomind/kernels/core/common.h"
#include "src/turbomind/kernels/core/math.h"
#include "src/turbomind/kernels/gemm/moe_utils_v2.h"

namespace turbomind {

template<int top_k, int block_dim>
__global__ void MoeGateKernel_V2(float*       scales,  // [e,n]
                                 int8_t*      masks,   // [E,n], padded
                                 int*         accum,   // [E,tiles]
                                 const float* logits,  // [E,n]
                                 int          log_tile,
                                 int          tiles,
                                 int          tokens,
                                 int          tokens_padded,
                                 int          experts)
{
    constexpr int max_tiles = kMoeGateMaxTiles;

    // Brute-force per thread top-k using a flat thread mapping
    const int ti = threadIdx.x + blockIdx.x * blockDim.x;

    // Clear masks
    for (int e = 0; e < experts; ++e) {
        if (ti < tokens_padded) {
            masks[e * tokens_padded + ti] = -1;
        }
    }

    __shared__ int shared_accum[32][max_tiles];

    for (int i = threadIdx.x; i < experts * max_tiles; i += block_dim) {
        int e = i / max_tiles;
        int t = i % max_tiles;
        if (e < experts && t < tiles) {
            shared_accum[e][t] = 0;
        }
    }

    __syncthreads();

    if (ti < tokens) {

        static_assert(top_k <= 32);
        int mask = -1;

        float max_logit = 0.f;

        // Find top-k
        PRAGMA_UNROLL
        for (int k = 0; k < top_k; ++k) {
            int   max_bit = 0;
            float max_val = -std::numeric_limits<float>::infinity();
            int   bit     = 1;
            for (int e = 0; e < experts; ++e) {
                const auto val = logits[ti * experts + e];
                // const auto val = logits[e * tokens + ti];
                if ((mask & bit) && val > max_val) {
                    max_bit = bit;
                    max_val = val;
                }
                bit *= 2;
            }
            mask -= max_bit;
            if (k == 0) {
                max_logit = max_val;
            }
        }

        mask = ~mask;

        Array<float, top_k> top_val;
        PRAGMA_UNROLL
        for (int i = 0; i < top_k; ++i) {
            const int lowbit = (mask & -mask);
            const int e      = 31 - __clz(lowbit);

            // printf("e = %d, ti = %d, idx = %d\n", e, ti, i);

            masks[e * tokens_padded + ti] = i;
            atomicAdd(&shared_accum[e][ti >> log_tile], 1);
            top_val[i] = logits[ti * experts + e];
            // top_val[i] = logits[e * tokens + ti];

            mask -= lowbit;
        }

        float prob_sum = 0.f;
        PRAGMA_UNROLL
        for (int i = 0; i < top_k; ++i) {
            top_val[i] = expf(top_val[i] - max_logit);
            prob_sum += top_val[i];
        }

        PRAGMA_UNROLL
        for (int i = 0; i < top_k; ++i) {
            scales[i * tokens + ti] = fdividef(top_val[i], prob_sum);
        }
    }

    __syncthreads();

    for (int i = threadIdx.x; i < experts * max_tiles; i += block_dim) {
        int e = i / max_tiles;
        int t = i % max_tiles;
        if (e < experts && t < tiles) {
            atomicAdd(accum + e * tiles + t, shared_accum[e][t]);
        }
    }
}

template<int block_dim, class Mask>
__global__ void MoeScanKernel_v2(int*       f2n,      // [e*n]
                                 int*       en2f,     // [e,n]
                                 int*       offsets,  // [E+1]
                                 Mask*      masks,    // [E,n], padded
                                 const int* accum,    // [E,tiles]
                                 int        log_tile,
                                 int        tiles,
                                 int        tokens,
                                 int        tokens_padded,
                                 int        experts)
{
    using BlockReduce = hipcub::BlockReduce<int, block_dim>;
    using BlockScan   = hipcub::BlockScan<int, block_dim>;

    __shared__ union TempStorage {
        typename BlockReduce::TempStorage reduce;
        typename BlockScan::TempStorage   scan;
    } temp_storage;

    constexpr int vec_size = kMoeGateVecSize;

    using Vec = Array<Mask, vec_size>;

    const int tile_id = blockIdx.x;
    const int ei      = blockIdx.y;

    const int  global_tile_id = ei * tiles + tile_id;
    const bool is_valid       = global_tile_id <= experts * tiles;

#if 0
    int vacc[4]{};
    {
        int idx = threadIdx.x;
        PRAGMA_UNROLL
        for (int i = 0; i < 4; ++i) {
            if (idx < global_tile_id) {
                vacc[i] = accum[idx];
            }
            idx += block_dim;
        }
    }

    int offset = BlockReduce{temp_storage.reduce}.Sum(vacc);
#else

    int vacc = 0;
    for (int i = threadIdx.x; i < global_tile_id; i += block_dim) {
        if (is_valid && i < global_tile_id) {
            vacc += accum[i];
        }
    }

    int offset = BlockReduce{temp_storage.reduce}.Sum(vacc);

#endif

    __shared__ int shared_offset;

    if (threadIdx.x == 0) {
        shared_offset = offset;
        if (tile_id == 0) {
            offsets[ei] = offset;
        }
    }

    if (ei == experts) {
        return;
    }

    __syncthreads();

    offset = shared_offset;

    const int token_vecs = tokens_padded / vec_size;

    const int tile_size     = 1 << log_tile;
    const int tile_vec_size = tile_size / vec_size;

    const int tile_vec_beg    = tile_id * tile_vec_size;
    const int tile_vec_end    = std::min(tile_vec_beg + tile_vec_size, token_vecs);
    const int tile_vec_padded = tile_vec_beg + round_up(tile_vec_size, block_dim);

    // if (threadIdx.x == 0) {
    //     printf("%d %d %d\n", tile_vec_beg, tile_vec_end, tile_vec_padded);
    // }

    auto mask_ptr = (Vec*)masks + ei * token_vecs;

    for (int vi = tile_vec_beg + threadIdx.x; vi < tile_vec_padded; vi += block_dim) {

        const bool pred = vi < tile_vec_end;

        Vec data;
        fill(data, Mask{-1});
        if (pred) {
            Ldg(data, mask_ptr[vi].data());
        }

        int prefix[vec_size];
        PRAGMA_UNROLL
        for (int i = 0; i < vec_size; ++i) {
            prefix[i] = int(data[i] >= 0);
        }

        int block_sum = 0;

        BlockScan{temp_storage.scan}.ExclusiveSum(prefix, prefix, block_sum);
        __syncthreads();

        PRAGMA_UNROLL
        for (int i = 0; i < vec_size; ++i) {
            if (pred && data[i] >= 0) {
                const int flat_id = prefix[i] + offset;
                const int ti      = vi * vec_size + i;
                f2n[flat_id]      = ti;
                // No ti is generated for padded tokens so we are safe
                en2f[data[i] * tokens + ti] = flat_id;
            }
        }

        offset += block_sum;
    }
}

template<int max_expert_num,
         int max_top_k,
         //  bool norm_top_k,
         int items_per_thread,
         int block_dim,
         int access_size,
         class Mask>
__global__ void MoeGateKernel_v8(float*       scales,  // [e,n]
                                 Mask*        masks,   // [E,n], padded
                                 int*         accum,   // [E,tiles]
                                 const float* logits,  // [n,E]
                                 int          log_tile,
                                 int          tiles,
                                 int          token_num,
                                 int          token_num_padded,
                                 int          expert_num,
                                 int          top_k,
                                 bool         norm_topk)
{
    constexpr int max_tiles         = kMoeGateMaxTiles;
    constexpr int threads_per_token = max_expert_num / items_per_thread;  // 8
    constexpr int tokens_per_cta    = block_dim / threads_per_token;

    // We use bits in a uint32_t to represent selected experts
    static_assert(items_per_thread <= 32);
    // We use warp-level primitives for reduction
    static_assert(threads_per_token <= 32);

    static_assert((threads_per_token & (threads_per_token - 1)) == 0);

    const int thread_idx = threadIdx.x + blockIdx.x * blockDim.x;

    const int ti = thread_idx / threads_per_token;
    const int ei = thread_idx % threads_per_token;

    const int bti = threadIdx.x / threads_per_token;

    const int warp_ti = threadIdx.x % WARP_SIZE / threads_per_token;

    const int warp_offset  = thread_idx / WARP_SIZE * WARP_SIZE / threads_per_token;
    const int block_offset = thread_idx / block_dim * block_dim / threads_per_token;

    float data[items_per_thread];
    int   idxs[items_per_thread];

#if 0
    PRAGMA_UNROLL
    for (int i = 0; i < items_per_thread; ++i) {
        data[i] = -std::numeric_limits<float>::infinity();
        idxs[i] = threads_per_token * (i / access_size * access_size) + i % access_size + ei * access_size;
    }
    if (ti < token_num) {
        PRAGMA_UNROLL
        for (int i = 0; i < items_per_thread; i += access_size) {
            const int e = threads_per_token * i + ei * access_size;
            if (e < expert_num) {
                Ldg((Array<float, access_size>&)data[i], &logits[ti * expert_num + e]);
            }
        }
    }

    __shared__ union {
        struct {
            // +1 padding greatly reduced (-80%) bank conflicts
            int   shared_accum[max_tiles][max_expert_num + 1];
            float shared_scales[max_top_k][tokens_per_cta];
            int   shared_exp_id[max_top_k][tokens_per_cta];
        };
    } smem;
#elif 1
    PRAGMA_UNROLL
    for (int i = 0; i < items_per_thread; ++i) {
        data[i] = -std::numeric_limits<float>::infinity();
        // idxs[i] = threads_per_token * (i / access_size * access_size) + i % access_size + ei * access_size;
        idxs[i] = ei * items_per_thread + i;
    }
    if (ti < token_num) {
        PRAGMA_UNROLL
        for (int i = 0; i < items_per_thread; i += access_size) {
            // const int e = threads_per_token * i + ei * access_size;
            const int e = ei * items_per_thread + i;
            if (e < expert_num) {
                Ldg((Array<float, access_size>&)data[i], &logits[ti * expert_num + e]);
            }
        }
    }

    __shared__ union {
        struct {
            // +1 padding greatly reduced (-80%) bank conflicts
            int   shared_accum[max_tiles][max_expert_num + 1];
            float shared_scales[max_top_k][tokens_per_cta];
            int   shared_exp_id[max_top_k][tokens_per_cta];
        };
    } smem;
#else

    const int warp_id = threadIdx.x / WARP_SIZE;
    const int lane_id = threadIdx.x % WARP_SIZE;

    constexpr int vecs_per_thread = items_per_thread / access_size;

    using Vec            = Array<float, access_size>;
    constexpr int banks  = 128 / sizeof(Vec);
    constexpr int chunks = 4;  // block_dim / WARP_SIZE;

    __shared__ union {
        Vec shared_data[chunks][vecs_per_thread * WARP_SIZE / banks][banks + 1];
        struct {
            // +1 padding greatly reduced (-80%) bank conflicts
            int   shared_accum[max_tiles][max_expert_num + 1];
            float shared_scales[max_top_k][tokens_per_cta];
            int   shared_exp_id[max_top_k][tokens_per_cta];
        };
    } smem;

    __align__(16) Vec vecs[vecs_per_thread];

    {
        const int warp_end = min(warp_offset + WARP_SIZE / threads_per_token, token_num) * expert_num;
        int       p        = warp_offset * expert_num + access_size * lane_id;
        PRAGMA_UNROLL
        for (int i = 0; i < vecs_per_thread; ++i) {
            fill(vecs[i], -std::numeric_limits<float>::infinity());
            // const int p = warp_offset * expert_num + access_size * (lane_id + i * WARP_SIZE);
            if (p < warp_end) {
                Ldg(vecs[i], &logits[p]);
            }
            p += access_size * WARP_SIZE;
        }
    }

    PRAGMA_UNROLL
    for (int c = 0; c < block_dim / WARP_SIZE; c += chunks) {
        PRAGMA_UNROLL
        for (int i = 0; i < vecs_per_thread; ++i) {
            int p = i * WARP_SIZE + lane_id;
            if (c <= warp_id && warp_id < c + chunks) {
                Store(smem.shared_data[warp_id - c][p / banks][p % banks].data(), vecs[i]);
            }
        }

        __syncwarp();

        PRAGMA_UNROLL
        for (int i = 0; i < vecs_per_thread; ++i) {
            int p = lane_id * vecs_per_thread + i;
            if (c <= warp_id && warp_id < c + chunks) {
                Load(vecs[i], smem.shared_data[warp_id - c][p / banks][p % banks].data());
            }
        }

        __syncthreads();
    }

    PRAGMA_UNROLL
    for (int i = 0; i < items_per_thread; ++i) {
        idxs[i] = ei * items_per_thread + i;
    }
    PRAGMA_UNROLL
    for (int i = 0; i < vecs_per_thread; ++i) {
        (Array<float, access_size>&)data[i * access_size] = vecs[i];
    }

#endif

    constexpr float kLog2e = 1.4426950408889634074;

    unsigned mask = (unsigned)-1;
    float    max_logit;

    int   count{};
    float sum_prob{};

    const int warp_ti_offset = warp_ti * threads_per_token;

    auto run = [&](int k) {
        unsigned bit     = 1;
        unsigned max_bit = 0;
        float    max_val = -std::numeric_limits<float>::infinity();
        // local maximum
        PRAGMA_UNROLL
        for (int i = 0; i < items_per_thread; ++i) {
            if ((mask & bit) && data[i] > max_val) {
                max_bit = bit;
                max_val = data[i];
            }
            asm("shl.b32 %0, %1, 1;\n" : "=r"(bit) : "r"(bit));
        }

        if (k == 0) {
            PRAGMA_UNROLL
            for (int i = 0; i < items_per_thread; ++i) {
                data[i] *= kLog2e;
            }
        }

        int   g_max_ei  = ei;
        float g_max_val = max_val;
        if constexpr (threads_per_token > 1) {
            // global maximum
            PRAGMA_UNROLL
            for (int m = threads_per_token / 2; m >= 1; m /= 2) {
                g_max_val = fmaxf(g_max_val, __shfl_xor_sync((uint32_t)-1, g_max_val, m));
            }
            // tie breaking
            const auto active = __ballot_sync((uint32_t)-1, max_val == g_max_val);
            g_max_ei          = __ffs(active >> (unsigned)warp_ti_offset) - 1;
        }
        if (k == 0) {
            max_logit = g_max_val;
        }
        if (ei == g_max_ei) {
            mask -= max_bit;
            ++count;
        }
    };

    run(0);

    for (int k = 1; k < top_k; ++k) {
        run(k);
    }

    mask = ~mask;

    int used[items_per_thread];
    {
        unsigned bit = 1;
        PRAGMA_UNROLL
        for (int i = 0; i < items_per_thread; ++i) {
            used[i] = (mask & bit) > 0;
            asm("shl.b32 %0, %1, 1;\n" : "=r"(bit) : "r"(bit));
        }
    }

    PRAGMA_UNROLL
    for (int i = 0; i < items_per_thread; ++i) {
        if (!norm_topk || used[i]) {
            data[i] = exp2f(data[i] - max_logit);
            sum_prob += data[i];
        }
    }

    PRAGMA_UNROLL
    for (int m = threads_per_token / 2; m >= 1; m /= 2) {
        sum_prob += __shfl_xor_sync((uint32_t)-1, sum_prob, m);
    }

    sum_prob = fdividef(1.f, sum_prob);

    using WarpScan = hipcub::WarpScan<int, threads_per_token>;
    __shared__ typename WarpScan::TempStorage temp_storage[tokens_per_cta];

    int idx{};
    WarpScan{temp_storage[bti]}.ExclusiveSum(count, idx);

    PRAGMA_UNROLL
    for (int i = 0; i < items_per_thread; ++i) {
        if (used[i]) {
            smem.shared_exp_id[idx][bti] = idxs[i];
            smem.shared_scales[idx][bti] = data[i] * sum_prob;
            ++idx;
        }
    }

    PRAGMA_UNROLL
    for (int i = 0; i < max_tiles * max_expert_num; i += block_dim) {
        int e                   = (i + threadIdx.x) % max_expert_num;
        int t                   = (i + threadIdx.x) / max_expert_num;
        smem.shared_accum[t][e] = 0;
    }

    __syncthreads();

    constexpr int k_per_thread = cdiv(max_top_k, threads_per_token);

    const int bti2 = threadIdx.x % tokens_per_cta;
    const int ei2  = threadIdx.x / tokens_per_cta;
    const int ti2  = blockIdx.x * tokens_per_cta + bti2;

    PRAGMA_UNROLL
    for (int i = 0; i < k_per_thread; ++i) {
        const int   idx       = ei2 * k_per_thread + i;
        const int   expert_id = smem.shared_exp_id[idx][bti2];
        const float scale     = smem.shared_scales[idx][bti2];

        if (ti2 < token_num && idx < top_k) {
            masks[expert_id * token_num_padded + ti2] = idx;
            scales[idx * token_num + ti2]             = scale;
            atomicAdd(&smem.shared_accum[ti2 >> log_tile][expert_id], 1);

            // printf("%d %d %f\n", idx, expert_id, scale);
        }
    }

    __syncthreads();

    for (int i = 0; i < max_expert_num * max_tiles; i += block_dim) {
        int t = (threadIdx.x + i) % max_tiles;
        int e = (threadIdx.x + i) / max_tiles;
        if (e < expert_num && t < tiles) {
            atomicAdd(accum + e * tiles + t, smem.shared_accum[t][e]);
        }
    }
}

template<int N>
inline constexpr std::integral_constant<int, N> _Int{};

void invokeMoeGate_V2(int*         f2n,            // [e*n]  -> n
                      int*         en2f,           // [e,n] -> n*e
                      int*         offsets,        // [E+1]
                      float*       scales,         // [e,n]
                      void*        masks,          // [E,n]
                      int*         accum,          // [E]
                      const float* logits,         // [e,n]
                      int          tokens,         //  n
                      int          tokens_padded,  //  round_up(n, 4)
                      int          experts,        //  E
                      int          experts_per_token,
                      bool         norm_topk,
                      hipStream_t st)
{
    constexpr int base_log_tile = 9;

    int log_tile = base_log_tile;
    while (((tokens_padded + (1 << log_tile) - 1) >> log_tile) > kMoeGateMaxTiles) {
        ++log_tile;
    }
    const int tiles = ceil_div(tokens_padded, 1 << log_tile);

    // std::cout << log_tile << " " << tiles << "\n";

    auto invoke = [&](auto max_expert_num, auto top_k, auto items_per_thread) {
        constexpr int thrs_per_tok = max_expert_num.value / items_per_thread.value;
        constexpr int threads      = 256;
        const int     blocks       = ceil_div(tokens, threads / thrs_per_tok);

        hipMemsetAsync(masks, -1, sizeof(int8_t) * experts * tokens_padded, st);

        MoeGateKernel_v8<max_expert_num.value, top_k.value, items_per_thread.value, threads, 4>
            <<<blocks, threads, 0, st>>>(  //
                scales,
                (int8_t*)masks,
                accum,
                logits,
                log_tile,
                tiles,
                tokens,
                tokens_padded,
                experts,
                experts_per_token,
                norm_topk);
    };

    auto fail = [&] {
        std::cerr << "unsupported moe config: expert_num=" << experts << ", top_k=" << experts_per_token << "\n";
        std::abort();
    };

    if (experts <= 8) {
        if (experts_per_token <= 2) {
            invoke(_Int<8>, _Int<2>, _Int<8>);
        }
        else {
            invoke(_Int<8>, _Int<8>, _Int<8>);
        }
    }
    else if (experts <= 64) {
        if (experts_per_token <= 4) {
            invoke(_Int<64>, _Int<4>, _Int<16>);
        }
        else if (experts_per_token <= 8) {
            invoke(_Int<64>, _Int<8>, _Int<16>);
        }
        else {
            fail();
        }
    }
    else {
        fail();
    }

    {
        constexpr int threads = (1 << base_log_tile) / kMoeGateVecSize;
        const dim3    blocks(tiles, experts + 1);

        MoeScanKernel_v2<threads><<<blocks, threads, 0, st>>>(f2n,  //
                                                              en2f,
                                                              offsets,
                                                              (int8_t*)masks,
                                                              accum,
                                                              log_tile,
                                                              tiles,
                                                              tokens,
                                                              tokens_padded,
                                                              experts);
    }
}

template<int vec_size, int block_dim, class T>
__global__ void MoeGatherKernel(T*         dst,  // [e*n, d]
                                const T*   src,  // [  n, d]
                                const int* f2n,  // [e*n] :: e*n -> n
                                int        dims)
{
    using Vec        = Array<T, vec_size>;
    const int64_t bi = blockIdx.x;

    auto src_ptr = (const Vec*)src + dims * f2n[bi];
    auto dst_ptr = (/* */ Vec*)dst + dims * bi;
    for (int i = threadIdx.x; i < dims; i += block_dim) {
        Vec v;
        Ldg(v, src_ptr[i].data());
        Store(dst_ptr[i].data(), v);
    }
}

template<class T>
void invokeMoeGather(T* dst, const T* src, const int* f2n, int tokens, int experts_per_token, int dims, hipStream_t st)
{
    constexpr int threads  = 256;
    constexpr int vec_size = 16 / sizeof(T);
    MoeGatherKernel<vec_size, threads><<<tokens * experts_per_token, threads, 0, st>>>(  //
        dst,
        src,
        f2n,
        dims / vec_size);
}

template void invokeMoeGather(uint16_t*, const uint16_t*, const int*, int, int, int, hipStream_t);

template<int vec_size, int exp_k, int block_dim, class T>
__global__ void MoeReduceKernel(T*           dst,         // [  n, d]
                                const T*     src,         // [e*n, d]
                                const float* scales,      // [  e, n]
                                const int*   en2f,        // [  e, n] :: (e,n) -> e*n
                                const float* dst_scales,  // [n]
                                int          dims,
                                int          tokens)
{
    using Vec = Array<T, vec_size>;

    const int64_t ti = blockIdx.x;

    auto dst_ptr = (Vec*)dst + dims * ti;

    float dst_scale = 0;
    if (dst_scales) {
        dst_scale = dst_scales[ti];
        dst_scale = fdividef(1.f, 1.f + expf(-dst_scale));
    }

    // Should be warp uniforms
    const Vec* src_ptr[exp_k];
    float      scale[exp_k];
    PRAGMA_UNROLL
    for (int e = 0; e < exp_k; ++e) {
        src_ptr[e] = (const Vec*)src + dims * en2f[e * tokens + ti];
        scale[e]   = scales ? scales[e * tokens + ti] : 1.f;
    }

    for (int i = threadIdx.x; i < dims; i += block_dim) {
        Array<float, vec_size> accum{};
        if (dst_scales) {
            Vec v;
            Ldg(v, dst_ptr[i].data());
            using namespace ops;
            accum = cast<float>(v) * dst_scale;
        }
        PRAGMA_UNROLL
        for (int e = 0; e < exp_k; ++e) {
            Vec v;
            Ldg(v, src_ptr[e][i].data());
            using namespace ops;
            const auto x = cast<float>(v) * scale[e];
            accum        = accum + x;
        }
        Store(dst_ptr[i].data(), cast<T>(accum));
    }
}

template<class T>
void invokeMoeReduce(T*           dst,
                     const T*     src,
                     const float* scales,
                     const int*   en2f,
                     const float* dst_scales,
                     int          tokens,
                     int          experts_per_token,
                     int          dims,
                     hipStream_t st)
{
    // std::cout << __PRETTY_FUNCTION__ << std::endl;

    const auto invoke = [&](auto e) {
        constexpr int threads     = 256;
        constexpr int vec_size    = 16 / sizeof(T);
        constexpr int exp_per_tok = decltype(e)::value;
        MoeReduceKernel<vec_size, exp_per_tok, threads><<<tokens, threads, 0, st>>>(  //
            dst,
            src,
            scales,
            en2f,
            dst_scales,
            dims / vec_size,
            tokens);
    };

    switch (experts_per_token) {
        case 1:
            return invoke(std::integral_constant<int, 1>{});
        case 2:
            return invoke(std::integral_constant<int, 2>{});
        case 4:
            return invoke(std::integral_constant<int, 4>{});
        case 6:
            return invoke(std::integral_constant<int, 6>{});
        case 8:
            return invoke(std::integral_constant<int, 8>{});
        default:
            fprintf(stderr, "Unsupported experts_per_token %d\n", experts_per_token);
            std::abort();
    }
}

template void invokeMoeReduce(half*, const half*, const float*, const int*, const float*, int, int, int, hipStream_t);
#ifdef ENABLE_BF16
template void
invokeMoeReduce(hip_bfloat16*, const hip_bfloat16*, const float*, const int*, const float*, int, int, int, hipStream_t);
#endif

std::vector<int> SampleUniform(int token_num, int expert_num, int exp_per_tok, std::mt19937& g)
{
    std::vector<int> idxs((size_t)token_num * exp_per_tok);
    std::vector<int> r(expert_num);
    std::iota(r.begin(), r.end(), 0);
    auto it = idxs.begin();
    for (int i = 0; i < token_num; ++i) {
        it = std::sample(r.cbegin(), r.cend(), it, exp_per_tok, g);
    }
    return idxs;
}

std::vector<int> SampleBalanced(int token_num, int expert_num, int exp_per_tok, std::mt19937& g)
{
    assert(exp_per_tok <= expert_num);
    std::vector<int> idxs((size_t)token_num * exp_per_tok);
    std::vector<int> q;

    std::vector<int> r(expert_num);
    std::iota(r.begin(), r.end(), 0);

    auto it = idxs.begin();
    for (int i = 0; i < token_num; ++i) {
        if ((int)q.size() < exp_per_tok) {
            const int k = q.size();
            // prepend the experts: [xxx] -> [yyy | xxx]
            q.insert(q.begin(), r.cbegin(), r.cend());
            // move duplicated experts to the front: [yyy | xxx] -> [xxx' | yyy' | xxx]
            int p = 0;
            std::for_each(q.cend() - k, q.cend(), [&](auto x) { std::swap(q[p++], q[x]); });
            // shuffle unique experts yyy'
            std::shuffle(q.begin() + p, q.end() - k, g);
        }
        it = std::copy(q.end() - exp_per_tok, q.end(), it);
        // remove used experts [xxx' | yyy' | xxx ] -> [xxx' | zzz]
        q.resize(q.size() - exp_per_tok);
        // alias [xxx] <- [xxx' | zzz]
    }
    assert(it == idxs.end());

    // shuffle to decorrelate adjacent tokens
    r.resize(token_num);
    std::iota(r.begin(), r.end(), 0);
    std::shuffle(r.begin(), r.end(), g);
    std::vector<int> ret(idxs.size());
    it = ret.begin();
    for (const auto& i : r) {
        it = std::copy_n(idxs.begin() + i * exp_per_tok, exp_per_tok, it);
    }
    assert(it == ret.end());
    return ret;
}

}  // namespace turbomind
