// Copyright (c) OpenMMLab. All rights reserved.

#include "src/turbomind/kernels/gemm/registry.h"

namespace turbomind::gemm {

Registry::Registry(std::shared_ptr<hipDeviceProp_t> device_prop):
    device_prop_{std::move(device_prop)}, arch_{device_prop_->major * 100 + device_prop_->minor * 10}
{
    f16_u4g128_f16_tnt_sm70_s884();
    f16_u4g128_f16_tnt_sm75_simt();
    f16_u4g128_f16_tnt_sm75_s16816();
    f16_u4g128_f16_tnt_sm80_s16816();
}

bool Registry::Add(std::unique_ptr<Kernel> kernel)
{
    if ((int)device_prop_->sharedMemPerBlockOptin < kernel->smem_size()) {
        return false;
    }
    if (arch_ < kernel->arch()) {
        return false;
    }
    std::cout << "register: " << kernel->name() << "\n";
    kernels_.push_back(std::move(kernel));
    return true;
}

}  // namespace turbomind::gemm