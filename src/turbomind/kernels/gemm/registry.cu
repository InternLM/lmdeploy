// Copyright (c) OpenMMLab. All rights reserved.

#include "src/turbomind/kernels/gemm/arch.h"
#include "src/turbomind/kernels/gemm/registry.h"

namespace turbomind::gemm {

Registry::Registry(std::shared_ptr<hipDeviceProp_t> device_prop):
    device_prop_{std::move(device_prop)}, arch_{device_prop_->major * 100 + device_prop_->minor * 10}
{
    f16_u4g128_f16_tnt_sm70_s884();
    f16_u4g128_f16_tnt_sm75_simt();
    f16_u4g128_f16_tnt_sm75_s16816();
    f16_u4g128_f16_tnt_sm80_s16816();
    f16_u4g128_f16_tnt_sm90_s16816();

    u4g128_f16_f16_nnn_sm80_s16816();
}

bool Registry::Add(std::unique_ptr<Kernel> kernel)
{
    if (!is_arch_compatible(kernel->arch(), arch_)) {
        return false;
    }
    if ((int)device_prop_->sharedMemPerBlockOptin < kernel->smem_size()) {
        return false;
    }
    // std::cout << "register: " << kernel->name()                                        //
    //           << ", shared: " << (kernel->smem_size() >> 10) << " KB"                  //
    //           << ", regs: " << kernel->desc().attr.numRegs                             //
    //           << ", local: " << (float)kernel->desc().attr.localSizeBytes << " bytes"  //
    //           << ", max_active_ctas: " << kernel->desc().max_active_ctas << " \n";

    kernels_.push_back(std::move(kernel));
    ptrs_.push_back(kernels_.back().get());
    return true;
}

}  // namespace turbomind::gemm
