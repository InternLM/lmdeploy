#include "hip/hip_runtime.h"
// Copyright (c) OpenMMLab. All rights reserved.

#include "src/turbomind/kernels/gemm/arch.h"
#include "src/turbomind/kernels/gemm/registry.h"

namespace turbomind::gemm {

Registry::Registry(std::shared_ptr<hipDeviceProp_t> device_prop):
    device_prop_{std::move(device_prop)}, arch_{device_prop_->major * 100 + device_prop_->minor * 10}
{
    f16_u4g128_f16_tnt_sm70_s884();
    f16_u4g128_f16_tnt_sm75_simt();
    f16_u4g128_f16_tnt_sm75_s16816();
    f16_u4g128_f16_tnt_sm80_s16816();
    f16_u4g128_f16_tnt_sm90_s16816();

    sm70_s884_dynamic();
    sm75_s16816_dynamic();
    sm80_s16816_dynamic<half>();
    sm90_s16816_dynamic<half>();
    sm80_s16816_dynamic<hip_bfloat16>();
    sm90_s16816_dynamic<hip_bfloat16>();

    // u4g128_f16_f16_nnn_sm80_s16816();
}

bool Registry::Add(std::unique_ptr<Kernel> kernel)
{
    bool is_valid = true;
    if (!is_arch_compatible(kernel->arch(), arch_)) {
        is_valid = false;
    }
    if ((int)device_prop_->sharedMemPerBlockOptin < kernel->smem_size()) {
        is_valid = false;
    }
    // if (is_valid) {
    //     std::cout << "register: " << kernel->name()                                        //
    //               << ", shared: " << (kernel->smem_size() >> 10) << " KB"                  //
    //               << ", regs: " << kernel->desc().attr.numRegs                             //
    //               << ", local: " << (float)kernel->desc().attr.localSizeBytes << " bytes"  //
    //               << ", max_active_ctas: " << kernel->desc().max_active_ctas * is_valid << " \n";
    // }
    if (is_valid) {
        kernels_.push_back(std::move(kernel));
        ptrs_.push_back(kernels_.back().get());
    }
    return true;
}

}  // namespace turbomind::gemm
