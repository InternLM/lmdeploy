#include "hip/hip_runtime.h"
// Copyright (c) OpenMMLab. All rights reserved.

#include "src/turbomind/kernels/gemm/test/test_utils.h"
#include <hipblas.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <fstream>
#include <iostream>

#define _CG_ABI_EXPERIMENTAL
#include <hip/hip_cooperative_groups.h>
#include <cooperative_groups/memcpy_async.h>
#include <cooperative_groups/reduce.h>

#include <thrust/device_ptr.h>
#include <thrust/iterator/zip_iterator.h>
#include <thrust/system/cuda/execution_policy.h>
#include <thrust/transform_reduce.h>

namespace turbomind {

hipblasHandle_t cublas_handle{};
hipStream_t   cublas_stream{};

template<typename T>
void Compare(const T* src, const T* ref, size_t stride, int dims, int bsz, bool show, float rtol, float atol)
{
    float asums{};
    float rsums{};
    int   outliers{};
    for (int nn = 0; nn < bsz; ++nn) {
        float abs_diff_sum{};
        float rel_diff_sum{};
        for (int mm = 0; mm < dims; ++mm) {
            auto x = float(src[nn * stride + mm]);
            auto y = float(ref[nn * stride + mm]);
            // if (show) {
            //     std::cout << x << "\t" << y << std::endl;
            // }
            auto abs_diff = std::abs(x - y);
            auto rel_diff = abs_diff / (std::max(std::abs(y), std::abs(x)) + 1e-8f);
            if (!(abs_diff <= atol + rtol * std::abs(y))) {
                ++outliers;
                if (show) {
                    std::cout << nn << "," << mm << "\t" << x << "\t" << y << std::endl;
                }
            }
            abs_diff_sum += abs_diff;
            rel_diff_sum += rel_diff;
        }
        asums += abs_diff_sum / dims;
        rsums += rel_diff_sum / dims;
    }
    const float abs_diff = asums / bsz;
    const float rel_diff = rsums / bsz;
    const float outlier  = outliers / (float)bsz;
    std::cout << "abs_diff = " << abs_diff << " rel_diff = " << rel_diff << " outliers = " << outlier << std::endl;
}

template void
Compare(const half* src, const half* ref, size_t stride, int dims, int bsz, bool show, float rtol, float atol);
template void
Compare(const float* src, const float* ref, size_t stride, int dims, int bsz, bool show, float rtol, float atol);
#if ENABLE_BF16
template void Compare(const hip_bfloat16* src,
                      const hip_bfloat16* ref,
                      size_t             stride,
                      int                dims,
                      int                bsz,
                      bool               show,
                      float              rtol,
                      float              atol);
#endif

template<class T>
std::vector<float>
FastCompare(const T* src, const T* ref, int dims, int bsz, hipStream_t stream, float rtol, float atol)
{
    auto       zip_iter = thrust::make_zip_iterator(src, ref);
    const auto count    = (size_t)dims * bsz;
    // nvcc-11.8: __host__ __device__ lambda can't be generic
    using Tuple = thrust::tuple<float, float, float, float, float, float, int64_t>;
    auto res    = thrust::transform_reduce(
        thrust::cuda::par.on(stream),
        zip_iter,
        zip_iter + count,
        [=] __device__(auto tup) {
            float   s        = thrust::get<0>(tup);
            float   r        = thrust::get<1>(tup);
            float   abs_diff = fabsf(s - r);
            float   abs_s    = fabsf(s);
            float   abs_r    = fabsf(r);
            float   rel_diff = abs_diff / (fmaxf(abs_r, abs_s) + 1e-8f);
            int64_t outlier  = !(abs_diff <= (atol + rtol * abs_r));
            return thrust::make_tuple(abs_s, abs_r, abs_diff, abs_diff, rel_diff, rel_diff, outlier);
        },
        thrust::make_tuple(0.f, 0.f, 0.f, 0.f, 0.f, 0.f, 0LL),
        [] __host__ __device__(const Tuple& a, const Tuple& b) {  // `__host__`: compiler needs the return type
            return thrust::make_tuple(thrust::get<0>(a) + thrust::get<0>(b),
                                      thrust::get<1>(a) + thrust::get<1>(b),
                                      thrust::get<2>(a) + thrust::get<2>(b),
                                      fmaxf(thrust::get<3>(a), thrust::get<3>(b)),
                                      thrust::get<4>(a) + thrust::get<4>(b),
                                      fmaxf(thrust::get<5>(a), thrust::get<5>(b)),
                                      thrust::get<6>(a) + thrust::get<6>(b));
        });
    return {thrust::get<0>(res) / dims / bsz,   // avg abs src
            thrust::get<1>(res) / dims / bsz,   // avg abs ref
            thrust::get<2>(res) / dims / bsz,   // avg abs diff
            thrust::get<3>(res),                // max abs diff
            thrust::get<4>(res) / dims / bsz,   // avg rel diff
            thrust::get<5>(res),                // max rel diff
            (float)thrust::get<6>(res) / bsz};  // outlier count
}

template std::vector<float>
FastCompare(const half* src, const half* ref, int dims, int bsz, hipStream_t stream, float rtol, float atol);

void LoadBinary(const std::string& path, size_t size, void* dst)
{
    std::ifstream ifs(path, std::ios::binary | std::ios::in);
    if (!ifs.is_open()) {
        std::cerr << "failed to open " << path << "\n";
        std::abort();
    }
    ifs.seekg(0, ifs.end);
    auto actual_size_in_bytes = ifs.tellg();
    ifs.seekg(0, ifs.beg);
    if (size != actual_size_in_bytes) {
        std::cerr << "[warning] file " << path << " has " << actual_size_in_bytes << " bytes, while " << size
                  << " bytes is requested\n";
    }
    ifs.read((char*)dst, size);
    std::cerr << "[info] " << path << " " << size << "\n";
}

namespace cg = cooperative_groups;

__global__ void hiprand_init(hiprandState* state)
{
    auto tid = cg::this_grid().thread_rank();
    hiprand_init(0xe4c45822e90461ddULL, tid, 0, state + tid);
}

template<typename T>
__global__ void hiprand_uniform(hiprandState* state, size_t count, T* result, float scale, float shift)
{
    auto grid = cg::this_grid();
    for (auto i = grid.thread_rank(); i < count; i += grid.size()) {
        float tmp = hiprand_uniform(state + grid.thread_rank());
        result[i] = T(scale * tmp + shift);
    }
}

template<typename T>
__global__ void hiprand_normal(hiprandState* state, size_t count, T* result, float scale, float shift)
{
    auto grid = cg::this_grid();
    for (auto i = grid.thread_rank(); i < count; i += grid.size()) {
        float tmp = hiprand_normal(state + grid.thread_rank());
        result[i] = T(scale * tmp + shift);
    }
}

__global__ void curand_bytes(hiprandState* state, size_t count, uint* result)
{
    auto grid = cg::this_grid();
    for (auto i = grid.thread_rank(); i < count; i += grid.size()) {
        result[i] = hiprand(state + grid.thread_rank());
    }
}

struct RNG::Impl {

    hiprandState* states{};

    Impl()
    {
        hipMalloc(&states, sizeof(hiprandState) * 64 * 64);
        hiprand_init<<<64, 64>>>(states);
    }

    ~Impl()
    {
        hipFree(states);
    }

    void GenerateUInt(uint* out, size_t count)
    {
        curand_bytes<<<64, 64, 0, stream_>>>(states, count, out);
    }

    template<typename T>
    void GenerateUniform(T* out, size_t count, float scale, float shift)
    {
        hiprand_uniform<<<64, 64, 0, stream_>>>(states, count, out, scale, shift);
    }

    template<typename T>
    void GenerateNormal(T* out, size_t count, float scale, float shift)
    {
        hiprand_normal<<<64, 64, 0, stream_>>>(states, count, out, scale, shift);
    }

    hipStream_t stream_{};
};

RNG::RNG(): impl_(std::make_unique<Impl>()) {}

RNG::~RNG() = default;

void RNG::GenerateUInt(uint* out, size_t count)
{
    impl_->GenerateUInt(out, count);
}

template<typename T>
void RNG::GenerateUniform(T* out, size_t count, float scale, float shift)
{
    impl_->GenerateUniform(out, count, scale, shift);
}

template<typename T>
void RNG::GenerateNormal(T* out, size_t count, float scale, float shift)
{
    impl_->GenerateNormal(out, count, scale, shift);
}

hipStream_t RNG::stream() const
{
    return impl_->stream_;
}

void RNG::set_stream(hipStream_t stream)
{
    impl_->stream_ = stream;
}

template void RNG::GenerateUniform(half* out, size_t count, float scale, float shift);
template void RNG::GenerateUniform(float* out, size_t count, float scale, float shift);
#if ENABLE_BF16
template void RNG::GenerateUniform(hip_bfloat16* out, size_t count, float scale, float shift);
#endif

template void RNG::GenerateNormal(half* out, size_t count, float scale, float shift);
template void RNG::GenerateNormal(float* out, size_t count, float scale, float shift);
#if ENABLE_BF16
template void RNG::GenerateNormal(hip_bfloat16* out, size_t count, float scale, float shift);
#endif

}  // namespace turbomind
