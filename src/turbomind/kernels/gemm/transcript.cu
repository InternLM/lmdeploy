// Copyright (c) OpenMMLab. All rights reserved.

#include "src/turbomind/kernels/attention/quantization.h"
#include "src/turbomind/kernels/gemm/cta_map.h"
#include "src/turbomind/kernels/gemm/format.h"
#include "src/turbomind/kernels/gemm/impl_81616.h"
#include "src/turbomind/kernels/gemm/tile_iterator.h"
#include "src/turbomind/kernels/gemm/transcript.h"
#include "src/turbomind/kernels/gemm/transcript_template.h"
#include <type_traits>

namespace turbomind::gemm {

namespace {

template<class Ti, class To>
struct _Converter {
    __device__ _Converter(): impl_(1, 0) {}
    template<class T>
    __device__ auto operator()(T&& t) const
    {
        return impl_((T&&)t);
    }
    ConvertKvCache<Ti, To> impl_;
};

struct BaseConfig {
    static constexpr int CTA_M = 64;
    static constexpr int CTA_N = 64;
    static constexpr int CTA_K = 32;

    static constexpr int WARP_M = 64;
    static constexpr int WARP_N = 64;
    static constexpr int WARP_K = 32;
};

template<class T, class TbI_, class TbO>
struct Config: BaseConfig {};

template<class T, class Tb>
struct Config<T, T, Tb>: BaseConfig {
    using Gemm0  = Impl<MMA_81616, T, T, CTA_M, CTA_N, CTA_K, WARP_M, WARP_N, WARP_K, 3, 0>;
    using Gemm1  = Impl<MMA_81616, T, Tb, CTA_M, CTA_N, CTA_K, WARP_M, WARP_N, WARP_K, 3, 0>;
    using Kernel = Transcript<void, Gemm0, Gemm1, _Converter<T, Tb>, CtaSwizzleMap<0>>;
};

template<class T>
struct Config<T, uint16_t, uint4_t>: BaseConfig {
    static_assert(sizeof(T) == 2);
    using Gemm0  = Impl<MMA_81616, T, T, CTA_M, CTA_N, CTA_K, WARP_M, WARP_N, WARP_K, 3, 0>;
    using Gemm1  = Impl<MMA_81616, T, uint4_t, CTA_M, CTA_N, CTA_K, WARP_M, WARP_N, WARP_K, 3, 0>;
    using Kernel = Transcript<void, Gemm0, Gemm1, Converter<uint16_t, uint4_t>, CtaSwizzleMap<0>>;
};

template<class T>
struct Config<T, uint16_t, uint8_t>: BaseConfig {
    static_assert(sizeof(T) == 2);
    using Gemm0  = Impl<MMA_81616, T, T, CTA_M, CTA_N, CTA_K, WARP_M, WARP_N, WARP_K, 3, 0>;
    using Gemm1  = Impl<MMA_81616, T, uint8_t, CTA_M, CTA_N, CTA_K, WARP_M, WARP_N, WARP_K, 3, 0>;
    using Kernel = Transcript<void, Gemm0, Gemm1, Converter<uint16_t, uint8_t>, CtaSwizzleMap<0>>;
};

}  // namespace

namespace detail {

template<class T>
auto cast(T* p)
{
    if constexpr (bitsof<T> % 8 == 0) {
        return p;
    }
    else {
        return (char*)p;
    }
}

}  // namespace detail

template<class T, class Ti, class To>
void transcript(To* dst, const Ti* src, int n, int k, hipStream_t st)
{
    using Kernel = typename Config<T, Ti, To>::Kernel;

    static constexpr int kSmemSize = sizeof(typename Kernel::SharedStorage);
    if constexpr (kSmemSize > (48 << 10)) {
        hipFuncSetAttribute(reinterpret_cast<const void*>(transcript_kernel<Kernel>), hipFuncAttributeMaxDynamicSharedMemorySize, kSmemSize);
    }

    using Map = typename Kernel::CtaMap;

    auto tiles = Map::get_tiled_shape(Kernel::CTA_M, n, k, Kernel::CTA_M, Kernel::CTA_N, 1);
    auto grid  = Map::get_grid_shape(tiles);
    auto block = Kernel::WARP_CNT * WARP_SIZE;

    std::cout << "P_K: " << Kernel::P_K << ", P_N: " << Kernel::P_N << std::endl;

    auto _src = [&] {
        if constexpr (std::is_same_v<Ti, uint16_t>) {
            return (const T*)src;
        }
        else {
            return src;
        }
    }();

    typename Kernel::Param params{nullptr, _src, detail::cast(dst), Kernel::CTA_M, n, k};

    transcript_kernel<Kernel><<<grid, block, kSmemSize, st>>>(params);
}

template void transcript<half>(half* dst, const half* src, int n, int k, hipStream_t st);

template void transcript<half>(uint4_t* dst, const half* src, int n, int k, hipStream_t st);
template void transcript<half>(uint8_t* dst, const half* src, int n, int k, hipStream_t st);

template void transcript<half>(uint4_t* dst, const uint16_t* src, int n, int k, hipStream_t st);
template void transcript<half>(uint8_t* dst, const uint16_t* src, int n, int k, hipStream_t st);

}  // namespace turbomind::gemm