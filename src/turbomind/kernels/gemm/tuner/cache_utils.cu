// Copyright (c) OpenMMLab. All rights reserved.

#include "src/turbomind/kernels/gemm/tuner/cache_utils.h"

namespace turbomind::gemm {

CacheFlushing::CacheFlushing()
{
    hipDeviceProp_t props{};
    hipGetDeviceProperties(&props, 0);

    size_ = props.l2CacheSize;

    hipMalloc(&buffer_, size_);
}

void CacheFlushing::flush(hipStream_t stream)
{
    thread_local CacheFlushing inst{};
    inst(stream);
}

void CacheFlushing::operator()(hipStream_t stream) const
{
    hipMemsetAsync(buffer_, 0, size_, stream);
}

}  // namespace turbomind::gemm
