// Copyright (c) OpenMMLab. All rights reserved.

#include "src/turbomind/kernels/gemm/kernel.h"
#include "src/turbomind/kernels/gemm/tuner/cache_utils.h"
#include "src/turbomind/kernels/gemm/tuner/measurer.h"
#include <iostream>

namespace turbomind::gemm {

Measurer::Measurer(std::unique_ptr<StoppingCriterion> stop_criterion): stop_criterion_{std::move(stop_criterion)}
{
    hipEventCreate(&ev_beg_);
    hipEventCreate(&ev_end_);
}

Measurer::~Measurer()
{
    hipEventDestroy(ev_beg_);
    hipEventDestroy(ev_end_);
    ev_beg_ = ev_end_ = {};
}

std::vector<Measurement>
Measurer::Measure(const std::vector<LaunchSpec>& specs, const Launcher& launcher, hipStream_t stream)
{
    std::vector<Measurement> m;
    m.reserve(specs.size());
    for (const auto& spec : specs) {
        auto measure = MeasureOne(spec, launcher, stream);
        if (measure.sample_count) {
            m.push_back(measure);
        }
        /// TODO: report error
    }
    return m;
}

Measurement Measurer::MeasureOne(LaunchSpec spec, const Launcher& launcher, hipStream_t stream)
{
    Stats       stats{};
    hipError_t status = hipSuccess;
    while (true) {
        float ms{};
        std::tie(ms, status) = ColdRun(spec, launcher, stream);
        if (status != hipSuccess) {
            break;
        }
        stats.add_sample(ms);
        // std::cout << spec.kernel->name() << " " << spec.swizzle << " " << stats.count() << " " << stats.mean() << " "
        //           << stats.get_variance() << "\n";
        if (stop_criterion_->should_stop(stats)) {
            break;
        }
    }
    return Measurement{
        status,
        stats.count(),
        stats.mean(),
        stats.get_variance(),
    };
}

std::pair<float, hipError_t> Measurer::ColdRun(LaunchSpec spec, const Launcher& launcher, hipStream_t stream)
{
    CacheFlushing::flush(stream);

    hipEventRecord(ev_beg_, stream);

    // std::cout << spec.kernel->name() << " " << spec.splits << " " << spec.swizzle << std::endl;

    launcher(spec, stream);

    hipEventRecord(ev_end_, stream);
    hipEventSynchronize(ev_end_);

    const auto status = hipGetLastError();
    float      ms{};

    if (status == hipSuccess) {
        hipEventElapsedTime(&ms, ev_beg_, ev_end_);
    }
    else {
        std::cerr << hipGetErrorString(status) << std::endl;
    }

    return {ms, status};
}

}  // namespace turbomind::gemm
