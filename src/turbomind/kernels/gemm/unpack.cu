#include "hip/hip_runtime.h"
// Copyright (c) OpenMMLab. All rights reserved.

#include "src/turbomind/kernels/core/array_ops.h"
#include "src/turbomind/kernels/core/common.h"
#include "src/turbomind/kernels/core/data_type.h"
#include <iostream>

namespace turbomind {

namespace {

__device__ void atomic_assign_u4(uint32_t* address, uint32_t index, uint32_t value)
{
    uint32_t old = *address;
    uint32_t assumed;
    do {
        assumed      = old;
        uint32_t tmp = (assumed & ~(0xfu << (index * 4u))) | (value << (index * 4u));
        old          = atomicCAS(address, assumed, tmp);
    } while (assumed != old);
}

__device__ uint32_t read_u4(const uint32_t* address, uint32_t index)
{
    return (*address >> (index * 4u)) & 0xfu;
}

template<int... Ds>
__global__ void permute_u4(uint* dst, const uint* src, Array<int, sizeof...(Ds)> dims)
{
    constexpr int N = sizeof...(Ds);

    size_t count = 1;
    PRAGMA_UNROLL
    for (int i = 0; i < N; ++i) {
        count *= dims[i];
    }

    constexpr int order[] = {Ds...};

    for (int i = threadIdx.x + blockDim.x * blockIdx.x; i < count; i += blockDim.x * gridDim.x) {

        int indices[N]{};

        PRAGMA_UNROLL
        for (int j = N - 1, ii = i; j >= 0; --j) {
            indices[j] = ii % dims[j];
            ii /= dims[j];
        }

        auto data = read_u4(src + i / 8, i % 8);

        int index = 0;

        PRAGMA_UNROLL
        for (int j = N - 1, stride = 1; j >= 0; --j) {
            index += indices[order[j]] * stride;
            stride *= dims[order[j]];
        }

        atomic_assign_u4(dst + index / 8, index % 8, data);
    }
}

}  // namespace

// col-major interleaved
void unpack_awq_gemm(uint4_t* dst, const uint4_t* src, int rows, int cols, hipStream_t st)
{
    Array<int, 4> shape{cols, rows / 8, 2, 4};
    permute_u4<0, 1, 3, 2><<<512, 512, 0, st>>>((uint*)dst, (const uint*)src, shape);
}

__global__ void transpose_u4_kernel(uint4_t* dst, const uint4_t* src, int s, int c)
{
    const int idx_c = 8 * (threadIdx.x + blockIdx.x * blockDim.x);
    const int idx_s = 8 * (threadIdx.y + blockIdx.y * blockDim.y);
    if (idx_c >= c || idx_s >= s) {
        return;
    }
    uint32_t ivec[8];
    PRAGMA_UNROLL
    for (int i = 0; i < 8; ++i) {
        ivec[i] = ((const uint32_t*)src)[((idx_s + i) * c + idx_c) / 8];
    }
    uint32_t ovec[8]{};
    PRAGMA_UNROLL
    for (int i = 0; i < 8; ++i) {
        PRAGMA_UNROLL
        for (int j = 0; j < 8; ++j) {
            ovec[i] |= (((ivec[j] >> (i * 4)) & 0xfu) << (j * 4));
        }
    }
    PRAGMA_UNROLL
    for (int i = 0; i < 8; ++i) {
        ((uint32_t*)dst)[((idx_c + i) * s + idx_s) / 8] = ovec[i];
    }
}

void transpose_u4(uint4_t* dst, const uint4_t* src, int s, int c, hipStream_t st)
{
    if (s % 8 || c % 8) {
        std::cerr << "transpose_u4: invalid shape (" << s << "," << c << "), must be multiple of 8" << std::endl;
        return;
    }
    // Array<int, 2> shape{s, c};
    // permute_u4<1, 0><<<512, 512, 0, st>>>((uint*)dst, (const uint*)src, shape);

    const dim3 block(16, 16);
    const dim3 grid((c + 15) / 16, (s + 15) / 16);
    transpose_u4_kernel<<<grid, block, 0, st>>>(dst, src, s, c);
}

// load -> unpack -> extend_to_u8 -> manipulation -> compat_to_u4 -> store
// load -> extend_to_u16 -> convert -> run

}  // namespace turbomind
