#include "hip/hip_runtime.h"
// Copyright (c) OpenMMLab. All rights reserved.

#include "common.h"
#include <iostream>

namespace turbomind {

__device__ void atomic_assign_u4(uint32_t* address, uint32_t index, uint32_t value)
{
    uint32_t old = *address;
    uint32_t assumed;
    do {
        assumed      = old;
        uint32_t tmp = (assumed & ~(0xfu << (index * 4u))) | (value << (index * 4u));
        old          = atomicCAS(address, assumed, tmp);
    } while (assumed != old);
}

__device__ uint32_t read_u4(const uint32_t* address, uint32_t index)
{
    return (*address >> (index * 4u)) & 0xfu;
}

template<int... Ds>
__global__ void permute_u4(uint* dst, const uint* src, Array<int, sizeof...(Ds)> dims)
{
    constexpr int N = sizeof...(Ds);

    size_t count = 1;
    PRAGMA_UNROLL
    for (int i = 0; i < N; ++i) {
        count *= dims[i];
    }

    constexpr int order[] = {Ds...};

    for (int i = threadIdx.x + blockDim.x * blockIdx.x; i < count; i += blockDim.x * gridDim.x) {

        int indices[N]{};

        PRAGMA_UNROLL
        for (int j = N - 1, ii = i; j >= 0; --j) {
            indices[j] = ii % dims[j];
            ii /= dims[j];
        }

        auto data = read_u4(src + i / 8, i % 8);

        int index = 0;

        PRAGMA_UNROLL
        for (int j = N - 1, stride = 1; j >= 0; --j) {
            index += indices[order[j]] * stride;
            stride *= dims[order[j]];
        }

        atomic_assign_u4(dst + index / 8, index % 8, data);
    }
}

void reformat_s4_k8_m(uint32_t* dst, const uint32_t* src, int m, int k, hipStream_t st)
{
    // permutation for [k/8, m] layout
    Array<int, 10> shape{k / 32, 2, 2, m / 32, 2, 2, 8, 2, 2, 2};
    //        |warp|  lane  | 2x2 |  a0-7  |
    permute_u4<0, 3, 6, 8, 9, 1, 4, 7, 2, 5><<<512, 512, 0, st>>>(dst, src, shape);
}

void reformat_s4_k_m8(uint32_t* dst, const uint32_t* src, int m, int k, hipStream_t st)
{
    // permutation for [k, m/8] layout
    Array<int, 10> shape{k / 32, 2, 2, 4, 2, m / 32, 2, 2, 2, 4};
    //        |warp|  lane  | 2x2 |  a0-7  |
    permute_u4<0, 5, 9, 8, 3, 1, 6, 4, 2, 7><<<512, 512, 0, st>>>(dst, src, shape);
}

__global__ void dequantize_s4_offset_64(uint4* dst, const uint32_t* src, size_t count)
{
    for (int i = threadIdx.x + blockDim.x * blockIdx.x; i < count; i += blockDim.x * gridDim.x) {
        dst[i] = dequantize_s4_to_fp16x2_v2(src[i]);
    }
}

__global__ void merge_Q(half2* Q, const half* scales, const half* zeros, int count)
{
    for (int i = threadIdx.x + blockDim.x * blockIdx.x; i < count; i += blockDim.x * gridDim.x) {
        if (TURBOMIND_S4_DEQUANT_USE_FMA) {
            // dequant via HFMA2 has numerical statbility issue
            Q[i] = __halves2half2(-zeros[i] * scales[i], scales[i]);
        }
        else {
            Q[i] = __halves2half2(zeros[i], scales[i]);
        }
    }
}

void convert_s4_k_m8(uint32_t*       A_dst,
                     half2*          Q_dst,
                     half*           workspace,
                     const uint32_t* A_src,
                     const half*     scales,
                     const uint32_t* qzeros,
                     int             m,
                     int             k,
                     int             group_size,
                     hipStream_t    st)
{
    dequantize_s4_offset_64<<<256, 256, 0, st>>>((uint4*)workspace, qzeros, k / group_size * m / 8);

    merge_Q<<<256, 256, 0, st>>>(Q_dst, scales, workspace, k / group_size * m);

    reformat_s4_k_m8(A_dst, A_src, m, k, st);
}

void transpose_qk_s4_k_m8_hf(uint32_t* dst, const uint32_t* src, int m, int k, int size_per_head, hipStream_t st)
{
    Array<int, 7> shape{k, m / size_per_head, 2, size_per_head / 2 / 8, 2, 2, 2};
    //      dequant   transpose    quant
    // 0123456 -> 0123564 -> 0135642 -> 0135264
    permute_u4<0, 1, 3, 5, 2, 6, 4><<<512, 512, 0, st>>>(dst, src, shape);
}

// [2, k, m/8] -> [k, m/8, 2]
void fuse_w1_w3_s4_k_m8(uint32_t* dst, const uint32_t* src, int m, int k, hipStream_t st)
{
    Array<int, 6> shape{2, k, m / 8, 2, 2, 2};
    //     dequant   transpose   quant
    // 012345 -> 012453 -> 124530 -> 124053
    permute_u4<1, 2, 4, 0, 5, 3><<<512, 512, 0, st>>>(dst, src, shape);
}

__global__ void dequantize_s4_kernel(uint4* dst, const uint* src, size_t count)
{
    for (int i = threadIdx.x + blockDim.x * blockIdx.x; i < count; i += blockDim.x * gridDim.x) {
        dst[i] = dequantize_s4_to_fp16x2(src[i]);
    }
}

void dequantize_s4(uint4* dst, const uint32_t* src, size_t count, hipStream_t st)
{
    dequantize_s4_kernel<<<512, 512>>>(dst, src, count);
}

}  // namespace turbomind
