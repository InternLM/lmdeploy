// Copyright (c) OpenMMLab. All rights reserved.

#include "gemm_s4_f16.h"
#include "gemm_s4_f16_kernel.h"
#include "metric.h"
#include <algorithm>
#include <iomanip>
#include <ios>
#include <iostream>
#include <limits>
#include <numeric>
#include <stdexcept>
#include <vector>

namespace turbomind {

bool g_dump_kernel_info_once = false;

struct GemmS4F16::Impl {

    template<int GS>
    void Generate()
    {
        std::vector<std::unique_ptr<IGemmKernel>> k;

        // k.emplace_back(new GemmKernel<Shape<256, 128, 32>, Shape<32, 128, 32>, 5, GS>{});
        // k.emplace_back(new GemmKernel<Shape<256, 128, 32>, Shape<32, 128, 32>, 4, GS>{});
        // k.emplace_back(new GemmKernel<Shape<256, 128, 32>, Shape<32, 128, 32>, 3, GS>{});
        // k.emplace_back(new GemmKernel<Shape<256, 128, 32>, Shape<32, 128, 32>, 2, GS>{});

        k.emplace_back(new GemmKernel<Shape<128, 128, 64>, Shape<32, 128, 32>, 4, GS>{});
        k.emplace_back(new GemmKernel<Shape<128, 128, 64>, Shape<32, 128, 32>, 3, GS>{});
        // k.emplace_back(new GemmKernel<Shape<128, 128, 64>, Shape<32, 128, 32>, 2, GS>{});
        k.emplace_back(new GemmKernel<Shape<128, 96, 32>, Shape<32, 96, 32>, 5, GS>{});
        k.emplace_back(new GemmKernel<Shape<128, 96, 32>, Shape<32, 96, 32>, 4, GS>{});
        k.emplace_back(new GemmKernel<Shape<128, 96, 32>, Shape<32, 96, 32>, 3, GS>{});
        // k.emplace_back(new GemmKernel<Shape<128, 96, 32>, Shape<32, 96, 32>, 2, GS>{});
        k.emplace_back(new GemmKernel<Shape<128, 64, 32>, Shape<32, 64, 32>, 5, GS>{});
        k.emplace_back(new GemmKernel<Shape<128, 64, 32>, Shape<32, 64, 32>, 4, GS>{});
        k.emplace_back(new GemmKernel<Shape<128, 64, 32>, Shape<32, 64, 32>, 3, GS>{});
        // k.emplace_back(new GemmKernel<Shape<128, 64, 32>, Shape<32, 64, 32>, 2, GS>{});
        k.emplace_back(new GemmKernel<Shape<128, 64, 64>, Shape<32, 64, 32>, 5, GS>{});
        k.emplace_back(new GemmKernel<Shape<128, 64, 64>, Shape<32, 64, 32>, 4, GS>{});
        k.emplace_back(new GemmKernel<Shape<128, 64, 64>, Shape<32, 64, 32>, 3, GS>{});
        // k.emplace_back(new GemmKernel<Shape<128, 64, 64>, Shape<32, 64, 32>, 2, GS>{});
        k.emplace_back(new GemmKernel<Shape<128, 32, 128>, Shape<32, 32, 64>, 4, GS>{});
        k.emplace_back(new GemmKernel<Shape<128, 32, 128>, Shape<32, 32, 64>, 3, GS>{});
        k.emplace_back(new GemmKernel<Shape<128, 32, 128>, Shape<32, 32, 64>, 2, GS>{});
        k.emplace_back(new GemmKernel<Shape<128, 16, 256>, Shape<32, 16, 64>, 3, GS>{});
        k.emplace_back(new GemmKernel<Shape<128, 16, 256>, Shape<32, 16, 64>, 2, GS>{});
        k.emplace_back(new GemmKernel<Shape<128, 8, 512>, Shape<32, 8, 128>, 2, GS>{});

        k.emplace_back(new GemmKernel<Shape<64, 128, 128>, Shape<32, 128, 32>, 3, GS>{});
        k.emplace_back(new GemmKernel<Shape<64, 128, 128>, Shape<32, 128, 32>, 2, GS>{});
        k.emplace_back(new GemmKernel<Shape<64, 96, 128>, Shape<32, 96, 32>, 4, GS>{});
        k.emplace_back(new GemmKernel<Shape<64, 96, 128>, Shape<32, 96, 32>, 3, GS>{});
        // k.emplace_back(new GemmKernel<Shape<64, 96, 128>, Shape<32, 96, 32>, 2, GS>{});
        k.emplace_back(new GemmKernel<Shape<64, 64, 128>, Shape<32, 64, 32>, 4, GS>{});
        k.emplace_back(new GemmKernel<Shape<64, 64, 128>, Shape<32, 64, 32>, 3, GS>{});
        // k.emplace_back(new GemmKernel<Shape<64, 64, 128>, Shape<32, 64, 32>, 2, GS>{});
        k.emplace_back(new GemmKernel<Shape<64, 32, 128>, Shape<32, 32, 32>, 4, GS>{});
        k.emplace_back(new GemmKernel<Shape<64, 32, 128>, Shape<32, 32, 32>, 3, GS>{});
        // k.emplace_back(new GemmKernel<Shape<64, 32, 128>, Shape<32, 32, 32>, 2, GS>{});
        k.emplace_back(new GemmKernel<Shape<64, 16, 256>, Shape<32, 16, 32>, 3, GS>{});
        // k.emplace_back(new GemmKernel<Shape<64, 16, 256>, Shape<32, 16, 32>, 2, GS>{});
        k.emplace_back(new GemmKernel<Shape<64, 8, 512>, Shape<32, 8, 128>, 2, GS>{});
        // k.emplace_back(new GemmKernel<Shape<64, 8, 256>, Shape<32, 8, 32>, 3, GS>{});
        k.emplace_back(new GemmKernel<Shape<64, 8, 256>, Shape<32, 8, 32>, 2, GS>{});

        kernels_.push_back(std::move(k));
        group_sizes_.push_back(GS);
    }

    void Measure(half*                C,
                 const uint*          A,
                 const half*          B,
                 const half2*         Q,
                 int                  m,
                 int                  n,
                 int                  k,
                 int                  group_size,
                 std::vector<Metric>& metrics,
                 hipStream_t         st)
    {
        int gid = -1;
        for (size_t i = 0; i < group_sizes_.size(); ++i) {
            if (group_sizes_[i] == group_size) {
                gid = i;
                break;
            }
        }
        if (gid < 0) {
            throw std::runtime_error("unsupported group size");
        }
        const auto& kernels = kernels_[gid];
        metrics             = std::vector<Metric>(kernels.size());

        int best = 0;

        for (size_t i = 0; i < kernels.size(); ++i) {
            metrics[i].id = i;
            kernels[i]->GetMetric(&metrics[i], m, n, k);
            if (!metrics[i].feasible) {
                metrics[i].time  = std::numeric_limits<float>::infinity();
                metrics[i].count = 1;
                continue;
            }
            if (Compare(metrics[i], metrics[best])) {
                best = i;
            }
            for (size_t j = 0; j < kWarmup + kMeasure; ++j) {
                if (j == kWarmup) {
                    hipEventRecord(ev_start_, st);
                }
                kernels[i]->Launch(C, A, B, Q, m, n, k, st);
            }
            hipEventRecord(ev_end_, st);
            hipEventSynchronize(ev_end_);
            float ms{};
            hipEventElapsedTime(&ms, ev_start_, ev_end_);
            metrics[i].time  = ms;
            metrics[i].count = kMeasure;
        }

        metrics[best].best = 1;

        // sort metrics
        std::vector<int> indices(kernels.size());
        std::iota(indices.begin(), indices.end(), 0);
        std::stable_sort(
            indices.begin(), indices.end(), [&](int i, int j) { return metrics[i].time < metrics[j].time; });

        if (g_dump_kernel_info_once) {
            DumpMetrics(std::cerr, metrics, indices);
            g_dump_kernel_info_once = 0;
        }

        std::vector<Metric> tmp;
        for (size_t i = 0; i < indices.size(); ++i) {
            tmp.push_back(metrics[indices[i]]);
        }
        metrics.swap(tmp);
    }

    static bool Compare(const Metric& a, const Metric& b)
    {
        if (a.feasible != b.feasible) {
            return a.feasible > b.feasible;
        }
        if (a.normalized != b.normalized) {
            return a.normalized < b.normalized;
        }
        if (a.cost != b.cost) {
            return a.cost < b.cost;
        }
        if (a.nice != b.nice) {
            return a.nice > b.nice;
        }
        return a.occupancy > b.occupancy;
    }

    int Estimate(int gid, int m, int n, int k)
    {
        const auto& kernels = kernels_.at(gid);

        int                 best = 0;
        std::vector<Metric> metrics(kernels.size());
        for (size_t i = 0; i < kernels.size(); ++i) {
            metrics[i].id = i;
            kernels[i]->GetMetric(&metrics[i], m, n, k);
            if (Compare(metrics[i], metrics[best])) {
                best = i;
            }
        }

        if (g_dump_kernel_info_once) {
            std::vector<int> indices(kernels.size());
            std::iota(indices.begin(), indices.end(), 0);
            std::stable_sort(
                indices.begin(), indices.end(), [&](int i, int j) { return Compare(metrics[i], metrics[j]); });
            DumpMetrics(std::cerr, metrics, indices);
            g_dump_kernel_info_once = 0;
        }

        return best;
    }

    void Run(half*        C,
             const uint*  A,
             const half*  B,
             const half2* Q,
             int          m,
             int          n,
             int          k,
             int          group_size,
             int          algo_id,
             hipStream_t st)
    {
        for (size_t i = 0; i < group_sizes_.size(); ++i) {
            if (group_sizes_[i] == group_size) {
                if (algo_id < 0) {
                    algo_id = Estimate(i, m, n, k);
                }
                if (algo_id < 0) {
                    throw std::runtime_error("no feasible kernel found");
                }
                kernels_[i].at(algo_id)->Launch(C, A, B, Q, m, n, k, st);
                return;
            }
        }
        throw std::runtime_error("unsupported group size");
    }

    Impl()
    {
        hipEventCreate(&ev_start_);
        hipEventCreate(&ev_end_);

        /// TODO: add more group sizes
        Generate<128>();
    }

    ~Impl()
    {
        hipEventDestroy(ev_end_);
        hipEventDestroy(ev_start_);
    }

    std::vector<std::vector<std::unique_ptr<IGemmKernel>>> kernels_;

    std::vector<int> group_sizes_;

    static constexpr int kWarmup  = 10;
    static constexpr int kMeasure = 100;

    hipEvent_t ev_start_{};
    hipEvent_t ev_end_{};
};

GemmS4F16::GemmS4F16(): impl_(std::make_unique<Impl>()) {}

GemmS4F16::~GemmS4F16() = default;

void GemmS4F16::Measure(half*                C,
                        const uint*          A,
                        const half*          B,
                        const half2*         Q,
                        int                  m,
                        int                  n,
                        int                  k,
                        int                  group_size,
                        std::vector<Metric>& metrics,
                        hipStream_t         st)
{
    impl_->Measure(C, A, B, Q, m, n, k, group_size, metrics, st);
}

void GemmS4F16::Run(half*        C,
                    const uint*  A,
                    const half*  B,
                    const half2* Q,
                    int          m,
                    int          n,
                    int          k,
                    int          group_size,
                    int          algo_id,
                    hipStream_t st)
{
    impl_->Run(C, A, B, Q, m, n, k, group_size, algo_id, st);
}

}  // namespace turbomind