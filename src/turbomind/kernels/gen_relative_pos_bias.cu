#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2020-2023, NVIDIA CORPORATION.  All rights reserved.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "hipblas.h"
#include "gen_relative_pos_bias.h"
#include "reduce_kernel_utils.cuh"
#include "src/fastertransformer/kernels/activation_kernels.h"
#include "src/fastertransformer/utils/cuda_utils.h"
#include <cstdio>

namespace fastertransformer {

/*******************  invokeGenRelativePosBias  ***********************/
// relative_position_bias_table is [(2*window_size-1)*(2*window_size-1), headNum]
// relative_position_bias is [head_num, window_size^2, window_size^2]
// grid(window_size*window_size, head_num)
// block(window_size*window_size)

template<typename T, typename Tindex>
__global__ void gen_relative_pos_bias(T*            relative_position_bias,
                                      const T*      relative_position_bias_table,
                                      const Tindex* relative_position_bias_index,
                                      const int     window_size,
                                      const int     head_num)
{
    const int    h_in_window           = blockIdx.x / window_size;
    const int    w_in_window           = blockIdx.x % window_size;
    const int    h_in_token            = threadIdx.x / window_size;
    const int    w_in_token            = threadIdx.x % window_size;
    const int    head_idx              = blockIdx.y;
    const int    elements_per_window   = window_size * window_size;
    const size_t elements_per_window_2 = elements_per_window * elements_per_window;
    const size_t output_idx = head_idx * elements_per_window_2 + blockIdx.x * elements_per_window + threadIdx.x;
    if (output_idx < head_num * elements_per_window_2) {
        const Tindex idx_in_table =
            relative_position_bias_index[(h_in_window * window_size + w_in_window) * elements_per_window
                                         + h_in_token * window_size + w_in_token];
        relative_position_bias[output_idx] = relative_position_bias_table[idx_in_table * head_num + head_idx];
    }
}

template<typename T, typename Tindex>
void invokeGenRelativePosBias(T*            relative_position_bias,
                              const T*      relative_position_bias_table,
                              const Tindex* relative_position_bias_index,
                              const int     window_size,
                              const int     head_num,
                              hipStream_t  stream)
{
    dim3 grid(window_size * window_size, head_num);
    dim3 block(window_size * window_size);

    if (block.x > 1024) {
        printf("[ERROR][invokeGenRelativePosBias] window_size*window_size > 1024.\n");
        exit(-1);
    }

    gen_relative_pos_bias<<<grid, block, 0, stream>>>(
        relative_position_bias, relative_position_bias_table, relative_position_bias_index, window_size, head_num);
}

/*******************  invokeGenRelativePosBiasV2  ***********************/
template<typename T, typename Tindex>
void invokeGenRelativePosBiasV2(T*            relative_position_bias,
                                const T*      relative_coords_table,
                                const Tindex* relative_position_bias_index,
                                const T*      cpb_mlp_weight1,
                                const T*      cpb_mlp_bias1,
                                const T*      cpb_mlp_weight2,
                                const int     window_size,
                                const int     cpb_mlp_in_dim,
                                const int     cpb_mlp_out_dim,
                                const int     head_num,
                                hipStream_t  stream)
{

    dim3 grid(window_size * window_size, head_num);
    dim3 block(window_size * window_size);

    if (block.x > 1024) {
        printf("[ERROR][invokeGenRelativePosBias] window_size*window_size > 1024.\n");
        exit(-1);
    }

    T* relative_position_bias_table;
    check_cuda_error(hipMalloc(&relative_position_bias_table,
                                ((2 * window_size - 1) * (2 * window_size - 1) * head_num) * sizeof(T)));
    T* cpb_mlp_1;
    check_cuda_error(
        hipMalloc(&cpb_mlp_1, ((2 * window_size - 1) * (2 * window_size - 1) * cpb_mlp_out_dim) * sizeof(T)));
    hipblasHandle_t cublas_handle;
    check_cuda_error(hipblasCreate(&cublas_handle));

    int            m     = (2 * window_size - 1) * (2 * window_size - 1);
    T              alpha = (T)1.0f;
    T              beta  = (T)0.0f;
    hipDataType type  = std::is_same<float, T>::value ? HIP_R_32F : HIP_R_16F;
#if (CUDART_VERSION >= 11000)
    hipblasComputeType_t compute_type = std::is_same<float, T>::value ? HIPBLAS_COMPUTE_32F : HIPBLAS_COMPUTE_16F;
#else
    hipDataType compute_type = std::is_same<float, T>::value ? HIP_R_32F : HIP_R_16F;
#endif
    hipblasGemmAlgo_t algo = std::is_same<float, T>::value ? HIPBLAS_GEMM_DEFAULT : CUBLAS_GEMM_DEFAULT_TENSOR_OP;
    check_cuda_error(hipblasGemmEx(cublas_handle,
                                  HIPBLAS_OP_T,
                                  HIPBLAS_OP_N,
                                  cpb_mlp_out_dim,
                                  m,
                                  cpb_mlp_in_dim,
                                  &alpha,
                                  cpb_mlp_weight1,
                                  type,
                                  cpb_mlp_in_dim,
                                  relative_coords_table,
                                  type,
                                  cpb_mlp_in_dim,
                                  &beta,
                                  cpb_mlp_1,
                                  type,
                                  cpb_mlp_out_dim,
                                  compute_type,
                                  algo));

    invokeGenericActivation<ReluActivation, T, T>(
        cpb_mlp_1, cpb_mlp_bias1, nullptr, nullptr, nullptr, nullptr, m, cpb_mlp_out_dim, 0, nullptr, nullptr, stream);

    check_cuda_error(hipblasGemmEx(cublas_handle,
                                  HIPBLAS_OP_T,
                                  HIPBLAS_OP_N,
                                  head_num,
                                  m,
                                  cpb_mlp_out_dim,
                                  &alpha,
                                  cpb_mlp_weight2,
                                  type,
                                  cpb_mlp_out_dim,
                                  cpb_mlp_1,
                                  type,
                                  cpb_mlp_out_dim,
                                  &beta,
                                  relative_position_bias_table,
                                  type,
                                  head_num,
                                  compute_type,
                                  algo));

    gen_relative_pos_bias<<<grid, block, 0, stream>>>(
        relative_position_bias, relative_position_bias_table, relative_position_bias_index, window_size, head_num);

    invokeSigmoid(
        relative_position_bias, window_size * window_size * window_size * window_size * head_num, 16.0f, stream);
    check_cuda_error(hipFree(relative_position_bias_table));
    check_cuda_error(hipFree(cpb_mlp_1));
    check_cuda_error(hipblasDestroy(cublas_handle));
}

/*******************  instantiation  ***********************/

template void invokeGenRelativePosBias(float*       relative_position_bias,
                                       const float* relative_position_bias_table,
                                       const int*   relative_position_bias_index,
                                       const int    window_size,
                                       const int    head_num,
                                       hipStream_t stream);

template void invokeGenRelativePosBias(half*        relative_position_bias,
                                       const half*  relative_position_bias_table,
                                       const int*   relative_position_bias_index,
                                       const int    window_size,
                                       const int    head_num,
                                       hipStream_t stream);

template void invokeGenRelativePosBias(float*         relative_position_bias,
                                       const float*   relative_position_bias_table,
                                       const int64_t* relative_position_bias_index,
                                       const int      window_size,
                                       const int      head_num,
                                       hipStream_t   stream);

template void invokeGenRelativePosBias(half*          relative_position_bias,
                                       const half*    relative_position_bias_table,
                                       const int64_t* relative_position_bias_index,
                                       const int      window_size,
                                       const int      head_num,
                                       hipStream_t   stream);

__host__ __device__ uint32_t pow2_rounddown(uint32_t x)
{
    x |= x >> 1;
    x |= x >> 2;
    x |= x >> 4;
    x |= x >> 8;
    x |= x >> 16;
    x >>= 1;
    return x + 1;
}

template<typename T>
__global__ void generate_alibi_slopes(T* alibi_slopes, const size_t num_heads)
{
    if (threadIdx.x < num_heads) {
        // The nearest power of 2 greater than num_heads followed by HF's implementation.
        int num_heads_pow2 = pow2_rounddown(num_heads);
        // Loop over the attention head.
        for (int h = threadIdx.x; h < num_heads; h += blockDim.x) {
            if (h < num_heads_pow2) {
                alibi_slopes[h] = static_cast<T>(powf(powf(0.5f, powf(0.5f, log2f(num_heads_pow2) - 3.f)), h + 1));
            }
            else {
                alibi_slopes[h] = static_cast<T>(
                    powf(powf(0.5f, powf(0.5f, log2f(num_heads_pow2 << 1) - 3.f)), (h - num_heads_pow2) * 2 + 1));
            }
        }
    }
}

template<typename T>
void invokeBuildAlibiSlopes(T* alibi_slopes, const size_t num_heads, hipStream_t stream)
{
    // Generate the slopes of a linear attention linear bias.
    //
    // Paper: https://arxiv.org/abs/2108.12409
    // HF's implementation
    //   https://github.com/huggingface/transformers/blob/56ef0ba44765162f830873c140bd40bdc975cc34/src/transformers/models/bloom/modeling_bloom.py#L86
    // Author's implementation
    //   https://github.com/ofirpress/attention_with_linear_biases/blob/02aa87e7a29e9340efd28d6d169018eafb3aa57a/fairseq/models/transformer.py#L760
    //
    // alibi_slopes: [num_heads],
    //     strictly follows how HF implements. which treats power-of-2 heads, and non-power-of-2 heads differently.
    //     what paper generates differs with HF's when number of heads is not a power of 2.
    // num_heads: the number of attention heads.
    // stream: a cuda stream.

    dim3 block(min((int)num_heads, 512));
    generate_alibi_slopes<<<1, block, 0, stream>>>(alibi_slopes, num_heads);
}

template void invokeBuildAlibiSlopes(float* alibi_slopes, const size_t num_heads, hipStream_t stream);
template void invokeBuildAlibiSlopes(half* alibi_slopes, const size_t num_heads, hipStream_t stream);
#ifdef ENABLE_BF16
template void invokeBuildAlibiSlopes(__hip_bfloat16* alibi_slopes, const size_t num_heads, hipStream_t stream);
#endif

template void invokeGenRelativePosBiasV2(float*       relative_position_bias,
                                         const float* relative_coords_table,
                                         const int*   relative_position_bias_index,
                                         const float* cpb_mlp_weight1,
                                         const float* cpb_mlp_bias1,
                                         const float* cpb_mlp_weight2,
                                         const int    window_size,
                                         const int    cpb_mlp_in_dim,
                                         const int    cpb_mlp_out_dim,
                                         const int    head_num,
                                         hipStream_t stream);

template void invokeGenRelativePosBiasV2(half*        relative_position_bias,
                                         const half*  relative_coords_table,
                                         const int*   relative_position_bias_index,
                                         const half*  cpb_mlp_weight1,
                                         const half*  cpb_mlp_bias1,
                                         const half*  cpb_mlp_weight2,
                                         const int    window_size,
                                         const int    cpb_mlp_in_dim,
                                         const int    cpb_mlp_out_dim,
                                         const int    head_num,
                                         hipStream_t stream);

template void invokeGenRelativePosBiasV2(float*         relative_position_bias,
                                         const float*   relative_coords_table,
                                         const int64_t* relative_position_bias_index,
                                         const float*   cpb_mlp_weight1,
                                         const float*   cpb_mlp_bias1,
                                         const float*   cpb_mlp_weight2,
                                         const int      window_size,
                                         const int      cpb_mlp_in_dim,
                                         const int      cpb_mlp_out_dim,
                                         const int      head_num,
                                         hipStream_t   stream);

template void invokeGenRelativePosBiasV2(half*          relative_position_bias,
                                         const half*    relative_coords_table,
                                         const int64_t* relative_position_bias_index,
                                         const half*    cpb_mlp_weight1,
                                         const half*    cpb_mlp_bias1,
                                         const half*    cpb_mlp_weight2,
                                         const int      window_size,
                                         const int      cpb_mlp_in_dim,
                                         const int      cpb_mlp_out_dim,
                                         const int      head_num,
                                         hipStream_t   stream);
}  // namespace fastertransformer
