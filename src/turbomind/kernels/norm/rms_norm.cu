#include "hip/hip_runtime.h"
// Copyright (c) OpenMMLab. All rights reserved.

#include "cub/block/block_reduce.cuh"

#include "src/turbomind/kernels/core/array_ops.h"
#include "src/turbomind/kernels/core/common.h"

namespace turbomind {

template<class T, class Accum, int block_dim, int vec_size>
__global__ void RMSNormKernel(T*       dst,
                              int      dst_ld,
                              const T* src,
                              int      src_ld,
                              const T* __restrict__ weights,
                              int   dims,
                              int   num,
                              float eps,
                              float inv_dims)
{
    const int ti = blockIdx.x;
    const int di = threadIdx.x * vec_size;

    if (ti >= num) {
        return;
    }

    src += src_ld * ti;

    Array<Accum, vec_size> accum{};
    Array<T, vec_size>     vec;

    for (int i = di; i < dims; i += block_dim * vec_size) {
        Load(vec, &src[i]);
        Array<Accum, vec_size> tmp = cast<Accum>(vec);
        using namespace ops;
        accum = accum + tmp * tmp;
    }

    float sum{};
    PRAGMA_UNROLL
    for (int i = 0; i < vec_size; ++i) {
        sum += accum[i];
    }

    using BlockReduce = hipcub::BlockReduce<Accum, block_dim>;
    __shared__ typename BlockReduce::TempStorage temp_storage;

    sum = BlockReduce{temp_storage}.Sum(sum);

    __shared__ float shared_sum;

    if (threadIdx.x == 0) {
        shared_sum = rsqrtf(sum * inv_dims + eps);
    }

    __syncthreads();

    sum = shared_sum;

    dst += dst_ld * ti;

    Array<T, vec_size> sv;
    for (int i = di; i < dims; i += block_dim * vec_size) {
        Load(vec, &src[i]);
        Ldg(sv, &weights[i]);
        PRAGMA_UNROLL
        for (int c = 0; c < vec_size; ++c) {
            vec[c] = (T)((float)vec[c] * sum) * sv[c];
            // vec[c] = (T)((float)vec[c] * sum * (float)sv[c]);
        }
        Store(&dst[i], vec);
    }
}

template<class T>
void invokeRMSNorm(
    T* dst, int dst_ld, const T* src, int src_ld, const T* weights, int dims, int num, float eps, hipStream_t st)
{
    constexpr int vec_size = 16 / sizeof(T);

    constexpr int threads = 512;
    const int     blocks  = num;

    RMSNormKernel<T, float, threads, vec_size><<<blocks, threads, 0, st>>>(dst,  //
                                                                           dst_ld,
                                                                           src,
                                                                           src_ld,
                                                                           weights,
                                                                           dims,
                                                                           num,
                                                                           eps,
                                                                           1.f / dims);
}

template void invokeRMSNorm(half*        dst,
                            int          dst_ld,
                            const half*  src,
                            int          src_ld,
                            const half*  weights,
                            int          dims,
                            int          num,
                            float        eps,
                            hipStream_t st);
#if ENABLE_BF16
template void invokeRMSNorm(hip_bfloat16*       dst,
                            int                dst_ld,
                            const hip_bfloat16* src,
                            int                src_ld,
                            const hip_bfloat16* weights,
                            int                dims,
                            int                num,
                            float              eps,
                            hipStream_t       st);
#endif

// r' <- r + (h + b)
// h' <- norm(r') * w
template<class T, class Tacc, int block_dim, int vec_size>
__global__ void BiasResidualRMSNormKernel(T* __restrict__ residual,
                                          T* __restrict__ hidden_states,
                                          const T* __restrict__ weights,
                                          const T* __restrict__ bias,
                                          int   dims,
                                          int   num,
                                          float eps,
                                          float inv_dims)
{
    const int ti = blockIdx.x;
    const int di = threadIdx.x * vec_size;

    if (ti >= num) {
        return;
    }

    residual += dims * ti;
    hidden_states += dims * ti;

    Array<Tacc, vec_size> accum{};

    Array<T, vec_size> r_vec;
    Array<T, vec_size> h_vec;
    Array<T, vec_size> b_vec;

    for (int i = di; i < dims; i += block_dim * vec_size) {
        Load(r_vec, &residual[i]);
        Load(h_vec, &hidden_states[i]);

        using namespace ops;
        r_vec = r_vec + h_vec;

        if (bias) {
            Ldg(b_vec, &bias[i]);
            r_vec = r_vec + b_vec;
        }

        Store(&residual[i], r_vec);

        Array<Tacc, vec_size> tmp = cast<Tacc>(r_vec);

        accum = accum + tmp * tmp;
    }

    float sum{};
    PRAGMA_UNROLL
    for (int i = 0; i < vec_size; ++i) {
        sum += accum[i];
    }

    using BlockReduce = hipcub::BlockReduce<Tacc, block_dim>;
    __shared__ typename BlockReduce::TempStorage temp_storage;

    sum = BlockReduce{temp_storage}.Sum(sum);

    __shared__ float shared_sum;

    if (threadIdx.x == 0) {
        shared_sum = rsqrtf(sum * inv_dims + eps);
    }

    __syncthreads();

    sum = shared_sum;

    Array<T, vec_size> w_vec;
    for (int i = di; i < dims; i += block_dim * vec_size) {
        Load(r_vec, &residual[i]);
        Ldg(w_vec, &weights[i]);
        PRAGMA_UNROLL
        for (int c = 0; c < vec_size; ++c) {
            r_vec[c] = (T)((float)r_vec[c] * sum) * w_vec[c];
        }
        Store(&hidden_states[i], r_vec);
    }
}

template<class T>
void invokeBiasResidualRMSNorm(
    T* residual, T* hidden_states, const T* weights, const T* bias, int dims, int num, float eps, hipStream_t st)
{
    constexpr int vec_size = 16 / sizeof(T);
    constexpr int threads  = 512;
    const int     blocks   = num;

    BiasResidualRMSNormKernel<T, float, threads, vec_size><<<blocks, threads, 0, st>>>(residual,  //
                                                                                       hidden_states,
                                                                                       weights,
                                                                                       bias,
                                                                                       dims,
                                                                                       num,
                                                                                       eps,
                                                                                       1.f / dims);
}

template void invokeBiasResidualRMSNorm(half*        residual,
                                        half*        hidden_states,
                                        const half*  weights,
                                        const half*  bias,
                                        int          dims,
                                        int          num,
                                        float        eps,
                                        hipStream_t st);

#if ENABLE_BF16
template void invokeBiasResidualRMSNorm(hip_bfloat16*       residual,
                                        hip_bfloat16*       hidden_states,
                                        const hip_bfloat16* weights,
                                        const hip_bfloat16* bias,
                                        int                dims,
                                        int                num,
                                        float              eps,
                                        hipStream_t       st);
#endif

}  // namespace turbomind
