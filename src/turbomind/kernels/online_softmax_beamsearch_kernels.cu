#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2020-2023, NVIDIA CORPORATION.  All rights reserved.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#ifndef CUDART_VERSION
#error CUDART_VERSION Undefined!
#elif (CUDART_VERSION >= 11050)
#include <hipcub/hipcub.hpp>
#else
#include "3rdparty/hipcub/hipcub.hpp"
#endif

#include "src/fastertransformer/kernels/online_softmax_beamsearch_kernels.h"
#include "src/fastertransformer/kernels/reduce_kernel_utils.cuh"
#include "src/fastertransformer/utils/cuda_utils.h"

namespace fastertransformer {

#define DO_SPLIT_SMALL_TOP_K_SOFTMAX
static const int SMALL_TOP_K_SOFTMAX_THREADBLOCK_SIZE = 256;

#define TOPK_FP16_STORAGE 0

template<typename T>
__device__ __forceinline__ T apply_length_penalty(T log_prob, int length, float length_penalty)
{
    // score = log(prob) / (length)^length_penalty.
    if (length_penalty == 0.0f || length == 1) {
        return log_prob;
    }
    return log_prob / static_cast<T>(powf(length, length_penalty));
}

template<typename T, int MAX_K, int THREADBLOCK_SIZE>
__launch_bounds__(THREADBLOCK_SIZE) __global__
    void batch_topK_kernel(int* topk_tmp_id_buf, T* topk_tmp_val_buf, int* id_buf)
{
    int            thread_id = threadIdx.x;
    int            block_id  = blockIdx.x;
    TopK<T, MAX_K> partial;
    if (thread_id == 0) {
        for (int i = 0; i < MAX_K; ++i) {
            partial.p[i] = -1;
            partial.u[i] = -FLT_MAX;
        }

        int index = block_id * MAX_K * MAX_K;
        for (int i = 0; i < MAX_K * MAX_K; i++) {
            partial.insert((T)topk_tmp_val_buf[index + i], topk_tmp_id_buf[index + i]);
        }

        index = block_id * MAX_K;
        for (int i = 0; i < MAX_K; i++) {
            id_buf[index + i] = partial.p[i];
        }
    }
}

template<typename T, int MAX_K, int THREADBLOCK_SIZE>
__launch_bounds__(THREADBLOCK_SIZE) __global__ void batch_topK_kernel(const int* __restrict topk_tmp_id_buf,
                                                                      const T* __restrict topk_tmp_val_buf,
                                                                      int* __restrict id_buf,
                                                                      T* __restrict val_buf)
{
    int            thread_id = threadIdx.x;
    int            block_id  = blockIdx.x;
    TopK<T, MAX_K> partial;
    if (thread_id == 0) {
        for (int i = 0; i < MAX_K; ++i) {
            partial.p[i] = -1;
            partial.u[i] = -FLT_MAX;
        }

        int index = block_id * MAX_K * MAX_K;
        for (int i = 0; i < MAX_K * MAX_K; i++) {
            partial.insert((T)topk_tmp_val_buf[index + i], topk_tmp_id_buf[index + i]);
        }

        index = block_id * MAX_K;
        for (int i = 0; i < MAX_K; i++) {
            id_buf[index + i]  = partial.p[i];
            val_buf[index + i] = partial.u[i];
        }
    }
}

template<typename T, int MAX_K, int THREADBLOCK_SIZE>
__launch_bounds__(THREADBLOCK_SIZE) __global__ void batch_topk_kernel(const int* __restrict x,
                                                                      const T* __restrict y,
                                                                      int* __restrict z,
                                                                      float* __restrict v,
                                                                      float*         output_log_probs,
                                                                      const bool*    finished,
                                                                      const int*     sequence_lengths,
                                                                      BeamHypotheses beam_hyps,
                                                                      const int      V,
                                                                      const int      K,
                                                                      const int      vocab_size,
                                                                      const float    length_penalty,
                                                                      const T        diversity_rate)
{
    int thread_id = threadIdx.x;
    int vector_id = blockIdx.x;

    // reposition x, y to data for the current vector
    x += vector_id * V;
    y += vector_id * V;

    typedef hipcub::BlockReduce<TopK<T, MAX_K>, THREADBLOCK_SIZE> BlockReduce;

    __shared__ typename BlockReduce::TempStorage temp_storage;
    __shared__ int                               selected_beams;
    __shared__ float                             old_cum_log_probs[MAX_K];

    if (thread_id == 0) {
        selected_beams = 0;
    }
    if (thread_id < K) {
        old_cum_log_probs[thread_id] = v[vector_id * K + thread_id];
    }
    __syncthreads();
    if (beam_hyps.num_beams != nullptr) {
        const int global_batch_idx = beam_hyps.ite * beam_hyps.local_batch_size + vector_id;
        if (beam_hyps.num_beams[global_batch_idx] == 0 && thread_id == 0) {
            beam_hyps.min_normed_scores[global_batch_idx] = FLT_MAX;
        }
        else if (beam_hyps.num_beams[global_batch_idx] == K) {
            return;
        }
    }

    TopK<T, MAX_K> partial;
    for (int i = 0; i < MAX_K; ++i) {
        partial.p[i] = -1;
        partial.u[i] = -FLT_MAX;
    }

    for (int elem_id = thread_id; elem_id < V; elem_id += THREADBLOCK_SIZE) {
        int i    = elem_id % K;
        T   elem = length_penalty == 0.0f ? y[elem_id] :
                                            apply_length_penalty(y[elem_id],
                                                               finished[vector_id] ? sequence_lengths[vector_id] :
                                                                                       sequence_lengths[vector_id] + 1,
                                                               length_penalty);
        elem += diversity_rate * (T)i;
        int elem_idx = elem_id;  // x[elem_id];
        partial.insert(elem, elem_idx);
    }

    TopK<T, MAX_K> total = BlockReduce(temp_storage).Reduce(partial, reduce_topk_op<T, MAX_K>);

    if (thread_id == 0) {
        z += vector_id * K;
        v += vector_id * K;

        for (int i = 0; i < MAX_K; ++i) {
            if (beam_hyps.num_beams != nullptr && x[total.p[i]] % vocab_size == beam_hyps.end_ids[vector_id]) {
                // if beam_token does not belong to top num_beams tokens, it should not be added. Refer from
                // https://github.com/huggingface/transformers/blob/v4.24.0/src/transformers/generation_beam_search.py#L257
                if (i >= K) {
                    // do nothing
                }
                else {
                    const int   global_batch_idx = beam_hyps.ite * beam_hyps.local_batch_size + vector_id;
                    const float normed_score     = (float)total.u[i];
                    const int   num_beam         = beam_hyps.num_beams[global_batch_idx];
                    int         beam_idx         = num_beam;
                    // If there are beam_width finished sentences, check that the score of selected candidatet
                    // is higher than min_normed_score or not. If current score is better, replace worst one
                    // and update the min_normed_score.
                    if (num_beam == K) {
                        if (normed_score < beam_hyps.min_normed_scores[global_batch_idx]) {
                            // end the tracing and exist this for loop
                            selected_beams = K;
                            break;
                        }
                        else {
                            // find the beam index which's score = min_normed_score, erase it.
                            for (int j = 0; j < K; j++) {
                                if (beam_hyps.normed_scores[global_batch_idx * (K * 2) + j]
                                    == beam_hyps.min_normed_scores[global_batch_idx]) {
                                    beam_idx = j;
                                    beam_hyps.num_beams[global_batch_idx]--;

                                    beam_hyps.min_normed_scores[global_batch_idx]           = FLT_MAX;
                                    beam_hyps.normed_scores[global_batch_idx * (K * 2) + j] = normed_score;
                                    for (int l = 0; l < K; l++) {
                                        beam_hyps.min_normed_scores[global_batch_idx] =
                                            min(beam_hyps.min_normed_scores[global_batch_idx],
                                                beam_hyps.normed_scores[global_batch_idx * (K * 2) + l]);
                                    }
                                    break;
                                }
                            }
                        }
                    }
                    const int tgt_id_offset =
                        ((vector_id + beam_hyps.ite * beam_hyps.local_batch_size) * (K * 2) + beam_idx)
                        * (beam_hyps.max_seq_len);
                    beam_hyps.output_ids_tgt[tgt_id_offset + beam_hyps.step] = beam_hyps.end_ids[vector_id];
                    if (beam_hyps.log_probs != nullptr) {
                        beam_hyps.log_probs[tgt_id_offset + beam_hyps.step] =
                            (float)y[total.p[i]] - old_cum_log_probs[(x[total.p[i]] / vocab_size) % K];
                    }

                    int prev_id = (x[total.p[i]] / vocab_size) % K;
                    for (int j = beam_hyps.step - 1; j >= 0; j--) {
                        const int src_idx = j * beam_hyps.batch_size * K
                                            + beam_hyps.ite * beam_hyps.local_batch_size * K + vector_id * K + prev_id;

                        beam_hyps.output_ids_tgt[tgt_id_offset + j] = beam_hyps.output_ids_src[src_idx];
                        if (beam_hyps.log_probs != nullptr && beam_hyps.log_probs_src != nullptr) {
                            beam_hyps.log_probs[tgt_id_offset + j] = beam_hyps.log_probs_src[src_idx];
                        }
                        prev_id = beam_hyps.parent_ids_src[src_idx];
                    }
                    const int tgt_beam_idx                       = global_batch_idx * (K * 2) + beam_idx;
                    beam_hyps.sequence_lengths_tgt[tgt_beam_idx] = beam_hyps.step;
                    beam_hyps.normed_scores[tgt_beam_idx]        = normed_score;
                    beam_hyps.min_normed_scores[global_batch_idx] =
                        min(beam_hyps.min_normed_scores[global_batch_idx], beam_hyps.normed_scores[tgt_beam_idx]);

                    beam_hyps.num_beams[global_batch_idx]++;
                    beam_hyps.cum_log_probs[tgt_beam_idx] = (float)y[total.p[i]];
                }
            }
            else if ((beam_hyps.num_beams != nullptr && i < 2 * K) || (beam_hyps.num_beams == nullptr && i < K)) {
                z[selected_beams] = x[total.p[i]];
                if (output_log_probs != nullptr) {
                    output_log_probs[vector_id * K + selected_beams] =
                        (float)y[total.p[i]] - old_cum_log_probs[(z[selected_beams] / vocab_size) % K];
                }
                v[selected_beams] = (float)y[total.p[i]];
                selected_beams++;
            }
            __syncthreads();
            if (selected_beams >= K) {
                break;
            }
        }
    }
    if (threadIdx.x == 0 && beam_hyps.num_beams != nullptr) {
        if (beam_hyps.num_beams[blockIdx.x] < K) {
            beam_hyps.is_done[blockIdx.x] = false;
        }
        else if (beam_hyps.early_stopping) {
            beam_hyps.is_done[blockIdx.x] = true;
        }
    }
}

struct __align__(8) MD
{
    float m;
    float d;
};

__device__ __forceinline__ MD reduce_md_op(MD a, MD b)
{
    bool a_bigger  = (a.m > b.m);
    MD   bigger_m  = a_bigger ? a : b;
    MD   smaller_m = a_bigger ? b : a;
    MD   res;
    res.d = bigger_m.d + smaller_m.d * __expf(smaller_m.m - bigger_m.m);
    res.m = bigger_m.m;
    return res;
}

template<typename T, int MAX_K>
struct TopKMD {
    MD             md;
    TopK<T, MAX_K> topk;
};

template<typename T, int MAX_K>
__device__ __forceinline__ TopKMD<T, MAX_K> reduce_topk_md_op(const TopKMD<T, MAX_K>& a, const TopKMD<T, MAX_K>& b)
{
    TopKMD<T, MAX_K> res;
    res.md   = reduce_md_op(a.md, b.md);
    res.topk = reduce_topk_op(a.topk, b.topk);
    return res;
}

template<typename T, int ITEMS_PER_THREAD, int MAX_K, int THREADBLOCK_SIZE>
__launch_bounds__(THREADBLOCK_SIZE) __global__ void beam_online_softmax_topk_kernel(const T* __restrict x,
                                                                                    const T* __restrict b,
                                                                                    const float* __restrict c,
                                                                                    const bool* __restrict finished,
                                                                                    int* __restrict z,
                                                                                    T* __restrict v,
                                                                                    int V,
                                                                                    int K,
                                                                                    const int* __restrict end_ids)
{
    int thread_id = threadIdx.x;
    int vector_id = blockIdx.x;

    const bool IS_FP16   = std::is_same<T, half>::value;
    const T    MAX_T_VAL = (IS_FP16) ? HALF_FLT_MAX : FLT_MAX;

    // reposition y to data for the current vector
    x += vector_id * V;

    typedef hipcub::BlockReduce<TopKMD<float, MAX_K>, THREADBLOCK_SIZE> BlockReduce;
    __shared__ typename BlockReduce::TempStorage                     temp_storage;

    TopKMD<float, MAX_K> partial;
    bool                 finish = finished[vector_id];
    for (int i = 0; i < MAX_K; ++i) {
        partial.topk.p[i] = -1;
        partial.topk.u[i] = -MAX_T_VAL;
    }
    partial.md.m = -MAX_T_VAL;
    partial.md.d = 0.0F;

    if (finish) {
        for (int elem_id = thread_id; elem_id < V; elem_id += THREADBLOCK_SIZE) {
            float elem = (elem_id == end_ids[vector_id / K]) ? MAX_T_VAL : -MAX_T_VAL;
            MD    new_elem{elem, 1.0F};
            partial.md = reduce_md_op(partial.md, new_elem);
            partial.topk.insert(elem, elem_id);
            // if (elem_id > THREADBLOCK_SIZE * MAX_K && (elem_id == E)) break;
        }
    }
    else {
        for (int elem_id = thread_id; elem_id < V; elem_id += THREADBLOCK_SIZE) {
            float elem = x[elem_id] + b[elem_id];
            MD    new_elem{elem, 1.0F};
            partial.md = reduce_md_op(partial.md, new_elem);
            partial.topk.insert(elem, elem_id);
        }
    }

    TopKMD<float, MAX_K> total = BlockReduce(temp_storage).Reduce(partial, reduce_topk_md_op<float, MAX_K>);

    if (thread_id == 0) {
        z += vector_id * K;
        v += vector_id * K;
        c += vector_id;

        // float d_total_inverse = __fdividef(1.0F, total.md.d);
        float d_total_log = logf(total.md.d);
        for (int i = 0; i < MAX_K; ++i) {
            // float val = __expf(total.topk.u[i] - total.md.m) * d_total_inverse;
            float val = total.topk.u[i] - total.md.m - d_total_log;
            if (i < K) {
                z[i] = total.topk.p[i] + vector_id * V;  // faster transformer needs absolute id
                v[i] = val + c[0];
            }
        }
    }
}

template<typename T, int ITEMS_PER_THREAD, int MAX_K, int THREADBLOCK_SIZE>
__launch_bounds__(THREADBLOCK_SIZE, 1) __global__
    void beam_online_softmax_topk_stage1_kernel(const T* __restrict x,
                                                const T* __restrict b,
                                                const bool* __restrict finished,
                                                float* __restrict t,
                                                int V,
                                                int K,
                                                const int* __restrict end_ids)
{
    int thread_id = threadIdx.x;
    int vector_id = blockIdx.x;  // batch beam index.

    const int PACKED_TOP_KMD_SIZE = 2 * MAX_K + 2;

    const bool IS_FP16   = std::is_same<T, half>::value;
    const T    MAX_T_VAL = (IS_FP16) ? HALF_FLT_MAX : FLT_MAX;

    // one will have multiple sections per V
    const int v_local       = (V + gridDim.y - 1) / gridDim.y;
    const int section_start = v_local * blockIdx.y;
    int       section_end   = section_start + v_local;
    section_end             = (section_end > V) ? V : section_end;

    // reposition x to data for the current vector
    x += vector_id * V;
#if TOPK_FP16_STORAGE == 1
    typedef hipcub::BlockReduce<TopKMD<__half, MAX_K>, THREADBLOCK_SIZE> BlockReduce;
#else
    typedef hipcub::BlockReduce<TopKMD<T, MAX_K>, THREADBLOCK_SIZE> BlockReduce;
#endif
    __shared__ typename BlockReduce::TempStorage temp_storage;
    __shared__ float                             buf_s[PACKED_TOP_KMD_SIZE];  // save intermediate result

#if TOPK_FP16_STORAGE == 1
    TopKMD<__half, MAX_K> partial;
#else
    TopKMD<T, MAX_K>                                             partial;
#endif
    bool finish = finished[vector_id];
    for (int i = 0; i < MAX_K; ++i) {
        partial.topk.p[i] = -1;
        partial.topk.u[i] = -MAX_T_VAL;
    }
    partial.md.m = -MAX_T_VAL;
    partial.md.d = 0.0F;

    if (finish) {
#pragma unroll 1
        for (int elem_id = section_start + thread_id; elem_id < section_end; elem_id += THREADBLOCK_SIZE) {
            float elem = (elem_id == end_ids[vector_id / K]) ? MAX_T_VAL : -MAX_T_VAL;
            MD    new_elem{elem, 1.0F};
            partial.md = reduce_md_op(partial.md, new_elem);
            partial.topk.insert(elem, elem_id);
        }
    }
    else {
#pragma unroll 1
        for (int elem_id = section_start + thread_id; elem_id < section_end; elem_id += THREADBLOCK_SIZE) {
            T  bias = b == nullptr ? (T)0.0f : b[elem_id];  // gpt-2 does not use bias
            T  elem = x[elem_id] + bias;
            MD new_elem{elem, 1.0F};
            partial.md = reduce_md_op(partial.md, new_elem);
            partial.topk.insert(elem, elem_id);
        }
    }

#if TOPK_FP16_STORAGE == 1
    TopKMD<__half, MAX_K> total = BlockReduce(temp_storage).Reduce(partial, reduce_topk_md_op<__half, MAX_K>);
#else
    TopKMD<T, MAX_K> total = BlockReduce(temp_storage).Reduce(partial, reduce_topk_md_op<T, MAX_K>);
#endif

    if (thread_id == 0) {
        for (int i = 0; i < 2 * K; i++) {
            reinterpret_cast<int*>(buf_s)[i] = total.topk.p[i] + vector_id * V;  // faster transformer needs absolute id
            buf_s[MAX_K + i]                 = total.topk.u[i];
        }
        buf_s[2 * MAX_K]     = total.md.d;
        buf_s[2 * MAX_K + 1] = total.md.m;
    }
    __syncthreads();
    for (int elem_id = thread_id; elem_id < PACKED_TOP_KMD_SIZE; elem_id += THREADBLOCK_SIZE) {
        t[blockIdx.x * PACKED_TOP_KMD_SIZE * gridDim.y + blockIdx.y * PACKED_TOP_KMD_SIZE + elem_id] = buf_s[elem_id];
    }
}

template<typename T, int MAX_K, int THREADBLOCK_SIZE>
__launch_bounds__(THREADBLOCK_SIZE) __global__ void beam_online_softmax_topk_stage2_kernel(
    const float* __restrict x, const float* __restrict c, int* __restrict z, T* __restrict v, int K, int parts_per_beam)
{
    const int vector_id           = blockIdx.x;
    const int thread_id           = threadIdx.x;
    const int PACKED_TOP_KMD_SIZE = 2 * MAX_K + 2;

    const bool IS_FP16   = std::is_same<T, half>::value;
    const T    MAX_T_VAL = (IS_FP16) ? HALF_FLT_MAX : FLT_MAX;

    extern __shared__ char buf_s_[];  // intermediate result
    float*                 buf_s = reinterpret_cast<float*>(buf_s_);
    //__shared__ float buf_s[PACKED_TOP_KMD_SIZE * THREADBLOCK_SIZE]; // intermediate result

    typedef hipcub::BlockReduce<TopKMD<T, MAX_K>, THREADBLOCK_SIZE> BlockReduce;
    __shared__ typename BlockReduce::TempStorage                 temp_storage;

    x += vector_id * PACKED_TOP_KMD_SIZE * parts_per_beam;

    TopKMD<T, MAX_K> partial;
    for (int i = 0; i < MAX_K; ++i) {
        partial.topk.p[i] = -1;
        partial.topk.u[i] = -MAX_T_VAL;
    }
    partial.md.m = -MAX_T_VAL;
    partial.md.d = 0.0F;

    // load and unpack into registers through smem
    for (int idx = thread_id; idx < PACKED_TOP_KMD_SIZE * parts_per_beam; idx += THREADBLOCK_SIZE) {
        buf_s[idx] = x[idx];
    }
    __syncthreads();

    if (threadIdx.x < parts_per_beam) {
        float* b_s = buf_s + thread_id * PACKED_TOP_KMD_SIZE;
        for (int i = 0; i < 2 * K; i++) {
            partial.topk.p[i] = reinterpret_cast<int*>(b_s)[i];
            partial.topk.u[i] = b_s[MAX_K + i];
        }
        partial.md.d = b_s[2 * MAX_K];
        partial.md.m = b_s[2 * MAX_K + 1];
    }
    __syncthreads();

    TopKMD<T, MAX_K> total = BlockReduce(temp_storage).Reduce(partial, reduce_topk_md_op<T, MAX_K>);

    if (thread_id == 0) {
        z += vector_id * 2 * K;
        v += vector_id * 2 * K;
        c += vector_id;

        float d_total_log = logf(total.md.d);
        for (int i = 0; i < MAX_K; ++i) {
            float val = (float)total.topk.u[i] - total.md.m - d_total_log;
            if (i < 2 * K) {
                z[i] = total.topk.p[i];
                v[i] = (float)val + (float)c[0];
            }
        }
    }
}

template<typename T, int MAX_K>
void beam_online_softmax_topk_stage2_kernelLauncher(const float* temp_storage,
                                                    const float* cum_log_probs,
                                                    int*         ids,
                                                    T*           vals,
                                                    int          batch_size,
                                                    int          beam_width,
                                                    int          parts_per_beam,
                                                    hipStream_t stream)
{
    // might rewrite beam_online_softmax_topk_stage2_kernel no to depend on constant block size
    // in oreder to reduce compilation time
    int smem_stage2_size = parts_per_beam * (2 * MAX_K + 2) * sizeof(float);

    if (parts_per_beam <= 32) {
        beam_online_softmax_topk_stage2_kernel<T, MAX_K, 32><<<batch_size * beam_width, 32, smem_stage2_size, stream>>>(
            temp_storage, cum_log_probs, ids, vals, beam_width, parts_per_beam);
        return;
    }
    if (parts_per_beam <= 64) {
        beam_online_softmax_topk_stage2_kernel<T, MAX_K, 64><<<batch_size * beam_width, 64, smem_stage2_size, stream>>>(
            temp_storage, cum_log_probs, ids, vals, beam_width, parts_per_beam);
        return;
    }
    if (parts_per_beam <= 128) {
        beam_online_softmax_topk_stage2_kernel<T, MAX_K, 128>
            <<<batch_size * beam_width, 128, smem_stage2_size, stream>>>(
                temp_storage, cum_log_probs, ids, vals, beam_width, parts_per_beam);
        return;
    }
    assert(0);
}

template<typename T, int MAX_K>
void topK_softMax_kernelLauncher(const T*        log_probs,
                                 const T*        bias,
                                 const bool*     finished,
                                 const int*      sequence_lengths,
                                 float*          cum_log_probs,
                                 float*          output_log_probs,
                                 int*            ids,
                                 void*           temp_storage,
                                 const int       temp_storage_size,
                                 BeamHypotheses* beam_hyps,
                                 const int       batch_size,
                                 const int       beam_width,
                                 const int       vocab_size,
                                 const int*      end_ids,
                                 T               diversity_rate,
                                 const float     length_penalty,
                                 hipStream_t    stream)
{
    const int items_per_thread = 1;
    const int block_sz         = (MAX_K < 16) ? (MAX_K < 8) ? SMALL_TOP_K_SOFTMAX_THREADBLOCK_SIZE : 128 : 64;
    // const int block_sz = SMALL_TOP_K_SOFTMAX_THREADBLOCK_SIZE;

    assert(temp_storage_size % 2 == 0);
    assert(temp_storage_size >= 2 * batch_size * beam_width * beam_width * 2);
    // Beam search needs the sequence lengths of beams to apply length penalty.
    assert(length_penalty == 0.0f || sequence_lengths != nullptr);

    const int topk_buf_offset  = ceil(batch_size * beam_width * beam_width * 2 / 4.) * 4;
    int*      topk_tmp_id_buf  = reinterpret_cast<int*>(temp_storage);
    T*        topk_tmp_val_buf = reinterpret_cast<T*>(topk_tmp_id_buf + topk_buf_offset);
    float*    tmp_buffer       = reinterpret_cast<float*>(topk_tmp_val_buf + topk_buf_offset);

#ifdef DO_SPLIT_SMALL_TOP_K_SOFTMAX
    int voc_parts = 4;
    if (batch_size * beam_width < 256) {
        // Volta has 80 SMs, so we aim for three waves
        voc_parts = (240 + batch_size * beam_width - 1) / (batch_size * beam_width);
        voc_parts = std::min(128, voc_parts);  // we implement up to 128
    }
    dim3 grid(batch_size * beam_width, voc_parts);
    hipFuncSetAttribute(reinterpret_cast<const void*>(beam_online_softmax_topk_stage1_kernel<T), items_per_thread, 2 * MAX_K, block_sz>,
                         hipFuncAttributePreferredSharedMemoryCarveout,
                         cudaSharedmemCarveoutMaxL1);
    beam_online_softmax_topk_stage1_kernel<T, items_per_thread, 2 * MAX_K, block_sz>
        <<<grid, block_sz, 0, stream>>>(log_probs, bias, finished, tmp_buffer, vocab_size, beam_width, end_ids);
    sync_check_cuda_error();
#endif
    if (beam_width > 1) {
#ifdef DO_SPLIT_SMALL_TOP_K_SOFTMAX
        beam_online_softmax_topk_stage2_kernelLauncher<T, 2 * MAX_K>(
            tmp_buffer, cum_log_probs, topk_tmp_id_buf, topk_tmp_val_buf, batch_size, beam_width, voc_parts, stream);
        sync_check_cuda_error();
#else
        beam_online_softmax_topk_kernel<T, items_per_thread, MAX_K, block_sz>
            <<<batch_size * beam_width, block_sz, 0, stream>>>(log_probs,
                                                               bias,
                                                               cum_log_probs,
                                                               finished,
                                                               topk_tmp_id_buf,
                                                               topk_tmp_val_buf,
                                                               vocab_size,
                                                               beam_width,
                                                               end_ids);
#endif
#if 0
            // wrong result with diversity_rate != 0.f
            batch_topK_kernel<T, MAX_K, 32><<<batch_size, 32, 0, stream>>>
                                (topk_tmp_id_buf, topk_tmp_val_buf, ids, cum_log_probs);
#else
        // We need 2*MAX_K candidates because at most k candidates are finished, and we
        // will not put them into next iteration
        batch_topk_kernel<T, MAX_K * 2, 32><<<batch_size, 32, 0, stream>>>(topk_tmp_id_buf,
                                                                           topk_tmp_val_buf,
                                                                           ids,
                                                                           cum_log_probs,
                                                                           output_log_probs,
                                                                           finished,
                                                                           sequence_lengths,
                                                                           *beam_hyps,
                                                                           beam_width * beam_width * 2,
                                                                           beam_width,
                                                                           vocab_size,
                                                                           length_penalty,
                                                                           diversity_rate);
        sync_check_cuda_error();
#endif
    }
    else {
        FT_CHECK(false);
#ifdef DO_SPLIT_SMALL_TOP_K_SOFTMAX
        beam_online_softmax_topk_stage2_kernelLauncher<float, MAX_K>(
            tmp_buffer, cum_log_probs, ids, cum_log_probs, batch_size, beam_width, voc_parts, stream);
#else
        beam_online_softmax_topk_kernel<T, items_per_thread, MAX_K, block_sz>
            <<<batch_size * beam_width, block_sz, 0, stream>>>(
                log_probs, bias, cum_log_probs, finished, ids, cum_log_probs, vocab_size, beam_width, end_ids);
#endif
    }
}

#define CASE_K(K, MAX_K)                                                                                               \
    case K ... MAX_K:                                                                                                  \
        topK_softMax_kernelLauncher<T, MAX_K>(log_probs,                                                               \
                                              bias,                                                                    \
                                              finished,                                                                \
                                              sequence_lengths,                                                        \
                                              cum_log_probs,                                                           \
                                              output_log_probs,                                                        \
                                              ids,                                                                     \
                                              temp_storage,                                                            \
                                              temp_storage_size,                                                       \
                                              beam_hyps,                                                               \
                                              batch_size,                                                              \
                                              beam_width,                                                              \
                                              vocab_size,                                                              \
                                              end_ids,                                                                 \
                                              diversity_rate,                                                          \
                                              length_penalty,                                                          \
                                              stream);                                                                 \
        break;

template<typename T>
void invokeTopkSoftMax(const T*        log_probs,
                       const T*        bias,
                       const bool*     finished,
                       const int*      sequence_lengths,
                       float*          cum_log_probs,
                       float*          output_log_probs,
                       int*            ids,
                       void*           temp_storage,
                       const int       temp_storage_size,
                       BeamHypotheses* beam_hyps,
                       const int       batch_size,
                       const int       beam_width,
                       const int       vocab_size,
                       const int*      end_ids,
                       const float     diversity_rate,
                       const float     length_penalty,
                       hipStream_t    stream)
{
    switch (beam_width) {
        CASE_K(1, 4);
        CASE_K(5, 8);
        CASE_K(9, 16);
        CASE_K(17, 32);
        CASE_K(33, 64);
        default:
            throw std::runtime_error(fmtstr("Topk kernel of beam search does not support beam_width=%d", beam_width));
    }
}

#undef CASE_K

template void invokeTopkSoftMax<float>(const float*    log_probs,
                                       const float*    bias,
                                       const bool*     finished,
                                       const int*      sequence_lengths,
                                       float*          cum_log_probs,
                                       float*          output_log_probs,
                                       int*            ids,
                                       void*           tmp_storage,
                                       const int       temp_storage_size,
                                       BeamHypotheses* beam_hyps,
                                       const int       batch_size,
                                       const int       beam_width,
                                       const int       vocab_size,
                                       const int*      end_ids,
                                       const float     diversity_rate,
                                       const float     length_penalty,
                                       hipStream_t    stream);

template void invokeTopkSoftMax<half>(const half*     log_probs,
                                      const half*     bias,
                                      const bool*     finished,
                                      const int*      sequence_lengths,
                                      float*          cum_log_probs,
                                      float*          output_log_probs,
                                      int*            ids,
                                      void*           tmp_storage,
                                      const int       temp_storage_size,
                                      BeamHypotheses* beam_hyps,
                                      const int       batch_size,
                                      const int       beam_width,
                                      const int       vocab_size,
                                      const int*      end_ids,
                                      const float     diversity_rate,
                                      const float     length_penalty,
                                      hipStream_t    stream);

}  // end of namespace fastertransformer
