#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2019-2023, NVIDIA CORPORATION.  All rights reserved.
 * Copyright (c) 2021, NAVER Corp.  Authored by CLOVA.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "src/turbomind/kernels/attention/array_ops.h"
#include "src/turbomind/kernels/reduce_kernel_utils.cuh"
#include "src/turbomind/kernels/unfused_attention_kernels.h"
#include "src/turbomind/utils/cuda_type_utils.cuh"
#include "src/turbomind/utils/cuda_utils.h"
#include "src/turbomind/utils/logger.h"

namespace turbomind {

__inline__ __device__ int target_index(int id1, int id2, int id3, int id4, int dim_1, int dim_2, int dim_3, int dim_4)
{
    return id1 * (dim_2 * dim_3 * dim_4) + id3 * (dim_2 * dim_4) + id2 * dim_4 + id4;
}

template<typename T, typename T_IN, int ITEMS_PER_THREAD>
__global__ void softmax_kernel(T*          attn_score,
                               const T_IN* qk,
                               const T*    attn_mask,
                               const T*    linear_bias_slopes,
                               const int   batch_size,
                               const int   head_num,
                               const int   q_length,
                               const int   k_length,
                               const float qk_scale)
{
    // attn_score, [batch_size, num_heads, q_length, k_length]
    // qk, [batch_size, num_heads, q_length, k_length]
    // attn_mask, [batch_size, q_length, k_length]
    // linear_bias_slopes, [num_heads]

    const long bi = blockIdx.y;  // Batch index.
    const int  hi = blockIdx.z;  // Head index.

    __shared__ float s_mean, s_max;

    const float linear_bias_slope = linear_bias_slopes != nullptr ? (float)linear_bias_slopes[hi] : 0.0f;

    // Loop along with Q dimension.
    for (int qi = blockIdx.x; qi < q_length; qi += gridDim.x) {

        float data[ITEMS_PER_THREAD];
        long  qk_offset;
        float local_max = -1e20f;

        // Loop along with K dimension.
        for (int i = 0; blockDim.x * i + threadIdx.x < k_length; i++) {
            int ki    = blockDim.x * i + threadIdx.x;  // Index of K dimension.
            qk_offset = ((bi * head_num + hi) * q_length + qi) * k_length + ki;

            float qk_val  = static_cast<float>(qk[qk_offset]);
            float qk_bias = 0.0f;

            if (linear_bias_slopes != nullptr) {
                // We don't handle the upper diagonal (ki > qi) separately, whose values
                // are negligible due to the negative infinity mask. And it matches with
                // the HF's implementation.
                qk_bias += static_cast<float>(linear_bias_slope * (ki - qi));
            }

            long  mask_offset = (bi * q_length + qi) * k_length + ki;
            float mask_val    = static_cast<float>(ldg(&attn_mask[mask_offset]));
            qk_bias += (1.0f - mask_val) * -10000.0f;

            data[i]   = qk_scale * qk_val + qk_bias;
            local_max = fmax(local_max, data[i]);
        }

        float max_val = blockDim.x <= 32 ? warpReduceMax(local_max) : blockReduceMax<float>(local_max);
        if (threadIdx.x == 0) {
            s_max = max_val;
        }
        __syncthreads();

        float local_sum = 0;
        for (int i = 0; blockDim.x * i + threadIdx.x < k_length; i++) {
            data[i] = __expf(data[i] - s_max);
            local_sum += data[i];
        }

        float sum_val = blockDim.x <= 32 ? warpReduceSum(local_sum) : blockReduceSum<float>(local_sum);
        if (threadIdx.x == 0) {
            s_mean = sum_val + 1e-6f;
            s_mean = __fdividef(1.0f, s_mean);
        }
        __syncthreads();

        for (int i = 0; blockDim.x * i + threadIdx.x < k_length; i++) {
            qk_offset             = ((bi * head_num + hi) * q_length + qi) * k_length + blockDim.x * i + threadIdx.x;
            attn_score[qk_offset] = (T)(data[i] * s_mean);
        }
    }
}

template<typename T, int ITEMS_PER_THREAD>
__global__ void softmax_kernel_h2(T*        attn_score,
                                  const T*  qk_buf,
                                  const T*  attn_mask,
                                  const T*  linear_bias_slopes,
                                  const int batch_size,
                                  const int head_num,
                                  const int q_length,
                                  const int k_length,
                                  const T   qk_scale)
{
    // attn_score, [batch_size, num_heads, q_length, k_length]
    // qk, [batch_size, num_heads, q_length, k_length]
    // attn_mask, [batch_size, q_length, k_length]
    // linear_bias_slopes, [num_heads]

    using T2 = typename TypeConverter<T>::Type;

    T2*       attn_score_h2 = reinterpret_cast<T2*>(attn_score);
    const T2* qk_buf_h2     = reinterpret_cast<const T2*>(qk_buf);
    const T2* attn_mask_h2  = reinterpret_cast<const T2*>(attn_mask);

    const long bi = blockIdx.y;  // Batch index
    const int  hi = blockIdx.z;  // Head index.

    __shared__ float s_mean, s_max;

    // Constant values that will be used repeately in the q/k loop.
    const T2 ONE       = cuda_cast<T2>(1.0f);
    const T2 ZERO      = cuda_cast<T2>(0.0f);
    const T2 NEG_INFTY = cuda_cast<T2>(-10000.0f);

    // The normalization factor of QK.
    const T2 qk_scale_h2 = cuda_cast<T2>(qk_scale);
    // The slope of a linear position bias of the current attention head.
    const T2 linear_bias_slope = linear_bias_slopes != nullptr ? cuda_cast<T2>(linear_bias_slopes[hi]) : ZERO;

    // Loop over q dimension.
    for (int qi = blockIdx.x; qi < q_length; qi += gridDim.x) {
        T2    data[ITEMS_PER_THREAD];
        long  qk_offset;
        float local_max = -1e20f;

        // Loop over k dimension.
        for (int i = 0; blockDim.x * i + threadIdx.x < (k_length / 2) && i < ITEMS_PER_THREAD; i++) {
            // The half of the index of k dimension. We will use the elements at {2 * ki, 2 * ki + 1}.
            int ki           = blockDim.x * i + threadIdx.x;
            qk_offset        = ((bi * head_num + hi) * q_length + qi) * (k_length / 2) + ki;
            long mask_offset = (bi * q_length + qi) * (k_length / 2) + ki;

            // The value of QK^T matrix at (qi, ki).
            T2 qk = qk_buf_h2[qk_offset];
            // The bias value to the position (qi, ki) including both mask and positional bias.
            T2 qk_bias = ZERO;

            if (linear_bias_slopes != nullptr) {
                // The position bias depends on the distance between qi/ki and is zero if qi >= 2*ki
                // or qi >= 2*ki+1. For T2 vectorization, we should handle every two elements along
                // with k-dim simultaneously. To do this, we check qi / 2 > ki at ones instead of
                // qi >= 2*ki or 2*ki+1. It works because an diagonal element for an odd qi will be
                // zero due to slope * (qi - 2*ki+1) = 0. Thus, we don't handle the upper diagonal
                // separately, whose values are negligible due to the negative infinity mask.
                T2 dist(2.0f * ki - qi, 2.0f * ki + 1 - qi);
                qk_bias = hadd2<T2>(qk_bias, hmul2<T2>(linear_bias_slope, dist));
            }

            T2 mask_val = ldg(&attn_mask_h2[mask_offset]);
            qk_bias     = hadd2<T2>(qk_bias, hmul2<T2>(hsub2<T2>(ONE, mask_val), NEG_INFTY));

            data[i]   = hadd2<T2>(hmul2<T2>(qk, qk_scale_h2), qk_bias);
            local_max = fmax(local_max, fmax((float)data[i].x, (float)data[i].y));
        }

        float max_val = blockDim.x <= 32 ? warpReduceMax(local_max) : blockReduceMax<float>(local_max);
        if (threadIdx.x == 0) {
            s_max = max_val;
        }
        __syncthreads();

        float local_sum = 0.0f;
        for (int i = 0; blockDim.x * i + threadIdx.x < (k_length / 2) && i < ITEMS_PER_THREAD; i++) {
            data[i] = hexp2<T2>(hsub2<T2>(data[i], cuda_cast<T2>(s_max)));
            local_sum += (float)(data[i].x + data[i].y);
        }

        float sum_val = blockDim.x <= 32 ? warpReduceSum(local_sum) : blockReduceSum<float>(local_sum);

        if (threadIdx.x == 0) {
            s_mean = sum_val + 1e-6f;
            s_mean = __fdividef(1.0f, s_mean);
        }
        __syncthreads();

        for (int i = 0; blockDim.x * i + threadIdx.x < (k_length / 2) && i < ITEMS_PER_THREAD; i++) {
            qk_offset = ((bi * head_num + hi) * q_length + qi) * (k_length / 2) + blockDim.x * i + threadIdx.x;
            attn_score_h2[qk_offset] = hmul2<T2>(data[i], cuda_cast<T2>(s_mean));
        }
    }
}

template<typename T, int K_ITEMS_PER_THREAD, int Q_ITEMS_PER_THREAD>
__global__ void softmax_kernel_h2_v2(T*        attn_score,
                                     const T*  qk_buf,
                                     const T*  attn_mask,
                                     const T*  linear_bias_slopes,
                                     const int batch_size,
                                     const int head_num,
                                     const int q_length,
                                     const int k_length,
                                     const T   scalar)
{
    // attn_score, [batch_size, num_heads, q_length, k_length]
    // qk, [batch_size, num_heads, q_length, k_length]
    // attn_mask, [batch_size, q_length, k_length]
    // linear_bias_slopes, [num_heads]

    using T2 = typename TypeConverter<T>::Type;

    // QK^T matrix of shape (batch_size, head_num, q_length, k_length / 2)
    T2*       attn_score_h2 = reinterpret_cast<T2*>(attn_score);
    const T2* qk_buf_h2     = reinterpret_cast<const T2*>(qk_buf);
    const T2* attn_mask_h2  = reinterpret_cast<const T2*>(attn_mask);

    const long bi = blockIdx.y;  // Batch index
    const int  hi = blockIdx.z;  // Head index.

    // Constant values that will be used repeately in the q/k loop.
    const T2 ONE       = cuda_cast<T2>(1.0f);
    const T2 ZERO      = cuda_cast<T2>(0.0f);
    const T2 NEG_INFTY = cuda_cast<T2>(-10000.0f);

    // The normalization factor of QK.
    const T2 qk_scale = cuda_cast<T2>(scalar);
    // The slope of a linear position bias of the current attention head.
    const T2 linear_bias_slope = linear_bias_slopes != nullptr ? cuda_cast<T2>(linear_bias_slopes[hi]) : ZERO;

    __shared__ float s_sum[Q_ITEMS_PER_THREAD], s_max[Q_ITEMS_PER_THREAD];

    // Loop over q dimension.
    for (int qi = blockIdx.x; qi < q_length; qi += gridDim.x * Q_ITEMS_PER_THREAD) {
        T2 data[Q_ITEMS_PER_THREAD][K_ITEMS_PER_THREAD];

        long qk_offset[Q_ITEMS_PER_THREAD];

        float local_max[Q_ITEMS_PER_THREAD];
#pragma unroll
        for (int j = 0; j < Q_ITEMS_PER_THREAD; j++) {
            local_max[j] = -1e20f;
        }

        // Loop over k dimension.
        const int Q_ITEMS = min((q_length - qi + gridDim.x - 1) / gridDim.x, Q_ITEMS_PER_THREAD);
        for (int i = 0; blockDim.x * i + threadIdx.x < k_length / 2 && i < K_ITEMS_PER_THREAD; ++i) {
            // The half of the index of k dimension. We will use the elements at {2 * ki, 2 * ki + 1}.
            int ki = blockDim.x * i + threadIdx.x;

            long mask_offset[Q_ITEMS_PER_THREAD];
#pragma unroll
            for (int j = 0; j < Q_ITEMS; j++) {
                qk_offset[j]   = ((bi * head_num + hi) * q_length + qi + j * gridDim.x) * (k_length / 2) + ki;
                mask_offset[j] = (bi * q_length + qi + j * gridDim.x) * (k_length / 2) + ki;
            }

            T2 mask_val[Q_ITEMS_PER_THREAD];
#pragma unroll
            for (int j = 0; j < Q_ITEMS; j++) {
                mask_val[j] = ldg(&attn_mask_h2[mask_offset[j]]);
            }

            T2 qk[Q_ITEMS_PER_THREAD];
#pragma unroll
            for (int j = 0; j < Q_ITEMS; j++) {
                qk[j] = qk_buf_h2[qk_offset[j]];
            }

            T2 pos_bias[Q_ITEMS_PER_THREAD];
            if (linear_bias_slopes != nullptr) {
#pragma unroll
                for (int j = 0; j < Q_ITEMS; j++) {
                    // The position bias depends on the distance between qi/ki and is zero if qi >= 2*ki
                    // or qi >= 2*ki+1. For T2 vectorization, we should handle every two elements along
                    // with k-dim simultaneously. To do this, we check qi / 2 > ki at ones instead of
                    // qi >= 2*ki or 2*ki+1. It works because an diagonal element for an odd qi will be
                    // zero due to slope * (qi - 2*ki+1) = 0. Thus, we don't handle the upper diagonal
                    // separately, whose values are negligible due to the negative infinity mask.
                    int qidx = qi + j * gridDim.x;
                    T2  dist(2.0f * ki - qidx, 2.0f * ki + 1 - qidx);
                    pos_bias[j] = hmul2<T2>(linear_bias_slope, dist);
                }
            }
#pragma unroll
            for (int j = 0; j < Q_ITEMS; j++) {
                mask_val[j] = hmul2<T2>(hsub2<T2>(ONE, mask_val[j]), NEG_INFTY);
            }

#pragma unroll
            for (int j = 0; j < Q_ITEMS; j++) {
                T2 val = hadd2<T2>(hmul2<T2>(qk_scale, qk[j]), mask_val[j]);
                if (linear_bias_slopes != nullptr) {
                    val = hadd2<T2>(val, pos_bias[j]);
                }
                data[j][i]   = val;
                local_max[j] = fmax(local_max[j], fmax((float)data[j][i].x, (float)data[j][i].y));
            }
        }

        if (blockDim.x <= 32) {
            warpReduceMaxV2<float, Q_ITEMS_PER_THREAD>(local_max);
        }
        else {
            blockReduceMaxV2<float, Q_ITEMS_PER_THREAD>(local_max);
        }

        if (threadIdx.x == 0) {
#pragma unroll
            for (int j = 0; j < Q_ITEMS_PER_THREAD; j++) {
                s_max[j] = local_max[j];
            }
        }
        __syncthreads();

        float local_sum[Q_ITEMS_PER_THREAD];
#pragma unroll
        for (int j = 0; j < Q_ITEMS_PER_THREAD; j++) {
            local_sum[j] = {0.f};
        }

        for (int i = 0; blockDim.x * i + threadIdx.x < k_length / 2 && i < K_ITEMS_PER_THREAD; ++i) {
#pragma unroll
            for (int j = 0; j < Q_ITEMS; ++j) {
                data[j][i] = hexp2<T2>(hsub2<T2>(data[j][i], cuda_cast<T2>(s_max[j])));
            }

#pragma unroll
            for (int j = 0; j < Q_ITEMS; j++) {
                local_sum[j] += (float)(data[j][i].x + data[j][i].y);
            }
        }

        if (blockDim.x <= 32) {
            warpReduceSumV2<float, Q_ITEMS_PER_THREAD>(local_sum);
        }
        else {
            blockReduceSumV2<float, Q_ITEMS_PER_THREAD>(local_sum);
        }

        if (threadIdx.x == 0) {
#pragma unroll
            for (int j = 0; j < Q_ITEMS_PER_THREAD; j++) {
                s_sum[j] = __fdividef(1.0f, local_sum[j] + 1e-6f);
            }
        }
        __syncthreads();

        for (int i = 0; blockDim.x * i + threadIdx.x < k_length / 2 && i < K_ITEMS_PER_THREAD; ++i) {
#pragma unroll
            for (int j = 0; j < Q_ITEMS; j++) {
                qk_offset[j] = ((bi * head_num + hi) * q_length + qi + j * gridDim.x) * (k_length / 2) + blockDim.x * i
                               + threadIdx.x;
            }

#pragma unroll
            for (int j = 0; j < Q_ITEMS; j++) {
                attn_score_h2[qk_offset[j]] = hmul2<T2>(data[j][i], cuda_cast<T2>(s_sum[j]));
            }
        }
    }
}

#define LAUNCH_MAKSED_SOFTMAX_(T_, ITEMS_PER_THREAD)                                                                   \
    block.x /= ITEMS_PER_THREAD;                                                                                       \
    block.x = (block.x + 31) / 32 * 32;                                                                                \
    assert(block.x <= 1024);                                                                                           \
    if (is_half2) {                                                                                                    \
        if (grid.x % 4 == 0) {                                                                                         \
            grid.x /= 4;                                                                                               \
            softmax_kernel_h2_v2<T_, ITEMS_PER_THREAD, 4>                                                              \
                <<<grid, block, 0, stream>>>((T_*)param.attention_score,                                               \
                                             (const T_*)param.qk,                                                      \
                                             (const T_*)param.attention_mask,                                          \
                                             (const T_*)param.linear_bias_slopes,                                      \
                                             param.batch_size,                                                         \
                                             param.num_heads,                                                          \
                                             param.q_length,                                                           \
                                             param.k_length,                                                           \
                                             (const T_)param.qk_scale);                                                \
        }                                                                                                              \
        else {                                                                                                         \
            softmax_kernel_h2<T_, ITEMS_PER_THREAD><<<grid, block, 0, stream>>>((T_*)param.attention_score,            \
                                                                                (const T_*)param.qk,                   \
                                                                                (const T_*)param.attention_mask,       \
                                                                                (const T_*)param.linear_bias_slopes,   \
                                                                                param.batch_size,                      \
                                                                                param.num_heads,                       \
                                                                                param.q_length,                        \
                                                                                param.k_length,                        \
                                                                                (const T_)param.qk_scale);             \
        }                                                                                                              \
    }                                                                                                                  \
    else {                                                                                                             \
        softmax_kernel<T, T_IN, ITEMS_PER_THREAD><<<grid, block, 0, stream>>>(param.attention_score,                   \
                                                                              param.qk,                                \
                                                                              param.attention_mask,                    \
                                                                              param.linear_bias_slopes,                \
                                                                              param.batch_size,                        \
                                                                              param.num_heads,                         \
                                                                              param.q_length,                          \
                                                                              param.k_length,                          \
                                                                              param.qk_scale);                         \
    }

#define LAUNCH_MAKSED_SOFTMAX(ITEMS_PER_THREAD) LAUNCH_MAKSED_SOFTMAX_(half, ITEMS_PER_THREAD)

template<typename T, typename T_IN>
void invokeMaskedSoftmax(MaskedSoftmaxParam<T, T_IN>& param, hipStream_t stream)
{
    // attention_score,    (batch_size, head_num, q_length, k_length), softmax output.
    // qk,                 (batch_size, head_num, q_length, k_length), QK^T.
    // attention_mask,     (batch_size, q_length, k_length), attention mask.
    // linear_bias_slopes, (head_num,) the slopes of the linear position bias.

    dim3 grid(param.q_length, param.batch_size, param.num_heads);
    if (param.batch_size * param.num_heads > 360) {
        grid.x = ceil(float(param.q_length) / 32.0f);
    }

    bool is_half2 = sizeof(T) == 2 && sizeof(T_IN) == 2 && param.k_length % 2 == 0;
    dim3 block((param.k_length / (is_half2 ? 2 : 1) + 31) / 32 * 32);

    if (block.x > 4096 && block.x <= 8192) {
        LAUNCH_MAKSED_SOFTMAX(8);
    }
    else if (block.x > 2048 && block.x <= 4096) {
        LAUNCH_MAKSED_SOFTMAX(4)
    }
    else if (block.x > 1024) {
        LAUNCH_MAKSED_SOFTMAX(2)
    }
    else if (block.x > 0) {
        LAUNCH_MAKSED_SOFTMAX(1)
    }
    else {
        FT_CHECK(param.k_length <= 8192);
    }
}

#if ENABLE_FP32
template void invokeMaskedSoftmax(MaskedSoftmaxParam<float, float>& param, hipStream_t stream);
#endif
template void invokeMaskedSoftmax(MaskedSoftmaxParam<half, float>& param, hipStream_t stream);
template void invokeMaskedSoftmax(MaskedSoftmaxParam<half, half>& param, hipStream_t stream);

#ifdef ENABLE_BF16
template<>
void invokeMaskedSoftmax(MaskedSoftmaxParam<__hip_bfloat16, float>& param, hipStream_t stream)
{
    // attention_score,    (batch_size, head_num, q_length, k_length), softmax output.
    // qk,                 (batch_size, head_num, q_length, k_length), QK^T.
    // attention_mask,     (batch_size, q_length, k_length), attention mask.
    // linear_bias_slopes, (head_num,) the slopes of the linear position bias.

    using T    = __hip_bfloat16;
    using T_IN = float;

    dim3 grid(param.q_length, param.batch_size, param.num_heads);
    if (param.batch_size * param.num_heads > 360) {
        grid.x = ceil(float(param.q_length) / 32.0f);
    }

    bool is_half2 = sizeof(T) == 2 && sizeof(T_IN) == 2 && param.k_length % 2 == 0;
    dim3 block((param.k_length / (is_half2 ? 2 : 1) + 31) / 32 * 32);

    if (block.x > 2048 && block.x <= 4096) {
        LAUNCH_MAKSED_SOFTMAX_(__hip_bfloat16, 4);
    }
    else if (block.x > 1024) {
        LAUNCH_MAKSED_SOFTMAX_(__hip_bfloat16, 2);
    }
    else if (block.x > 0) {
        LAUNCH_MAKSED_SOFTMAX_(__hip_bfloat16, 1);
    }
    else {
        FT_CHECK(param.k_length <= 4096);
    }
}
template<>
void invokeMaskedSoftmax(MaskedSoftmaxParam<__hip_bfloat16, __hip_bfloat16>& param, hipStream_t stream)
{
    // attention_score,    (batch_size, head_num, q_length, k_length), softmax output.
    // qk,                 (batch_size, head_num, q_length, k_length), QK^T.
    // attention_mask,     (batch_size, q_length, k_length), attention mask.
    // linear_bias_slopes, (head_num,) the slopes of the linear position bias.

    using T    = __hip_bfloat16;
    using T_IN = __hip_bfloat16;

    dim3 grid(param.q_length, param.batch_size, param.num_heads);
    if (param.batch_size * param.num_heads > 360) {
        grid.x = ceil(float(param.q_length) / 32.0f);
    }

    bool is_half2 = sizeof(T) == 2 && sizeof(T_IN) == 2 && param.k_length % 2 == 0;
    dim3 block((param.k_length / (is_half2 ? 2 : 1) + 31) / 32 * 32);

    if (block.x > 2048 && block.x <= 4096) {
        LAUNCH_MAKSED_SOFTMAX_(__hip_bfloat16, 4);
    }
    else if (block.x > 1024) {
        LAUNCH_MAKSED_SOFTMAX_(__hip_bfloat16, 2);
    }
    else if (block.x > 0) {
        LAUNCH_MAKSED_SOFTMAX_(__hip_bfloat16, 1);
    }
    else {
        FT_CHECK(param.k_length <= 4096);
    }
}

#endif

#undef LAUNCH_MAKSED_SOFTMAX
#undef LAUNCH_MAKSED_SOFTMAX_

template<typename T>
__global__ void transpose_remove_padding(const T*     src,
                                         T*           dst,
                                         const int    batch_size,
                                         const int    seq_len,
                                         const int    head_num,
                                         const int    size_per_head,
                                         const int*   mask_offset,
                                         const float* scale,
                                         const int    int8_mode)
{
    // TODO: optimize this kernel?
    // do remove_sequence_length_padding
    const int bid = blockIdx.x;  // batch * seq_len or valid_word_num

    const int token_offset = mask_offset ? mask_offset[bid] : 0;

    const int src_batch_id = (bid + token_offset) / seq_len;
    const int src_seq_id   = (bid + token_offset) % seq_len;

    const int dst_seq_id = bid;

    const int src_offset_base = src_batch_id * seq_len * head_num * size_per_head + src_seq_id * size_per_head;
    const int dst_offset_base = dst_seq_id * head_num * size_per_head;

    using Int8_Packed_T  = typename packed_as<int8_t, num_elems<T>::value>::type;
    using Float_Packed_T = typename packed_as<float, num_elems<T>::value>::type;
    const Float_Packed_T scale_val =
        int8_mode == 2 ? cuda_cast<Float_Packed_T>(*scale) : cuda_cast<Float_Packed_T>(0.0f);

    for (int idx = threadIdx.x; idx < head_num * size_per_head; idx += blockDim.x) {
        const int head_id   = idx / size_per_head;
        const int hidden_id = idx % size_per_head;
        const T   src_elem  = ldg(&src[src_offset_base + head_id * seq_len * size_per_head + hidden_id]);
        if (int8_mode == 2) {
            reinterpret_cast<Int8_Packed_T*>(dst)[dst_offset_base + idx] =
                cuda_cast<Int8_Packed_T>(cuda_cast<Float_Packed_T>(src_elem) * scale_val);
        }
        else {
            dst[dst_offset_base + idx] = src_elem;
        }
    }
}

// clang-format off
template<typename T>
void invokeTransposeAttentionOutRemovePadding(T*           src,
                                              T*           dst,
                                              const int    valid_word_num,
                                              const int    batch_size,
                                              const int    seq_len,
                                              const int    head_num,
                                              const int    size_per_head,
                                              const int*   mask_offset,
                                              const float* scale,
                                              const int    int8_mode,
                                              hipStream_t stream)
{
#ifdef ENABLE_BF16
    bool is_half2 = (std::is_same<T, half>::value || std::is_same<T, __hip_bfloat16>::value) && (size_per_head % 2 == 0);
#else
    bool is_half2 = (std::is_same<T, half>::value) && (size_per_head % 2 == 0);
#endif
    using T2       = typename TypeConverter<T>::Type;  // fp16 to half2, bf16 to bf162
    int block_size = head_num * size_per_head;
    if (is_half2) {
        while (block_size > 512) {
            if (block_size % 2 == 0) {
                block_size /= 2;
            }
            else {
                is_half2   = false;
                block_size = std::min(block_size, 1024);
                break;
            }
        }
    }
    else {
        block_size = std::min(block_size, 1024);
    }

    if (is_half2) {
        transpose_remove_padding<T2><<<valid_word_num, block_size, 0, stream>>>(
            (T2*)src, (T2*)dst, batch_size, seq_len, head_num, size_per_head / 2, mask_offset, scale, int8_mode);
    }
    else {
        transpose_remove_padding<<<valid_word_num, block_size, 0, stream>>>(
            src, dst, batch_size, seq_len, head_num, size_per_head, mask_offset, scale, int8_mode);
    }
}
// clang-format on

#define INSTANTIATETRANSPOSEATTENTIONOUTREMOVEPADDING(T)                                                               \
    template void invokeTransposeAttentionOutRemovePadding(T*           src,                                           \
                                                           T*           dst,                                           \
                                                           const int    valid_word_num,                                \
                                                           const int    batch_size,                                    \
                                                           const int    seq_len,                                       \
                                                           const int    head_num,                                      \
                                                           const int    size_per_head,                                 \
                                                           const int*   mask_offset,                                   \
                                                           const float* scale,                                         \
                                                           const int    int8_mode,                                     \
                                                           hipStream_t stream)
#ifdef ENABLE_FP32
INSTANTIATETRANSPOSEATTENTIONOUTREMOVEPADDING(float);
#endif
INSTANTIATETRANSPOSEATTENTIONOUTREMOVEPADDING(half);
#ifdef ENABLE_BF16
INSTANTIATETRANSPOSEATTENTIONOUTREMOVEPADDING(__hip_bfloat16);
#endif
#undef INSTANTIATETRANSPOSEATTENTIONOUTREMOVEPADDING

template<typename T>
__global__ void addRelativeAttentionBias(
    T* qk_buf, const T* relative_attention_bias, const int batch_size, const int head_num, const int seq_len)
{
    for (int i = threadIdx.x; i < batch_size * seq_len; i += blockDim.x) {
        int batch_id = i / seq_len;
        int seq_id   = i % seq_len;

        const int bias_index = blockIdx.x * seq_len + seq_id;
        const int qk_index   = batch_id * gridDim.x * seq_len + bias_index;
        qk_buf[qk_index]     = add(qk_buf[qk_index], relative_attention_bias[bias_index]);
    }
}

template<typename T>
void invokeAddRelativeAttentionBias(T*           qk_buf,
                                    const T*     relative_attention_bias,
                                    const int    batch_size,
                                    const int    head_num,
                                    const int    seq_len,
                                    hipStream_t stream)
{
    // qk_buf: [batch_size, head_num, seq_len, seq_len]
    // relative_attention_bias: [1, head_num, seq_len, seq_len]
    dim3 grid(head_num * seq_len);
    dim3 block(512);
    using T2 = typename TypeConverter<T>::Type;
#ifdef ENABLE_BF16
    const bool is_half2 = (std::is_same<T, half>::value || std::is_same<T, __hip_bfloat16>::value) && (seq_len % 2 == 0);
#else
    const bool is_half2 = (std::is_same<T, half>::value) && (seq_len % 2 == 0);
#endif
    if (is_half2) {
        addRelativeAttentionBias<T2><<<grid, block, 0, stream>>>(
            (T2*)qk_buf, (const T2*)relative_attention_bias, batch_size, head_num, seq_len / 2);
    }
    else {
        addRelativeAttentionBias<<<grid, block, 0, stream>>>(
            qk_buf, relative_attention_bias, batch_size, head_num, seq_len);
    }
}

#define INSTANTIATEADDRELATIVEATTENTIONBIAS(T)                                                                         \
    template void invokeAddRelativeAttentionBias(T*           qk_buf,                                                  \
                                                 const T*     relative_attention_bias,                                 \
                                                 const int    batch_size,                                              \
                                                 const int    head_num,                                                \
                                                 const int    seq_len,                                                 \
                                                 hipStream_t stream)
#if 0
#ifdef ENABLE_FP32
INSTANTIATEADDRELATIVEATTENTIONBIAS(float);
#endif
INSTANTIATEADDRELATIVEATTENTIONBIAS(half);
#ifdef ENABLE_BF16
INSTANTIATEADDRELATIVEATTENTIONBIAS(__hip_bfloat16);
#endif
#undef INSTANTIATEADDRELATIVEATTENTIONBIAS
#endif

}  // namespace turbomind
