#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2019-2023, NVIDIA CORPORATION.  All rights reserved.
 * Copyright (c) 2021, NAVER Corp.  Authored by CLOVA.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "src/turbomind/kernels/decoder_masked_multihead_attention_utils.h"
#include "src/turbomind/kernels/reduce_kernel_utils.cuh"
#include "src/turbomind/kernels/unfused_attention_kernels.h"
#include "src/turbomind/utils/cuda_type_utils.cuh"
#include "src/turbomind/utils/cuda_utils.h"
#include "src/turbomind/utils/logger.h"

namespace turbomind {

__inline__ __device__ int target_index(int id1, int id2, int id3, int id4, int dim_1, int dim_2, int dim_3, int dim_4)
{
    return id1 * (dim_2 * dim_3 * dim_4) + id3 * (dim_2 * dim_4) + id2 * dim_4 + id4;
}

template<typename T, typename T_IN, int ITEMS_PER_THREAD>
__global__ void softmax_kernel(T*          attn_score,
                               const T_IN* qk,
                               const T*    attn_mask,
                               const T*    linear_bias_slopes,
                               const int   batch_size,
                               const int   head_num,
                               const int   q_length,
                               const int   k_length,
                               const float qk_scale)
{
    // attn_score, [batch_size, num_heads, q_length, k_length]
    // qk, [batch_size, num_heads, q_length, k_length]
    // attn_mask, [batch_size, q_length, k_length]
    // linear_bias_slopes, [num_heads]

    const int bi = blockIdx.y;  // Batch index.
    const int hi = blockIdx.z;  // Head index.

    __shared__ float s_mean, s_max;

    const float linear_bias_slope = linear_bias_slopes != nullptr ? (float)linear_bias_slopes[hi] : 0.0f;

    // Loop along with Q dimension.
    for (int qi = blockIdx.x; qi < q_length; qi += gridDim.x) {

        float data[ITEMS_PER_THREAD];
        int   qk_offset;
        float local_max = -1e20f;

        // Loop along with K dimension.
        for (int i = 0; blockDim.x * i + threadIdx.x < k_length; i++) {
            int ki    = blockDim.x * i + threadIdx.x;  // Index of K dimension.
            qk_offset = ((bi * head_num + hi) * q_length + qi) * k_length + ki;

            float qk_val  = static_cast<float>(qk[qk_offset]);
            float qk_bias = 0.0f;

            if (linear_bias_slopes != nullptr) {
                // We don't handle the upper diagonal (ki > qi) separately, whose values
                // are negligible due to the negative infinity mask. And it matches with
                // the HF's implementation.
                qk_bias += static_cast<float>(linear_bias_slope * (ki - qi));
            }

            int   mask_offset = (bi * q_length + qi) * k_length + ki;
            float mask_val    = static_cast<float>(ldg(&attn_mask[mask_offset]));
            qk_bias += (1.0f - mask_val) * -10000.0f;

            data[i]   = qk_scale * qk_val + qk_bias;
            local_max = fmax(local_max, data[i]);
        }

        float max_val = blockDim.x <= 32 ? warpReduceMax(local_max) : blockReduceMax<float>(local_max);
        if (threadIdx.x == 0) {
            s_max = max_val;
        }
        __syncthreads();

        float local_sum = 0;
        for (int i = 0; blockDim.x * i + threadIdx.x < k_length; i++) {
            data[i] = __expf(data[i] - s_max);
            local_sum += data[i];
        }

        float sum_val = blockDim.x <= 32 ? warpReduceSum(local_sum) : blockReduceSum<float>(local_sum);
        if (threadIdx.x == 0) {
            s_mean = sum_val + 1e-6f;
            s_mean = __fdividef(1.0f, s_mean);
        }
        __syncthreads();

        for (int i = 0; blockDim.x * i + threadIdx.x < k_length; i++) {
            qk_offset             = ((bi * head_num + hi) * q_length + qi) * k_length + blockDim.x * i + threadIdx.x;
            attn_score[qk_offset] = (T)(data[i] * s_mean);
        }
    }
}

template<typename T, int ITEMS_PER_THREAD>
__global__ void softmax_kernel_h2(T*        attn_score,
                                  const T*  qk_buf,
                                  const T*  attn_mask,
                                  const T*  linear_bias_slopes,
                                  const int batch_size,
                                  const int head_num,
                                  const int q_length,
                                  const int k_length,
                                  const T   qk_scale)
{
    // attn_score, [batch_size, num_heads, q_length, k_length]
    // qk, [batch_size, num_heads, q_length, k_length]
    // attn_mask, [batch_size, q_length, k_length]
    // linear_bias_slopes, [num_heads]

    using T2 = typename TypeConverter<T>::Type;

    T2*       attn_score_h2 = reinterpret_cast<T2*>(attn_score);
    const T2* qk_buf_h2     = reinterpret_cast<const T2*>(qk_buf);
    const T2* attn_mask_h2  = reinterpret_cast<const T2*>(attn_mask);

    const int bi = blockIdx.y;  // Batch index
    const int hi = blockIdx.z;  // Head index.

    __shared__ float s_mean, s_max;

    // Constant values that will be used repeately in the q/k loop.
    const T2 ONE       = cuda_cast<T2>(1.0f);
    const T2 ZERO      = cuda_cast<T2>(0.0f);
    const T2 NEG_INFTY = cuda_cast<T2>(-10000.0f);

    // The normalization factor of QK.
    const T2 qk_scale_h2 = cuda_cast<T2>(qk_scale);
    // The slope of a linear position bias of the current attention head.
    const T2 linear_bias_slope = linear_bias_slopes != nullptr ? cuda_cast<T2>(linear_bias_slopes[hi]) : ZERO;

    // Loop over q dimension.
    for (int qi = blockIdx.x; qi < q_length; qi += gridDim.x) {
        T2    data[ITEMS_PER_THREAD];
        int   qk_offset;
        float local_max = -1e20f;

        // Loop over k dimension.
        for (int i = 0; blockDim.x * i + threadIdx.x < (k_length / 2) && i < ITEMS_PER_THREAD; i++) {
            // The half of the index of k dimension. We will use the elements at {2 * ki, 2 * ki + 1}.
            int ki          = blockDim.x * i + threadIdx.x;
            qk_offset       = ((bi * head_num + hi) * q_length + qi) * (k_length / 2) + ki;
            int mask_offset = (bi * q_length + qi) * (k_length / 2) + ki;

            // The value of QK^T matrix at (qi, ki).
            T2 qk = qk_buf_h2[qk_offset];
            // The bias value to the position (qi, ki) including both mask and positional bias.
            T2 qk_bias = ZERO;

            if (linear_bias_slopes != nullptr) {
                // The position bias depends on the distance between qi/ki and is zero if qi >= 2*ki
                // or qi >= 2*ki+1. For T2 vectorization, we should handle every two elements along
                // with k-dim simultaneously. To do this, we check qi / 2 > ki at ones instead of
                // qi >= 2*ki or 2*ki+1. It works because an diagonal element for an odd qi will be
                // zero due to slope * (qi - 2*ki+1) = 0. Thus, we don't handle the upper diagonal
                // separately, whose values are negligible due to the negative infinity mask.
                T2 dist(2.0f * ki - qi, 2.0f * ki + 1 - qi);
                qk_bias = hadd2<T2>(qk_bias, hmul2<T2>(linear_bias_slope, dist));
            }

            T2 mask_val = ldg(&attn_mask_h2[mask_offset]);
            qk_bias     = hadd2<T2>(qk_bias, hmul2<T2>(hsub2<T2>(ONE, mask_val), NEG_INFTY));

            data[i]   = hadd2<T2>(hmul2<T2>(qk, qk_scale_h2), qk_bias);
            local_max = fmax(local_max, fmax((float)data[i].x, (float)data[i].y));
        }

        float max_val = blockDim.x <= 32 ? warpReduceMax(local_max) : blockReduceMax<float>(local_max);
        if (threadIdx.x == 0) {
            s_max = max_val;
        }
        __syncthreads();

        float local_sum = 0.0f;
        for (int i = 0; blockDim.x * i + threadIdx.x < (k_length / 2) && i < ITEMS_PER_THREAD; i++) {
            data[i] = hexp2<T2>(hsub2<T2>(data[i], cuda_cast<T2>(s_max)));
            local_sum += (float)(data[i].x + data[i].y);
        }

        float sum_val = blockDim.x <= 32 ? warpReduceSum(local_sum) : blockReduceSum<float>(local_sum);

        if (threadIdx.x == 0) {
            s_mean = sum_val + 1e-6f;
            s_mean = __fdividef(1.0f, s_mean);
        }
        __syncthreads();

        for (int i = 0; blockDim.x * i + threadIdx.x < (k_length / 2) && i < ITEMS_PER_THREAD; i++) {
            qk_offset = ((bi * head_num + hi) * q_length + qi) * (k_length / 2) + blockDim.x * i + threadIdx.x;
            attn_score_h2[qk_offset] = hmul2<T2>(data[i], cuda_cast<T2>(s_mean));
        }
    }
}

template<typename T, int K_ITEMS_PER_THREAD, int Q_ITEMS_PER_THREAD>
__global__ void softmax_kernel_h2_v2(T*        attn_score,
                                     const T*  qk_buf,
                                     const T*  attn_mask,
                                     const T*  linear_bias_slopes,
                                     const int batch_size,
                                     const int head_num,
                                     const int q_length,
                                     const int k_length,
                                     const T   scalar)
{
    // attn_score, [batch_size, num_heads, q_length, k_length]
    // qk, [batch_size, num_heads, q_length, k_length]
    // attn_mask, [batch_size, q_length, k_length]
    // linear_bias_slopes, [num_heads]

    using T2 = typename TypeConverter<T>::Type;

    // QK^T matrix of shape (batch_size, head_num, q_length, k_length / 2)
    T2*       attn_score_h2 = reinterpret_cast<T2*>(attn_score);
    const T2* qk_buf_h2     = reinterpret_cast<const T2*>(qk_buf);
    const T2* attn_mask_h2  = reinterpret_cast<const T2*>(attn_mask);

    const int bi = blockIdx.y;  // Batch index
    const int hi = blockIdx.z;  // Head index.

    // Constant values that will be used repeately in the q/k loop.
    const T2 ONE       = cuda_cast<T2>(1.0f);
    const T2 ZERO      = cuda_cast<T2>(0.0f);
    const T2 NEG_INFTY = cuda_cast<T2>(-10000.0f);

    // The normalization factor of QK.
    const T2 qk_scale = cuda_cast<T2>(scalar);
    // The slope of a linear position bias of the current attention head.
    const T2 linear_bias_slope = linear_bias_slopes != nullptr ? cuda_cast<T2>(linear_bias_slopes[hi]) : ZERO;

    __shared__ float s_sum[Q_ITEMS_PER_THREAD], s_max[Q_ITEMS_PER_THREAD];

    // Loop over q dimension.
    for (int qi = blockIdx.x; qi < q_length; qi += gridDim.x * Q_ITEMS_PER_THREAD) {
        T2 data[Q_ITEMS_PER_THREAD][K_ITEMS_PER_THREAD];

        int qk_offset[Q_ITEMS_PER_THREAD];

        float local_max[Q_ITEMS_PER_THREAD];
#pragma unroll
        for (int j = 0; j < Q_ITEMS_PER_THREAD; j++) {
            local_max[j] = -1e20f;
        }

        // Loop over k dimension.
        const int Q_ITEMS = min((q_length - qi + gridDim.x - 1) / gridDim.x, Q_ITEMS_PER_THREAD);
        for (int i = 0; blockDim.x * i + threadIdx.x < k_length / 2 && i < K_ITEMS_PER_THREAD; ++i) {
            // The half of the index of k dimension. We will use the elements at {2 * ki, 2 * ki + 1}.
            int ki = blockDim.x * i + threadIdx.x;

            int mask_offset[Q_ITEMS_PER_THREAD];
#pragma unroll
            for (int j = 0; j < Q_ITEMS; j++) {
                qk_offset[j]   = ((bi * head_num + hi) * q_length + qi + j * gridDim.x) * (k_length / 2) + ki;
                mask_offset[j] = (bi * q_length + qi + j * gridDim.x) * (k_length / 2) + ki;
            }

            T2 mask_val[Q_ITEMS_PER_THREAD];
#pragma unroll
            for (int j = 0; j < Q_ITEMS; j++) {
                mask_val[j] = ldg(&attn_mask_h2[mask_offset[j]]);
            }

            T2 qk[Q_ITEMS_PER_THREAD];
#pragma unroll
            for (int j = 0; j < Q_ITEMS; j++) {
                qk[j] = qk_buf_h2[qk_offset[j]];
            }

            T2 pos_bias[Q_ITEMS_PER_THREAD];
            if (linear_bias_slopes != nullptr) {
#pragma unroll
                for (int j = 0; j < Q_ITEMS; j++) {
                    // The position bias depends on the distance between qi/ki and is zero if qi >= 2*ki
                    // or qi >= 2*ki+1. For T2 vectorization, we should handle every two elements along
                    // with k-dim simultaneously. To do this, we check qi / 2 > ki at ones instead of
                    // qi >= 2*ki or 2*ki+1. It works because an diagonal element for an odd qi will be
                    // zero due to slope * (qi - 2*ki+1) = 0. Thus, we don't handle the upper diagonal
                    // separately, whose values are negligible due to the negative infinity mask.
                    int qidx = qi + j * gridDim.x;
                    T2  dist(2.0f * ki - qidx, 2.0f * ki + 1 - qidx);
                    pos_bias[j] = hmul2<T2>(linear_bias_slope, dist);
                }
            }
#pragma unroll
            for (int j = 0; j < Q_ITEMS; j++) {
                mask_val[j] = hmul2<T2>(hsub2<T2>(ONE, mask_val[j]), NEG_INFTY);
            }

#pragma unroll
            for (int j = 0; j < Q_ITEMS; j++) {
                T2 val = hadd2<T2>(hmul2<T2>(qk_scale, qk[j]), mask_val[j]);
                if (linear_bias_slopes != nullptr) {
                    val = hadd2<T2>(val, pos_bias[j]);
                }
                data[j][i]   = val;
                local_max[j] = fmax(local_max[j], fmax((float)data[j][i].x, (float)data[j][i].y));
            }
        }

        if (blockDim.x <= 32) {
            warpReduceMaxV2<float, Q_ITEMS_PER_THREAD>(local_max);
        }
        else {
            blockReduceMaxV2<float, Q_ITEMS_PER_THREAD>(local_max);
        }

        if (threadIdx.x == 0) {
#pragma unroll
            for (int j = 0; j < Q_ITEMS_PER_THREAD; j++) {
                s_max[j] = local_max[j];
            }
        }
        __syncthreads();

        float local_sum[Q_ITEMS_PER_THREAD];
#pragma unroll
        for (int j = 0; j < Q_ITEMS_PER_THREAD; j++) {
            local_sum[j] = {0.f};
        }

        for (int i = 0; blockDim.x * i + threadIdx.x < k_length / 2 && i < K_ITEMS_PER_THREAD; ++i) {
#pragma unroll
            for (int j = 0; j < Q_ITEMS; ++j) {
                data[j][i] = hexp2<T2>(hsub2<T2>(data[j][i], cuda_cast<T2>(s_max[j])));
            }

#pragma unroll
            for (int j = 0; j < Q_ITEMS; j++) {
                local_sum[j] += (float)(data[j][i].x + data[j][i].y);
            }
        }

        if (blockDim.x <= 32) {
            warpReduceSumV2<float, Q_ITEMS_PER_THREAD>(local_sum);
        }
        else {
            blockReduceSumV2<float, Q_ITEMS_PER_THREAD>(local_sum);
        }

        if (threadIdx.x == 0) {
#pragma unroll
            for (int j = 0; j < Q_ITEMS_PER_THREAD; j++) {
                s_sum[j] = __fdividef(1.0f, local_sum[j] + 1e-6f);
            }
        }
        __syncthreads();

        for (int i = 0; blockDim.x * i + threadIdx.x < k_length / 2 && i < K_ITEMS_PER_THREAD; ++i) {
#pragma unroll
            for (int j = 0; j < Q_ITEMS; j++) {
                qk_offset[j] = ((bi * head_num + hi) * q_length + qi + j * gridDim.x) * (k_length / 2) + blockDim.x * i
                               + threadIdx.x;
            }

#pragma unroll
            for (int j = 0; j < Q_ITEMS; j++) {
                attn_score_h2[qk_offset[j]] = hmul2<T2>(data[j][i], cuda_cast<T2>(s_sum[j]));
            }
        }
    }
}

#define LAUNCH_MAKSED_SOFTMAX_(T_, ITEMS_PER_THREAD)                                                                   \
    block.x /= ITEMS_PER_THREAD;                                                                                       \
    block.x = (block.x + 31) / 32 * 32;                                                                                \
    assert(block.x <= 1024);                                                                                           \
    if (is_half2) {                                                                                                    \
        if (grid.x % 4 == 0) {                                                                                         \
            grid.x /= 4;                                                                                               \
            softmax_kernel_h2_v2<T_, ITEMS_PER_THREAD, 4>                                                              \
                <<<grid, block, 0, stream>>>((T_*)param.attention_score,                                               \
                                             (const T_*)param.qk,                                                      \
                                             (const T_*)param.attention_mask,                                          \
                                             (const T_*)param.linear_bias_slopes,                                      \
                                             param.batch_size,                                                         \
                                             param.num_heads,                                                          \
                                             param.q_length,                                                           \
                                             param.k_length,                                                           \
                                             (const T_)param.qk_scale);                                                \
        }                                                                                                              \
        else {                                                                                                         \
            softmax_kernel_h2<T_, ITEMS_PER_THREAD><<<grid, block, 0, stream>>>((T_*)param.attention_score,            \
                                                                                (const T_*)param.qk,                   \
                                                                                (const T_*)param.attention_mask,       \
                                                                                (const T_*)param.linear_bias_slopes,   \
                                                                                param.batch_size,                      \
                                                                                param.num_heads,                       \
                                                                                param.q_length,                        \
                                                                                param.k_length,                        \
                                                                                (const T_)param.qk_scale);             \
        }                                                                                                              \
    }                                                                                                                  \
    else {                                                                                                             \
        softmax_kernel<T, T_IN, ITEMS_PER_THREAD><<<grid, block, 0, stream>>>(param.attention_score,                   \
                                                                              param.qk,                                \
                                                                              param.attention_mask,                    \
                                                                              param.linear_bias_slopes,                \
                                                                              param.batch_size,                        \
                                                                              param.num_heads,                         \
                                                                              param.q_length,                          \
                                                                              param.k_length,                          \
                                                                              param.qk_scale);                         \
    }

#define LAUNCH_MAKSED_SOFTMAX(ITEMS_PER_THREAD) LAUNCH_MAKSED_SOFTMAX_(half, ITEMS_PER_THREAD)

template<typename T, typename T_IN>
void invokeMaskedSoftmax(MaskedSoftmaxParam<T, T_IN>& param, hipStream_t stream)
{
    // attention_score,    (batch_size, head_num, q_length, k_length), softmax output.
    // qk,                 (batch_size, head_num, q_length, k_length), QK^T.
    // attention_mask,     (batch_size, q_length, k_length), attention mask.
    // linear_bias_slopes, (head_num,) the slopes of the linear position bias.

    dim3 grid(param.q_length, param.batch_size, param.num_heads);
    if (param.batch_size * param.num_heads > 360) {
        grid.x = ceil(float(param.q_length) / 32.0f);
    }

    bool is_half2 = sizeof(T) == 2 && sizeof(T_IN) == 2 && param.k_length % 2 == 0;
    dim3 block((param.k_length / (is_half2 ? 2 : 1) + 31) / 32 * 32);

    if (block.x > 2048 && block.x <= 4096) {
        LAUNCH_MAKSED_SOFTMAX(4)
    }
    else if (block.x > 1024) {
        LAUNCH_MAKSED_SOFTMAX(2)
    }
    else if (block.x > 0) {
        LAUNCH_MAKSED_SOFTMAX(1)
    }
    else {
        FT_CHECK(param.k_length <= 4096);
    }
}

template void invokeMaskedSoftmax(MaskedSoftmaxParam<float, float>& param, hipStream_t stream);
template void invokeMaskedSoftmax(MaskedSoftmaxParam<half, float>& param, hipStream_t stream);
template void invokeMaskedSoftmax(MaskedSoftmaxParam<half, half>& param, hipStream_t stream);

#ifdef ENABLE_BF16
template<>
void invokeMaskedSoftmax(MaskedSoftmaxParam<__hip_bfloat16, float>& param, hipStream_t stream)
{
    // attention_score,    (batch_size, head_num, q_length, k_length), softmax output.
    // qk,                 (batch_size, head_num, q_length, k_length), QK^T.
    // attention_mask,     (batch_size, q_length, k_length), attention mask.
    // linear_bias_slopes, (head_num,) the slopes of the linear position bias.

    using T    = __hip_bfloat16;
    using T_IN = float;

    dim3 grid(param.q_length, param.batch_size, param.num_heads);
    if (param.batch_size * param.num_heads > 360) {
        grid.x = ceil(float(param.q_length) / 32.0f);
    }

    bool is_half2 = sizeof(T) == 2 && sizeof(T_IN) == 2 && param.k_length % 2 == 0;
    dim3 block((param.k_length / (is_half2 ? 2 : 1) + 31) / 32 * 32);

    if (block.x > 2048 && block.x <= 4096) {
        LAUNCH_MAKSED_SOFTMAX_(__hip_bfloat16, 4);
    }
    else if (block.x > 1024) {
        LAUNCH_MAKSED_SOFTMAX_(__hip_bfloat16, 2);
    }
    else if (block.x > 0) {
        LAUNCH_MAKSED_SOFTMAX_(__hip_bfloat16, 1);
    }
    else {
        FT_CHECK(param.k_length <= 4096);
    }
}
template<>
void invokeMaskedSoftmax(MaskedSoftmaxParam<__hip_bfloat16, __hip_bfloat16>& param, hipStream_t stream)
{
    // attention_score,    (batch_size, head_num, q_length, k_length), softmax output.
    // qk,                 (batch_size, head_num, q_length, k_length), QK^T.
    // attention_mask,     (batch_size, q_length, k_length), attention mask.
    // linear_bias_slopes, (head_num,) the slopes of the linear position bias.

    using T    = __hip_bfloat16;
    using T_IN = __hip_bfloat16;

    dim3 grid(param.q_length, param.batch_size, param.num_heads);
    if (param.batch_size * param.num_heads > 360) {
        grid.x = ceil(float(param.q_length) / 32.0f);
    }

    bool is_half2 = sizeof(T) == 2 && sizeof(T_IN) == 2 && param.k_length % 2 == 0;
    dim3 block((param.k_length / (is_half2 ? 2 : 1) + 31) / 32 * 32);

    if (block.x > 2048 && block.x <= 4096) {
        LAUNCH_MAKSED_SOFTMAX_(__hip_bfloat16, 4);
    }
    else if (block.x > 1024) {
        LAUNCH_MAKSED_SOFTMAX_(__hip_bfloat16, 2);
    }
    else if (block.x > 0) {
        LAUNCH_MAKSED_SOFTMAX_(__hip_bfloat16, 1);
    }
    else {
        FT_CHECK(param.k_length <= 4096);
    }
}

#endif

#undef LAUNCH_MAKSED_SOFTMAX
#undef LAUNCH_MAKSED_SOFTMAX_

template<typename T>
__global__ void transpose(const T*     src,
                          T*           dst,
                          const int    batch_size,
                          const int    seq_len,
                          const int    head_num,
                          const int    size_per_head,
                          const float* scale,
                          int          int8_mode)
{
    int tid = blockIdx.x * blockDim.x + threadIdx.x;

    int batch_id = tid / (head_num * seq_len * size_per_head);
    int head_id  = (tid % (head_num * seq_len * size_per_head)) / (seq_len * size_per_head);
    int seq_id   = (tid % (seq_len * size_per_head)) / size_per_head;
    int id       = tid % size_per_head;

    int target_id = target_index(batch_id, head_id, seq_id, id, batch_size, head_num, seq_len, size_per_head);

    if (int8_mode == 2) {
        using Int8_Packed_T  = typename packed_as<int8_t, num_elems<T>::value>::type;
        using Float_Packed_T = typename packed_as<float, num_elems<T>::value>::type;

        const Float_Packed_T scale_val = cuda_cast<Float_Packed_T>(*scale);
        reinterpret_cast<Int8_Packed_T*>(dst)[target_id] =
            cuda_cast<Int8_Packed_T>(cuda_cast<Float_Packed_T>(src[tid]) * scale_val);
    }
    else {
        dst[target_id] = src[tid];
    }
}

template<>
__global__ void transpose(const float* src,
                          float*       dst,
                          const int    batch_size,
                          const int    seq_len,
                          const int    head_num,
                          const int    size_per_head,
                          const float* scale,
                          int          int8_mode)
{
    int batch_id = blockIdx.x / (head_num * seq_len);
    int seq_id   = blockIdx.x % seq_len;
    int head_id  = (blockIdx.x % (head_num * seq_len)) / seq_len;

    const int target_id = batch_id * (head_num * seq_len * size_per_head) + seq_id * head_num * size_per_head
                          + head_id * size_per_head + threadIdx.x;
    const int src_id = blockIdx.x * size_per_head + threadIdx.x;

    if (int8_mode == 2) {
        const float scale_val                     = *scale;
        reinterpret_cast<int8_t*>(dst)[target_id] = cuda_cast<int8_t>(src[src_id] * scale_val);
    }
    else {
        dst[target_id] = src[src_id];
    }
}

template<typename T>
void invokeTransposeQKV(T*           dst,
                        T*           src,
                        const int    batch_size,
                        const int    seq_len,
                        const int    head_num,
                        const int    size_per_head,
                        const float* scale,
                        const int    int8_mode,
                        hipStream_t stream)
{
    dim3 grid, block;
    if (sizeof(T) == 2) {
        int seq_per_block = 1;
        grid.x            = batch_size * head_num * seq_len / seq_per_block;
        while (seq_per_block < 4 && grid.x % 2 == 0) {
            grid.x /= 2;
            seq_per_block *= 2;
        }

        FT_CHECK(grid.x * seq_per_block == (size_t)batch_size * head_num * seq_len);

        if (seq_per_block * size_per_head % 2 == 0) {
            block.x = seq_per_block * size_per_head / 2;
            if (std::is_same<T, half>::value) {
                transpose<half2><<<grid, block, 0, stream>>>(
                    (half2*)src, (half2*)dst, batch_size, seq_len, head_num, size_per_head / 2, scale, int8_mode);
            }
#ifdef ENABLE_BF16
            else {
                transpose<__hip_bfloat162><<<grid, block, 0, stream>>>((__hip_bfloat162*)src,
                                                                      (__hip_bfloat162*)dst,
                                                                      batch_size,
                                                                      seq_len,
                                                                      head_num,
                                                                      size_per_head / 2,
                                                                      scale,
                                                                      int8_mode);
            }
#endif
        }
        else {
            block.x = seq_per_block * size_per_head;
            transpose<T>
                <<<grid, block, 0, stream>>>(src, dst, batch_size, seq_len, head_num, size_per_head, scale, int8_mode);
        }
    }
    else {
        const int seq_per_block = 1;
        grid.x                  = batch_size * head_num * seq_len / seq_per_block;
        block.x                 = seq_per_block * size_per_head;
        transpose<T>
            <<<grid, block, 0, stream>>>(src, dst, batch_size, seq_len, head_num, size_per_head, scale, int8_mode);
    }
}

#define INSTANTIATETRANSPOSEQKV(T)                                                                                     \
    template void invokeTransposeQKV(T*           src,                                                                 \
                                     T*           dst,                                                                 \
                                     const int    batch_size,                                                          \
                                     const int    seq_len,                                                             \
                                     const int    head_num,                                                            \
                                     const int    size_per_head,                                                       \
                                     const float* scale,                                                               \
                                     const int    int8_mode,                                                           \
                                     hipStream_t stream)
INSTANTIATETRANSPOSEQKV(float);
INSTANTIATETRANSPOSEQKV(half);
#ifdef ENABLE_BF16
INSTANTIATETRANSPOSEQKV(__hip_bfloat16);
#endif
#undef INSTANTIATETRANSPOSEQKV

template<typename T>
__global__ void transpose_remove_padding(const T*     src,
                                         T*           dst,
                                         const int    batch_size,
                                         const int    seq_len,
                                         const int    head_num,
                                         const int    size_per_head,
                                         const int*   mask_offset,
                                         const float* scale,
                                         const int    int8_mode)
{
    // TODO: optimize this kernel?
    // do remove_sequence_length_padding
    const int bid = blockIdx.x;  // batch * seq_len or valid_word_num

    const int src_batch_id = (bid + mask_offset[bid]) / seq_len;
    const int src_seq_id   = (bid + mask_offset[bid]) % seq_len;

    const int dst_seq_id = bid;

    const int src_offset_base = src_batch_id * seq_len * head_num * size_per_head + src_seq_id * size_per_head;
    const int dst_offset_base = dst_seq_id * head_num * size_per_head;

    using Int8_Packed_T  = typename packed_as<int8_t, num_elems<T>::value>::type;
    using Float_Packed_T = typename packed_as<float, num_elems<T>::value>::type;
    const Float_Packed_T scale_val =
        int8_mode == 2 ? cuda_cast<Float_Packed_T>(*scale) : cuda_cast<Float_Packed_T>(0.0f);

    for (int idx = threadIdx.x; idx < head_num * size_per_head; idx += blockDim.x) {
        const int head_id   = idx / size_per_head;
        const int hidden_id = idx % size_per_head;
        const T   src_elem  = ldg(&src[src_offset_base + head_id * seq_len * size_per_head + hidden_id]);
        if (int8_mode == 2) {
            reinterpret_cast<Int8_Packed_T*>(dst)[dst_offset_base + idx] =
                cuda_cast<Int8_Packed_T>(cuda_cast<Float_Packed_T>(src_elem) * scale_val);
        }
        else {
            dst[dst_offset_base + idx] = src_elem;
        }
    }
}

// clang-format off
template<typename T>
void invokeTransposeAttentionOutRemovePadding(T*           src,
                                              T*           dst,
                                              const int    valid_word_num,
                                              const int    batch_size,
                                              const int    seq_len,
                                              const int    head_num,
                                              const int    size_per_head,
                                              const int*   mask_offset,
                                              const float* scale,
                                              const int    int8_mode,
                                              hipStream_t stream)
{
#ifdef ENABLE_BF16
    bool is_half2 = (std::is_same<T, half>::value || std::is_same<T, __hip_bfloat16>::value) && (size_per_head % 2 == 0);
#else
    bool is_half2 = (std::is_same<T, half>::value) && (size_per_head % 2 == 0);
#endif
    using T2       = typename TypeConverter<T>::Type;  // fp16 to half2, bf16 to bf162
    int block_size = head_num * size_per_head;
    if (is_half2) {
        while (block_size > 512) {
            if (block_size % 2 == 0) {
                block_size /= 2;
            }
            else {
                is_half2   = false;
                block_size = std::min(block_size, 1024);
                break;
            }
        }
    }
    else {
        block_size = std::min(block_size, 1024);
    }

    if (is_half2) {
        transpose_remove_padding<T2><<<valid_word_num, block_size, 0, stream>>>(
            (T2*)src, (T2*)dst, batch_size, seq_len, head_num, size_per_head / 2, mask_offset, scale, int8_mode);
    }
    else {
        transpose_remove_padding<<<valid_word_num, block_size, 0, stream>>>(
            src, dst, batch_size, seq_len, head_num, size_per_head, mask_offset, scale, int8_mode);
    }
}
// clang-format on

#define INSTANTIATETRANSPOSEATTENTIONOUTREMOVEPADDING(T)                                                               \
    template void invokeTransposeAttentionOutRemovePadding(T*           src,                                           \
                                                           T*           dst,                                           \
                                                           const int    valid_word_num,                                \
                                                           const int    batch_size,                                    \
                                                           const int    seq_len,                                       \
                                                           const int    head_num,                                      \
                                                           const int    size_per_head,                                 \
                                                           const int*   mask_offset,                                   \
                                                           const float* scale,                                         \
                                                           const int    int8_mode,                                     \
                                                           hipStream_t stream)
INSTANTIATETRANSPOSEATTENTIONOUTREMOVEPADDING(float);
INSTANTIATETRANSPOSEATTENTIONOUTREMOVEPADDING(half);
#ifdef ENABLE_BF16
INSTANTIATETRANSPOSEATTENTIONOUTREMOVEPADDING(__hip_bfloat16);
#endif
#undef INSTANTIATETRANSPOSEATTENTIONOUTREMOVEPADDING

template<typename T>
__global__ void add_fusedQKV_bias_transpose_kernel(T* q_buf,
                                                   T* k_buf,
                                                   T* v_buf,
                                                   T* QKV,
                                                   const T* __restrict qkv_bias,
                                                   const int*   padding_offset,
                                                   const int    batch_size,
                                                   const int    seq_len,
                                                   const int    token_num,
                                                   const int    head_num,
                                                   const int    size_per_head,
                                                   const float* scale,
                                                   const int    int8_mode)
{
    // QKV: [token_num, 3, n]
    // qkv_bias: [3, n]
    // q_buf, k_buf, v_buf: [batch, head_num, seq_len, size_per_head]

    T*        qkv_ptr[3] = {q_buf, k_buf, v_buf};
    const int n          = head_num * size_per_head;
    for (int index = blockDim.x * blockIdx.x + threadIdx.x; index < token_num * 3 * n;
         index += gridDim.x * blockDim.x) {
        const int bias_id = index % (3 * n);

        const int token_idx        = index / (3 * n);
        const int token_padded_idx = token_idx + (padding_offset == nullptr ? 0 : padding_offset[token_idx]);
        const int target_batch_id  = token_padded_idx / seq_len;
        const int seq_id           = token_padded_idx % seq_len;

        const int qkv_id  = (index % (3 * n)) / n;
        const int head_id = (index % n) / size_per_head;
        const int size_id = index % size_per_head;

        T val;
        if (int8_mode == 2) {
            val = cuda_cast<T>(cuda_cast<float>(reinterpret_cast<const int8_t*>(QKV)[index]) * scale[qkv_id]);
        }
        else {
            val = ldg(&QKV[index]);
        }
        val = val + ldg(&qkv_bias[bias_id]);

        if (int8_mode == 2) {
            // TODO(mseznec): add support for int8 BMM with FusedAtt
        }
        else {
            QKV[index] = val;
        }

        qkv_ptr[qkv_id][target_batch_id * head_num * seq_len * size_per_head + head_id * seq_len * size_per_head
                        + seq_id * size_per_head + size_id] = val;
    }
}

template<typename T>
struct Vec_t {
    static constexpr int size = 0;
};

template<>
struct Vec_t<float> {
    using Type                = float2;
    static constexpr int size = 2;
};

template<>
struct Vec_t<half> {
    using Type                = uint32_t;
    static constexpr int size = 2;
};

#ifdef ENABLE_BF16
template<>
struct Vec_t<__hip_bfloat16> {
    using Type                = __hip_bfloat162;
    static constexpr int size = 2;
};
#endif

/// TODO: support batch step offset
template<typename T, bool PREFIX_PROMPT>
__global__ void add_fusedQKV_bias_transpose_kernel(T* q_buf,
                                                   T* k_buf,
                                                   T* v_buf,
                                                   T* QKV,
                                                   const T* __restrict qkv_bias,
                                                   const int* padding_offset,
                                                   const int* history_length,
                                                   const int* input_length,
                                                   int        batch_size,
                                                   int        seq_len,
                                                   int        head_num,
                                                   int        kv_head_num,
                                                   int        size_per_head,
                                                   int        rotary_embedding_dim,
                                                   float      rotary_embedding_base,
                                                   int        max_position_embeddings,
                                                   bool       use_dynamic_ntk,
                                                   bool       use_logn_attn)
{
    // This kernel add bias to QKV, which has shape [batch_size, seq_len, 3, head_num, size_per_head], and
    // QKV split to 3 split buffer q, k, v and transpose them to [batch_size, head_num, seq_len, size_per_head].
    // For q and k, also apply the rotary embedding.

    // NOTE: QKV src shape (batch_size, seq_len, 3, head_num, size_per_head)
    //  QKV dst shape (3, batch_size, head_num, seq_len, size_per_head)
    extern __shared__ __align__(sizeof(float2)) char smem_[];  // align on largest vector type

    constexpr int vec_size         = Vec_t<T>::size;
    using Vec_t                    = typename Vec_t<T>::Type;
    const int token_idx            = blockIdx.x;
    const int token_padding_offset = (padding_offset == nullptr || token_idx < 0) ? 0 : padding_offset[token_idx];
    const int tgt_token_idx        = token_idx + token_padding_offset;

    const int batch_idx = tgt_token_idx / seq_len;
    const int seq_idx   = tgt_token_idx % seq_len;

    const int head_idx = blockIdx.y;
    const int tidx     = threadIdx.x;

    const int total_seq_len = seq_len;

    const bool is_masked = tidx * vec_size >= size_per_head;

    const int hidden_idx = head_idx * size_per_head + tidx * vec_size;

    const int q_kv_head_num = head_num + 2 * kv_head_num;

    const int k_offset = head_num * size_per_head;
    const int v_offset = k_offset + kv_head_num * size_per_head;

    // src QKV: [batch, time, q_kv_head_num, hidden]
    const int src_q_idx = token_idx * q_kv_head_num * size_per_head + hidden_idx;
    const int src_k_idx = token_idx * q_kv_head_num * size_per_head + hidden_idx + k_offset;
    const int src_v_idx = token_idx * q_kv_head_num * size_per_head + hidden_idx + v_offset;

    Vec_t q, k, v;
    Vec_t q_bias, k_bias, v_bias;

    // load Q and apply bias
    if (!is_masked) {
        q = *reinterpret_cast<const Vec_t*>(&QKV[src_q_idx]);
        if (qkv_bias) {
            q_bias = *reinterpret_cast<const Vec_t*>(&qkv_bias[hidden_idx]);
            q      = mmha::add(q, q_bias);
        }
    }

    // load KV and apply bias
    if (!is_masked && head_idx < kv_head_num) {
        k = *reinterpret_cast<const Vec_t*>(&QKV[src_k_idx]);
        v = *reinterpret_cast<const Vec_t*>(&QKV[src_v_idx]);
        if (qkv_bias) {
            k_bias = *reinterpret_cast<const Vec_t*>(&qkv_bias[hidden_idx + k_offset]);
            v_bias = *reinterpret_cast<const Vec_t*>(&qkv_bias[hidden_idx + v_offset]);
            k      = mmha::add(k, k_bias);
            v      = mmha::add(v, v_bias);
        }
    }

    const int history_len = history_length[batch_idx];
    const int context_len = history_len + input_length[batch_idx];
    const int timestep    = history_len + seq_idx;

    if (use_dynamic_ntk) {
        rotary_embedding_base = mmha::rotary_embedding_get_base(
            context_len, max_position_embeddings, rotary_embedding_dim, rotary_embedding_base);
    }

    // TODO: unused computation on k if GQA is used
    mmha::apply_rotary_embedding(q, k, tidx, rotary_embedding_dim, rotary_embedding_base, timestep);

    if (use_logn_attn) {
        // +1 to convert to context length at the timestep
        float logn_scaling = mmha::logn_attn_get_scaling(timestep + 1, max_position_embeddings);
        if constexpr (std::is_same_v<T, float>) {
            q = mmha::mul<Vec_t, float, Vec_t>(logn_scaling, q);
        }
        else if constexpr (std::is_same_v<T, half>) {
            half tmp = __float2half(logn_scaling);
            q        = mmha::mul<Vec_t, uint16_t, Vec_t>((uint16_t&)tmp, q);
        }
    }

    if (!is_masked && !q_buf) {  // also skip modifying QKV if q/k/v_buf are present
        *reinterpret_cast<Vec_t*>(&QKV[src_q_idx]) = q;
        if (head_idx < kv_head_num) {
            *reinterpret_cast<Vec_t*>(&QKV[src_k_idx]) = k;
            *reinterpret_cast<Vec_t*>(&QKV[src_v_idx]) = v;
        }
    }

    const int dest_q_idx = batch_idx * size_per_head * seq_len * head_num + head_idx * size_per_head * seq_len
                           + seq_idx * size_per_head + tidx * vec_size;

    const int dest_kv_idx = batch_idx * size_per_head * total_seq_len * kv_head_num
                            + head_idx * size_per_head * total_seq_len + seq_idx * size_per_head + tidx * vec_size;

    if (!is_masked) {
        *reinterpret_cast<Vec_t*>(&q_buf[dest_q_idx]) = q;
        if (head_idx < kv_head_num) {
            *reinterpret_cast<Vec_t*>(&k_buf[dest_kv_idx]) = k;
            *reinterpret_cast<Vec_t*>(&v_buf[dest_kv_idx]) = v;
        }
    }
}

#define FUSED_QKV_BIAS_TRANSPOSE_LAUNCH(T, PREFIX_PROMPT)                                                              \
    add_fusedQKV_bias_transpose_kernel<T, PREFIX_PROMPT><<<grid, block, smem_size, stream>>>(q_buf,                    \
                                                                                             k_buf,                    \
                                                                                             v_buf,                    \
                                                                                             QKV,                      \
                                                                                             qkv_bias,                 \
                                                                                             padding_offset,           \
                                                                                             history_length,           \
                                                                                             input_length,             \
                                                                                             batch_size,               \
                                                                                             seq_len,                  \
                                                                                             head_num,                 \
                                                                                             kv_head_num,              \
                                                                                             size_per_head,            \
                                                                                             rotary_embedding_dim,     \
                                                                                             rotary_embedding_base,    \
                                                                                             max_position_embeddings,  \
                                                                                             use_dynamic_ntk,          \
                                                                                             use_logn_attn);

template<typename T>
void invokeAddFusedQKVBiasTranspose(T*           q_buf,
                                    T*           k_buf,
                                    T*           v_buf,
                                    T*           QKV,
                                    const T*     qkv_bias,
                                    const int*   padding_offset,
                                    const int*   history_length,
                                    const int*   input_length,
                                    const int    batch_size,
                                    const int    seq_len,
                                    const int    token_num,
                                    const int    head_num,
                                    const int    kv_head_num,
                                    const int    size_per_head,
                                    const int    rotary_embedding_dim,
                                    float        rotary_embedding_base,
                                    int          max_position_embeddings,
                                    bool         use_dynamic_ntk,
                                    bool         use_logn_attn,
                                    hipStream_t stream)
{
    FT_CHECK(rotary_embedding_dim);
    // To implement rotary embeddings, each thread processes two QKV elems:
    dim3   block((size_per_head / Vec_t<T>::size + 31) / 32 * 32);
    dim3   grid(token_num, head_num);
    size_t smem_size = 0;
    FUSED_QKV_BIAS_TRANSPOSE_LAUNCH(T, false);
}

#define INSTANTIATEADDFUSEDQKVBIASTRANSPOSE(T)                                                                         \
    template void invokeAddFusedQKVBiasTranspose(T*           q_buf,                                                   \
                                                 T*           k_buf,                                                   \
                                                 T*           v_buf,                                                   \
                                                 T*           QKV,                                                     \
                                                 const T*     qkv_bias,                                                \
                                                 const int*   padding_offset,                                          \
                                                 const int*   history_length,                                          \
                                                 const int*   input_length,                                            \
                                                 const int    batch_size,                                              \
                                                 const int    seq_len,                                                 \
                                                 const int    token_num,                                               \
                                                 const int    head_num,                                                \
                                                 const int    kv_head_num,                                             \
                                                 const int    size_per_head,                                           \
                                                 const int    rotary_embedding_dim,                                    \
                                                 float        rotary_embedding_base,                                   \
                                                 int          max_position_embeddings,                                 \
                                                 bool         use_dynamic_ntk,                                         \
                                                 bool         use_logn_attn,                                           \
                                                 hipStream_t stream)
INSTANTIATEADDFUSEDQKVBIASTRANSPOSE(float);
INSTANTIATEADDFUSEDQKVBIASTRANSPOSE(half);
#ifdef ENABLE_BF16
INSTANTIATEADDFUSEDQKVBIASTRANSPOSE(__hip_bfloat16);
#endif
#undef INSTANTIATEADDFUSEDQKVBIASTRANSPOSE

template<typename T>
__global__ void transpose_4d(T*        dst,
                             T*        src,
                             const int dim0,
                             const int dim1,
                             const int dim2,
                             const int dim3,
                             const int dim0_leading_dim,
                             const int ite)
{
    // transpose from [dim0, dim1, dim2, dim3] to [dim2, X, dim1, dim3]
    // where the dimension of X is dim0_leading_dim, and offset is ite * dim0
    for (int i = threadIdx.x + blockIdx.x * blockDim.x; i < dim0 * dim1 * dim2 * dim3; i += blockDim.x * gridDim.x) {
        int       index = i;
        const int d3    = index % dim3;
        index           = (index - d3) / dim3;
        const int d2    = index % dim2;
        index           = (index - d2) / dim2;
        const int d1    = index % dim1;
        index           = (index - d1) / dim1;
        const int d0    = index % dim0;
        index           = (index - d0) / dim0;
        dst[d2 * dim0_leading_dim * dim1 * dim3 + (d0 + dim0 * ite) * dim1 * dim3 + d1 * dim3 + d3] = src[i];
    }
}

template<>
__global__ void transpose_4d(half*     dst,
                             half*     src,
                             const int dim0,
                             const int dim1,
                             const int dim2,
                             const int dim3,
                             const int dim0_leading_dim,
                             const int ite)
{
    half2*    dst_ptr   = (half2*)dst;
    half2*    src_ptr   = (half2*)src;
    const int half_dim3 = dim3 / 2;
    // transpose from [dim0, dim1, dim2, half_dim3] to [dim2, dim0, dim1, half_dim3]
    // where the dimension of X is dim0_leading_dim, and offset is ite * dim0
    for (int i = threadIdx.x + blockIdx.x * blockDim.x; i < dim0 * dim1 * dim2 * half_dim3;
         i += blockDim.x * gridDim.x) {
        int       index = i;
        const int d3    = index % half_dim3;
        index           = (index - d3) / half_dim3;
        const int d2    = index % dim2;
        index           = (index - d2) / dim2;
        const int d1    = index % dim1;
        index           = (index - d1) / dim1;
        const int d0    = index % dim0;
        index           = (index - d0) / dim0;
        dst_ptr[d2 * dim0_leading_dim * dim1 * half_dim3 + (d0 + dim0 * ite) * dim1 * half_dim3 + d1 * half_dim3 + d3] =
            src_ptr[i];
    }
}

template<typename T>
void invokeTranspose4d(T*           dst,
                       T*           src,
                       const int    local_batch_size,
                       const int    seq_len,
                       const int    size_per_head,
                       const int    local_hidden_units,
                       const int    local_head_num,
                       const int    batch_size,
                       const int    ite,
                       hipStream_t stream)
{
    transpose_4d<<<local_batch_size * seq_len * local_hidden_units / 512, 512 / (4 / (sizeof(T))), 0, stream>>>(
        dst, src, local_batch_size, local_head_num, seq_len, size_per_head, batch_size, ite);
}

#define INSTANTIATETRANSPOSE4D(T)                                                                                      \
    template void invokeTranspose4d(T*           dst,                                                                  \
                                    T*           src,                                                                  \
                                    const int    local_batch_size,                                                     \
                                    const int    seq_len,                                                              \
                                    const int    size_per_head,                                                        \
                                    const int    local_hidden_units,                                                   \
                                    const int    local_head_num,                                                       \
                                    const int    batch_size,                                                           \
                                    const int    ite,                                                                  \
                                    hipStream_t stream)
INSTANTIATETRANSPOSE4D(float);
INSTANTIATETRANSPOSE4D(half);
#undef INSTANTIATETRANSPOSE4D
template<typename T>
__global__ void transpose_4d_batch_major_k_cache(
    T* k_dst, const T* k_src, const int head_num, const int size_per_head, const int seq_len, const int max_seq_len)
{
    const int     batch_id = blockIdx.y;
    const int     head_id  = blockIdx.z;
    constexpr int X_ELEMS  = (sizeof(T) == 4) ? 4 : 8;

    auto key_src = reinterpret_cast<const uint4*>(k_src + batch_id * head_num * size_per_head * seq_len
                                                  + head_id * size_per_head * seq_len);
    auto key_dst = reinterpret_cast<uint4*>(k_dst + batch_id * head_num * size_per_head * max_seq_len
                                            + head_id * size_per_head * max_seq_len);

    const int out_idx             = blockIdx.x * blockDim.x + threadIdx.x;
    int       size_per_head_div_x = size_per_head / X_ELEMS;
    if (out_idx >= size_per_head_div_x * max_seq_len) {
        return;
    }

    int       idx            = out_idx;
    const int k_seq_len_id   = idx % max_seq_len;
    idx                      = (idx - k_seq_len_id) / max_seq_len;
    const int k_head_size_id = idx % size_per_head_div_x;

    if (k_seq_len_id < seq_len) {
        key_dst[out_idx] = key_src[k_seq_len_id * size_per_head_div_x + k_head_size_id];
    }
}

template<typename T>
__global__ void transpose_4d_batch_major_v_cache(
    T* v_dst, const T* v_src, const int head_num, const int size_per_head, const int seq_len, const int max_seq_len)
{
    const int batch_id = blockIdx.y;
    const int head_id  = blockIdx.z;

    // 16 byte loads will handle "x" dimension
    auto val_src = reinterpret_cast<const uint4*>(v_src + batch_id * head_num * size_per_head * seq_len
                                                  + head_id * size_per_head * seq_len);
    auto val_dst = reinterpret_cast<uint4*>(v_dst + batch_id * head_num * size_per_head * max_seq_len
                                            + head_id * size_per_head * max_seq_len);

    // idx is over output dimension L * size_per_head / x for values
    const int idx = blockIdx.x * blockDim.x + threadIdx.x;

    constexpr int X_ELEMS             = (sizeof(T) == 4) ? 4 : 8;
    const int     size_per_head_div_x = size_per_head / X_ELEMS;

    if (idx >= size_per_head_div_x * seq_len) {
        return;
    }

    val_dst[idx] = val_src[idx];
}

template<typename T>
void invokeTranspose4dBatchMajor(T*           k_dst,
                                 T*           v_dst,
                                 const T*     k_src,
                                 const T*     v_src,
                                 const int    local_batch_size,
                                 const int    seq_len,
                                 const int    max_seq_len,
                                 const int    size_per_head,
                                 const int    local_head_num,
                                 hipStream_t stream)
{
    constexpr int block_sz = 128;
    constexpr int x        = (sizeof(T) == 4) ? 4 : 8;
    int           size     = max_seq_len * size_per_head / x;
    dim3          grid((size + block_sz - 1) / block_sz, local_batch_size, local_head_num);
    dim3          grid_v((seq_len * size_per_head / x + block_sz - 1) / block_sz, local_batch_size, local_head_num);

    transpose_4d_batch_major_k_cache<<<grid, block_sz, 0, stream>>>(
        k_dst, k_src, local_head_num, size_per_head, seq_len, max_seq_len);

    transpose_4d_batch_major_v_cache<<<grid_v, block_sz, 0, stream>>>(
        v_dst, v_src, local_head_num, size_per_head, seq_len, max_seq_len);
}

#define INSTANTIATETRANSPOSE4DBATCHMAJOR(T)                                                                            \
    template void invokeTranspose4dBatchMajor(T*           k_dst,                                                      \
                                              T*           v_dst,                                                      \
                                              const T*     k_src,                                                      \
                                              const T*     v_src,                                                      \
                                              const int    local_batch_size,                                           \
                                              const int    seq_len,                                                    \
                                              const int    max_seq_len,                                                \
                                              const int    size_per_head,                                              \
                                              const int    local_head_num,                                             \
                                              hipStream_t stream)
INSTANTIATETRANSPOSE4DBATCHMAJOR(float);
INSTANTIATETRANSPOSE4DBATCHMAJOR(half);
#ifdef ENABLE_BF16
INSTANTIATETRANSPOSE4DBATCHMAJOR(__hip_bfloat16);
#endif
#undef INSTANTIATETRANSPOSE4DBATCHMAJOR

template<typename T>
__global__ void addRelativeAttentionBias(
    T* qk_buf, const T* relative_attention_bias, const int batch_size, const int head_num, const int seq_len)
{
    for (int i = threadIdx.x; i < batch_size * seq_len; i += blockDim.x) {
        int batch_id = i / seq_len;
        int seq_id   = i % seq_len;

        const int bias_index = blockIdx.x * seq_len + seq_id;
        const int qk_index   = batch_id * gridDim.x * seq_len + bias_index;
        qk_buf[qk_index]     = add(qk_buf[qk_index], relative_attention_bias[bias_index]);
    }
}

template<typename T>
void invokeAddRelativeAttentionBias(T*           qk_buf,
                                    const T*     relative_attention_bias,
                                    const int    batch_size,
                                    const int    head_num,
                                    const int    seq_len,
                                    hipStream_t stream)
{
    // qk_buf: [batch_size, head_num, seq_len, seq_len]
    // relative_attention_bias: [1, head_num, seq_len, seq_len]
    dim3 grid(head_num * seq_len);
    dim3 block(512);
    using T2 = typename TypeConverter<T>::Type;
#ifdef ENABLE_BF16
    const bool is_half2 = (std::is_same<T, half>::value || std::is_same<T, __hip_bfloat16>::value) && (seq_len % 2 == 0);
#else
    const bool is_half2 = (std::is_same<T, half>::value) && (seq_len % 2 == 0);
#endif
    if (is_half2) {
        addRelativeAttentionBias<T2><<<grid, block, 0, stream>>>(
            (T2*)qk_buf, (const T2*)relative_attention_bias, batch_size, head_num, seq_len / 2);
    }
    else {
        addRelativeAttentionBias<<<grid, block, 0, stream>>>(
            qk_buf, relative_attention_bias, batch_size, head_num, seq_len);
    }
}

#define INSTANTIATEADDRELATIVEATTENTIONBIAS(T)                                                                         \
    template void invokeAddRelativeAttentionBias(T*           qk_buf,                                                  \
                                                 const T*     relative_attention_bias,                                 \
                                                 const int    batch_size,                                              \
                                                 const int    head_num,                                                \
                                                 const int    seq_len,                                                 \
                                                 hipStream_t stream)
INSTANTIATEADDRELATIVEATTENTIONBIAS(float);
INSTANTIATEADDRELATIVEATTENTIONBIAS(half);
#ifdef ENABLE_BF16
INSTANTIATEADDRELATIVEATTENTIONBIAS(__hip_bfloat16);
#endif
#undef INSTANTIATEADDRELATIVEATTENTIONBIAS

/*******************  invokeAddHead3SizeQKVBias  ***********************/
// m = batch*window_num*window_len
// mm_qkv is [m, head*3*size_per_head] row-major
// bias_qkv is [head*3*size_per_head]
// q_buf_, k_buf_, v_buf_ is [batch*window_num, num_head, window_len, size_per_head] row-major
// grid(window_len, window_num, 3*batch);
// block(num_head * size_per_head)
template<typename T>
__global__ void add_head3Size_QKV_bias(const T*  mm_qkv,
                                       const T*  bias_qkv,
                                       T*        q_buf_,
                                       T*        k_buf_,
                                       T*        v_buf_,
                                       const int batch,
                                       const int window_num,
                                       const int window_len,
                                       const int num_head,
                                       const int size_per_head)
{

    T*  buf_ptr;
    int qkv_id = blockIdx.z / batch;
    if (qkv_id == 0) {
        buf_ptr = q_buf_;
    }
    else if (qkv_id == 1) {
        buf_ptr = k_buf_;
    }
    else {
        buf_ptr = v_buf_;
    }

    const int batch_id   = blockIdx.z % batch;
    const int token_id   = blockIdx.x;
    const int window_id  = blockIdx.y;
    const int head_id    = threadIdx.x / size_per_head;
    const int id_in_head = threadIdx.x % size_per_head;

    const int bias_idx = (head_id * 3 + qkv_id) * size_per_head + id_in_head;
    const T   bias     = ldg(bias_qkv + bias_idx);

    const int input_idx =
        ((batch_id * window_num + window_id) * window_len + token_id) * num_head * 3 * size_per_head + bias_idx;
    T tmp = mm_qkv[input_idx] + bias;

    int target_id = (((batch_id * window_num + window_id) * num_head + head_id) * window_len + token_id) * size_per_head
                    + id_in_head;
    ;
    buf_ptr[target_id] = tmp;
}

// for float2, size_per_head /= 2
// m = batch*window_num*window_len
// mm_qkv is [m, head*3*size_per_head] row-major
// bias_qkv is [head*3*size_per_head]
// q_buf_, k_buf_, v_buf_ is [batch*window_num, num_head, window_len, size_per_head] row-major
// grid(window_len, window_num, 3*batch);
// block(num_head * size_per_head)
template<>
__global__ void add_head3Size_QKV_bias(const float2* mm_qkv,
                                       const float2* bias_qkv,
                                       float2*       q_buf_,
                                       float2*       k_buf_,
                                       float2*       v_buf_,
                                       const int     batch,
                                       const int     window_num,
                                       const int     window_len,
                                       const int     num_head,
                                       const int     size_per_head)
{

    float2* buf_ptr;
    int     qkv_id = blockIdx.z / batch;
    if (qkv_id == 0) {
        buf_ptr = q_buf_;
    }
    else if (qkv_id == 1) {
        buf_ptr = k_buf_;
    }
    else {
        buf_ptr = v_buf_;
    }

    const int batch_id   = blockIdx.z % batch;
    const int token_id   = blockIdx.x;
    const int window_id  = blockIdx.y;
    const int head_id    = threadIdx.x / size_per_head;
    const int id_in_head = threadIdx.x % size_per_head;

    const int    bias_idx = (head_id * 3 + qkv_id) * size_per_head + id_in_head;
    const float2 bias     = ldg(bias_qkv + bias_idx);

    const int input_idx =
        ((batch_id * window_num + window_id) * window_len + token_id) * num_head * 3 * size_per_head + bias_idx;
    float2 tmp = mm_qkv[input_idx];
    tmp.x += bias.x;
    tmp.y += bias.y;

    int target_id = (((batch_id * window_num + window_id) * num_head + head_id) * window_len + token_id) * size_per_head
                    + id_in_head;
    ;
    buf_ptr[target_id] = tmp;
}

// for half2, size_per_head /= 2
// m = batch*window_num*window_len
// mm_qkv is [m, head*3*size_per_head] row-major
// bias_qkv is [head*3*size_per_head]
// q_buf_, k_buf_, v_buf_ is [batch*window_num, num_head, window_len, size_per_head] row-major
// grid(window_len, window_num, batch);
// block(num_head * size_per_head)
template<>
__global__ void add_head3Size_QKV_bias(const half2* mm_qkv,
                                       const half2* bias_qkv,
                                       half2*       q_buf_,
                                       half2*       k_buf_,
                                       half2*       v_buf_,
                                       const int    batch,
                                       const int    window_num,
                                       const int    window_len,
                                       const int    num_head,
                                       const int    size_per_head)
{

    const int batch_id   = blockIdx.z;
    const int token_id   = blockIdx.x;
    const int window_id  = blockIdx.y;
    const int head_id    = threadIdx.x / size_per_head;
    const int id_in_head = threadIdx.x % size_per_head;

    const int input_offset =
        ((batch_id * window_num + window_id) * window_len + token_id) * num_head * 3 * size_per_head;
    const int target_id =
        (((batch_id * window_num + window_id) * num_head + head_id) * window_len + token_id) * size_per_head
        + id_in_head;

    int   qkv_id      = 0;
    int   bias_idx    = (head_id * 3 + qkv_id) * size_per_head + id_in_head;
    half2 bias        = __ldg(bias_qkv + bias_idx);
    int   input_idx   = input_offset + bias_idx;
    half2 tmp         = mm_qkv[input_idx];
    tmp               = __hadd2(tmp, bias);
    q_buf_[target_id] = tmp;

    qkv_id            = 1;
    bias_idx          = (head_id * 3 + qkv_id) * size_per_head + id_in_head;
    bias              = __ldg(bias_qkv + bias_idx);
    input_idx         = input_offset + bias_idx;
    tmp               = mm_qkv[input_idx];
    tmp               = __hadd2(tmp, bias);
    k_buf_[target_id] = tmp;

    qkv_id            = 2;
    bias_idx          = (head_id * 3 + qkv_id) * size_per_head + id_in_head;
    bias              = __ldg(bias_qkv + bias_idx);
    input_idx         = input_offset + bias_idx;
    tmp               = mm_qkv[input_idx];
    tmp               = __hadd2(tmp, bias);
    v_buf_[target_id] = tmp;
}

#ifdef ENABLE_BF16
template<>
__global__ void add_head3Size_QKV_bias(const __hip_bfloat162* mm_qkv,
                                       const __hip_bfloat162* bias_qkv,
                                       __hip_bfloat162*       q_buf_,
                                       __hip_bfloat162*       k_buf_,
                                       __hip_bfloat162*       v_buf_,
                                       const int             batch,
                                       const int             window_num,
                                       const int             window_len,
                                       const int             num_head,
                                       const int             size_per_head)
{

    const int batch_id   = blockIdx.z;
    const int token_id   = blockIdx.x;
    const int window_id  = blockIdx.y;
    const int head_id    = threadIdx.x / size_per_head;
    const int id_in_head = threadIdx.x % size_per_head;

    const int input_offset =
        ((batch_id * window_num + window_id) * window_len + token_id) * num_head * 3 * size_per_head;
    const int target_id =
        (((batch_id * window_num + window_id) * num_head + head_id) * window_len + token_id) * size_per_head
        + id_in_head;

    int            qkv_id    = 0;
    int            bias_idx  = (head_id * 3 + qkv_id) * size_per_head + id_in_head;
    __hip_bfloat162 bias      = ldg(bias_qkv + bias_idx);
    int            input_idx = input_offset + bias_idx;
    __hip_bfloat162 tmp       = mm_qkv[input_idx];
    tmp                      = bf16hadd2(tmp, bias);
    q_buf_[target_id]        = tmp;

    qkv_id            = 1;
    bias_idx          = (head_id * 3 + qkv_id) * size_per_head + id_in_head;
    bias              = ldg(bias_qkv + bias_idx);
    input_idx         = input_offset + bias_idx;
    tmp               = mm_qkv[input_idx];
    tmp               = bf16hadd2(tmp, bias);
    k_buf_[target_id] = tmp;

    qkv_id            = 2;
    bias_idx          = (head_id * 3 + qkv_id) * size_per_head + id_in_head;
    bias              = ldg(bias_qkv + bias_idx);
    input_idx         = input_offset + bias_idx;
    tmp               = mm_qkv[input_idx];
    tmp               = bf16hadd2(tmp, bias);
    v_buf_[target_id] = tmp;
}
#endif

template<typename T>
void invokeAddHead3SizeQKVBias(const T*     mm_qkv,
                               const T*     bias_qkv,
                               T*           q_buf_,
                               T*           k_buf_,
                               T*           v_buf_,
                               const int    batch,
                               const int    window_num,
                               const int    window_len,
                               const int    num_head,
                               const int    size_per_head,
                               hipStream_t stream)
{
    if (std::is_same<T, float>::value) {
        dim3 grid(window_len, window_num, 3 * batch);
        dim3 block(num_head * size_per_head);

        if (block.x < 1024) {
            add_head3Size_QKV_bias<<<grid, block, 0, stream>>>(
                mm_qkv, bias_qkv, q_buf_, k_buf_, v_buf_, batch, window_num, window_len, num_head, size_per_head);
        }
        else if ((block.x % 2 == 0) && (block.x / 2 < 1024)) {
            block.x /= 2;
            add_head3Size_QKV_bias<<<grid, block, 0, stream>>>((const float2*)mm_qkv,
                                                               (const float2*)bias_qkv,
                                                               (float2*)q_buf_,
                                                               (float2*)k_buf_,
                                                               (float2*)v_buf_,
                                                               batch,
                                                               window_num,
                                                               window_len,
                                                               num_head,
                                                               size_per_head / 2);
        }
        else {
            printf("[ERROR][invokeAddHead3SizeQKVBias] unsupported block.x!\n");
            exit(-1);
        }
    }
#ifdef ENABLE_BF16
    else if (std::is_same<T, half>::value || std::is_same<T, __hip_bfloat16>::value) {
#else
    else if (std::is_same<T, half>::value) {
#endif
        dim3 grid(window_len, window_num, batch);
        dim3 block(num_head * size_per_head / 2);

        using T2 = typename TypeConverter<T>::Type;  // half2 or bfloat16

        if (block.x > 1024) {
            printf("[ERROR][invokeAddHead3SizeQKVBias] block.x > 1024!\n");
            exit(-1);
        }

        add_head3Size_QKV_bias<<<grid, block, 0, stream>>>((const T2*)mm_qkv,
                                                           (const T2*)bias_qkv,
                                                           (T2*)q_buf_,
                                                           (T2*)k_buf_,
                                                           (T2*)v_buf_,
                                                           batch,
                                                           window_num,
                                                           window_len,
                                                           num_head,
                                                           size_per_head / 2);
    }
}

#define INSTANTIATEADDHEAD3SIZEQKVBIAS(T)                                                                              \
    template void invokeAddHead3SizeQKVBias<T>(const T*     mm_qkv,                                                    \
                                               const T*     bias_qkv,                                                  \
                                               T*           q_buf_,                                                    \
                                               T*           k_buf_,                                                    \
                                               T*           v_buf_,                                                    \
                                               const int    batch,                                                     \
                                               const int    window_num,                                                \
                                               const int    window_len,                                                \
                                               const int    num_head,                                                  \
                                               const int    size_per_head,                                             \
                                               hipStream_t stream)
INSTANTIATEADDHEAD3SIZEQKVBIAS(float);
INSTANTIATEADDHEAD3SIZEQKVBIAS(half);
#ifdef ENABLE_BF16
INSTANTIATEADDHEAD3SIZEQKVBIAS(__hip_bfloat16);
#endif
#undef INSTANTIATEADDHEAD3SIZEQKVBIAS

/*******************  invokeMaskedSoftMaxWithRelPosBias  ***********************/

// grid = (window_len/word_per_thread, window_num*num_head, batch_size)
// block.x = max(32, (window_len + 31)/32*32)
// qk_buf is [batch, window_num, num_head, window_len, window_len]
// attn_mask is [window_num, window_len, window_len] + row-major
// relative_pos_bias is [num_head, window_len, window_len] + row-majot
template<typename T>
__global__ void softmax_withRelPosBias_element1_kernel(T*          qk_buf,
                                                       const T*    attn_mask,
                                                       const T*    relative_pos_bias,
                                                       const int   batch_size,
                                                       const int   num_head,
                                                       const int   window_num,
                                                       const int   window_len,
                                                       const int   window_len_x_window_len,
                                                       const float qk_scale)
{

    bool qual = threadIdx.x < window_len;
    for (int window_id = blockIdx.x; window_id < window_len; window_id += gridDim.x) {
        float            tmp = -1e20f;
        __shared__ float s_mean, s_max;
        int              qk_offset;
        if (qual) {
            const int offset_in_window = window_id * window_len + threadIdx.x;
            qk_offset = (blockIdx.z * gridDim.y + blockIdx.y) * window_len_x_window_len + offset_in_window;
            const int relative_pos_bias_offset = (blockIdx.y % num_head) * window_len_x_window_len + offset_in_window;
            float     mask_val =
                (attn_mask == nullptr) ?
                        0.0f :
                        static_cast<float>(
                        ldg(attn_mask + ((blockIdx.y / num_head) * window_len_x_window_len + offset_in_window)));
            tmp = qk_scale * static_cast<float>(qk_buf[qk_offset]) + mask_val
                  + static_cast<float>(ldg(relative_pos_bias + relative_pos_bias_offset));
        }

        float max_val = blockReduceMax<float>(tmp);
        if (threadIdx.x == 0) {
            s_max = max_val;
        }
        __syncthreads();

        float qk_tmp  = qual ? __expf(tmp - s_max) : 0.0f;
        float sum_val = blockReduceSum<float>(qk_tmp);
        if (threadIdx.x == 0) {
            s_mean = sum_val + 1e-6f;
            s_mean = __fdividef(1.0f, s_mean);
        }
        __syncthreads();
        if (qual) {
            qk_buf[qk_offset] = (T)(qk_tmp * s_mean);
        }
    }
}

// grid = (window_len/word_per_thread, window_num*num_head, batch_size)
// block.x = max(32, (window_len/2 + 31)/32*32)
// qk_buf is [batch, window_num, num_head, window_len, window_len]
// attn_mask is [window_num, window_len, window_len] + row-major
// relative_pos_bias is [num_head, window_len, window_len] + row-majot
template<typename T2, typename T>
__global__ void softmax_withRelPosBias_element2_kernel(T2*         qk_buf,
                                                       const T2*   attn_mask,
                                                       const T2*   relative_pos_bias,
                                                       const int   batch_size,
                                                       const int   num_head,
                                                       const int   window_num,
                                                       const int   window_len,
                                                       const int   window_len_x_window_len,
                                                       const float qk_scale)
{
    const int window_len_2 = window_len / 2;
    const int tidx         = threadIdx.x;
    bool      qual         = tidx < window_len_2;
    const T2  zero         = {T(0.0f), T(0.0f)};
    const int bdim         = blockDim.x;
    for (int window_id = blockIdx.x; window_id < window_len; window_id += gridDim.x) {
        float            tmp = -1e20f;
        __shared__ float s_mean, s_max;
        int              qk_offset;
        float2           local_qk_val;
        T2               qk_val;
        if (qual) {
            const int offset_in_window = window_id * window_len + 2 * tidx;
            qk_offset = ((blockIdx.z * gridDim.y + blockIdx.y) * window_len_x_window_len + offset_in_window) / 2;
            const int relative_pos_bias_offset =
                ((blockIdx.y % num_head) * window_len_x_window_len + offset_in_window) / 2;
            T2 mask_val =
                (attn_mask == nullptr) ?
                    zero :
                    ldg(attn_mask + ((blockIdx.y / num_head) * window_len_x_window_len + offset_in_window) / 2);
            qk_val            = qk_buf[qk_offset];
            local_qk_val.x    = static_cast<float>(qk_val.x);
            local_qk_val.y    = static_cast<float>(qk_val.y);
            const T2 bias_val = ldg(relative_pos_bias + relative_pos_bias_offset);
            local_qk_val.x =
                qk_scale * local_qk_val.x + static_cast<float>(mask_val.x) + static_cast<float>(bias_val.x);
            local_qk_val.y =
                qk_scale * local_qk_val.y + static_cast<float>(mask_val.y) + static_cast<float>(bias_val.y);
            tmp = local_qk_val.x > local_qk_val.y ? local_qk_val.x : local_qk_val.y;
        }

        float max_val = bdim <= 32 ? warpReduceMax<float>(tmp) : blockReduceMax<float>(tmp);
        if (tidx == 0) {
            s_max = max_val;
        }
        __syncthreads();

        local_qk_val.x = qual ? __expf(local_qk_val.x - s_max) : 0.0f;
        local_qk_val.y = qual ? __expf(local_qk_val.y - s_max) : 0.0f;

        float sum_val = bdim <= 32 ? warpReduceSum<float>(local_qk_val.x + local_qk_val.y) :
                                     blockReduceSum<float>(local_qk_val.x + local_qk_val.y);
        if (tidx == 0) {
            s_mean = sum_val + 1e-6f;
            s_mean = __fdividef(1.0f, s_mean);
        }
        __syncthreads();
        if (qual) {
            local_qk_val.x    = local_qk_val.x * s_mean;
            local_qk_val.y    = local_qk_val.y * s_mean;
            qk_val.x          = T(local_qk_val.x);
            qk_val.y          = T(local_qk_val.y);
            qk_buf[qk_offset] = qk_val;
        }
    }
}

// grid = (window_len/word_per_thread, window_num*num_head, batch_size)
// block.x = max(32, (window_len/4 + 31)/32*32)
// qk_buf is [batch, window_num, num_head, window_len, window_len]
// attn_mask is [window_num, window_len, window_len] + row-major
// relative_pos_bias is [num_head, window_len, window_len] + row-majot
template<typename T4, typename T>
__global__ void softmax_withRelPosBias_element4_kernel(T4*         qk_buf,
                                                       const T4*   attn_mask,
                                                       const T4*   relative_pos_bias,
                                                       const int   batch_size,
                                                       const int   num_head,
                                                       const int   window_num,
                                                       const int   window_len,
                                                       const int   window_len_x_window_len,
                                                       const float qk_scale)
{
    const int window_len_4 = window_len / 4;
    const int tidx         = threadIdx.x;
    bool      qual         = tidx < window_len_4;
    const T4  zero         = {T(0.0f), T(0.0f), T(0.0f), T(0.0f)};
    const int bdim         = blockDim.x;
    for (int window_id = blockIdx.x; window_id < window_len; window_id += gridDim.x) {
        float            tmp = -1e20f;
        __shared__ float s_mean, s_max;
        int              qk_offset;
        float4           local_qk_val;
        T4               qk_val;
        if (qual) {
            const int offset_in_window = window_id * window_len + 4 * tidx;
            qk_offset = ((blockIdx.z * gridDim.y + blockIdx.y) * window_len_x_window_len + offset_in_window) / 4;
            const int relative_pos_bias_offset =
                ((blockIdx.y % num_head) * window_len_x_window_len + offset_in_window) / 4;
            T4 mask_val       = (attn_mask == nullptr) ?
                                    zero :
                                    attn_mask[((blockIdx.y / num_head) * window_len_x_window_len + offset_in_window) / 4];
            qk_val            = qk_buf[qk_offset];
            local_qk_val.x    = static_cast<float>(qk_val.x);
            local_qk_val.y    = static_cast<float>(qk_val.y);
            local_qk_val.z    = static_cast<float>(qk_val.z);
            local_qk_val.w    = static_cast<float>(qk_val.w);
            const T4 bias_val = relative_pos_bias[relative_pos_bias_offset];
            local_qk_val.x =
                qk_scale * local_qk_val.x + static_cast<float>(mask_val.x) + static_cast<float>(bias_val.x);
            local_qk_val.y =
                qk_scale * local_qk_val.y + static_cast<float>(mask_val.y) + static_cast<float>(bias_val.y);
            local_qk_val.z =
                qk_scale * local_qk_val.z + static_cast<float>(mask_val.z) + static_cast<float>(bias_val.z);
            local_qk_val.w =
                qk_scale * local_qk_val.w + static_cast<float>(mask_val.w) + static_cast<float>(bias_val.w);
            tmp = local_qk_val.x > local_qk_val.y ? local_qk_val.x : local_qk_val.y;
            tmp = tmp > local_qk_val.z ? tmp : local_qk_val.z;
            tmp = tmp > local_qk_val.w ? tmp : local_qk_val.w;
        }

        float max_val = bdim <= 32 ? warpReduceMax<float>(tmp) : blockReduceMax<float>(tmp);
        if (tidx == 0) {
            s_max = max_val;
        }
        __syncthreads();

        local_qk_val.x = qual ? __expf(local_qk_val.x - s_max) : 0.0f;
        local_qk_val.y = qual ? __expf(local_qk_val.y - s_max) : 0.0f;
        local_qk_val.z = qual ? __expf(local_qk_val.z - s_max) : 0.0f;
        local_qk_val.w = qual ? __expf(local_qk_val.w - s_max) : 0.0f;

        float sum_val = bdim <= 32 ?
                            warpReduceSum<float>(local_qk_val.x + local_qk_val.y + local_qk_val.z + local_qk_val.w) :
                            blockReduceSum<float>(local_qk_val.x + local_qk_val.y + local_qk_val.z + local_qk_val.w);
        if (tidx == 0) {
            s_mean = sum_val + 1e-6f;
            s_mean = __fdividef(1.0f, s_mean);
        }
        __syncthreads();
        if (qual) {
            local_qk_val.x    = local_qk_val.x * s_mean;
            local_qk_val.y    = local_qk_val.y * s_mean;
            local_qk_val.z    = local_qk_val.z * s_mean;
            local_qk_val.w    = local_qk_val.w * s_mean;
            qk_val.x          = T(local_qk_val.x);
            qk_val.y          = T(local_qk_val.y);
            qk_val.z          = T(local_qk_val.z);
            qk_val.w          = T(local_qk_val.w);
            qk_buf[qk_offset] = qk_val;
        }
    }
}

template<typename T>
void invokeMaskedSoftMaxWithRelPosBias(T*           qk_buf,
                                       const T*     attn_mask,
                                       const T*     relative_pos_bias,
                                       const int    batch_size,
                                       const int    num_head,
                                       const int    window_num,
                                       const int    window_len,
                                       float        qk_scale,
                                       hipStream_t stream)
{
    const int word_per_thread = 1;
    dim3      grid((window_len + word_per_thread - 1) / word_per_thread, window_num * num_head, batch_size);
    if ((window_len % 4 == 0) && window_len / 4 >= 32) {
        dim3 block((window_len / 4 + 31) / 32 * 32);
        if (std::is_same<T, float>::value) {
            softmax_withRelPosBias_element4_kernel<float4, float>
                <<<grid, block, 0, stream>>>((float4*)qk_buf,
                                             (const float4*)attn_mask,
                                             (const float4*)relative_pos_bias,
                                             batch_size,
                                             num_head,
                                             window_num,
                                             window_len,
                                             window_len * window_len,
                                             qk_scale);
        }
        else if (std::is_same<T, half>::value) {
            softmax_withRelPosBias_element4_kernel<half4, half>
                <<<grid, block, 0, stream>>>((half4*)qk_buf,
                                             (const half4*)attn_mask,
                                             (const half4*)relative_pos_bias,
                                             batch_size,
                                             num_head,
                                             window_num,
                                             window_len,
                                             window_len * window_len,
                                             qk_scale);
        }
#ifdef ENABLE_BF16
        else {
            dim3 block((window_len + 31) / 32 * 32);
            softmax_withRelPosBias_element1_kernel<<<grid, block, 0, stream>>>(qk_buf,
                                                                               attn_mask,
                                                                               relative_pos_bias,
                                                                               batch_size,
                                                                               num_head,
                                                                               window_num,
                                                                               window_len,
                                                                               window_len * window_len,
                                                                               qk_scale);
        }
#endif
    }
    else if (window_len % 2 == 0) {
        dim3 block((window_len / 2 + 31) / 32 * 32);
        if (std::is_same<T, float>::value) {
            softmax_withRelPosBias_element2_kernel<float2, float>
                <<<grid, block, 0, stream>>>((float2*)qk_buf,
                                             (const float2*)attn_mask,
                                             (const float2*)relative_pos_bias,
                                             batch_size,
                                             num_head,
                                             window_num,
                                             window_len,
                                             window_len * window_len,
                                             qk_scale);
        }
        else if (std::is_same<T, half>::value) {
            softmax_withRelPosBias_element2_kernel<half2, half>
                <<<grid, block, 0, stream>>>((half2*)qk_buf,
                                             (const half2*)attn_mask,
                                             (const half2*)relative_pos_bias,
                                             batch_size,
                                             num_head,
                                             window_num,
                                             window_len,
                                             window_len * window_len,
                                             qk_scale);
        }
#ifdef ENABLE_BF16
        else {
            dim3 block((window_len + 31) / 32 * 32);
            softmax_withRelPosBias_element1_kernel<<<grid, block, 0, stream>>>(qk_buf,
                                                                               attn_mask,
                                                                               relative_pos_bias,
                                                                               batch_size,
                                                                               num_head,
                                                                               window_num,
                                                                               window_len,
                                                                               window_len * window_len,
                                                                               qk_scale);
        }
#endif
    }
    else {
        dim3 block((window_len + 31) / 32 * 32);
        softmax_withRelPosBias_element1_kernel<<<grid, block, 0, stream>>>(qk_buf,
                                                                           attn_mask,
                                                                           relative_pos_bias,
                                                                           batch_size,
                                                                           num_head,
                                                                           window_num,
                                                                           window_len,
                                                                           window_len * window_len,
                                                                           qk_scale);
    }
}

#define INSTANTIATEMASKEDSOFTMAXWITHRELPOSBIAS(T)                                                                      \
    template void invokeMaskedSoftMaxWithRelPosBias(T*           qk_buf,                                               \
                                                    const T*     attn_mask,                                            \
                                                    const T*     relative_pos_bias,                                    \
                                                    const int    batch_size,                                           \
                                                    const int    num_head,                                             \
                                                    const int    window_num,                                           \
                                                    const int    window_len,                                           \
                                                    const float  qk_scale,                                             \
                                                    hipStream_t stream)
INSTANTIATEMASKEDSOFTMAXWITHRELPOSBIAS(float);
INSTANTIATEMASKEDSOFTMAXWITHRELPOSBIAS(half);
#ifdef ENABLE_BF16
INSTANTIATEMASKEDSOFTMAXWITHRELPOSBIAS(__hip_bfloat16);
#endif
#undef INSTANTIATEMASKEDSOFTMAXWITHRELPOSBIAS

template<typename T>
__global__ void transpose_attentions(
    T* attentions_out, const T* attentions_in, size_t batch_size, size_t num_layers, size_t num_heads, size_t seq_len)
{
    // attentions_in  shape [B, H, S, S]
    // attentions_out shape [B, L, H, S, S].
    // Note that we write the L dimension as if it was index 0.
    // In reality, the pointer has already been shifted to point to the correct layer.

    const auto batch_idx = blockIdx.x;
    const auto head_idx  = blockIdx.y;

    const auto dst_offset = (batch_idx * num_layers * num_heads + head_idx) * seq_len * seq_len;
    const auto src_offset = (batch_idx * num_heads + head_idx) * seq_len * seq_len;

    for (auto x = threadIdx.x; x < seq_len * seq_len; x += blockDim.x) {
        attentions_out[dst_offset + x] = attentions_in[src_offset + x];
    }
}

template<typename T>
void invokeTransposeAttentions(Tensor& attentions_out, const Tensor& attentions_in, hipStream_t stream)
{
    const size_t batch_size = attentions_in.shape[0];
    const size_t num_heads  = attentions_in.shape[1];
    const size_t seq_len    = attentions_in.shape[2];
    const size_t num_layers = attentions_out.shape[1];

    const dim3 gridSize(batch_size, num_heads);
    const dim3 blockSize(512);

    transpose_attentions<<<gridSize, blockSize, 0, stream>>>(
        attentions_out.getPtr<T>(), attentions_in.getPtr<const T>(), batch_size, num_layers, num_heads, seq_len);
}

#define INSTANTIATETRANSPOSEATTENTIONS(T)                                                                              \
    template void invokeTransposeAttentions<T>(                                                                        \
        Tensor & attentions_out, const Tensor& attentions_in, hipStream_t stream)
INSTANTIATETRANSPOSEATTENTIONS(float);
INSTANTIATETRANSPOSEATTENTIONS(half);
#ifdef ENABLE_BF16
INSTANTIATETRANSPOSEATTENTIONS(__hip_bfloat16);
#endif
#undef INSTANTIATETRANSPOSEATTENTIONS

}  // namespace turbomind
