#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2019-2023, NVIDIA CORPORATION.  All rights reserved.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "src/fastertransformer/layers/beam_search_layers/OnlineBeamSearchLayer.h"

namespace fastertransformer {

static const int SMALL_TOP_K_SOFTMAX_MAX_VOC_PARTS = 128;
static const int MAX_K                             = 4;

template<typename T>
__global__ void update_kernel(bool*          finished,
                              int*           parent_ids,
                              int*           sequence_length,
                              int*           word_ids,
                              int*           output_ids,
                              BeamHypotheses beam_hyps,
                              const int      vocab_size,
                              const int*     end_ids,
                              const int      local_batch_size,
                              const int      beam_width)
{
    for (int index = blockIdx.x * blockDim.x + threadIdx.x; index < local_batch_size * beam_width;
         index += blockDim.x * gridDim.x) {

        int batch_id           = index / beam_width;
        sequence_length[index] = finished[index] ? sequence_length[index] : sequence_length[index] + 1;

        int beam_id = (word_ids[index] / vocab_size) % beam_width;
        int word_id = word_ids[index] % vocab_size;

        sequence_length[index] = sequence_length[batch_id * beam_width + beam_id];
        finished[index]        = word_id == end_ids[index / beam_width] ? 1 : 0;
        parent_ids[index]      = beam_id;
        word_ids[index]        = word_id;
        output_ids[index]      = word_id;

        if (beam_hyps.num_beams != nullptr) {
            if (beam_hyps.num_beams[beam_hyps.ite * beam_hyps.local_batch_size + batch_id] == beam_width) {
                for (int i = 0; i < beam_width; i++) {
                    finished[batch_id * beam_width + i] = true;
                }
            }
        }
    }
}

void invokeUpdate(bool*           finished,
                  int*            parent_ids,
                  int*            sequence_length,
                  int*            word_ids,
                  int*            output_ids,
                  BeamHypotheses* beam_hyps,
                  const int       local_batch_size,
                  const int       beam_width,
                  const int       vocab_size_padded,
                  const int*      end_ids,
                  hipStream_t    stream)
{
    dim3 grid((int)ceil(local_batch_size * beam_width * 1.0 / 256));
    dim3 block(256);

    update_kernel<float><<<grid, block, 0, stream>>>(finished,
                                                     parent_ids,
                                                     sequence_length,
                                                     word_ids,
                                                     output_ids,
                                                     *beam_hyps,
                                                     vocab_size_padded,
                                                     end_ids,
                                                     local_batch_size,
                                                     beam_width);
}

template<typename T>
void OnlineBeamSearchLayer<T>::invokeSoftMax(TensorMap* output_tensors, TensorMap* input_tensors)
{
    // input_tensors:
    //      logits [local_batch_size, beam_width, vocab_size_padded]
    //      embedding_bias [vocab_size_padded]
    //      step [1] on cpu
    //      src_cache_indirection [local_batch_size, beam_width, max_seq_len]
    //      max_input_length [1] on cpu
    //      input_lengths [local_batch_size * beam_width]
    //      ite [1] on cpu
    //      beam_search_diversity_rate [1] on cpu, optional
    //      temperature [1] on cpu, optional
    //      len_penalty [1] on cpu, optional
    //      repetition_penalty [1] on cpu, optional

    // output_tensors:
    //      output_ids [max_seq_len, batch_size, beam_width]
    //      finished [local_batch_size * beam_width]
    //      cum_log_probs [local_batch_size * beam_width]
    //      parent_ids [max_seq_len, batch_size * beam_width]
    //      sequence_length [local_batch_size * beam_width]
    //      tgt_cache_indirection [local_batch_size, beam_width, max_seq_len]
    //      output_log_probs [max_seq_len, batch_size, beam_width]

    FT_CHECK(input_tensors->size() >= 7);
    FT_CHECK(output_tensors->size() >= 6);

    const int   batch_size       = output_tensors->at("output_ids").shape[1];
    const int   beam_width       = output_tensors->at("output_ids").shape[2];
    const int   step             = input_tensors->at("step").getVal<int>();
    const int   ite              = input_tensors->at("ite").getVal<int>();
    const int   local_batch_size = input_tensors->at("logits").shape[0];
    const float diversity_rate   = input_tensors->isExist("beam_search_diversity_rate") ?
                                       input_tensors->at("beam_search_diversity_rate").getVal<float>() :
                                       0.0f;
    const float length_penalty =
        input_tensors->isExist("len_penalty") ? input_tensors->at("len_penalty").getVal<float>() : 0.0f;

    const int id_offset = step * batch_size * beam_width + local_batch_size * ite * beam_width;

    BeamHypotheses beam_hyps;
    if (output_tensors->isExist("beam_hyps")) {
        beam_hyps                      = *((BeamHypotheses*)(output_tensors->at("beam_hyps").getPtr<void>()));
        beam_hyps.step                 = step;
        beam_hyps.ite                  = ite;
        beam_hyps.local_batch_size     = local_batch_size;
        beam_hyps.batch_size           = output_tensors->at("output_ids").shape[1];
        beam_hyps.max_seq_len          = output_tensors->at("output_ids").shape[0];
        beam_hyps.output_ids_src       = output_tensors->at("output_ids").getPtr<int>();
        beam_hyps.parent_ids_src       = output_tensors->at("parent_ids").getPtr<int>();
        beam_hyps.sequence_lengths_src = output_tensors->at("sequence_length").getPtr<int>();
        beam_hyps.log_probs_src        = output_tensors->getPtr<float>("output_log_probs", nullptr);
        beam_hyps.length_penalty       = length_penalty;
        beam_hyps.end_ids              = input_tensors->at("end_id").getPtr<int>();
    }

    invokeTopkSoftMax(input_tensors->at("logits").getPtr<T>(),
                      (const T*)(nullptr),
                      output_tensors->at("finished").getPtr<bool>(),
                      output_tensors->at("sequence_length").getPtr<int>(),
                      output_tensors->at("cum_log_probs").getPtr<float>(),
                      output_tensors->getPtrWithOffset<float>("output_log_probs", id_offset, nullptr),
                      output_tensors->at("output_ids").getPtrWithOffset<int>(id_offset),
                      topk_softmax_workspace_,
                      topk_softmax_workspace_size_,
                      &beam_hyps,
                      local_batch_size,
                      beam_width,
                      vocab_size_padded_,
                      input_tensors->at("end_id").getPtr<int>(),
                      diversity_rate,
                      length_penalty,
                      stream_);
    sync_check_cuda_error();

    invokeUpdate(output_tensors->at("finished").getPtr<bool>(),
                 output_tensors->at("parent_ids").getPtrWithOffset<int>(id_offset),
                 output_tensors->at("sequence_length").getPtr<int>(),
                 output_tensors->at("output_ids").getPtrWithOffset<int>(id_offset),
                 output_tensors->at("output_ids").getPtrWithOffset<int>(id_offset),
                 &beam_hyps,
                 local_batch_size,
                 beam_width,
                 vocab_size_padded_,
                 input_tensors->at("end_id").getPtr<const int>(),
                 stream_);
    sync_check_cuda_error();
}

template<typename T>
void OnlineBeamSearchLayer<T>::allocateBuffer()
{
    FT_CHECK(false);
}

template<typename T>
void OnlineBeamSearchLayer<T>::allocateBuffer(size_t batch_size, size_t beam_width)
{
    FT_LOG_DEBUG(__PRETTY_FUNCTION__);
    // we need to check 2 * beam_width candidates each time
    // 64 is the max beam width we support now.
    topk_softmax_workspace_size_ =
        (size_t)(ceil(batch_size * 64 * (64 * 2) / 4.) * 4 * 2
                 + ceil(batch_size * (64 * 2) * SMALL_TOP_K_SOFTMAX_MAX_VOC_PARTS * (2 * (MAX_K * 2) + 2) / 4.) * 4);

    topk_softmax_workspace_ = reinterpret_cast<float*>(
        allocator_->reMalloc(topk_softmax_workspace_, sizeof(float) * topk_softmax_workspace_size_, true));
    is_allocate_buffer_ = true;
}

template<typename T>
OnlineBeamSearchLayer<T>::OnlineBeamSearchLayer(size_t           max_batch_size,
                                                size_t           head_num,
                                                size_t           size_per_head,
                                                size_t           beam_width,
                                                size_t           vocab_size,
                                                size_t           vocab_size_padded,
                                                int              end_id,
                                                float            diversity_rate,
                                                float            temperature,
                                                float            len_penalty,
                                                float            repetition_penalty,
                                                hipStream_t     stream,
                                                cublasMMWrapper* cublas_wrapper,
                                                IAllocator*      allocator,
                                                bool             is_free_buffer_after_forward):
    BaseBeamSearchLayer<T>(max_batch_size,
                           head_num,
                           size_per_head,
                           beam_width,
                           vocab_size,
                           vocab_size_padded,
                           end_id,
                           diversity_rate,
                           temperature,
                           len_penalty,
                           repetition_penalty,
                           stream,
                           cublas_wrapper,
                           allocator,
                           is_free_buffer_after_forward)
{
}

template<typename T>
OnlineBeamSearchLayer<T>::OnlineBeamSearchLayer(OnlineBeamSearchLayer<T> const& beam_search_layer):
    BaseBeamSearchLayer<T>(beam_search_layer)
{
    FT_LOG_DEBUG(__PRETTY_FUNCTION__);
}

template<typename T>
OnlineBeamSearchLayer<T>::~OnlineBeamSearchLayer()
{
    FT_LOG_DEBUG(__PRETTY_FUNCTION__);
}

template class OnlineBeamSearchLayer<float>;
template class OnlineBeamSearchLayer<half>;

}  // namespace fastertransformer
