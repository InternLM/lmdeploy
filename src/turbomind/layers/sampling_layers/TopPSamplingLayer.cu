#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2019-2023, NVIDIA CORPORATION.  All rights reserved.
 * Copyright (c) 2021, NAVER Corp.  Authored by CLOVA.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <algorithm>
#include <float.h>

#include "src/fastertransformer/kernels/reduce_kernel_utils.cuh"
#include "src/fastertransformer/kernels/sampling_topk_kernels.h"
#include "src/fastertransformer/kernels/sampling_topp_kernels.h"
#include "src/fastertransformer/layers/sampling_layers/TopPSamplingLayer.h"
#include "src/fastertransformer/utils/logger.h"
#include "src/fastertransformer/utils/memory_utils.h"

namespace fastertransformer {

static __global__ void set_topp_runtime_args(int             batch_size,
                                             uint            top_k,
                                             uint*           top_ks,
                                             int             top_ks_size,
                                             float           top_p,
                                             float*          top_ps,
                                             int             top_ps_size,
                                             bool*           skip_decode,
                                             float*          initial_top_p_buf,
                                             float*          top_p_decay_buf,
                                             const float*    top_p_decay,
                                             float*          top_p_min_buf,
                                             const float*    top_p_min,
                                             int32_t*        top_p_reset_ids_buf,
                                             const uint32_t* top_p_reset_ids)
{
    /**
     * @brief Setup the runtime arguments for topp, broadcasting top_p to top_ps
                and top_k to top_ks, copying top_p_decay/top_p_min/top_p_reset_ids
                to internal buffers.
     *
     * \param batch_size            [batch_size]
     * \param op_k                  [batch_size]
     * \param top_ks                [batch_size]
     * \param top_ks_size           [batch_size]
     * \param top_p                 [batch_size]
     * \param top_ps                [batch_size]
     * \param top_ps_size           [batch_size]
     * \param skip_decode           [batch_size]
     * \param initial_top_p_buf     [batch_size]
     * \param top_p_decay_buf       [batch_size]
     * \param top_p_decay           [batch_size], optional, must between [0, 1]
     * \param top_p_min_buf         [batch_size]
     * \param top_p_min             [batch_size], optional, must between [0, 1]
     * \param top_p_reset_ids_buf    [batch_size]
     * \param top_p_reset_ids        [batch_size], optional
     *
     */

    int index = blockIdx.x * blockDim.x + threadIdx.x;
    for (int i = index; i < batch_size; i += gridDim.x * blockDim.x) {
        uint  k = top_ks_size > 1 ? top_ks[i] : top_k;
        float p = top_ps_size > 1 ? top_ps[i] : top_p;
        if (k == 0 && p == 0.0f) {
            // FT's topp implementation does not support topp = 0.0f, but it equivalent to greedy search.
            // So, we set the topk = 1 as an alternative solution.
            k = 1;
        }
        top_ks[i] = k;
        // Clip p value if it is out of range. range = [0.0, 1.0].
        top_ps[i] = p < 0.0f ? 0.0f : (p > 1.0f ? 1.0f : p);
        if (p < 0.0f || p > 1.0f) {
            printf("[WARNING] topp (%f) is out of range ([0.0, 1.0f]) for token %d"
                   " clip to closest number %f.\n",
                   p,
                   i,
                   top_ps[i]);
        }
        skip_decode[i] = k > 0;

        initial_top_p_buf[i] = top_ps[i];
        top_p_decay_buf[i]   = top_p_decay == nullptr ? 1.0f : top_p_decay[i];
        if (top_p_decay_buf[i] > 1.0f || top_p_decay_buf[i] <= 0.0f) {
            printf("[WARNING] top_p_decay_buf (%f) is out of range ([0.0, 1.0f]) for token %d,"
                   " change to 1.0f.\n",
                   top_p_decay_buf[i],
                   i);
            top_p_decay_buf[i] = 1.0f;
        }
        top_p_min_buf[i] = top_p_min == nullptr ? 1e-6f : top_p_min[i];  // prevent topp becoming 0.0
        if (top_p_min_buf[i] > 1.0f || top_p_min_buf[i] <= 0.0f) {
            printf("[WARNING] top_p_min_buf (%f) is out of range ([0.0, 1.0f]) for token %d,"
                   " change to 0.5f.\n",
                   top_p_min_buf[i],
                   i);
            top_p_min_buf[i] = 0.5f;
        }
        top_p_reset_ids_buf[i] = (int32_t)(top_p_reset_ids == nullptr ? -1 : top_p_reset_ids[i]);
    }
}

template<typename T>
void TopPSamplingLayer<T>::allocateBuffer()
{
    FT_CHECK(false);
}

template<typename T>
void TopPSamplingLayer<T>::allocateBuffer(size_t batch_size, Tensor top_k, Tensor top_p)
{
    FT_LOG_DEBUG(__PRETTY_FUNCTION__);
    BaseSamplingLayer<T>::allocateBuffer(batch_size, top_k, top_p);
    invokeTopPSampling<T>(nullptr,  // workspace
                          sampling_workspace_size_,
                          cub_temp_storage_size_,
                          nullptr,  // output_ids
                          nullptr,  // sequence_length
                          nullptr,  // finished_buffer
                          nullptr,  // cum_log_probs
                          nullptr,  // output_log_probs
                          nullptr,  // log_probs
                          topp_id_vals_buf_,
                          topp_offset_buf_,
                          begin_topp_offset_buf_,
                          curandstate_buf_,
                          batch_size,
                          vocab_size_padded_,
                          nullptr,
                          top_p.size() > 0 ? top_p.max<float>() : 0.0f,
                          stream_,
                          cuda_device_prop_,
                          skip_decode_buf_);
    sampling_workspace_ = allocator_->reMalloc(sampling_workspace_, sampling_workspace_size_, true);
    runtime_top_k_buf_ =
        reinterpret_cast<uint*>(allocator_->reMalloc(runtime_top_k_buf_, sizeof(uint) * batch_size, false));
    runtime_top_p_buf_ =
        reinterpret_cast<float*>(allocator_->reMalloc(runtime_top_p_buf_, sizeof(float) * batch_size, false));
    initial_top_p_buf_ =
        reinterpret_cast<float*>(allocator_->reMalloc(initial_top_p_buf_, sizeof(float) * batch_size, false));
    top_p_decay_buf_ =
        reinterpret_cast<float*>(allocator_->reMalloc(top_p_decay_buf_, sizeof(float) * batch_size, false));
    top_p_min_buf_ = reinterpret_cast<float*>(allocator_->reMalloc(top_p_min_buf_, sizeof(float) * batch_size, false));
    top_p_reset_ids_buf_ =
        reinterpret_cast<int32_t*>(allocator_->reMalloc(top_p_reset_ids_buf_, sizeof(int32_t) * batch_size, false));
    topp_id_vals_buf_ = reinterpret_cast<int*>(
        allocator_->reMalloc(topp_id_vals_buf_, sizeof(int) * batch_size * vocab_size_padded_, false));
    topp_offset_buf_ =
        reinterpret_cast<int*>(allocator_->reMalloc(topp_offset_buf_, sizeof(int) * (batch_size + 1), false));
    begin_topp_offset_buf_ =
        reinterpret_cast<int*>(allocator_->reMalloc(begin_topp_offset_buf_, sizeof(int) * (batch_size + 1), false));
    is_allocate_buffer_ = true;
}

template<typename T>
void TopPSamplingLayer<T>::freeBuffer()
{
    FT_LOG_DEBUG(__PRETTY_FUNCTION__);
    if (is_allocate_buffer_) {
        allocator_->free((void**)(&sampling_workspace_));
        allocator_->free((void**)(&topp_id_vals_buf_));
        allocator_->free((void**)(&topp_offset_buf_));
        allocator_->free((void**)(&begin_topp_offset_buf_));
        allocator_->free((void**)(&runtime_top_k_buf_));
        allocator_->free((void**)(&runtime_top_p_buf_));
        allocator_->free((void**)(&initial_top_p_buf_));
        allocator_->free((void**)(&top_p_decay_buf_));
        allocator_->free((void**)(&top_p_min_buf_));
        allocator_->free((void**)(&top_p_reset_ids_buf_));
    }
    BaseSamplingLayer<T>::freeBuffer();
    is_allocate_buffer_ = false;
}

template<typename T>
void TopPSamplingLayer<T>::setup(const size_t batch_size, const size_t beam_width, TensorMap* runtime_args)
{
    /**
    * @brief Set up the sampling layer for given runtime arguments.

    * runtime_args:
    *   \param  runtime_top_k [1] or [batch_size] on cpu, optional.
    *   \param  runtime_top_p [1] or [batch_size] on cpu, optional
    *   \param  temperature [1] or [batch_size] on cpu, optional
    *   \param  repetition_penalty [1] or [batch_size] on cpu, optional
    *   \param  top_p_decay [batch_size] on gpu, float, optional
    *   \param  top_p_min [batch_size] on gpu, float, optional
    *   \param  top_p_reset_ids [batch_size] on gpu, uint32, optional
    **/

    FT_LOG_DEBUG(__PRETTY_FUNCTION__);
    BaseSamplingLayer<T>::setup(batch_size, beam_width, runtime_args);
    const Tensor runtime_top_p = runtime_args->isExist("runtime_top_p") ? runtime_args->at("runtime_top_p") : Tensor();
    const size_t runtime_top_p_size = runtime_top_p.size();
    if (runtime_top_p_size == 0) {
        std::fill_n(skip_decode_, batch_size, true);
        return;
    }

    uint         tmp_top_k          = 0;
    const Tensor runtime_top_k      = runtime_args->isExist("runtime_top_k") ?
                                          runtime_args->at("runtime_top_k") :
                                          Tensor(MEMORY_CPU, TYPE_UINT32, {1}, &tmp_top_k);
    const size_t runtime_top_k_size = runtime_top_k.size();

    uint  top_k = runtime_top_k.getVal<uint>();
    float top_p = runtime_top_p.getVal<float>();

    if (runtime_top_k_size > 1) {
        FT_CHECK(runtime_top_k.size() == batch_size);
        cudaH2Dcpy(runtime_top_k_buf_, runtime_top_k.getPtr<uint>(), batch_size);
    }
    if (runtime_top_p_size > 1) {
        FT_CHECK(runtime_top_p.size() == batch_size);
        cudaH2Dcpy(runtime_top_p_buf_, runtime_top_p.getPtr<float>(), batch_size);
    }

    dim3 block(std::min((int)batch_size, 256));
    dim3 grid(div_up((int)batch_size, (int)block.x));

    const float*    top_p_decay     = runtime_args->getPtr<float>("top_p_decay", nullptr);
    const float*    top_p_min       = runtime_args->getPtr<float>("top_p_min", nullptr);
    const uint32_t* top_p_reset_ids = runtime_args->getPtr<uint32_t>("top_p_reset_ids", nullptr);
    set_topp_runtime_args<<<grid, block, 0, stream_>>>(batch_size,
                                                       top_k,
                                                       runtime_top_k_buf_,
                                                       runtime_top_k_size,
                                                       top_p,
                                                       runtime_top_p_buf_,
                                                       runtime_top_p_size,
                                                       skip_decode_buf_,
                                                       initial_top_p_buf_,
                                                       top_p_decay_buf_,
                                                       top_p_decay,
                                                       top_p_min_buf_,
                                                       top_p_min,
                                                       top_p_reset_ids_buf_,
                                                       top_p_reset_ids);
    sync_check_cuda_error();
    cudaAutoCpy(skip_decode_, skip_decode_buf_, batch_size, stream_);
    float* runtime_top_ps = new float[batch_size];
    cudaAutoCpy(runtime_top_ps, runtime_top_p_buf_, batch_size, stream_);
    runtime_max_top_p_ = *std::max_element(runtime_top_ps, runtime_top_ps + batch_size);
    delete[] runtime_top_ps;
}

template<typename T>
void TopPSamplingLayer<T>::runSampling(TensorMap* output_tensors, TensorMap* input_tensors)
{
    /**
    * input_tensors:
    *   \param  logits [local_batch_size, vocab_size_padded]
    *   \param  embedding_bias [vocab_size_padded], optional
    *   \param  step [1] on cpu
    *   \param  max_input_length [1] on cpu
    *   \param  input_lengths [local_batch_size], optional
    *   \param  ite [1] on cpu

    * output_tensors:
    *   \param  output_ids [max_seq_len, batch_size]
    *   \param  finished [local_batch_size], optional
    *   \param  sequence_length [local_batch_size], optional
    *   \param  cum_log_probs [batch_size], must be float*, optional
    *   \param  The cumultative log probability of generated tokens.
    *   \param  output_log_probs [local_batch_size], must be float*, optional
                    log probs at the current step.
    **/

    FT_LOG_DEBUG("%s start", __PRETTY_FUNCTION__);
    FT_CHECK(input_tensors->size() >= 4);
    FT_CHECK(output_tensors->size() >= 1);

    const int batch_size       = output_tensors->at("output_ids").shape[1];
    const int local_batch_size = input_tensors->at("logits").shape[0];
    const int step             = input_tensors->at("step").getVal<int>();
    const int ite              = input_tensors->at("ite").getVal<int>();

    // in case of skip any, the logit value is already copied and processed.
    T* logits = !skip_any_ ? input_tensors->at("logits").getPtr<T>() : runtime_logits_buf_;

    invokeTopPInitialize(
        topp_id_vals_buf_, topp_offset_buf_, begin_topp_offset_buf_, local_batch_size, vocab_size_padded_, stream_);
    sync_check_cuda_error();

    invokeAddBiasSoftMax(logits,
                         (T*)(nullptr),
                         input_tensors->at("end_id").getPtr<int>(),
                         output_tensors->at("finished", Tensor{MEMORY_GPU, TYPE_INVALID, {}, nullptr}).getPtr<bool>(),
                         local_batch_size,
                         vocab_size_padded_,
                         vocab_size_,
                         stream_);
    sync_check_cuda_error();

    float* cum_log_probs =
        output_tensors->isExist("cum_log_probs") ? output_tensors->at("cum_log_probs").getPtr<float>() : nullptr;
    float* output_log_probs =
        output_tensors->isExist("output_log_probs") ? output_tensors->at("output_log_probs").getPtr<float>() : nullptr;

    invokeBatchTopPSampling<T>(
        sampling_workspace_,
        sampling_workspace_size_,
        cub_temp_storage_size_,
        output_tensors->at("output_ids").getPtrWithOffset<int>(step * batch_size + ite * local_batch_size),
        output_tensors->at("sequence_length", Tensor{MEMORY_GPU, TYPE_INVALID, {}, nullptr}).getPtr<int>(),
        output_tensors->at("finished", Tensor{MEMORY_GPU, TYPE_INVALID, {}, nullptr}).getPtr<bool>(),
        cum_log_probs,
        output_log_probs,
        logits,
        topp_id_vals_buf_,
        topp_offset_buf_,
        begin_topp_offset_buf_,
        curandstate_buf_ + ite * local_batch_size,
        local_batch_size,
        vocab_size_padded_,
        input_tensors->at("end_id").getPtr<int>(),
        runtime_max_top_p_,
        runtime_top_p_buf_ + ite * local_batch_size,
        stream_,
        cuda_device_prop_,
        skip_decode_buf_ + ite * local_batch_size);
    sync_check_cuda_error();

    invokeComputeToppDecay(
        runtime_top_p_buf_ + ite * local_batch_size,
        initial_top_p_buf_ + ite * local_batch_size,
        output_tensors->getPtrWithOffset<int>("output_ids", step * batch_size + ite * local_batch_size),
        top_p_decay_buf_ + ite * local_batch_size,
        top_p_min_buf_ + ite * local_batch_size,
        top_p_reset_ids_buf_ + ite * local_batch_size,
        local_batch_size,
        stream_);
    sync_check_cuda_error();
    FT_LOG_DEBUG("%s stop", __PRETTY_FUNCTION__);
}

template<typename T>
TopPSamplingLayer<T>::TopPSamplingLayer(size_t             max_batch_size,
                                        size_t             vocab_size,
                                        size_t             vocab_size_padded,
                                        int                end_id,
                                        float              top_p,
                                        unsigned long long random_seed,
                                        float              temperature,
                                        float              len_penalty,
                                        float              repetition_penalty,
                                        hipStream_t       stream,
                                        cublasMMWrapper*   cublas_wrapper,
                                        IAllocator*        allocator,
                                        bool               is_free_buffer_after_forward,
                                        hipDeviceProp_t*    cuda_device_prop):
    BaseSamplingLayer<T>(max_batch_size,
                         vocab_size,
                         vocab_size_padded,
                         end_id,
                         0,
                         top_p,
                         random_seed,
                         temperature,
                         len_penalty,
                         repetition_penalty,
                         stream,
                         cublas_wrapper,
                         allocator,
                         is_free_buffer_after_forward,
                         cuda_device_prop)
{
}

template<typename T>
TopPSamplingLayer<T>::TopPSamplingLayer(TopPSamplingLayer<T> const& top_p_sampling_layer):
    BaseSamplingLayer<T>(top_p_sampling_layer)
{
}

template<typename T>
TopPSamplingLayer<T>::~TopPSamplingLayer()
{
    freeBuffer();
}

template class TopPSamplingLayer<float>;
template class TopPSamplingLayer<half>;

}  // namespace fastertransformer
