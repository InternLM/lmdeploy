// Copyright (c) OpenMMLab. All rights reserved.

#include "src/turbomind/kernels/gemm/gemm.h"
#include "src/turbomind/kernels/gemm/types.h"
#include "src/turbomind/models/llama/LlamaLinear.h"
#include "src/turbomind/utils/cuda_utils.h"

#include "src/turbomind/core/cuda_data_type.h"

namespace turbomind {

struct LlamaLinear::Impl {

    explicit Impl(hipStream_t stream): stream_(stream)
    {
        workspace_ = {};

        workspace_.barriers_size = gemm::Gemm::kBarriersSize;
        workspace_.partials_size = gemm::Gemm::kPartialsSize;

        check_cuda_error(hipMallocAsync(&workspace_.barriers, workspace_.barriers_size, stream_));
        check_cuda_error(hipMallocAsync(&workspace_.partials, workspace_.partials_size, stream_));
        check_cuda_error(hipMemsetAsync(workspace_.barriers, 0, workspace_.barriers_size, stream_));

        check_cuda_error(hipblasCreate(&cublas_));
        check_cuda_error(hipblasSetStream(cublas_, stream_));
        check_cuda_error(hipblasSetWorkspace(cublas_, workspace_.partials, workspace_.partials_size));

        if (0) {
            check_cuda_error(hipblasSetMathMode(cublas_, HIPBLAS_MATH_DISALLOW_REDUCED_PRECISION_REDUCTION));
        }
    }

    ~Impl()
    {
        hipblasDestroy(cublas_);
        hipFreeAsync(workspace_.barriers, stream_);
        hipFreeAsync(workspace_.partials, stream_);
        workspace_ = {};
    }

    void forward(Tensor& output, const Tensor& input, const LlamaDenseWeight& dense, Type type)
    {
        switch (dense.weight_type) {
            case kFloat16:
            case kFloat32:
            case kBfloat16:
                return forwardFp(output, input, dense.weight);
            case kUint4:
                return forwardInt4(output, input, dense, type);
            default:
                TM_CHECK(0) << "not implemented for weight type: " << dense.weight_type;
        }
    }

    void forwardFp(Ref<Tensor> output_, const Tensor& input, const Tensor& weight)
    {
        auto& output = output_.get();
        TM_CHECK_EQ(weight.ndim(), 2);
        TM_CHECK_EQ(input.ndim(), 2);
        TM_CHECK_EQ(output.ndim(), 2);

        int m, n, k;
        std::tie(k, m) = weight.shapes(0, 1);
        n              = input.shape(0);

        TM_CHECK_EQ(input.shape(1), k);
        TM_CHECK_EQ(output.shape(0), n);
        TM_CHECK_EQ(output.shape(1), m);

        // [k, m]
        hipblasOperation_t transa = weight.stride(1) == 1 ? HIPBLAS_OP_N : HIPBLAS_OP_T;
        // [n, k]
        hipblasOperation_t transb = input.stride(1) == 1 ? HIPBLAS_OP_N : HIPBLAS_OP_T;

        const float alpha = 1.f;
        const float beta  = 0.f;

        check_cuda_error(hipblasGemmEx(cublas_,
                                      transa,
                                      transb,
                                      m,
                                      n,
                                      k,
                                      &alpha,
                                      weight.raw_data(),
                                      to_cuda_dtype(weight.dtype()),
                                      weight.stride(0) * weight.stride(1),  // one of these is 1
                                      input.raw_data(),
                                      to_cuda_dtype(input.dtype()),
                                      input.stride(0) * input.stride(1),  // one of these is 1
                                      &beta,
                                      output.raw_data(),
                                      to_cuda_dtype(output.dtype()),
                                      output.stride(0) * output.stride(1),  // one of these is 1
                                      HIP_R_32F,
                                      CUBLAS_GEMM_DEFAULT_TENSOR_OP));
    }

    void forwardInt4(Tensor& output, const Tensor& input, const LlamaDenseWeight& dense, Type type)
    {
        TM_CHECK_EQ(output.ndim(), 2);  // A [m, k]
        TM_CHECK_EQ(input.ndim(), 2);   // C [m, n]

        TM_CHECK_EQ(input.stride(1), 1) << "input must be row-major";
        TM_CHECK_EQ(output.stride(1), 1) << "output must be row-major";

        TM_CHECK_EQ(output.shape(0), input.shape(0));
        TM_CHECK_EQ(input.shape(1), dense.input_dim);
        // TM_CHECK_EQ(output.shape(1), dense.output_dim);

        using namespace gemm;

        const Operation operation{dispatch_policy_,
                                  type == kFusedSiluFfn ? Epilogue::kGatedSilu : Epilogue::kNone,
                                  {QuantType::kNone},
                                  {QuantType::kDefault, dense.group_size},
                                  0,
                                  {},
                                  nullptr};

        const MatrixLayout a_desc{
            input.dtype(),
            kRowMajor,
            (int)input.shape(0),
            dense.input_dim,
            (int)input.stride(0),
        };

        const MatrixLayout c_desc{
            output.dtype(),  //
            kRowMajor,
            (int)output.shape(0),
            dense.output_dim,
            (int)output.stride(0),
            // type == kFusedSiluFfn ? (int)weight.output_dim / 2 : (int)weight.output_dim,
        };

        auto ec = gemm_.Run(operation,
                            1.f,
                            input.raw_data(),
                            a_desc,
                            nullptr,
                            {},
                            dense.weight.raw_data(),
                            dense.k_desc,
                            dense.scales_zeros.raw_data(),
                            dense.q_desc,
                            type == kFusedAdd ? 1.0f : 0.0f,
                            output.raw_data(),
                            c_desc,
                            output.raw_data(),
                            c_desc,
                            workspace_,
                            stream_);

        if (ec) {
            TM_LOG_ERROR("%s: %d", __PRETTY_FUNCTION__, ec);
        }
    }

    void forward_moe(Tensor&                 output,
                     const Tensor&           input,
                     const int*              indexes,
                     const int*              offsets,
                     const LlamaDenseWeight& dense,
                     Type                    type,
                     gemm::Context*          context)
    {
        using namespace gemm;

        QuantDesc quant_b{};
        if (dense.k_desc.type == kUint4) {
            quant_b.type       = QuantType::kDefault;
            quant_b.group_size = dense.group_size;
        }

        const Operation operation{dispatch_policy_,
                                  type == kFusedSiluFfn ? Epilogue::kGatedSilu : Epilogue::kNone,
                                  {QuantType::kNone},
                                  quant_b,
                                  0,
                                  context,
                                  nullptr};

        MatrixLayout a_desc{
            input.dtype(),
            kRowMajor,
            (int)output.shape(0),  // batch size
            dense.input_dim,       // k
            (int)input.stride(0),
        };

        a_desc.offsets = (int*)offsets;
        a_desc.idxs    = (int*)indexes;

        // std::cout << "m" << batch_size << "n" << weight.output_dims << "k" << weight.input_dims << " "
        //           << input_data.pitch << "\n";

        MatrixLayout c_desc{
            output.dtype(),  //
            kRowMajor,
            (int)output.shape(0),  // batch size
            dense.output_dim,
            (int)output.stride(0),
            // type == kFusedSiluFfn ? (int)weight.output_dims / 2 : (int)weight.output_dims,
        };

        c_desc.offsets = (int*)offsets;

        a_desc.num = c_desc.num = dense.k_desc.num;

        auto ec = gemm_.Run(operation,
                            1.f,
                            input.raw_data(),
                            a_desc,
                            nullptr,
                            {},
                            dense.weight.raw_data(),
                            dense.k_desc,
                            dense.scales_zeros.data_or((void*)nullptr),
                            dense.q_desc,
                            type == kFusedAdd ? 1.0f : 0.0f,
                            output.raw_data(),
                            c_desc,
                            output.raw_data(),
                            c_desc,
                            workspace_,
                            stream_);

        if (ec) {
            TM_LOG_ERROR("%s: %d", __PRETTY_FUNCTION__, ec);
        }
    }

    // cublasMMWrapper*     cublas_wrapper_;
    hipblasHandle_t       cublas_;
    gemm::Gemm           gemm_;
    gemm::DispatchPolicy dispatch_policy_{gemm::DispatchPolicy::kDefault};
    hipStream_t         stream_{};

    gemm::Workspace workspace_;
};

LlamaLinear::LlamaLinear(hipStream_t stream): impl_{std::make_shared<Impl>(stream)} {}

Tensor LlamaLinear::forward(const Tensor&           input,  //
                            const LlamaDenseWeight& dense,
                            Type                    type,
                            std::optional<Tensor>   output)
{
    ssize_t output_dim = type == kFusedSiluFfn ? dense.output_dim / 2 : dense.output_dim;

    Tensor in = input.view({-1, input.shape(-1)});
    Tensor out;

    if (output) {
        out = output->view({in.shape(0), output_dim});
    }
    else {
        out = Tensor({in.shape(0), output_dim}, input.dtype(), input.device());
    }

    impl_->forward(out, in, dense, type);

    auto shape   = input.shape();
    shape.back() = out.shape(-1);

    return out.view(shape);
}

void LlamaLinear::forward_moe(Tensor&                 output,
                              const Tensor&           input,
                              const int*              indexes,
                              const int*              offsets,
                              const LlamaDenseWeight& dense,
                              Type                    type,
                              gemm::Context*          context)
{
    return impl_->forward_moe(output, input, indexes, offsets, dense, type, context);
}

void LlamaLinear::set_measure(bool measure)
{
    impl_->dispatch_policy_ = measure ? gemm::DispatchPolicy::kMeasure : gemm::DispatchPolicy::kReuse;
}

int LlamaLinear::Export(std::ostream& os)
{
    if (os) {
        return impl_->gemm_.Export(os);
    }
    return 0;
}

int LlamaLinear::Import(std::istream& is)
{
    auto n_records = 0;
    if (is) {
        n_records = impl_->gemm_.Import(is);
    }
    if (n_records) {
        impl_->dispatch_policy_ = gemm::DispatchPolicy::kReuse;
    };
    return n_records;
}

std::vector<int> LlamaLinear::GetTuningSeq() const
{
    return impl_->gemm_.GetTuningSeq();
}

}  // namespace turbomind
