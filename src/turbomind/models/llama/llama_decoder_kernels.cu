#include "hip/hip_runtime.h"
// Copyright (c) OpenMMLab. All rights reserved.

#include "src/turbomind/macro.h"
#include "src/turbomind/models/llama/llama_decoder_kernels.h"
#include "src/turbomind/utils/cuda_utils.h"
#include <hip/hip_cooperative_groups.h>
#include <cooperative_groups/reduce.h>
#include <hip/hip_fp16.h>

namespace cg = cooperative_groups;

namespace turbomind {

template<typename T>
struct res_norm_ops_t {
};

template<typename T>
struct res_norm_t {
    res_norm_ops_t<T> f;
    __device__ uint4  addvec(const uint4& a, const uint4& b, const uint4& bias, float& accum) const
    {
        uint4 c;
        c.x = f.cast(f.add(f.cast(a.x), f.cast(b.x), f.cast(bias.x), accum));
        c.y = f.cast(f.add(f.cast(a.y), f.cast(b.y), f.cast(bias.y), accum));
        c.z = f.cast(f.add(f.cast(a.z), f.cast(b.z), f.cast(bias.z), accum));
        c.w = f.cast(f.add(f.cast(a.w), f.cast(b.w), f.cast(bias.w), accum));
        return c;
    }
    __device__ uint4 normvec(const uint4& u, const uint4& s, float factor) const
    {
        uint4 v;
        v.x = f.cast(f.norm(f.cast(u.x), f.cast(s.x), factor));
        v.y = f.cast(f.norm(f.cast(u.y), f.cast(s.y), factor));
        v.z = f.cast(f.norm(f.cast(u.z), f.cast(s.z), factor));
        v.w = f.cast(f.norm(f.cast(u.w), f.cast(s.w), factor));
        return v;
    }
};

template<>
struct res_norm_ops_t<half> {
    __device__ float2 cast(const uint& x) const
    {
        return __half22float2(reinterpret_cast<const half2&>(x));
    }
    __device__ uint cast(const float2& x) const
    {
        auto y = __float22half2_rn(x);
        return reinterpret_cast<uint&>(y);
    }
    __device__ float2 add(const float2& a, const float2& b, const float2& bias, float& accum) const
    {
        float2 c{a.x + b.x + bias.x, a.y + b.y + bias.y};
        accum += c.x * c.x + c.y * c.y;
        return c;
    }
    __device__ float2 norm(const float2& a, const float2& s, float factor) const
    {
        return {a.x * s.x * factor, a.y * s.y * factor};
    }
};

template<>
struct res_norm_ops_t<float> {
    __device__ float cast(const uint& x) const
    {
        return reinterpret_cast<const float&>(x);
    }
    __device__ uint cast(const float& x) const
    {
        return reinterpret_cast<const uint&>(x);
    }
    __device__ float add(const float& a, const float& b, const float& bias, float& accum) const
    {
        float c = a + b + bias;
        accum += c * c;
        return c;
    }
    __device__ float norm(const float& a, const float& s, float factor) const
    {
        return a * s * factor;
    }
};

template<typename T>
__device__ T blockReduceSum(const cg::thread_block& block, T value)
{
    __shared__ float partial[32];

    auto tile = cg::tiled_partition<32>(block);
    value     = cg::reduce(tile, value, cg::plus<float>{});

    if (tile.thread_rank() == 0) {
        partial[tile.meta_group_rank()] = value;
    }

    block.sync();

    value = tile.thread_rank() < tile.meta_group_size() ? partial[tile.thread_rank()] : T{};
    return cg::reduce(tile, value, cg::plus<float>{});
}

template<typename T>
__global__ void fusedAddBiasResidualNorm(T* __restrict__ r_data,
                                         T* __restrict__ x_data,
                                         const T* __restrict__ bias,
                                         const T* __restrict__ scale,
                                         float eps,
                                         int   batch_size,
                                         int   n_dims)
{
    auto block = cg::this_thread_block();
    auto grid  = cg::this_grid();

    constexpr int PACK_DIM = sizeof(uint4) / sizeof(T);

    const auto batch_idx            = block.group_index().x;
    uint4* __restrict__ r_ptr       = reinterpret_cast<uint4*>(r_data + batch_idx * n_dims);
    uint4* __restrict__ x_ptr       = reinterpret_cast<uint4*>(x_data + batch_idx * n_dims);
    const uint4* __restrict__ b_ptr = reinterpret_cast<const uint4*>(bias);

    res_norm_t<T> ops;

    float thread_sum{};
    for (auto i = block.thread_rank(); i < n_dims / PACK_DIM; i += block.size()) {
        auto  r  = r_ptr[i];
        auto  x  = x_ptr[i];
        uint4 b  = b_ptr ? b_ptr[i] : uint4{};
        r        = ops.addvec(r, x, b, thread_sum);
        r_ptr[i] = r;
    }

    auto total_sum = blockReduceSum(block, thread_sum);

    float s_inv_mean = rsqrt(total_sum / n_dims + eps);

    const uint4* __restrict__ s_ptr = reinterpret_cast<const uint4*>(scale);
    for (uint i = block.thread_rank(); i < n_dims / PACK_DIM; i += block.size()) {
        auto r   = r_ptr[i];
        auto s   = s_ptr[i];
        auto o   = ops.normvec(r, s, s_inv_mean);
        x_ptr[i] = o;
    }
}

template<typename T>
void invokeFusedAddBiasResidualRMSNorm(
    T* residual, T* in_out, const T* bias, const T* scale, float eps, int batch_size, int n_dims, hipStream_t stream)
{
    constexpr int PACK_DIM = sizeof(uint4) / sizeof(T);
    FT_CHECK(n_dims % PACK_DIM == 0);
    const int n_pack    = n_dims / PACK_DIM;
    const int n_iter    = ((n_pack + 1023) / 1024);        // iterations when block size == 1024
    int       n_threads = (n_pack + n_iter - 1) / n_iter;  // adjust block size to avoid tail effect
    n_threads           = (n_threads + 31) / 32 * 32;      // round up to the nearest multiple of warp size

    fusedAddBiasResidualNorm<<<batch_size, n_threads, 0, stream>>>(
        residual, in_out, bias, scale, eps, batch_size, n_dims);
}

template void
invokeFusedAddBiasResidualRMSNorm(float*, float*, const float*, const float*, float, int, int, hipStream_t);
template void invokeFusedAddBiasResidualRMSNorm(half*, half*, const half*, const half*, float, int, int, hipStream_t);

}  // namespace turbomind
