#include "hip/hip_runtime.h"
// Copyright (c) OpenMMLab. All rights reserved.

#include "src/turbomind/kernels/decoder_masked_multihead_attention_utils.h"
#include "src/turbomind/kernels/decoder_multihead_attention/array_ops.h"
#include "src/turbomind/kernels/gemm_s_f16/common.h"
#include "src/turbomind/kernels/reduce_kernel_utils.cuh"
#include "src/turbomind/macro.h"
#include "src/turbomind/models/llama/llama_kernels.h"
#include "src/turbomind/models/llama/llama_utils.h"
#include "src/turbomind/utils/cuda_type_utils.cuh"
#include "src/turbomind/utils/cuda_utils.h"
#include "src/turbomind/utils/dispatch.h"
#include "src/turbomind/utils/logger.h"
#include <algorithm>
#include <cstdint>
#include <cub/block/block_reduce.cuh>
#include <type_traits>
#include <utility>

namespace turbomind {

// fp16, bf16
// n is divided by 2 for this impl
template<typename T>
__global__ void rootMeanSquareNorm(T* out, const T* input, const T* scale, float eps, int m, int n)
{
    using T2 = typename TypeConverter<T>::Type;
    __shared__ float s_inv_mean;
    float            mean = 0.f;

    T2*       out_ptr   = (T2*)out;
    const T2* input_ptr = (const T2*)input;
    const T2* scale_ptr = (const T2*)scale;

    for (uint idx = threadIdx.x; idx < n; idx += blockDim.x) {
        float2 tmp2 = cuda_cast<float2>(input_ptr[blockIdx.x * n + idx]);
        mean += tmp2.x * tmp2.x;
        mean += tmp2.y * tmp2.y;
    }

    mean = blockReduceSum<float>(mean);
    if (threadIdx.x == 0) {
        s_inv_mean = rsqrt(.5f * mean / (float)n + eps);
    }
    __syncthreads();

    for (uint idx = threadIdx.x; idx < n; idx += blockDim.x) {
        float2 tmp2                   = cuda_cast<float2>(input_ptr[blockIdx.x * n + idx]);
        float2 sca2                   = cuda_cast<float2>(scale_ptr[idx]);
        tmp2.x                        = tmp2.x * s_inv_mean * sca2.x;
        tmp2.y                        = tmp2.y * s_inv_mean * sca2.y;
        out_ptr[blockIdx.x * n + idx] = cuda_cast<T2>(tmp2);
    }
}

template<>
__global__ void rootMeanSquareNorm(float* out, const float* input, const float* scale, float eps, int m, int n)
{
    __shared__ float s_inv_mean;
    float            mean = 0.f;

    for (uint idx = threadIdx.x; idx < n; idx += blockDim.x) {
        float tmp = input[blockIdx.x * n + idx];
        mean += tmp * tmp;
    }

    mean = blockReduceSum<float>(mean);
    if (threadIdx.x == 0) {
        s_inv_mean = rsqrt(mean / static_cast<float>(n) + eps);
    }
    __syncthreads();

    for (uint idx = threadIdx.x; idx < n; idx += blockDim.x) {
        float tmp                 = input[blockIdx.x * n + idx];
        out[blockIdx.x * n + idx] = tmp * s_inv_mean * scale[idx];
    }
}

template<typename T>
void invokeRootMeanSquareNorm(T* out, const T* input, const T* scale, float eps, int m, int n, hipStream_t stream)
{
    if (sizeof(T) == 2) {
        FT_CHECK(n % 2 == 0);
        n /= 2;
    }
    dim3 grid(m);
    dim3 block(std::min(n, 1024));
    rootMeanSquareNorm<<<grid, block, 0, stream>>>(out, input, scale, eps, m, n);
}

template void invokeRootMeanSquareNorm(float*, const float*, const float*, float, int, int, hipStream_t);
template void invokeRootMeanSquareNorm(half*, const half*, const half*, float, int, int, hipStream_t);
#ifdef ENABLE_BF16
template void
invokeRootMeanSquareNorm(__hip_bfloat16*, const __hip_bfloat16*, const __hip_bfloat16*, float, int, int, hipStream_t);
#endif

// #ifdef ENABLE_BF16

// template void invokeRootMeanSquareNorm(__hip_bfloat16*, const __hip_bfloat16*, float, int, int, hipStream_t);

// #endif

template<typename T, typename T0>
__device__ T saturate_cast(T0 x)
{
    return x;
}

template<>
__device__ half saturate_cast<half, float>(float x)
{
    return (x > 64512.f || x < -64512.f) ? (x > 0.f ? 64512.f : -64512.f) : x;
}

template<typename T>
__global__ void addResidual(T* out, const T* in, size_t n)
{
    auto idx = threadIdx.x + (size_t)blockIdx.x * blockDim.x;
    if (idx < n) {
        out[idx] = static_cast<T>(static_cast<float>(out[idx]) + static_cast<float>(in[idx]));
    }
}

template<typename T>
void invokeAddResidual(T* out, const T* in, int m, int n, hipStream_t stream)
{
    auto total = static_cast<size_t>(m) * n;
    dim3 block(std::min((unsigned long)total, 1024UL));
    dim3 grid((total + block.x - 1) / block.x);

    addResidual<<<grid, block, 0, stream>>>(out, in, total);
}

template void invokeAddResidual(float*, const float*, int, int, hipStream_t);
template void invokeAddResidual(half*, const half*, int, int, hipStream_t);

// ids [seq_len, batch_size]
// input_ids [batch_size, max_input_len]
__global__ void
fixInputIds(int* ids, const int* input_ids, const int* input_lengths, int batch_size, int seq_len, int max_input_len)
{
    int seq_id   = threadIdx.x;
    int batch_id = blockIdx.x;
    for (; seq_id < input_lengths[batch_id]; seq_id += blockDim.x) {
        ids[seq_id * batch_size + batch_id] = input_ids[batch_id * max_input_len + seq_id];
    }
}

void invokeFixInputIds(int*         ids,
                       const int*   input_ids,
                       const int*   input_lengths,
                       int          batch_size,
                       int          seq_len,
                       int          max_input_len,
                       hipStream_t st)
{
    dim3 block(std::min(1024, max_input_len));
    dim3 grid(batch_size);
    fixInputIds<<<grid, block, 0, st>>>(ids, input_ids, input_lengths, batch_size, seq_len, max_input_len);
}

template<typename T>
__global__ void sliceCausalMask(T* mask, int seq_len, int key_len, int step)
{
    mask += (size_t)blockIdx.x * seq_len * key_len;
    for (int i = threadIdx.x; i < seq_len * key_len; i += blockDim.x) {
        int row = i / key_len;
        int col = i % key_len;
        if (col <= row + step) {
            mask[i] = static_cast<T>(1.f);
        }
        else {
            mask[i] = static_cast<T>(0.f);
        }
    }
}

// [step: step+Q, :] of the K*K causal mask
template<typename T>
void invokeSliceCausalMask(T* mask, int seq_len, int key_len, int step, int batch_size, hipStream_t stream)
{
    FT_CHECK(step == key_len - seq_len);
    sliceCausalMask<<<batch_size, 256, 0, stream>>>(mask, seq_len, key_len, step);
}

template void invokeSliceCausalMask(half*, int, int, int, int, hipStream_t);
template void invokeSliceCausalMask(float*, int, int, int, int, hipStream_t);

// mask [bsz, max_q_len, max_k_len]

template<typename T>
__global__ void createCausalMasks(T* mask, const int* q_lens, const int* k_lens, int max_q_len, int max_k_len)
{
    const auto q_len = q_lens[blockIdx.x];
    const auto k_len = k_lens[blockIdx.x];
    mask += blockIdx.x * max_q_len * max_k_len;
    for (int i = threadIdx.x; i < max_q_len * max_k_len; i += blockDim.x) {
        const int q        = i / max_k_len;  // [0, max_q_len)
        const int k        = i % max_k_len;  // [0, max_k_len)
        bool      is_valid = q < q_len && k < k_len && k <= q + (k_len - q_len);
        mask[i]            = static_cast<T>(is_valid);
    }
}

template<typename T>
void invokeCreateCausalMasks(
    T* mask, const int* q_lens, const int* k_lens, int max_q_len, int max_k_len, int batch_size, hipStream_t stream)
{
    createCausalMasks<<<batch_size, 512, 0, stream>>>(mask, q_lens, k_lens, max_q_len, max_k_len);
}

template void invokeCreateCausalMasks(float* mask, const int*, const int*, int, int, int, hipStream_t);
template void invokeCreateCausalMasks(half* mask, const int*, const int*, int, int, int, hipStream_t);
#ifdef ENABLE_BF16
template<>
__global__ void createCausalMasks<__hip_bfloat16>(
    __hip_bfloat16* mask, const int* q_lens, const int* k_lens, int max_q_len, int max_k_len)
{
    const auto q_len = q_lens[blockIdx.x];
    const auto k_len = k_lens[blockIdx.x];
    mask += blockIdx.x * max_q_len * max_k_len;
    for (int i = threadIdx.x; i < max_q_len * max_k_len; i += blockDim.x) {
        const int q        = i / max_k_len;  // [0, max_q_len)
        const int k        = i % max_k_len;  // [0, max_k_len)
        bool      is_valid = q < q_len && k < k_len && k <= q + (k_len - q_len);
        mask[i]            = static_cast<__hip_bfloat16>(float(is_valid));
    }
}
template void invokeCreateCausalMasks(__hip_bfloat16* mask, const int*, const int*, int, int, int, hipStream_t);
#endif

template<typename Ti, typename To>
struct ExtendKvCache {

    static constexpr int MaxElemSize = std::max(sizeof(Ti), sizeof(To));
    static constexpr int X_ELEMS     = 16 / MaxElemSize;

    using Vi = Array<Ti, X_ELEMS>;
    using Vo = Array<To, X_ELEMS>;

    using Transform = ConvertKvCache<Ti, To>;

    struct Params {
        To**       k_dst_ptrs;
        To**       v_dst_ptrs;
        const Ti*  k_src;
        const Ti*  v_src;
        const int* cu_block_counts;
        const int* query_length;
        const int* context_length;
        int        block_length;
        size_t     dst_layer_offset;
        int        max_q_len;
        int        head_num;
        int        head_dim;
        Transform  transform_k;
        Transform  transform_v;
    };

    __device__ void operator()(const Params& params) const
    {
        const int batch_id = blockIdx.y;

        const int query_len    = params.query_length[batch_id];
        const int history_len  = params.context_length[batch_id] - query_len;
        const int cu_block_cnt = params.cu_block_counts[batch_id];

        const int head_id = blockIdx.z;

        const int size_per_head_div_x = params.head_dim / X_ELEMS;
        const int idx                 = blockIdx.x * blockDim.x + threadIdx.x;
        const int head_size_id        = idx % size_per_head_div_x;
        const int seq_len_id          = idx / size_per_head_div_x;

        const int cache_block_index  = (seq_len_id + history_len) / params.block_length;
        const int cache_block_offset = (seq_len_id + history_len) % params.block_length;

        const auto k_val_src = params.k_src;
        const auto v_val_src = params.v_src;

        const auto k_val_dst = (params.k_dst_ptrs + cu_block_cnt)[cache_block_index] + params.dst_layer_offset;
        const auto v_val_dst = (params.v_dst_ptrs + cu_block_cnt)[cache_block_index] + params.dst_layer_offset;

        if (seq_len_id < query_len) {
            // [B, H, s, D/x] -> [H, S[t:t+s], D/x]
            const int64_t dst_idx = head_id * params.block_length * size_per_head_div_x +  // H
                                    cache_block_offset * size_per_head_div_x +             // s + offset
                                    head_size_id;                                          // D/x

            const int64_t src_idx = batch_id * params.head_num * params.max_q_len * size_per_head_div_x +  // B
                                    head_id * params.max_q_len * size_per_head_div_x +                     // H
                                    seq_len_id * size_per_head_div_x +                                     // s
                                    head_size_id;                                                          // D/x

            Vi k_vi;
            Vi v_vi;

            Ldg(k_vi, k_val_src + src_idx * X_ELEMS);
            Ldg(v_vi, v_val_src + src_idx * X_ELEMS);

            Vo k_vo = params.transform_k(k_vi);
            Vo v_vo = params.transform_v(v_vi);

            Store(k_val_dst + dst_idx * X_ELEMS, k_vo);
            Store(v_val_dst + dst_idx * X_ELEMS, v_vo);
        }
    }
};

namespace {

template<class Kernel, class Params>
__global__ void KernelWrapper(Params params)
{
    Kernel{}(params);
};

}  // namespace

template<typename T>
void invokeExtendKVCache(void**       k_dst_ptrs,
                         void**       v_dst_ptrs,
                         const T*     k_src,
                         const T*     v_src,
                         const int*   cu_block_counts,
                         const int*   query_length,
                         const int*   context_length,
                         int          batch_size,
                         int          block_length,
                         size_t       dst_layer_offset,
                         int          max_q_len,
                         int          head_dim,
                         int          head_num,
                         int          quant,
                         const float* kv_params,
                         hipStream_t stream)
{
    constexpr int block_sz = 128;

    auto fn = [&](auto value) {
        using Tout   = decltype(value);
        using Kernel = ExtendKvCache<T, Tout>;

        dim3 grid((max_q_len * head_dim / Kernel::X_ELEMS + block_sz - 1) / block_sz, batch_size, head_num);

        typename Kernel::Params params{(Tout**)k_dst_ptrs,
                                       (Tout**)v_dst_ptrs,
                                       k_src,
                                       v_src,
                                       cu_block_counts,
                                       query_length,
                                       context_length,
                                       block_length,
                                       dst_layer_offset,
                                       max_q_len,
                                       head_num,
                                       head_dim,
                                       {kv_params[0], kv_params[1]},
                                       {kv_params[2], kv_params[3]}};

        KernelWrapper<Kernel><<<grid, block_sz, 0, stream>>>(params);
    };

    (quant & QuantPolicy::kCacheKVInt8) ? fn(int8_t{}) : fn(T{});
}

template void invokeExtendKVCache(void**       k_dst_ptrs,
                                  void**       v_dst_ptrs,
                                  const float* k_src,
                                  const float* v_src,
                                  const int*   cu_block_counts,
                                  const int*   query_length,
                                  const int*   history_length,
                                  int          batch_size,
                                  int          block_length,
                                  size_t       dst_layer_offset,
                                  int          max_q_len,
                                  int          head_dim,
                                  int          head_num,
                                  int          quant,
                                  const float* kv_scale,
                                  hipStream_t stream);

template void invokeExtendKVCache(void**       k_dst_ptrs,
                                  void**       v_dst_ptrs,
                                  const half*  k_src,
                                  const half*  v_src,
                                  const int*   cu_block_counts,
                                  const int*   query_length,
                                  const int*   history_length,
                                  int          batch_size,
                                  int          block_length,
                                  size_t       dst_layer_offset,
                                  int          max_q_len,
                                  int          head_dim,
                                  int          head_num,
                                  int          quant,
                                  const float* kv_scale,
                                  hipStream_t stream);
#ifdef ENABLE_BF16
template void invokeExtendKVCache(void**               k_dst_ptrs,
                                  void**               v_dst_ptrs,
                                  const __hip_bfloat16* k_src,
                                  const __hip_bfloat16* v_src,
                                  const int*           cu_block_counts,
                                  const int*           query_length,
                                  const int*           history_length,
                                  int                  batch_size,
                                  int                  block_length,
                                  size_t               dst_layer_offset,
                                  int                  max_q_len,
                                  int                  head_dim,
                                  int                  head_num,
                                  int                  quant,
                                  const float*         kv_scale,
                                  hipStream_t         stream);
#endif

template<typename Ti, typename To>
struct TransposeKvCache {
    static constexpr int MaxElemSize = std::max(sizeof(Ti), sizeof(To));
    static constexpr int X_ELEMS     = 16 / MaxElemSize;

    using Vi = Array<Ti, X_ELEMS>;
    using Vo = Array<To, X_ELEMS>;

    using Transform = ConvertKvCache<Ti, To>;

    struct Params {
        To*        k_dst;
        To*        v_dst;
        const Ti** k_src;
        const Ti** v_src;
        size_t     src_offset;
        int        head_num;
        int        head_n_rep;
        int        size_per_head;
        const int* seq_length;
        int        max_kv_len;
        int        max_seq_len;
        Transform  transform_k;
        Transform  transform_v;
        // float      k_scale;
        // float      k_zp;
        // float      v_scale;
        // float      v_zp;
    };

    __device__ void operator()(const Params& params) const
    {
        const int batch_id = blockIdx.y;
        const int head_id  = blockIdx.z;

        const int idx                 = blockIdx.x * blockDim.x + threadIdx.x;
        const int size_per_head_div_x = params.size_per_head / X_ELEMS;

        const auto k_src = params.k_src[batch_id] + params.src_offset;
        const auto v_src = params.v_src[batch_id] + params.src_offset;
        const auto k_dst = params.k_dst;
        const auto v_dst = params.v_dst;

        const auto seq_len = params.seq_length[batch_id];

        const int v_head_size_id = idx % size_per_head_div_x;
        const int v_seq_len_id   = idx / size_per_head_div_x;

        if (v_seq_len_id < seq_len) {
            // [B, H, s, D/x] <- [B, H, S[:s], D/x]
            const int64_t src_idx = head_id / params.head_n_rep * size_per_head_div_x * params.max_seq_len +  // H
                                    v_seq_len_id * size_per_head_div_x +                                      // s
                                    v_head_size_id;                                                           // D/x

            const int64_t dst_idx = batch_id * params.head_num * size_per_head_div_x * params.max_kv_len +  // B
                                    head_id * size_per_head_div_x * params.max_kv_len +                     // H
                                    v_seq_len_id * size_per_head_div_x +                                    // s
                                    v_head_size_id;                                                         // D/x

            Vi k_vi;
            Vi v_vi;

            Ldg(k_vi, k_src + src_idx * X_ELEMS);
            Ldg(v_vi, v_src + src_idx * X_ELEMS);

            Vo k_vo = params.transform_k(k_vi);
            Vo v_vo = params.transform_v(v_vi);

            Store(k_dst + dst_idx * X_ELEMS, k_vo);
            Store(v_dst + dst_idx * X_ELEMS, v_vo);
        }
    }
};

template<typename T>
void invokeTransposeKVCache(T*           key_cache_trans,
                            T*           val_cache_trans,
                            const T**    key_cache,
                            const T**    val_cache,
                            size_t       src_offset,
                            int          batch_size,
                            const int*   key_length,
                            int          max_kv_len,
                            int          max_seq_len,
                            int          size_per_head,
                            int          head_num,
                            int          head_n_rep,
                            hipStream_t stream,
                            int          quant,
                            const float* kv_params)
{
    constexpr int block_sz = 128;

    auto fn = [&](auto value) {
        using Tin    = decltype(value);
        using Kernel = TransposeKvCache<Tin, T>;

        dim3 grid((max_kv_len * size_per_head / Kernel::X_ELEMS + block_sz - 1) / block_sz, batch_size, head_num);

        typename Kernel::Params params{key_cache_trans,
                                       val_cache_trans,
                                       (const Tin**)key_cache,
                                       (const Tin**)val_cache,
                                       src_offset,
                                       head_num,
                                       head_n_rep,
                                       size_per_head,
                                       key_length,
                                       max_kv_len,
                                       max_seq_len,
                                       {kv_params[0], kv_params[1]},
                                       {kv_params[2], kv_params[3]}};

        KernelWrapper<Kernel><<<grid, block_sz, 0, stream>>>(params);
    };

    (quant & QuantPolicy::kCacheKVInt8) ? fn(int8_t{}) : fn(T{});
}

template void invokeTransposeKVCache(float*,
                                     float*,
                                     const float**,
                                     const float**,
                                     size_t,
                                     int,
                                     const int*,
                                     int,
                                     int,
                                     int,
                                     int,
                                     int,
                                     hipStream_t stream,
                                     int,
                                     const float*);
template void invokeTransposeKVCache(half*,
                                     half*,
                                     const half**,
                                     const half**,
                                     size_t,
                                     int,
                                     const int*,
                                     int,
                                     int,
                                     int,
                                     int,
                                     int,
                                     hipStream_t stream,
                                     int,
                                     const float*);
#ifdef ENABLE_BF16
template void invokeTransposeKVCache(__hip_bfloat16*,
                                     __hip_bfloat16*,
                                     const __hip_bfloat16**,
                                     const __hip_bfloat16**,
                                     size_t,
                                     int,
                                     const int*,
                                     int,
                                     int,
                                     int,
                                     int,
                                     int,
                                     hipStream_t stream,
                                     int,
                                     const float*);
#endif

__global__ void gatherOutput(int*       output_ids,
                             const int* ids,
                             const int* context_length,
                             int        max_context_len,
                             int        max_gen_step,
                             int        max_output_len,
                             int        batch_size)
{
    const int batch_id    = blockIdx.x;
    const int context_len = context_length[batch_id];
    output_ids += batch_id * max_output_len;
    for (int src_idx = threadIdx.x; src_idx < max_gen_step; src_idx += blockDim.x) {
        // skip padding for src
        if (context_len <= src_idx && src_idx < max_context_len) {
            continue;
        }
        // skip padding for dst
        const int dst_idx = src_idx < context_len ? src_idx : src_idx - (max_context_len - context_len);
        if (dst_idx < max_output_len) {
            output_ids[dst_idx] = ids[src_idx * batch_size + batch_id];
        }
    }
}

void invokeGatherOutput(int*         output_ids,
                        const int*   ids,
                        const int*   context_length,
                        int          max_context_len,
                        int          max_gen_step,
                        int          max_output_len,
                        int          batch_size,
                        hipStream_t stream)
{
    int block_size = 128;
    int grid_size  = batch_size;
    gatherOutput<<<grid_size, block_size, 0, stream>>>(
        output_ids, ids, context_length, max_context_len, max_gen_step, max_output_len, batch_size);
}

__global__ void updateOutput(int**      request_output_ids_ptrs,
                             int**      request_seqlen_ptrs,
                             const int* output_ids,
                             const int* sequence_lengths,
                             const int* request_output_ids_lens,
                             int        max_session_len,
                             bool       token_generated)
{
    const int batch_id = blockIdx.x;

    auto request_output_ids = request_output_ids_ptrs[batch_id];
    auto request_seqlen     = request_seqlen_ptrs[batch_id];

    output_ids += max_session_len * batch_id;

    const int seqlen     = sequence_lengths[batch_id] + (int)token_generated;
    const int output_len = min(seqlen, request_output_ids_lens[batch_id]);

    for (int i = threadIdx.x; i < output_len; i += blockDim.x) {
        request_output_ids[i] = output_ids[i];
    }

    *request_seqlen = seqlen;
}

void invokeUpdateOutput(int**        request_output_ids_ptrs,
                        int**        request_seqlen_ptrs,
                        const int*   output_ids,
                        const int*   sequence_lengths,
                        const int*   request_output_ids_lens,
                        int          max_session_len,
                        bool         token_generated,
                        int          batch_size,
                        hipStream_t stream)
{
    constexpr int block_size = 128;
    const int     grid_size  = batch_size;

    updateOutput<<<grid_size, block_size, 0, stream>>>(request_output_ids_ptrs,
                                                       request_seqlen_ptrs,
                                                       output_ids,
                                                       sequence_lengths,
                                                       request_output_ids_lens,
                                                       max_session_len,
                                                       token_generated);
}

template<int BLOCK_DIM>
__global__ void compactOutputIds(
    int* cu_output_ids, const int* output_ids, const int* sequence_lengths, int session_len, bool token_generated)
{
    typedef hipcub::BlockReduce<int, BLOCK_DIM>     BlockReduce;
    __shared__ typename BlockReduce::TempStorage temp_storage;

    const int batch_idx = blockIdx.x;

    int end   = (batch_idx + BLOCK_DIM - 1) / BLOCK_DIM * BLOCK_DIM;  // align to BLOCK_DIM boundary
    int count = 0;
    for (int i = threadIdx.x; i < end; i += blockDim.x) {
        int x = threadIdx.x < batch_idx ? sequence_lengths[threadIdx.x] : 0;
        count += BlockReduce(temp_storage).Sum(x);
        // https://nvlabs.github.io/cub/classcub_1_1_block_reduce.html
        __syncthreads();
    }

    __shared__ int offset;

    if (threadIdx.x == 0) {
        offset = count;
    }

    __syncthreads();

    auto dst = cu_output_ids + offset;

    const int seq_len = sequence_lengths[batch_idx];

    for (int i = threadIdx.x; i < seq_len; i += blockDim.x) {
        dst[i] = output_ids[batch_idx * session_len + i];
    }
}

void invokeCompactOutputIds(int*         cu_output_ids,
                            const int*   output_ids,
                            const int*   sequence_lengths,
                            int          max_session_len,
                            bool         token_generated,
                            int          batch_size,
                            hipStream_t stream)
{
    constexpr int BLOCK_DIM = 128;
    compactOutputIds<BLOCK_DIM><<<batch_size, BLOCK_DIM, 0, stream>>>(
        cu_output_ids, output_ids, sequence_lengths, max_session_len, token_generated);
}

template<int N, int C>
struct IndexedCopyParam {
    Array<void*, N> src_ptr;
    Array<void*, N> dst_ptr;
    Array<int, N>   stride;
    Array<int, C>   src_idx;
    Array<int, C>   dst_idx;
    int             max_stride;
};

template<class T, int N, int C>
__global__ void indexedCopy(IndexedCopyParam<N, C> param)
{
    const int bi = blockIdx.x;
    const int si = param.src_idx[bi];
    const int di = param.dst_idx[bi];
    for (int i = threadIdx.x; i < param.max_stride; i += blockDim.x) {
        PRAGMA_UNROLL
        for (int k = 0; k < N; ++k) {
            if (i < param.stride[k]) {
                *((T*)param.dst_ptr[k] + param.stride[k] * di + i) =
                    *((const T*)param.src_ptr[k] + param.stride[k] * si + i);
            }
        }
    }
}

template<class T, int N>
void invokeIndexedCopyImpl(void**       h_src_ptr,
                           void**       h_dst_ptr,
                           const int*   h_elem_sz,
                           const int*   h_src_idx,
                           const int*   h_dst_idx,
                           int          count,
                           hipStream_t st)
{
    dispatch(  // dispatch for num of copy operations
        std::integer_sequence<int, 4, 8, 16, 32, 64, 128, 256>{},
        [&](auto C) { return count <= C; },
        [&](auto C) {
            // maximum parameter size: sm<70: 4kB, sm>=70: 32kB
            static_assert(sizeof(IndexedCopyParam<N, C>) <= 4096);
            IndexedCopyParam<N, C> param{};
            std::copy_n(h_src_ptr, N, param.src_ptr.data());
            std::copy_n(h_dst_ptr, N, param.dst_ptr.data());
            std::transform(h_elem_sz, h_elem_sz + N, param.stride.data(), [](int size) {
                // Basic alignment check
                FT_CHECK_WITH_INFO(size % sizeof(T) == 0, fmtstr("misalignment: %d %% %d", size, (int)sizeof(T)));
                return size / sizeof(T);
            });
            param.max_stride = *std::max_element(param.stride.begin(), param.stride.end());
            auto copy_idx    = [](const int* src, int offset, int n, auto dst) {
                return src ? (void)std::copy_n(src + offset, n, dst) : std::iota(dst, dst + n, offset);
            };
            for (int c = 0; c < count; c += C) {
                int batch_size = std::min(count - c, (int)C);
                copy_idx(h_src_idx, c, batch_size, param.src_idx.data());
                copy_idx(h_dst_idx, c, batch_size, param.dst_idx.data());
                indexedCopy<T><<<batch_size, 128, 0, st>>>(param);
            }
        });
}

void invokeIndexedCopy(void**       h_src_ptr,
                       void**       h_dst_ptr,
                       const int*   h_elem_sz,
                       const int*   h_src_idx,
                       const int*   h_dst_idx,
                       int          count,
                       int          n_copys,
                       hipStream_t st)
{
    auto success = dispatch(std::integer_sequence<int, 1, 2, 3, 4>{}, [&](auto N) {
        if (N == n_copys) {
            invokeIndexedCopyImpl<uint32_t, N>(h_src_ptr, h_dst_ptr, h_elem_sz, h_src_idx, h_dst_idx, count, st);
            return true;
        }
        return false;
    });
    FT_CHECK(success);
}

__global__ void padLastTokenIds(int* token_ids, const int* context_length, int max_context_len, int batch_size)
{
    for (int bi = threadIdx.x; bi < batch_size; bi += blockDim.x) {
        token_ids[(max_context_len - 1) * batch_size + bi] = token_ids[(context_length[bi] - 1) * batch_size + bi];
    }
}

void invokePadLastTokenIds(
    int* token_ids, const int* context_length, int max_context_len, int batch_size, hipStream_t stream)
{
    padLastTokenIds<<<1, 512, 0, stream>>>(token_ids, context_length, max_context_len, batch_size);
}

template<typename T>
__global__ void getFeatureOfLastToken(T* output, const T* input, const int* cu_seqlens, int dims)
{
    int bi = blockIdx.x;
    int ti = cu_seqlens[bi + 1] - 1;
    for (int i = threadIdx.x; i < dims; i += blockDim.x) {
        output[dims * bi + i] = input[dims * ti + i];
    }
}

template<typename T>
void invokeGetFeatureOfLastToken(
    T* output, const T* input, const int* cu_seqlens, int dims, int batch_size, hipStream_t stream)
{
    getFeatureOfLastToken<<<batch_size, 256, 0, stream>>>(output, input, cu_seqlens, dims);
}

template void invokeGetFeatureOfLastToken(half*, const half*, const int*, int, int, hipStream_t);
template void invokeGetFeatureOfLastToken(float*, const float*, const int*, int, int, hipStream_t);
#ifdef ENABLE_BF16
template void invokeGetFeatureOfLastToken(__hip_bfloat16*, const __hip_bfloat16*, const int*, int, int, hipStream_t);
#endif  // ENABLE_BF16

template<class T, int C>
struct BatchedCopyParam {
    Array<T*, C>  src_ptr;
    Array<T*, C>  dst_ptr;
    Array<int, C> size;
    int           count;
};

template<int kThrPerCpy, class T, int C>
__global__ void batchedCopy(BatchedCopyParam<T, C> param)
{
    const int ti = threadIdx.x + blockIdx.x * blockDim.x;
    const int bi = ti / kThrPerCpy;
    if (bi >= param.count) {
        return;
    }
    const T* __restrict__ src = param.src_ptr[bi];
    T* __restrict__ dst       = param.dst_ptr[bi];
    int size                  = param.size[bi];
    for (int i = ti % kThrPerCpy; i < size; i += kThrPerCpy) {
        dst[i] = src[i];
    }
}

// MSVC does not like CUDA kernel launch inside nested lambdas
template<class P>
struct BatchedCopyLauncher {
    int          max_size;
    int          count;
    const P*     params;
    hipStream_t st;

    template<int S>
    void operator()(std::integral_constant<int, S>) const
    {
        constexpr int threads         = 128;
        constexpr int items_per_block = threads / S;
        const int     blocks          = (count + items_per_block - 1) / items_per_block;
        batchedCopy<S><<<blocks, threads, 0, st>>>(*params);
    }
};

void invokeBatchedCopy(void** src_ptr, void** dst_ptr, int* size, int count, hipStream_t st)
{
    dispatch(
        std::integer_sequence<int, 1, 8, 32, 128>{},
        [&](auto C) { return count <= C; },
        [&](auto C) {
            using T = uint32_t;
            BatchedCopyParam<T, C> params{};
            // TODO: on CUDA 12.1 and sm_70+ this can be 32K
            static_assert(sizeof(params) <= 4096);
            for (int c = 0; c < count; c += C) {
                const int bsz = std::min<int>(count - c, C);
                params.count  = bsz;
                for (int i = 0; i < bsz; ++i) {
                    params.src_ptr[i] = (T*)src_ptr[c + i];
                    params.dst_ptr[i] = (T*)dst_ptr[c + i];
                    FT_CHECK(size[c + i] % sizeof(T) == 0);
                    params.size[i] = size[c + i] / sizeof(T);
                }
                const int max_size = *std::max_element(params.size.begin(), params.size.end());
                dispatch(
                    std::integer_sequence<int, 1, 2, 4, 8, 16, 32, 64, 128>{},
                    [&](auto S) { return max_size <= S; },
                    BatchedCopyLauncher<BatchedCopyParam<T, C>>{max_size, count, &params, st});
            }
        });
}

#define VERSION_SWITCH(VERSION, CONST_NAME, ...)                                                                       \
    [&] {                                                                                                              \
        if (VERSION == 2) {                                                                                            \
            constexpr static int CONST_NAME = 2;                                                                       \
            return __VA_ARGS__();                                                                                      \
        }                                                                                                              \
        else {                                                                                                         \
            constexpr static int CONST_NAME = 1;                                                                       \
            return __VA_ARGS__();                                                                                      \
        }                                                                                                              \
    }()

template<typename T>
FlashAttentionOp<T>::FlashAttentionOp(int batch_size, int head_num, int key_len, int seq_len, int size_per_head):
    batch_size_(batch_size), head_num_(head_num), key_len_(key_len), seq_len_(seq_len), size_per_head_(size_per_head)
{
#ifdef _MSC_VER
    op_version_ = 1;
#else
    op_version_ = std::is_same<float, typename std::decay<T>::type>::value ? 1 : 2;
    if (op_version_ == 2 && getSMVersion() < 80) {
        op_version_ = 1;
    }
#endif
}

template<typename T>
int FlashAttentionOp<T>::get_workspace_size() const
{
#ifdef _MSC_VER
    FlashAttentionOpImpl<T, 1> attention_op(batch_size_, head_num_, key_len_, seq_len_, size_per_head_);
    return attention_op.get_workspace_size();
#else
    return VERSION_SWITCH(op_version_, OP_VERSION, [&]() {
        FlashAttentionOpImpl<T, OP_VERSION> attention_op(batch_size_, head_num_, key_len_, seq_len_, size_per_head_);
        return attention_op.get_workspace_size();
    });
#endif
}

template<typename T>
void FlashAttentionOp<T>::operator()(Params& params, hipStream_t st) const
{
#ifdef _MSC_VER
    FlashAttentionOpImpl<T, 1> attention_op(batch_size_, head_num_, key_len_, seq_len_, size_per_head_);
    return attention_op(params, st);
#else
    return VERSION_SWITCH(op_version_, OP_VERSION, [&]() {
        FlashAttentionOpImpl<T, OP_VERSION> attention_op(batch_size_, head_num_, key_len_, seq_len_, size_per_head_);
        return attention_op(params, st);
    });
#endif
}

template class FlashAttentionOp<float>;
template class FlashAttentionOp<half>;
#ifdef ENABLE_BF16
template class FlashAttentionOp<__hip_bfloat16>;
#endif

}  // namespace turbomind
