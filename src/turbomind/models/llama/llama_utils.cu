#include "hip/hip_runtime.h"
// Copyright (c) OpenMMLab. All rights reserved.

#include "src/turbomind/kernels/reduce_kernel_utils.cuh"
#include "src/turbomind/models/llama/llama_utils.h"
#include "src/turbomind/utils/cuda_utils.h"
#include <cmath>
#include <cstdio>
#include <cstdlib>
#include <cstring>
#include <hip/hip_fp16.h>
#include <hiprand/hiprand_kernel.h>
#include <thrust/device_vector.h>
#include <thrust/execution_policy.h>
#include <thrust/host_vector.h>
#include <vector>

namespace turbomind {

CmpMode compare_mode = kCmpNone;

template<typename T>
struct abs_diff_t {
    using type = T;
};

template<>
struct abs_diff_t<half> {
    using type = float;
};

template<typename T>
struct abs_diff: public thrust::unary_function<thrust::tuple<T, T>, typename abs_diff_t<T>::type> {
    __host__ __device__ float operator()(thrust::tuple<T, T> x) const
    {
        using R = typename abs_diff_t<T>::type;
        auto r  = R(thrust::get<0>(x)) - R(thrust::get<1>(x));
        return r < R(0) ? -r : r;
    }
};

template<typename T>
void CheckNan(const T* ptr, size_t size, std::string key, hipStream_t stream)
{
    std::vector<T> h_data(size);
    hipMemcpyAsync(h_data.data(), ptr, sizeof(T) * size, hipMemcpyDefault, stream);

    check_cuda_error(hipStreamSynchronize(stream));

    size_t nan_cnt = 0;
    for (const auto& x : h_data) {
        nan_cnt += std::isnan(static_cast<float>(x));
    }
    if (nan_cnt) {
        std::cerr << key << ": NaN count " << nan_cnt << "\n";
    }
}

template<typename T>
void CmpRead(T* ptr, size_t size, std::string key, hipStream_t stream)
{
    // wait for b
    check_cuda_error(hipStreamSynchronize(stream));
    // read a from file
    thrust::host_vector<T> h_a(size);
    {
        const auto    filename = "tmp/" + key + ".cmp";
        std::ifstream ifs(filename, std::ios::binary);
        if (!ifs.is_open()) {
            std::cerr << key << ": failed to open " + filename << "\n";
            return;
        }
        ifs.seekg(0, ifs.end);
        const auto actual_size_in_bytes = ifs.tellg();
        ifs.seekg(0, ifs.beg);
        const auto expect_size_in_bytes = sizeof(T) * size;
        if (actual_size_in_bytes != expect_size_in_bytes) {
            std::cerr << key << ": file size in bytes mismatch, expect " << expect_size_in_bytes << ", got "
                      << actual_size_in_bytes << "\n";
            return;
        }
        ifs.read((char*)h_a.data(), sizeof(T) * h_a.size());
    }
    // copy a to device
    thrust::device_vector<T> a = h_a;
    // create abs(a - b) iterator
    thrust::device_ptr<T> dev_ptr(ptr);
    auto                  zip_iter       = thrust::make_zip_iterator(thrust::make_tuple(a.begin(), dev_ptr));
    auto                  transform_iter = thrust::make_transform_iterator(zip_iter, abs_diff<T>{});
    // sum(abs(a - b))
    auto asum = thrust::reduce(thrust::device, transform_iter, transform_iter + size);
    std::cerr << key << ": " << asum << " " << asum / size << "\n";
}

template<typename T>
void CmpWrite(T* ptr, size_t size, std::string key, hipStream_t stream)
{
    std::vector<T> a(size);
    // copy a to host
    check_cuda_error(hipMemcpyAsync(a.data(), ptr, sizeof(T) * size, hipMemcpyDefault, stream));
    check_cuda_error(hipStreamSynchronize(stream));
    // write to file
    {
        std::ofstream ofs("tmp/" + key + ".cmp", std::ios::binary);
        ofs.write((char*)a.data(), sizeof(T) * a.size());
    }
}

template<typename T>
void Compare(T* ptr, size_t size, std::string key, CmpMode mode, hipStream_t stream)
{
    // std::cerr << "Comparing " << key << "\n";
    if (mode == kCmpRead) {
        CmpRead(ptr, size, key, stream);
    }
    else if (mode == kCmpWrite) {
        CmpWrite(ptr, size, key, stream);
    }
    else {
        // kCmpNone
    }
}

template void Compare(int* ptr, size_t size, std::string key, CmpMode mode, hipStream_t stream);
template void Compare(float* ptr, size_t size, std::string key, CmpMode mode, hipStream_t stream);
template void Compare(half* ptr, size_t size, std::string key, CmpMode mode, hipStream_t stream);

template void CheckNan(const float* ptr, size_t size, std::string key, hipStream_t stream);
template void CheckNan(const half* ptr, size_t size, std::string key, hipStream_t stream);

std::string format(const std::pair<std::string, Tensor>& p)
{
    std::stringstream ss;
    ss << p.first << " [";
    bool first = true;
    for (const auto& x : p.second.shape) {
        ss << (first ? "" : ", ") << x;
        first = false;
    }
    ss << "]";
    return ss.str();
}

size_t curandStateGetSize()
{
    return sizeof(hiprandState_t);
}

bool isDebug()
{
    static const bool is_debug = [] {
        const auto level = std::getenv("TM_DEBUG_LEVEL");
        if (level && level == std::string("DEBUG")) {
            return true;
        }
        return false;
    }();
    return is_debug;
}

}  // namespace turbomind
