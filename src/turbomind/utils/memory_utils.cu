#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2019-2023, NVIDIA CORPORATION.  All rights reserved.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "src/turbomind/macro.h"
#include "src/turbomind/utils/Tensor.h"
#include "src/turbomind/utils/cuda_type_utils.cuh"
#include "src/turbomind/utils/logger.h"
#include "src/turbomind/utils/memory_utils.h"
#include <hiprand/hiprand_kernel.h>
#include <sys/stat.h>
#include <unordered_map>

namespace turbomind {

template<typename T>
void deviceMalloc(T** ptr, size_t size, bool is_random_initialize)
{
    FT_CHECK_WITH_INFO(size >= ((size_t)0), "Ask deviceMalloc size " + std::to_string(size) + "< 0 is invalid.");
    check_cuda_error(hipMalloc((void**)(ptr), sizeof(T) * size));
    if (is_random_initialize) {
        cudaRandomUniform(*ptr, size);
    }
}

template void deviceMalloc(float** ptr, size_t size, bool is_random_initialize);
template void deviceMalloc(half** ptr, size_t size, bool is_random_initialize);
#ifdef ENABLE_BF16
template void deviceMalloc(__hip_bfloat16** ptr, size_t size, bool is_random_initialize);
#endif
template void deviceMalloc(uint16_t** ptr, size_t size, bool is_random_initialize);
template void deviceMalloc(int** ptr, size_t size, bool is_random_initialize);
template void deviceMalloc(bool** ptr, size_t size, bool is_random_initialize);
template void deviceMalloc(char** ptr, size_t size, bool is_random_initialize);
template void deviceMalloc(int8_t** ptr, size_t size, bool is_random_initialize);
#ifdef ENABLE_FP8
template void deviceMalloc(__hip_fp8_e4m3_fnuz** ptr, size_t size, bool is_random_initialize);
#endif

template<typename T>
void deviceMemSetZero(T* ptr, size_t size)
{
    check_cuda_error(hipMemset(static_cast<void*>(ptr), 0, sizeof(T) * size));
}

template void deviceMemSetZero(float* ptr, size_t size);
template void deviceMemSetZero(half* ptr, size_t size);
template void deviceMemSetZero(int* ptr, size_t size);
template void deviceMemSetZero(uint32_t* ptr, size_t size);
template void deviceMemSetZero(bool* ptr, size_t size);
#ifdef ENABLE_FP8
template void deviceMemSetZero(__hip_fp8_e4m3_fnuz* ptr, size_t size);
#endif
#ifdef ENABLE_BF16
template void deviceMemSetZero(__hip_bfloat16* ptr, size_t size);
#endif

template<typename T>
void deviceFree(T*& ptr)
{
    if (ptr != NULL) {
        check_cuda_error(hipFree(ptr));
        ptr = NULL;
    }
}

template void deviceFree(float*& ptr);
template void deviceFree(half*& ptr);
#ifdef ENABLE_BF16
template void deviceFree(__hip_bfloat16*& ptr);
#endif
template void deviceFree(unsigned short*& ptr);
template void deviceFree(int*& ptr);
template void deviceFree(bool*& ptr);
template void deviceFree(char*& ptr);
template void deviceFree(int8_t*& ptr);
#ifdef ENABLE_FP8
template void deviceFree(__hip_fp8_e4m3_fnuz*& ptr);
#endif

template<typename T>
void deviceFill(T* devptr, size_t size, T value, hipStream_t stream)
{
    T* arr = new T[size];
    std::fill(arr, arr + size, value);
    check_cuda_error(hipMemcpyAsync(devptr, arr, sizeof(T) * size, hipMemcpyHostToDevice, stream));
    delete[] arr;
}

template void deviceFill(float* devptr, size_t size, float value, hipStream_t stream);
template void deviceFill(half* devptr, size_t size, half value, hipStream_t stream);
#ifdef ENABLE_BF16
template void deviceFill(__hip_bfloat16* devptr, size_t size, __hip_bfloat16 value, hipStream_t stream);
#endif
template void deviceFill(int* devptr, size_t size, int value, hipStream_t stream);
template void deviceFill(bool* devptr, size_t size, bool value, hipStream_t stream);

template<typename T>
void cudaD2Hcpy(T* tgt, const T* src, const size_t size)
{
    check_cuda_error(hipMemcpy(tgt, src, sizeof(T) * size, hipMemcpyDeviceToHost));
}

template void cudaD2Hcpy(float* tgt, const float* src, size_t size);
template void cudaD2Hcpy(half* tgt, const half* src, size_t size);
#ifdef ENABLE_BF16
template void cudaD2Hcpy(__hip_bfloat16* tgt, const __hip_bfloat16* src, size_t size);
#endif
template void cudaD2Hcpy(int* tgt, const int* src, size_t size);
template void cudaD2Hcpy(bool* tgt, const bool* src, size_t size);
#ifdef ENABLE_FP8
template void cudaD2Hcpy(__hip_fp8_e4m3_fnuz* tgt, const __hip_fp8_e4m3_fnuz* src, size_t size);
#endif
template void cudaD2Hcpy(unsigned long long* tgt, const unsigned long long* src, size_t size);
template void cudaD2Hcpy(unsigned int* tgt, const unsigned int* src, size_t size);
template void cudaD2Hcpy(int8_t* tgt, const int8_t* src, size_t size);

template<typename T>
void cudaH2Dcpy(T* tgt, const T* src, const size_t size)
{
    if (tgt == nullptr || src == nullptr) {
        TM_LOG_ERROR("cudaH2Dcpy: dst=%p src=%p, size=%d", tgt, src, (int)(sizeof(T) * size));
    }
    check_cuda_error(hipMemcpy(tgt, src, sizeof(T) * size, hipMemcpyHostToDevice));
}

template void cudaH2Dcpy(float* tgt, const float* src, size_t size);
template void cudaH2Dcpy(half* tgt, const half* src, size_t size);
#ifdef ENABLE_BF16
template void cudaH2Dcpy(__hip_bfloat16* tgt, const __hip_bfloat16* src, size_t size);
#endif
template void cudaH2Dcpy(int* tgt, const int* src, size_t size);
template void cudaH2Dcpy(bool* tgt, const bool* src, size_t size);
#ifdef ENABLE_FP8
template void cudaH2Dcpy(__hip_fp8_e4m3_fnuz* tgt, const __hip_fp8_e4m3_fnuz* src, size_t size);
#endif
template void cudaH2Dcpy(unsigned long long* tgt, const unsigned long long* src, size_t size);
template void cudaH2Dcpy(unsigned int* tgt, const unsigned int* src, size_t size);
template void cudaH2Dcpy(int8_t* tgt, const int8_t* src, size_t size);

template<typename T>
void cudaD2Dcpy(T* tgt, const T* src, const size_t size)
{
    check_cuda_error(hipMemcpy(tgt, src, sizeof(T) * size, hipMemcpyDeviceToDevice));
}

template void cudaD2Dcpy(float* tgt, const float* src, size_t size);
template void cudaD2Dcpy(half* tgt, const half* src, size_t size);
#ifdef ENABLE_BF16
template void cudaD2Dcpy(__hip_bfloat16* tgt, const __hip_bfloat16* src, size_t size);
#endif
template void cudaD2Dcpy(int* tgt, const int* src, size_t size);
template void cudaD2Dcpy(bool* tgt, const bool* src, size_t size);
template void cudaD2Dcpy(int8_t* tgt, const int8_t* src, size_t size);
#ifdef ENABLE_FP8
template void cudaD2Dcpy(__hip_fp8_e4m3_fnuz* tgt, const __hip_fp8_e4m3_fnuz* src, size_t size);
#endif
template void cudaD2Dcpy(unsigned long long* tgt, const unsigned long long* src, size_t size);

template<typename T_OUT, typename T_IN>
__global__ void cudaCast(T_OUT* dst, T_IN* src, const size_t size)
{
    for (size_t tid = threadIdx.x + blockIdx.x * blockDim.x; tid < size; tid += blockDim.x * gridDim.x) {
        dst[tid] = (T_OUT)((float)(src[tid]));
    }
}

template<typename T_OUT, typename T_IN>
void invokeCudaCast(T_OUT* dst, T_IN const* const src, const size_t size, hipStream_t stream)
{
    cudaCast<<<256, 256, 0, stream>>>(dst, src, size);
}

template void invokeCudaCast(float* dst, half const* const src, const size_t size, hipStream_t stream);
#ifdef ENABLE_BF16
template void invokeCudaCast(float* dst, __hip_bfloat16 const* const src, const size_t size, hipStream_t stream);
template void invokeCudaCast(__hip_bfloat16* dst, float const* const src, const size_t size, hipStream_t stream);
template void invokeCudaCast(__hip_bfloat16* dst, half const* const src, const size_t size, hipStream_t stream);
template void invokeCudaCast(half* dst, __hip_bfloat16 const* const src, const size_t size, hipStream_t stream);
#endif
#ifdef ENABLE_FP8
template void invokeCudaCast(float* dst, __hip_fp8_e4m3_fnuz const* const src, const size_t size, hipStream_t stream);
template void
invokeCudaCast(__hip_bfloat16* dst, __hip_fp8_e4m3_fnuz const* const src, const size_t size, hipStream_t stream);
template void invokeCudaCast(half* dst, __hip_fp8_e4m3_fnuz const* const src, const size_t size, hipStream_t stream);
template void invokeCudaCast(__hip_fp8_e4m3_fnuz* dst, float const* const src, const size_t size, hipStream_t stream);
template void
invokeCudaCast(__hip_fp8_e4m3_fnuz* dst, __hip_bfloat16 const* const src, const size_t size, hipStream_t stream);
template void invokeCudaCast(__hip_fp8_e4m3_fnuz* dst, half const* const src, const size_t size, hipStream_t stream);
#endif

template<typename T>
void cudaAutoCpy(T* tgt, const T* src, const size_t size, hipStream_t stream)
{
    if (stream != NULL) {
        check_cuda_error(hipMemcpyAsync(tgt, src, sizeof(T) * size, hipMemcpyDefault, stream));
    }
    else {
        check_cuda_error(hipMemcpy(tgt, src, sizeof(T) * size, hipMemcpyDefault));
    }
}

template void cudaAutoCpy(float* tgt, const float* src, size_t size, hipStream_t stream);
template void cudaAutoCpy(half* tgt, const half* src, size_t size, hipStream_t stream);
#ifdef ENABLE_BF16
template void cudaAutoCpy(__hip_bfloat16* tgt, const __hip_bfloat16* src, size_t size, hipStream_t stream);
#endif
template void cudaAutoCpy(int* tgt, const int* src, size_t size, hipStream_t stream);
template void cudaAutoCpy(bool* tgt, const bool* src, size_t size, hipStream_t stream);
template void cudaAutoCpy(int8_t* tgt, const int8_t* src, size_t size, hipStream_t stream);
template void cudaAutoCpy(uint* tgt, const uint* src, size_t size, hipStream_t stream);
template void cudaAutoCpy(unsigned long long* tgt, const unsigned long long* src, size_t size, hipStream_t stream);
template void cudaAutoCpy(char* tgt, const char* src, size_t size, hipStream_t stream);

template void cudaAutoCpy(float const** tgt, float const* const* src, size_t size, hipStream_t stream);
template void cudaAutoCpy(half const** tgt, half const* const* src, size_t size, hipStream_t stream);
#ifdef ENABLE_BF16
template void cudaAutoCpy(__hip_bfloat16 const** tgt, __hip_bfloat16 const* const* src, size_t size, hipStream_t stream);
#endif
template void cudaAutoCpy(int const** tgt, int const* const* src, size_t size, hipStream_t stream);
template void cudaAutoCpy(bool const** tgt, bool const* const* src, size_t size, hipStream_t stream);
template void cudaAutoCpy(int8_t const** tgt, int8_t const* const* src, size_t size, hipStream_t stream);
template void
cudaAutoCpy(unsigned long long const** tgt, unsigned long long const* const* src, size_t size, hipStream_t stream);

template<typename T>
__global__ void cuda_random_uniform_kernel(T* buffer, const size_t size, const int seq_offset)
{
    const int     idx = blockIdx.x * blockDim.x + threadIdx.x;
    hiprandState_t local_state;
    hiprand_init((unsigned long long int)1337, idx + seq_offset, 0, &local_state);
    for (size_t index = idx; index < size; index += blockDim.x * gridDim.x) {
        buffer[index] = (T)(hiprand_uniform(&local_state) * 0.2f - 0.1f);
    }
}

template<>
__global__ void cuda_random_uniform_kernel<int>(int* buffer, const size_t size, const int seq_offset)
{
    const int     idx = blockIdx.x * blockDim.x + threadIdx.x;
    hiprandState_t local_state;
    hiprand_init((float)1337.f, idx + seq_offset, 0, &local_state);
    for (size_t index = idx; index < size; index += blockDim.x * gridDim.x) {
        buffer[index] = hiprand(&local_state);
    }
}

template<>
__global__ void cuda_random_uniform_kernel<bool>(bool* buffer, const size_t size, const int seq_offset)
{
    const int     idx = blockIdx.x * blockDim.x + threadIdx.x;
    hiprandState_t local_state;
    hiprand_init((float)1337.f, idx + seq_offset, 0, &local_state);
    for (size_t index = idx; index < size; index += blockDim.x * gridDim.x) {
        buffer[index] = (hiprand(&local_state) % 2 == 0);
    }
}

template<>
__global__ void cuda_random_uniform_kernel<char>(char* buffer, const size_t size, const int seq_offset)
{
    const int     idx = blockIdx.x * blockDim.x + threadIdx.x;
    hiprandState_t local_state;
    hiprand_init((float)1337.f, idx + seq_offset, 0, &local_state);
    for (size_t index = idx; index < size; index += blockDim.x * gridDim.x) {
        buffer[index] = hiprand(&local_state) % 0xFF;
    }
}

template<typename T>
void cudaRandomUniform(T* buffer, const size_t size)
{
    static int seq_offset = 0;
    cuda_random_uniform_kernel<T><<<256, 256>>>(buffer, size, seq_offset);
    seq_offset += 256 * 256;
}

template void cudaRandomUniform(float* buffer, const size_t size);
template void cudaRandomUniform(half* buffer, const size_t size);
#ifdef ENABLE_BF16
template void cudaRandomUniform(__hip_bfloat16* buffer, const size_t size);
#endif
template void cudaRandomUniform(int* buffer, const size_t size);
template void cudaRandomUniform(bool* buffer, const size_t size);
template void cudaRandomUniform(char* buffer, const size_t size);
#ifdef ENABLE_FP8
template void cudaRandomUniform(__hip_fp8_e4m3_fnuz* buffer, const size_t size);
#endif

// loads data from binary file. If it succeeds, returns a non-empty vector. If loading fails or
// the product of the elements in shape is 0, this function will return an empty vector.
template<typename T>
std::vector<T>
loadWeightFromBinHelper(std::vector<size_t> shape, std::string filename, std::vector<ConcateSlice> slices = {})
{
    if (shape.size() > 2) {
        printf("[ERROR] shape should have less than two dims \n");
        return std::vector<T>();
    }

    size_t dim0 = shape[0], dim1 = 1;
    if (shape.size() == 2) {
        dim1 = shape[1];
    }

    if (slices.size() == 0) {
        size_t size = dim0 * dim1;
        if (size == 0) {
            TM_LOG_WARNING("shape is zero, skip loading weight from file %s \n", filename.c_str());
            return std::vector<T>();
        }

        std::vector<T> host_array(size);
        std::ifstream  in(filename, std::ios::in | std::ios::binary);
        if (!in.is_open()) {
            TM_LOG_WARNING("file %s cannot be opened, loading model fails! \n", filename.c_str());
            return std::vector<T>();
        }

        size_t loaded_data_size = sizeof(T) * size;
        in.seekg(0, in.end);
        const auto file_size_in_bytes = (size_t)in.tellg();
        in.seekg(0, in.beg);

        TM_LOG_DEBUG("Read " + std::to_string(loaded_data_size) + " bytes from " + filename);
        in.read((char*)host_array.data(), loaded_data_size);

        if (file_size_in_bytes != loaded_data_size) {
            TM_LOG_WARNING("file %s has %ld, but request %ld, loading model fails!",
                           filename.c_str(),
                           file_size_in_bytes,
                           loaded_data_size);
            return std::vector<T>();
        }
        in.close();
        // If we succeed, return an array with values.
        return host_array;
    }
    else {
        // concate all slices on the same dims

        if (slices.size() != shape.size()) {
            printf("[ERROR] slices should have same dims as shape \n");
            return std::vector<T>();
        }

        // get slices
        ConcateSlice slice0{{{0, dim0}}};
        ConcateSlice slice1{{{0, dim1}}};
        if (slices.size() > 0 && slices[0].slices.size() > 0) {
            slice0 = slices[0];
        }
        if (shape.size() == 2 && slices[1].slices.size() > 0) {
            slice1 = slices[1];
        }

        size_t w0 = 0;
        for (auto& s : slice0.slices) {
            if (s.second > dim0) {
                s.second = dim0;
            }
            if (s.second < s.first) {
                printf("[ERROR] slice0: end < start \n");
                return std::vector<T>();
            }
            w0 += s.second - s.first;
        }

        size_t w1 = 0;
        for (auto& s : slice1.slices) {
            if (s.second > dim1) {
                s.second = dim1;
            }
            if (s.second < s.first) {
                printf("[ERROR] slice1: end < start \n");
                return std::vector<T>();
            }
            w1 += s.second - s.first;
        }

        size_t size             = w0 * w1;
        size_t loaded_data_size = size * sizeof(T);

        TM_LOG_DEBUG("Read " + std::to_string(loaded_data_size) + " bytes from " + filename + " with slice.");
        if (size == 0) {
            TM_LOG_WARNING("shape is zero, skip loading weight from file %s \n", filename.c_str());
            return std::vector<T>();
        }

        std::vector<T> host_array(size);
        std::ifstream  in(filename, std::ios::in | std::ios::binary);
        if (!in.is_open()) {
            TM_LOG_WARNING("file %s cannot be opened, loading model fails! \n", filename.c_str());
            return std::vector<T>();
        }

        char* host_ptr = (char*)host_array.data();
        if (slice1.slices.size() == 0
            || (slice1.slices.size() == 1 && slice1.slices[0].second - slice1.slices[0].first == dim1)) {
            for (auto& s : slice0.slices) {
                size_t read_size = (s.second - s.first) * dim1 * sizeof(T);
                size_t pos       = s.first * dim1;
                in.seekg(pos * sizeof(T));
                in.read((char*)host_ptr, read_size);
                host_ptr += read_size;
            }
            in.close();
            return host_array;
        }

        {
            for (auto& s0 : slice0.slices) {
                // loop over outer slice
                for (size_t line_id = s0.first; line_id < s0.second; ++line_id) {
                    // loop over lines
                    size_t pos0 = line_id * dim1;
                    for (auto& s1 : slice1.slices) {
                        // loop over inner slice
                        size_t pos       = pos0 + s1.first;
                        size_t read_size = (s1.second - s1.first) * sizeof(T);
                        in.seekg(pos * sizeof(T));
                        in.read(host_ptr, read_size);
                        host_ptr += read_size;
                    }
                }
            }
            in.close();
        }
        return host_array;
    }
}

std::vector<float> loadArrayFromBin(std::vector<size_t> shape, std::string filename, std::vector<ConcateSlice> slices)
{
    return loadWeightFromBinHelper<float>(shape, filename, slices);
}

template<typename T, typename T_IN>
int loadWeightFromBinFunc(T*                        ptr,
                          std::vector<size_t>       shape,
                          std::string               filename,
                          std::vector<ConcateSlice> slices = std::vector<ConcateSlice>())
{
    std::vector<T_IN> host_array = loadWeightFromBinHelper<T_IN>(shape, filename, slices);

    if (host_array.empty()) {
        return 0;
    }

    if (std::is_same<T, T_IN>::value == true) {
        cudaH2Dcpy(ptr, (T*)host_array.data(), host_array.size());
    }
    else {
        T_IN* ptr_2 = nullptr;
        deviceMalloc(&ptr_2, host_array.size(), false);
        cudaH2Dcpy(ptr_2, host_array.data(), host_array.size());
        invokeCudaD2DcpyConvert(ptr, ptr_2, host_array.size());
        deviceFree(ptr_2);
    }
    return 0;
}

template int loadWeightFromBinFunc<float, float>(float*                    ptr,
                                                 std::vector<size_t>       shape,
                                                 std::string               filename,
                                                 std::vector<ConcateSlice> slices);
template int loadWeightFromBinFunc<half, float>(half*                     ptr,
                                                std::vector<size_t>       shape,
                                                std::string               filename,
                                                std::vector<ConcateSlice> slices);
template int loadWeightFromBinFunc<float, half>(float*                    ptr,
                                                std::vector<size_t>       shape,
                                                std::string               filename,
                                                std::vector<ConcateSlice> slices);
template int loadWeightFromBinFunc<half, half>(half*                     ptr,
                                               std::vector<size_t>       shape,
                                               std::string               filename,
                                               std::vector<ConcateSlice> slices);
template int loadWeightFromBinFunc<int8_t, int8_t>(int8_t*                   ptr,
                                                   std::vector<size_t>       shape,
                                                   std::string               filename,
                                                   std::vector<ConcateSlice> slices);
#ifdef ENABLE_BF16
template int loadWeightFromBinFunc<__hip_bfloat16, float>(__hip_bfloat16*            ptr,
                                                         std::vector<size_t>       shape,
                                                         std::string               filename,
                                                         std::vector<ConcateSlice> slices);
template int loadWeightFromBinFunc<__hip_bfloat16, half>(__hip_bfloat16*            ptr,
                                                        std::vector<size_t>       shape,
                                                        std::string               filename,
                                                        std::vector<ConcateSlice> slices);
template int loadWeightFromBinFunc<float, __hip_bfloat16>(float*                    ptr,
                                                         std::vector<size_t>       shape,
                                                         std::string               filename,
                                                         std::vector<ConcateSlice> slices);
template int loadWeightFromBinFunc<half, __hip_bfloat16>(half*                     ptr,
                                                        std::vector<size_t>       shape,
                                                        std::string               filename,
                                                        std::vector<ConcateSlice> slices);
template int loadWeightFromBinFunc<__hip_bfloat16, __hip_bfloat16>(__hip_bfloat16*            ptr,
                                                                 std::vector<size_t>       shape,
                                                                 std::string               filename,
                                                                 std::vector<ConcateSlice> slices);
#endif  // ENABLE_BF16
template int loadWeightFromBinFunc<int, int>(int*                      ptr,
                                             std::vector<size_t>       shape,
                                             std::string               filename,
                                             std::vector<ConcateSlice> slices);
#ifdef ENABLE_FP8
template int loadWeightFromBinFunc<__hip_fp8_e4m3_fnuz, float>(__hip_fp8_e4m3_fnuz*            ptr,
                                                         std::vector<size_t>       shape,
                                                         std::string               filename,
                                                         std::vector<ConcateSlice> slices);
#endif  // ENABLE_FP8

template<typename T>
int loadWeightFromBin(T*                        ptr,
                      std::vector<size_t>       shape,
                      std::string               filename,
                      FtCudaDataType            model_file_type,
                      std::vector<ConcateSlice> slices)
{
    switch (model_file_type) {
        case FtCudaDataType::FP32:
            loadWeightFromBinFunc<T, float>(ptr, shape, filename, slices);
            break;
        case FtCudaDataType::FP16:
            loadWeightFromBinFunc<T, half>(ptr, shape, filename, slices);
            break;
        case FtCudaDataType::INT8:
            loadWeightFromBinFunc<T, int8_t>(ptr, shape, filename, slices);
            break;
#ifdef ENABLE_BF16
        case FtCudaDataType::BF16:
            loadWeightFromBinFunc<T, __hip_bfloat16>(ptr, shape, filename, slices);
            break;
#endif
#ifdef ENABLE_FP8
        case FtCudaDataType::FP8:
            loadWeightFromBinFunc<T, float>(ptr, shape, filename, slices);
            break;
#endif
        default:
            TM_LOG_ERROR("Does not support FtCudaDataType=%d", model_file_type);
            FT_CHECK(false);
    }
    return 0;
}

template<>
int loadWeightFromBin(int*                      ptr,
                      std::vector<size_t>       shape,
                      std::string               filename,
                      FtCudaDataType            model_file_type,
                      std::vector<ConcateSlice> slices)
{
    loadWeightFromBinFunc<int, int>(ptr, shape, filename, slices);
    return 0;
}

template int loadWeightFromBin(float*                    ptr,
                               std::vector<size_t>       shape,
                               std::string               filename,
                               FtCudaDataType            model_file_type,
                               std::vector<ConcateSlice> slices);
template int loadWeightFromBin(half*                     ptr,
                               std::vector<size_t>       shape,
                               std::string               filename,
                               FtCudaDataType            model_file_type,
                               std::vector<ConcateSlice> slices);
template int loadWeightFromBin(int8_t*                   ptr,
                               std::vector<size_t>       shape,
                               std::string               filename,
                               FtCudaDataType            model_file_type,
                               std::vector<ConcateSlice> slices);
#ifdef ENABLE_BF16
template int loadWeightFromBin(__hip_bfloat16*            ptr,
                               std::vector<size_t>       shape,
                               std::string               filename,
                               FtCudaDataType            model_file_type,
                               std::vector<ConcateSlice> slices);
#endif
#ifdef ENABLE_FP8
template int loadWeightFromBin(__hip_fp8_e4m3_fnuz*            ptr,
                               std::vector<size_t>       shape,
                               std::string               filename,
                               FtCudaDataType            model_file_type,
                               std::vector<ConcateSlice> slices);
#endif
template int loadWeightFromBin(int*                      ptr,
                               std::vector<size_t>       shape,
                               std::string               filename,
                               FtCudaDataType            model_file_type,
                               std::vector<ConcateSlice> slices);

template<typename T_IN, typename T_OUT>
__global__ void cudaD2DcpyConvert(T_OUT* dst, const T_IN* src, const size_t size)
{
    for (size_t tid = threadIdx.x + blockIdx.x * blockDim.x; tid < size; tid += blockDim.x * gridDim.x) {
        dst[tid] = cuda_cast<T_OUT>(src[tid]);
    }
}

template<typename T_IN, typename T_OUT>
void invokeCudaD2DcpyConvert(T_OUT* tgt, const T_IN* src, const size_t size, hipStream_t stream)
{
    cudaD2DcpyConvert<<<256, 256, 0, stream>>>(tgt, src, size);
}

template void invokeCudaD2DcpyConvert(int8_t* tgt, const float* src, const size_t size, hipStream_t stream);
template void invokeCudaD2DcpyConvert(float* tgt, const int8_t* src, const size_t size, hipStream_t stream);
template void invokeCudaD2DcpyConvert(float* tgt, const int* src, const size_t size, hipStream_t stream);
template void invokeCudaD2DcpyConvert(half* tgt, const int* src, const size_t size, hipStream_t stream);
template void invokeCudaD2DcpyConvert(float* tgt, const float* src, const size_t size, hipStream_t stream);
template void invokeCudaD2DcpyConvert(half* tgt, const float* src, const size_t size, hipStream_t stream);
template void invokeCudaD2DcpyConvert(float* tgt, const half* src, const size_t size, hipStream_t stream);
template void invokeCudaD2DcpyConvert(uint* tgt, const int* src, const size_t size, hipStream_t stream);
template void invokeCudaD2DcpyConvert(int* tgt, const uint* src, const size_t size, hipStream_t stream);
template void invokeCudaD2DcpyConvert(int* tgt, const float* src, const size_t size, hipStream_t stream);
template void invokeCudaD2DcpyConvert(int* tgt, const half* src, const size_t size, hipStream_t stream);

#ifdef ENABLE_BF16
template void invokeCudaD2DcpyConvert(__hip_bfloat16* tgt, const float* src, const size_t size, hipStream_t stream);
template void invokeCudaD2DcpyConvert(__hip_bfloat16* tgt, const int* src, const size_t size, hipStream_t stream);
template void invokeCudaD2DcpyConvert(float* tgt, const __hip_bfloat16* src, const size_t size, hipStream_t stream);
template void invokeCudaD2DcpyConvert(int* tgt, const __hip_bfloat16* src, const size_t size, hipStream_t stream);
#endif  // ENABLE_BF16

template<typename T_IN, typename T_OUT>
__global__ void
cudaD2DScaleCpyConvert(T_OUT* dst, const T_IN* src, const float* scale, bool invert_scale, const size_t size)
{
    const float scale_value = invert_scale ? 1.0f / scale[0] : scale[0];
    for (size_t tid = threadIdx.x + blockIdx.x * blockDim.x; tid < size; tid += blockDim.x * gridDim.x) {
        dst[tid] = cuda_cast<T_OUT>(cuda_cast<float>(src[tid]) * scale_value);
    }
}

template<typename T_IN, typename T_OUT>
void invokeCudaD2DScaleCpyConvert(
    T_OUT* tgt, const T_IN* src, const float* scale, bool invert_scale, const size_t size, hipStream_t stream)
{
    cudaD2DScaleCpyConvert<<<256, 256, 0, stream>>>(tgt, src, scale, invert_scale, size);
}

// clang-format off
template void invokeCudaD2DScaleCpyConvert(float* tgt, const int32_t* src, const float* scale, bool invert_scale, const size_t size, hipStream_t stream);
template void invokeCudaD2DScaleCpyConvert(int32_t* tgt, const float* src, const float* scale, bool invert_scale, const size_t size, hipStream_t stream);
template void invokeCudaD2DScaleCpyConvert(half* tgt, const int32_t* src, const float* scale, bool invert_scale, const size_t size, hipStream_t stream);
template void invokeCudaD2DScaleCpyConvert(int32_t* tgt, const half* src, const float* scale, bool invert_scale, const size_t size, hipStream_t stream);
#ifdef ENABLE_BF16
template void invokeCudaD2DScaleCpyConvert(__hip_bfloat16* tgt, const int32_t* src, const float* scale, bool invert_scale, const size_t size, hipStream_t stream);
template void invokeCudaD2DScaleCpyConvert(int32_t* tgt, const __hip_bfloat16* src, const float* scale, bool invert_scale, const size_t size, hipStream_t stream);
#endif  // ENABLE_BF16
#ifdef ENABLE_FP8
template void invokeCudaD2DScaleCpyConvert(float* tgt, const __hip_fp8_e4m3_fnuz* src, const float* scale, bool invert_scale, const size_t size, hipStream_t stream);
#endif  // ENABLE_FP8
// clang-format on

void invokeCudaD2DcpyHalf2Float(float* dst, half* src, const size_t size, hipStream_t stream)
{
    invokeCudaD2DcpyConvert(dst, src, size, stream);
}

void invokeCudaD2DcpyFloat2Half(half* dst, float* src, const size_t size, hipStream_t stream)
{
    invokeCudaD2DcpyConvert(dst, src, size, stream);
}

template<typename T>
void saveToBinary(const T* ptr, const size_t size, std::string filename)
{

    std::vector<T> h_ptr(size);
    cudaD2Hcpy(h_ptr.data(), ptr, size);
    std::vector<float> float_ptr(size);
    for (size_t i = 0; i < size; i++) {
        float_ptr[i] = (float)h_ptr[i];
    }

    std::ofstream out(filename, std::ios::out | std::ios::binary);
    FT_CHECK_WITH_INFO(out.is_open(), "Fail to open file " + filename);

    out.write((char*)float_ptr.data(), size * sizeof(float));
}

template void saveToBinary(const float* ptr, const size_t size, std::string filename);
template void saveToBinary(const half* ptr, const size_t size, std::string filename);
#ifdef ENABLE_BF16
template void saveToBinary(const __hip_bfloat16* ptr, const size_t size, std::string filename);
#endif  // ENABLE_BF16

template<>
void saveToBinary(const int* ptr, const size_t size, std::string filename)
{
    std::vector<int> h_ptr(size);
    cudaD2Hcpy(h_ptr.data(), ptr, size);
    std::ofstream out(filename, std::ios::out | std::ios::binary);
    FT_CHECK_WITH_INFO(out.is_open(), "Fail to open file " + filename);
    out.write((char*)h_ptr.data(), size * sizeof(int));
}

template<typename T_IN, typename T_fake_type>
__global__ void fakeCast(T_IN* input_ptr, const size_t size)
{
    for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < size; i += blockDim.x * gridDim.x) {
        T_fake_type tmp_val = (T_fake_type)((float)input_ptr[i]);
        input_ptr[i]        = (T_IN)((float)tmp_val);
    }
}

template<typename T_IN, typename T_fake_type>
void invokeFakeCast(T_IN* input_ptr, const size_t size, hipStream_t stream)
{
    dim3 block(256);
    dim3 grid((size + 255) / 256);
    fakeCast<T_IN, T_fake_type><<<grid, block, 0, stream>>>(input_ptr, size);
}

#ifdef ENABLE_FP8
__global__ void cudaD2Dcpyfp82Float(float* dst, __hip_fp8_e4m3_fnuz* src, const size_t size)
{
    for (size_t tid = threadIdx.x + blockIdx.x * blockDim.x; tid < size; tid += blockDim.x * gridDim.x) {
        dst[tid] = (float)(src[tid]);
    }
}

void invokeCudaD2Dcpyfp82Float(float* dst, __hip_fp8_e4m3_fnuz* src, const size_t size, hipStream_t stream)
{
    cudaD2Dcpyfp82Float<<<256, 256, 0, stream>>>(dst, src, size);
}

__global__ void cudaD2Dcpyfp82Half(half* dst, __hip_fp8_e4m3_fnuz* src, const size_t size)
{
    for (size_t tid = threadIdx.x + blockIdx.x * blockDim.x; tid < size; tid += blockDim.x * gridDim.x) {
        dst[tid] = (half)((float)(src[tid]));
    }
}

void invokeCudaD2Dcpyfp82Half(half* dst, __hip_fp8_e4m3_fnuz* src, const size_t size, hipStream_t stream)
{
    cudaD2Dcpyfp82Half<<<256, 256, 0, stream>>>(dst, src, size);
}

__global__ void cudaD2DcpyFloat2fp8(__hip_fp8_e4m3_fnuz* dst, float* src, const size_t size)
{
    for (size_t tid = threadIdx.x + blockIdx.x * blockDim.x; tid < size; tid += blockDim.x * gridDim.x) {
        dst[tid] = (__hip_fp8_e4m3_fnuz)src[tid];
    }
}

void invokeCudaD2DcpyFloat2fp8(__hip_fp8_e4m3_fnuz* dst, float* src, const size_t size, hipStream_t stream)
{
    cudaD2DcpyFloat2fp8<<<256, 256, 0, stream>>>(dst, src, size);
}

__global__ void cudaD2DcpyHalf2fp8(__hip_fp8_e4m3_fnuz* dst, half* src, const size_t size)
{
    for (size_t tid = threadIdx.x + blockIdx.x * blockDim.x; tid < size; tid += blockDim.x * gridDim.x) {
        dst[tid] = (__hip_fp8_e4m3_fnuz)src[tid];
    }
}

void invokeCudaD2DcpyHalf2fp8(__hip_fp8_e4m3_fnuz* dst, half* src, const size_t size, hipStream_t stream)
{
    cudaD2DcpyHalf2fp8<<<256, 256, 0, stream>>>(dst, src, size);
}

__global__ void cudaD2DcpyBfloat2fp8(__hip_fp8_e4m3_fnuz* dst, __hip_bfloat16* src, const size_t size)
{
    for (size_t tid = threadIdx.x + blockIdx.x * blockDim.x; tid < size; tid += blockDim.x * gridDim.x) {
        dst[tid] = (__hip_fp8_e4m3_fnuz)src[tid];
    }
}

void invokeCudaD2DcpyBfloat2fp8(__hip_fp8_e4m3_fnuz* dst, __hip_bfloat16* src, const size_t size, hipStream_t stream)
{
    cudaD2DcpyBfloat2fp8<<<256, 256, 0, stream>>>(dst, src, size);
}

#endif  // ENABLE_FP8

template<typename T_OUT, typename T_IN>
__global__ void transpose(T_OUT* dst, T_IN* src, const int dim0, const int dim1)
{
    for (size_t tid = threadIdx.x + blockIdx.x * blockDim.x; tid < dim0 * dim1; tid += blockDim.x * gridDim.x) {
        const int src_col_id                = tid % dim1;
        const int src_row_id                = tid / dim1;
        dst[src_col_id * dim0 + src_row_id] = (T_OUT)(src[tid]);
    }
}

template<typename T>
void invokeInPlaceTranspose(T* data, T* workspace, const int dim0, const int dim1)
{
    // copy data to workspace, and then transpose from workspace to data
    cudaD2Dcpy(workspace, data, dim0 * dim1);
    transpose<<<256, 256>>>(data, workspace, dim0, dim1);
}

#ifdef ENABLE_FP8
template void invokeInPlaceTranspose(__hip_fp8_e4m3_fnuz* data, __hip_fp8_e4m3_fnuz* workspace, const int dim0, const int dim1);
#endif  // ENABLE_FP8
#ifdef ENABLE_BF16
template void invokeInPlaceTranspose(__hip_bfloat16* data, __hip_bfloat16* workspace, const int dim0, const int dim1);
#endif  // ENABLE_BF16
template void invokeInPlaceTranspose(float* data, float* workspace, const int dim0, const int dim1);

template<typename T_OUT, typename T_IN>
__global__ void transpose0213(T_OUT* dst, T_IN* src, const int dim0, const int dim1, const int dim2, const int dim3)
{
    // src permutation: [0, 1, 2, 3]
    // dst permutation: [0, 2, 1, 3]
    for (size_t tid = threadIdx.x + blockIdx.x * blockDim.x; tid < dim0 * dim1 * dim2 * dim3;
         tid += blockDim.x * gridDim.x) {
        int       tmp_idx   = tid;
        const int dim_3_idx = tmp_idx % dim3;
        tmp_idx             = (tmp_idx - dim_3_idx) / dim3;
        const int dim_2_idx = tmp_idx % dim2;
        tmp_idx             = (tmp_idx - dim_2_idx) / dim2;
        const int dim_1_idx = tmp_idx % dim1;
        tmp_idx             = (tmp_idx - dim_1_idx) / dim1;
        const int dim_0_idx = tmp_idx % dim0;
        dst[dim_0_idx * dim1 * dim2 * dim3 + dim_2_idx * dim1 * dim3 + dim_1_idx * dim3 + dim_3_idx] = src[tid];
    }
}

template<typename T>
void invokeInPlaceTranspose0213(T* data, T* workspace, const int dim0, const int dim1, const int dim2, const int dim3)
{
    // copy data to workspace, and then transpose from workspace to data
    // Note that this kernel is used for pre-processing and not very efficient.
    cudaD2Dcpy(workspace, data, dim0 * dim1 * dim2 * dim3);
    transpose0213<<<256, 256>>>(data, workspace, dim0, dim1, dim2, dim3);
}

#ifdef ENABLE_FP8
template void invokeInPlaceTranspose0213(
    __hip_fp8_e4m3_fnuz* data, __hip_fp8_e4m3_fnuz* workspace, const int dim0, const int dim1, const int dim2, const int dim3);
#endif  // ENABLE_FP8
#ifdef ENABLE_BF16
template void invokeInPlaceTranspose0213(
    __hip_bfloat16* data, __hip_bfloat16* workspace, const int dim0, const int dim1, const int dim2, const int dim3);
#endif  // ENABLE_BF16
template void invokeInPlaceTranspose0213(
    float* data, float* workspace, const int dim0, const int dim1, const int dim2, const int dim3);

template<typename T_OUT, typename T_IN>
__global__ void transpose102(T_OUT* dst, T_IN* src, const int dim0, const int dim1, const int dim2)
{
    // src permutation: [0, 1, 2]
    // dst permutation: [1, 0, 2]
    for (size_t tid = threadIdx.x + blockIdx.x * blockDim.x; tid < dim0 * dim1 * dim2; tid += blockDim.x * gridDim.x) {
        int       tmp_idx                                           = tid;
        const int dim_2_idx                                         = tmp_idx % dim2;
        tmp_idx                                                     = (tmp_idx - dim_2_idx) / dim2;
        const int dim_1_idx                                         = tmp_idx % dim1;
        tmp_idx                                                     = (tmp_idx - dim_1_idx) / dim1;
        const int dim_0_idx                                         = tmp_idx % dim0;
        dst[dim_1_idx * dim0 * dim2 + dim_0_idx * dim2 + dim_2_idx] = src[tid];
    }
}

template<typename T>
void invokeInPlaceTranspose102(T* data, T* workspace, const int dim0, const int dim1, const int dim2)
{
    // copy data to workspace, and then transpose from workspace to data
    // Note that this kernel is used for pre-processing and not very efficient.
    cudaD2Dcpy(workspace, data, dim0 * dim1 * dim2);
    transpose102<<<256, 256>>>(data, workspace, dim0, dim1, dim2);
}

#ifdef ENABLE_FP8
template void invokeInPlaceTranspose102(
    __hip_fp8_e4m3_fnuz* data, __hip_fp8_e4m3_fnuz* workspace, const int dim0, const int dim1, const int dim2);
#endif  // ENABLE_FP8
#ifdef ENABLE_BF16
template void invokeInPlaceTranspose102(
    __hip_bfloat16* data, __hip_bfloat16* workspace, const int dim0, const int dim1, const int dim2);
#endif  // ENABLE_BF16
template void invokeInPlaceTranspose102(float* data, float* workspace, const int dim0, const int dim1, const int dim2);

template<typename T>
void __global__ multiplyScale(T* tensor, float scale, const size_t size)
{
    for (size_t index = threadIdx.x + blockIdx.x * blockDim.x; index < size; index += blockDim.x * gridDim.x) {
        tensor[index] = (T)(((float)tensor[index]) * scale);
    }
}

template<typename T>
void invokeMultiplyScale(T* tensor, float scale, const size_t size, hipStream_t stream)
{
    int block = 256;
    int grid  = (size + 255) / 256;
    multiplyScale<<<grid, block, 0, stream>>>(tensor, scale, size);
}

template void invokeMultiplyScale(float* tensor, float scale, const size_t size, hipStream_t stream);
template void invokeMultiplyScale(half* tensor, float scale, const size_t size, hipStream_t stream);
#ifdef ENABLE_BF16
template void invokeMultiplyScale(__hip_bfloat16* tensor, float scale, const size_t size, hipStream_t stream);
#endif
#ifdef ENABLE_FP8
template void invokeMultiplyScale(__hip_fp8_e4m3_fnuz* tensor, float scale, const size_t size, hipStream_t stream);
#endif

template<typename T>
void __global__ divideScale(T* tensor, float scale, const size_t size)
{
    for (size_t index = threadIdx.x + blockIdx.x * blockDim.x; index < size; index += blockDim.x * gridDim.x) {
        tensor[index] = (T)(((float)tensor[index]) / scale);
    }
}

template<typename T>
void invokeDivideScale(T* tensor, float scale, const size_t size, hipStream_t stream)
{
    int block = 256;
    int grid  = (size + 255) / 256;
    divideScale<<<grid, block, 0, stream>>>(tensor, scale, size);
}

template void invokeDivideScale(float* tensor, float scale, const size_t size, hipStream_t stream);
template void invokeDivideScale(half* tensor, float scale, const size_t size, hipStream_t stream);
#ifdef ENABLE_BF16
template void invokeDivideScale(__hip_bfloat16* tensor, float scale, const size_t size, hipStream_t stream);
#endif
#ifdef ENABLE_FP8
template void invokeDivideScale(__hip_fp8_e4m3_fnuz* tensor, float scale, const size_t size, hipStream_t stream);
#endif
#ifdef ENABLE_BF16
template void invokeFakeCast<float, __hip_bfloat16>(float* input_ptr, const size_t size, hipStream_t stream);
template void
invokeFakeCast<__hip_bfloat16, __hip_bfloat16>(__hip_bfloat16* input_ptr, const size_t size, hipStream_t stream);
template void invokeFakeCast<half, __hip_bfloat16>(half* input_ptr, const size_t size, hipStream_t stream);
#endif
template void invokeFakeCast<float, half>(float* input_ptr, const size_t size, hipStream_t stream);
template void invokeFakeCast<float, float>(float* input_ptr, const size_t size, hipStream_t stream);
#ifdef ENABLE_FP8
template void invokeFakeCast<float, __hip_fp8_e4m3_fnuz>(float* input_ptr, const size_t size, hipStream_t stream);
template void invokeFakeCast<half, __hip_fp8_e4m3_fnuz>(half* input_ptr, const size_t size, hipStream_t stream);
template void
invokeFakeCast<__hip_bfloat16, __hip_fp8_e4m3_fnuz>(__hip_bfloat16* input_ptr, const size_t size, hipStream_t stream);
#endif

size_t cuda_datatype_size(FtCudaDataType dt)
{
    static const std::unordered_map<FtCudaDataType, size_t> sizes{{FtCudaDataType::FP32, sizeof(float)},
                                                                  {FtCudaDataType::FP16, sizeof(half)}
#ifdef ENABLE_BF16
                                                                  ,
                                                                  {FtCudaDataType::BF16, sizeof(__hip_bfloat16)}
#endif
    };

    return sizes.at(dt);
}

template<typename T>
__global__ void check_range(T* buffer, size_t size, T min, T max, bool* d_within_range)
{
    for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < size; i += blockDim.x * gridDim.x) {
        const T val = buffer[i];
        if (val < min || val > max) {
            *d_within_range = false;
        }
    }
}

template<typename T>
bool invokeCheckRange(T* buffer, const size_t size, T min, T max, bool* d_within_range, hipStream_t stream)
{
    hipMemsetAsync(d_within_range, true, sizeof(bool), stream);

    dim3 block(256);
    dim3 grid((size + 255) / 256);
    check_range<T><<<grid, block, 0, stream>>>(buffer, size, min, max, d_within_range);

    bool result;
    cudaD2Hcpy(&result, d_within_range, 1);
    return result;
}

template bool
invokeCheckRange<int>(int* buffer, const size_t size, int min, int max, bool* d_within_range, hipStream_t stream);

}  // namespace turbomind
