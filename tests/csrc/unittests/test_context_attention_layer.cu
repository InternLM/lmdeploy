#include "hip/hip_runtime.h"
// Copyright (c) OpenMMLab. All rights reserved.

#include <assert.h>
#include <cstdlib>
#include <math.h>
#include <numeric>
#include <random>
#include <thrust/device_vector.h>
#include <thrust/execution_policy.h>
#include <thrust/host_vector.h>
#include <thrust/transform.h>

#undef TORCH_CUDA

#include "src/turbomind/kernels/bert_preprocess_kernels.h"
#include "src/turbomind/kernels/unfused_attention_kernels.h"
#include "src/turbomind/models/llama/llama_kernels.h"
#include "src/turbomind/utils/allocator.h"
#include "src/turbomind/utils/cublasMMWrapper.h"
#include "src/turbomind/utils/cuda_utils.h"
#include "src/turbomind/utils/logger.h"
#include "src/turbomind/utils/memory_utils.h"
#include "unittest_utils.h"

using namespace turbomind;

template<typename scalar_t>
__global__ void pad_query_kernel(
    scalar_t* query_ptr, const int* cu_seqlens, int batch_size, int batch_stride, int seq_stride, int max_seq_length)
{
    int batch_id = blockIdx.x;
    int seqlen   = cu_seqlens[batch_id + 1] - cu_seqlens[batch_id];

    query_ptr += batch_id * batch_stride;
    for (int tid = threadIdx.x; tid < batch_stride; tid += blockDim.x) {
        int seq_id = (tid / seq_stride) % max_seq_length;
        if (seq_id >= seqlen) {
            query_ptr[tid] = scalar_t(0.0f);
        }
    }
}

template<typename scalar_t>
void pad_query(scalar_t*    query_ptr,
               const int*   cu_seqlens,
               int          batch_size,
               int          batch_stride,
               int          seq_stride,
               int          max_seq_length,
               hipStream_t stream)
{
    pad_query_kernel<<<batch_size, 512, 0, stream>>>(
        query_ptr, cu_seqlens, batch_size, batch_stride, seq_stride, max_seq_length);
}

template<typename scalar_t>
__global__ void
pad_out_kernel(scalar_t* out_ptr, const int* cu_seqlens, int batch_size, int batch_stride, int seq_stride)
{
    int seqlen = cu_seqlens[batch_size];

    for (int tid = blockIdx.x * blockDim.x + threadIdx.x; tid < batch_size * batch_stride;
         tid += blockDim.x * gridDim.x) {
        int seq_id = (tid / seq_stride);
        if (seq_id >= seqlen) {
            out_ptr[tid] = scalar_t(0.0f);
        }
    }
}

template<typename scalar_t>
void pad_out(
    scalar_t* out_ptr, const int* cu_seqlens, int batch_size, int batch_stride, int seq_stride, hipStream_t stream)
{
    pad_out_kernel<<<batch_size, 512, 0, stream>>>(out_ptr, cu_seqlens, batch_size, batch_stride, seq_stride);
}

template<typename scalar_t>
void naive_mha(scalar_t*        out_ptr,
               scalar_t*        query_ptr,
               scalar_t*        key_ptr,
               scalar_t*        val_ptr,
               scalar_t*        mask_ptr,
               scalar_t*        q_buf_ptr,
               scalar_t*        k_buf_ptr,
               scalar_t*        v_buf_ptr,
               scalar_t*        qk_buf_ptr,
               scalar_t*        out_buf_ptr,
               int*             padding_offset,
               int*             cu_seqlens,
               int              batch_size,
               int              head_num,
               int              key_len,
               int              seq_len,
               int              size_per_head,
               hipStream_t     stream,
               cublasMMWrapper* cublas_wrapper_)
{
    const scalar_t qk_scale = static_cast<scalar_t>(1.f / sqrtf(size_per_head * 1.f));
    // create

    //////////////////////////////////////////////
    /// Q,K,V
    /// transpose <B,s,h,D> -> <B,h,s,D>
    /// TODO: remove padding
    // invokeTransposeQKV(q_buf_ptr,
    //                    query_ptr,
    //                    batch_size,
    //                    head_num,
    //                    seq_len,
    //                    size_per_head,
    //                    nullptr,  // scale, only used in int8 mode
    //                    0,        // int8_mode
    //                    stream);
    // invokeTransposeQKV(k_buf_ptr,
    //                    key_ptr,
    //                    batch_size,
    //                    head_num,
    //                    key_len,
    //                    size_per_head,
    //                    nullptr,  // scale, only used in int8 mode
    //                    0,        // int8_mode
    //                    stream);
    // invokeTransposeQKV(v_buf_ptr,
    //                    val_ptr,
    //                    batch_size,
    //                    head_num,
    //                    key_len,
    //                    size_per_head,
    //                    nullptr,  // scale, only used in int8 mode
    //                    0,        // int8_mode
    //                    stream);

    q_buf_ptr = query_ptr;
    k_buf_ptr = key_ptr;
    v_buf_ptr = val_ptr;

    //////////////////////////////////////////////
    /// Q*K batch gemm
    /// -> [B, H, s, t + s]
    cublas_wrapper_->stridedBatchedGemm(HIPBLAS_OP_T,
                                        HIPBLAS_OP_N,
                                        key_len,                  // m
                                        seq_len,                  // n
                                        size_per_head,            // k
                                        k_buf_ptr,                // A
                                        size_per_head,            // lda
                                        key_len * size_per_head,  // strideA
                                        q_buf_ptr,                // B
                                        size_per_head,            // ldb
                                        seq_len * size_per_head,  // strideB
                                        qk_buf_ptr,               // C
                                        key_len,                  // ldc
                                        seq_len * key_len,        // strideC
                                        batch_size * head_num);   // batchCount

    //////////////////////////////////////////////
    /// ! masked softmax (kernel asserts k_length <= 4096)
    MaskedSoftmaxParam<scalar_t, scalar_t> param{};
    param.attention_score    = qk_buf_ptr;
    param.qk                 = qk_buf_ptr;
    param.attention_mask     = mask_ptr;
    param.batch_size         = batch_size;
    param.q_length           = seq_len;
    param.k_length           = key_len;
    param.num_heads          = head_num;
    param.qk_scale           = qk_scale;
    param.linear_bias_slopes = nullptr;
    invokeMaskedSoftmax(param, stream);

    //////////////////////////////////////////////
    /// softmax(QK)*V batch gemm
    cublas_wrapper_->stridedBatchedGemm(HIPBLAS_OP_N,
                                        HIPBLAS_OP_N,
                                        size_per_head,            // m
                                        seq_len,                  // n
                                        key_len,                  // k
                                        v_buf_ptr,                // A
                                        size_per_head,            // lda
                                        key_len * size_per_head,  // strideA,
                                        qk_buf_ptr,               // B
                                        key_len,                  // ldb
                                        key_len * seq_len,        // strideB
                                        out_buf_ptr,              // C
                                        size_per_head,            // ldc,
                                        seq_len * size_per_head,  // strideC
                                        batch_size * head_num);   // batchCount

    //////////////////////////////////////////////
    /// transpose <B,h,s,D> -> <B,s,h,D>
    int num_token = batch_size * seq_len;
    invokeTransposeAttentionOutRemovePadding(out_buf_ptr,
                                             out_ptr,
                                             num_token,
                                             batch_size,
                                             seq_len,
                                             head_num,
                                             size_per_head,
                                             padding_offset,
                                             nullptr,
                                             0,
                                             stream);

    pad_out(out_ptr, cu_seqlens, batch_size, head_num * seq_len * size_per_head, head_num * size_per_head, stream);
}

static const char* usage = "Usage: %s <batch-size> <num-heads> <key-len> <query-len> <size-per-head>\n"
                           "Example: $test_context_attention_layer 2, 8, 1024, 512, 128\n";

int main(int argc, const char* argv[])
{
    using namespace turbomind;
    using scalar_t                            = half;
    static const hipDataType kCudaDataType = std::is_same<scalar_t, half>::value ? HIP_R_16F : HIP_R_32F;

    Logger::getLogger().setLevel(Logger::INFO);

    if (argc != 6) {
        printf(usage, argv[0]);
        return EXIT_FAILURE;
    }

    // First create an instance of an engine.
    std::random_device rnd_device;
    // Specify the engine and distribution.
    std::mt19937 mersenne_engine{rnd_device()};  // Generates random integers

    int batch_size    = atoi(argv[1]);
    int num_heads     = atoi(argv[2]);
    int key_len       = atoi(argv[3]);
    int seq_len       = atoi(argv[4]);
    int size_per_head = atoi(argv[5]);

    // Create stream and handle
    hipStream_t     stream;
    hipblasHandle_t   cublas_handle;
    hipblasLtHandle_t cublaslt_handle;
    hipStreamCreate(&stream);
    hipblasCreate(&cublas_handle);
    hipblasLtCreate(&cublaslt_handle);
    hipblasSetStream(cublas_handle, stream);

    cublasAlgoMap* cublas_algo_map = new cublasAlgoMap("gemm_config.in");

    Allocator<AllocatorType::CUDA> allocator(getDevice());
    allocator.setStream(stream);
    std::mutex*     cublas_wrapper_mutex = new std::mutex();
    cublasMMWrapper cublas_wrapper =
        cublasMMWrapper(cublas_handle, cublaslt_handle, stream, cublas_algo_map, cublas_wrapper_mutex, &allocator);
    cublas_wrapper.setGemmConfig(kCudaDataType, kCudaDataType, kCudaDataType, kCudaDataType);

    // initialize device
    scalar_t* query_ptr =
        (scalar_t*)allocator.malloc(batch_size * num_heads * seq_len * size_per_head * sizeof(scalar_t));
    scalar_t* key_ptr =
        (scalar_t*)allocator.malloc(batch_size * num_heads * key_len * size_per_head * sizeof(scalar_t));
    scalar_t* val_ptr =
        (scalar_t*)allocator.malloc(batch_size * num_heads * key_len * size_per_head * sizeof(scalar_t));
    scalar_t* mask_ptr = (scalar_t*)allocator.malloc(batch_size * seq_len * key_len * sizeof(scalar_t));
    scalar_t* expect_out_ptr =
        (scalar_t*)allocator.malloc(batch_size * num_heads * seq_len * size_per_head * sizeof(scalar_t), true);
    scalar_t* actual_out_ptr =
        (scalar_t*)allocator.malloc(batch_size * num_heads * seq_len * size_per_head * sizeof(scalar_t), true);
    scalar_t* q_buf_ptr =
        (scalar_t*)allocator.malloc(batch_size * num_heads * seq_len * size_per_head * sizeof(scalar_t), true);
    scalar_t* k_buf_ptr =
        (scalar_t*)allocator.malloc(batch_size * num_heads * key_len * size_per_head * sizeof(scalar_t), true);
    scalar_t* v_buf_ptr =
        (scalar_t*)allocator.malloc(batch_size * num_heads * key_len * size_per_head * sizeof(scalar_t), true);
    scalar_t* qk_buf_ptr =
        (scalar_t*)allocator.malloc(batch_size * num_heads * seq_len * key_len * sizeof(scalar_t), true);
    scalar_t* out_buf_ptr =
        (scalar_t*)allocator.malloc(batch_size * num_heads * seq_len * size_per_head * sizeof(scalar_t), true);

    auto* h_pinned_token_num_ptr = (size_t*)allocator.malloc(sizeof(size_t), true);
    auto* padding_offset_ptr     = (int*)allocator.malloc(sizeof(int) * batch_size * seq_len, false);
    auto* cu_seqlens_ptr         = (int*)allocator.malloc(sizeof(int) * (batch_size + 1), false);
    // auto* input_lengths  = (int*)allocator.malloc(sizeof(int) * batch_size, false);
    thrust::device_vector<int> input_lengths(batch_size);
    thrust::host_vector<int>   input_lengths_host(batch_size);

    cudaRandomUniform<scalar_t>(query_ptr, batch_size * num_heads * seq_len * size_per_head);
    cudaRandomUniform<scalar_t>(key_ptr, batch_size * num_heads * key_len * size_per_head);
    cudaRandomUniform<scalar_t>(val_ptr, batch_size * num_heads * key_len * size_per_head);
    cudaRandomUniform<scalar_t>(mask_ptr, batch_size * seq_len * key_len);

    // create random length for batch
    std::uniform_int_distribution<int> dist{seq_len / 2, seq_len};
    auto                               gen = [&dist, &mersenne_engine]() { return dist(mersenne_engine); };
    std::generate(begin(input_lengths_host), end(input_lengths_host), gen);
    // for(int batch_id=0;batch_id<batch_size;++batch_id){
    //     input_lengths_host[batch_id] = seq_len;
    // }
    thrust::copy(input_lengths_host.begin(), input_lengths_host.end(), input_lengths.begin());
    size_t  h_token_num = 0;
    size_t* h_pinned_token_num;
    auto    input_lengths_ptr = thrust::raw_pointer_cast(input_lengths.data());
    hipHostMalloc((void**)&h_pinned_token_num, sizeof(size_t));
    invokeGetPaddingOffsetAndCuSeqLens(h_pinned_token_num,
                                       &h_token_num,
                                       padding_offset_ptr,
                                       cu_seqlens_ptr,
                                       input_lengths_ptr,
                                       batch_size,
                                       seq_len,
                                       stream);
    hipHostFree((void*)h_pinned_token_num);

    int* k_lens = (int*)allocator.malloc(batch_size * sizeof(int));
    deviceFill(k_lens, batch_size, key_len, stream);

    invokeCreateCausalMasks(mask_ptr, input_lengths_ptr, k_lens, seq_len, key_len, batch_size, stream);
    // deviceFill(mask_ptr, batch_size*key_len*seq_len, scalar_t(1), stream);

    // compute gt
    naive_mha<scalar_t>(expect_out_ptr,
                        query_ptr,
                        key_ptr,
                        val_ptr,
                        mask_ptr,
                        q_buf_ptr,
                        k_buf_ptr,
                        v_buf_ptr,
                        qk_buf_ptr,
                        out_buf_ptr,
                        padding_offset_ptr,
                        cu_seqlens_ptr,
                        batch_size,
                        num_heads,
                        key_len,
                        seq_len,
                        size_per_head,
                        stream,
                        &cublas_wrapper);

    // compute actual
#ifdef _MSC_VER
    static constexpr int FMHA_VERSION = 1;
#else
    static constexpr int FMHA_VERSION = 2;
#endif
    using AttentionOp = FlashAttentionOpImpl<scalar_t, FMHA_VERSION>;
    using Layout      = typename AttentionOp::AttentionLayout;
    Layout      layout_q{num_heads * seq_len * size_per_head, size_per_head, seq_len * size_per_head};
    Layout      layout_k{num_heads * key_len * size_per_head, size_per_head, key_len * size_per_head};
    Layout      layout_v{num_heads * key_len * size_per_head, size_per_head, key_len * size_per_head};
    Layout      layout_o{num_heads * seq_len * size_per_head, num_heads * size_per_head, size_per_head, true};
    AttentionOp flash_attention(batch_size, num_heads, key_len, seq_len, size_per_head);
    float*      accum_buf_ptr = (float*)allocator.malloc(flash_attention.get_workspace_size(), true);

    typename AttentionOp::Params attn_params{actual_out_ptr,
                                             query_ptr,
                                             key_ptr,
                                             val_ptr,
                                             mask_ptr,
                                             accum_buf_ptr,
                                             cu_seqlens_ptr,
                                             nullptr,
                                             1,
                                             layout_q,
                                             layout_k,
                                             layout_v,
                                             layout_o};
    flash_attention(attn_params, stream);
    sync_check_cuda_error();

    int num_rows = 8;
    // printf("query:\n");
    // printMatrix(query_ptr, num_rows, 8, size_per_head, true);
    printf("expect:\n");
    printMatrix(expect_out_ptr, num_rows, 8, size_per_head, true);
    printf("actual:\n");
    printMatrix(actual_out_ptr, num_rows, 8, size_per_head, true);
    checkResult(
        "all close:", actual_out_ptr, expect_out_ptr, batch_size * num_heads * seq_len * size_per_head, true, true);

    delete cublas_algo_map;
    delete cublas_wrapper_mutex;
}
