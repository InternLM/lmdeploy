#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2022-2023, NVIDIA CORPORATION.  All rights reserved.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <hipblas.h>
#include <iostream>
#include <vector>
#include <cstdlib>
#include <chrono>

#include "torch/csrc/cuda/Stream.h"
#include <torch/custom_class.h>
#include <torch/script.h>

#include "src/fastertransformer/kernels/cutlass_kernels/int8_gemm/int8_gemm.h"
#include "src/fastertransformer/th_op/th_utils.h"
#include "src/fastertransformer/utils/cuda_bf16_wrapper.h"
#include "src/fastertransformer/utils/logger.h"

#include "cutlass/numeric_types.h"

using torch::Tensor;
using torch_ext::get_ptr;

namespace ft = fastertransformer;

template<typename T>
void int8_gemm_test(
    const int m, 
    const int n, 
    const int k, 
    const at::ScalarType output_data_type,
    const QuantMode quant_mode,
    const int iters)
{
     const bool per_token_quant = quant_mode == QuantMode::PerTokenChannelQuant
        || quant_mode == QuantMode::PerTokenQuant;
    const bool per_channel_quant = quant_mode == QuantMode::PerTokenChannelQuant
        || quant_mode == QuantMode::PerChannelQuant;
    const int row_scale_size = per_token_quant ? m : 1;
    const int col_scale_size = per_channel_quant ? n : 1;

    const at::ScalarType at_int32 = at::ScalarType::Int;
    const at::ScalarType at_int8  = at::ScalarType::Char;
    const at::ScalarType at_fp16  = at::ScalarType::Half;
    const at::ScalarType at_bf16  = at::ScalarType::BFloat16;
    const at::ScalarType at_fp32  = at::ScalarType::Float;

    using std::chrono::high_resolution_clock;
    using std::chrono::duration_cast;
    using std::chrono::microseconds;

    torch::manual_seed(0);

    auto x = torch::randint(-128, 128, {m, k}, torch::dtype(at_int32).requires_grad(false));
    auto w = torch::randint(-128, 128, {k, n}, torch::dtype(at_int32).requires_grad(false));

    ft::FT_CHECK(torch::allclose(x, x.to(at_int8).to(at_int32)));
    ft::FT_CHECK(torch::allclose(w, w.to(at_int8).to(at_int32)));

    auto y = torch::matmul(x, w);

    ft::Tensor{ft::MEMORY_CPU, ft::TYPE_INT32, {(size_t)m, (size_t)k}, get_ptr<int32_t>(x)}.saveNpy("x.npy");
    ft::Tensor{ft::MEMORY_CPU, ft::TYPE_INT32, {(size_t)k, (size_t)n}, get_ptr<int32_t>(w)}.saveNpy("w.npy");
    ft::Tensor{ft::MEMORY_CPU, ft::TYPE_INT32, {(size_t)m, (size_t)n}, get_ptr<int32_t>(y)}.saveNpy("y.npy");

    auto x_gpu = x.to(at_int8).to(torch::kCUDA);
    auto w_T_gpu = w.to(at_int8).to(torch::kCUDA).t().contiguous();
    auto w_gpu = w.to(at_int8).to(torch::kCUDA);
    auto y_gpu = torch::zeros({m, n}, torch::dtype(output_data_type).device(torch::kCUDA).requires_grad(false));
    auto y_gpu_int32 = torch::zeros({m, n}, torch::dtype(at_int32).device(torch::kCUDA).requires_grad(false));

    auto alpha_row_cultass = torch::ones({row_scale_size, 1}, torch::dtype(at_fp32).requires_grad(false)) * (1.0 / 100) *
        torch::randint(1, 10, {row_scale_size, 1}, torch::dtype(at_fp32));
    auto alpha_col_cutlass = torch::ones({1, col_scale_size}, torch::dtype(at_fp32).requires_grad(false)) * (1.0 / 100) *
        torch::randint(1, 10, {1, col_scale_size}, torch::dtype(at_fp32));

    auto alpha_row_torch = alpha_row_cultass.expand({m, 1});
    auto alpha_col_torch = alpha_col_cutlass.expand({1, n});

    // std::cout << alpha_row << std::endl;
    auto alpha_row_gpu = alpha_row_cultass.to(torch::kCUDA);
    auto alpha_col_gpu = alpha_col_cutlass.to(torch::kCUDA);

    auto alpha_row_col_scale_gpu = torch::matmul(alpha_row_torch, alpha_col_torch).to(torch::kCUDA);

    ft::CutlassInt8GemmRunner<T> cutlass_runner_half;

    auto stream = at::cuda::getCurrentCUDAStream().stream();
    // warm_up
    cutlass_runner_half.gemm(get_ptr<int8_t>(x_gpu),
            get_ptr<int8_t>(w_T_gpu),
            quant_mode,
            get_ptr<float>(alpha_col_gpu),
            get_ptr<float>(alpha_row_gpu),
            get_ptr<T>(y_gpu),
            m,
            n,
            k,
            nullptr,
            0,
            stream);

    ft::Tensor{ft::MEMORY_GPU, ft::TYPE_INT8, {(size_t)m, (size_t)k}, get_ptr<int8_t>(x_gpu)}.saveNpy("x_gpu.npy");
    ft::Tensor{ft::MEMORY_GPU, ft::TYPE_INT8, {(size_t)n, (size_t)k}, get_ptr<int8_t>(w_T_gpu)}.saveNpy("w_T_gpu.npy");
    ft::Tensor{ft::MEMORY_GPU, ft::TYPE_INT8, {(size_t)k, (size_t)n}, get_ptr<int8_t>(w_gpu)}.saveNpy("w_gpu.npy");
    ft::Tensor{ft::MEMORY_GPU, ft::TYPE_FP16, {(size_t)m, (size_t)n}, get_ptr<T>(y_gpu)}.saveNpy("y_gpu.npy");
    ft::Tensor{ft::MEMORY_GPU, ft::TYPE_INT32, {(size_t)m, (size_t)n}, get_ptr<int32_t>(y_gpu_int32)}.saveNpy("y_gpu_int32.npy");

    ft::check_cuda_error(hipStreamSynchronize(stream));
    auto start = high_resolution_clock::now();

    for (int i = 0; i < iters; ++i) {
        cutlass_runner_half.gemm(get_ptr<int8_t>(x_gpu),
            get_ptr<int8_t>(w_T_gpu),
            quant_mode,
            get_ptr<float>(alpha_col_gpu),
            get_ptr<float>(alpha_row_gpu),
            get_ptr<T>(y_gpu),
            m,
            n,
            k,
            nullptr,
            0,
            stream);
    }

    ft::check_cuda_error(hipStreamSynchronize(stream));
    auto end = high_resolution_clock::now();

    auto duration = duration_cast<microseconds>(end - start);

    if (torch::allclose((y.to(torch::kCUDA).to(at_fp32) * alpha_row_col_scale_gpu.to(torch::kCUDA)).to(output_data_type), y_gpu)) {
        FT_LOG_INFO("SUCCESS " + std::to_string((double(duration.count()) / iters) / 1000) + " ms");
    } else {
        FT_LOG_ERROR("FAILED " + std::to_string((double(duration.count()) / iters) / 1000) + " ms");
        // std::cout << "diff " << (y.to(torch::kCUDA).to(at_fp32) * alpha_row_col_scale_gpu.to(torch::kCUDA)).to(at_fp16) - y_gpu << std::endl;
    }
}

int main(int argc, char **argv)
{
    if (argc != 7) {
        FT_LOG_ERROR("arguments missing, needs m, n, k, data_type(fp16=0, bf16=1), quant_mode (perTensor=0, perToken=1, perChannel=2, perTokenChannel=3), iters.");
        return 0;
    }

    const int m = atoi(argv[1]);
    const int n = atoi(argv[2]);
    const int k = atoi(argv[3]);
    const at::ScalarType output_data_type = atoi(argv[4]) == 0 ?
        at::ScalarType::Half : at::ScalarType::BFloat16;
    const QuantMode quant_mode = static_cast<QuantMode>(atoi(argv[5]));
    if (quant_mode == QuantMode::PerChannelQuant) {
        printf("per channel quant \n");
    }
    const int iters = atoi(argv[6]);

    if (output_data_type == at::ScalarType::Half) {
        int8_gemm_test<half>(m, n, k, output_data_type, quant_mode, iters);
    } else {
#if ENABLE_BF16
        int8_gemm_test<__hip_bfloat16>(m, n, k, output_data_type, quant_mode, iters);
#endif
    }

    return 0;
}
