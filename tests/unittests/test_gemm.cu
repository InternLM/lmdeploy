#include "hip/hip_runtime.h"
#include <assert.h>
#include <math.h>
#include <hipblas.h>
#include <numeric>
#include <stdexcept>
#include <tuple>
#include <vector>

#include "src/fastertransformer/layers/DenseWeight.h"
#include "src/fastertransformer/utils/allocator.h"
#include "src/fastertransformer/utils/cublasMMWrapper.h"
#include "src/fastertransformer/utils/cuda_utils.h"
#include "src/fastertransformer/utils/gemm.h"
#include "src/fastertransformer/utils/logger.h"
#include "src/fastertransformer/utils/memory_utils.h"

using namespace fastertransformer;

// Can be replaced by the function provided by a test framework

class TestFailureError : public std::exception {
private:
    std::string msg_;
public:
    explicit TestFailureError() = default;
    explicit TestFailureError(std::string name, std::string msg = "") {
        msg_ = fmtstr("TEST FAIL [%s] %s", name.c_str(), msg.c_str());
    }
    const char* what () const throw () {
        return msg_.c_str();
    }
};

#define EXPECT_TRUE(cond)                           \
    do { if(!(cond)) {                              \
        FT_LOG_ERROR("TEST FAIL [%s] at %s:%d",     \
                     __func__, __FILE__, __LINE__); \
        throw TestFailureError(__func__);           \
    } } while(false)

#define EXPECT_ALMOST_EQUAL(name, dtype, ctype, out, ref)       \
    do {                                                        \
        bool is_ok = checkResult<dtype,ctype>(name, out, ref);  \
        if(!is_ok) {                                            \
            FT_LOG_ERROR("TEST FAIL [%s] at %s:%d",             \
                        __func__, __FILE__, __LINE__);          \
            throw TestFailureError(__func__);                   \
        }                                                       \
    } while(false)

////////////////////////////////////////////////////////////////////////////////////

// TensorWrapper is to handle a tensor object as well as its memory buffer,
// because tensor.data is const we cannot set values.
class TensorWrapper {
private:
    IAllocator* allocator;

public:
    std::vector<size_t> shape;
    DataType type;
    Tensor* tensor;
    void* data;

    TensorWrapper(IAllocator* allocator, DataType dtype, std::vector<size_t> shape, bool zero_init = false)
    {
        this->allocator = allocator;
        this->type = dtype;
        this->shape = shape;

        size_t tensor_memsize = this->memsize();
        this->data = this->allocator->malloc(tensor_memsize, false);
        if (zero_init) {
            check_cuda_error(hipMemset(data, 0x0, tensor_memsize));
        } else {
            setRandomValues();
        }
        this->tensor = new Tensor(MEMORY_GPU, dtype, shape, data);
    }

    TensorWrapper(TensorWrapper const& other)
        : allocator(other.allocator), shape(other.shape), type(other.type), data(other.data), tensor(other.tensor)
    {
        FT_LOG_DEBUG("TensorWrapper copy: this=%p other=%p", data, other.data);
    }
    ~TensorWrapper()
    {
        delete tensor;
        allocator->free((void**)(&data));
    }

    void setInvalidValues()
    {
        size_t type_size = tensor->type == TYPE_FP32 ? sizeof(float) : sizeof(half);
        size_t tensor_size = type_size * tensor->size();
        // Fill by a random number to guarantee invalid values
        check_cuda_error(hipMemset(data, 0xdc, tensor_size));
    }

    void setRandomValues() {
        // random initialization
        size_t num_elements = this->size();
        switch (this->type) {
            case TYPE_FP32:
                cudaRandomUniform((float*)data, num_elements);
                break;
            case TYPE_FP16:
                cudaRandomUniform((half*)data, num_elements);
                break;
            default:
                // Will be added more if needed.
                throw std::runtime_error("Not supported data type");
        }
    }

    size_t size() {
        size_t n_elements = 1;
        for (size_t s : this->shape) {
            n_elements *= s;
        }
        return n_elements;
    }

    size_t memsize() {
        size_t type_size = 0;
        switch (this->type) {
            case TYPE_FP32:
                type_size = sizeof(float);
                break;
            case TYPE_FP16:
                type_size = sizeof(half);
                break;
            default:
                throw std::runtime_error("Not supported data type.");
        }
        return type_size * this->size();
    }
};

template<DataType computeType>
void computeReference(GemmOp transa,
                      GemmOp transb,
                      TensorWrapper& C,
                      TensorWrapper& A,
                      TensorWrapper& B,
                      float alpha = 1.0f,
                      float beta = 0.0f)
{
    size_t m = C.shape[0];
    size_t n = C.shape[1];
    size_t k = A.shape[1];

    size_t lda = (transa == GEMM_OP_N) ? k : m;
    size_t ldb = (transb == GEMM_OP_N) ? n : k;
    size_t ldc = n;

    hipDataType atype = (A.type == TYPE_FP16) ? HIP_R_16F : HIP_R_32F;
    hipDataType btype = (B.type == TYPE_FP16) ? HIP_R_16F : HIP_R_32F;
    hipDataType ctype = (C.type == TYPE_FP16) ? HIP_R_16F : HIP_R_32F;
    hipDataType compute_type = (computeType == TYPE_FP16) ? HIP_R_16F : HIP_R_32F;

    hipblasHandle_t cublas_handle;
    check_cuda_error(hipblasCreate(&cublas_handle));

    half h_alpha = (half)alpha;
    half h_beta = (half)beta;
    const void* _alpha = (computeType == TYPE_FP16) ? (const void*)&h_alpha : (const void*)&alpha;
    const void* _beta = (computeType == TYPE_FP16) ? (const void*)&h_beta : (const void*)&beta;

    check_cuda_error(hipblasGemmEx(cublas_handle,
                                  getCublasOperation(transb),
                                  getCublasOperation(transa),
                                  n, m, k,
                                  _alpha,
                                  (const void*)B.data, btype, ldb,
                                  (const void*)A.data, atype, lda,
                                  _beta,
                                  (void*)C.data, ctype, ldc,
                                  compute_type,
                                  HIPBLAS_GEMM_DEFAULT));
    check_cuda_error(hipblasDestroy(cublas_handle));
    hipDeviceSynchronize();
}

bool almostEqual(float a, float b, float atol = 1e-5, float rtol = 1e-8)
{
    // Params: a = value to compare and b = reference
    // This function follows implementation of numpy.isclose(), which checks
    //   abs(a - b) <= (atol + rtol * abs(b)).
    // Note that the inequality above is asymmetric where b is considered as
    // a reference value. To account into both absolute/relative errors, it
    // uses absolute tolerance and relative tolerance at the same time. The
    // default values of atol and rtol borrowed from numpy.isclose(). For the
    // case of nan value, the result will be true.
    if (isnan(a) && isnan(b)) {
        return true;
    }
    return fabs(a - b) <= (atol + rtol * fabs(b));
}

template<typename T>
bool _checkResult(std::string name, TensorWrapper& out, TensorWrapper& ref, float atol, float rtol) {
    assert(out.type == ref.type);

    size_t out_size = out.size();
    size_t ref_size = ref.size();
    T* h_out = reinterpret_cast<T*>(malloc(sizeof(T) * out_size));
    T* h_ref = reinterpret_cast<T*>(malloc(sizeof(T) * ref_size));

    hipMemcpy(h_out, out.data, sizeof(T) * out_size, hipMemcpyDeviceToHost);
    hipMemcpy(h_ref, ref.data, sizeof(T) * ref_size, hipMemcpyDeviceToHost);
    hipDeviceSynchronize();

    size_t failures = 0;
    for (size_t i = 0; i < out_size; ++i) {
        // The values for the output and the reference.
        float a = (float)h_out[i];
        float b = (float)h_ref[i];

        bool ok = almostEqual(a, b, atol, rtol);
        // Print the error.
        if( !ok && failures < 4 ) {
            FT_LOG_ERROR(">> invalid result for i=%lu:", i);
            FT_LOG_ERROR(">>    found......: %10.6f", a);
            FT_LOG_ERROR(">>    expected...: %10.6f", b);
            FT_LOG_ERROR(">>    error......: %.6f", fabsf(a - b));
            FT_LOG_ERROR(">>    tol........: %.6f", atol + rtol * fabs(b));
        }

        // Update the number of failures.
        failures += ok ? 0 : 1;
    }

    // Allow not matched up to 1% elements.
    size_t tol_failures = (size_t)(0.01 * out_size);
    FT_LOG_INFO("check....... %30s : %s (failures: %.2f%% atol: %.2e rtol: %.2e)",
                name.c_str(), failures <= tol_failures ? "OK" : "FAILED",
                100. * failures / out_size, atol, rtol);
    return failures <= tol_failures;
}

template<typename T, DataType computeType>
bool checkResult(std::string name, TensorWrapper& out, TensorWrapper& ref) {
    float atol = (computeType == TYPE_FP32) ? 1e-6f : 1e-3f;
    float rtol = (computeType == TYPE_FP32) ? 1e-4f : 1e-1f;
    bool is_ok = false;
    if (sizeof(T) == 4) {
        is_ok = _checkResult<float>(name, out, ref, atol, rtol);
    } else {
        is_ok = _checkResult<half>(name, out, ref, atol, rtol);
    }
    return is_ok;
}

template<typename T, DataType computeType>
bool checkResult(TensorWrapper& out, TensorWrapper& ref) {
    return checkResult<T, computeType>("", out, ref);
}

template<typename T>
std::string toString() {
    std::string str = "dtype=";
    str += std::is_same<T, float>::value ? "FP32" : "FP16";
    return str;
}

template<typename T, DataType ctype>
std::string toString() {
    std::string str = "dtype=";
    str += std::is_same<T, float>::value ? "FP32" : "FP16";
    str += ", compute_type=";
    str += (ctype == TYPE_FP32) ? "FP32" : "FP16";
    return str;
}

std::string toString(GemmOp op) {
    return op == GEMM_OP_N ? "N" : "T";
}

struct GemmOpPair {
    GemmOp transa;
    GemmOp transb;
};

static const std::vector<GemmOpPair> op_pairs {{GEMM_OP_N, GEMM_OP_N},
                                               {GEMM_OP_N, GEMM_OP_T},
                                               {GEMM_OP_T, GEMM_OP_N},
                                               {GEMM_OP_T, GEMM_OP_T}};

static inline std::string getTestName(const char* func_name, GemmOp transa, GemmOp transb,
                                      size_t m, size_t n, size_t k)
{
    return fmtstr("%s [opA=%s, opB=%s, m=%ld, n=%ld, k=%ld]",
                  func_name, getGemmOpString(transa).c_str(), getGemmOpString(transb).c_str(),
                  m, n, k);
}

static inline std::string getTestName(const char* func_name, GemmOpPair op_pairs,
                                      size_t m, size_t n, size_t k)
{
    return getTestName(func_name, op_pairs.transa, op_pairs.transb, m, n, k);
}


/////////////////////////////////// Unittests //////////////////////////////////////////

template<typename T, DataType computeType>
void testGemmCorrectnessMatmul(size_t m, size_t n, size_t k) {
    FT_LOG_INFO("Matmul function correctness test [m=%ld, n=%ld, k=%ld, %s]",
                m, n, k, toString<T, computeType>().c_str());
    hipStream_t stream;
    check_cuda_error(hipStreamCreate(&stream));

    Allocator<AllocatorType::CUDA> allocator(getDevice());

    DataType dtype = getTensorType<T>();
    TensorWrapper a_tensor(&allocator, dtype, {m, k}, false);
    TensorWrapper b_tensor(&allocator, dtype, {k, n}, false);
    TensorWrapper c_tensor(&allocator, dtype, {m, n}, true);
    TensorWrapper expected(&allocator, dtype, {m, n}, true);

    std::shared_ptr<Gemm> gemm = createGemm(&allocator, stream, false, false);
    gemm->setTypes(a_tensor.type, b_tensor.type, c_tensor.type, computeType);

    for (auto &op_pair : op_pairs) {
        std::string tc_name = getTestName(__func__, op_pair, m, n, k);
        FT_LOG_DEBUG(tc_name);
        computeReference<computeType>(op_pair.transa, op_pair.transb,
                                      expected, a_tensor, b_tensor);

        size_t lda = (op_pair.transa == GEMM_OP_N) ? k : m;
        size_t ldb = (op_pair.transb == GEMM_OP_N) ? n : k;
        size_t ldc = n;

        c_tensor.setInvalidValues(); // to guarantee C has invalid data
        gemm->gemm(op_pair.transa, op_pair.transb, m, n, k,
                   a_tensor.data, a_tensor.type, lda,
                   b_tensor.data, b_tensor.type, ldb,
                   c_tensor.data, c_tensor.type, ldc);
        EXPECT_ALMOST_EQUAL(tc_name + " api1", T, computeType, c_tensor, expected);

        c_tensor.setInvalidValues();
        gemm->gemm(op_pair.transa, op_pair.transb, m, n, k,
                   a_tensor.data, lda,
                   b_tensor.data, ldb,
                   c_tensor.data, ldc);
        EXPECT_ALMOST_EQUAL(tc_name + " api2", T, computeType, c_tensor, expected);

        c_tensor.setInvalidValues();
        gemm->gemm(op_pair.transa, op_pair.transb, m, n, k,
                   a_tensor.data, b_tensor.data, c_tensor.data);
        EXPECT_ALMOST_EQUAL(tc_name + " api3", T, computeType, c_tensor, expected);

        c_tensor.setInvalidValues();
        gemm->gemm(op_pair.transa, op_pair.transb, m, n, k,
                    a_tensor.data, DenseWeight<T>{(const T*)b_tensor.data, nullptr, nullptr}, c_tensor.data);
        EXPECT_ALMOST_EQUAL(tc_name + " api4", T, computeType, c_tensor, expected);
    }
    check_cuda_error(hipStreamDestroy(stream));
}

template<typename T, DataType computeType>
void testGemmConsistencyMatmul(size_t m, size_t n, size_t k) {
    // Test if Gemm is consistent with cublasWrapper
    FT_LOG_INFO("Matmul function consistency test [m=%ld, n=%ld, k=%ld, %s]",
                m, n, k, toString<T, computeType>().c_str());

    Allocator<AllocatorType::CUDA> allocator(getDevice());
    hipStream_t stream;
    check_cuda_error(hipStreamCreate(&stream));

    DataType dtype = getTensorType<T>();
    TensorWrapper a_tensor(&allocator, dtype, {m, k}, false);
    TensorWrapper b_tensor(&allocator, dtype, {k, n}, false);
    TensorWrapper c_tensor(&allocator, dtype, {m, n}, true);
    TensorWrapper expected(&allocator, dtype, {m, n}, true);

    hipblasHandle_t cublas_handle;
    hipblasLtHandle_t cublaslt_handle;
    check_cuda_error(hipblasCreate(&cublas_handle));
    check_cuda_error(hipblasLtCreate(&cublaslt_handle));
    check_cuda_error(hipblasSetStream(cublas_handle, stream));
    cublasAlgoMap cublas_algo_map(GEMM_CONFIG);
    std::mutex* cublas_wrapper_mutex = new std::mutex();
    cublasMMWrapper cublas_wrapper(cublas_handle,
                                   cublaslt_handle,
                                   stream,
                                   &cublas_algo_map,
                                   cublas_wrapper_mutex,
                                   &allocator);

    hipDataType cuda_dtype = std::is_same<float, T>::value ? HIP_R_32F : HIP_R_16F;
    hipDataType cuda_ctype = (DataType::TYPE_FP32 == computeType) ? HIP_R_32F : HIP_R_16F;
    cublas_wrapper.setGemmConfig(cuda_dtype, cuda_dtype, cuda_dtype, cuda_ctype);

    std::shared_ptr<Gemm> gemm = createGemm(&allocator, stream, false, false);
    gemm->setTypes(a_tensor.type, b_tensor.type, c_tensor.type, computeType);

    for (auto &op_pair : op_pairs) {
        std::string tc_name = getTestName(__func__, op_pair, m, n, k);

        // Switch A/B because Gemm expects column major layout as cublas does.
        size_t lda = (op_pair.transa == GEMM_OP_N) ? k : m;
        size_t ldb = (op_pair.transb == GEMM_OP_N) ? n : k;
        size_t ldc = n;
        cublas_wrapper.Gemm(getCublasOperation(op_pair.transb),
                            getCublasOperation(op_pair.transa),
                            n, m, k,
                            b_tensor.data, ldb,
                            a_tensor.data, lda,
                            expected.data, ldc);

        c_tensor.setInvalidValues(); // to guarantee C has invalid data
        gemm->gemm(op_pair.transa, op_pair.transb, m, n, k,
                   a_tensor.data, a_tensor.type, lda,
                   b_tensor.data, b_tensor.type, ldb,
                   c_tensor.data, c_tensor.type, ldc);
        EXPECT_ALMOST_EQUAL(tc_name + " api1", T, computeType, c_tensor, expected);

        c_tensor.setInvalidValues();
        gemm->gemm(op_pair.transa, op_pair.transb, m, n, k,
                   a_tensor.data, lda,
                   b_tensor.data, ldb,
                   c_tensor.data, ldc);
        EXPECT_ALMOST_EQUAL(tc_name + " api2", T, computeType, c_tensor, expected);

        c_tensor.setInvalidValues();
        gemm->gemm(op_pair.transa, op_pair.transb, m, n, k,
                   a_tensor.data, b_tensor.data, c_tensor.data);
        EXPECT_ALMOST_EQUAL(tc_name + " api3", T, computeType, c_tensor, expected);

        c_tensor.setInvalidValues();
        gemm->gemm(op_pair.transa, op_pair.transb, m, n, k,
                    a_tensor.data, DenseWeight<T>{(const T*)b_tensor.data, nullptr, nullptr}, c_tensor.data);
        EXPECT_ALMOST_EQUAL(tc_name + " api4", T, computeType, c_tensor, expected);
    }

    delete cublas_wrapper_mutex;
    check_cuda_error(hipblasLtDestroy(cublaslt_handle));
    check_cuda_error(hipblasDestroy(cublas_handle));
    check_cuda_error(hipStreamDestroy(stream));
}

template<typename T, DataType computeType>
void testGemmConsistencyBatchedMatmul(size_t m, size_t n, size_t k) {
    // Test if Gemm is consistent with cublasWrapper
    FT_LOG_INFO("Batched gemm function consistency test [m=%ld, n=%ld, k=%ld, %s]",
                m, n, k, toString<T, computeType>().c_str());

    Allocator<AllocatorType::CUDA> allocator(getDevice());
    hipStream_t stream;
    check_cuda_error(hipStreamCreate(&stream));

    // batch of in/out tensors
    DataType a_type = getTensorType<T>();
    DataType b_type = getTensorType<T>();
    DataType c_type = getTensorType<T>();
    std::vector<TensorWrapper*> a_tensors;
    std::vector<TensorWrapper*> b_tensors;
    std::vector<TensorWrapper*> c_tensors;
    std::vector<TensorWrapper*> expecteds;
    const size_t batch_size = 3;
    for (size_t i = 0; i < batch_size; ++i) {
        a_tensors.push_back(new TensorWrapper(&allocator, a_type, {m, k}, false));
        b_tensors.push_back(new TensorWrapper(&allocator, b_type, {k, n}, false));
        c_tensors.push_back(new TensorWrapper(&allocator, c_type, {m, n}, true));
        expecteds.push_back(new TensorWrapper(&allocator, c_type, {m, n}, true));
    }

    const T* hA[]{(const T*)a_tensors[0]->data,
                  (const T*)a_tensors[1]->data,
                  (const T*)a_tensors[2]->data,
                  nullptr,  // for memory alignment.
                  (const T*)b_tensors[0]->data,
                  (const T*)b_tensors[1]->data,
                  (const T*)b_tensors[2]->data,
                  nullptr,  // for memory alignment.
                  (const T*)c_tensors[0]->data,
                  (const T*)c_tensors[1]->data,
                  (const T*)c_tensors[2]->data,
                  nullptr,  // for memory alignment.
                  (const T*)expecteds[0]->data,
                  (const T*)expecteds[1]->data,
                  (const T*)expecteds[2]->data};

    T** batch_tensor_ptrs = reinterpret_cast<T**>(allocator.malloc(sizeof(T*) * 16, false));
    check_cuda_error(hipMemcpyAsync(
        (void*)batch_tensor_ptrs, hA, sizeof(T*) * 16, hipMemcpyHostToDevice, stream));
    const void* const* batch_a = reinterpret_cast<const void* const*>(batch_tensor_ptrs);
    const void* const* batch_b = reinterpret_cast<const void* const*>(batch_tensor_ptrs + 4);
    void* const* batch_c = reinterpret_cast<void* const*>(batch_tensor_ptrs + 8);
    void* const* batch_expected = reinterpret_cast<void* const*>(batch_tensor_ptrs + 12);

    hipblasHandle_t cublas_handle;
    hipblasLtHandle_t cublaslt_handle;
    check_cuda_error(hipblasCreate(&cublas_handle));
    check_cuda_error(hipblasLtCreate(&cublaslt_handle));
    check_cuda_error(hipblasSetStream(cublas_handle, stream));
    cublasAlgoMap cublas_algo_map(GEMM_CONFIG);
    std::mutex* cublas_wrapper_mutex = new std::mutex();
    cublasMMWrapper cublas_wrapper(cublas_handle,
                                   cublaslt_handle,
                                   stream,
                                   &cublas_algo_map,
                                   cublas_wrapper_mutex,
                                   &allocator);

    hipDataType dtype = std::is_same<float, T>::value ? HIP_R_32F : HIP_R_16F;
    hipDataType ctype = (computeType == DataType::TYPE_FP32) ? HIP_R_32F : HIP_R_16F;
    cublas_wrapper.setGemmConfig(dtype, dtype, dtype, ctype);

    std::shared_ptr<Gemm> gemm = createGemm(&allocator, stream, false, false);
    gemm->setTypes(a_type, b_type, c_type, computeType);

    for (auto &op_pair : op_pairs) {
        std::string tc_name = getTestName(__func__, op_pair, m, n, k);
        FT_LOG_DEBUG(tc_name);

        size_t lda = (op_pair.transa == GEMM_OP_N) ? k : m;
        size_t ldb = (op_pair.transb == GEMM_OP_N) ? n : k;
        size_t ldc = n;

        // Switch A/B because Gemm expects column major layout as cublas does.
        cublas_wrapper.batchedGemm(getCublasOperation(op_pair.transb),  // N
                                   getCublasOperation(op_pair.transa),  // T
                                   n,
                                   m,
                                   k,
                                   (const void* const*)batch_b, ldb,
                                   (const void* const*)batch_a, lda,
                                   (void* const*)batch_expected, ldc,
                                   batch_size);

        gemm->batchedGemm(op_pair.transa, op_pair.transb, m, n, k,
                          batch_a, a_type, lda,
                          batch_b, b_type, ldb,
                          batch_c, c_type, ldc,
                          batch_size);
        for (size_t i = 0; i < batch_size; ++i) {
            EXPECT_ALMOST_EQUAL(tc_name + " api1 batch" + std::to_string(i),
                                T, computeType, *c_tensors[i], *expecteds[i]);
        }

        for (size_t i = 0; i < batch_size; ++i) {
            c_tensors[i]->setInvalidValues();
        }
        gemm->batchedGemm(op_pair.transa, op_pair.transb, m, n, k,
                          batch_a, lda,
                          batch_b, ldb,
                          batch_c, ldc,
                          batch_size);
        for (size_t i = 0; i < batch_size; ++i) {
            EXPECT_ALMOST_EQUAL(tc_name + " api2 batch" + std::to_string(i),
                                T, computeType, *c_tensors[i], *expecteds[i]);
        }

        for (size_t i = 0; i < batch_size; ++i) {
            c_tensors[i]->setInvalidValues();
        }
        gemm->batchedGemm(op_pair.transa, op_pair.transb, m, n, k,
                          batch_a, batch_b, batch_c, batch_size);
        for (size_t i = 0; i < batch_size; ++i) {
            EXPECT_ALMOST_EQUAL(tc_name + " api3 batch" + std::to_string(i),
                                T, computeType, *c_tensors[i], *expecteds[i]);
        }
    }
    a_tensors.clear();
    b_tensors.clear();
    c_tensors.clear();
    expecteds.clear();
    delete cublas_wrapper_mutex;
    check_cuda_error(hipblasLtDestroy(cublaslt_handle));
    check_cuda_error(hipblasDestroy(cublas_handle));
    check_cuda_error(hipStreamDestroy(stream));
}


template<typename T, DataType computeType>
void testGemmConsistencyStridedBatchedMatmul(size_t batch_size, size_t m, size_t n, size_t k) {
    // Test if Gemm is consistent with cublasWrapper
    FT_LOG_INFO("Strided batched gemm function consistency test [bsz=%ld, m=%ld, n=%ld, k=%ld, %s]",
                batch_size, m, n, k, toString<T, computeType>().c_str());

    Allocator<AllocatorType::CUDA> allocator(getDevice());
    hipStream_t stream;
    check_cuda_error(hipStreamCreate(&stream));

    DataType data_type = getTensorType<T>();
    TensorWrapper a_tensor(&allocator, data_type, {batch_size, m, k}, false);
    TensorWrapper b_tensor(&allocator, data_type, {batch_size, k, n}, false);
    TensorWrapper c_tensor(&allocator, data_type, {batch_size, m, n}, true);
    TensorWrapper expected(&allocator, data_type, {batch_size, m, n}, true);

    hipblasHandle_t cublas_handle;
    hipblasLtHandle_t cublaslt_handle;
    check_cuda_error(hipblasCreate(&cublas_handle));
    check_cuda_error(hipblasLtCreate(&cublaslt_handle));
    check_cuda_error(hipblasSetStream(cublas_handle, stream));
    cublasAlgoMap cublas_algo_map(GEMM_CONFIG);
    std::mutex* cublas_wrapper_mutex = new std::mutex();
    cublasMMWrapper cublas_wrapper(cublas_handle,
                                   cublaslt_handle,
                                   stream,
                                   &cublas_algo_map,
                                   cublas_wrapper_mutex,
                                   &allocator);

    hipDataType dtype = std::is_same<float, T>::value ? HIP_R_32F : HIP_R_16F;
    hipDataType ctype = (computeType == DataType::TYPE_FP32) ? HIP_R_32F : HIP_R_16F;
    cublas_wrapper.setGemmConfig(dtype, dtype, dtype, ctype);

    std::shared_ptr<Gemm> gemm = createGemm(&allocator, stream, false, false);
    gemm->setTypes(a_tensor.type, b_tensor.type, c_tensor.type, computeType);

    for (auto &op_pair : op_pairs) {
        std::string tc_name = getTestName(__func__, op_pair, m, n, k);

        // Switch A/B because Gemm expects column major layout as cublas does.
        size_t lda = (op_pair.transa == GEMM_OP_N) ? k : m;
        size_t ldb = (op_pair.transb == GEMM_OP_N) ? n : k;
        size_t ldc = n;

        int64_t stridea = m * k;
        int64_t strideb = k * n;
        int64_t stridec = m * n;

        float alpha = 1.0f;
        float beta = 0.0f;

        cublas_wrapper.stridedBatchedGemm(getCublasOperation(op_pair.transb),
                                          getCublasOperation(op_pair.transa),
                                          n,
                                          m,
                                          k,
                                          alpha,
                                          b_tensor.data,
                                          getCublasDataType(b_tensor.type),
                                          ldb,
                                          strideb,
                                          a_tensor.data,
                                          getCublasDataType(a_tensor.type),
                                          lda,
                                          stridea,
                                          beta,
                                          expected.data,
                                          getCublasDataType(expected.type),
                                          ldc,
                                          stridec,
                                          batch_size,
                                          getCublasDataType(computeType));

        c_tensor.setInvalidValues();  // to guarantee C has invalid data
        gemm->stridedBatchedGemm(op_pair.transa, op_pair.transb, m, n, k,
                                 a_tensor.data, a_tensor.type, lda, stridea,
                                 b_tensor.data, b_tensor.type, ldb, strideb,
                                 c_tensor.data, c_tensor.type, ldc, stridec,
                                 batch_size, computeType, alpha, beta);
        EXPECT_ALMOST_EQUAL(tc_name + " api1", T, computeType, c_tensor, expected);

        c_tensor.setInvalidValues();
        gemm->stridedBatchedGemm(op_pair.transa, op_pair.transb, m, n, k,
                                 a_tensor.data, lda, stridea,
                                 b_tensor.data, ldb, strideb,
                                 c_tensor.data, ldc, stridec,
                                 batch_size, alpha, beta);
        EXPECT_ALMOST_EQUAL(tc_name + " api2", T, computeType, c_tensor, expected);

        c_tensor.setInvalidValues();
        gemm->stridedBatchedGemm(op_pair.transa, op_pair.transb, m, n, k,
                                 a_tensor.data, stridea,
                                 b_tensor.data, strideb,
                                 c_tensor.data, stridec,
                                 batch_size, alpha, beta);
        EXPECT_ALMOST_EQUAL(tc_name + " api3", T, computeType, c_tensor, expected);

        c_tensor.setInvalidValues();
        gemm->stridedBatchedGemm(op_pair.transa, op_pair.transb, m, n, k,
                                 a_tensor.data,
                                 b_tensor.data,
                                 c_tensor.data,
                                 batch_size, alpha, beta);
        EXPECT_ALMOST_EQUAL(tc_name + " api4", T, computeType, c_tensor, expected);
    }

    delete cublas_wrapper_mutex;
    check_cuda_error(hipblasLtDestroy(cublaslt_handle));
    check_cuda_error(hipblasDestroy(cublas_handle));
    check_cuda_error(hipStreamDestroy(stream));
}

#ifdef SPARSITY_ENABLED
// The current SpGemm only supports TYPE_FP16 for T, computeType,
// but let us keep these template variables for later use.
template<typename T, DataType computeType>
void testSpGemmCorrectnessMatmul(size_t m, size_t n, size_t k) {
    FT_LOG_INFO("Sparse gemm function correctness test [m=%ld, n=%ld, k=%ld, %s]",
                m, n, k, toString<T, computeType>().c_str());
    hipStream_t stream;
    check_cuda_error(hipStreamCreate(&stream));

    Allocator<AllocatorType::CUDA> allocator(getDevice());

    DataType dtype = getTensorType<T>();
    TensorWrapper a_tensor(&allocator, dtype, {m, k}, false);
    TensorWrapper b_tensor(&allocator, dtype, {k, n}, false);
    TensorWrapper c_tensor(&allocator, dtype, {m, n}, true);
    TensorWrapper expected(&allocator, dtype, {m, n}, true);

    std::shared_ptr<Gemm> gemm = createGemm(&allocator, stream, true, false);
    gemm->setTypes(a_tensor.type, b_tensor.type, c_tensor.type, computeType);

    for (auto &op_pair : op_pairs) {
        // A/B will be switched in SpGemm.
        std::string tc_name = getTestName(__func__, op_pair, m, n, k);
        FT_LOG_DEBUG(tc_name);

        b_tensor.setRandomValues();
        pruneMatrixB(b_tensor.data, stream,
                     b_tensor.shape[0], b_tensor.shape[1], op_pair.transb);
        computeReference<computeType>(op_pair.transa, op_pair.transb,
                                      expected, a_tensor, b_tensor);

        void* b_compressed;
        compressMatrixB(&b_compressed, allocator, stream,
                        b_tensor.data, b_tensor.shape[0], b_tensor.shape[1],
                        op_pair.transb);

        size_t lda = (op_pair.transa == GEMM_OP_N) ? k : m;
        size_t ldb = (op_pair.transb == GEMM_OP_N) ? n : k;
        size_t ldc = n;

        c_tensor.setInvalidValues(); // to guarantee C has invalid data
        gemm->gemm(op_pair.transa, op_pair.transb, m, n, k,
                   a_tensor.data, a_tensor.type, lda,
                   b_compressed, b_tensor.type, ldb,
                   c_tensor.data, c_tensor.type, ldc);
        EXPECT_ALMOST_EQUAL(tc_name + " api1", T, computeType, c_tensor, expected);

        c_tensor.setInvalidValues();
        gemm->gemm(op_pair.transa, op_pair.transb, m, n, k,
                   a_tensor.data, lda,
                   b_compressed, ldb,
                   c_tensor.data, ldc);
        EXPECT_ALMOST_EQUAL(tc_name + " api2", T, computeType, c_tensor, expected);

        c_tensor.setInvalidValues();
        gemm->gemm(op_pair.transa, op_pair.transb, m, n, k,
                   a_tensor.data, b_compressed, c_tensor.data);
        EXPECT_ALMOST_EQUAL(tc_name + " api3", T, computeType, c_tensor, expected);

        c_tensor.setInvalidValues();
        gemm->gemm(op_pair.transa, op_pair.transb, m, n, k,
                   a_tensor.data,
                   DenseWeight<T>{(const T*)b_tensor.data, nullptr, (const T*)b_compressed},
                   c_tensor.data);
        EXPECT_ALMOST_EQUAL(tc_name + " api4", T, computeType, c_tensor, expected);

        allocator.free((void**)(&b_compressed));
    }
    check_cuda_error(hipStreamDestroy(stream));
}

template<typename T, DataType computeType>
void testSpGemmConsistencyMatmul(size_t m, size_t n, size_t k) {
    // Test if Gemm is consistent with cublasWrapper
    FT_LOG_INFO("Sparse Matmul function consistency test [m=%ld, n=%ld, k=%ld, %s]",
                m, n, k, toString<T, computeType>().c_str());

    Allocator<AllocatorType::CUDA> allocator(getDevice());
    hipStream_t stream;
    check_cuda_error(hipStreamCreate(&stream));

    DataType dtype = getTensorType<T>();
    TensorWrapper a_tensor(&allocator, dtype, {m, k}, false);
    TensorWrapper b_tensor(&allocator, dtype, {k, n}, false);
    TensorWrapper c_tensor(&allocator, dtype, {m, n}, true);
    TensorWrapper expected(&allocator, dtype, {m, n}, true);

    hipblasHandle_t cublas_handle;
    hipblasLtHandle_t cublaslt_handle;
    check_cuda_error(hipblasCreate(&cublas_handle));
    check_cuda_error(hipblasLtCreate(&cublaslt_handle));
    check_cuda_error(hipblasSetStream(cublas_handle, stream));
    cublasAlgoMap cublas_algo_map(GEMM_CONFIG);
    std::mutex* cublas_wrapper_mutex = new std::mutex();
    cublasMMWrapper cublas_wrapper(cublas_handle,
                                   cublaslt_handle,
                                   stream,
                                   &cublas_algo_map,
                                   cublas_wrapper_mutex,
                                   &allocator);

    hipDataType cu_dtype = std::is_same<float, T>::value ? HIP_R_32F : HIP_R_16F;
    hipDataType cu_ctype = (DataType::TYPE_FP32 == computeType) ? HIP_R_32F : HIP_R_16F;
    cublas_wrapper.setGemmConfig(cu_dtype, cu_dtype, cu_dtype, cu_ctype);

    std::shared_ptr<Gemm> gemm = createGemm(&allocator, stream, true, false);
    gemm->setTypes(a_tensor.type, b_tensor.type, c_tensor.type, computeType);

    for (auto &op_pair : op_pairs) {
        std::string tc_name = getTestName(__func__, op_pair, m, n, k);
        FT_LOG_DEBUG(tc_name);

        b_tensor.setRandomValues();
        pruneMatrixB(b_tensor.data, stream,
                     b_tensor.shape[0], b_tensor.shape[1], op_pair.transb);

        // Switch A/B because Gemm expects column major layout as cublas does.
        size_t lda = (op_pair.transa == GEMM_OP_N) ? k : m;
        size_t ldb = (op_pair.transb == GEMM_OP_N) ? n : k;
        size_t ldc = n;
        cublas_wrapper.Gemm(getCublasOperation(op_pair.transb),
                            getCublasOperation(op_pair.transa),
                            n,
                            m,
                            k,
                            b_tensor.data, ldb,
                            a_tensor.data, lda,
                            expected.data, ldc);

        void* b_compressed;
        compressMatrixB(&b_compressed, allocator, stream,
                        b_tensor.data, b_tensor.shape[0], b_tensor.shape[1],
                        op_pair.transb);

        c_tensor.setInvalidValues();  // to guarantee C has invalid data
        gemm->gemm(op_pair.transa, op_pair.transb, m, n, k,
                   a_tensor.data, a_tensor.type, lda,
                   b_compressed, b_tensor.type, ldb,
                   c_tensor.data, c_tensor.type, ldc);
        EXPECT_ALMOST_EQUAL(tc_name + " api1", T, computeType, c_tensor, expected);

        c_tensor.setInvalidValues();
        gemm->gemm(op_pair.transa, op_pair.transb,  m, n, k,
                   a_tensor.data, lda,
                   b_compressed, ldb,
                   c_tensor.data, ldc);
        EXPECT_ALMOST_EQUAL(tc_name + " api1", T, computeType, c_tensor, expected);

        c_tensor.setInvalidValues();
        gemm->gemm(op_pair.transa, op_pair.transb, m, n, k,
                   a_tensor.data, b_compressed, c_tensor.data);
        EXPECT_ALMOST_EQUAL(tc_name + " api3", T, computeType, c_tensor, expected);
    }

    delete cublas_wrapper_mutex;
    check_cuda_error(hipblasLtDestroy(cublaslt_handle));
    check_cuda_error(hipblasDestroy(cublas_handle));
    check_cuda_error(hipStreamDestroy(stream));
}
#endif

int main(int argc, char* argv[]) {
    // testGemmCreate();
    using testcase_t = std::tuple<size_t, size_t, size_t>;

    std::vector<testcase_t> testcases = {{16, 32, 64},
                                         {255, 255, 255},
                                         {1041, 2047, 9999},
                                         {1041, 1, 9999},
                                         {1041, 999, 1}};

    // Computation correctness tests
    for (testcase_t &tc : testcases) {
        size_t m = std::get<0>(tc);
        size_t n = std::get<1>(tc);
        size_t k = std::get<2>(tc);

        testGemmCorrectnessMatmul<float, TYPE_FP32>(m, n, k);
        testGemmCorrectnessMatmul<half, TYPE_FP32>(m, n, k);
        testGemmCorrectnessMatmul<half, TYPE_FP16>(m, n, k);

        testGemmConsistencyMatmul<float, TYPE_FP32>(m, n, k);
        testGemmConsistencyMatmul<half, TYPE_FP32>(m, n, k);
        testGemmConsistencyMatmul<half, TYPE_FP16>(m, n, k);

        testGemmConsistencyBatchedMatmul<float, TYPE_FP32>(m, n, k);
        testGemmConsistencyBatchedMatmul<half, TYPE_FP32>(m, n, k);
        testGemmConsistencyBatchedMatmul<half, TYPE_FP16>(m, n, k);

        testGemmConsistencyStridedBatchedMatmul<float, TYPE_FP32>(7, m, n, k);
        testGemmConsistencyStridedBatchedMatmul<half, TYPE_FP32>(7, m, n, k);
        testGemmConsistencyStridedBatchedMatmul<half, TYPE_FP16>(7, m, n, k);
    }

#ifdef SPARSITY_ENABLED
    // Reset for SpGemm test.
    testcases.clear();
    testcases.insert(testcases.end(),
                    {{8, 32, 32},  // minimum possible example.
                     {8, 32, 64},
                     {64, 64, 64},
                     {16, 32, 64},
                     {1024, 32, 1024},
                     {1024, 1024, 32},
                     {16, 1024, 1024},
                     {1024, 1024, 1024}});

    for (testcase_t &tc : testcases) {
        size_t m = std::get<0>(tc);
        size_t n = std::get<1>(tc);
        size_t k = std::get<2>(tc);
        testSpGemmCorrectnessMatmul<half, TYPE_FP16>(m, n, k);
        testSpGemmConsistencyMatmul<half, TYPE_FP16>(m, n, k);
    }
#endif
    FT_LOG_INFO("Test done");
    return 0;
}
