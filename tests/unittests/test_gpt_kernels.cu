#include <vector>
#include <random>

#include "src/fastertransformer/kernels/gpt_kernels.h"
#include "src/fastertransformer/utils/memory_utils.h"

#include "unittest_utils.h"

int test_find_context_dups();
int test_compact();
int test_uncompact();

int main(int argc, char* argv[])
{
    bool all_passed = true;
    bool passed;

    passed = test_find_context_dups() == EXIT_SUCCESS;
    all_passed |= passed;
    printf("%s", passed ? "." : "X");
    if (!passed) {
        puts("\ntest_find_context_dups: FAILED");
    }

    passed = test_compact() == EXIT_SUCCESS;
    all_passed |= passed;
    printf("%s", passed ? "." : "X");
    if (!passed) {
        puts("\ntest_compact: FAILED");
    }

    passed = test_uncompact() == EXIT_SUCCESS;
    all_passed |= passed;
    printf("%s", passed ? "." : "X");
    if (!passed) {
        puts("\ntest_uncompact: FAILED");
    }

    puts("");
    return all_passed ? EXIT_SUCCESS : EXIT_FAILURE;
}

int test_find_context_dups()
{
    const size_t vec_size = 1234;
    const size_t batch_size = 8;
    // Reference to the first unique vector
    const std::vector<int> shared_contexts_ref {0, 0, 2, 3, 4, 4, 3, 3};

    // Which compact index belong to what vector
    const std::vector<int> batch_idx_to_compact_idx {0, 0, 1, 2, 3, 3, 2, 2};
    std::vector<int> batch_idx_to_compact_idx_test(batch_size);

    // Reverse map of batch_idx_to_compact_idx
    const std::vector<int> compact_idx_to_batch_idx {0, 2, 3, 4, -1, -1, -1, -1};
    std::vector<int> compact_idx_to_batch_idx_test(batch_size, -1);

    std::vector<int> input_ids;
    std::vector<int> default_vector(vec_size, 0);

    for (size_t i = 0; i < batch_size; ++i) {
        default_vector[vec_size - 1] = shared_contexts_ref[i];
        input_ids.insert(input_ids.end(), default_vector.begin(), default_vector.end());
    }

    std::vector<int> shared_contexts_test(batch_size);

    int* d_input_ids;
    int* d_shared_contexts_test;
    int* d_batch_idx_to_compact_idx;
    int* d_compact_to_batch;
    int* d_compact_size;
    hipMalloc(&d_input_ids, batch_size * vec_size * sizeof(int));
    hipMalloc(&d_shared_contexts_test, batch_size * sizeof(int));
    hipMalloc(&d_batch_idx_to_compact_idx, batch_size * sizeof(int));
    hipMalloc(&d_compact_to_batch, batch_size * sizeof(int));
    hipMalloc(&d_compact_size, sizeof(int));

    cudaH2Dcpy(d_input_ids, input_ids.data(), batch_size * vec_size);
    cudaH2Dcpy(d_compact_to_batch, compact_idx_to_batch_idx_test.data(), batch_size);

    invokeFindContextDups(d_shared_contexts_test,
            d_batch_idx_to_compact_idx,
            d_compact_to_batch,
            d_compact_size,
            d_input_ids,
            batch_size,
            vec_size);

    int compact_size;
    cudaD2Hcpy(shared_contexts_test.data(), d_shared_contexts_test, batch_size);
    cudaD2Hcpy(batch_idx_to_compact_idx_test.data(), d_batch_idx_to_compact_idx, batch_size);
    cudaD2Hcpy(compact_idx_to_batch_idx_test.data(), d_compact_to_batch, batch_size);
    cudaD2Hcpy(&compact_size, d_compact_size, 1);

    hipFree(d_input_ids);
    hipFree(d_shared_contexts_test);

    EXPECT_TRUE(shared_contexts_test == shared_contexts_ref);
    EXPECT_TRUE(batch_idx_to_compact_idx == batch_idx_to_compact_idx_test);
    EXPECT_TRUE(compact_idx_to_batch_idx_test == compact_idx_to_batch_idx);
    EXPECT_TRUE(compact_size == 4);

    return EXIT_SUCCESS;
}

int test_compact()
{
    size_t batch_size = 128;
    size_t compact_size = 5;
    size_t seq_len = 40;
    size_t hidden_dimension = 8;
    auto generator_f = std::bind(std::uniform_real_distribution<float>(-1.0, 1.0), std::mt19937());
    auto generator_i = std::bind(std::uniform_int_distribution<int>(0, 128), std::mt19937());

    // decoder_input [batch_size, seq_len, hidden_dimension] ->
    // compact_decoder_input [compact_size, seq_len, hidden_dimension]
    std::vector<float> decoder_input(batch_size * seq_len * hidden_dimension);
    std::vector<float> compact_decoder_input(compact_size * seq_len * hidden_dimension);
    std::generate(decoder_input.begin(), decoder_input.end(), generator_f);
    float *d_decoder_input, *d_compact_decoder_input;
    hipMalloc(&d_decoder_input, decoder_input.size() * sizeof(float));
    hipMalloc(&d_compact_decoder_input, compact_decoder_input.size() * sizeof(float));
    cudaH2Dcpy(d_decoder_input, decoder_input.data(), decoder_input.size());

    // attention_mask [batch_size, seq_len, seq_len] ->
    // compact_attention_mask [compact_size, seq_len, seq_len]
    std::vector<float> attention_mask(batch_size * seq_len * seq_len);
    std::vector<float> compact_attention_mask(compact_size * seq_len * seq_len);
    std::generate(attention_mask.begin(), attention_mask.end(), generator_f);
    float *d_attention_mask, *d_compact_attention_mask;
    hipMalloc(&d_attention_mask, attention_mask.size() * sizeof(float));
    hipMalloc(&d_compact_attention_mask, compact_attention_mask.size() * sizeof(float));
    cudaH2Dcpy(d_attention_mask, attention_mask.data(), attention_mask.size());

    // input_lengths [batch_size] -> compact_input_lengths [compact_size]
    std::vector<int> input_lengths(batch_size);
    std::vector<int> compact_input_lengths(compact_size);
    std::generate(input_lengths.begin(), input_lengths.end(), generator_i);
    int *d_input_lengths, *d_compact_input_lengths;
    hipMalloc(&d_input_lengths, input_lengths.size() * sizeof(int));
    hipMalloc(&d_compact_input_lengths, compact_input_lengths.size() * sizeof(int));
    cudaH2Dcpy(d_input_lengths, input_lengths.data(), input_lengths.size());

    // compact_idx [compact_size]
    /* std::vector<int> compact_idx {0, 3}; */
    std::vector<int> compact_idx {0, 29, 42, 44, 100};
    int *d_compact_idx;
    hipMalloc(&d_compact_idx, compact_idx.size() * sizeof(int));
    cudaH2Dcpy(d_compact_idx, compact_idx.data(), compact_idx.size());

    invokeCompactInputs<float>(d_compact_decoder_input,
                               d_compact_attention_mask,
                               d_compact_input_lengths,
                               d_decoder_input,
                               d_attention_mask,
                               d_input_lengths,
                               d_compact_idx,
                               compact_size,
                               seq_len,
                               hidden_dimension);

    cudaD2Hcpy(compact_decoder_input.data(), d_compact_decoder_input, compact_decoder_input.size());
    cudaD2Hcpy(compact_attention_mask.data(), d_compact_attention_mask, compact_attention_mask.size());
    cudaD2Hcpy(compact_input_lengths.data(), d_compact_input_lengths, compact_input_lengths.size());

    for (size_t i = 0; i < compact_size; i++) {
        for (size_t t = 0; t < seq_len; t++) {
            for (size_t h = 0; h < hidden_dimension; h++) {
                EXPECT_TRUE(compact_decoder_input[(i * seq_len + t) * hidden_dimension + h] ==
                            decoder_input[(compact_idx[i] * seq_len + t) * hidden_dimension + h]);
            }
        }
    }

    for (size_t i = 0; i < compact_size; i++) {
        for (size_t t1 = 0; t1 < seq_len; t1++) {
            for (size_t t2 = 0; t2 < seq_len; t2++) {
                EXPECT_TRUE(compact_attention_mask[(i * seq_len + t1) * seq_len + t2] ==
                            attention_mask[(compact_idx[i] * seq_len + t1) * seq_len + t2]);
            }
        }
    }

    for (size_t i = 0; i < compact_size; i++) {
        EXPECT_TRUE(compact_input_lengths[i] == input_lengths[compact_idx[i]]);
    }

    hipFree(d_decoder_input);
    hipFree(d_compact_decoder_input);
    hipFree(d_attention_mask);
    hipFree(d_compact_attention_mask);
    hipFree(d_input_lengths);
    hipFree(d_compact_input_lengths);
    hipFree(d_compact_idx);

    return EXIT_SUCCESS;
}

int test_uncompact()
{
    // compact_decoder_outputs [compact_size, seq_len, hidden_dimension] ->
    // decoder_outputs [batch_size, seq_len, hidden_dimension]
    size_t batch_size = 128;
    size_t compact_size = 6;
    size_t local_batch_size = compact_size / 2;
    size_t seq_len = 40;
    size_t max_seq_len = 60;
    size_t hidden_dimension = 8;
    size_t num_layer = 2;
    size_t num_head = 2;
    size_t size_per_head = 4;
    auto generator_f = std::bind(std::uniform_real_distribution<float>(-1.0, 1.0), std::mt19937());
    auto generator_i = std::bind(std::uniform_int_distribution<int>(0, compact_size - 1), std::mt19937());

    std::vector<float> compact_decoder_outputs(compact_size * seq_len * hidden_dimension);
    std::vector<float> decoder_outputs(batch_size * seq_len * hidden_dimension);
    std::vector<float> k_cache_compact(num_layer * compact_size * num_head * size_per_head * seq_len);
    std::vector<float> v_cache_compact(num_layer * compact_size * num_head * seq_len * size_per_head);
    std::vector<float> k_cache_out(num_layer * batch_size * num_head * size_per_head * max_seq_len);
    std::vector<float> v_cache_out(num_layer * batch_size * num_head * max_seq_len * size_per_head);

    std::generate(compact_decoder_outputs.begin(), compact_decoder_outputs.end(), generator_f);
    std::generate(k_cache_compact.begin(), k_cache_compact.end(), generator_f);
    std::generate(v_cache_compact.begin(), v_cache_compact.end(), generator_f);

    std::vector<int> batch_to_compact_idx(batch_size);
    std::generate(batch_to_compact_idx.begin(), batch_to_compact_idx.end(), generator_i);

    float *d_compact_decoder_outputs, *d_decoder_outputs, *d_k_cache, *d_v_cache;
    float *d_k_cache_compact, *d_v_cache_compact;

    hipMalloc(&d_compact_decoder_outputs, compact_decoder_outputs.size() * sizeof(float));
    cudaH2Dcpy(d_compact_decoder_outputs, compact_decoder_outputs.data(), compact_decoder_outputs.size());

    hipMalloc(&d_k_cache_compact, k_cache_compact.size() * sizeof(float));
    hipMalloc(&d_v_cache_compact, v_cache_compact.size() * sizeof(float));
    cudaH2Dcpy(d_k_cache_compact, k_cache_compact.data(), k_cache_compact.size());
    cudaH2Dcpy(d_v_cache_compact, v_cache_compact.data(), v_cache_compact.size());

    hipMalloc(&d_k_cache, k_cache_out.size() * sizeof(float));
    hipMalloc(&d_v_cache, v_cache_out.size() * sizeof(float));
    hipMemset(d_k_cache, 0, k_cache_out.size() * sizeof(float));
    hipMemset(d_v_cache, 0, v_cache_out.size() * sizeof(float));

    hipMalloc(&d_decoder_outputs, decoder_outputs.size() * sizeof(float));

    int *d_batch_to_compact_idx;
    hipMalloc(&d_batch_to_compact_idx, batch_to_compact_idx.size() * sizeof(int));
    cudaH2Dcpy(d_batch_to_compact_idx, batch_to_compact_idx.data(), batch_to_compact_idx.size());

    const size_t cache_stride_dst = max_seq_len * hidden_dimension;
    const size_t cache_stride_src = seq_len * hidden_dimension;
    for (size_t ite = 0; ite < (batch_size / local_batch_size); ite++) {
        for (size_t l = 0; l < num_layer; l++) {

            const float *k_cache_offset = d_k_cache_compact + (l * compact_size + ite * local_batch_size) * cache_stride_src;
            const float *v_cache_offset = d_v_cache_compact + (l * compact_size + ite * local_batch_size) * cache_stride_src;

            invokeUnCompactCaches(d_k_cache + l * batch_size * cache_stride_dst,
                                  d_v_cache + l * batch_size * cache_stride_dst,
                                  k_cache_offset,
                                  v_cache_offset,
                                  d_batch_to_compact_idx,
                                  batch_size,
                                  num_head,
                                  max_seq_len,
                                  seq_len,
                                  size_per_head,
                                  local_batch_size,
                                  ite);
        }
    }

    invokeUnCompactOutputs(d_decoder_outputs,
                           d_compact_decoder_outputs,
                           d_batch_to_compact_idx,
                           batch_size,
                           cache_stride_src);

    cudaD2Hcpy(decoder_outputs.data(), d_decoder_outputs, decoder_outputs.size());
    cudaD2Hcpy(k_cache_out.data(), d_k_cache, k_cache_out.size());
    cudaD2Hcpy(v_cache_out.data(), d_v_cache, v_cache_out.size());

    for (size_t i = 0; i < batch_size; i++) {
        for (size_t t = 0; t < seq_len; t++) {
            for (size_t h = 0; h < hidden_dimension; h++) {
                EXPECT_TRUE(decoder_outputs[(i * seq_len + t) * hidden_dimension] ==
                            compact_decoder_outputs[(batch_to_compact_idx[i] * seq_len + t) * hidden_dimension]);
            }
        }
    }

    size_t x_size = (16 / sizeof(float));
    for (size_t l = 0; l < num_layer; l++) {
        for (size_t i = 0; i < batch_size; i++) {
            for (size_t h = 0; h < num_head; h++) {
                for (size_t dh = 0; dh < size_per_head / x_size; dh++) {
                    for (size_t t = 0; t < seq_len; t++) {
                        for (size_t x = 0; x < x_size; x++) {
                            auto src = batch_to_compact_idx[i];
                            EXPECT_TRUE(
                                    k_cache_out[((((l * batch_size + i  ) * num_head + h) * (size_per_head / x_size) + dh) *
                                        max_seq_len + t) * x_size + x] ==
                                    k_cache_compact[((((l * compact_size + src) * num_head + h) * (size_per_head / x_size) + dh) *
                                        seq_len + t) * x_size + x]);
                        }
                    }
                }
            }
        }
    }

    for (size_t l = 0; l < num_layer; l++) {
        for (size_t i = 0; i < batch_size; i++) {
            for (size_t h = 0; h < num_head; h++) {
                for (size_t t = 0; t < seq_len; t++) {
                    for (size_t dh = 0; dh < size_per_head; dh++) {
                        auto src = batch_to_compact_idx[i];
                        EXPECT_TRUE(
                                v_cache_out[(((l * batch_size + i  ) * num_head + h) * max_seq_len + t) * size_per_head + dh] ==
                                v_cache_compact[(((l * compact_size + src) * num_head + h) * seq_len + t) * size_per_head + dh]);
                    }
                }
            }
        }
    }

    hipFree(d_compact_decoder_outputs);
    hipFree(d_k_cache_compact);
    hipFree(d_v_cache_compact);
    hipFree(d_k_cache);
    hipFree(d_v_cache);
    hipFree(d_decoder_outputs);
    hipFree(d_batch_to_compact_idx);

    return EXIT_SUCCESS;
}
