#include "hip/hip_runtime.h"
#include <algorithm>   // std::fill_n
#include <iostream>    // snprintf
#include <math.h>      // expf, log
#include <stdlib.h>    // rand
#include <string>      // std::string
#include <vector>      // std::vector

#include <hipblas.h>
#include <hipblaslt.h>
#include <hip/hip_runtime.h>
#include <gtest/gtest.h>

#include "src/fastertransformer/kernels/sampling_topk_kernels.h"
#include "src/fastertransformer/kernels/sampling_topp_kernels.h"
#include "src/fastertransformer/layers/DynamicDecodeLayer.h"
#include "src/fastertransformer/layers/sampling_layers/TopKSamplingLayer.h"
#include "src/fastertransformer/utils/Tensor.h"
#include "src/fastertransformer/utils/cublasMMWrapper.h"
#include "src/fastertransformer/utils/cuda_utils.h"
#include "src/fastertransformer/utils/memory_utils.h"

#include "tests/unittests/gtest_utils.h"

using namespace fastertransformer;

namespace {

struct SamplingKernelTestParam {
    size_t batch_size;
    size_t vocab_size;
    size_t beam_width;
    uint   top_k;
    float  top_p;
    size_t output_len;

    std::string toString()
    {
        return fmtstr("SamplingKernelTestParam[batch=%ld, vocab=%ld, beam=%ld, k=%u, p=%3.1f, output_len=%ld]",
                      batch_size,
                      vocab_size,
                      beam_width,
                      top_k,
                      top_p,
                      output_len);
    }
};

/////////////////////////////////// Tests //////////////////////////////////////////

template<typename T>
void computeProb(T* probs, T* logits, int batch_size, int vocab_size)
{
    // Compute the log probability from logits.
    //   logits = batch_size x vocab_size.
    //   probs =  softmax(logits) (softmax along with vocab dimension)
    // float is used for either T=float or half, since operations of half are
    // not fully supported in a host function.
    for (int bidx = 0; bidx < batch_size; ++bidx) {
        float maxval = -FLT_MAX;
        for (int i = 0; i < vocab_size; ++i) {
            float logit = static_cast<float>(logits[bidx * vocab_size + i]);
            if (logit > maxval) {
                maxval = logit;
            }
        }
        float sum = 0.0f;
        for (int i = 0; i < vocab_size; ++i) {
            sum += expf(static_cast<float>(logits[bidx * vocab_size + i]) - maxval);
        }
        for (int i = 0; i < vocab_size; ++i) {
            int idx = bidx * vocab_size + i;
            float logit = static_cast<float>(logits[idx]) - maxval;
            probs[idx] = static_cast<T>(expf(logit) / (sum + EPSILON));
        }
    }
}

template<typename T>
void computeLogProb(T* logprobs, T* logits, int batch_size, int vocab_size)
{
    // Compute the log probability from logits.
    //   logits = batch_size x vocab_size.
    //   logprobs = log(softmax(logits)) (softmax along with vocab dimension)
    // float is used for either T=float or half, since operations of half are
    // not fully supported in a host function.
    for (int bidx = 0; bidx < batch_size; ++bidx) {
        float maxval = -FLT_MAX;
        for (int i = 0; i < vocab_size; ++i) {
            float logit = static_cast<float>(logits[bidx * vocab_size + i]);
            if (logit > maxval) {
                maxval = logit;
            }
        }
        float sum = 0.0f;
        for (int i = 0; i < vocab_size; ++i) {
            sum += expf(static_cast<float>(logits[bidx * vocab_size + i]) - maxval);
        }
        for (int i = 0; i < vocab_size; ++i) {
            int idx = bidx * vocab_size + i;
            float logit = static_cast<float>(logits[idx]) - maxval;
            logprobs[idx] = static_cast<T>(logit - logf(sum + EPSILON));
        }
    }
}

template<typename T>
class SamplingKernelTest: public testing::Test {
public:
    void SetUp() override
    {
        check_cuda_error(hipStreamCreate(&stream));
        allocator = new Allocator<AllocatorType::CUDA>(getDevice());
        allocator->setStream(stream);
    }
    void TearDown() override
    {
        delete allocator;
        check_cuda_error(hipStreamDestroy(stream));
    }

protected:
    unsigned long long seed = 0;
    hipStream_t stream;
    Allocator<AllocatorType::CUDA>* allocator;
    hiprandState_t* curand_states;
};

template<typename T>
class TopKSamplingKernelTest: public SamplingKernelTest<T> {

protected:
    const int end_id = 0;
    using SamplingKernelTest<T>::seed;
    using SamplingKernelTest<T>::stream;
    using SamplingKernelTest<T>::allocator;
    using SamplingKernelTest<T>::curand_states;

public:
    void runTest(SamplingKernelTestParam param)
    {
        size_t batch_size  = param.batch_size;
        size_t vocab_size  = param.vocab_size;
        size_t output_len  = param.output_len;
        size_t max_seq_len = output_len;

        uint  top_k = param.top_k;
        float top_p = param.top_p;

        // Logit values in the host of shape (batch_size x vocab_size).
        T* h_logits = new T[batch_size * vocab_size];
        T* h_probs  = new T[batch_size * vocab_size];
        T* h_lprobs = new T[batch_size * vocab_size];

        int*  h_output_ids  = new int[batch_size];
        int*  h_seq_lengths = new int[batch_size];
        bool* h_finished    = new bool[batch_size];

        float* expected_cum_lprobs = new float[batch_size];
        std::fill_n(expected_cum_lprobs, batch_size, 0);

        hiprandState_t* curand_states =
            reinterpret_cast<hiprandState_t*>(allocator->malloc(sizeof(hiprandState_t) * batch_size, false));
        invokeCurandInitialize(curand_states, batch_size, seed, stream);

        size_t workspace_size = 0;
        // retrieve the workspace size of the top-k sampling kernel.
        invokeTopKSampling<T>(nullptr,
                              workspace_size,
                              nullptr,
                              nullptr,
                              nullptr,
                              nullptr,
                              nullptr,
                              nullptr,
                              nullptr,
                              top_k,
                              1.0f,
                              vocab_size,
                              nullptr,
                              stream,
                              batch_size,
                              nullptr);
        void* workspace = allocator->malloc(workspace_size);

        int*  end_ids     = reinterpret_cast<int*>(allocator->malloc(sizeof(int) * batch_size));
        int*  seq_lengths = reinterpret_cast<int*>(allocator->malloc(sizeof(int) * batch_size));
        bool* finished    = reinterpret_cast<bool*>(allocator->malloc(sizeof(bool) * batch_size));

        T*     probs         = reinterpret_cast<T*>(allocator->malloc(sizeof(T) * batch_size * vocab_size));
        float* cum_lprobs    = reinterpret_cast<float*>(allocator->malloc(sizeof(float) * batch_size));
        float* output_lprobs = reinterpret_cast<float*>(allocator->malloc(sizeof(float) * output_len * batch_size));
        int*   output_ids    = reinterpret_cast<int*>(allocator->malloc(sizeof(int) * max_seq_len * batch_size));

        // Init by zero.
        deviceFill(seq_lengths, batch_size, 0);
        deviceFill(finished, batch_size, false);
        deviceFill(end_ids, batch_size, end_id);

        deviceFill(cum_lprobs, batch_size, 0.0f);
        deviceFill(output_lprobs, output_len * batch_size, 0.0f);
        deviceFill(output_ids, max_seq_len * batch_size, 0);

        for (size_t step = 0; step < output_len; ++step) {
            initRandom(h_logits, batch_size * vocab_size, -3.0f, 3.0f);
            computeProb(h_probs, h_logits, batch_size, vocab_size);
            cudaH2Dcpy(probs, h_probs, batch_size * vocab_size);
            invokeTopKSampling(workspace,
                               workspace_size,
                               // Note that the kernel needs vocab probs instead of
                               // log-prob if cum_log_probs or output_log_probs are
                               // provided. It's because the sampling layer already
                               // preprocesses log_prob_buf when those are provided.
                               probs,
                               output_ids + step * batch_size,
                               seq_lengths,
                               finished,
                               cum_lprobs,
                               output_lprobs + step * batch_size,
                               curand_states,
                               top_k,
                               top_p,
                               vocab_size,
                               end_ids,
                               stream,
                               batch_size,
                               nullptr);

            // Compute reference.
            cudaD2Hcpy(h_output_ids, output_ids + step * batch_size, batch_size);
            cudaD2Hcpy(h_seq_lengths, seq_lengths, batch_size);
            cudaD2Hcpy(h_finished, finished, batch_size);
            computeLogProb(h_lprobs, h_logits, batch_size, vocab_size);
            for (size_t i = 0; i < batch_size; ++i) {
                int idx = i * vocab_size + h_output_ids[i];
                expected_cum_lprobs[i] += (int)step < h_seq_lengths[i] ? (float)h_lprobs[idx] : 0.0f;
                EXPECT_EQ(h_finished[i], h_output_ids[i] == end_id);
            }
        }
        bool passed = checkResult(param.toString(), cum_lprobs, expected_cum_lprobs, batch_size);
        EXPECT_TRUE(passed);

        delete[] expected_cum_lprobs;
        delete[] h_seq_lengths;
        delete[] h_logits;
        delete[] h_lprobs;
        delete[] h_probs;
        delete[] h_output_ids;
    }

    void runBatchTest(SamplingKernelTestParam param, bool has_diff_runtime_args, bool use_skip_decode)
    {
        size_t batch_size = param.batch_size;
        size_t vocab_size = param.vocab_size;
        size_t output_len = param.output_len;
        size_t seq_len    = output_len;

        int   top_k = param.top_k;
        float top_p = param.top_p;

        int*   h_top_ks = new int[batch_size];
        float* h_top_ps = new float[batch_size];
        for (size_t i = 0; i < batch_size; ++i) {
            h_top_ks[i] = (!has_diff_runtime_args || i % 3 == 0) ? top_k : 1;
            h_top_ps[i] = (!has_diff_runtime_args || i % 3 == 0) ? top_p : 0.1 * top_p;
        }
        int max_top_k = *std::max_element(h_top_ks, h_top_ks + batch_size);

        // Logit values in the host of shape (batch_size x vocab_size).
        T* h_logits = new T[batch_size * vocab_size];
        T* h_probs  = new T[batch_size * vocab_size];
        T* h_lprobs = new T[batch_size * vocab_size];

        float* expected_cum_lprobs = new float[batch_size];

        int*  h_output_ids  = new int[batch_size];
        int*  h_seq_lengths = new int[batch_size];
        bool* h_finished    = new bool[batch_size];
        bool* h_skip_decode = new bool[batch_size];

        initRandom(h_logits, batch_size * vocab_size, -3.0f, 3.0f);
        std::fill_n(expected_cum_lprobs, batch_size, 0);
        for (size_t i = 0; i < batch_size; ++i) {
            h_skip_decode[i] = use_skip_decode && (i % 2 == 0);
        }

        hiprandState_t* curand_states =
            reinterpret_cast<hiprandState_t*>(allocator->malloc(sizeof(hiprandState_t) * batch_size, false));
        invokeCurandInitialize(curand_states, batch_size, seed, stream);

        size_t workspace_size = 0;
        // retrieve the workspace size of the top-k sampling kernel.
        invokeBatchTopKSampling<T>(nullptr,  // workspace
                                   workspace_size,
                                   nullptr,  // log_probs
                                   nullptr,  // ids
                                   nullptr,  // sequence_lengths
                                   nullptr,  // finished
                                   nullptr,  // cum_log_probs
                                   nullptr,  // output_log_probs
                                   nullptr,  // curandstates
                                   max_top_k,
                                   nullptr,  // top_ks
                                   1.0f,
                                   nullptr,
                                   vocab_size,
                                   nullptr,  // end_ids
                                   stream,
                                   batch_size,
                                   nullptr);
        void* workspace = allocator->malloc(workspace_size, false);

        int*   top_ks = reinterpret_cast<int*>(allocator->malloc(sizeof(int) * batch_size));
        float* top_ps = reinterpret_cast<float*>(allocator->malloc(sizeof(float) * batch_size));

        int*  end_ids     = reinterpret_cast<int*>(allocator->malloc(sizeof(int) * batch_size));
        int*  seq_lengths = reinterpret_cast<int*>(allocator->malloc(sizeof(int) * batch_size));
        int*  output_ids  = reinterpret_cast<int*>(allocator->malloc(sizeof(int) * seq_len * batch_size));
        bool* finished    = reinterpret_cast<bool*>(allocator->malloc(sizeof(bool) * batch_size));
        bool* skip_decode = reinterpret_cast<bool*>(allocator->malloc(sizeof(bool) * batch_size));

        T*     probs         = reinterpret_cast<T*>(allocator->malloc(sizeof(T) * batch_size * vocab_size, true));
        float* cum_lprobs    = reinterpret_cast<float*>(allocator->malloc(sizeof(float) * batch_size));
        float* output_lprobs = reinterpret_cast<float*>(allocator->malloc(sizeof(float) * output_len * batch_size));

        // Initialize.
        cudaH2Dcpy(top_ks, h_top_ks, batch_size);
        cudaH2Dcpy(top_ps, h_top_ps, batch_size);
        cudaH2Dcpy(skip_decode, h_skip_decode, batch_size);

        deviceFill(end_ids, batch_size, end_id);
        deviceFill(seq_lengths, batch_size, 0);
        deviceFill(finished, batch_size, false);
        deviceFill(cum_lprobs, batch_size, 0.0f);
        deviceFill(output_lprobs, output_len * batch_size, 0.0f);
        deviceFill(output_ids, seq_len * batch_size, 0);

        for (size_t step = 0; step < output_len; ++step) {
            initRandom(h_logits, batch_size * vocab_size, -3.0f, 3.0f);
            computeProb(h_probs, h_logits, batch_size, vocab_size);
            cudaH2Dcpy(probs, h_probs, batch_size * vocab_size);

            invokeBatchTopKSampling(workspace,
                                    workspace_size,
                                    // Note that the kernel needs vocab probs instead of
                                    // log-prob if cum_log_probs or output_log_probs are
                                    // provided. It's because the sampling layer already
                                    // preprocesses log_prob_buf when those are provided.
                                    probs,
                                    output_ids + step * batch_size,
                                    seq_lengths,
                                    finished,
                                    cum_lprobs,
                                    output_lprobs + step * batch_size,
                                    curand_states,
                                    max_top_k,
                                    top_ks,
                                    1.0f,
                                    nullptr,
                                    vocab_size,
                                    end_ids,
                                    stream,
                                    batch_size,
                                    skip_decode);

            // Compute reference.
            cudaD2Hcpy(h_output_ids, output_ids + step * batch_size, batch_size);
            cudaD2Hcpy(h_seq_lengths, seq_lengths, batch_size);
            cudaD2Hcpy(h_finished, finished, batch_size);
            computeLogProb(h_lprobs, h_logits, batch_size, vocab_size);
            for (size_t i = 0; i < batch_size; ++i) {
                if (!h_skip_decode[i]) {
                    int idx = i * vocab_size + h_output_ids[i];
                    expected_cum_lprobs[i] += (int)step < h_seq_lengths[i] ? (float)h_lprobs[idx] : 0.0f;
                    EXPECT_EQ(h_finished[i], h_output_ids[i] == end_id);
                }
            }
        }
        bool passed = checkResult(param.toString(), cum_lprobs, expected_cum_lprobs, batch_size);
        EXPECT_TRUE(passed) << "Fail subtest (has_diff_runtime_args: " << has_diff_runtime_args
                            << ", skip_decode: " << use_skip_decode << ")";

        delete[] expected_cum_lprobs;
        delete[] h_seq_lengths;
        delete[] h_logits;
        delete[] h_lprobs;
        delete[] h_probs;
        delete[] h_output_ids;
        delete[] h_top_ks;
        delete[] h_skip_decode;
    }

    void runBatchTest(SamplingKernelTestParam param)
    {
        this->runBatchTest(param, false, false);
        this->runBatchTest(param, false, true);
        this->runBatchTest(param, true,  false);
        this->runBatchTest(param, true,  true);
    }
};

TYPED_TEST_SUITE(TopKSamplingKernelTest, FloatAndHalfTypes);

TYPED_TEST(TopKSamplingKernelTest, CorrectnessGreedy)
{
    this->runTest({6, 4, 1, 1, 1.0f, 1});
};

TYPED_TEST(TopKSamplingKernelTest, CorrectnessAncestral)
{
    this->runTest({6, 4, 1, 4, 1.0f, 1});
};


TYPED_TEST(TopKSamplingKernelTest, CorrectnessLargeK63)
{
    this->runTest({16, 51200, 1, 63, 1.0f, 8});
};

TYPED_TEST(TopKSamplingKernelTest, CorrectnessLargeK1024)
{
    this->runTest({16, 51200, 1, 1024, 1.0f, 8});
};

TYPED_TEST(TopKSamplingKernelTest, CorrectnessTopKTopP)
{
    this->runTest({16, 4000, 1, 63, 0.3f, 8});
};

TYPED_TEST(TopKSamplingKernelTest, NotSupportedLargerThanK1024)
{
    EXPECT_THROW(this->runTest({16, 4000, 1, 1025, 1.0f, 8}), std::domain_error);
};

TYPED_TEST(TopKSamplingKernelTest, BatchCorrectnessGreedy)
{
    this->runBatchTest({6, 4, 1, 1, 1.0f, 1});
};

TYPED_TEST(TopKSamplingKernelTest, BatchCorrectnessAncestral)
{
    this->runBatchTest({6, 4, 1, 4, 1.0f, 1});
};

TYPED_TEST(TopKSamplingKernelTest, BatchCorrectnessLargeK63)
{
    this->runBatchTest({8, 4000, 1, 63, 1.0f, 8});
};

TYPED_TEST(TopKSamplingKernelTest, BatchCorrectnessLargeK1024)
{
    this->runBatchTest({8, 4000, 1, 1024, 0.0f, 8});
};

TYPED_TEST(TopKSamplingKernelTest, BatchCorrectnessTopKTopP)
{
    this->runBatchTest({8, 4000, 1, 63, 0.3f, 8});
};


template<typename T>
class TopPSamplingKernelTest: public SamplingKernelTest<T> {

protected:
    const int end_id = 0;
    using SamplingKernelTest<T>::seed;
    using SamplingKernelTest<T>::stream;
    using SamplingKernelTest<T>::allocator;
    using SamplingKernelTest<T>::curand_states;

public:
    void runTest(SamplingKernelTestParam param)
    {
        size_t batch_size = param.batch_size;
        size_t vocab_size = param.vocab_size;
        size_t output_len = param.output_len;
        size_t seq_len = output_len;

        float top_p = param.top_p;

        // Logit values in the host of shape (batch_size x vocab_size).
        T* h_logits = new T[batch_size * vocab_size];
        T* h_probs  = new T[batch_size * vocab_size];
        T* h_lprobs = new T[batch_size * vocab_size];

        float* expected_cum_lprobs = new float[batch_size];
        std::fill_n(expected_cum_lprobs, batch_size, 0);

        int*  h_output_ids  = new int[batch_size];
        int*  h_seq_lengths = new int[batch_size];
        bool* h_finished    = new bool[batch_size];

        initRandom(h_logits, batch_size * vocab_size, -3.0f, 3.0f);

        int device;
        hipGetDevice(&device);
        struct hipDeviceProp_t device_prop;
        hipGetDeviceProperties(&device_prop, device);

        hiprandState_t* curand_states = reinterpret_cast<hiprandState_t*>(
            allocator->malloc(sizeof(hiprandState_t) * batch_size, false));
        invokeCurandInitialize(curand_states, batch_size, seed, stream);

        int* end_ids     = reinterpret_cast<int*>(allocator->malloc(sizeof(int) * batch_size));
        int* seq_lengths = reinterpret_cast<int*>(allocator->malloc(sizeof(int) * batch_size));
        int* output_ids  = reinterpret_cast<int*>(allocator->malloc(sizeof(int) * seq_len * batch_size));

        bool* finished    = reinterpret_cast<bool*>(allocator->malloc(sizeof(bool) * batch_size));
        bool* skip_decode = reinterpret_cast<bool*>(allocator->malloc(sizeof(bool) * batch_size));

        T*     probs         = reinterpret_cast<T*>(allocator->malloc(sizeof(T) * batch_size * vocab_size));
        float* cum_lprobs    = reinterpret_cast<float*>(allocator->malloc(sizeof(float) * batch_size));
        float* output_lprobs = reinterpret_cast<float*>(allocator->malloc(sizeof(float) * output_len * batch_size));

        int* begin_offsets    = reinterpret_cast<int*>(allocator->malloc(sizeof(int) * (batch_size + 1)));
        int* end_offsets      = reinterpret_cast<int*>(allocator->malloc(sizeof(int) * (batch_size + 1)));
        int* topp_id_vals_buf = reinterpret_cast<int*>(allocator->malloc(sizeof(int) * batch_size * vocab_size));

        size_t workspace_size = 0;
        size_t cub_temp_storage_size = 0;
        // retrieve the workspace size of the top-p sampling kernel.
        invokeTopPSampling<T>(nullptr,  // workspace
                              workspace_size,
                              cub_temp_storage_size,
                              nullptr,  // output_ids
                              nullptr,  // sequence_length
                              nullptr,  // finished_buffer
                              nullptr,  // cum_log_probs
                              nullptr,  // output_log_probs
                              (T*)nullptr,  // log_probs
                              topp_id_vals_buf,
                              end_offsets,
                              begin_offsets,
                              curand_states,
                              batch_size,
                              vocab_size,
                              nullptr,
                              top_p,
                              stream,
                              &device_prop,
                              nullptr);
        void* workspace = allocator->malloc(workspace_size);

        // Initialize.
        deviceFill(end_ids, batch_size, end_id);
        deviceFill(seq_lengths, batch_size, 0);
        deviceFill(finished, batch_size, false);
        deviceFill(cum_lprobs, batch_size, 0.0f);
        deviceFill(output_lprobs, output_len * batch_size, 0.0f);
        deviceFill(output_ids, seq_len * batch_size, 0);

        for (size_t step = 0; step < output_len; ++step) {
            initRandom(h_logits, batch_size * vocab_size, -3.0f, 3.0f);
            computeProb(h_probs, h_logits, batch_size, vocab_size);
            cudaH2Dcpy(probs, h_probs, batch_size * vocab_size);

            invokeTopPInitialize(topp_id_vals_buf,
                                 end_offsets,
                                 begin_offsets,
                                 batch_size,
                                 vocab_size,
                                 stream);

            invokeTopPSampling<T>(workspace,
                                  workspace_size,
                                  cub_temp_storage_size,
                                  output_ids + step * batch_size,
                                  seq_lengths,
                                  finished,
                                  cum_lprobs,
                                  output_lprobs + step * batch_size,
                                  // Note that the kernel needs vocab probs instead of
                                  // log-prob if cum_log_probs or output_log_probs are
                                  // provided. It's because the sampling layer already
                                  // preprocesses log_prob_buf when those are provided.
                                  probs,
                                  topp_id_vals_buf,
                                  end_offsets,
                                  begin_offsets,
                                  curand_states,
                                  batch_size,
                                  vocab_size,
                                  end_ids,
                                  top_p,
                                  stream,
                                  &device_prop,
                                  nullptr);

            // Compute reference.
            cudaD2Hcpy(h_output_ids, output_ids + step * batch_size, batch_size);
            cudaD2Hcpy(h_seq_lengths, seq_lengths, batch_size);
            cudaD2Hcpy(h_finished, finished, batch_size);
            computeLogProb(h_lprobs, h_logits, batch_size, vocab_size);
            for (size_t i = 0; i < batch_size; ++i) {
                int idx = i * vocab_size + h_output_ids[i];
                expected_cum_lprobs[i] += (int)step < h_seq_lengths[i] ? (float)h_lprobs[idx] : 0.0f;
                EXPECT_EQ(h_finished[i], h_output_ids[i] == end_id);
            }
        }
        bool passed = checkResult(param.toString(), cum_lprobs, expected_cum_lprobs, batch_size);
        EXPECT_TRUE(passed);

        delete[] expected_cum_lprobs;
        delete[] h_seq_lengths;
        delete[] h_logits;
        delete[] h_lprobs;
        delete[] h_probs;
        delete[] h_output_ids;
    }

    void runBatchTest(SamplingKernelTestParam param, bool has_diff_runtime_args, bool use_skip_decode)
    {
        size_t batch_size = param.batch_size;
        size_t vocab_size = param.vocab_size;

        float top_p = param.top_p;
        float* h_top_ps = new float[batch_size];
        // Initialize runtime top k values.
        for (size_t i = 0; i < batch_size; ++i) {
            h_top_ps[i] = (!has_diff_runtime_args || i % 3 == 0) ? top_p : 0.1 * top_p;
        }
        float max_top_p = *std::max_element(h_top_ps, h_top_ps + batch_size);

        size_t output_len = param.output_len;
        size_t seq_len = output_len;

        // Logit values in the host of shape (batch_size x vocab_size).
        T* h_logits = new T[batch_size * vocab_size];
        T* h_probs  = new T[batch_size * vocab_size];
        T* h_lprobs = new T[batch_size * vocab_size];

        float* expected_cum_lprobs = new float[batch_size];
        std::fill_n(expected_cum_lprobs, batch_size, 0);

        int*  h_output_ids  = new int[batch_size];
        int*  h_seq_lengths = new int[batch_size];
        bool* h_finished    = new bool[batch_size];
        bool* h_skip_decode = new bool[batch_size];

        initRandom(h_logits, batch_size * vocab_size, -3.0f, 3.0f);
        std::fill_n(expected_cum_lprobs, batch_size, 0);
        for (size_t i = 0; i < batch_size; ++i) {
            h_skip_decode[i] = use_skip_decode && (i % 2 == 0);
        }

        int device;
        hipGetDevice(&device);
        struct hipDeviceProp_t device_prop;
        hipGetDeviceProperties(&device_prop, device);

        hiprandState_t* curand_states = reinterpret_cast<hiprandState_t*>(
            allocator->malloc(sizeof(hiprandState_t) * batch_size, false));
        invokeCurandInitialize(curand_states, batch_size, seed, stream);

        float* top_ps = reinterpret_cast<float*>(allocator->malloc(sizeof(float) * batch_size));

        int* end_ids     = reinterpret_cast<int*>(allocator->malloc(sizeof(int) * batch_size));
        int* seq_lengths = reinterpret_cast<int*>(allocator->malloc(sizeof(int) * batch_size));
        int* output_ids  = reinterpret_cast<int*>(allocator->malloc(sizeof(int) * seq_len * batch_size));

        bool* finished    = reinterpret_cast<bool*>(allocator->malloc(sizeof(bool) * batch_size));
        bool* skip_decode = reinterpret_cast<bool*>(allocator->malloc(sizeof(bool) * batch_size));

        T*     probs         = reinterpret_cast<T*>(allocator->malloc(sizeof(T) * batch_size * vocab_size));
        float* cum_lprobs    = reinterpret_cast<float*>(allocator->malloc(sizeof(float) * batch_size));
        float* output_lprobs = reinterpret_cast<float*>(allocator->malloc(sizeof(float) * output_len * batch_size));

        int* begin_offsets    = reinterpret_cast<int*>(allocator->malloc(sizeof(int) * (batch_size + 1)));
        int* end_offsets      = reinterpret_cast<int*>(allocator->malloc(sizeof(int) * (batch_size + 1)));
        int* topp_id_vals_buf = reinterpret_cast<int*>(allocator->malloc(sizeof(int) * batch_size * vocab_size));

        size_t workspace_size = 0;
        size_t cub_temp_storage_size = 0;
        // retrieve the workspace size of the top-p sampling kernel.
        invokeBatchTopPSampling<T>(nullptr,  // workspace
                                   workspace_size,
                                   cub_temp_storage_size,
                                   nullptr,  // output_ids
                                   nullptr,  // sequence_length
                                   nullptr,  // finished_buffer
                                   nullptr,  // cum_log_probs
                                   nullptr,  // output_log_probs
                                   (T*)nullptr,  // log_probs
                                   topp_id_vals_buf,
                                   end_offsets,
                                   begin_offsets,
                                   curand_states,
                                   batch_size,
                                   vocab_size,
                                   nullptr,
                                   max_top_p,
                                   top_ps,
                                   stream,
                                   &device_prop,
                                   nullptr);
        void* workspace = allocator->malloc(workspace_size);

        // Initialize.
        cudaH2Dcpy(top_ps, h_top_ps, batch_size);
        cudaH2Dcpy(skip_decode, h_skip_decode, batch_size);
        deviceFill(end_ids, batch_size, end_id);
        deviceFill(seq_lengths, batch_size, 0);
        deviceFill(finished, batch_size, false);
        deviceFill(cum_lprobs, batch_size, 0.0f);
        deviceFill(output_lprobs, output_len * batch_size, 0.0f);
        deviceFill(output_ids, seq_len * batch_size, 0);

        for (size_t step = 0; step < output_len; ++step) {
            initRandom(h_logits, batch_size * vocab_size, -3.0f, 3.0f);
            computeProb(h_probs, h_logits, batch_size, vocab_size);
            cudaH2Dcpy(probs, h_probs, batch_size * vocab_size);

            invokeTopPInitialize(topp_id_vals_buf,
                                 end_offsets,
                                 begin_offsets,
                                 batch_size,
                                 vocab_size,
                                 stream);

            invokeBatchTopPSampling<T>(workspace,
                                       workspace_size,
                                       cub_temp_storage_size,
                                       output_ids + step * batch_size,
                                       seq_lengths,
                                       finished,
                                       cum_lprobs,
                                       output_lprobs + step * batch_size,
                                       // Note that the kernel needs vocab probs instead of
                                       // log-prob if cum_log_probs or output_log_probs are
                                       // provided. It's because the sampling layer already
                                       // preprocesses log_prob_buf when those are provided.
                                       probs,
                                       topp_id_vals_buf,
                                       end_offsets,
                                       begin_offsets,
                                       curand_states,
                                       batch_size,
                                       vocab_size,
                                       end_ids,
                                       max_top_p,
                                       top_ps,
                                       stream,
                                       &device_prop,
                                       skip_decode);

            // Compute reference.
            cudaD2Hcpy(h_output_ids, output_ids + step * batch_size, batch_size);
            cudaD2Hcpy(h_seq_lengths, seq_lengths, batch_size);
            cudaD2Hcpy(h_finished, finished, batch_size);
            computeLogProb(h_lprobs, h_logits, batch_size, vocab_size);
            for (size_t i = 0; i < batch_size; ++i) {
                if (!h_skip_decode[i]) {
                    int idx = i * vocab_size + h_output_ids[i];
                    expected_cum_lprobs[i] += (int)step < h_seq_lengths[i] ? (float)h_lprobs[idx] : 0.0f;
                    EXPECT_EQ(h_finished[i], h_output_ids[i] == end_id);
                }
            }
        }
        bool passed = checkResult(param.toString(), cum_lprobs, expected_cum_lprobs, batch_size);
        EXPECT_TRUE(passed) << "Fail subtest (has_diff_runtime_args: " << has_diff_runtime_args
                            << ", skip_decode: " << use_skip_decode << ")";

        delete[] expected_cum_lprobs;
        delete[] h_seq_lengths;
        delete[] h_logits;
        delete[] h_lprobs;
        delete[] h_probs;
        delete[] h_output_ids;
        delete[] h_top_ps;
        delete[] h_skip_decode;
    }

    void runBatchTest(SamplingKernelTestParam param)
    {
        this->runBatchTest(param, false, false);
        this->runBatchTest(param, false, true);
        this->runBatchTest(param, true,  false);
        this->runBatchTest(param, true,  true);
    }
};

TYPED_TEST_SUITE(TopPSamplingKernelTest, FloatAndHalfTypes);

TYPED_TEST(TopPSamplingKernelTest, CorrectnessSmallP)
{
    this->runTest({6, 4, 1, 0, 0.2f, 1});
};

TYPED_TEST(TopPSamplingKernelTest, CorrectnessLargeP)
{
    this->runTest({6, 4, 1, 0, 0.9f, 1});
};

TYPED_TEST(TopPSamplingKernelTest, CorrectnessAncestral)
{
    this->runTest({6, 4, 1, 0, 1.0f, 1});
};

TYPED_TEST(TopPSamplingKernelTest, CorrectnessLargeVocabSmallP)
{
    this->runTest({32, 51200, 1, 0, 0.2f, 16});
};

TYPED_TEST(TopPSamplingKernelTest, CorrectnessLargeVocabLargeP)
{
    this->runTest({32, 51200, 1, 0, 0.9f, 16});
};

TYPED_TEST(TopPSamplingKernelTest, BatchCorrectnessSmallP)
{
    this->runBatchTest({6, 4, 1, 0, 0.2f, 1});
};

TYPED_TEST(TopPSamplingKernelTest, BatchCorrectnessLargeP)
{
    this->runBatchTest({6, 4, 1, 0, 0.9f, 1});
};

TYPED_TEST(TopPSamplingKernelTest, BatchCorrectnessSmallP2)
{
    this->runBatchTest({8, 4000, 1, 0, 0.2f, 16});
};

TYPED_TEST(TopPSamplingKernelTest, BatchCorrectnessLargeP2)
{
    this->runBatchTest({8, 4000, 1, 0, 0.9f, 16});
};

__global__
void generateRandomNumber(unsigned int *vals, hiprandState_t *states, const int batch_size) {
    int idx = threadIdx.x;
    if (idx < batch_size) {
        vals[idx] = hiprand(states + idx);
    }
}

TEST(SamplingKernelTest, CurandBatchInitialize) {
    size_t batch_size = 127;
    hipStream_t stream;
    hipStreamCreate(&stream);

    hiprandState_t* curand_states;
    check_cuda_error(hipMalloc(&curand_states, sizeof(hiprandState_t) * batch_size));
    unsigned long long* h_random_seeds = new unsigned long long[batch_size];
    const size_t period_size = 3;
    for (size_t i = 0; i < batch_size; ++i) {
        h_random_seeds[i] = i / period_size;
    }
    unsigned long long* d_random_seeds;
    check_cuda_error(hipMalloc(&d_random_seeds, sizeof(unsigned long long) * batch_size));
    check_cuda_error(hipMemcpy(d_random_seeds, h_random_seeds,
                                sizeof(unsigned long long) * batch_size, hipMemcpyHostToDevice));

    // Initialize hiprand states.
    invokeCurandBatchInitialize(curand_states, batch_size, d_random_seeds, stream);
    sync_check_cuda_error();

    // Generate random numbers using initialized hiprand states.
    unsigned int* d_rand_vals;
    unsigned int* h_rand_vals = new unsigned int[batch_size];
    check_cuda_error(hipMalloc(&d_rand_vals, sizeof(unsigned int) * batch_size));
    generateRandomNumber<<<1, batch_size, 0, stream>>>(d_rand_vals, curand_states, batch_size);
    check_cuda_error(hipMemcpyAsync(
        h_rand_vals, d_rand_vals, sizeof(unsigned int) * batch_size, hipMemcpyDeviceToHost, stream));
    check_cuda_error(hipStreamSynchronize(stream));

    // The same seed produces the same random number.
    for (size_t i = 0; i + period_size - 1 < batch_size; i += period_size) {
        for (size_t j = 1; j < period_size; ++j) {
            EXPECT_TRUE(h_rand_vals[i] == h_rand_vals[i + j])
                << fmtstr("Fail at val[%d]=%d <> val[%d]=%d", i, h_rand_vals[i], i + j, h_rand_vals[i + j]);
        }
    }

    delete h_rand_vals;
    delete h_random_seeds;
    check_cuda_error(hipFree(d_rand_vals));
    check_cuda_error(hipFree(d_random_seeds));
    check_cuda_error(hipFree(curand_states));
    check_cuda_error(hipStreamDestroy(stream));
}

}  // end of namespace
